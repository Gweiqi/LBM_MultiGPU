#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define BLOCKSIZEX 64
#define BLOCKSIZEY 1
#define BLOCKSIZEZ 1
#define BLOCKSIZELRX 64
#define BLOCKSIZELRY 1
#define BLOCKSIZELRZ 1
#define XDIM 128
#define YDIM 128
#define ZDIM 128
#define TMAX 10000

#define LRFACTOR 0.5f
#define LRX0 31.75f	//minimum x coord of LR
#define XLRDIM 128	//number of nodes in x
#define LRY0 31.75f
#define YLRDIM 128
#define LRZ0 31.75f
#define ZLRDIM 128

#define RE 100.f
#define UMAX 0.08f
#define METHOD "HYB" //SINGLE,HYB,TEXT,SHARED
#define REFINEMENT "YES" //YES,NO
#define MODEL "MRT" //BGK,MRT,STREAM
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>

inline __device__ int ImageFcn(float x, float y, float z){
	if(y < 0.1f || z < 0.1f || (XDIM-x) < 0.1f || (YDIM-y) < 0.1f || (ZDIM-z) < 0.1f)
		return 1;
	else if(x < 0.1f)
		return 3;
	else
		return 0;
}

inline __device__ int ImageFcn(int x, int y, int z){
	if(y == 0 || z == 0 || x == XDIM-1 || y == YDIM-1 || z == ZDIM-1)
		return 1;
	else if(x == 0)
		return 3;
	else
		return 0;
}

texture<float,2,hipReadModeElementType> texRef_f0A;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;
texture<float,2,hipReadModeElementType> texRef_f9A;
texture<float,2,hipReadModeElementType> texRef_f10A;
texture<float,2,hipReadModeElementType> texRef_f11A;
texture<float,2,hipReadModeElementType> texRef_f12A;
texture<float,2,hipReadModeElementType> texRef_f13A;
texture<float,2,hipReadModeElementType> texRef_f14A;
texture<float,2,hipReadModeElementType> texRef_f15A;
texture<float,2,hipReadModeElementType> texRef_f16A;
texture<float,2,hipReadModeElementType> texRef_f17A;
texture<float,2,hipReadModeElementType> texRef_f18A;

texture<float,2,hipReadModeElementType> texRef_f0B;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;
texture<float,2,hipReadModeElementType> texRef_f9B;
texture<float,2,hipReadModeElementType> texRef_f10B;
texture<float,2,hipReadModeElementType> texRef_f11B;
texture<float,2,hipReadModeElementType> texRef_f12B;
texture<float,2,hipReadModeElementType> texRef_f13B;
texture<float,2,hipReadModeElementType> texRef_f14B;
texture<float,2,hipReadModeElementType> texRef_f15B;
texture<float,2,hipReadModeElementType> texRef_f16B;
texture<float,2,hipReadModeElementType> texRef_f17B;
texture<float,2,hipReadModeElementType> texRef_f18B;


texture<float,2,hipReadModeElementType> texRef_f0C;
texture<float,2,hipReadModeElementType> texRef_f1C;
texture<float,2,hipReadModeElementType> texRef_f2C;
texture<float,2,hipReadModeElementType> texRef_f3C;
texture<float,2,hipReadModeElementType> texRef_f4C;
texture<float,2,hipReadModeElementType> texRef_f5C;
texture<float,2,hipReadModeElementType> texRef_f6C;
texture<float,2,hipReadModeElementType> texRef_f7C;
texture<float,2,hipReadModeElementType> texRef_f8C;
texture<float,2,hipReadModeElementType> texRef_f9C;
texture<float,2,hipReadModeElementType> texRef_f10C;
texture<float,2,hipReadModeElementType> texRef_f11C;
texture<float,2,hipReadModeElementType> texRef_f12C;
texture<float,2,hipReadModeElementType> texRef_f13C;
texture<float,2,hipReadModeElementType> texRef_f14C;
texture<float,2,hipReadModeElementType> texRef_f15C;
texture<float,2,hipReadModeElementType> texRef_f16C;
texture<float,2,hipReadModeElementType> texRef_f17C;
texture<float,2,hipReadModeElementType> texRef_f18C;

texture<float,2,hipReadModeElementType> texRef_f0D;
texture<float,2,hipReadModeElementType> texRef_f1D;
texture<float,2,hipReadModeElementType> texRef_f2D;
texture<float,2,hipReadModeElementType> texRef_f3D;
texture<float,2,hipReadModeElementType> texRef_f4D;
texture<float,2,hipReadModeElementType> texRef_f5D;
texture<float,2,hipReadModeElementType> texRef_f6D;
texture<float,2,hipReadModeElementType> texRef_f7D;
texture<float,2,hipReadModeElementType> texRef_f8D;
texture<float,2,hipReadModeElementType> texRef_f9D;
texture<float,2,hipReadModeElementType> texRef_f10D;
texture<float,2,hipReadModeElementType> texRef_f11D;
texture<float,2,hipReadModeElementType> texRef_f12D;
texture<float,2,hipReadModeElementType> texRef_f13D;
texture<float,2,hipReadModeElementType> texRef_f14D;
texture<float,2,hipReadModeElementType> texRef_f15D;
texture<float,2,hipReadModeElementType> texRef_f16D;
texture<float,2,hipReadModeElementType> texRef_f17D;
texture<float,2,hipReadModeElementType> texRef_f18D;


int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

	f0 = f0 -omega*(f0 -0.3333333333f*(rho-1.5f*usqr));
	f1 = f1 -omega*(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 = f2 -omega*(f2 -0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 = f3 -omega*(f3 -0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 = f4 -omega*(f4 -0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 = f5 -omega*(f5 -0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 = f6 -omega*(f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 = f7 -omega*(f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 = f8 -omega*(f8 -0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 = f9 -omega*(f9 -0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10= f10-omega*(f10-0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11= f11-omega*(f11-0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12= f12-omega*(f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13= f13-omega*(f13-0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14= f14-omega*(f14-0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15= f15-omega*(f15-0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16= f16-omega*(f16-0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17= f17-omega*(f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18= f18-omega*(f18-0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));


}


inline __device__ void mrt_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
//{
//	float rho,u,v,w;	
//	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
//	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
//	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
//	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
//	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//	u = u*rho;
//	v = v*rho;
//	w = w*rho;
//
//	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//
//	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
//	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+  1.f*f8+ -4.f*f9+    f10+  1.f*f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
//	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
//	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
//	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
//	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
//	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
//	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
//	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
//	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
//	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
//	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
//	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
//	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
//	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;
//
//	m1 -= -11.f*rho+19.f*(u*u+v*v+w*w);
//	m2 -= -7.53968254f*(u*u+v*v+w*w);
//	m4 -= -0.66666667f*u;//qx_eq
//	m6 -= -0.66666667f*v;//qx_eq
//	m8 -= -0.66666667f*w;//qx_eq
//	m9 -= (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
//	m11-= (v*v-w*w);//pww_eq
//	m13-= u*v;//pxy_eq
//	m14-= v*w;//pyz_eq
//	m15-= u*w;//pxz_eq
//
//
//f0  -= - 0.012531328f*(m1)+  0.047619048f*(m2);
//f1  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+   -0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
//f2  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +   -0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f3  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+    0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
//f4  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +    0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f5  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f6  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f7  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f8  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f9  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +   -0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f10 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f11 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//f12 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f13 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//f14 -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +    0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f15 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f16 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//f17 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f18 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//
//f2  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
//f4  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
//f5  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
//f6  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
//f7  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
//f8  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
//f9  -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
//f10 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
//f11 -=                                                                         +(                         0.25f*(m14)                        )*omega ;
//f12 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
//f13 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
//f14 -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
//f15 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
//f16 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
//f17 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
//f18 -=                                                                         +(                         0.25f*(m14)                        )*omega ;
//
//f5  -=  0.125f*(m16)+ -0.125f*(m17);                        
//f6  -= -0.125f*(m16)+ -0.125f*(m17);                        
//f7  -= -0.125f*(m16)+  0.125f*(m17);                        
//f8  -=  0.125f*(m16)+  0.125f*(m17);                        
//f10 -= -0.125f*(m16)                              +  0.125f*(m18);
//f11 -=                             +  0.125f*(m17)+ -0.125f*(m18);
//f12 -=  0.125f*(m16)                              +  0.125f*(m18);
//f13 -=                             + -0.125f*(m17)+ -0.125f*(m18);
//f15 -= -0.125f*(m16)                              + -0.125f*(m18);
//f16 -=                             +  0.125f*(m17)+  0.125f*(m18);
//f17 -=  0.125f*(m16)                              + -0.125f*(m18);
//f18 -=                             + -0.125f*(m17)+  0.125f*(m18);
//}


{
	float u,v,w;	
//	float rho = f1+f2+f4+f6+f8+f9+f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//	u = u*rho;
//	v = v*rho;
//	w = w*rho;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;


	//COMPUTE M-MEQ
	m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
	m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m9  =            2.f*f1+   -  f2+  2.f*f3+   -  f4+      f5+      f6+      f7+      f8+ -    f9+    f10+   -2.f*f11+      f12+ -2.f*f13+ -    f14+      f15+ -2.f*f16+      f17+ -2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+      f5+      f6+      f7+      f8+  2.f*f9+    f10+   -2.f*f11+      f12+ -2.f*f13+  2.f*f14+      f15+ -2.f*f16+      f17+ -2.f*f18;
	m11 =                         f2         +      f4+      f5+      f6+      f7+      f8+ -    f9+ -  f10            + -    f12          + -    f14+ -    f15          + -    f17          -(v*v-w*w);
	m12 =                    -2.f*f2           -2.f*f4+      f5+      f6+      f7+      f8+  2.f*f9+ -  f10            + -    f12          +  2.f*f14+ -    f15          + -    f17          ;
	m13 =                                                    f5+   -  f6+      f7+ -    f8                                                                                                     -u*v;
	m14 =                                                                                                           f11          + -    f13                    + -    f16          +      f18  -v*w;
	m15 =                                                                                               f10          + -      f12                    + -    f15          +      f17            -u*w;  
	m16 =                                                    f5+   -  f6+ -    f7+      f8           -  f10          +        f12                    + -    f15          +      f17          ;  
	m17 =                                                 -  f5+   -  f6+      f7+      f8                   +      f11          + -    f13                    +      f16          + -    f18;  
	m18 =                                                                                               f10+ -      f11+      f12+ -    f13          + -    f15+      f16+ -    f17+      f18;


f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
}

inline __device__ int f_mem(int f_num, int x, int y, int z, size_t pitch)
{

	return (x+y*pitch+z*YDIM*pitch)+f_num*pitch*YDIM*ZDIM;
}

inline __device__ int f_memLR(int f_num, int x, int y, int z, size_t pitch)
{

	return (x+y*pitch+z*YLRDIM*pitch)+f_num*pitch*YLRDIM*ZLRDIM;
}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}

__global__ void simple_copy(float* fA, float* fB,
							int *image, float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)

	fB[j] = fA[j];//+0.01f;
}


__global__ void ExtractFromC_d(float* fout,
							size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//	if(x < LRX0+1 || x > LRX0+XLRDIM-2 || y < LRY0+1 || y > LRY0+YLRDIM-2 || z < LRZ0+1 || z > LRZ0+ZLRDIM-2)
//	//if(x < LRX0+2 || x > LRX0+XLRDIM-3 || y < LRY0+2 || y > LRY0+YLRDIM-3 || z < LRZ0+2 || z > LRZ0+ZLRDIM-3)
//	{
//	//do nothing
//	}
//	else{
	if( (x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1 && z > LRZ0+1 && z < LRZ0+ZLRDIM*LRFACTOR-1) && 
	(x == int(LRX0+2) || x == int(LRX0+XLRDIM*LRFACTOR-1) || y == int(LRY0+2) || y == int(LRY0+YLRDIM*LRFACTOR-1) || z == int(LRZ0+2) || z == int(LRY0+ZLRDIM*LRFACTOR-1)) )
	{
//	if(x > 10 && y > 10 && z > 10 && x < 20 && y < 20 && z < 20)
//	{
	float xcoord = 2.0f*(x-LRX0)+0.5f;
	float ycoord = 2.0f*(y-LRY0)+0.5f;
	float zcoord = 2.0f*(z-LRZ0);
	int zminus = int(zcoord);
	int zplus = zminus+1;
	f0 = (zplus-zcoord)*tex2D(texRef_f0C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0C ,xcoord,ycoord+YLRDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2C ,xcoord,ycoord+YLRDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4C ,xcoord,ycoord+YLRDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9C ,xcoord,ycoord+YLRDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11C,xcoord,ycoord+YLRDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13C,xcoord,ycoord+YLRDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14C,xcoord,ycoord+YLRDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16C,xcoord,ycoord+YLRDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18C,xcoord,ycoord+YLRDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1C ,xcoord,ycoord+YLRDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3C ,xcoord,ycoord+YLRDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5C ,xcoord,ycoord+YLRDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6C ,xcoord,ycoord+YLRDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7C ,xcoord,ycoord+YLRDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8C ,xcoord,ycoord+YLRDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15C,xcoord,ycoord+YLRDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17C,xcoord,ycoord+YLRDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10C,xcoord,ycoord+YLRDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12C,xcoord,ycoord+YLRDIM*(zplus));

	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = fma(u,u,fma(v,v,w*w));

//	float m1 -= -11.f*rho+19.f*(u*u+v*v+w*w);
//	float m4 = -0.66666667f*u;//qx_eq
//	float m6 = -0.66666667f*v;//qx_eq
//	float m8 = -0.66666667f*w;//qx_eq
//	float m9 -= (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
//	float m11-= (v*v-w*w);//pww_eq
//	float m13-= u*v;//pxy_eq
//	float m14-= v*w;//pyz_eq
//	float m15-= u*w;//pxz_eq

	if(MODEL == "MRT")
	{
	float meq0 = rho;
	float meq1 = -11.f*rho+19.f*(u*u+v*v+w*w);
	float meq2 = -7.53968254f*(u*u+v*v+w*w);
	float meq3 = u;
	float meq4 = -0.66666667f*u;//qx_eq
	float meq5 = v;
	float meq6 = -0.66666667f*v;//qx_eq
	float meq7 = w;
	float meq8 = -0.66666667f*w;//qx_eq
	float meq9 = (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
	float meq11= (v*v-w*w);//pww_eq
	float meq13= u*v;//pxy_eq
	float meq14= v*w;//pyz_eq
	float meq15= u*w;//pxz_eq

float feq0 = 0.052631579f*meq0+  -0.012531328f*meq1+  0.047619048f*meq2                                                                                    ;
float feq1 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+  0.1f*meq3+   -0.1f*meq4                                                        ;
float feq2 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            +  0.1f*meq5+   -0.1f*meq6                            ;
float feq3 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+ -0.1f*meq3+    0.1f*meq4                                                        ;
float feq4 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            + -0.1f*meq5+    0.1f*meq6                            ;
float feq5 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*meq3+  0.025f*meq4+  0.1f*meq5+  0.025f*meq6                            ;
float feq6 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*meq3+ -0.025f*meq4+  0.1f*meq5+  0.025f*meq6                            ;
float feq7 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*meq3+ -0.025f*meq4+ -0.1f*meq5+ -0.025f*meq6                            ;
float feq8 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*meq3+  0.025f*meq4+ -0.1f*meq5+ -0.025f*meq6                            ;
float feq9 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                        +  0.1f*meq7+   -0.1f*meq8;
float feq10= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*meq3+  0.025f*meq4                            +  0.1f*meq7+  0.025f*meq8;
float feq11= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*meq5+  0.025f*meq6+  0.1f*meq7+  0.025f*meq8;
float feq12= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*meq3+ -0.025f*meq4                            +  0.1f*meq7+  0.025f*meq8;
float feq13= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            + -0.1f*meq5+ -0.025f*meq6+  0.1f*meq7+  0.025f*meq8;
float feq14= 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                        + -0.1f*meq7+    0.1f*meq8;
float feq15= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*meq3+  0.025f*meq4                            + -0.1f*meq7+ -0.025f*meq8;
float feq16= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*meq5+  0.025f*meq6+ -0.1f*meq7+ -0.025f*meq8;
float feq17= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*meq3+ -0.025f*meq4                            + -0.1f*meq7+ -0.025f*meq8;
float feq18= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+                           + -0.1f*meq5+ -0.025f*meq6+ -0.1f*meq7+ -0.025f*meq8;

feq1 +=  0.055555556f*meq9;
feq2 += -0.027777778f*meq9+  0.083333333f*meq11;
feq3 +=  0.055555556f*meq9;
feq4 += -0.027777778f*meq9+  0.083333333f*meq11;
feq5 +=  0.027777778f*meq9+  0.083333333f*meq11+  0.25f*meq13                              ;
feq6 +=  0.027777778f*meq9+  0.083333333f*meq11+ -0.25f*meq13                              ;
feq7 +=  0.027777778f*meq9+  0.083333333f*meq11+  0.25f*meq13                              ;
feq8 +=  0.027777778f*meq9+  0.083333333f*meq11+ -0.25f*meq13                              ;
feq9 += -0.027777778f*meq9+ -0.083333333f*meq11                                            ;
feq10+=  0.027777778f*meq9+ -0.083333333f*meq11                              +  0.25f*meq15;
feq11+= -0.055555556f*meq9                                      +  0.25f*meq14             ;
feq12+=  0.027777778f*meq9+ -0.083333333f*meq11                              + -0.25f*meq15;
feq13+= -0.055555556f*meq9                                        -0.25f*meq14             ;
feq14+= -0.027777778f*meq9+ -0.083333333f*meq11                                            ;
feq15+=  0.027777778f*meq9+ -0.083333333f*meq11                              + -0.25f*meq15;
feq16+= -0.055555556f*meq9                                      + -0.25f*meq14             ;
feq17+=  0.027777778f*meq9+ -0.083333333f*meq11                              +  0.25f*meq15;
feq18+= -0.055555556f*meq9                                                           +  0.25f*meq14             ;


f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;






//	float m2 = -7.53968254f*(u*u+v*v+w*w);
//	//scale
//	f0 =SF*f0 +(1.0f-SF)*(0.052631579f*rho+ -0.012531328f*(-11.f*rho+19.f*usqr)+ 0.047619048f*m2);
//	f1 =SF*f1 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//	f2 =SF*f2 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//	f3 =SF*f3 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//	f4 =SF*f4 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//	f5 =SF*f5 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+v)+0.01666666667f*(-u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//	f6 =SF*f6 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+v)+0.01666666667f*( u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//	f7 =SF*f7 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-v)+0.01666666667f*( u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//	f8 =SF*f8 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-v)+0.01666666667f*(-u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//	f9 =SF*f9 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*w)                -0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
//	f10=SF*f10+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+w)+0.01666666667f*(-u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f11=SF*f11+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//	f12=SF*f12+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+w)+0.01666666667f*( u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      -0.25f*u*v);
//	f13=SF*f13+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         -0.25f*v*w      );
//	f14=SF*f14+(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*w)                +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
//	f15=SF*f15+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-w)+0.01666666667f*(-u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f16=SF*f16+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//	f17=SF*f17+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-w)+0.01666666667f*( u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f18=SF*f18+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	}
	else
	{
//	//scale
//f0 =SF*f0 +(1.0f-SF)*(0.052631579f*rho+ -0.012531328f*(-11.f*rho+19.f*usqr)+ 0.047619048f*m2);
//f1 =SF*f1 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +0.1f*(u+0.666666667f*u)                  +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//f2 =SF*f2 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +0.1f*(v+0.666666667f*v)                  -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//f3 =SF*f3 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +0.1f*(-0.666666667f*u-u)                  +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//f4 =SF*f4 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +0.1f*(-0.666666667f*v-v)                  -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//f5 =SF*f5 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+v)+0.025f*( -0.666666667f*u+-0.666666667f*v)  +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//f6 =SF*f6 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+v)+0.025f*(+0.666666667f*u+-0.666666667f*v)  +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//f7 =SF*f7 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-v)+0.025f*(+0.666666667f*u+0.666666667f*v)  +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//f8 =SF*f8 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-v)+0.025f*( -0.666666667f*u+0.666666667f*v)  +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//f9 =SF*f9 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +0.1f*(w+0.666666667f*w)                  -0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
//f10=SF*f10+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+w)+0.025f*( -0.666666667f*u+-0.666666667f*w)  +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//f11=SF*f11+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v+w)+0.025f*( -0.666666667f*v+-0.666666667f*w)  -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//f12=SF*f12+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+w)+0.025f*(+0.666666667f*u+-0.666666667f*w)  +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      -0.25f*u*v);
//f13=SF*f13+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v+w)+0.025f*( -0.666666667f*v+-0.666666667f*w)  -0.055555556f*(2.f*u*u-(v*v+w*w))                         -0.25f*v*w      );
//f14=SF*f14+(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +0.1f*(-0.666666667f*w-w)                  +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//f15=SF*f15+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-w)+0.025f*( -0.666666667f*u+0.666666667f*w)  +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//f16=SF*f16+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v-w)+0.025f*( -0.666666667f*v+0.666666667f*w)  -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//f17=SF*f17+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-w)+0.025f*(+0.666666667f*u+0.666666667f*w)  +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//f18=SF*f18+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v-w)+0.025f*( -0.666666667f*v+0.666666667f*w)  -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.3333333333f*(rho-1.5f*usqr));
	f1 =SF*f1 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 =SF*f2 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 =SF*f6 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 =SF*f9 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10=SF*f10+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11=SF*f11+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12=SF*f12+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13=SF*f13+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14=SF*f14+(1.0f-SF)*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15=SF*f15+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16=SF*f16+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17=SF*f17+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18=SF*f18+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));
	}

	fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
	fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
	fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
	fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
	fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
	fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
	fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
	fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
	fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
	fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
	fout[f_mem(10,x,y,z,pitch)] = f10;
	fout[f_mem(11,x,y,z,pitch)] = f11;
	fout[f_mem(12,x,y,z,pitch)] = f12;
	fout[f_mem(13,x,y,z,pitch)] = f13;
	fout[f_mem(14,x,y,z,pitch)] = f14;
	fout[f_mem(15,x,y,z,pitch)] = f15;
	fout[f_mem(16,x,y,z,pitch)] = f16;
	fout[f_mem(17,x,y,z,pitch)] = f17;
	fout[f_mem(18,x,y,z,pitch)] = f18;

	}
}

__global__ void LR_d_hybABCD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
	}
	f1 = tex2D(texRef_f1C ,x-1+0.5f,y  +0.5f+YDIM*(z));
	f3 = tex2D(texRef_f3C ,x+1+0.5f,y  +0.5f+YDIM*(z));
	f5 = tex2D(texRef_f5C ,x-1+0.5f,y-1+0.5f+YDIM*(z));
	f6 = tex2D(texRef_f6C ,x+1+0.5f,y-1+0.5f+YDIM*(z));
	f7 = tex2D(texRef_f7C ,x+1+0.5f,y+1+0.5f+YDIM*(z));
	f8 = tex2D(texRef_f8C ,x-1+0.5f,y+1+0.5f+YDIM*(z));
	f15= tex2D(texRef_f15C,x-1+0.5f,y  +0.5f+YDIM*(z+1));
	f17= tex2D(texRef_f17C,x+1+0.5f,y  +0.5f+YDIM*(z+1));
	f10= tex2D(texRef_f10C,x-1+0.5f,y  +0.5f+YDIM*(z-1));
	f12= tex2D(texRef_f12C,x+1+0.5f,y  +0.5f+YDIM*(z-1));

	if(im == 1){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_hybABDC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;//dont need to +0.5f because z is not using texture interpolation
	int zminus = int(zcoord);
	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3 || z < 2 || z > ZLRDIM-3)
	{
	//interpolate for next time step. from B //YDIM and not YLRDIM
	f0 = (zplus-zcoord)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));

	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	u = rho*u;
	v = rho*v;
	w = rho*w;
	float usqr = fma(u,u,fma(v,v,w*w));

	if(MODEL == "MRT")
	{
	float meq0 = rho;
	float meq1 = -11.f*rho+19.f*(u*u+v*v+w*w);
	float meq2 = -7.53968254f*(u*u+v*v+w*w);
	float meq3 = u;
	float meq4 = -0.66666667f*u;//qx_eq
	float meq5 = v;
	float meq6 = -0.66666667f*v;//qx_eq
	float meq7 = w;
	float meq8 = -0.66666667f*w;//qx_eq
	float meq9 = (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
	float meq11= (v*v-w*w);//pww_eq
	float meq13= u*v;//pxy_eq
	float meq14= v*w;//pyz_eq
	float meq15= u*w;//pxz_eq

//float feq0 = 0.052631579f*meq0+  -0.012531328f*meq1+  0.047619048f*meq2                                                                      ;
//float feq1 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+  (0.166666667f*u)                                                   ;
//float feq2 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                       +  (0.166666667f*v)                       ;
//float feq3 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+ -(0.166666667f*u)                                                   ;
//float feq4 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            + -(0.166666667f*v)                       ;
//float feq5 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3+meq5)+  0.025f*(meq4+meq6)                            ;
//float feq6 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3-meq5)+ -0.025f*(meq4-meq6)                            ;
//float feq7 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3+meq5)+ -0.025f*(meq4+meq6)                            ;
//float feq8 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3-meq5)+  0.025f*(meq4-meq6)                            ;
//float feq9 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      +  (0.166666667f*w);
//float feq10= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3+meq7)+  0.025f*(meq4+meq8)                                 ;
//float feq11= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*(meq5+meq7)+  0.025f*(meq6+meq8);
//float feq12= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3-meq7)+ -0.025f*(meq4-meq8)                            ;
//float feq13= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            + -0.1f*(meq5-meq7)+ -0.025f*(meq6-meq8);
//float feq14= 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      + -(0.166666667f*w);
//float feq15= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3-meq7)+  0.025f*(meq4-meq8)                            ;
//float feq16= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*(meq5-meq7)+  0.025f*(meq6-meq8);
//float feq17= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3+meq7)+ -0.025f*(meq4+meq8)                            ;
//float feq18= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+                           + -0.1f*(meq5+meq7)+ -0.025f*(meq6+meq8);

float feq0 = 0.052631579f*meq0+  -0.012531328f*meq1+  0.047619048f*meq2                                                                      ;
float feq1 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+  0.1f*(meq3-meq4)                                                   ;
float feq2 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            +  0.1f*(meq5-meq6)                       ;
float feq3 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+ -0.1f*(meq3-meq4)                                                   ;
float feq4 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            + -0.1f*(meq5-meq6)                       ;
float feq5 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3+meq5)+  0.025f*(meq4+meq6)                            ;
float feq6 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3-meq5)+ -0.025f*(meq4-meq6)                            ;
float feq7 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3+meq5)+ -0.025f*(meq4+meq6)                            ;
float feq8 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3-meq5)+  0.025f*(meq4-meq6)                            ;
float feq9 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      +  0.1f*(meq7-meq8);
float feq10= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3+meq7)+  0.025f*(meq4+meq8)                                 ;
float feq11= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*(meq5+meq7)+  0.025f*(meq6+meq8);
float feq12= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3-meq7)+ -0.025f*(meq4-meq8)                            ;
float feq13= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            + -0.1f*(meq5-meq7)+ -0.025f*(meq6-meq8);
float feq14= 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      + -0.1f*(meq7-meq8);
float feq15= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3-meq7)+  0.025f*(meq4-meq8)                            ;
float feq16= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*(meq5-meq7)+  0.025f*(meq6-meq8);
float feq17= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3+meq7)+ -0.025f*(meq4+meq8)                            ;
float feq18= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+                           + -0.1f*(meq5+meq7)+ -0.025f*(meq6+meq8);

feq1 +=  0.055555556f*meq9;
feq2 += -0.027777778f*meq9+  0.083333333f*meq11;
feq3 +=  0.055555556f*meq9;
feq4 += -0.027777778f*meq9+  0.083333333f*meq11;
feq5 +=  0.027777778f*meq9+  0.083333333f*meq11+  0.25f*meq13                              ;
feq6 +=  0.027777778f*meq9+  0.083333333f*meq11+ -0.25f*meq13                              ;
feq7 +=  0.027777778f*meq9+  0.083333333f*meq11+  0.25f*meq13                              ;
feq8 +=  0.027777778f*meq9+  0.083333333f*meq11+ -0.25f*meq13                              ;
feq9 += -0.027777778f*meq9+ -0.083333333f*meq11                                            ;
feq10+=  0.027777778f*meq9+ -0.083333333f*meq11                              +  0.25f*meq15;
feq11+= -0.055555556f*meq9                                      +  0.25f*meq14             ;
feq12+=  0.027777778f*meq9+ -0.083333333f*meq11                              + -0.25f*meq15;
feq13+= -0.055555556f*meq9                                        -0.25f*meq14             ;
feq14+= -0.027777778f*meq9+ -0.083333333f*meq11                                            ;
feq15+=  0.027777778f*meq9+ -0.083333333f*meq11                              + -0.25f*meq15;
feq16+= -0.055555556f*meq9                                      + -0.25f*meq14             ;
feq17+=  0.027777778f*meq9+ -0.083333333f*meq11                              +  0.25f*meq15;
feq18+= -0.055555556f*meq9                                      +  0.25f*meq14             ;


f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;






//	float m2 = -7.53968254f*(u*u+v*v+w*w);
//	//scale
//	f0 =SF*f0 +(1.0f-SF)*(0.052631579f*rho+ -0.012531328f*(-11.f*rho+19.f*usqr)+ 0.047619048f*m2);
//	f1 =SF*f1 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//	f2 =SF*f2 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//	f3 =SF*f3 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//	f4 =SF*f4 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//	f5 =SF*f5 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+v)+0.01666666667f*(-u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//	f6 =SF*f6 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+v)+0.01666666667f*( u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//	f7 =SF*f7 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-v)+0.01666666667f*( u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//	f8 =SF*f8 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-v)+0.01666666667f*(-u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//	f9 =SF*f9 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*w)                -0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
//	f10=SF*f10+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+w)+0.01666666667f*(-u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f11=SF*f11+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//	f12=SF*f12+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+w)+0.01666666667f*( u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      -0.25f*u*v);
//	f13=SF*f13+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         -0.25f*v*w      );
//	f14=SF*f14+(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*w)                +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
//	f15=SF*f15+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-w)+0.01666666667f*(-u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f16=SF*f16+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//	f17=SF*f17+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-w)+0.01666666667f*( u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f18=SF*f18+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	}
	else
	{
	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.3333333333f*(rho-1.5f*usqr));
	f1 =SF*f1 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 =SF*f2 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 =SF*f6 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 =SF*f9 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10=SF*f10+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11=SF*f11+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12=SF*f12+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13=SF*f13+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14=SF*f14+(1.0f-SF)*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15=SF*f15+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16=SF*f16+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17=SF*f17+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18=SF*f18+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));
	}

	fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
	fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
	fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
	fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
	fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
	fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
	fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
	fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
	fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
	fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
	fout[f_memLR(10,x,y,z,pitch)] = f10;
	fout[f_memLR(11,x,y,z,pitch)] = f11;
	fout[f_memLR(12,x,y,z,pitch)] = f12;
	fout[f_memLR(13,x,y,z,pitch)] = f13;
	fout[f_memLR(14,x,y,z,pitch)] = f14;
	fout[f_memLR(15,x,y,z,pitch)] = f15;
	fout[f_memLR(16,x,y,z,pitch)] = f16;
	fout[f_memLR(17,x,y,z,pitch)] = f17;
	fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
	}
	f1 = tex2D(texRef_f1D ,x-1+0.5f,y  +0.5f+YDIM*(z));
	f3 = tex2D(texRef_f3D ,x+1+0.5f,y  +0.5f+YDIM*(z));
	f5 = tex2D(texRef_f5D ,x-1+0.5f,y-1+0.5f+YDIM*(z));
	f6 = tex2D(texRef_f6D ,x+1+0.5f,y-1+0.5f+YDIM*(z));
	f7 = tex2D(texRef_f7D ,x+1+0.5f,y+1+0.5f+YDIM*(z));
	f8 = tex2D(texRef_f8D ,x-1+0.5f,y+1+0.5f+YDIM*(z));
	f15= tex2D(texRef_f15D,x-1+0.5f,y  +0.5f+YDIM*(z+1));
	f17= tex2D(texRef_f17D,x+1+0.5f,y  +0.5f+YDIM*(z+1));
	f10= tex2D(texRef_f10D,x-1+0.5f,y  +0.5f+YDIM*(z-1));
	f12= tex2D(texRef_f12D,x+1+0.5f,y  +0.5f+YDIM*(z-1));

	//else f18 = 0.1f;
	if(im == 1){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}
__global__ void LR_d_hybBADC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;//dont need to +0.5f because z is not using texture interpolation
	int zminus = int(zcoord);
	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3 || z < 2 || z > ZLRDIM-3)
	{
	//interpolate for next time step. from B
	f0 = (zplus-zcoord)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));

	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	u = rho*u;
	v = rho*v;
	w = rho*w;
	float usqr = fma(u,u,fma(v,v,w*w));

	if(MODEL == "MRT")
	{
	float meq0 = rho;
	float meq1 = -11.f*rho+19.f*(u*u+v*v+w*w);
	float meq2 = -7.53968254f*(u*u+v*v+w*w);
	float meq3 = u;
	float meq4 = -0.66666667f*u;//qx_eq
	float meq5 = v;
	float meq6 = -0.66666667f*v;//qx_eq
	float meq7 = w;
	float meq8 = -0.66666667f*w;//qx_eq
	float meq9 = (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
	float meq11= (v*v-w*w);//pww_eq
	float meq13= u*v;//pxy_eq
	float meq14= v*w;//pyz_eq
	float meq15= u*w;//pxz_eq

//float feq0 = 0.052631579f*meq0+  -0.012531328f*meq1+  0.047619048f*meq2                                                                      ;
//float feq1 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+  (0.166666667f*u)                                                   ;
//float feq2 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                       +  (0.166666667f*v)                       ;
//float feq3 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+ -(0.166666667f*u)                                                   ;
//float feq4 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            + -(0.166666667f*v)                       ;
//float feq5 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3+meq5)+  0.025f*(meq4+meq6)                            ;
//float feq6 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3-meq5)+ -0.025f*(meq4-meq6)                            ;
//float feq7 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3+meq5)+ -0.025f*(meq4+meq6)                            ;
//float feq8 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3-meq5)+  0.025f*(meq4-meq6)                            ;
//float feq9 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      +  (0.166666667f*w);
//float feq10= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3+meq7)+  0.025f*(meq4+meq8)                                 ;
//float feq11= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*(meq5+meq7)+  0.025f*(meq6+meq8);
//float feq12= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3-meq7)+ -0.025f*(meq4-meq8)                            ;
//float feq13= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            + -0.1f*(meq5-meq7)+ -0.025f*(meq6-meq8);
//float feq14= 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      + -(0.166666667f*w);
//float feq15= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3-meq7)+  0.025f*(meq4-meq8)                            ;
//float feq16= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*(meq5-meq7)+  0.025f*(meq6-meq8);
//float feq17= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3+meq7)+ -0.025f*(meq4+meq8)                            ;
//float feq18= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+                           + -0.1f*(meq5+meq7)+ -0.025f*(meq6+meq8);

float feq0 = 0.052631579f*meq0+  -0.012531328f*meq1+  0.047619048f*meq2                                                                                    ;
float feq1 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+  0.1f*(meq3-meq4)                                                           ;
float feq2 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            +  0.1f*(meq5-meq6)                            ;
float feq3 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2+ -0.1f*(meq3-meq4)                                                        ;
float feq4 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                            + -0.1f*(meq5-meq6)                            ;
float feq5 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3+meq5)+  0.025f*(meq4+meq6)                            ;
float feq6 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3-meq5)+ -0.025f*(meq4-meq6)                            ;
float feq7 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*(meq3+meq5)+ -0.025f*(meq4+meq6)                            ;
float feq8 = 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*(meq3-meq5)+  0.025f*(meq4-meq6)                            ;
float feq9 = 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      +  0.1f*meq7+   -0.1f*meq8;
float feq10= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*meq3+  0.025f*meq4                            +  0.1f*meq7+  0.025f*meq8;
float feq11= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*meq5+  0.025f*meq6+  0.1f*meq7+  0.025f*meq8;
float feq12= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*meq3+ -0.025f*meq4                            +  0.1f*meq7+  0.025f*meq8;
float feq13= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            + -0.1f*meq5+ -0.025f*meq6+  0.1f*meq7+  0.025f*meq8;
float feq14= 0.052631579f*meq0+ -0.0045948204f*meq1+ -0.015873016f*meq2                                                      + -0.1f*meq7+    0.1f*meq8;
float feq15= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+  0.1f*meq3+  0.025f*meq4                            + -0.1f*meq7+ -0.025f*meq8;
float feq16= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2                            +  0.1f*meq5+  0.025f*meq6+ -0.1f*meq7+ -0.025f*meq8;
float feq17= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+ -0.1f*meq3+ -0.025f*meq4                            + -0.1f*meq7+ -0.025f*meq8;
float feq18= 0.052631579f*meq0+  0.0033416876f*meq1+  0.003968254f*meq2+                           + -0.1f*meq5+ -0.025f*meq6+ -0.1f*meq7+ -0.025f*meq8;

feq1 +=  0.055555556f*meq9;
feq2 += -0.027777778f*meq9+  0.083333333f*meq11;
feq3 +=  0.055555556f*meq9;
feq4 += -0.027777778f*meq9+  0.083333333f*meq11;
feq5 +=  0.027777778f*meq9+  0.083333333f*meq11+  0.25f*meq13                              ;
feq6 +=  0.027777778f*meq9+  0.083333333f*meq11+ -0.25f*meq13                              ;
feq7 +=  0.027777778f*meq9+  0.083333333f*meq11+  0.25f*meq13                              ;
feq8 +=  0.027777778f*meq9+  0.083333333f*meq11+ -0.25f*meq13                              ;
feq9 += -0.027777778f*meq9+ -0.083333333f*meq11                                            ;
feq10+=  0.027777778f*meq9+ -0.083333333f*meq11                              +  0.25f*meq15;
feq11+= -0.055555556f*meq9                                      +  0.25f*meq14             ;
feq12+=  0.027777778f*meq9+ -0.083333333f*meq11                              + -0.25f*meq15;
feq13+= -0.055555556f*meq9                                        -0.25f*meq14             ;
feq14+= -0.027777778f*meq9+ -0.083333333f*meq11                                            ;
feq15+=  0.027777778f*meq9+ -0.083333333f*meq11                              + -0.25f*meq15;
feq16+= -0.055555556f*meq9                                      + -0.25f*meq14             ;
feq17+=  0.027777778f*meq9+ -0.083333333f*meq11                              +  0.25f*meq15;
feq18+= -0.055555556f*meq9                                      +  0.25f*meq14             ;


f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;






//	float m2 = -7.53968254f*(u*u+v*v+w*w);
//	//scale
//	f0 =SF*f0 +(1.0f-SF)*(0.052631579f*rho+ -0.012531328f*(-11.f*rho+19.f*usqr)+ 0.047619048f*m2);
//	f1 =SF*f1 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//	f2 =SF*f2 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//	f3 =SF*f3 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
//	f4 =SF*f4 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
//	f5 =SF*f5 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+v)+0.01666666667f*(-u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//	f6 =SF*f6 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+v)+0.01666666667f*( u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//	f7 =SF*f7 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-v)+0.01666666667f*( u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
//	f8 =SF*f8 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-v)+0.01666666667f*(-u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
//	f9 =SF*f9 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*w)                -0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
//	f10=SF*f10+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+w)+0.01666666667f*(-u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f11=SF*f11+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//	f12=SF*f12+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+w)+0.01666666667f*( u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      -0.25f*u*v);
//	f13=SF*f13+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         -0.25f*v*w      );
//	f14=SF*f14+(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*w)                +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
//	f15=SF*f15+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-w)+0.01666666667f*(-u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f16=SF*f16+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
//	f17=SF*f17+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-w)+0.01666666667f*( u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
//	f18=SF*f18+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	}
	else
	{
	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.3333333333f*(rho-1.5f*usqr));
	f1 =SF*f1 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 =SF*f2 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 =SF*f6 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 =SF*f9 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10=SF*f10+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11=SF*f11+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12=SF*f12+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13=SF*f13+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14=SF*f14+(1.0f-SF)*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15=SF*f15+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16=SF*f16+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17=SF*f17+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18=SF*f18+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));
	}

	fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
	fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
	fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
	fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
	fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
	fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
	fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
	fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
	fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
	fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
	fout[f_memLR(10,x,y,z,pitch)] = f10;
	fout[f_memLR(11,x,y,z,pitch)] = f11;
	fout[f_memLR(12,x,y,z,pitch)] = f12;
	fout[f_memLR(13,x,y,z,pitch)] = f13;
	fout[f_memLR(14,x,y,z,pitch)] = f14;
	fout[f_memLR(15,x,y,z,pitch)] = f15;
	fout[f_memLR(16,x,y,z,pitch)] = f16;
	fout[f_memLR(17,x,y,z,pitch)] = f17;
	fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
	}
	f1 = tex2D(texRef_f1D ,x-1+0.5f,y  +0.5f+YDIM*(z));
	f3 = tex2D(texRef_f3D ,x+1+0.5f,y  +0.5f+YDIM*(z));
	f5 = tex2D(texRef_f5D ,x-1+0.5f,y-1+0.5f+YDIM*(z));
	f6 = tex2D(texRef_f6D ,x+1+0.5f,y-1+0.5f+YDIM*(z));
	f7 = tex2D(texRef_f7D ,x+1+0.5f,y+1+0.5f+YDIM*(z));
	f8 = tex2D(texRef_f8D ,x-1+0.5f,y+1+0.5f+YDIM*(z));
	f15= tex2D(texRef_f15D,x-1+0.5f,y  +0.5f+YDIM*(z+1));
	f17= tex2D(texRef_f17D,x+1+0.5f,y  +0.5f+YDIM*(z+1));
	f10= tex2D(texRef_f10D,x-1+0.5f,y  +0.5f+YDIM*(z-1));
	f12= tex2D(texRef_f12D,x+1+0.5f,y  +0.5f+YDIM*(z-1));

	//else f18 = 0.1f;
	if(im == 1){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABCD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
//	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
//	im = -1;//not valid for extraction
//	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
//	{
//	im = -2;//not valid for second TS
//	}

	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,dmin(y+1,YLRDIM),dmin(z+1,ZLRDIM),pitch)];
	if(im == 1){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;//dont need to +0.5f because z is not using texture interpolation
	int zminus = int(zcoord);
	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
//	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
//	im = -1;//not valid for extraction
//	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
//	{
//	im = -2;//not valid for second TS
//	}

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3 || z < 2 || z > ZLRDIM-3)
	{
	//interpolate for next time step. from B
	f0 = (zplus-zcoord)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));

	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = fma(u,u,fma(v,v,w*w));

	if(MODEL == "MRT")
	{
	float m2 = -7.53968254f*(u*u+v*v+w*w);
	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.052631579f*rho+ -0.012531328f*(-11.f*rho+19.f*usqr)+ 0.047619048f*m2);
	f1 =SF*f1 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
	f2 =SF*f2 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
	f3 =SF*f3 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
	f4 =SF*f4 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
	f5 =SF*f5 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+v)+0.01666666667f*(-u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
	f6 =SF*f6 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+v)+0.01666666667f*( u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
	f7 =SF*f7 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-v)+0.01666666667f*( u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
	f8 =SF*f8 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-v)+0.01666666667f*(-u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
	f9 =SF*f9 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*w)                -0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
	f10=SF*f10+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+w)+0.01666666667f*(-u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
	f11=SF*f11+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	f12=SF*f12+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+w)+0.01666666667f*( u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      -0.25f*u*v);
	f13=SF*f13+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         -0.25f*v*w      );
	f14=SF*f14+(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*w)                +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
	f15=SF*f15+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-w)+0.01666666667f*(-u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
	f16=SF*f16+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	f17=SF*f17+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-w)+0.01666666667f*( u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
	f18=SF*f18+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	}
	else
	{

	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.3333333333f*(rho-1.5f*usqr));
	f1 =SF*f1 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 =SF*f2 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 =SF*f6 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 =SF*f9 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10=SF*f10+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11=SF*f11+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12=SF*f12+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13=SF*f13+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14=SF*f14+(1.0f-SF)*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15=SF*f15+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16=SF*f16+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17=SF*f17+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18=SF*f18+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));
	}

	fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
	fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
	fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
	fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
	fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
	fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
	fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
	fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
	fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
	fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
	fout[f_memLR(10,x,y,z,pitch)] = f10;
	fout[f_memLR(11,x,y,z,pitch)] = f11;
	fout[f_memLR(12,x,y,z,pitch)] = f12;
	fout[f_memLR(13,x,y,z,pitch)] = f13;
	fout[f_memLR(14,x,y,z,pitch)] = f14;
	fout[f_memLR(15,x,y,z,pitch)] = f15;
	fout[f_memLR(16,x,y,z,pitch)] = f16;
	fout[f_memLR(17,x,y,z,pitch)] = f17;
	fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	//if(z != ZLRDIM-1){
	f18= fin[f_memLR(18,x  ,dmin(y+1,YLRDIM),dmin(z+1,ZLRDIM),pitch)];
	//}
	//else f18 = 0.1f;
	if(im == 1){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BACD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
//	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
//	im = -1;//not valid for extraction
//	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
//	{
//	im = -2;//not valid for second TS
//	}

	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,dmin(y+1,YLRDIM),dmin(z+1,ZLRDIM),pitch)];
	if(im == 1){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BADC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int zminus = int(zcoord);
	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
//	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
//	im = -1;//not valid for extraction
//	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
//	{
//	im = -2;//not valid for second TS
//	}

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3 || z < 2 || z > ZLRDIM-3)
	{
	//interpolate for next time step. from A
	f0 = (zplus-zcoord)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));

	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = fma(u,u,fma(v,v,w*w));

	if(MODEL == "MRT")
	{
	float m2 = -7.53968254f*(u*u+v*v+w*w);
	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.052631579f*rho+ -0.012531328f*(-11.f*rho+19.f*usqr)+ 0.047619048f*m2);
	f1 =SF*f1 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
	f2 =SF*f2 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
	f3 =SF*f3 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*u)                +0.055555556f*(2.f*u*u-(v*v+w*w))                          );
	f4 =SF*f4 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*v)                -0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
	f5 =SF*f5 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+v)+0.01666666667f*(-u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
	f6 =SF*f6 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+v)+0.01666666667f*( u-v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
	f7 =SF*f7 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-v)+0.01666666667f*( u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)+0.25f*(u*v)    );
	f8 =SF*f8 +(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-v)+0.01666666667f*(-u+v) +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)-0.25f*(u*v)    );
	f9 =SF*f9 +(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +( 0.1666666667f*w)                -0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)            );
	f10=SF*f10+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u+w)+0.01666666667f*(-u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
	f11=SF*f11+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	f12=SF*f12+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u+w)+0.01666666667f*( u-w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      -0.25f*u*v);
	f13=SF*f13+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v+w)+0.01666666667f*(-v-w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         -0.25f*v*w      );
	f14=SF*f14+(1.0f-SF)*(0.052631579f*rho+-0.0045948204f*(-11.f*rho+19.f*usqr)+-0.015873016f*m2 +(-0.1666666667f*w)                +0.027777778f*(2.f*u*u-(v*v+w*w))  +0.0833333333f*(v*v-w*w)            );
	f15=SF*f15+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( u-w)+0.01666666667f*(-u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
	f16=SF*f16+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*( v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	f17=SF*f17+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-u-w)+0.01666666667f*( u+w) +0.027777778f*(2.f*u*u-(v*v+w*w))  -0.0833333333f*(v*v-w*w)      +0.25f*u*v);
	f18=SF*f18+(1.0f-SF)*(0.052631579f*rho+ 0.0033416876f*(-11.f*rho+19.f*usqr)+ 0.003968254f*m2 +0.1f*(-v-w)+0.01666666667f*(-v+w) -0.055555556f*(2.f*u*u-(v*v+w*w))                         +0.25f*v*w      );
	}
	else
	{

	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.3333333333f*(rho-1.5f*usqr));
	f1 =SF*f1 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 =SF*f2 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 =SF*f6 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 =SF*f9 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10=SF*f10+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11=SF*f11+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12=SF*f12+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13=SF*f13+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14=SF*f14+(1.0f-SF)*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15=SF*f15+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16=SF*f16+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17=SF*f17+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18=SF*f18+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));
	}

	fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
	fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
	fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
	fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
	fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
	fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
	fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
	fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
	fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
	fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
	fout[f_memLR(10,x,y,z,pitch)] = f10;
	fout[f_memLR(11,x,y,z,pitch)] = f11;
	fout[f_memLR(12,x,y,z,pitch)] = f12;
	fout[f_memLR(13,x,y,z,pitch)] = f13;
	fout[f_memLR(14,x,y,z,pitch)] = f14;
	fout[f_memLR(15,x,y,z,pitch)] = f15;
	fout[f_memLR(16,x,y,z,pitch)] = f16;
	fout[f_memLR(17,x,y,z,pitch)] = f17;
	fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	//if(z != ZLRDIM-1){
	f18= fin[f_memLR(18,x  ,dmin(y+1,YLRDIM),dmin(z+1,ZLRDIM),pitch)];
	//}
	//else f18 = 0.1f;
	if(im == 1){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}


__global__ void mrt_d_hybAB(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch)];
	f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f11= fin[f_mem(11,x  ,y-1,z-1,pitch)];
	f13= fin[f_mem(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_mem(14,x  ,y  ,z+1,pitch)];
	f16= fin[f_mem(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fin[f_mem(18,x  ,y+1,z+1,pitch)];
	}
	f1 = tex2D(texRef_f1A ,x-1+0.5f,y  +0.5f+YDIM*(z));
	f3 = tex2D(texRef_f3A ,x+1+0.5f,y  +0.5f+YDIM*(z));
	f5 = tex2D(texRef_f5A ,x-1+0.5f,y-1+0.5f+YDIM*(z));
	f6 = tex2D(texRef_f6A ,x+1+0.5f,y-1+0.5f+YDIM*(z));
	f7 = tex2D(texRef_f7A ,x+1+0.5f,y+1+0.5f+YDIM*(z));
	f8 = tex2D(texRef_f8A ,x-1+0.5f,y+1+0.5f+YDIM*(z));
	f15= tex2D(texRef_f15A,x-1+0.5f,y  +0.5f+YDIM*(z+1));
	f17= tex2D(texRef_f17A,x+1+0.5f,y  +0.5f+YDIM*(z+1));
	f10= tex2D(texRef_f10A,x-1+0.5f,y  +0.5f+YDIM*(z-1));
	f12= tex2D(texRef_f12A,x+1+0.5f,y  +0.5f+YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);

		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}
__global__ void mrt_d_hybBA(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch)];
	f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f11= fin[f_mem(11,x  ,y-1,z-1,pitch)];
	f13= fin[f_mem(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_mem(14,x  ,y  ,z+1,pitch)];
	f16= fin[f_mem(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fin[f_mem(18,x  ,y+1,z+1,pitch)];
	}
	f1 = tex2D(texRef_f1B ,x-1+0.5f,y  +0.5f+YDIM*(z));
	f3 = tex2D(texRef_f3B ,x+1+0.5f,y  +0.5f+YDIM*(z));
	f5 = tex2D(texRef_f5B ,x-1+0.5f,y-1+0.5f+YDIM*(z));
	f6 = tex2D(texRef_f6B ,x+1+0.5f,y-1+0.5f+YDIM*(z));
	f7 = tex2D(texRef_f7B ,x+1+0.5f,y+1+0.5f+YDIM*(z));
	f8 = tex2D(texRef_f8B ,x-1+0.5f,y+1+0.5f+YDIM*(z));
	f15= tex2D(texRef_f15B,x-1+0.5f,y  +0.5f+YDIM*(z+1));
	f17= tex2D(texRef_f17B,x+1+0.5f,y  +0.5f+YDIM*(z+1));
	f10= tex2D(texRef_f10B,x-1+0.5f,y  +0.5f+YDIM*(z-1));
	f12= tex2D(texRef_f12B,x+1+0.5f,y  +0.5f+YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}


__global__ void mrt_d_textAB(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = tex2D(texRef_f2A ,x  ,y-1+YDIM*(z));
	f4 = tex2D(texRef_f4A ,x  ,y+1+YDIM*(z));
	f9 = tex2D(texRef_f9A ,x  ,y+1+YDIM*(z-1));
	f11= tex2D(texRef_f11A,x  ,y-1+YDIM*(z-1));
	f13= tex2D(texRef_f13A,x  ,y+1+YDIM*(z-1));
	f14= tex2D(texRef_f14A,x  ,y  +YDIM*(z+1));
	f16= tex2D(texRef_f16A,x  ,y-1+YDIM*(z+1));
	f18= tex2D(texRef_f18A,x  ,y+1+YDIM*(z+1));

	f1 = tex2D(texRef_f1A ,x-1,y  +YDIM*(z));
	f3 = tex2D(texRef_f3A ,x+1,y  +YDIM*(z));
	f5 = tex2D(texRef_f5A ,x-1,y-1+YDIM*(z));
	f6 = tex2D(texRef_f6A ,x+1,y-1+YDIM*(z));
	f7 = tex2D(texRef_f7A ,x+1,y+1+YDIM*(z));
	f8 = tex2D(texRef_f8A ,x-1,y+1+YDIM*(z));
	f15= tex2D(texRef_f15A,x-1,y  +YDIM*(z+1));
	f17= tex2D(texRef_f17A,x+1,y  +YDIM*(z+1));
	f10= tex2D(texRef_f10A,x-1,y  +YDIM*(z-1));
	f12= tex2D(texRef_f12A,x+1,y  +YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}

__global__ void mrt_d_textBA(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = tex2D(texRef_f2B ,x  ,y-1+YDIM*(z));
	f4 = tex2D(texRef_f4B ,x  ,y+1+YDIM*(z));
	f9 = tex2D(texRef_f9B ,x  ,y+1+YDIM*(z-1));
	f11= tex2D(texRef_f11B,x  ,y-1+YDIM*(z-1));
	f13= tex2D(texRef_f13B,x  ,y+1+YDIM*(z-1));
	f14= tex2D(texRef_f14B,x  ,y  +YDIM*(z+1));
	f16= tex2D(texRef_f16B,x  ,y-1+YDIM*(z+1));
	f18= tex2D(texRef_f18B,x  ,y+1+YDIM*(z+1));

	f1 = tex2D(texRef_f1B ,x-1,y  +YDIM*(z));
	f3 = tex2D(texRef_f3B ,x+1,y  +YDIM*(z));
	f5 = tex2D(texRef_f5B ,x-1,y-1+YDIM*(z));
	f6 = tex2D(texRef_f6B ,x+1,y-1+YDIM*(z));
	f7 = tex2D(texRef_f7B ,x+1,y+1+YDIM*(z));
	f8 = tex2D(texRef_f8B ,x-1,y+1+YDIM*(z));
	f15= tex2D(texRef_f15B,x-1,y  +YDIM*(z+1));
	f17= tex2D(texRef_f17B,x+1,y  +YDIM*(z+1));
	f10= tex2D(texRef_f10B,x-1,y  +YDIM*(z-1));
	f12= tex2D(texRef_f12B,x+1,y  +YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}



__global__ void mrt_d_shared(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = blockIdx.y;//threadIdx.y+blockIdx.y*blockDim.y;
	int z = blockIdx.z;//threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,z);

	__shared__ float  f1_s[BLOCKSIZEX];
	__shared__ float  f3_s[BLOCKSIZEX];
	__shared__ float  f5_s[BLOCKSIZEX];
	__shared__ float  f7_s[BLOCKSIZEX];
	__shared__ float  f6_s[BLOCKSIZEX];
	__shared__ float  f8_s[BLOCKSIZEX];
	__shared__ float f10_s[BLOCKSIZEX];
	__shared__ float f12_s[BLOCKSIZEX];
	__shared__ float f15_s[BLOCKSIZEX];
	__shared__ float f17_s[BLOCKSIZEX];

	 f1_s[threadIdx.x] = fA[f_mem(1 ,x ,y  ,z               ,pitch)];//dmax(x-1)      
	 f3_s[threadIdx.x] = fA[f_mem(3 ,x ,y  ,z               ,pitch)];//dmin(x+1,XDIM)
//	if(y != 0){//takin these out was good
	 f5_s[threadIdx.x] = fA[f_mem(5 ,x ,y-1,z               ,pitch)];//dmax(x-1)      
	 f6_s[threadIdx.x] = fA[f_mem(6 ,x ,y-1,z               ,pitch)];//dmin(x+1,XDIM)
//	}
//	else{
//	f5_s[threadIdx.x] = 0.f;
//	f6_s[threadIdx.x] = 0.f;
//	}
//	if(y != YDIM-1){
	 f7_s[threadIdx.x] = fA[f_mem(7 ,x ,y+1,z               ,pitch)];//dmin(x+1,XDIM)
	 f8_s[threadIdx.x] = fA[f_mem(8 ,x ,y+1,z               ,pitch)];//dmax(x-1)      
//	}
//	else{
//	f7_s[threadIdx.x] = 0.f;
//	f8_s[threadIdx.x] = 0.f;
//	}
//	if(z != 0){
	f10_s[threadIdx.x] = fA[f_mem(10,x ,y  ,z-1,pitch)];//dmax(x-1)      
	f12_s[threadIdx.x] = fA[f_mem(12,x ,y  ,z-1,pitch)];//dmin(x+1,XDIM)
//	}
//	else{
//	f10_s[threadIdx.x] = 0.f;
//	f12_s[threadIdx.x] = 0.f;
//	}
//	if(z != ZDIM-1){
	f15_s[threadIdx.x] = fA[f_mem(15,x ,y  ,z+1,pitch)];//dmax(x-1)      
	f17_s[threadIdx.x] = fA[f_mem(17,x ,y  ,z+1,pitch)];//dmin(x+1,XDIM)
//	}
//	else{
//	f15_s[threadIdx.x] = 0.f;
//	f17_s[threadIdx.x] = 0.f;
//	}

	float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	//f0  = 0.f;
	f1  = 0.f;
	f2  = 0.f;
	f3  = 0.f;
	f4  = 0.f;
	f5  = 0.f;
	f6  = 0.f;
	f7  = 0.f;
	f8  = 0.f;
	f9  = 0.f;
	f10 = 0.f;
	f11 = 0.f;
	f12 = 0.f;
	f13 = 0.f;
	f14 = 0.f;
	f15 = 0.f;
	f16 = 0.f;
	f17 = 0.f;
	f18 = 0.f;

//	if(y != 0){
	f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch)];
//	}
//	if(y != YDIM-1){
	f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch)];
//	}
	f14= fA[f_mem(14,x  ,y  ,z+1,pitch)];
	f16= fA[f_mem(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fA[f_mem(18,x  ,y+1,z+1,pitch)];
	}
//	else{
//	f18 = 0.0f;
//	}
	
//		if(z != 0){
	f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f11= fA[f_mem(11,x  ,y-1,z-1,pitch)];
	f13= fA[f_mem(13,x  ,y+1,z-1,pitch)];
//		}

	__syncthreads();

	if(threadIdx.x != XDIM-1){
	f3 = f3_s[threadIdx.x+1];//fA[f_mem(3 ,x+1,y  ,z  ,pitch,YDIM,ZDIM)];
	f6 = f6_s[threadIdx.x+1];//fA[f_mem(6 ,x+1,y-1,z  ,pitch,YDIM,ZDIM)];
	f7 = f7_s[threadIdx.x+1];//fA[f_mem(7 ,x+1,y+1,z  ,pitch,YDIM,ZDIM)];
	f12=f12_s[threadIdx.x+1];//fA[f_mem(12,x+1,y  ,z-1,pitch,YDIM,ZDIM)];
	f17=f17_s[threadIdx.x+1];//fA[f_mem(17,x+1,y  ,z+1,pitch,YDIM,ZDIM)];
	}
//	else{
//	f3 =0.f;
//	f6 =0.f;
//	f7 =0.f;
//	f12=0.f;
//	f17=0.f;
//	}
	if(threadIdx.x != 0){
	f1 = f1_s[threadIdx.x-1];//fA[f_mem(1 ,x-1,y  ,z  ,pitch,YDIM,ZDIM)];
	f5 = f5_s[threadIdx.x-1];//fA[f_mem(5 ,x-1,y-1,z  ,pitch,YDIM,ZDIM)];
	f8 = f8_s[threadIdx.x-1];//fA[f_mem(8 ,x-1,y+1,z  ,pitch,YDIM,ZDIM)];
	f10=f10_s[threadIdx.x-1];//fA[f_mem(10,x-1,y  ,z-1,pitch,YDIM,ZDIM)];
	f15=f15_s[threadIdx.x-1];//fA[f_mem(15,x-1,y  ,z+1,pitch,YDIM,ZDIM)];
	}
//	else{
//	f1 =0.f;
//	f5 =0.f;
//	f8 =0.f;
//	f10=0.f;
//	f15=0.f;
//	}




	if(im == 1){//BB

//	float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//
//	f4 = fA[f_mem(2 ,x  ,y-1,z  ,pitch)];
//	f2 = fA[f_mem(4 ,x  ,y+1,z  ,pitch)];
//	f9 = fA[f_mem(14,x  ,y  ,z+1,pitch)];
//	f13= fA[f_mem(16,x  ,y-1,z+1,pitch)];
//	if(z != ZDIM-1){
//	f11= fA[f_mem(18,x  ,y+1,z+1,pitch)];
//	}
//	f14= fA[f_mem(9 ,x  ,y  ,z-1,pitch)];
//	f18= fA[f_mem(11,x  ,y-1,z-1,pitch)];
//	f16= fA[f_mem(13,x  ,y+1,z-1,pitch)];
//
//	if(threadIdx.x != XDIM-1){
//	f1 = f3_s[threadIdx.x+1];//fA[f_mem(3 ,x+1,y  ,z  ,pitch,YDIM,ZDIM)];
//	f5 = f7_s[threadIdx.x+1];//fA[f_mem(6 ,x+1,y-1,z  ,pitch,YDIM,ZDIM)];
//	f8 = f6_s[threadIdx.x+1];//fA[f_mem(7 ,x+1,y+1,z  ,pitch,YDIM,ZDIM)];
//	f10=f17_s[threadIdx.x+1];//fA[f_mem(12,x+1,y  ,z-1,pitch,YDIM,ZDIM)];
//	f15=f12_s[threadIdx.x+1];//fA[f_mem(17,x+1,y  ,z+1,pitch,YDIM,ZDIM)];
//	}
//	if(threadIdx.x != 0){
//	f3 = f1_s[threadIdx.x-1];//fA[f_mem(1 ,x-1,y  ,z  ,pitch,YDIM,ZDIM)];
//	f7 = f5_s[threadIdx.x-1];//fA[f_mem(5 ,x-1,y-1,z  ,pitch,YDIM,ZDIM)];
//	f6 = f8_s[threadIdx.x-1];//fA[f_mem(8 ,x-1,y+1,z  ,pitch,YDIM,ZDIM)];
//	f17=f10_s[threadIdx.x-1];//fA[f_mem(10,x-1,y  ,z-1,pitch,YDIM,ZDIM)];
//	f12=f15_s[threadIdx.x-1];//fA[f_mem(15,x-1,y  ,z+1,pitch,YDIM,ZDIM)];
//	}

	//	fB[j] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f14;
		fB[f_mem(10,x,y,z,pitch)] = f17;
		fB[f_mem(11,x,y,z,pitch)] = f18;
		fB[f_mem(12,x,y,z,pitch)] = f15;
		fB[f_mem(13,x,y,z,pitch)] = f16;
		fB[f_mem(14,x,y,z,pitch)] = f9 ;
		fB[f_mem(15,x,y,z,pitch)] = f12;
		fB[f_mem(16,x,y,z,pitch)] = f13;
		fB[f_mem(17,x,y,z,pitch)] = f10;
		fB[f_mem(18,x,y,z,pitch)] = f11;

//		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
//		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
//		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
//		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
//		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
//		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
//		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
//		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
//		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
//		fB[f_mem(10,x,y,z,pitch)] = f10;
//		fB[f_mem(11,x,y,z,pitch)] = f11;
//		fB[f_mem(12,x,y,z,pitch)] = f12;
//		fB[f_mem(13,x,y,z,pitch)] = f13;
//		fB[f_mem(14,x,y,z,pitch)] = f14;
//		fB[f_mem(15,x,y,z,pitch)] = f15;
//		fB[f_mem(16,x,y,z,pitch)] = f16;
//		fB[f_mem(17,x,y,z,pitch)] = f17;
//		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
	else{


	float f0 = fA[j];


		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);

		}
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fB[f_mem(10,x,y,z,pitch)] = f10;
		fB[f_mem(11,x,y,z,pitch)] = f11;
		fB[f_mem(12,x,y,z,pitch)] = f12;
		fB[f_mem(13,x,y,z,pitch)] = f13;
		fB[f_mem(14,x,y,z,pitch)] = f14;
		fB[f_mem(15,x,y,z,pitch)] = f15;
		fB[f_mem(16,x,y,z,pitch)] = f16;
		fB[f_mem(17,x,y,z,pitch)] = f17;
		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
}

__global__ void mrt_d_single(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,z);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 
		&& y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1 && z > LRZ0+1 && z < LRZ0+(ZLRDIM-1)*LRFACTOR-1){
	}
	else{

	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y  ,z  ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y  ,z  ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1,z  ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1,z  ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1,z  ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1,z  ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1,z  ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,y+1,z  ,pitch)];
	f9 = fA[f_mem(9 ,x             ,y  ,z-1,pitch)];
	f10= fA[f_mem(10,dmax(x-1)     ,y  ,z-1,pitch)];
	f11= fA[f_mem(11,x             ,y-1,z-1,pitch)];
	f12= fA[f_mem(12,dmin(x+1,XDIM),y  ,z-1,pitch)];
	f13= fA[f_mem(13,x             ,y+1,z-1,pitch)];
	f14= fA[f_mem(14,x             ,y  ,z+1,pitch)];
	f15= fA[f_mem(15,dmax(x-1)     ,y  ,z+1,pitch)];
	f16= fA[f_mem(16,x             ,y-1,z+1,pitch)];
	f17= fA[f_mem(17,dmin(x+1,XDIM),y  ,z+1,pitch)];
	//f18= fA[f_mem(18,x  ,dmin(y+1,YDIM),dmin(z+1,ZDIM),pitch)];
	f18= fA[f_mem(18,x             ,dmin(y+1,YDIM),dmin(z+1,ZDIM),pitch)];
	if(im == 1){//BB
		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f14;
		fB[f_mem(10,x,y,z,pitch)] = f17;
		fB[f_mem(11,x,y,z,pitch)] = f18;
		fB[f_mem(12,x,y,z,pitch)] = f15;
		fB[f_mem(13,x,y,z,pitch)] = f16;
		fB[f_mem(14,x,y,z,pitch)] = f9 ;
		fB[f_mem(15,x,y,z,pitch)] = f12;
		fB[f_mem(16,x,y,z,pitch)] = f13;
		fB[f_mem(17,x,y,z,pitch)] = f10;
		fB[f_mem(18,x,y,z,pitch)] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);

		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fB[f_mem(10,x,y,z,pitch)] = f10;
		fB[f_mem(11,x,y,z,pitch)] = f11;
		fB[f_mem(12,x,y,z,pitch)] = f12;
		fB[f_mem(13,x,y,z,pitch)] = f13;
		fB[f_mem(14,x,y,z,pitch)] = f14;
		fB[f_mem(15,x,y,z,pitch)] = f15;
		fB[f_mem(16,x,y,z,pitch)] = f16;
		fB[f_mem(17,x,y,z,pitch)] = f17;
		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
	}
}




__global__ void initialize_single(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	f[j+0 *pitch*YDIM*ZDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f[j+12*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f[j+14*pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}
__global__ void initialize_LR(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	f[j+0 *pitch*YLRDIM*ZLRDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f[j+12*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f[j+14*pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}
__global__ void initialize(float* f0, float* f1, float* f2,
							float* f3, float* f4, float* f5,
							float* f6, float* f7, float* f8, float* f9,
							float* f10, float* f11, float* f12,
							float* f13, float* f14, float* f15,
							float* f16, float* f17, float* f18,
							size_t pitch)//pitch in elements
//__global__ void initialize(void** f0in, void** f1in, 
//							int w, int h, int pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int i = x+y*XDIM+z*XDIM*YDIM;//index on linear mem
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
	float u,v,w,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.1f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	feq = 1.0f/3.0f*(rho-1.5f*usqr);
	f0[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[j] = feq;

	feq = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f9[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f10[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f11[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f12[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f13[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f14[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f15[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f16[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f17[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
	f18[j] = feq;
}



int main(int argc, char *argv[])
{

	//int *image_d, *image_h;

	ofstream output;
	output.open ("LBM1_out.dat");

	size_t memsize, memsize2;
	size_t pitch = 0;
	size_t pitch2 = 0;
	int i, n, nBlocks, nBlocks2, n2;
	float omega, CharLength, omega2;

	CharLength = XDIM-2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));
	float SF_cf = omega*(1.0f-omega2)/((1.0f-omega)*omega2/LRFACTOR);
	float SF_fc = 1.f/SF_cf;

	cout<<"omega: "<<omega<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<"x"<<BLOCKSIZEZ<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<"x"<<ZDIM<<endl;
	cout<<"LRblocksize: "<<BLOCKSIZELRX<<"x"<<BLOCKSIZELRY<<"x"<<BLOCKSIZELRZ<<endl;
	cout<<"LRgrid: "<<XLRDIM<<"x"<<YLRDIM<<"x"<<ZLRDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

	nBlocks = (XDIM/BLOCKSIZEX+XDIM%BLOCKSIZEX)*(YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY)
				*(ZDIM/BLOCKSIZEZ+ZDIM%BLOCKSIZEZ);
	nBlocks2 = (XLRDIM/BLOCKSIZELRX+XLRDIM%BLOCKSIZELRX)*(YLRDIM/BLOCKSIZELRY+YLRDIM%BLOCKSIZELRY)
				*(ZLRDIM/BLOCKSIZELRZ+ZLRDIM%BLOCKSIZELRZ);
	int B = BLOCKSIZEX*BLOCKSIZELRY*BLOCKSIZELRZ;
	int B2 = BLOCKSIZELRX*BLOCKSIZELRY*BLOCKSIZELRZ;
	n = nBlocks*B;
	n2 = nBlocks2*B2;
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
    dim3 grid(XDIM/BLOCKSIZEX,YDIM/BLOCKSIZEY,ZDIM/BLOCKSIZEZ);

    dim3 threads2(BLOCKSIZELRX, BLOCKSIZELRY, BLOCKSIZELRZ);
    dim3 grid2(XLRDIM/BLOCKSIZELRX,YLRDIM/BLOCKSIZELRY,ZLRDIM/BLOCKSIZELRZ);

	memsize = n*sizeof(float);
	//memsize_int = n*sizeof(int);
	memsize2 = n2*sizeof(float);

	//hipExtent extent = make_hipExtent(XDIM*sizeof(float),YDIM,ZDIM);

	//image_h = (int *)malloc(memsize_int);

	float *fA_h,*fA_d,*fB_d,*fC_h,*fC_d,*fD_d;
	fA_h = (float *)malloc(memsize*19);
	fC_h = (float *)malloc(memsize2*19);
	hipMallocPitch((void **) &fA_d, &pitch, XDIM*sizeof(float), YDIM*ZDIM*19);
	hipMallocPitch((void **) &fB_d, &pitch, XDIM*sizeof(float), YDIM*ZDIM*19);

//	if(REFINEMENT == "YES"){
	hipMallocPitch((void **) &fC_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM*19);
	hipMallocPitch((void **) &fD_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM*19);
//	}

	//hipMalloc((void **) &image_d, memsize_int);

	cout<<pitch<<", "<<pitch2<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);
	size_t pitch_elements2 = pitch2/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = i;
		fC_h[i] = 0;
	}
//	for (i = 0; i < n; i++)
//	{
//		int x = i%XDIM;
//		int y = (i/XDIM)%YDIM;
//		int z = (i/XDIM)/YDIM;
////		image_h[i] = 0;
////		if(x < 1) image_h[i] = 1;//DirichletWest
////		if(x > XDIM-2) image_h[i] = 1;//BB
////		if(y < 1) image_h[i] = 1;//BB
////		if(y > YDIM-2) image_h[i] = 1;//BB
////		if(z < 1) image_h[i] = 1;//DirichletWest
////		if(z > ZDIM-2) image_h[i] = 1;//BB
//	}
	//hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;
	texRef_f9B.normalized = false;
	texRef_f10B.normalized = false;
	texRef_f11B.normalized = false;
	texRef_f12B.normalized = false;
	texRef_f13B.normalized = false;
	texRef_f14B.normalized = false;
	texRef_f15B.normalized = false;
	texRef_f16B.normalized = false;
	texRef_f17B.normalized = false;
	texRef_f18B.normalized = false;
	texRef_f0B.filterMode = hipFilterModeLinear;
	texRef_f1B.filterMode = hipFilterModeLinear;
	texRef_f2B.filterMode = hipFilterModeLinear;
	texRef_f3B.filterMode = hipFilterModeLinear;
	texRef_f4B.filterMode = hipFilterModeLinear;
	texRef_f5B.filterMode = hipFilterModeLinear;
	texRef_f6B.filterMode = hipFilterModeLinear;
	texRef_f7B.filterMode = hipFilterModeLinear;
	texRef_f8B.filterMode = hipFilterModeLinear;
	texRef_f9B.filterMode = hipFilterModeLinear;
	texRef_f10B.filterMode = hipFilterModeLinear;
	texRef_f11B.filterMode = hipFilterModeLinear;
	texRef_f12B.filterMode = hipFilterModeLinear;
	texRef_f13B.filterMode = hipFilterModeLinear;
	texRef_f14B.filterMode = hipFilterModeLinear;
	texRef_f15B.filterMode = hipFilterModeLinear;
	texRef_f16B.filterMode = hipFilterModeLinear;
	texRef_f17B.filterMode = hipFilterModeLinear;
	texRef_f18B.filterMode = hipFilterModeLinear;
	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f9A.normalized = false;
	texRef_f10A.normalized = false;
	texRef_f11A.normalized = false;
	texRef_f12A.normalized = false;
	texRef_f13A.normalized = false;
	texRef_f14A.normalized = false;
	texRef_f15A.normalized = false;
	texRef_f16A.normalized = false;
	texRef_f17A.normalized = false;
	texRef_f18A.normalized = false;
	texRef_f0A.filterMode = hipFilterModeLinear;
	texRef_f1A.filterMode = hipFilterModeLinear;
	texRef_f2A.filterMode = hipFilterModeLinear;
	texRef_f3A.filterMode = hipFilterModeLinear;
	texRef_f4A.filterMode = hipFilterModeLinear;
	texRef_f5A.filterMode = hipFilterModeLinear;
	texRef_f6A.filterMode = hipFilterModeLinear;
	texRef_f7A.filterMode = hipFilterModeLinear;
	texRef_f8A.filterMode = hipFilterModeLinear;
	texRef_f9A.filterMode = hipFilterModeLinear;
	texRef_f10A.filterMode = hipFilterModeLinear;
	texRef_f11A.filterMode = hipFilterModeLinear;
	texRef_f12A.filterMode = hipFilterModeLinear;
	texRef_f13A.filterMode = hipFilterModeLinear;
	texRef_f14A.filterMode = hipFilterModeLinear;
	texRef_f15A.filterMode = hipFilterModeLinear;
	texRef_f16A.filterMode = hipFilterModeLinear;
	texRef_f17A.filterMode = hipFilterModeLinear;
	texRef_f18A.filterMode = hipFilterModeLinear;

//	if(REFINEMENT == "YES"){
	texRef_f0C.normalized = false;
	texRef_f1C.normalized = false;
	texRef_f2C.normalized = false;
	texRef_f3C.normalized = false;
	texRef_f4C.normalized = false;
	texRef_f5C.normalized = false;
	texRef_f6C.normalized = false;
	texRef_f7C.normalized = false;
	texRef_f8C.normalized = false;
	texRef_f9C.normalized = false;
	texRef_f10C.normalized = false;
	texRef_f11C.normalized = false;
	texRef_f12C.normalized = false;
	texRef_f13C.normalized = false;
	texRef_f14C.normalized = false;
	texRef_f15C.normalized = false;
	texRef_f16C.normalized = false;
	texRef_f17C.normalized = false;
	texRef_f18C.normalized = false;
	texRef_f0C.filterMode = hipFilterModeLinear;
	texRef_f1C.filterMode = hipFilterModeLinear;
	texRef_f2C.filterMode = hipFilterModeLinear;
	texRef_f3C.filterMode = hipFilterModeLinear;
	texRef_f4C.filterMode = hipFilterModeLinear;
	texRef_f5C.filterMode = hipFilterModeLinear;
	texRef_f6C.filterMode = hipFilterModeLinear;
	texRef_f7C.filterMode = hipFilterModeLinear;
	texRef_f8C.filterMode = hipFilterModeLinear;
	texRef_f9C.filterMode = hipFilterModeLinear;
	texRef_f10C.filterMode = hipFilterModeLinear;
	texRef_f11C.filterMode = hipFilterModeLinear;
	texRef_f12C.filterMode = hipFilterModeLinear;
	texRef_f13C.filterMode = hipFilterModeLinear;
	texRef_f14C.filterMode = hipFilterModeLinear;
	texRef_f15C.filterMode = hipFilterModeLinear;
	texRef_f16C.filterMode = hipFilterModeLinear;
	texRef_f17C.filterMode = hipFilterModeLinear;
	texRef_f18C.filterMode = hipFilterModeLinear;
	texRef_f0D.normalized = false;
	texRef_f1D.normalized = false;
	texRef_f2D.normalized = false;
	texRef_f3D.normalized = false;
	texRef_f4D.normalized = false;
	texRef_f5D.normalized = false;
	texRef_f6D.normalized = false;
	texRef_f7D.normalized = false;
	texRef_f8D.normalized = false;
	texRef_f9D.normalized = false;
	texRef_f10D.normalized = false;
	texRef_f11D.normalized = false;
	texRef_f12D.normalized = false;
	texRef_f13D.normalized = false;
	texRef_f14D.normalized = false;
	texRef_f15D.normalized = false;
	texRef_f16D.normalized = false;
	texRef_f17D.normalized = false;
	texRef_f18D.normalized = false;
	texRef_f0D.filterMode = hipFilterModeLinear;
	texRef_f1D.filterMode = hipFilterModeLinear;
	texRef_f2D.filterMode = hipFilterModeLinear;
	texRef_f3D.filterMode = hipFilterModeLinear;
	texRef_f4D.filterMode = hipFilterModeLinear;
	texRef_f5D.filterMode = hipFilterModeLinear;
	texRef_f6D.filterMode = hipFilterModeLinear;
	texRef_f7D.filterMode = hipFilterModeLinear;
	texRef_f8D.filterMode = hipFilterModeLinear;
	texRef_f9D.filterMode = hipFilterModeLinear;
	texRef_f10D.filterMode = hipFilterModeLinear;
	texRef_f11D.filterMode = hipFilterModeLinear;
	texRef_f12D.filterMode = hipFilterModeLinear;
	texRef_f13D.filterMode = hipFilterModeLinear;
	texRef_f14D.filterMode = hipFilterModeLinear;
	texRef_f15D.filterMode = hipFilterModeLinear;
	texRef_f16D.filterMode = hipFilterModeLinear;
	texRef_f17D.filterMode = hipFilterModeLinear;
	texRef_f18D.filterMode = hipFilterModeLinear;
//	}
	}
	
	hipMemcpy2D(fA_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyHostToDevice);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*ZLRDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fD_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*ZLRDIM*19,hipMemcpyHostToDevice);
	}
	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = 0;
		fC_h[i] = 1;
	}


	if(true)//bind texture
	{
	hipBindTexture2D(0,&texRef_f0A, fA_d                            ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f1A, fA_d+pitch_elements*YDIM*ZDIM   ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f2A, fA_d+pitch_elements*YDIM*ZDIM*2 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f3A, fA_d+pitch_elements*YDIM*ZDIM*3 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f4A, fA_d+pitch_elements*YDIM*ZDIM*4 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f5A, fA_d+pitch_elements*YDIM*ZDIM*5 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f6A, fA_d+pitch_elements*YDIM*ZDIM*6 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f7A, fA_d+pitch_elements*YDIM*ZDIM*7 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f8A, fA_d+pitch_elements*YDIM*ZDIM*8 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f9A, fA_d+pitch_elements*YDIM*ZDIM*9 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f10A,fA_d+pitch_elements*YDIM*ZDIM*10,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f11A,fA_d+pitch_elements*YDIM*ZDIM*11,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f12A,fA_d+pitch_elements*YDIM*ZDIM*12,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f13A,fA_d+pitch_elements*YDIM*ZDIM*13,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f14A,fA_d+pitch_elements*YDIM*ZDIM*14,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f15A,fA_d+pitch_elements*YDIM*ZDIM*15,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f16A,fA_d+pitch_elements*YDIM*ZDIM*16,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f17A,fA_d+pitch_elements*YDIM*ZDIM*17,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f18A,fA_d+pitch_elements*YDIM*ZDIM*18,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f0B, fB_d                            ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f1B, fB_d+pitch_elements*YDIM*ZDIM   ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f2B, fB_d+pitch_elements*YDIM*ZDIM*2 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f3B, fB_d+pitch_elements*YDIM*ZDIM*3 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f4B, fB_d+pitch_elements*YDIM*ZDIM*4 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f5B, fB_d+pitch_elements*YDIM*ZDIM*5 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f6B, fB_d+pitch_elements*YDIM*ZDIM*6 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f7B, fB_d+pitch_elements*YDIM*ZDIM*7 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f8B, fB_d+pitch_elements*YDIM*ZDIM*8 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f9B, fB_d+pitch_elements*YDIM*ZDIM*9 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f10B,fB_d+pitch_elements*YDIM*ZDIM*10,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f11B,fB_d+pitch_elements*YDIM*ZDIM*11,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f12B,fB_d+pitch_elements*YDIM*ZDIM*12,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f13B,fB_d+pitch_elements*YDIM*ZDIM*13,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f14B,fB_d+pitch_elements*YDIM*ZDIM*14,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f15B,fB_d+pitch_elements*YDIM*ZDIM*15,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f16B,fB_d+pitch_elements*YDIM*ZDIM*16,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f17B,fB_d+pitch_elements*YDIM*ZDIM*17,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f18B,fB_d+pitch_elements*YDIM*ZDIM*18,&desc,XDIM,YDIM*ZDIM,pitch);

//	if(REFINEMENT == "YES"){
	hipBindTexture2D(0,&texRef_f0C, fC_d                                 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1C, fC_d+pitch_elements2*YLRDIM*ZLRDIM   ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*2 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*3 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*4 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*5 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*6 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*7 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*8 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f9C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*9 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f10C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*10,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f11C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*11,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f12C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*12,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f13C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*13,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f14C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*14,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f15C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*15,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f16C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*16,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f17C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*17,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f18C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*18,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f0D, fD_d                                 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1D, fD_d+pitch_elements2*YLRDIM*ZLRDIM   ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*2 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*3 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*4 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*5 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*6 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*7 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*8 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f9D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*9 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f10D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*10,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f11D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*11,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f12D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*12,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f13D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*13,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f14D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*14,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f15D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*15,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f16D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*16,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f17D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*17,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f18D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*18,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
//	}
	}


	initialize_single<<<grid, threads>>>(fA_d,pitch_elements);
	initialize_single<<<grid, threads>>>(fB_d,pitch_elements);
	if(REFINEMENT == "YES"){
	initialize_LR<<<grid2, threads2>>>(fC_d,pitch_elements2);
	}

//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(mrt_d_single),hipFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<TMAX; t=t+2){
		if(METHOD == "SINGLE"){
		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		
		if(REFINEMENT == "YES"){
			LR_d_ABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
			LR_d_ABDC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
			
			//ExtractFromC_d<<<grid, threads>>>(fB_d,pitch_elements,SF_fc);
		}
		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);

		if(REFINEMENT == "YES"){
			LR_d_BACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
			LR_d_BADC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
	
			//ExtractFromC_d<<<grid, threads>>>(fA_d,pitch_elements,SF_fc);
		}
		}

		else if(METHOD == "HYB"){
		mrt_d_hybAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);

		if(REFINEMENT == "YES"){
		LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
		LR_d_hybABDC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
		
		ExtractFromC_d<<<grid, threads>>>(fB_d,pitch_elements,SF_fc);
		}


		mrt_d_hybBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);

		if(REFINEMENT == "YES"){
		LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
		LR_d_hybBADC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
		
		ExtractFromC_d<<<grid, threads>>>(fA_d,pitch_elements,SF_fc);
		}
		}

		else if(METHOD == "TEXT"){
		mrt_d_textAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		mrt_d_textBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
		}

		else if(METHOD == "SHARED"){
		mrt_d_shared<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		mrt_d_shared<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
		}

//		simple_copy<<<grid, threads>>>(fA_d,fB_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);
//		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);

		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	int Nodes;
	if(REFINEMENT == "YES"){
		Nodes = (XDIM*YDIM*ZDIM+XLRDIM*YLRDIM*ZLRDIM*2);
	}
	else{
		Nodes = XDIM*YDIM*ZDIM;
	}
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
//			<<double((XDIM*YDIM*ZDIM)*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<XDIM<<","<<YDIM<<","<<ZDIM<<","<<TMAX<<","<<restime<<endl;

	if(true){
	hipUnbindTexture(texRef_f0A);
	hipUnbindTexture(texRef_f1A);
	hipUnbindTexture(texRef_f2A);
	hipUnbindTexture(texRef_f3A);
	hipUnbindTexture(texRef_f4A);
	hipUnbindTexture(texRef_f5A);
	hipUnbindTexture(texRef_f6A);
	hipUnbindTexture(texRef_f7A);
	hipUnbindTexture(texRef_f8A);
	hipUnbindTexture(texRef_f9A);
	hipUnbindTexture(texRef_f10A);
	hipUnbindTexture(texRef_f11A);
	hipUnbindTexture(texRef_f12A);
	hipUnbindTexture(texRef_f13A);
	hipUnbindTexture(texRef_f14A);
	hipUnbindTexture(texRef_f15A);
	hipUnbindTexture(texRef_f16A);
	hipUnbindTexture(texRef_f17A);
	hipUnbindTexture(texRef_f18A);
	hipUnbindTexture(texRef_f0B);
	hipUnbindTexture(texRef_f1B);
	hipUnbindTexture(texRef_f2B);
	hipUnbindTexture(texRef_f3B);
	hipUnbindTexture(texRef_f4B);
	hipUnbindTexture(texRef_f5B);
	hipUnbindTexture(texRef_f6B);
	hipUnbindTexture(texRef_f7B);
	hipUnbindTexture(texRef_f8B);
	hipUnbindTexture(texRef_f9B);
	hipUnbindTexture(texRef_f10B);
	hipUnbindTexture(texRef_f11B);
	hipUnbindTexture(texRef_f12B);
	hipUnbindTexture(texRef_f13B);
	hipUnbindTexture(texRef_f14B);
	hipUnbindTexture(texRef_f15B);
	hipUnbindTexture(texRef_f16B);
	hipUnbindTexture(texRef_f17B);
	hipUnbindTexture(texRef_f18B);

	hipUnbindTexture(texRef_f0C);
	hipUnbindTexture(texRef_f1C);
	hipUnbindTexture(texRef_f2C);
	hipUnbindTexture(texRef_f3C);
	hipUnbindTexture(texRef_f4C);
	hipUnbindTexture(texRef_f5C);
	hipUnbindTexture(texRef_f6C);
	hipUnbindTexture(texRef_f7C);
	hipUnbindTexture(texRef_f8C);
	hipUnbindTexture(texRef_f9C);
	hipUnbindTexture(texRef_f10C);
	hipUnbindTexture(texRef_f11C);
	hipUnbindTexture(texRef_f12C);
	hipUnbindTexture(texRef_f13C);
	hipUnbindTexture(texRef_f14C);
	hipUnbindTexture(texRef_f15C);
	hipUnbindTexture(texRef_f16C);
	hipUnbindTexture(texRef_f17C);
	hipUnbindTexture(texRef_f18C);
	hipUnbindTexture(texRef_f0D);
	hipUnbindTexture(texRef_f1D);
	hipUnbindTexture(texRef_f2D);
	hipUnbindTexture(texRef_f3D);
	hipUnbindTexture(texRef_f4D);
	hipUnbindTexture(texRef_f5D);
	hipUnbindTexture(texRef_f6D);
	hipUnbindTexture(texRef_f7D);
	hipUnbindTexture(texRef_f8D);
	hipUnbindTexture(texRef_f9D);
	hipUnbindTexture(texRef_f10D);
	hipUnbindTexture(texRef_f11D);
	hipUnbindTexture(texRef_f12D);
	hipUnbindTexture(texRef_f13D);
	hipUnbindTexture(texRef_f14D);
	hipUnbindTexture(texRef_f15D);
	hipUnbindTexture(texRef_f16D);
	hipUnbindTexture(texRef_f17D);
	hipUnbindTexture(texRef_f18D);
	}

	hipMemcpy2D(fA_h,XDIM*sizeof(float),fA_d,pitch,XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyDeviceToHost);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_h,XLRDIM*sizeof(float),fC_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM*19,hipMemcpyDeviceToHost);
	}

	//hipMemcpy(image_h, image_d, memsize_int, hipMemcpyDeviceToHost);

	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<", K="<<ZDIM<<"\n";
	
	int row = 0;
	int col = 0;
	int dep = 0;
	i = 0;
	float rho, u, v, w;
	int j;

	for(dep = 0; dep<ZDIM; dep++){
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = dep*XDIM*YDIM+row*XDIM+col;
//			rho = 0;
			rho = fA_h[i];
			for(j = 1; j<19; j++)
				rho+=fA_h[i+XDIM*YDIM*ZDIM*j];
//			rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i]+f9_h[i]+
//			      f10_h[i]+f11_h[i]+f12_h[i]+f13_h[i]+f14_h[i]+f15_h[i]+f16_h[i]+f17_h[i]+f18_h[i];
			u = fA_h[i+XDIM*YDIM*ZDIM*1]-fA_h[i+XDIM*YDIM*ZDIM*3]+fA_h[i+XDIM*YDIM*ZDIM*5]-fA_h[i+XDIM*YDIM*ZDIM*6]-
				fA_h[i+XDIM*YDIM*ZDIM*7]+fA_h[i+XDIM*YDIM*ZDIM*8]+fA_h[i+XDIM*YDIM*ZDIM*10]-fA_h[i+XDIM*YDIM*ZDIM*12]
				+fA_h[i+XDIM*YDIM*ZDIM*15]-fA_h[i+XDIM*YDIM*ZDIM*17];
			v = fA_h[i+XDIM*YDIM*ZDIM*2]-fA_h[i+XDIM*YDIM*ZDIM*4]+fA_h[i+XDIM*YDIM*ZDIM*5]+fA_h[i+XDIM*YDIM*ZDIM*6]-fA_h[i+XDIM*YDIM*ZDIM*7]-fA_h[i+XDIM*YDIM*ZDIM*8]+fA_h[i+XDIM*YDIM*ZDIM*11]-fA_h[i+XDIM*YDIM*ZDIM*13]+fA_h[i+XDIM*YDIM*ZDIM*16]-fA_h[i+XDIM*YDIM*ZDIM*18];
			w = fA_h[i+XDIM*YDIM*ZDIM*9]+fA_h[i+XDIM*YDIM*ZDIM*10]+fA_h[i+XDIM*YDIM*ZDIM*11]+fA_h[i+XDIM*YDIM*ZDIM*12]+fA_h[i+XDIM*YDIM*ZDIM*13]-fA_h[i+XDIM*YDIM*ZDIM*14]-fA_h[i+XDIM*YDIM*ZDIM*15]-fA_h[i+XDIM*YDIM*ZDIM*16]-fA_h[i+XDIM*YDIM*ZDIM*17]-fA_h[i+XDIM*YDIM*ZDIM*18];
			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;
//			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<fA_h[i+XDIM*YDIM*ZDIM*1]<<","<<rho<<endl;
		}
	}
	}

	if(REFINEMENT == "YES"){
	output<<endl<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<XLRDIM-0<<", J="<<YLRDIM-0<<", K="<<ZLRDIM-0<<"\n";
	
	for(dep = 0; dep<ZLRDIM-0; dep++){
	for(row = 0; row<YLRDIM-0; row++){
		for(col = 0; col<XLRDIM-0; col++){
			i = dep*XLRDIM*YLRDIM+row*XLRDIM+col;
			rho = fC_h[i];
			for(j = 1; j<19; j++)
			
				rho+=fC_h[i+XLRDIM*YLRDIM*ZLRDIM*j];
				u = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*1]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*3]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*5]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*6]-
				fC_h[i+XLRDIM*YLRDIM*ZLRDIM*7]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*8]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*10]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*12]
				+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*15]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*17];
				v = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*2]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*4]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*5]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*6]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*7]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*8]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*11]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*13]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*16]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*18];
				w = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*9]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*10]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*11]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*12]+fC_h[i+XLRDIM*YLRDIM*ZLRDIM*13]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*14]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*15]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*16]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*17]-fC_h[i+XLRDIM*YLRDIM*ZLRDIM*18];
			output<<LRX0+col*LRFACTOR<<", "<<LRY0+row*LRFACTOR<<", "<<LRZ0+dep*LRFACTOR<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;


		}
	}
	}
	}

	output.close();


	//hipFree(image_d);


	hipFree(fA_d);
	hipFree(fB_d);
	hipFree(fC_d);
	hipFree(fD_d);
	return(0);

}
