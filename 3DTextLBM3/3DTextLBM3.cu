#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;

//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>


int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

__device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

	f0 = f0 -omega*(f0 -0.3333333333f*(rho-1.5f*usqr));
	f1 = f1 -omega*(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 = f2 -omega*(f2 -0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 = f3 -omega*(f3 -0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 = f4 -omega*(f4 -0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 = f5 -omega*(f5 -0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 = f6 -omega*(f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 = f7 -omega*(f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 = f8 -omega*(f8 -0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 = f9 -omega*(f9 -0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10= f10-omega*(f10-0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11= f11-omega*(f11-0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12= f12-omega*(f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13= f13-omega*(f13-0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14= f14-omega*(f14-0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15= f15-omega*(f15-0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16= f16-omega*(f16-0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17= f17-omega*(f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18= f18-omega*(f18-0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));

}


__device__ void mrt_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+  1.f*f8+ -4.f*f9+    f10+  1.f*f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;

	m1 -= -11.f*rho+19.f*(u*u+v*v+w*w);
	//m2 -= -475.f/63.f*(u*u+v*v+w*w);
	m2 -= -7.53968254f*(u*u+v*v+w*w);
	m4 -= -0.66666667f*u;//qx_eq
	m6 -= -0.66666667f*v;//qx_eq
	m8 -= -0.66666667f*w;//qx_eq
	m9 -= (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
//	m10-= 0.f;//.f.f;//.f.5*meq[9];/.f.f;//.f.5*meq[9];/.f.f;//pixx
	m11-= (v*v-w*w);//pww_eq
//	m12-= 0.f;//.f.f;//.f.5*meq[11];/.f.f;//.f.5*meq[9];/.f.f;//piww
	m13-= u*v;//pxy_eq
	m14-= v*w;//pyz_eq
	m15-= u*w;//pxz_eq
//	m16-= 0.0;//mx_eq
//	m17-= 0.0;//my_eq
//	m18-= 0.0;//mz_eq


f0  -= - 0.012531328f*(m1)+  0.047619048f*(m2);
f1  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+   -0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
f2  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +   -0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f3  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+    0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
f4  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +    0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f5  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f6  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f7  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f8  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f9  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +   -0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f10 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f11 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f12 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f13 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f14 -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +    0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f15 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f16 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f17 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f18 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);

f2  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
f4  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
f5  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
f6  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
f7  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
f8  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
f9  -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
f10 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
f11 -=                                                                         +(                         0.25f*(m14)                        )*omega ;
f12 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
f13 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
f14 -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
f15 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
f16 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
f17 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
f18 -=                                                                         +(                         0.25f*(m14)                        )*omega ;

f5  -=  0.125f*(m16)+ -0.125f*(m17);                        
f6  -= -0.125f*(m16)+ -0.125f*(m17);                        
f7  -= -0.125f*(m16)+  0.125f*(m17);                        
f8  -=  0.125f*(m16)+  0.125f*(m17);                        
f10 -= -0.125f*(m16)                              +  0.125f*(m18);
f11 -=                             +  0.125f*(m17)+ -0.125f*(m18);
f12 -=  0.125f*(m16)                              +  0.125f*(m18);
f13 -=                             + -0.125f*(m17)+ -0.125f*(m18);
f15 -= -0.125f*(m16)                              + -0.125f*(m18);
f16 -=                             +  0.125f*(m17)+  0.125f*(m18);
f17 -=  0.125f*(m16)                              + -0.125f*(m18);
f18 -=                             + -0.125f*(m17)+  0.125f*(m18);
}

//{
//	float u,v,w;	//rho,
////		rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
////		      f10+f11+f12+f13+f14+f15+f16+f17+f18;
//		u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
//		v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
//		w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//
//		float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//
//		//m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
////		m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
////				+11.f*(f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+f10+f11+f12+f13+f14+f15+f16+f17+f18) - 19.f*(u*u+v*v+w*w);
////		m1  = -19.f*f0+  19.f*f5+  19.f*f6+  19.f*f7+  19.f*f8+19.f*f10+  19.f*f11+  19.f*f12+  19.f*f13+  19.f*f15+  19.f*f16+  19.f*f17+  19.f*f18- 19.f*(u*u+v*v+w*w);
//		m1  = 19.f*(-f0+f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18-(u*u+v*v+w*w));
//		m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+    f8+ -4.f*f9+    f10+      f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
////		m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
////		m4  =-4.f*f1+4.f*f3+f5-f6-f7+f8+f10-f12+f15-f17+0.66666667f*(f1-f3+f5-f6-f7+f8+f10-f12+f15-f17);
//		m4  =-3.33333333f*f1+3.33333333f*f3 +1.66666667f*(f5+f8+f10+f15) -0.33333333f*(f3+f6+f7+f12+f17);
////		m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
////		m6  =-4.f*f2+4.f*f4+f5+f6-f7-f8+f11-f13+f16-f18+0.66666667f*(f2-f4+f5+f6-f7-f8+f11-f13+f16-f18);
//		m6  =-3.33333333f*f2+3.33333333f*f4+1.66666667f*(f5+f6+f11+f16) -0.33333333f*(f7+f8+f13+f18);//-0.66666667f*(f2-f4+f5+f6-f7-f8+f11-f13+f16-f18);
////		m8  =-4.f*f9+f10+f11+f12+f13+4.f*f14-f15-f16-f17-f18;
////		m8  =-4.f*f9+f10+f11+f12+f13+4.f*f14-f15-f16-f17-f18  +0.66666667f*(f9+f10+f11+f12+f13-f14-f15-f16-f17-f18);
//		m8  =-3.33333333f*f9+3.33333333f*f14+1.66666667f*(f10+f11+f12+f13)-0.33333333f*(f15+f16+f17+f18);//-0.33333333f*(f10+f11+f12+f13-f15-f16-f17-f18);
////		m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
////		m9  = 2.f*(f1+f3-f11-f13-f16-f18) -  f2-  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+    f12-  f14+    f15+    f17;
//		m9  = (f1+f3-f11-f13-f16-f18)+(f1+f3-f11-f13-f16-f18) -  f2-  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+    f12-  f14+    f15+    f17;
////		m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
//		m10 = (-f1-f3)+(-f1-f3)+(-f1-f3)+(-f1-f3)+(f2+f4+f9-f11-f13-f16-f18)+(f2+f4+f9-f11-f13-f16-f18)+    f5+    f6+    f7+    f8+    f10+    f12+    f15+    f17;
//		m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
////		m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
////		m12 = 2.f*(-f2-f4+f9+f14)                   +    f5+    f6+    f7+    f8-  f10          + -  f12-  f15          -  f17          ;
//		m12 = (-f2-f4+f9+f14)+(-f2-f4+f9+f14)                   +    f5+    f6+    f7+    f8-  f10          + -  f12-  f15          -  f17          ;
//		m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
//		m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
//		m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
//		m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
//		m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
//		m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;
//
////		m1 -= -11.f*rho+19.f*(u*u+v*v+w*w);
//		//m2 -= -475.f/63.f*(u*u+v*v+w*w);
//		m2 -= -7.53968254f*(u*u+v*v+w*w);
////		m4 -= -0.66666667f*u;//qx_eq
////		m6 -= -0.66666667f*v;//qx_eq
////		m8 -= -0.66666667f*w;//qx_eq
//		m9 -= (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
//		m11-= (v*v-w*w);//pww_eq
//		m13-= u*v;//pxy_eq
//		m14-= v*w;//pyz_eq
//		m15-= u*w;//pxz_eq
//
//
//f0  -= - 0.012531328f*(m1)+  0.047619048f*(m2);
//f1  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+   -0.1f*(m4)                            +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
//f2  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +    -0.1f*(m6)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f3  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+    0.1f*(m4)                            +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
//f4  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +     0.1f*(m6)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f5  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+  0.025f*(m6)              +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f6  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+  0.025f*(m6)              +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f7  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+ -0.025f*(m6)              +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f8  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+ -0.025f*(m6)              +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f9  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                            +   -0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f10 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)              +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f11 -=  0.0033416876f*(m1)+  0.003968254f*(m2)              +  0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//f12 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)              +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f13 -=  0.0033416876f*(m1)+  0.003968254f*(m2)              + -0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//f14 -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                            +    0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
//f15 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)              + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f16 -=  0.0033416876f*(m1)+  0.003968254f*(m2)              +  0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//f17 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)              + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
//f18 -=  0.0033416876f*(m1)+  0.003968254f*(m2)              + -0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
//
//f2  -=  0.083333333f*((m11)*omega-m12);// + -0.083333333f*(m12);
//f4  -=  0.083333333f*((m11)*omega-m12);// + -0.083333333f*(m12);
//f5  -=  0.083333333f*((m11)*omega + 0.5f*(m12))+ ( 0.25f*(m13))*omega; 
//f6  -=  0.083333333f*((m11)*omega + 0.5f*(m12))+ (-0.25f*(m13))*omega; 
//f7  -=  0.083333333f*((m11)*omega + 0.5f*(m12))+ ( 0.25f*(m13))*omega; 
//f8  -=  0.083333333f*((m11)*omega + 0.5f*(m12))+ (-0.25f*(m13))*omega; 
//f9  -= -0.083333333f*((m11)*omega +  (m12));
//f10 -= -0.083333333f*((m11)*omega + -0.5f*(m12)) +( 0.25f*(m15))*omega ;
//f11 -=       ( 0.25f*(m14))*omega ;
//f12 -= -0.083333333f*((m11)*omega + -0.5f*(m12)) +(-0.25f*(m15))*omega ;
//f13 -=  (-0.25f*(m14))*omega ;
//f14 -= -0.083333333f*((m11)*omega +  (m12));
//f15 -= -0.083333333f*((m11)*omega + -0.5f*(m12)) +(-0.25f*(m15))*omega ;
//f16 -=       (-0.25f*(m14))*omega ;
//f17 -= -0.083333333f*((m11)*omega + -0.5f*(m12)) +( 0.25f*(m15))*omega ;
//f18 -=       ( 0.25f*(m14))*omega ;
//
//f5  -=  0.125f*(m16)+ -0.125f*(m17);                        
//f6  -= -0.125f*(m16)+ -0.125f*(m17);                        
//f7  -= -0.125f*(m16)+  0.125f*(m17);                        
//f8  -=  0.125f*(m16)+  0.125f*(m17);                        
////f10 -= -0.125f*(m16)               +  0.125f*(m18);
//f10 -= -0.125f*(m16-m18);
//f11 -=              +  0.125f*(m17-m18);//+ -0.125f*(m18);
//f12 -=  0.125f*(m16+m18);//               +  0.125f*(m18);
//f13 -=              + -0.125f*(m17+m18);//+ -0.125f*(m18);
//f15 -= -0.125f*(m16+m18);//               + -0.125f*(m18);
//f16 -=              +  0.125f*(m17+m18);//+  0.125f*(m18);
//f17 -=  0.125f*(m16-m18);//               + -0.125f*(m18);
//f18 -=              + -0.125f*(m17-m18);//+  0.125f*(m18);
//
//
//
//
//
//
//}

inline __device__ int f_mem(int f_num, int x, int y, int z, size_t pitch, int height, int depth)
{
//	if (x<0 || x>pitch || y<0 || y>height || z<0 || z>depth) return 0;
//	else
		return (x+y*pitch+z*height*pitch)+f_num*pitch*height*depth;
}
__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}

__global__ void simple_copy(float* fA, float* fB,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)

//	fB[f_mem(1 ,x,y,z,pitch,height,depth)] = fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(2 ,x,y,z,pitch,height,depth)] = fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(3 ,x,y,z,pitch,height,depth)] = fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(4 ,x,y,z,pitch,height,depth)] = fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(5 ,x,y,z,pitch,height,depth)] = fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(6 ,x,y,z,pitch,height,depth)] = fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(7 ,x,y,z,pitch,height,depth)] = fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(8 ,x,y,z,pitch,height,depth)] = fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(9 ,x,y,z,pitch,height,depth)] = fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(10,x,y,z,pitch,height,depth)] = fA[f_mem(10,x,y,z,pitch,height,depth)];
//	fB[f_mem(11,x,y,z,pitch,height,depth)] = fA[f_mem(11,x,y,z,pitch,height,depth)];
//	fB[f_mem(12,x,y,z,pitch,height,depth)] = fA[f_mem(12,x,y,z,pitch,height,depth)];
//	fB[f_mem(13,x,y,z,pitch,height,depth)] = fA[f_mem(13,x,y,z,pitch,height,depth)];
//	fB[f_mem(14,x,y,z,pitch,height,depth)] = fA[f_mem(14,x,y,z,pitch,height,depth)];
//	fB[f_mem(15,x,y,z,pitch,height,depth)] = fA[f_mem(15,x,y,z,pitch,height,depth)];
//	fB[f_mem(16,x,y,z,pitch,height,depth)] = fA[f_mem(16,x,y,z,pitch,height,depth)];
//	fB[f_mem(17,x,y,z,pitch,height,depth)] = fA[f_mem(17,x,y,z,pitch,height,depth)];
//	fB[f_mem(18,x,y,z,pitch,height,depth)] = fA[f_mem(18,x,y,z,pitch,height,depth)];
	
	float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f1  = fA[j+pitch*height*depth];
	f2  = fA[j+pitch*height*depth+pitch*height*depth];
//	f1  = fA[(x+y*pitch+z*height*pitch)+pitch*height*depth];
//	f1  = fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//	f2  = fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//	f3  = fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//	f4  = fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//	f5  = fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//	f6  = fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//	f7  = fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//	f8  = fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//	f9  = fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//	f10 = fA[f_mem(10,x,y,z,pitch,height,depth)];
//	f11 = fA[f_mem(11,x,y,z,pitch,height,depth)];
//	f12 = fA[f_mem(12,x,y,z,pitch,height,depth)];
//	f13 = fA[f_mem(13,x,y,z,pitch,height,depth)];
//	f14 = fA[f_mem(14,x,y,z,pitch,height,depth)];
//	f15 = fA[f_mem(15,x,y,z,pitch,height,depth)];
//	f16 = fA[f_mem(16,x,y,z,pitch,height,depth)];
//	f17 = fA[f_mem(17,x,y,z,pitch,height,depth)];
//	f18 = fA[f_mem(18,x,y,z,pitch,height,depth)];

	fB[j+pitch*height*depth] = f1 ;//+0.01f;
	fB[j+pitch*height*depth+pitch*height*depth] = f2;
//	fB[(x+y*pitch+z*height*pitch)+pitch*height*depth] = f1 ;//+0.01f;
//	fB[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 +0.01f;
//	fB[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;//+0.01f;
//	fB[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;//+0.01f;
//	fB[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;//+0.01f;
//	fB[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;//+0.01f;
//	fB[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;//+0.01f;
//	fB[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;//+0.01f;
//	fB[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;//+0.01f;
//	fB[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;//+0.01f;
//	fB[f_mem(10,x,y,z,pitch,height,depth)] = f10;//+0.01f;
//	fB[f_mem(11,x,y,z,pitch,height,depth)] = f11;//+0.01f;
//	fB[f_mem(12,x,y,z,pitch,height,depth)] = f12;//+0.01f;
//	fB[f_mem(13,x,y,z,pitch,height,depth)] = f13;//+0.01f;
//	fB[f_mem(14,x,y,z,pitch,height,depth)] = f14;//+0.01f;
//	fB[f_mem(15,x,y,z,pitch,height,depth)] = f15;//+0.01f;
//	fB[f_mem(16,x,y,z,pitch,height,depth)] = f16;//+0.01f;
//	fB[f_mem(17,x,y,z,pitch,height,depth)] = f17;//+0.01f;
//	fB[f_mem(18,x,y,z,pitch,height,depth)] = f18;//+0.01f;

}

//int const blockx = 192;
//int const blocky = 1;

__global__ void mrt_d_single(float* fA, float* fB,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
//	int i = x+y*blockDim.x*gridDim.x;
	//float u,v,w,rho;//,usqr;
	int im = image[i];
	if(im == 1){//BB
		//__shared__ float f0_s [blockDim.x][blockDim.y];
//		__shared__ float f1_s [blockx][blocky];
//		__shared__ float f2_s [blockx][blocky];
//		__shared__ float f3_s [blockx][blocky];
//		__shared__ float f4_s [blockx][blocky];
//		__shared__ float f5_s [blockx][blocky];
//		__shared__ float f7_s [blockx][blocky];
//		__shared__ float f6_s [blockx][blocky];
//		__shared__ float f8_s [blockx][blocky];
//		__shared__ float f9_s [blockx][blocky];
//		__shared__ float f10_s[blockx][blocky];
//		__shared__ float f11_s[blockx][blocky];
//		__shared__ float f12_s[blockx][blocky];
//		__shared__ float f13_s[blockx][blocky];
//		__shared__ float f14_s[blockx][blocky];
//		__shared__ float f15_s[blockx][blocky];
//		__shared__ float f16_s[blockx][blocky];
//		__shared__ float f17_s[blockx][blocky];
//		__shared__ float f18_s[blockx][blocky];
//
//		f1_s [threadIdx.x][threadIdx.y] = fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f2_s [threadIdx.x][threadIdx.y] = fA[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//		f3_s [threadIdx.x][threadIdx.y] = fA[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];//fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//		f4_s [threadIdx.x][threadIdx.y] = fA[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//		f5_s [threadIdx.x][threadIdx.y] = fA[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//		f7_s [threadIdx.x][threadIdx.y] = fA[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//		f6_s [threadIdx.x][threadIdx.y] = fA[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//		f8_s [threadIdx.x][threadIdx.y] = fA[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//		f9_s [threadIdx.x][threadIdx.y] = fA[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//		f10_s[threadIdx.x][threadIdx.y] = fA[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(10,x,y,z,pitch,height,depth)];
//		f11_s[threadIdx.x][threadIdx.y] = fA[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(11,x,y,z,pitch,height,depth)];
//		f12_s[threadIdx.x][threadIdx.y] = fA[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(12,x,y,z,pitch,height,depth)];
//		f13_s[threadIdx.x][threadIdx.y] = fA[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(13,x,y,z,pitch,height,depth)];
//		f14_s[threadIdx.x][threadIdx.y] = fA[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(14,x,y,z,pitch,height,depth)];
//		f15_s[threadIdx.x][threadIdx.y] = fA[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(15,x,y,z,pitch,height,depth)];
//		f16_s[threadIdx.x][threadIdx.y] = fA[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(16,x,y,z,pitch,height,depth)];
//		f17_s[threadIdx.x][threadIdx.y] = fA[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(17,x,y,z,pitch,height,depth)];
//		f18_s[threadIdx.x][threadIdx.y] = fA[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(18,x,y,z,pitch,height,depth)];
//		fB[j+pitch*height*depth*1 ] = f1_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*2 ] = f2_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*3 ] = f3_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*4 ] = f4_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*5 ] = f5_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*6 ] = f7_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*7 ] = f6_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*8 ] = f8_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*9 ] = f9_s [threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*10] = f10_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*11] = f11_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*12] = f12_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*13] = f13_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*14] = f14_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*15] = f15_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*16] = f16_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*17] = f17_s[threadIdx.x][threadIdx.y];
//		fB[j+pitch*height*depth*18] = f18_s[threadIdx.x][threadIdx.y];

		float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f1 = fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f2 = fA[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(2 ,x,y,z,pitch,height,depth)];
		f3 = fA[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];//fA[f_mem(3 ,x,y,z,pitch,height,depth)];
		f4 = fA[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(4 ,x,y,z,pitch,height,depth)];
		f5 = fA[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(5 ,x,y,z,pitch,height,depth)];
		f7 = fA[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(7 ,x,y,z,pitch,height,depth)];
		f6 = fA[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(6 ,x,y,z,pitch,height,depth)];
		f8 = fA[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(8 ,x,y,z,pitch,height,depth)];
		f9 = fA[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(9 ,x,y,z,pitch,height,depth)];
		f10= fA[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(10,x,y,z,pitch,height,depth)];
		f11= fA[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(11,x,y,z,pitch,height,depth)];
		f12= fA[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(12,x,y,z,pitch,height,depth)];
		f13= fA[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(13,x,y,z,pitch,height,depth)];
		f14= fA[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(14,x,y,z,pitch,height,depth)];
		f15= fA[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(15,x,y,z,pitch,height,depth)];
		f16= fA[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(16,x,y,z,pitch,height,depth)];
		f17= fA[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(17,x,y,z,pitch,height,depth)];
		f18= fA[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(18,x,y,z,pitch,height,depth)];
		fB[j+pitch*height*depth*1 ] = f1 ;
		fB[j+pitch*height*depth*2 ] = f2 ;
		fB[j+pitch*height*depth*3 ] = f3 ;
		fB[j+pitch*height*depth*4 ] = f4 ;
		fB[j+pitch*height*depth*5 ] = f5 ;
		fB[j+pitch*height*depth*6 ] = f6 ;
		fB[j+pitch*height*depth*7 ] = f7 ;
		fB[j+pitch*height*depth*8 ] = f8 ;
		fB[j+pitch*height*depth*9 ] = f9 ;
		fB[j+pitch*height*depth*10] = f10;
		fB[j+pitch*height*depth*11] = f11;
		fB[j+pitch*height*depth*12] = f12;
		fB[j+pitch*height*depth*13] = f13;
		fB[j+pitch*height*depth*14] = f14;
		fB[j+pitch*height*depth*15] = f15;
		fB[j+pitch*height*depth*16] = f16;
		fB[j+pitch*height*depth*17] = f17;
		fB[j+pitch*height*depth*18] = f18;

//		fB[j+pitch*height*depth*1 ] = fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*2 ] = fA[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*3 ] = fA[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*4 ] = fA[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*5 ] = fA[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*6 ] = fA[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*7 ] = fA[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*8 ] = fA[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];
//		fB[j+pitch*height*depth*9 ] = fA[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[j+pitch*height*depth*10] = fA[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[j+pitch*height*depth*11] = fA[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];
//		fB[j+pitch*height*depth*12] = fA[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[j+pitch*height*depth*13] = fA[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[j+pitch*height*depth*14] = fA[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		fB[j+pitch*height*depth*15] = fA[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];
//		fB[j+pitch*height*depth*16] = fA[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];
//		fB[j+pitch*height*depth*17] = fA[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		fB[j+pitch*height*depth*18] = fA[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];

//		fB[f_mem(1 ,x,y,z,pitch,height,depth)] = fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(3 ,x,y,z,pitch,height,depth)] = fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(2 ,x,y,z,pitch,height,depth)] = fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(4 ,x,y,z,pitch,height,depth)] = fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(5 ,x,y,z,pitch,height,depth)] = fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(7 ,x,y,z,pitch,height,depth)] = fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(6 ,x,y,z,pitch,height,depth)] = fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(8 ,x,y,z,pitch,height,depth)] = fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(9 ,x,y,z,pitch,height,depth)] = fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//		fB[f_mem(14,x,y,z,pitch,height,depth)] = fA[f_mem(14,x,y,z,pitch,height,depth)];
//		fB[f_mem(10,x,y,z,pitch,height,depth)] = fA[f_mem(10,x,y,z,pitch,height,depth)];
//		fB[f_mem(17,x,y,z,pitch,height,depth)] = fA[f_mem(17,x,y,z,pitch,height,depth)];
//		fB[f_mem(11,x,y,z,pitch,height,depth)] = fA[f_mem(11,x,y,z,pitch,height,depth)];
//		fB[f_mem(18,x,y,z,pitch,height,depth)] = fA[f_mem(18,x,y,z,pitch,height,depth)];
//		fB[f_mem(12,x,y,z,pitch,height,depth)] = fA[f_mem(12,x,y,z,pitch,height,depth)];
//		fB[f_mem(15,x,y,z,pitch,height,depth)] = fA[f_mem(15,x,y,z,pitch,height,depth)];
//		fB[f_mem(13,x,y,z,pitch,height,depth)] = fA[f_mem(13,x,y,z,pitch,height,depth)];
//		fB[f_mem(16,x,y,z,pitch,height,depth)] = fA[f_mem(16,x,y,z,pitch,height,depth)];

//		fB[j] = fA[j];
//		fB[f_mem(1 ,x,y,z,pitch,height,depth)] = fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];
//		fB[f_mem(3 ,x,y,z,pitch,height,depth)] = fA[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];
//		fB[f_mem(2 ,x,y,z,pitch,height,depth)] = fA[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];
//		fB[f_mem(4 ,x,y,z,pitch,height,depth)] = fA[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		fB[f_mem(5 ,x,y,z,pitch,height,depth)] = fA[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];
//		fB[f_mem(7 ,x,y,z,pitch,height,depth)] = fA[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		fB[f_mem(6 ,x,y,z,pitch,height,depth)] = fA[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];
//		fB[f_mem(8 ,x,y,z,pitch,height,depth)] = fA[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];
//		fB[f_mem(9 ,x,y,z,pitch,height,depth)] = fA[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[f_mem(14,x,y,z,pitch,height,depth)] = fA[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		fB[f_mem(10,x,y,z,pitch,height,depth)] = fA[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[f_mem(17,x,y,z,pitch,height,depth)] = fA[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		fB[f_mem(11,x,y,z,pitch,height,depth)] = fA[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];
//		fB[f_mem(18,x,y,z,pitch,height,depth)] = fA[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];
//		fB[f_mem(12,x,y,z,pitch,height,depth)] = fA[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[f_mem(15,x,y,z,pitch,height,depth)] = fA[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];
//		fB[f_mem(13,x,y,z,pitch,height,depth)] = fA[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];
//		fB[f_mem(16,x,y,z,pitch,height,depth)] = fA[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];
	}
	else{


		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//		float meq1,meq2,meq4,meq6,meq7,meq8;
//		f0 = fA[j];
//		f1 = fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f2 = fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//		f3 = fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//		f4 = fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//		f5 = fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//		f6 = fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//		f7 = fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//		f8 = fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//		f9 = fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//		f10= fA[f_mem(10,x,y,z,pitch,height,depth)];
//		f11= fA[f_mem(11,x,y,z,pitch,height,depth)];
//		f12= fA[f_mem(12,x,y,z,pitch,height,depth)];
//		f13= fA[f_mem(13,x,y,z,pitch,height,depth)];
//		f14= fA[f_mem(14,x,y,z,pitch,height,depth)];
//		f15= fA[f_mem(15,x,y,z,pitch,height,depth)];
//		f16= fA[f_mem(16,x,y,z,pitch,height,depth)];
//		f17= fA[f_mem(17,x,y,z,pitch,height,depth)];
//		f18= fA[f_mem(18,x,y,z,pitch,height,depth)];


		f0 = fA[j];
		f1 = fA[f_mem(1 ,x-1,y  ,z  ,pitch,height,depth)];
		f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
		f3 = fA[f_mem(3 ,x+1,y  ,z  ,pitch,height,depth)];
		f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
		f5 = fA[f_mem(5 ,x-1,y-1,z  ,pitch,height,depth)];
		f6 = fA[f_mem(6 ,x+1,y-1,z  ,pitch,height,depth)];
		f7 = fA[f_mem(7 ,x+1,y+1,z  ,pitch,height,depth)];
		f8 = fA[f_mem(8 ,x-1,y+1,z  ,pitch,height,depth)];
		f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
		f10= fA[f_mem(10,x-1,y  ,z-1,pitch,height,depth)];
		f11= fA[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
		f12= fA[f_mem(12,x+1,y  ,z-1,pitch,height,depth)];
		f13= fA[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
		f14= fA[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
		f15= fA[f_mem(15,x-1,y  ,z+1,pitch,height,depth)];
		f16= fA[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
		f17= fA[f_mem(17,x+1,y  ,z+1,pitch,height,depth)];
		f18= fA[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];


//		f1 = fA[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];
//		f3 = fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];
//		f2 = fA[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		f4 = fA[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];
//		f5 = fA[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		f7 = fA[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];
//		f6 = fA[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];
//		f8 = fA[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];
//		f9 = fA[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		f14= fA[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		f10= fA[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		f17= fA[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		f11= fA[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];
//		f18= fA[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];
//		f12= fA[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];
//		f15= fA[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		f13= fA[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];
//		f16= fA[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];

		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == height-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			if(z == depth-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
//			float fInt1,fInt2;//,fDiff;
			float u,v,w,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = uMax;//0.0;
			w = 0.0f;
	        
//			fInt1 = f0+f2+f4+f9+f11+f13+f14+f16+f18;
//	        fInt2 = f3+f6+f7+f12+f17;
//        	rho = u+(fInt1+2.0f*fInt2); //D2Q9i
			
        	rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			float usqr = u*u+v*v+w*w;
		
//			f0 -= 1.0f/3.0f*(rho-1.5f*usqr);
//			f2 -= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//			f3 -= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//			f4 -= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//			f6 -= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//			f7 -= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//			f9 -= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//			f11-= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
//			f12-= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//			f13-= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
//			f14-= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//			f16-= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
//			f17-= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//			f18-= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

			f1 = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr)+f3-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr)+f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr)+f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr)+f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr)+f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		//bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fB[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fB[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fB[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fB[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fB[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fB[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fB[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fB[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fB[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fB[f_mem(18,x,y,z,pitch,height,depth)] = f18;
	}
}


__global__ void initialize_single(float *f, 
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	f[j+0 *pitch*height*depth]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*height*depth]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*height*depth]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*height*depth]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*height*depth]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*height*depth]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f[j+12*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f[j+14*pitch*height*depth]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}

__global__ void initialize(float* f0, float* f1, float* f2,
							float* f3, float* f4, float* f5,
							float* f6, float* f7, float* f8, float* f9,
							float* f10, float* f11, float* f12,
							float* f13, float* f14, float* f15,
							float* f16, float* f17, float* f18,
							int width, int height, size_t pitch)//pitch in elements
//__global__ void initialize(void** f0in, void** f1in, 
//							int w, int h, int pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
	float u,v,w,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	feq = 1.0f/3.0f*(rho-1.5f*usqr);
	f0[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[j] = feq;

	feq = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f9[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f10[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f11[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f12[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f13[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f14[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f15[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f16[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f17[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
	f18[j] = feq;
}
__global__ void copytest(hipPitchedPtr devPitchedPtr, float * test_d, int w, int h, int d)
//__global__ void copytest(float *test)//, int w, int h, int d)
//__global__ void copytest(int * image)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	char* devPtr = (char*)devPitchedPtr.ptr;
	int pitch = devPitchedPtr.pitch;
//	int slicepitch = pitch*height;
////	int pitch = devPitchedPtr.pitch;
//	char *slice = devPtr + blockIdx.x*slicepitch;
	float* test = (float *)(devPtr);
//	//int slicePitch = pitch*extent.height;
	//int i = threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	int i = x+y*w+z*w*h;//index on linear mem
	//int j = threadIdx.x+threadIdx.y*pitch+threadIdx.z*blockDim.y;
	int j = x+y*pitch/sizeof(float)+z*h*pitch/sizeof(float);//index on padded mem
	//if(test[i] == 2)
	//test[0] = 2.f;//test[i];
	test_d[i] = test[j];
	test[j] += 100;

}



int main(int argc, char *argv[])
{

//	float *f0_h, *f1_h, *f2_h, *f3_h, *f4_h, *f5_h, *f6_h, *f7_h, *f8_h, *f9_h;
//	float *f10_h, *f11_h, *f12_h, *f13_h, *f14_h, *f15_h, *f16_h, *f17_h, *f18_h;
//	float *f0_dA, *f1_dA, *f2_dA, *f3_dA, *f4_dA, *f5_dA, *f6_dA, *f7_dA, *f8_dA, *f9_dA;
//	float *f10_dA, *f11_dA, *f12_dA, *f13_dA, *f14_dA, *f15_dA, *f16_dA, *f17_dA, *f18_dA;
//	float *f0_dB, *f1_dB, *f2_dB, *f3_dB, *f4_dB, *f5_dB, *f6_dB, *f7_dB, *f8_dB, *f9_dB;
//	float *f10_dB, *f11_dB, *f12_dB, *f13_dB, *f14_dB, *f15_dB, *f16_dB, *f17_dB, *f18_dB;
	int *image_d, *image_h;



	//cudaPitchedPtr f0_d;


	ofstream output;
	output.open ("LBM1_out.dat");

	size_t memsize, memsize_int;
	size_t pitch;
	int i, n, nBlocks, xDim, yDim, zDim,tMax;
	float Re, omega, uMax, CharLength;
	
	int BLOCKSIZEx = 256;
	int BLOCKSIZEy = 1;
	int BLOCKSIZEz = 1;
	xDim = 256;
	yDim = 128;
	zDim = 32;
	tMax = 100;
	Re = 500.f;//100.f;
	uMax = 0.08f;
	CharLength = xDim-2.f;
	omega = 1.0f/(3.0f*(uMax*CharLength/Re)+0.5f);

	cout<<"omega: "<<omega<<endl;

	nBlocks = (xDim/BLOCKSIZEx+xDim%BLOCKSIZEx)*(yDim/BLOCKSIZEy+yDim%BLOCKSIZEy)
				*(zDim/BLOCKSIZEz+zDim%BLOCKSIZEz);
	int B = BLOCKSIZEx*BLOCKSIZEy*BLOCKSIZEz;
	n = nBlocks*B;//block*dimx*dimy
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEx, BLOCKSIZEy, BLOCKSIZEz);
    dim3 grid(xDim/BLOCKSIZEx,yDim/BLOCKSIZEy,zDim/BLOCKSIZEz);

	memsize = n*sizeof(float);
	memsize_int = n*sizeof(int);

	hipExtent extent = make_hipExtent(xDim*sizeof(float),yDim,zDim);

//	f0_h   = (float *)malloc(memsize);
//	f1_h   = (float *)malloc(memsize);
//	f2_h   = (float *)malloc(memsize);
//	f3_h   = (float *)malloc(memsize);
//	f4_h   = (float *)malloc(memsize);
//	f5_h   = (float *)malloc(memsize);
//	f6_h   = (float *)malloc(memsize);
//	f7_h   = (float *)malloc(memsize);
//	f8_h   = (float *)malloc(memsize);
//	f9_h   = (float *)malloc(memsize);
//	f10_h  = (float *)malloc(memsize);
//	f11_h  = (float *)malloc(memsize);
//	f12_h  = (float *)malloc(memsize);
//	f13_h  = (float *)malloc(memsize);
//	f14_h  = (float *)malloc(memsize);
//	f15_h  = (float *)malloc(memsize);
//	f16_h  = (float *)malloc(memsize);
//	f17_h  = (float *)malloc(memsize);
//	f18_h  = (float *)malloc(memsize);
//
	image_h = (int *)malloc(memsize_int);



	float *fA_h,*fA_d,*fB_d;
	fA_h = (float *)malloc(memsize*19);
	hipMallocPitch((void **) &fA_d, &pitch, xDim*sizeof(float), yDim*zDim*19);
	hipMallocPitch((void **) &fB_d, &pitch, xDim*sizeof(float), yDim*zDim*19);
	




	hipMalloc((void **) &image_d, memsize_int);
//	cudaMallocPitch((void **) &f0_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f1_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f2_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f3_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f4_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f5_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f6_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f7_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f8_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f9_dA , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f10_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f11_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f12_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f13_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f14_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f15_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f16_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f17_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f18_dA, &pitch, xDim*sizeof(float), yDim*zDim);
//	cout<<pitch<<endl;
//	cudaMallocPitch((void **) &f0_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f1_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f2_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f3_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f4_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f5_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f6_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f7_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f8_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f9_dB , &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f10_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f11_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f12_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f13_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f14_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f15_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f16_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f17_dB, &pitch, xDim*sizeof(float), yDim*zDim);
//	cudaMallocPitch((void **) &f18_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	cout<<pitch<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = i;
	}
	for (i = 0; i < n; i++)
	{
		int x = i%xDim;
		int y = (i/xDim)%yDim;
		int z = (i/xDim)/yDim;
//		f0_h[i]  = i;
//		f1_h[i]  = i;
//		f2_h[i]  = i;
//		f3_h[i]  = i;
//		f4_h[i]  = i;
//		f5_h[i]  = i;
//		f6_h[i]  = i;
//		f7_h[i]  = i;
//		f8_h[i]  = i;
//		f9_h[i]  = i;
//		f10_h[i] = i;
//		f11_h[i] = i;
//		f12_h[i] = i;
//		f13_h[i] = i;
//		f14_h[i] = i;
//		f15_h[i] = i;
//		f16_h[i] = i;
//		f17_h[i] = i;
//		f18_h[i] = i;
		image_h[i] = 0;
		if(x < 1) image_h[i] = 3;//DirichletWest
		if(x > xDim-2) image_h[i] = 1;//BB
		if(y < 1) image_h[i] = 1;//BB
		if(y > yDim-2) image_h[i] = 1;//BB
		if(z < 1) image_h[i] = 1;//DirichletWest
		if(z > zDim-2) image_h[i] = 1;//BB
	}
	hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
//	texRef_f0B.normalized = false;
//	texRef_f1B.normalized = false;
//	texRef_f2B.normalized = false;
//	texRef_f3B.normalized = false;
//	texRef_f4B.normalized = false;
//	texRef_f5B.normalized = false;
//	texRef_f6B.normalized = false;
//	texRef_f7B.normalized = false;
//	texRef_f8B.normalized = false;
//	texRef_f9B.normalized = false;
//	texRef_f10B.normalized = false;
//	texRef_f11B.normalized = false;
//	texRef_f12B.normalized = false;
//	texRef_f13B.normalized = false;
//	texRef_f14B.normalized = false;
//	texRef_f15B.normalized = false;
//	texRef_f16B.normalized = false;
//	texRef_f17B.normalized = false;
//	texRef_f18B.normalized = false;
//	texRef_f0B.filterMode = cudaFilterModePoint;
//	texRef_f1B.filterMode = cudaFilterModePoint;
//	texRef_f2B.filterMode = cudaFilterModePoint;
//	texRef_f3B.filterMode = cudaFilterModePoint;
//	texRef_f4B.filterMode = cudaFilterModePoint;
//	texRef_f5B.filterMode = cudaFilterModePoint;
//	texRef_f6B.filterMode = cudaFilterModePoint;
//	texRef_f7B.filterMode = cudaFilterModePoint;
//	texRef_f8B.filterMode = cudaFilterModePoint;
//	texRef_f9B.filterMode = cudaFilterModePoint;
//	texRef_f10B.filterMode = cudaFilterModePoint;
//	texRef_f11B.filterMode = cudaFilterModePoint;
//	texRef_f12B.filterMode = cudaFilterModePoint;
//	texRef_f13B.filterMode = cudaFilterModePoint;
//	texRef_f14B.filterMode = cudaFilterModePoint;
//	texRef_f15B.filterMode = cudaFilterModePoint;
//	texRef_f16B.filterMode = cudaFilterModePoint;
//	texRef_f17B.filterMode = cudaFilterModePoint;
//	texRef_f18B.filterMode = cudaFilterModePoint;
//	texRef_f0A.normalized = false;
//	texRef_f1A.normalized = false;
//	texRef_f2A.normalized = false;
//	texRef_f3A.normalized = false;
//	texRef_f4A.normalized = false;
//	texRef_f5A.normalized = false;
//	texRef_f6A.normalized = false;
//	texRef_f7A.normalized = false;
//	texRef_f8A.normalized = false;
//	texRef_f9A.normalized = false;
//	texRef_f10A.normalized = false;
//	texRef_f11A.normalized = false;
//	texRef_f12A.normalized = false;
//	texRef_f13A.normalized = false;
//	texRef_f14A.normalized = false;
//	texRef_f15A.normalized = false;
//	texRef_f16A.normalized = false;
//	texRef_f17A.normalized = false;
//	texRef_f18A.normalized = false;
//	texRef_f0A.filterMode = cudaFilterModePoint;
//	texRef_f1A.filterMode = cudaFilterModePoint;
//	texRef_f2A.filterMode = cudaFilterModePoint;
//	texRef_f3A.filterMode = cudaFilterModePoint;
//	texRef_f4A.filterMode = cudaFilterModePoint;
//	texRef_f5A.filterMode = cudaFilterModePoint;
//	texRef_f6A.filterMode = cudaFilterModePoint;
//	texRef_f7A.filterMode = cudaFilterModePoint;
//	texRef_f8A.filterMode = cudaFilterModePoint;
//	texRef_f9A.filterMode = cudaFilterModePoint;
//	texRef_f10A.filterMode = cudaFilterModePoint;
//	texRef_f11A.filterMode = cudaFilterModePoint;
//	texRef_f12A.filterMode = cudaFilterModePoint;
//	texRef_f13A.filterMode = cudaFilterModePoint;
//	texRef_f14A.filterMode = cudaFilterModePoint;
//	texRef_f15A.filterMode = cudaFilterModePoint;
//	texRef_f16A.filterMode = cudaFilterModePoint;
//	texRef_f17A.filterMode = cudaFilterModePoint;
//	texRef_f18A.filterMode = cudaFilterModePoint;
	}
	
	hipMemcpy2D(fA_d ,pitch,fA_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d ,pitch,fA_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim*19,hipMemcpyHostToDevice);

	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = 0;
	}



//	if(true)//mem copy host to dev
//	{
//	cudaMemcpy2D(f0_dA ,pitch,f0_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f1_dA ,pitch,f1_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f2_dA ,pitch,f2_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f3_dA ,pitch,f3_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f4_dA ,pitch,f4_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f5_dA ,pitch,f5_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f6_dA ,pitch,f6_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f7_dA ,pitch,f7_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f8_dA ,pitch,f8_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f9_dA ,pitch,f9_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f10_dA,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f11_dA,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f12_dA,pitch,f12_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f13_dA,pitch,f13_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f14_dA,pitch,f14_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f15_dA,pitch,f15_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f16_dA,pitch,f16_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f17_dA,pitch,f17_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f18_dA,pitch,f18_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f0_dB ,pitch,f0_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f1_dB ,pitch,f1_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f2_dB ,pitch,f2_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f3_dB ,pitch,f3_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f4_dB ,pitch,f4_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f5_dB ,pitch,f5_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f6_dB ,pitch,f6_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f7_dB ,pitch,f7_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f8_dB ,pitch,f8_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f9_dB ,pitch,f9_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f10_dB,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f11_dB,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f12_dB,pitch,f12_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f13_dB,pitch,f13_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f14_dB,pitch,f14_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f15_dB,pitch,f15_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f16_dB,pitch,f16_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f17_dB,pitch,f17_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	cudaMemcpy2D(f18_dB,pitch,f18_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,cudaMemcpyHostToDevice);
//	}
//	if(true)//bind texture
//	{
//	cudaBindTexture2D(0,&texRef_f0A, f0_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f1A, f1_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f2A, f2_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f3A, f3_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f4A, f4_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f5A, f5_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f6A, f6_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f7A, f7_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f8A, f8_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f9A, f9_dA ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f10A,f10_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f11A,f11_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f12A,f12_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f13A,f13_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f14A,f14_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f15A,f15_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f16A,f16_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f17A,f17_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f18A,f18_dA,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f0B, f0_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f1B, f1_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f2B, f2_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f3B, f3_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f4B, f4_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f5B, f5_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f6B, f6_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f7B, f7_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f8B, f8_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f9B, f9_dB ,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f10B,f10_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f11B,f11_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f12B,f12_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f13B,f13_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f14B,f14_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f15B,f15_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f16B,f16_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f17B,f17_dB,&desc,xDim,yDim*zDim,pitch);
//	cudaBindTexture2D(0,&texRef_f18B,f18_dB,&desc,xDim,yDim*zDim,pitch);
//	}
//
//
//	initialize<<<grid, threads>>>(f0_dA.ptr, f1_dA.ptr, f2_dA.ptr, f3_dA.ptr, f4_dA.ptr, f5_dA.ptr, f6_dA.ptr, f7_dA.ptr, f8_dA.ptr, f9_dA.ptr,
//									f10_dA.ptr, f11_dA.ptr, f12_dA.ptr, f13_dA.ptr, f14_dA.ptr, f15_dA.ptr, f16_dA.ptr, f17_dA.ptr, f18_dA.ptr,
//									xDim,yDim,pitch);
//	initialize<<<grid, threads>>>(f0_dA, f1_dA, f2_dA, f3_dA, f4_dA, f5_dA, f6_dA, f7_dA, f8_dA, f9_dA,
//									f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//									xDim,yDim,pitch_elements);

	initialize_single<<<grid, threads>>>(fA_d,xDim,yDim,zDim,pitch_elements);

//	cudaFuncSetCacheConfig(mrt_d_single,cudaFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<tMax; t=t+2){
	//for(int t = 0; t<tMax; t=t+1){
		//mrt_d<<<grid, threads>>>(f0_d,f1_d,f2_d,f3_d,f4_d,f5_d,f6_d,f7_d,f8_d,n,image_d,omega,uMax);
		//test<<<grid, threads>>>(f0_d,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,
//		mrt_d_textAB<<<grid, threads>>>(f0_dB,f1_dB,f2_dB,f3_dB,f4_dB,f5_dB,f6_dB,f7_dB,f8_dB,f9_dB,
//										f10_dB, f11_dB, f12_dB, f13_dB, f14_dB, f15_dB, f16_dB, f17_dB, f18_dB,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);
//
//		mrt_d_textBA<<<grid, threads>>>(f0_dA,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,f9_dA,
//										f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);
//
//
//		mrt_d_hybAB<<<grid, threads>>>(f0_dB,f1_dB,f2_dB,f3_dB,f4_dB,f5_dB,f6_dB,f7_dB,f8_dB,f9_dB,
//										f10_dB, f11_dB, f12_dB, f13_dB, f14_dB, f15_dB, f16_dB, f17_dB, f18_dB,
//										f2_dA,f4_dA,f9_dA,
//										f11_dA, f13_dA, f14_dA, f16_dA, f18_dA,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);
//
//		mrt_d_hybBA<<<grid, threads>>>(f0_dA,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,f9_dA,
//										f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//										f2_dB,f4_dB,f9_dB,
//										f11_dB, f13_dB, f14_dB, f16_dB, f18_dB,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);

//		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);
//		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);

		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);
		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);

		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(xDim*yDim*zDim*double(tMax/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<xDim<<","<<yDim<<","<<zDim<<","<<tMax<<","<<restime<<endl;




//	copytest<<<grid, threads>>>(f10_dA,test_d,xDim,yDim,zDim);
	//copytest<<<grid, threads>>>(test_d);
	//copytest<<<grid, threads>>>(image_d);
//	cudaUnbindTexture(texRef_f0A);
//	cudaUnbindTexture(texRef_f1A);
//	cudaUnbindTexture(texRef_f2A);
//	cudaUnbindTexture(texRef_f3A);
//	cudaUnbindTexture(texRef_f4A);
//	cudaUnbindTexture(texRef_f5A);
//	cudaUnbindTexture(texRef_f6A);
//	cudaUnbindTexture(texRef_f7A);
//	cudaUnbindTexture(texRef_f8A);
//	cudaUnbindTexture(texRef_f9A);
//	cudaUnbindTexture(texRef_f10A);
//	cudaUnbindTexture(texRef_f11A);
//	cudaUnbindTexture(texRef_f12A);
//	cudaUnbindTexture(texRef_f13A);
//	cudaUnbindTexture(texRef_f14A);
//	cudaUnbindTexture(texRef_f15A);
//	cudaUnbindTexture(texRef_f16A);
//	cudaUnbindTexture(texRef_f17A);
//	cudaUnbindTexture(texRef_f18A);
//	cudaUnbindTexture(texRef_f0B);
//	cudaUnbindTexture(texRef_f1B);
//	cudaUnbindTexture(texRef_f2B);
//	cudaUnbindTexture(texRef_f3B);
//	cudaUnbindTexture(texRef_f4B);
//	cudaUnbindTexture(texRef_f5B);
//	cudaUnbindTexture(texRef_f6B);
//	cudaUnbindTexture(texRef_f7B);
//	cudaUnbindTexture(texRef_f8B);
//	cudaUnbindTexture(texRef_f9B);
//	cudaUnbindTexture(texRef_f10B);
//	cudaUnbindTexture(texRef_f11B);
//	cudaUnbindTexture(texRef_f12B);
//	cudaUnbindTexture(texRef_f13B);
//	cudaUnbindTexture(texRef_f14B);
//	cudaUnbindTexture(texRef_f15B);
//	cudaUnbindTexture(texRef_f16B);
//	cudaUnbindTexture(texRef_f17B);
//	cudaUnbindTexture(texRef_f18B);

//	cudaMemcpy2D(f0_h,xDim*sizeof(float) , f0_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f1_h,xDim*sizeof(float) , f1_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f2_h,xDim*sizeof(float) , f2_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f3_h,xDim*sizeof(float) , f3_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f4_h,xDim*sizeof(float) , f4_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f5_h,xDim*sizeof(float) , f5_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f6_h,xDim*sizeof(float) , f6_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f7_h,xDim*sizeof(float) , f7_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f8_h,xDim*sizeof(float) , f8_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f9_h,xDim*sizeof(float) , f9_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f10_h,xDim*sizeof(float),f10_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f11_h,xDim*sizeof(float),f11_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f12_h,xDim*sizeof(float),f12_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f13_h,xDim*sizeof(float),f13_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f14_h,xDim*sizeof(float),f14_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f15_h,xDim*sizeof(float),f15_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f16_h,xDim*sizeof(float),f16_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f17_h,xDim*sizeof(float),f17_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);
//	cudaMemcpy2D(f18_h,xDim*sizeof(float),f18_dA,pitch,xDim*sizeof(float),yDim*zDim,cudaMemcpyDeviceToHost);

	hipMemcpy2D(fA_h,xDim*sizeof(float),fA_d,pitch,xDim*sizeof(float),yDim*zDim*19,hipMemcpyDeviceToHost);

//	cout<<"f1_h is "<<f1_h[0]<<endl;
	//cudaMemcpy(f0_h, f0_d.ptr, memsize, cudaMemcpyDeviceToHost);

	hipMemcpy(image_h, image_d, memsize_int, hipMemcpyDeviceToHost);

//	cout<<image_h[0]<<endl;
//	cout<<"test_d: "<<test_h[0]<<endl;
//	for(i = 0; i<n; i++){
//	cout<<f0_h[i]<<",";
//	}


	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<xDim<<", J="<<yDim<<", K="<<zDim<<"\n";
	
	int row = 0;
	int col = 0;
	int dep = 0;
	i = 0;
	float rho, u, v, w;
	int j;

	for(dep = 0; dep<zDim; dep++){
	for(row = 0; row<yDim; row++){
		for(col = 0; col<xDim; col++){
			i = dep*xDim*yDim+row*xDim+col;
//			rho = 0;
			rho = fA_h[i];
			for(j = 1; j<19; j++)
				rho+=fA_h[i+xDim*yDim*zDim*j];
//			rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i]+f9_h[i]+
//			      f10_h[i]+f11_h[i]+f12_h[i]+f13_h[i]+f14_h[i]+f15_h[i]+f16_h[i]+f17_h[i]+f18_h[i];
			u = fA_h[i+xDim*yDim*zDim*1]-fA_h[i+xDim*yDim*zDim*3]+fA_h[i+xDim*yDim*zDim*5]-fA_h[i+xDim*yDim*zDim*6]-
				fA_h[i+xDim*yDim*zDim*7]+fA_h[i+xDim*yDim*zDim*8]+fA_h[i+xDim*yDim*zDim*10]-fA_h[i+xDim*yDim*zDim*12]
				+fA_h[i+xDim*yDim*zDim*15]-fA_h[i+xDim*yDim*zDim*17];
			v = fA_h[i+xDim*yDim*zDim*2]-fA_h[i+xDim*yDim*zDim*4]+fA_h[i+xDim*yDim*zDim*5]+fA_h[i+xDim*yDim*zDim*6]-fA_h[i+xDim*yDim*zDim*7]-fA_h[i+xDim*yDim*zDim*8]+fA_h[i+xDim*yDim*zDim*11]-fA_h[i+xDim*yDim*zDim*13]+fA_h[i+xDim*yDim*zDim*16]-fA_h[i+xDim*yDim*zDim*18];
			w = fA_h[i+xDim*yDim*zDim*9]+fA_h[i+xDim*yDim*zDim*10]+fA_h[i+xDim*yDim*zDim*11]+fA_h[i+xDim*yDim*zDim*12]+fA_h[i+xDim*yDim*zDim*13]-fA_h[i+xDim*yDim*zDim*14]-fA_h[i+xDim*yDim*zDim*15]-fA_h[i+xDim*yDim*zDim*16]-fA_h[i+xDim*yDim*zDim*17]-fA_h[i+xDim*yDim*zDim*18];
//			output<<col<<", "<<row<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;
			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;
//			output<<row<<", "<<col<<", "<<dep<<", "<<u<<","<<v<<","<<fA_h[i+xDim*yDim*zDim*4]<<","<<rho<<endl;
		}
	}
	}
	output.close();




	hipFree(image_d);
//	cudaFree(f0_dA);
//	cudaFree(f1_dA);
//	cudaFree(f2_dA);
//	cudaFree(f3_dA);
//	cudaFree(f4_dA);
//	cudaFree(f5_dA);
//	cudaFree(f6_dA);
//	cudaFree(f7_dA);
//	cudaFree(f8_dA);
//	cudaFree(f9_dA);
//	cudaFree(f10_dA);
//	cudaFree(f11_dA);
//	cudaFree(f12_dA);
//	cudaFree(f13_dA);
//	cudaFree(f14_dA);
//	cudaFree(f15_dA);
//	cudaFree(f16_dA);
//	cudaFree(f17_dA);
//	cudaFree(f18_dA);
//	cudaFree(f0_dB);
//	cudaFree(f1_dB);
//	cudaFree(f2_dB);
//	cudaFree(f3_dB);
//	cudaFree(f4_dB);
//	cudaFree(f5_dB);
//	cudaFree(f6_dB);
//	cudaFree(f7_dB);
//	cudaFree(f8_dB);
//	cudaFree(f9_dB);
//	cudaFree(f10_dB);
//	cudaFree(f11_dB);
//	cudaFree(f12_dB);
//	cudaFree(f13_dB);
//	cudaFree(f14_dB);
//	cudaFree(f15_dB);
//	cudaFree(f16_dB);
//	cudaFree(f17_dB);
//	cudaFree(f18_dB);

	hipFree(fA_d);
	hipFree(fB_d);
	return(0);

}
