#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;

//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

//int Xcoord(int i, int XDim){
//	int B = BLOCKSIZE*BLOCKSIZE;
//	return (i%B)%BLOCKSIZE+((i/B)%(XDim/BLOCKSIZE))*BLOCKSIZE;
//}
//int Ycoord(int i, int XDim){
//	int B = BLOCKSIZE*BLOCKSIZE;
//	return (i%B)/BLOCKSIZE+((i/B)/(XDim/BLOCKSIZE))*BLOCKSIZE;
//}

//texture
//texture<float,1,hipReadModeElementType> texRef_f1;
texture<float,2,hipReadModeElementType> texRef_f0A;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;
texture<float,2,hipReadModeElementType> texRef_f9A;
texture<float,2,hipReadModeElementType> texRef_f10A;
texture<float,2,hipReadModeElementType> texRef_f11A;
texture<float,2,hipReadModeElementType> texRef_f12A;
texture<float,2,hipReadModeElementType> texRef_f13A;
texture<float,2,hipReadModeElementType> texRef_f14A;
texture<float,2,hipReadModeElementType> texRef_f15A;
texture<float,2,hipReadModeElementType> texRef_f16A;
texture<float,2,hipReadModeElementType> texRef_f17A;
texture<float,2,hipReadModeElementType> texRef_f18A;

texture<float,2,hipReadModeElementType> texRef_f0B;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;
texture<float,2,hipReadModeElementType> texRef_f9B;
texture<float,2,hipReadModeElementType> texRef_f10B;
texture<float,2,hipReadModeElementType> texRef_f11B;
texture<float,2,hipReadModeElementType> texRef_f12B;
texture<float,2,hipReadModeElementType> texRef_f13B;
texture<float,2,hipReadModeElementType> texRef_f14B;
texture<float,2,hipReadModeElementType> texRef_f15B;
texture<float,2,hipReadModeElementType> texRef_f16B;
texture<float,2,hipReadModeElementType> texRef_f17B;
texture<float,2,hipReadModeElementType> texRef_f18B;

#include <sys/time.h>
#include <time.h>



int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}


//__global__ void test(float *f0A, float *f1A, float *f2A,
//						   float *f3A, float *f4A, float *f5A,
//						   float *f6A, float *f7A, float *f8A,
//						               float *f1B, float *f2B,
//						   float *f3B, float *f4B, float *f5B,
//						   float *f6B, float *f7B, float *f8B, //int pitch)
//						   int n, int *image, float omega, float uMax, int pitch)
//{
//	int x = blockIdx.x*blockDim.x+threadIdx.x;
//	int y = blockIdx.y*blockDim.y+threadIdx.y;
////	int i = x+y*blockDim.x*gridDim.x;
////	j = threadIdx.y*blockDim.x+threadIdx.x;//local block index (for shared mem)
////	i = j+blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
////	i += blockDim.x*blockDim.y*blockIdx.x;//global memory index
//
//		//f0A[i] = f;
//		float f1,f2,f3,f4,f5,f6,f7,f8;
////		f1=tex2D(texRef_f1,x+1,y  );
////		f2=tex2D(texRef_f2,x-1,y  );
////		f3=tex2D(texRef_f3,x  ,y+1);
////		f4=tex2D(texRef_f4,x  ,y-1);
////		f5=tex2D(texRef_f5,x+1,y+1);
////		f7=tex2D(texRef_f7,x-1,y-1);
////		f6=tex2D(texRef_f6,x-1,y+1);
////		f8=tex2D(texRef_f8,x+1,y-1);
//		//if(image[i] == 0){
//
////		f1 = tex2D(texRef_f1,x-1,y  );
////		f2 = tex2D(texRef_f2,x  ,y-1);
////		f3 = tex2D(texRef_f3,x+1,y  );
////		f4 = tex2D(texRef_f4,x  ,y+1);
////		f5 = tex2D(texRef_f5,x-1,y-1);
////		f6 = tex2D(texRef_f6,x+1,y-1);
////		f7 = tex2D(texRef_f7,x+1,y+1);
////		f8 = tex2D(texRef_f8,x-1,y+1);
//
////		f1 = f1A[x-1+y*pitch];
////		f2 = f2A[x  +(y-1)*pitch];
////		f3 = f3A[x+1+y*pitch];
////		f4 = f4A[x  +(y+1)*pitch];
////		f5 = f5A[x-1+(y-1)*pitch];
////		f6 = f6A[x+1+(y-1)*pitch];
////		f7 = f7A[x+1+(y+1)*pitch];
////		f8 = f8A[x-1+(y+1)*pitch];
//
////		f1=f1A[x+1+(y  )*pitch];
////		f2=f2A[x-1+(y  )*pitch];
////		f3=f3A[x  +(y+1)*pitch];
////		f4=f4A[x  +(y-1)*pitch];
////		f5=f5A[x+1+(y+1)*pitch];
////		f7=f7A[x-1+(y-1)*pitch];
////		f6=f6A[x-1+(y+1)*pitch];
////		f8=f8A[x+1+(y-1)*pitch];
//		f1=f1A[x+y*pitch];
//		f2=f2A[x+y*pitch];
//		f3=f3A[x+y*pitch];
//		f4=f4A[x+y*pitch];
//		f5=f5A[x+y*pitch];
//		f6=f6A[x+y*pitch];
//		f7=f7A[x+y*pitch];
//		f8=f8A[x+y*pitch];
////		f1B[x+y*pitch]=f1+1;
////		f3B[x+y*pitch]=f3+1;
////		f2B[x+y*pitch]=f2+1;
////		f4B[x+y*pitch]=f4+1;
////		f5B[x+y*pitch]=f5+1;
////		f6B[x+y*pitch]=f6+1;
////		f7B[x+y*pitch]=f7+1;
////		f8B[x+y*pitch]=f8+1;
//		//}
////		f1=f1A[x+y*pitch];
////		f3=f3A[x+y*pitch];
////		f2=f2A[x+y*pitch];
////		f4=f4A[x+y*pitch];
////		f5=f5A[x+y*pitch];
////		f7=f7A[x+y*pitch];
////		f6=f6A[x+y*pitch];
////		f8=f8A[x+y*pitch];
//		f1B[x+y*pitch]=f1;
//		f2B[x+y*pitch]=f2;
//		f3B[x+y*pitch]=f3;
//		f4B[x+y*pitch]=f4;
//		f5B[x+y*pitch]=f5;
//		f6B[x+y*pitch]=f6;
//		f7B[x+y*pitch]=f7;
//		f8B[x+y*pitch]=f8;
//
//
//}
//__global__ void mrt_d_textAB(float *f0A, float *f1A, float *f2A,
//						   float *f3A, float *f4A, float *f5A,
//						   float *f6A, float *f7A, float *f8A,
//						               float *f1B, float *f2B,
//						   float *f3B, float *f4B, float *f5B,
//						   float *f6B, float *f7B, float *f8B,
//						   int n, int *image, float omega, float uMax, int pitch)

__device__ void collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
		rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
		      f10+f11+f12+f13+f14+f15+f16+f17+f18;
		u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
		v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
		w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

		float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

		m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
		m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+  1.f*f8+ -4.f*f9+    f10+  1.f*f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
		m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
		m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
		m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
		m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
		m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
		m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
		m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
		m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
		m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
		m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
		m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
		m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
		m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;

		m1 -= -11.f*rho+19.f*(u*u+v*v+w*w);
		m2 -= -475.f/63.f*(u*u+v*v+w*w);
		m4 -= -2.f/3.f*u;//qx_eq
		m6 -= -2.f/3.f*v;//qx_eq
		m8 -= -2.f/3.f*w;//qx_eq
		m9 -= (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
		m10-= 0.f;//.f.f;//.f.5*meq[9];/.f.f;//.f.5*meq[9];/.f.f;//pixx
		m11-= (v*v-w*w);//pww_eq
		m12-= 0.f;//.f.f;//.f.5*meq[11];/.f.f;//.f.5*meq[9];/.f.f;//piww
		m13-= u*v;//pxy_eq
		m14-= v*w;//pyz_eq
		m15-= u*w;//pxz_eq
		m16-= 0.0;//mx_eq
		m17-= 0.0;//my_eq
		m18-= 0.0;//mz_eq


f0  -= - 0.012531328f*(m1)+  0.047619048f*(m2);
f1  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+   -0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
f2  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +   -0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f3  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+    0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
f4  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +    0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f5  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f6  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f7  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f8  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f9  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +   -0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f10 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f11 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f12 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f13 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f14 -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +    0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f15 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f16 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f17 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f18 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);

f2  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
f4  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
f5  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
f6  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
f7  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
f8  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
f9  -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
f10 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
f11 -=                                                                         +(                         0.25f*(m14)                        )*omega ;
f12 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
f13 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
f14 -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
f15 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
f16 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
f17 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
f18 -=                                                                         +(                         0.25f*(m14)                        )*omega ;

f5  -=  0.125f*(m16)+ -0.125f*(m17);                        
f6  -= -0.125f*(m16)+ -0.125f*(m17);                        
f7  -= -0.125f*(m16)+  0.125f*(m17);                        
f8  -=  0.125f*(m16)+  0.125f*(m17);                        
f10 -= -0.125f*(m16)                              +  0.125f*(m18);
f11 -=                             +  0.125f*(m17)+ -0.125f*(m18);
f12 -=  0.125f*(m16)                              +  0.125f*(m18);
f13 -=                             + -0.125f*(m17)+ -0.125f*(m18);
f15 -= -0.125f*(m16)                              + -0.125f*(m18);
f16 -=                             +  0.125f*(m17)+  0.125f*(m18);
f17 -=  0.125f*(m16)                              + -0.125f*(m18);
f18 -=                             + -0.125f*(m17)+  0.125f*(m18);






}



__global__ void mrt_d_textAB(float* f0out, float* f1out, float* f2out,
							float* f3out, float* f4out, float* f5out,
							float* f6out, float* f7out, float* f8out, float* f9out,
							float* f10out, float* f11out, float* f12out,
							float* f13out, float* f14out, float* f15out,
							float* f16out, float* f17out, float* f18out,
							int *image, float omega, float uMax,
							int width, int height, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
//	int i = x+y*blockDim.x*gridDim.x;
	//float u,v,w,rho;//,usqr;
	int im = image[i];
	if(im == 1){//BB
		//f0A[i] = f0A[i];
		f0out[j] = tex2D(texRef_f0A,x  ,(y  )+height*(z));
		f1out[j] = tex2D(texRef_f3A,x+1,(y  )+height*(z));
		f3out[j] = tex2D(texRef_f1A,x-1,(y  )+height*(z));
		f2out[j] = tex2D(texRef_f4A,x  ,(y+1)+height*(z));
		f4out[j] = tex2D(texRef_f2A,x  ,(y-1)+height*(z));
		f5out[j] = tex2D(texRef_f7A,x+1,(y+1)+height*(z));
		f7out[j] = tex2D(texRef_f5A,x-1,(y-1)+height*(z));
		f6out[j] = tex2D(texRef_f8A,x-1,(y+1)+height*(z));
		f8out[j] = tex2D(texRef_f6A,x+1,(y-1)+height*(z));
		f9out[j] = tex2D(texRef_f14A,x  ,(y  )+height*(z+1));
		f14out[j]= tex2D(texRef_f9A ,x  ,(y  )+height*(z-1));
		f10out[j]= tex2D(texRef_f17A,x+1,(y  )+height*(z+1));
		f17out[j]= tex2D(texRef_f10A,x-1,(y  )+height*(z-1));
		f11out[j]= tex2D(texRef_f18A,x  ,(y+1)+height*(z+1));
		f18out[j]= tex2D(texRef_f11A,x  ,(y-1)+height*(z-1));
		f12out[j]= tex2D(texRef_f15A,x-1,(y  )+height*(z+1));
		f15out[j]= tex2D(texRef_f12A,x+1,(y  )+height*(z-1));
		f13out[j]= tex2D(texRef_f16A,x  ,(y-1)+height*(z+1));
		f16out[j]= tex2D(texRef_f13A,x  ,(y+1)+height*(z-1));

	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		//float u,v,w,rho;
//		float meq1,meq2,meq4,meq6,meq7,meq8;
//		f0 = f0A[i];

		f0 = tex2D(texRef_f0A ,x  ,y  +height*(z));
		f1 = tex2D(texRef_f1A ,x-1,y  +height*(z));
		f2 = tex2D(texRef_f2A ,x  ,y-1+height*(z));
		f3 = tex2D(texRef_f3A ,x+1,y  +height*(z));
		f4 = tex2D(texRef_f4A ,x  ,y+1+height*(z));
		f5 = tex2D(texRef_f5A ,x-1,y-1+height*(z));
		f6 = tex2D(texRef_f6A ,x+1,y-1+height*(z));
		f7 = tex2D(texRef_f7A ,x+1,y+1+height*(z));
		f8 = tex2D(texRef_f8A ,x-1,y+1+height*(z));
		f9 = tex2D(texRef_f9A ,x  ,y  +height*(z-1));
		f10= tex2D(texRef_f10A,x-1,y  +height*(z-1));
		f11= tex2D(texRef_f11A,x  ,y-1+height*(z-1));
		f12= tex2D(texRef_f12A,x+1,y  +height*(z-1));
		f13= tex2D(texRef_f13A,x  ,y+1+height*(z-1));
		f14= tex2D(texRef_f14A,x  ,y  +height*(z+1));
		f15= tex2D(texRef_f15A,x-1,y  +height*(z+1));
		f16= tex2D(texRef_f16A,x  ,y-1+height*(z+1));
		f17= tex2D(texRef_f17A,x+1,y  +height*(z+1));
		f18= tex2D(texRef_f18A,x  ,y+1+height*(z+1));

		collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		f0out[j] =  f0 ;
		f1out[j] = f1 ;
		f2out[j] = f2 ;
		f3out[j] = f3 ;
		f4out[j] = f4 ;
		f5out[j] = f5 ;
		f6out[j] = f6 ;
		f7out[j] = f7 ;
		f8out[j] = f8 ;
		f9out[j] = f9 ;
		f10out[j]= f10;
		f11out[j]= f11;
		f12out[j]= f12;
		f13out[j]= f13;
		f14out[j]= f14;
		f15out[j]= f15;
		f16out[j]= f16;
		f17out[j]= f17;
		f18out[j]= f18;


	}
}

__global__ void mrt_d_textBA(float* f0out, float* f1out, float* f2out,
							float* f3out, float* f4out, float* f5out,
							float* f6out, float* f7out, float* f8out, float* f9out,
							float* f10out, float* f11out, float* f12out,
							float* f13out, float* f14out, float* f15out,
							float* f16out, float* f17out, float* f18out,
							int *image, float omega, float uMax,
							int width, int height, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
//	int i = x+y*blockDim.x*gridDim.x;
	//float u,v,w,rho;//,usqr;
	int im = image[i];
	if(im == 1){//BB
		//f0A[i] = f0A[i];
		f0out[j] = tex2D(texRef_f0B,x  ,(y  )+height*(z));
		f1out[j] = tex2D(texRef_f3B,x+1,(y  )+height*(z));
		f3out[j] = tex2D(texRef_f1B,x-1,(y  )+height*(z));
		f2out[j] = tex2D(texRef_f4B,x  ,(y+1)+height*(z));
		f4out[j] = tex2D(texRef_f2B,x  ,(y-1)+height*(z));
		f5out[j] = tex2D(texRef_f7B,x+1,(y+1)+height*(z));
		f7out[j] = tex2D(texRef_f5B,x-1,(y-1)+height*(z));
		f6out[j] = tex2D(texRef_f8B,x-1,(y+1)+height*(z));
		f8out[j] = tex2D(texRef_f6B,x+1,(y-1)+height*(z));
		f9out[j] = tex2D(texRef_f14B,x  ,(y  )+height*(z+1));
		f14out[j]= tex2D(texRef_f9B ,x  ,(y  )+height*(z-1));
		f10out[j]= tex2D(texRef_f17B,x+1,(y  )+height*(z+1));
		f17out[j]= tex2D(texRef_f10B,x-1,(y  )+height*(z-1));
		f11out[j]= tex2D(texRef_f18B,x  ,(y+1)+height*(z+1));
		f18out[j]= tex2D(texRef_f11B,x  ,(y-1)+height*(z-1));
		f12out[j]= tex2D(texRef_f15B,x-1,(y  )+height*(z+1));
		f15out[j]= tex2D(texRef_f12B,x+1,(y  )+height*(z-1));
		f13out[j]= tex2D(texRef_f16B,x  ,(y-1)+height*(z+1));
		f16out[j]= tex2D(texRef_f13B,x  ,(y+1)+height*(z-1));


	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		//float u,v,w,rho;
//		float meq1,meq2,meq4,meq6,meq7,meq8;
//		f0 = f0A[i];

		f0 = tex2D(texRef_f0B ,x  ,y  +height*(z));
		f1 = tex2D(texRef_f1B ,x-1,y  +height*(z));
		f2 = tex2D(texRef_f2B ,x  ,y-1+height*(z));
		f3 = tex2D(texRef_f3B ,x+1,y  +height*(z));
		f4 = tex2D(texRef_f4B ,x  ,y+1+height*(z));
		f5 = tex2D(texRef_f5B ,x-1,y-1+height*(z));
		f6 = tex2D(texRef_f6B ,x+1,y-1+height*(z));
		f7 = tex2D(texRef_f7B ,x+1,y+1+height*(z));
		f8 = tex2D(texRef_f8B ,x-1,y+1+height*(z));
		f9 = tex2D(texRef_f9B ,x  ,y  +height*(z-1));
		f10= tex2D(texRef_f10B,x-1,y  +height*(z-1));
		f11= tex2D(texRef_f11B,x  ,y-1+height*(z-1));
		f12= tex2D(texRef_f12B,x+1,y  +height*(z-1));
		f13= tex2D(texRef_f13B,x  ,y+1+height*(z-1));
		f14= tex2D(texRef_f14B,x  ,y  +height*(z+1));
		f15= tex2D(texRef_f15B,x-1,y  +height*(z+1));
		f16= tex2D(texRef_f16B,x  ,y-1+height*(z+1));
		f17= tex2D(texRef_f17B,x+1,y  +height*(z+1));
		f18= tex2D(texRef_f18B,x  ,y+1+height*(z+1));

		collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
	
		f0out[j] = f0 ;
		f1out[j] = f1 ;
		f2out[j] = f2 ;
		f3out[j] = f3 ;
		f4out[j] = f4 ;
		f5out[j] = f5 ;
		f6out[j] = f6 ;
		f7out[j] = f7 ;
		f8out[j] = f8 ;
		f9out[j] = f9 ;
		f10out[j]= f10;
		f11out[j]= f11;
		f12out[j]= f12;
		f13out[j]= f13;
		f14out[j]= f14;
		f15out[j]= f15;
		f16out[j]= f16;
		f17out[j]= f17;
		f18out[j]= f18;


	}
}
//

__global__ void initialize(float* f0, float* f1, float* f2,
							float* f3, float* f4, float* f5,
							float* f6, float* f7, float* f8, float* f9,
							float* f10, float* f11, float* f12,
							float* f13, float* f14, float* f15,
							float* f16, float* f17, float* f18,
							int width, int height, size_t pitch)//pitch in elements
//__global__ void initialize(void** f0in, void** f1in, 
//							int w, int h, int pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
	float u,v,w,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	feq = 1.0f/3.0f*(rho-1.5f*usqr);
	f0[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[j] = feq;

	feq = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f9[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f10[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f11[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f12[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f13[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f14[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f15[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f16[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f17[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
	f18[j] = feq;
}
__global__ void copytest(hipPitchedPtr devPitchedPtr, float * test_d, int w, int h, int d)
//__global__ void copytest(float *test)//, int w, int h, int d)
//__global__ void copytest(int * image)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	char* devPtr = (char*)devPitchedPtr.ptr;
	int pitch = devPitchedPtr.pitch;
//	int slicepitch = pitch*height;
////	int pitch = devPitchedPtr.pitch;
//	char *slice = devPtr + blockIdx.x*slicepitch;
	float* test = (float *)(devPtr);
//	//int slicePitch = pitch*extent.height;
	//int i = threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	int i = x+y*w+z*w*h;//index on linear mem
	//int j = threadIdx.x+threadIdx.y*pitch+threadIdx.z*blockDim.y;
	int j = x+y*pitch/sizeof(float)+z*h*pitch/sizeof(float);//index on padded mem
	//if(test[i] == 2)
	//test[0] = 2.f;//test[i];
	test_d[i] = test[j];
	test[j] += 100;

}



int main(int argc, char *argv[])
{

	float *f0_h, *f1_h, *f2_h, *f3_h, *f4_h, *f5_h, *f6_h, *f7_h, *f8_h, *f9_h;
	float *f10_h, *f11_h, *f12_h, *f13_h, *f14_h, *f15_h, *f16_h, *f17_h, *f18_h;
	float *f0_dA, *f1_dA, *f2_dA, *f3_dA, *f4_dA, *f5_dA, *f6_dA, *f7_dA, *f8_dA, *f9_dA;
	float *f10_dA, *f11_dA, *f12_dA, *f13_dA, *f14_dA, *f15_dA, *f16_dA, *f17_dA, *f18_dA;
	float *f0_dB, *f1_dB, *f2_dB, *f3_dB, *f4_dB, *f5_dB, *f6_dB, *f7_dB, *f8_dB, *f9_dB;
	float *f10_dB, *f11_dB, *f12_dB, *f13_dB, *f14_dB, *f15_dB, *f16_dB, *f17_dB, *f18_dB;
	int *image_d, *image_h;

	//hipPitchedPtr f0_d;

	float *test_h,*test_d;

	ofstream output;
	output.open ("LBM1_out.dat");

	size_t memsize, memsize_int;
	size_t pitch;
	int i, n, nBlocks, xDim, yDim, zDim,tMax;
	float Re, omega, uMax, CharLength;
	
	int BLOCKSIZEx = 256;
	int BLOCKSIZEy = 1;
	int BLOCKSIZEz = 1;
	xDim = 256;
	yDim = 256;
	zDim = 4;
	tMax = 100;
	Re = 500.f;//100.f;
	uMax = 0.08f;
	CharLength = xDim-2.f;
	omega = 1.0f/(3.0f*(uMax*CharLength/Re)+0.5f);

	cout<<"omega: "<<omega<<endl;

	nBlocks = (xDim/BLOCKSIZEx+xDim%BLOCKSIZEx)*(yDim/BLOCKSIZEy+yDim%BLOCKSIZEy)
				*(zDim/BLOCKSIZEz+zDim%BLOCKSIZEz);
	int B = BLOCKSIZEx*BLOCKSIZEy*BLOCKSIZEz;
	n = nBlocks*B;//block*dimx*dimy
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEx, BLOCKSIZEy, BLOCKSIZEz);
    dim3 grid(xDim/BLOCKSIZEx,yDim/BLOCKSIZEy,zDim/BLOCKSIZEz);

	memsize = n*sizeof(float);
	memsize_int = n*sizeof(int);

	hipExtent extent = make_hipExtent(xDim*sizeof(float),yDim,zDim);

	f0_h   = (float *)malloc(memsize);
	f1_h   = (float *)malloc(memsize);
	f2_h   = (float *)malloc(memsize);
	f3_h   = (float *)malloc(memsize);
	f4_h   = (float *)malloc(memsize);
	f5_h   = (float *)malloc(memsize);
	f6_h   = (float *)malloc(memsize);
	f7_h   = (float *)malloc(memsize);
	f8_h   = (float *)malloc(memsize);
	f9_h   = (float *)malloc(memsize);
	f10_h  = (float *)malloc(memsize);
	f11_h  = (float *)malloc(memsize);
	f12_h  = (float *)malloc(memsize);
	f13_h  = (float *)malloc(memsize);
	f14_h  = (float *)malloc(memsize);
	f15_h  = (float *)malloc(memsize);
	f16_h  = (float *)malloc(memsize);
	f17_h  = (float *)malloc(memsize);
	f18_h  = (float *)malloc(memsize);

	image_h = (int *)malloc(memsize_int);
	test_h = (float *)malloc(memsize);

	hipMalloc((void **) &test_d, memsize);
	hipMalloc((void **) &image_d, memsize_int);
	hipMallocPitch((void **) &f0_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f1_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f2_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f3_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f4_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f5_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f6_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f7_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f8_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f9_dA , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f10_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f11_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f12_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f13_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f14_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f15_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f16_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f17_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f18_dA, &pitch, xDim*sizeof(float), yDim*zDim);
	cout<<pitch<<endl;
	hipMallocPitch((void **) &f0_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f1_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f2_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f3_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f4_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f5_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f6_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f7_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f8_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f9_dB , &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f10_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f11_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f12_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f13_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f14_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f15_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f16_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f17_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	hipMallocPitch((void **) &f18_dB, &pitch, xDim*sizeof(float), yDim*zDim);
	cout<<pitch<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	for (i = 0; i < n; i++)
	{
		int x = i%xDim;
		int y = (i/xDim)%yDim;
		int z = (i/xDim)/yDim;
		f0_h[i]  = i;
		f1_h[i]  = i;
		f2_h[i]  = i;
		f3_h[i]  = i;
		f4_h[i]  = i;
		f5_h[i]  = i;
		f6_h[i]  = i;
		f7_h[i]  = i;
		f8_h[i]  = i;
		f9_h[i]  = i;
		f10_h[i] = i;
		f11_h[i] = i;
		f12_h[i] = i;
		f13_h[i] = i;
		f14_h[i] = i;
		f15_h[i] = i;
		f16_h[i] = i;
		f17_h[i] = i;
		f18_h[i] = i;
		image_h[i] = 0;
		test_h[i] = 0;
		if(x < 1) image_h[i] = 1;//DirichletWest
		if(x > xDim-2) image_h[i] = 1;//BB
		if(y < 1) image_h[i] = 1;//BB
		if(y > yDim-2) image_h[i] = 1;//BB
//		if(z < 1) image_h[i] = 1;//DirichletWest
//		if(z > zDim-2) image_h[i] = 1;//BB
	}
	hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);

	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;
	texRef_f9B.normalized = false;
	texRef_f10B.normalized = false;
	texRef_f11B.normalized = false;
	texRef_f12B.normalized = false;
	texRef_f13B.normalized = false;
	texRef_f14B.normalized = false;
	texRef_f15B.normalized = false;
	texRef_f16B.normalized = false;
	texRef_f17B.normalized = false;
	texRef_f18B.normalized = false;
	texRef_f0B.filterMode = hipFilterModePoint;
	texRef_f1B.filterMode = hipFilterModePoint;
	texRef_f2B.filterMode = hipFilterModePoint;
	texRef_f3B.filterMode = hipFilterModePoint;
	texRef_f4B.filterMode = hipFilterModePoint;
	texRef_f5B.filterMode = hipFilterModePoint;
	texRef_f6B.filterMode = hipFilterModePoint;
	texRef_f7B.filterMode = hipFilterModePoint;
	texRef_f8B.filterMode = hipFilterModePoint;
	texRef_f9B.filterMode = hipFilterModePoint;
	texRef_f10B.filterMode = hipFilterModePoint;
	texRef_f11B.filterMode = hipFilterModePoint;
	texRef_f12B.filterMode = hipFilterModePoint;
	texRef_f13B.filterMode = hipFilterModePoint;
	texRef_f14B.filterMode = hipFilterModePoint;
	texRef_f15B.filterMode = hipFilterModePoint;
	texRef_f16B.filterMode = hipFilterModePoint;
	texRef_f17B.filterMode = hipFilterModePoint;
	texRef_f18B.filterMode = hipFilterModePoint;
	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f9A.normalized = false;
	texRef_f10A.normalized = false;
	texRef_f11A.normalized = false;
	texRef_f12A.normalized = false;
	texRef_f13A.normalized = false;
	texRef_f14A.normalized = false;
	texRef_f15A.normalized = false;
	texRef_f16A.normalized = false;
	texRef_f17A.normalized = false;
	texRef_f18A.normalized = false;
	texRef_f0A.filterMode = hipFilterModePoint;
	texRef_f1A.filterMode = hipFilterModePoint;
	texRef_f2A.filterMode = hipFilterModePoint;
	texRef_f3A.filterMode = hipFilterModePoint;
	texRef_f4A.filterMode = hipFilterModePoint;
	texRef_f5A.filterMode = hipFilterModePoint;
	texRef_f6A.filterMode = hipFilterModePoint;
	texRef_f7A.filterMode = hipFilterModePoint;
	texRef_f8A.filterMode = hipFilterModePoint;
	texRef_f9A.filterMode = hipFilterModePoint;
	texRef_f10A.filterMode = hipFilterModePoint;
	texRef_f11A.filterMode = hipFilterModePoint;
	texRef_f12A.filterMode = hipFilterModePoint;
	texRef_f13A.filterMode = hipFilterModePoint;
	texRef_f14A.filterMode = hipFilterModePoint;
	texRef_f15A.filterMode = hipFilterModePoint;
	texRef_f16A.filterMode = hipFilterModePoint;
	texRef_f17A.filterMode = hipFilterModePoint;
	texRef_f18A.filterMode = hipFilterModePoint;
	}
	
	if(true)//mem copy host to dev
	{
	hipMemcpy2D(f0_dA ,pitch,f0_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f1_dA ,pitch,f1_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f2_dA ,pitch,f2_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f3_dA ,pitch,f3_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f4_dA ,pitch,f4_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f5_dA ,pitch,f5_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f6_dA ,pitch,f6_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f7_dA ,pitch,f7_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f8_dA ,pitch,f8_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f9_dA ,pitch,f9_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f10_dA,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f11_dA,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f12_dA,pitch,f12_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f13_dA,pitch,f13_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f14_dA,pitch,f14_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f15_dA,pitch,f15_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f16_dA,pitch,f16_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f17_dA,pitch,f17_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f18_dA,pitch,f18_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f0_dB ,pitch,f0_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f1_dB ,pitch,f1_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f2_dB ,pitch,f2_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f3_dB ,pitch,f3_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f4_dB ,pitch,f4_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f5_dB ,pitch,f5_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f6_dB ,pitch,f6_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f7_dB ,pitch,f7_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f8_dB ,pitch,f8_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f9_dB ,pitch,f9_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f10_dB,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f11_dB,pitch,f11_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f12_dB,pitch,f12_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f13_dB,pitch,f13_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f14_dB,pitch,f14_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f15_dB,pitch,f15_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f16_dB,pitch,f16_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f17_dB,pitch,f17_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f18_dB,pitch,f18_h,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim,hipMemcpyHostToDevice);
	}
//	if(true)//bind texture
//	{
//	hipBindTexture2D(0,&texRef_f0A, f0_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f1A, f1_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f2A, f2_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f3A, f3_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f4A, f4_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f5A, f5_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f6A, f6_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f7A, f7_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f8A, f8_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f9A, f9_dA ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f10A,f10_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f11A,f11_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f12A,f12_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f13A,f13_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f14A,f14_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f15A,f15_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f16A,f16_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f17A,f17_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f18A,f18_dA,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f0B, f0_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f1B, f1_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f2B, f2_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f3B, f3_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f4B, f4_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f5B, f5_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f6B, f6_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f7B, f7_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f8B, f8_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f9B, f9_dB ,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f10B,f10_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f11B,f11_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f12B,f12_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f13B,f13_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f14B,f14_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f15B,f15_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f16B,f16_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f17B,f17_dB,&desc,xDim,yDim*zDim,pitch);
//	hipBindTexture2D(0,&texRef_f18B,f18_dB,&desc,xDim,yDim*zDim,pitch);
//	}


//	initialize<<<grid, threads>>>(f0_dA.ptr, f1_dA.ptr, f2_dA.ptr, f3_dA.ptr, f4_dA.ptr, f5_dA.ptr, f6_dA.ptr, f7_dA.ptr, f8_dA.ptr, f9_dA.ptr,
//									f10_dA.ptr, f11_dA.ptr, f12_dA.ptr, f13_dA.ptr, f14_dA.ptr, f15_dA.ptr, f16_dA.ptr, f17_dA.ptr, f18_dA.ptr,
//									xDim,yDim,pitch);
	initialize<<<grid, threads>>>(f0_dA, f1_dA, f2_dA, f3_dA, f4_dA, f5_dA, f6_dA, f7_dA, f8_dA, f9_dA,
									f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
									xDim,yDim,pitch_elements);


	struct timeval tdr0,tdr1;
	double restime;
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<tMax; t=t+2){
	//for(int t = 0; t<tMax; t=t+1){
		//mrt_d<<<grid, threads>>>(f0_d,f1_d,f2_d,f3_d,f4_d,f5_d,f6_d,f7_d,f8_d,n,image_d,omega,uMax);
		//test<<<grid, threads>>>(f0_d,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,
//		mrt_d_textAB<<<grid, threads>>>(f0_dB,f1_dB,f2_dB,f3_dB,f4_dB,f5_dB,f6_dB,f7_dB,f8_dB,f9_dB,
//										f10_dB, f11_dB, f12_dB, f13_dB, f14_dB, f15_dB, f16_dB, f17_dB, f18_dB,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);
//
//		mrt_d_textBA<<<grid, threads>>>(f0_dA,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,f9_dA,
//										f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);
//		mrt_d_AB<<<grid, threads>>>(f0_dB,f1_dB,f2_dB,f3_dB,f4_dB,f5_dB,f6_dB,f7_dB,f8_dB,f9_dB,
//										f10_dB, f11_dB, f12_dB, f13_dB, f14_dB, f15_dB, f16_dB, f17_dB, f18_dB,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);
//
//		mrt_d_BA<<<grid, threads>>>(f0_dA,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,f9_dA,
//										f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//										image_d,omega,uMax,xDim,yDim,pitch_elements);





		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(xDim*yDim*zDim*double(tMax/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<xDim<<","<<yDim<<","<<xDim<<","<<tMax<<","<<restime<<endl;




//	copytest<<<grid, threads>>>(f10_dA,test_d,xDim,yDim,zDim);
//	//copytest<<<grid, threads>>>(test_d);
//	//copytest<<<grid, threads>>>(image_d);
//	hipUnbindTexture(texRef_f0A);
//	hipUnbindTexture(texRef_f1A);
//	hipUnbindTexture(texRef_f2A);
//	hipUnbindTexture(texRef_f3A);
//	hipUnbindTexture(texRef_f4A);
//	hipUnbindTexture(texRef_f5A);
//	hipUnbindTexture(texRef_f6A);
//	hipUnbindTexture(texRef_f7A);
//	hipUnbindTexture(texRef_f8A);
//	hipUnbindTexture(texRef_f9A);
//	hipUnbindTexture(texRef_f10A);
//	hipUnbindTexture(texRef_f11A);
//	hipUnbindTexture(texRef_f12A);
//	hipUnbindTexture(texRef_f13A);
//	hipUnbindTexture(texRef_f14A);
//	hipUnbindTexture(texRef_f15A);
//	hipUnbindTexture(texRef_f16A);
//	hipUnbindTexture(texRef_f17A);
//	hipUnbindTexture(texRef_f18A);
//	hipUnbindTexture(texRef_f0B);
//	hipUnbindTexture(texRef_f1B);
//	hipUnbindTexture(texRef_f2B);
//	hipUnbindTexture(texRef_f3B);
//	hipUnbindTexture(texRef_f4B);
//	hipUnbindTexture(texRef_f5B);
//	hipUnbindTexture(texRef_f6B);
//	hipUnbindTexture(texRef_f7B);
//	hipUnbindTexture(texRef_f8B);
//	hipUnbindTexture(texRef_f9B);
//	hipUnbindTexture(texRef_f10B);
//	hipUnbindTexture(texRef_f11B);
//	hipUnbindTexture(texRef_f12B);
//	hipUnbindTexture(texRef_f13B);
//	hipUnbindTexture(texRef_f14B);
//	hipUnbindTexture(texRef_f15B);
//	hipUnbindTexture(texRef_f16B);
//	hipUnbindTexture(texRef_f17B);
//	hipUnbindTexture(texRef_f18B);
//
	hipMemcpy2D(f0_h,xDim*sizeof(float) , f0_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f1_h,xDim*sizeof(float) , f1_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f2_h,xDim*sizeof(float) , f2_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f3_h,xDim*sizeof(float) , f3_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f4_h,xDim*sizeof(float) , f4_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f5_h,xDim*sizeof(float) , f5_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f6_h,xDim*sizeof(float) , f6_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f7_h,xDim*sizeof(float) , f7_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f8_h,xDim*sizeof(float) , f8_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f9_h,xDim*sizeof(float) , f9_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f10_h,xDim*sizeof(float),f10_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f11_h,xDim*sizeof(float),f11_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f12_h,xDim*sizeof(float),f12_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f13_h,xDim*sizeof(float),f13_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f14_h,xDim*sizeof(float),f14_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f15_h,xDim*sizeof(float),f15_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f16_h,xDim*sizeof(float),f16_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f17_h,xDim*sizeof(float),f17_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
	hipMemcpy2D(f18_h,xDim*sizeof(float),f18_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);

	cout<<"f1_h is "<<f1_h[0]<<endl;
	//hipMemcpy(f0_h, f0_d.ptr, memsize, hipMemcpyDeviceToHost);

	hipMemcpy(image_h, image_d, memsize_int, hipMemcpyDeviceToHost);
	hipMemcpy(test_h, test_d, memsize, hipMemcpyDeviceToHost);

//	cout<<image_h[0]<<endl;
//	cout<<"test_d: "<<test_h[0]<<endl;
//	for(i = 0; i<n; i++){
//	cout<<f0_h[i]<<",";
//	}


	output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"w\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<xDim<<", J="<<yDim*zDim<<"\n";
	
	int row = 0;
	int col = 0;
	i = 0;
	float rho, u, v, w;

	for(row = 0; row<yDim*zDim; row++){
		for(col = 0; col<xDim; col++){
			i = row*xDim+col;
			rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i]+f9_h[i]+
			      f10_h[i]+f11_h[i]+f12_h[i]+f13_h[i]+f14_h[i]+f15_h[i]+f16_h[i]+f17_h[i]+f18_h[i];
			u = f1_h[i]-f3_h[i]+f5_h[i]-f6_h[i]-f7_h[i]+f8_h[i]+f10_h[i]-f12_h[i]+f15_h[i]-f17_h[i];
			v = f2_h[i]-f4_h[i]+f5_h[i]+f6_h[i]-f7_h[i]-f8_h[i]+f11_h[i]-f13_h[i]+f16_h[i]-f18_h[i];
			w = f9_h[i]+f10_h[i]+f11_h[i]+f12_h[i]+f13_h[i]-f14_h[i]-f15_h[i]-f16_h[i]-f17_h[i]-f18_h[i];
			output<<col<<", "<<row<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;
		}
	}
	output.close();




	hipFree(image_d);
	hipFree(test_d);
	hipFree(f0_dA);
	hipFree(f1_dA);
	hipFree(f2_dA);
	hipFree(f3_dA);
	hipFree(f4_dA);
	hipFree(f5_dA);
	hipFree(f6_dA);
	hipFree(f7_dA);
	hipFree(f8_dA);
	hipFree(f9_dA);
	hipFree(f10_dA);
	hipFree(f11_dA);
	hipFree(f12_dA);
	hipFree(f13_dA);
	hipFree(f14_dA);
	hipFree(f15_dA);
	hipFree(f16_dA);
	hipFree(f17_dA);
	hipFree(f18_dA);
	hipFree(f0_dB);
	hipFree(f1_dB);
	hipFree(f2_dB);
	hipFree(f3_dB);
	hipFree(f4_dB);
	hipFree(f5_dB);
	hipFree(f6_dB);
	hipFree(f7_dB);
	hipFree(f8_dB);
	hipFree(f9_dB);
	hipFree(f10_dB);
	hipFree(f11_dB);
	hipFree(f12_dB);
	hipFree(f13_dB);
	hipFree(f14_dB);
	hipFree(f15_dB);
	hipFree(f16_dB);
	hipFree(f17_dB);
	hipFree(f18_dB);
	return(0);

}
