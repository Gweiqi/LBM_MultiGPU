#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;

//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>

texture<float,2,hipReadModeElementType> texRef_f0A;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;
texture<float,2,hipReadModeElementType> texRef_f9A;
texture<float,2,hipReadModeElementType> texRef_f10A;
texture<float,2,hipReadModeElementType> texRef_f11A;
texture<float,2,hipReadModeElementType> texRef_f12A;
texture<float,2,hipReadModeElementType> texRef_f13A;
texture<float,2,hipReadModeElementType> texRef_f14A;
texture<float,2,hipReadModeElementType> texRef_f15A;
texture<float,2,hipReadModeElementType> texRef_f16A;
texture<float,2,hipReadModeElementType> texRef_f17A;
texture<float,2,hipReadModeElementType> texRef_f18A;

texture<float,2,hipReadModeElementType> texRef_f0B;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;
texture<float,2,hipReadModeElementType> texRef_f9B;
texture<float,2,hipReadModeElementType> texRef_f10B;
texture<float,2,hipReadModeElementType> texRef_f11B;
texture<float,2,hipReadModeElementType> texRef_f12B;
texture<float,2,hipReadModeElementType> texRef_f13B;
texture<float,2,hipReadModeElementType> texRef_f14B;
texture<float,2,hipReadModeElementType> texRef_f15B;
texture<float,2,hipReadModeElementType> texRef_f16B;
texture<float,2,hipReadModeElementType> texRef_f17B;
texture<float,2,hipReadModeElementType> texRef_f18B;



int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//	float usqr = u*u+v*v+w*w;
	float usqr = fma(u,u,fma(v,v,w*w));

	f0 -= omega*fma(-0.3333333333f,(fma(-1.5f,usqr,rho)),f0);//(f0 -0.3333333333f*(fma(-1.5f,usqr,rho)));//rho-1.5f*usqr));
	f1 -= omega*fma(-0.0555555556f,fma(3.0f,    u ,rho)+fma(4.5f,u*u,-1.5f*usqr),f1);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 -= omega*fma(-0.0555555556f,fma(3.0f,    v ,rho)+fma(4.5f,v*v,-1.5f*usqr),f2);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f3 -= omega*fma(-0.0555555556f,fma(3.0f,    u ,rho)+fma(4.5f,u*u,-1.5f*usqr),f3);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f4 -= omega*fma(-0.0555555556f,fma(3.0f,    v ,rho)+fma(4.5f,v*v,-1.5f*usqr),f4);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f5 -= omega*fma(-0.0555555556f,fma(3.0f,( u+v),rho)+fma(4.5f,( u+v)*( u+v),-1.5f*usqr),f5 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f6 -= omega*fma(-0.0555555556f,fma(3.0f,(-u+v),rho)+fma(4.5f,(-u+v)*(-u+v),-1.5f*usqr),f6 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f7 -= omega*fma(-0.0555555556f,fma(3.0f,(-u-v),rho)+fma(4.5f,(-u-v)*(-u-v),-1.5f*usqr),f7 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f8 -= omega*fma(-0.0555555556f,fma(3.0f,( u-v),rho)+fma(4.5f,( u-v)*( u-v),-1.5f*usqr),f8 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f9 -= omega*fma(-0.0555555556f,fma(3.0f,(   w),rho)+fma(4.5f,(   w)*(   w),-1.5f*usqr),f9 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f10-= omega*fma(-0.0277777778f,fma(3.0f,( u+w),rho)+fma(4.5f,( u+w)*( u+w),-1.5f*usqr),f10);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f11-= omega*fma(-0.0277777778f,fma(3.0f,( v+w),rho)+fma(4.5f,( v+w)*( v+w),-1.5f*usqr),f11);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f12-= omega*fma(-0.0277777778f,fma(3.0f,(-u+w),rho)+fma(4.5f,(-u+w)*(-u+w),-1.5f*usqr),f12);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f13-= omega*fma(-0.0277777778f,fma(3.0f,(-v+w),rho)+fma(4.5f,(-v+w)*(-v+w),-1.5f*usqr),f13);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f14-= omega*fma(-0.0555555556f,fma(3.0f,(  -w),rho)+fma(4.5f,(  -w)*(  -w),-1.5f*usqr),f14);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f15-= omega*fma(-0.0277777778f,fma(3.0f,( u-w),rho)+fma(4.5f,( u-w)*( u-w),-1.5f*usqr),f15);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f16-= omega*fma(-0.0277777778f,fma(3.0f,( v-w),rho)+fma(4.5f,( v-w)*( v-w),-1.5f*usqr),f16);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f17-= omega*fma(-0.0277777778f,fma(3.0f,(-u-w),rho)+fma(4.5f,(-u-w)*(-u-w),-1.5f*usqr),f17);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f18-= omega*fma(-0.0277777778f,fma(3.0f,(-v-w),rho)+fma(4.5f,(-v-w)*(-v-w),-1.5f*usqr),f18);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	
//	f0 = f0 -omega*(f0 -0.3333333333f*(rho-1.5f*usqr));
//	f1 = f1 -omega*(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f2 = f2 -omega*(f2 -0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 = f3 -omega*(f3 -0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 = f4 -omega*(f4 -0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 = f5 -omega*(f5 -0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
//	f6 = f6 -omega*(f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 = f7 -omega*(f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 = f8 -omega*(f8 -0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
//	f9 = f9 -omega*(f9 -0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
//	f10= f10-omega*(f10-0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
//	f11= f11-omega*(f11-0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
//	f12= f12-omega*(f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
//	f13= f13-omega*(f13-0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
//	f14= f14-omega*(f14-0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
//	f15= f15-omega*(f15-0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
//	f16= f16-omega*(f16-0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
//	f17= f17-omega*(f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
//	f18= f18-omega*(f18-0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));


}


__device__ void mrt_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+  1.f*f8+ -4.f*f9+    f10+  1.f*f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;

	m1 -= -11.f*rho+19.f*(u*u+v*v+w*w);
	m2 -= -7.53968254f*(u*u+v*v+w*w);
	m4 -= -0.66666667f*u;//qx_eq
	m6 -= -0.66666667f*v;//qx_eq
	m8 -= -0.66666667f*w;//qx_eq
	m9 -= (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
	m11-= (v*v-w*w);//pww_eq
	m13-= u*v;//pxy_eq
	m14-= v*w;//pyz_eq
	m15-= u*w;//pxz_eq

f0  -= - 0.012531328f*(m1)+  0.047619048f*(m2);
f1  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+   -0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
f2  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +   -0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f3  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)+    0.1f*(m4)                                                      +  0.055555556f*(m9)*omega + -0.055555556f*(m10);
f4  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                           +    0.1f*(m6)                           + -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f5  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f6  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+  0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f7  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f8  -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)+ -0.025f*(m6)                           +  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f9  -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +   -0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f10 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f11 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f12 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           +  0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f13 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+  0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f14 -= -0.0045948204f*(m1)+ -0.015873016f*(m2)                                                      +    0.1f*(m8)+ -0.027777778f*(m9)*omega +  0.027777778f*(m10);
f15 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+  0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f16 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           +  0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);
f17 -=  0.0033416876f*(m1)+  0.003968254f*(m2)+ -0.025f*(m4)                           + -0.025f*(m8)+  0.027777778f*(m9)*omega +  0.013888889f*(m10);
f18 -=  0.0033416876f*(m1)+  0.003968254f*(m2)                           + -0.025f*(m6)+ -0.025f*(m8)+ -0.055555556f*(m9)*omega + -0.027777778f*(m10);

f2  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
f4  -=  0.083333333f*(m11)*omega + -0.083333333f*(m12);
f5  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
f6  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
f7  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ ( 0.25f*(m13)                                                )*omega; 
f8  -=  0.083333333f*(m11)*omega +  0.041666667f*(m12)+ (-0.25f*(m13)                                                )*omega; 
f9  -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
f10 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
f11 -=                                                                         +(                         0.25f*(m14)                        )*omega ;
f12 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
f13 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
f14 -= -0.083333333f*(m11)*omega +  0.083333333f*(m12);
f15 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              + -0.25f*(m15))*omega ;
f16 -=                                                                         +(                        -0.25f*(m14)                        )*omega ;
f17 -= -0.083333333f*(m11)*omega + -0.041666667f*(m12) +(                                              +  0.25f*(m15))*omega ;
f18 -=                                                                         +(                         0.25f*(m14)                        )*omega ;

f5  -=  0.125f*(m16)+ -0.125f*(m17);                        
f6  -= -0.125f*(m16)+ -0.125f*(m17);                        
f7  -= -0.125f*(m16)+  0.125f*(m17);                        
f8  -=  0.125f*(m16)+  0.125f*(m17);                        
f10 -= -0.125f*(m16)                              +  0.125f*(m18);
f11 -=                             +  0.125f*(m17)+ -0.125f*(m18);
f12 -=  0.125f*(m16)                              +  0.125f*(m18);
f13 -=                             + -0.125f*(m17)+ -0.125f*(m18);
f15 -= -0.125f*(m16)                              + -0.125f*(m18);
f16 -=                             +  0.125f*(m17)+  0.125f*(m18);
f17 -=  0.125f*(m16)                              + -0.125f*(m18);
f18 -=                             + -0.125f*(m17)+  0.125f*(m18);
}


__device__ int f_mem(int f_num, int x, int y, int z, size_t pitch, int height, int depth)
{
//	if (x<0 || x>pitch || y<0 || y>height || z<0 || z>depth) return 0;
//	else
		return (x+y*pitch+z*height*pitch)+f_num*pitch*height*depth;
}
__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}

__global__ void simple_copy(float* fA, float* fB,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)

//	fB[f_mem(1 ,x,y,z,pitch,height,depth)] = fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(2 ,x,y,z,pitch,height,depth)] = fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(3 ,x,y,z,pitch,height,depth)] = fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(4 ,x,y,z,pitch,height,depth)] = fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(5 ,x,y,z,pitch,height,depth)] = fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(6 ,x,y,z,pitch,height,depth)] = fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(7 ,x,y,z,pitch,height,depth)] = fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(8 ,x,y,z,pitch,height,depth)] = fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(9 ,x,y,z,pitch,height,depth)] = fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(10,x,y,z,pitch,height,depth)] = fA[f_mem(10,x,y,z,pitch,height,depth)];
//	fB[f_mem(11,x,y,z,pitch,height,depth)] = fA[f_mem(11,x,y,z,pitch,height,depth)];
//	fB[f_mem(12,x,y,z,pitch,height,depth)] = fA[f_mem(12,x,y,z,pitch,height,depth)];
//	fB[f_mem(13,x,y,z,pitch,height,depth)] = fA[f_mem(13,x,y,z,pitch,height,depth)];
//	fB[f_mem(14,x,y,z,pitch,height,depth)] = fA[f_mem(14,x,y,z,pitch,height,depth)];
//	fB[f_mem(15,x,y,z,pitch,height,depth)] = fA[f_mem(15,x,y,z,pitch,height,depth)];
//	fB[f_mem(16,x,y,z,pitch,height,depth)] = fA[f_mem(16,x,y,z,pitch,height,depth)];
//	fB[f_mem(17,x,y,z,pitch,height,depth)] = fA[f_mem(17,x,y,z,pitch,height,depth)];
//	fB[f_mem(18,x,y,z,pitch,height,depth)] = fA[f_mem(18,x,y,z,pitch,height,depth)];
	
//	float f0;//,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//	float f0  = fA[j+pitch*height*depth];
//	float f0  = fA[f_mem(0 ,x,y,z,pitch,height,depth)];
//	f0  = tex2D(texRef_f0A ,x,y+height*z);
//	f1  = tex2D(texRef_f1A ,x,y+height*z);
//	f2  = tex2D(texRef_f2A ,x,y+height*z);
//	f3  = tex2D(texRef_f3A ,x,y+height*z);
//	f4  = tex2D(texRef_f4A ,x,y+height*z);
//	f5  = tex2D(texRef_f5A ,x,y+height*z);
//	f6  = tex2D(texRef_f6A ,x,y+height*z);
//	f7  = tex2D(texRef_f7A ,x,y+height*z);
//	f8  = tex2D(texRef_f8A ,x,y+height*z);
//	f9  = tex2D(texRef_f9A ,x,y+height*z);
//	f10 = tex2D(texRef_f10A,x,y+height*z);
//	f11 = tex2D(texRef_f11A,x,y+height*z);
//	f12 = tex2D(texRef_f12A,x,y+height*z);
//	f13 = tex2D(texRef_f13A,x,y+height*z);
//	f14 = tex2D(texRef_f14A,x,y+height*z);
//	f15 = tex2D(texRef_f15A,x,y+height*z);
//	f16 = tex2D(texRef_f16A,x,y+height*z);
//	f17 = tex2D(texRef_f17A,x,y+height*z);
//	f18 = tex2D(texRef_f18A,x,y+height*z);
//	float f1  = fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//	f1  = fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//	f2  = fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//	f3  = fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//	f4  = fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//	f5  = fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//	f6  = fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//	f7  = fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//	f8  = fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//	f9  = fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//	f10 = fA[f_mem(10,x,y,z,pitch,height,depth)];
//	f11 = fA[f_mem(11,x,y,z,pitch,height,depth)];
//	f12 = fA[f_mem(12,x,y,z,pitch,height,depth)];
//	f13 = fA[f_mem(13,x,y,z,pitch,height,depth)];
//	f14 = fA[f_mem(14,x,y,z,pitch,height,depth)];
//	f15 = fA[f_mem(15,x,y,z,pitch,height,depth)];
//	f16 = fA[f_mem(16,x,y,z,pitch,height,depth)];
//	f17 = fA[f_mem(17,x,y,z,pitch,height,depth)];
//	f18 = fA[f_mem(18,x,y,z,pitch,height,depth)];

//	fB[f_mem(0 ,x,y,z,pitch,height,depth)] = fA[f_mem(0 ,x,y,z,pitch,height,depth)];//+0.01f;
	fB[j] = fA[j];//+0.01f;
//	fB[j+pitch*height*depth+pitch*height*depth] = f2;
//	fB[(x+y*pitch+z*height*pitch)+pitch*height*depth] = f1 ;//+0.01f;
//	fB[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
//	fB[f_mem(1 ,x,y,z,pitch,height,depth)] = f1;//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//	fB[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;//+0.01f;
//	fB[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;//+0.01f;
//	fB[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;//+0.01f;
//	fB[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;//+0.01f;
//	fB[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;//+0.01f;
//	fB[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;//+0.01f;
//	fB[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;//+0.01f;
//	fB[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;//+0.01f;
//	fB[f_mem(10,x,y,z,pitch,height,depth)] = f10;//+0.01f;
//	fB[f_mem(11,x,y,z,pitch,height,depth)] = f11;//+0.01f;
//	fB[f_mem(12,x,y,z,pitch,height,depth)] = f12;//+0.01f;
//	fB[f_mem(13,x,y,z,pitch,height,depth)] = f13;//+0.01f;
//	fB[f_mem(14,x,y,z,pitch,height,depth)] = f14;//+0.01f;
//	fB[f_mem(15,x,y,z,pitch,height,depth)] = f15;//+0.01f;
//	fB[f_mem(16,x,y,z,pitch,height,depth)] = f16;//+0.01f;
//	fB[f_mem(17,x,y,z,pitch,height,depth)] = f17;//+0.01f;
//	fB[f_mem(18,x,y,z,pitch,height,depth)] = f18;//+0.01f;

}

//int const blockx = 192;
//int const blocky = 1;

__global__ void mrt_d_hybAB(float* fin, float* fout,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
//	int i = x+y*blockDim.x*gridDim.x;
	//float u,v,w,rho;//,usqr;
	int im = image[i];
	if(im == 1){//BB
		float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//		f1 = fin[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];
//		f3 = fin[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];
//		f5 = fin[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];
//		f7 = fin[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		f6 = fin[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];
//		f8 = fin[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];
//		f10= fin[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		f12= fin[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		f15= fin[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];
//		f17= fin[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];
		f2 = fin[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];
		f4 = fin[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];
		f9 = fin[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
		f11= fin[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];
		f13= fin[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];
		f14= fin[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];
		f16= fin[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];
		f18= fin[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];
		f3 = tex2D(texRef_f1A ,x-1,(y  )+height*(z));
		f1 = tex2D(texRef_f3A ,x+1,(y  )+height*(z));
		f5 = tex2D(texRef_f7A ,x+1,(y+1)+height*(z));
		f6 = tex2D(texRef_f8A ,x-1,(y+1)+height*(z));
		f7 = tex2D(texRef_f5A ,x-1,(y-1)+height*(z));
		f8 = tex2D(texRef_f6A ,x+1,(y-1)+height*(z));
		f10= tex2D(texRef_f17A,x+1,(y  )+height*(z+1));
		f12= tex2D(texRef_f15A,x-1,(y  )+height*(z+1));
		f17= tex2D(texRef_f10A,x-1,(y  )+height*(z-1));
		f15= tex2D(texRef_f12A,x+1,(y  )+height*(z-1));
		fout[j+pitch*height*depth*1 ] = f1 ;
		fout[j+pitch*height*depth*2 ] = f2 ;
		fout[j+pitch*height*depth*3 ] = f3 ;
		fout[j+pitch*height*depth*4 ] = f4 ;
		fout[j+pitch*height*depth*5 ] = f5 ;
		fout[j+pitch*height*depth*6 ] = f6 ;
		fout[j+pitch*height*depth*7 ] = f7 ;
		fout[j+pitch*height*depth*8 ] = f8 ;
		fout[j+pitch*height*depth*9 ] = f9 ;
		fout[j+pitch*height*depth*10] = f10;
		fout[j+pitch*height*depth*11] = f11;
		fout[j+pitch*height*depth*12] = f12;
		fout[j+pitch*height*depth*13] = f13;
		fout[j+pitch*height*depth*14] = f14;
		fout[j+pitch*height*depth*15] = f15;
		fout[j+pitch*height*depth*16] = f16;
		fout[j+pitch*height*depth*17] = f17;
		fout[j+pitch*height*depth*18] = f18;
	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//		f1 = fin[f_mem(1 ,x-1,y  ,z  ,pitch,height,depth)];
//		f3 = fin[f_mem(3 ,x+1,y  ,z  ,pitch,height,depth)];
//		f5 = fin[f_mem(5 ,x-1,y-1,z  ,pitch,height,depth)];
//		f6 = fin[f_mem(6 ,x+1,y-1,z  ,pitch,height,depth)];
//		f7 = fin[f_mem(7 ,x+1,y+1,z  ,pitch,height,depth)];
//		f8 = fin[f_mem(8 ,x-1,y+1,z  ,pitch,height,depth)];
//		f10= fin[f_mem(10,x-1,y  ,z-1,pitch,height,depth)];
//		f12= fin[f_mem(12,x+1,y  ,z-1,pitch,height,depth)];
//		f15= fin[f_mem(15,x-1,y  ,z+1,pitch,height,depth)];
//		f17= fin[f_mem(17,x+1,y  ,z+1,pitch,height,depth)];

		f0 = fin[j];
		f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
		f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
		f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
		f11= fin[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
		f13= fin[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
		f14= fin[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
		f16= fin[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
		f18= fin[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];
		f1 = tex2D(texRef_f1A ,x-1,y  +height*(z));
		f3 = tex2D(texRef_f3A ,x+1,y  +height*(z));
		f5 = tex2D(texRef_f5A ,x-1,y-1+height*(z));
		f6 = tex2D(texRef_f6A ,x+1,y-1+height*(z));
		f7 = tex2D(texRef_f7A ,x+1,y+1+height*(z));
		f8 = tex2D(texRef_f8A ,x-1,y+1+height*(z));
		f15= tex2D(texRef_f15A,x-1,y  +height*(z+1));
		f17= tex2D(texRef_f17A,x+1,y  +height*(z+1));
		f10= tex2D(texRef_f10A,x-1,y  +height*(z-1));
		f12= tex2D(texRef_f12A,x+1,y  +height*(z-1));

		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == height-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			if(z == depth-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
//			float fInt1,fInt2;//,fDiff;
			float u,v,w,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = uMax;//0.0;
			w = 0.0f;
	        
//			fInt1 = f0+f2+f4+f9+f11+f13+f14+f16+f18;
//	        fInt2 = f3+f6+f7+f12+f17;
//        	rho = u+(fInt1+2.0f*fInt2); //D2Q9i
			
        	rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			float usqr = u*u+v*v+w*w;
			f1 = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr)+f3-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr)+f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr)+f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr)+f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr)+f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		//mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fout[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fout[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fout[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fout[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fout[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fout[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fout[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fout[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fout[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fout[f_mem(18,x,y,z,pitch,height,depth)] = f18;
	}
}
__global__ void mrt_d_hybBA(float* fin, float* fout,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
	int im = image[i];
	if(im == 1){//BB
		float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f2 = fin[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];
		f4 = fin[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];
		f9 = fin[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
		f11= fin[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];
		f13= fin[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];
		f14= fin[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];
		f16= fin[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];
		f18= fin[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];
		f3 = tex2D(texRef_f1B ,x-1,(y  )+height*(z));
		f1 = tex2D(texRef_f3B ,x+1,(y  )+height*(z));
		f5 = tex2D(texRef_f7B ,x+1,(y+1)+height*(z));
		f6 = tex2D(texRef_f8B ,x-1,(y+1)+height*(z));
		f7 = tex2D(texRef_f5B ,x-1,(y-1)+height*(z));
		f8 = tex2D(texRef_f6B ,x+1,(y-1)+height*(z));
		f10= tex2D(texRef_f17B,x+1,(y  )+height*(z+1));
		f12= tex2D(texRef_f15B,x-1,(y  )+height*(z+1));
		f17= tex2D(texRef_f10B,x-1,(y  )+height*(z-1));
		f15= tex2D(texRef_f12B,x+1,(y  )+height*(z-1));
		fout[j+pitch*height*depth*1 ] = f1 ;
		fout[j+pitch*height*depth*2 ] = f2 ;
		fout[j+pitch*height*depth*3 ] = f3 ;
		fout[j+pitch*height*depth*4 ] = f4 ;
		fout[j+pitch*height*depth*5 ] = f5 ;
		fout[j+pitch*height*depth*6 ] = f6 ;
		fout[j+pitch*height*depth*7 ] = f7 ;
		fout[j+pitch*height*depth*8 ] = f8 ;
		fout[j+pitch*height*depth*9 ] = f9 ;
		fout[j+pitch*height*depth*10] = f10;
		fout[j+pitch*height*depth*11] = f11;
		fout[j+pitch*height*depth*12] = f12;
		fout[j+pitch*height*depth*13] = f13;
		fout[j+pitch*height*depth*14] = f14;
		fout[j+pitch*height*depth*15] = f15;
		fout[j+pitch*height*depth*16] = f16;
		fout[j+pitch*height*depth*17] = f17;
		fout[j+pitch*height*depth*18] = f18;
	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f0 = fin[j];
		f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
		f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
		f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
		f11= fin[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
		f13= fin[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
		f14= fin[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
		f16= fin[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
		f18= fin[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];
		f1 = tex2D(texRef_f1B ,x-1,y  +height*(z));
		f3 = tex2D(texRef_f3B ,x+1,y  +height*(z));
		f5 = tex2D(texRef_f5B ,x-1,y-1+height*(z));
		f6 = tex2D(texRef_f6B ,x+1,y-1+height*(z));
		f7 = tex2D(texRef_f7B ,x+1,y+1+height*(z));
		f8 = tex2D(texRef_f8B ,x-1,y+1+height*(z));
		f15= tex2D(texRef_f15B,x-1,y  +height*(z+1));
		f17= tex2D(texRef_f17B,x+1,y  +height*(z+1));
		f10= tex2D(texRef_f10B,x-1,y  +height*(z-1));
		f12= tex2D(texRef_f12B,x+1,y  +height*(z-1));

		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == height-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			if(z == depth-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
//			float fInt1,fInt2;//,fDiff;
			float u,v,w,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = uMax;//0.0;
			w = 0.0f;
	        
//			fInt1 = f0+f2+f4+f9+f11+f13+f14+f16+f18;
//	        fInt2 = f3+f6+f7+f12+f17;
//        	rho = u+(fInt1+2.0f*fInt2); //D2Q9i
			
        	rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			float usqr = u*u+v*v+w*w;
			f1 = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr)+f3-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr)+f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr)+f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr)+f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr)+f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		//mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fout[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fout[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fout[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fout[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fout[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fout[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fout[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fout[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fout[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fout[f_mem(18,x,y,z,pitch,height,depth)] = f18;
	}
}

__global__ void mrt_d_textAB(float* fin, float* fout,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
	int im = image[i];
	if(im == 1){//BB
		float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f2 = tex2D(texRef_f4A ,x  ,(y+1)+height*(z  ));
		f4 = tex2D(texRef_f2A ,x  ,(y-1)+height*(z  ));
		f9 = tex2D(texRef_f14A,x  ,(y  )+height*(z+1));
		f14= tex2D(texRef_f9A ,x  ,(y  )+height*(z-1));
		f11= tex2D(texRef_f18A,x  ,(y+1)+height*(z+1));
		f18= tex2D(texRef_f11A,x  ,(y-1)+height*(z-1));
		f16= tex2D(texRef_f13A,x  ,(y+1)+height*(z-1));
		f13= tex2D(texRef_f16A,x  ,(y-1)+height*(z+1));

//		f2 = fin[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];
//		f4 = fin[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		f9 = fin[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		f11= fin[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];
//		f13= fin[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];
//		f14= fin[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		f16= fin[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];
//		f18= fin[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];
		f3 = tex2D(texRef_f1A ,x-1,(y  )+height*(z));
		f1 = tex2D(texRef_f3A ,x+1,(y  )+height*(z));
		f5 = tex2D(texRef_f7A ,x+1,(y+1)+height*(z));
		f6 = tex2D(texRef_f8A ,x-1,(y+1)+height*(z));
		f7 = tex2D(texRef_f5A ,x-1,(y-1)+height*(z));
		f8 = tex2D(texRef_f6A ,x+1,(y-1)+height*(z));
		f10= tex2D(texRef_f17A,x+1,(y  )+height*(z+1));
		f12= tex2D(texRef_f15A,x-1,(y  )+height*(z+1));
		f17= tex2D(texRef_f10A,x-1,(y  )+height*(z-1));
		f15= tex2D(texRef_f12A,x+1,(y  )+height*(z-1));
		fout[j+pitch*height*depth*1 ] = f1 ;
		fout[j+pitch*height*depth*2 ] = f2 ;
		fout[j+pitch*height*depth*3 ] = f3 ;
		fout[j+pitch*height*depth*4 ] = f4 ;
		fout[j+pitch*height*depth*5 ] = f5 ;
		fout[j+pitch*height*depth*6 ] = f6 ;
		fout[j+pitch*height*depth*7 ] = f7 ;
		fout[j+pitch*height*depth*8 ] = f8 ;
		fout[j+pitch*height*depth*9 ] = f9 ;
		fout[j+pitch*height*depth*10] = f10;
		fout[j+pitch*height*depth*11] = f11;
		fout[j+pitch*height*depth*12] = f12;
		fout[j+pitch*height*depth*13] = f13;
		fout[j+pitch*height*depth*14] = f14;
		fout[j+pitch*height*depth*15] = f15;
		fout[j+pitch*height*depth*16] = f16;
		fout[j+pitch*height*depth*17] = f17;
		fout[j+pitch*height*depth*18] = f18;
	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f0 = fin[j];
		f2 = tex2D(texRef_f2A ,x  ,y-1+height*(z));
		f4 = tex2D(texRef_f4A ,x  ,y+1+height*(z));
		f9 = tex2D(texRef_f9A ,x  ,y+1+height*(z-1));
		f11= tex2D(texRef_f11A,x  ,y-1+height*(z-1));
		f13= tex2D(texRef_f13A,x  ,y+1+height*(z-1));
		f14= tex2D(texRef_f14A,x  ,y  +height*(z+1));
		f16= tex2D(texRef_f16A,x  ,y-1+height*(z+1));
		f18= tex2D(texRef_f18A,x  ,y+1+height*(z+1));
//		f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
//		f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
//		f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
//		f11= fin[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
//		f13= fin[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
//		f14= fin[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
//		f16= fin[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
//		f18= fin[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];
		f1 = tex2D(texRef_f1A ,x-1,y  +height*(z));
		f3 = tex2D(texRef_f3A ,x+1,y  +height*(z));
		f5 = tex2D(texRef_f5A ,x-1,y-1+height*(z));
		f6 = tex2D(texRef_f6A ,x+1,y-1+height*(z));
		f7 = tex2D(texRef_f7A ,x+1,y+1+height*(z));
		f8 = tex2D(texRef_f8A ,x-1,y+1+height*(z));
		f15= tex2D(texRef_f15A,x-1,y  +height*(z+1));
		f17= tex2D(texRef_f17A,x+1,y  +height*(z+1));
		f10= tex2D(texRef_f10A,x-1,y  +height*(z-1));
		f12= tex2D(texRef_f12A,x+1,y  +height*(z-1));

		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == height-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			if(z == depth-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
//			float fInt1,fInt2;//,fDiff;
			float u,v,w,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = uMax;//0.0;
			w = 0.0f;
	        
//			fInt1 = f0+f2+f4+f9+f11+f13+f14+f16+f18;
//	        fInt2 = f3+f6+f7+f12+f17;
//        	rho = u+(fInt1+2.0f*fInt2); //D2Q9i
			
        	rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			float usqr = u*u+v*v+w*w;
			f1 = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr)+f3-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr)+f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr)+f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr)+f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr)+f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		//mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fout[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fout[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fout[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fout[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fout[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fout[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fout[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fout[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fout[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fout[f_mem(18,x,y,z,pitch,height,depth)] = f18;
	}
}

__global__ void mrt_d_textBA(float* fin, float* fout,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
	int im = image[i];
	if(im == 1){//BB
		float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f2 = tex2D(texRef_f4B ,x  ,(y+1)+height*(z  ));
		f4 = tex2D(texRef_f2B ,x  ,(y-1)+height*(z  ));
		f9 = tex2D(texRef_f14B,x  ,(y  )+height*(z+1));
		f14= tex2D(texRef_f9B ,x  ,(y  )+height*(z-1));
		f11= tex2D(texRef_f18B,x  ,(y+1)+height*(z+1));
		f18= tex2D(texRef_f11B,x  ,(y-1)+height*(z-1));
		f16= tex2D(texRef_f13B,x  ,(y+1)+height*(z-1));
		f13= tex2D(texRef_f16B,x  ,(y-1)+height*(z+1));

//		f2 = fin[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];
//		f4 = fin[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];
//		f9 = fin[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];
//		f11= fin[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];
//		f13= fin[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];
//		f14= fin[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];
//		f16= fin[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];
//		f18= fin[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];
		f3 = tex2D(texRef_f1B ,x-1,(y  )+height*(z));
		f1 = tex2D(texRef_f3B ,x+1,(y  )+height*(z));
		f5 = tex2D(texRef_f7B ,x+1,(y+1)+height*(z));
		f6 = tex2D(texRef_f8B ,x-1,(y+1)+height*(z));
		f7 = tex2D(texRef_f5B ,x-1,(y-1)+height*(z));
		f8 = tex2D(texRef_f6B ,x+1,(y-1)+height*(z));
		f10= tex2D(texRef_f17B,x+1,(y  )+height*(z+1));
		f12= tex2D(texRef_f15B,x-1,(y  )+height*(z+1));
		f17= tex2D(texRef_f10B,x-1,(y  )+height*(z-1));
		f15= tex2D(texRef_f12B,x+1,(y  )+height*(z-1));
		fout[j+pitch*height*depth*1 ] = f1 ;
		fout[j+pitch*height*depth*2 ] = f2 ;
		fout[j+pitch*height*depth*3 ] = f3 ;
		fout[j+pitch*height*depth*4 ] = f4 ;
		fout[j+pitch*height*depth*5 ] = f5 ;
		fout[j+pitch*height*depth*6 ] = f6 ;
		fout[j+pitch*height*depth*7 ] = f7 ;
		fout[j+pitch*height*depth*8 ] = f8 ;
		fout[j+pitch*height*depth*9 ] = f9 ;
		fout[j+pitch*height*depth*10] = f10;
		fout[j+pitch*height*depth*11] = f11;
		fout[j+pitch*height*depth*12] = f12;
		fout[j+pitch*height*depth*13] = f13;
		fout[j+pitch*height*depth*14] = f14;
		fout[j+pitch*height*depth*15] = f15;
		fout[j+pitch*height*depth*16] = f16;
		fout[j+pitch*height*depth*17] = f17;
		fout[j+pitch*height*depth*18] = f18;
	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f0 = fin[j];
		f2 = tex2D(texRef_f2B ,x  ,y-1+height*(z));
		f4 = tex2D(texRef_f4B ,x  ,y+1+height*(z));
		f9 = tex2D(texRef_f9B ,x  ,y+1+height*(z-1));
		f11= tex2D(texRef_f11B,x  ,y-1+height*(z-1));
		f13= tex2D(texRef_f13B,x  ,y+1+height*(z-1));
		f14= tex2D(texRef_f14B,x  ,y  +height*(z+1));
		f16= tex2D(texRef_f16B,x  ,y-1+height*(z+1));
		f18= tex2D(texRef_f18B,x  ,y+1+height*(z+1));
//		f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
//		f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
//		f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
//		f11= fin[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
//		f13= fin[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
//		f14= fin[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
//		f16= fin[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
//		f18= fin[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];
		f1 = tex2D(texRef_f1B ,x-1,y  +height*(z));
		f3 = tex2D(texRef_f3B ,x+1,y  +height*(z));
		f5 = tex2D(texRef_f5B ,x-1,y-1+height*(z));
		f6 = tex2D(texRef_f6B ,x+1,y-1+height*(z));
		f7 = tex2D(texRef_f7B ,x+1,y+1+height*(z));
		f8 = tex2D(texRef_f8B ,x-1,y+1+height*(z));
		f15= tex2D(texRef_f15B,x-1,y  +height*(z+1));
		f17= tex2D(texRef_f17B,x+1,y  +height*(z+1));
		f10= tex2D(texRef_f10B,x-1,y  +height*(z-1));
		f12= tex2D(texRef_f12B,x+1,y  +height*(z-1));

		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == height-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			if(z == depth-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
//			float fInt1,fInt2;//,fDiff;
			float u,v,w,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = uMax;//0.0;
			w = 0.0f;
	        
//			fInt1 = f0+f2+f4+f9+f11+f13+f14+f16+f18;
//	        fInt2 = f3+f6+f7+f12+f17;
//        	rho = u+(fInt1+2.0f*fInt2); //D2Q9i
			
        	rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			float usqr = u*u+v*v+w*w;
			f1 = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr)+f3-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr)+f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr)+f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr)+f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr)+f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		//mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fout[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fout[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fout[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fout[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fout[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fout[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fout[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fout[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fout[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fout[f_mem(18,x,y,z,pitch,height,depth)] = f18;
	}
}


__global__ void mrt_d_shared(float* fA, float* fB,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
//	int i = x+y*blockDim.x*gridDim.x;
	//float u,v,w,rho;//,usqr;
//	int im = image[i];
	int im = 0;
	if(y == 0 || z == 0 || x == width-1 || y == height-1 || z == depth-1) im = 1;
	else if (x == 0) im = 3;

	__shared__ float  f1_s[256];
	__shared__ float  f3_s[256];
	__shared__ float  f5_s[256];
	__shared__ float  f7_s[256];
	__shared__ float  f6_s[256];
	__shared__ float  f8_s[256];
	__shared__ float f10_s[256];
	__shared__ float f12_s[256];
	__shared__ float f15_s[256];
	__shared__ float f17_s[256];

	 f1_s[threadIdx.x] = fA[f_mem(1 ,x ,y  ,z               ,pitch,height,depth)];//dmax(x-1)      
	 f3_s[threadIdx.x] = fA[f_mem(3 ,x ,y  ,z               ,pitch,height,depth)];//dmin(x+1,width)
	if(y != 0){
	 f5_s[threadIdx.x] = fA[f_mem(5 ,x ,y-1,z               ,pitch,height,depth)];//dmax(x-1)      
	 f8_s[threadIdx.x] = fA[f_mem(8 ,x ,y-1,z               ,pitch,height,depth)];//dmax(x-1)      
	}
	else if(y != height){
	 f7_s[threadIdx.x] = fA[f_mem(7 ,x ,y+1,z               ,pitch,height,depth)];//dmin(x+1,width)
	 f6_s[threadIdx.x] = fA[f_mem(6 ,x ,y+1,z               ,pitch,height,depth)];//dmin(x+1,width)
	}
	if(z != 0){
	f10_s[threadIdx.x] = fA[f_mem(10,x ,y  ,z-1,pitch,height,depth)];//dmax(x-1)      
	f12_s[threadIdx.x] = fA[f_mem(12,x ,y  ,z-1,pitch,height,depth)];//dmin(x+1,width)
	}
	else if(z != depth-1){
	f15_s[threadIdx.x] = fA[f_mem(15,x ,y  ,z+1,pitch,height,depth)];//dmax(x-1)      
	f17_s[threadIdx.x] = fA[f_mem(17,x ,y  ,z+1,pitch,height,depth)];//dmin(x+1,width)
	}

//	 f1_s[threadIdx.x] = fA[f_mem(1 ,x ,y               ,z               ,pitch,height,depth)];//dmax(x-1)      
//	 f3_s[threadIdx.x] = fA[f_mem(3 ,x ,y               ,z               ,pitch,height,depth)];//dmin(x+1,width)
//	 f5_s[threadIdx.x] = fA[f_mem(5 ,x ,dmax(y-1)       ,z               ,pitch,height,depth)];//dmax(x-1)      
//	 f7_s[threadIdx.x] = fA[f_mem(7 ,x ,dmin(y+1,height),z               ,pitch,height,depth)];//dmin(x+1,width)
//	 f6_s[threadIdx.x] = fA[f_mem(6 ,x ,dmin(y+1,height),z               ,pitch,height,depth)];//dmin(x+1,width)
//	 f8_s[threadIdx.x] = fA[f_mem(8 ,x ,dmax(y-1)       ,z               ,pitch,height,depth)];//dmax(x-1)      
//	f10_s[threadIdx.x] = fA[f_mem(10,x ,y               ,dmax(z-1)       ,pitch,height,depth)];//dmax(x-1)      
//	f12_s[threadIdx.x] = fA[f_mem(12,x ,y               ,dmax(z-1)       ,pitch,height,depth)];//dmin(x+1,width)
//	f15_s[threadIdx.x] = fA[f_mem(15,x ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//dmax(x-1)      
//	f17_s[threadIdx.x] = fA[f_mem(17,x ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//dmin(x+1,width)
//
	__syncthreads();

	if(im == 1){//BB
		float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		if(y != 0){
		f4 = fA[f_mem(2 ,x  ,y-1  ,z  ,pitch,height,depth)];//fA[f_mem(4 ,x,y,z,pitch,height,depth)];
		}
		else if(y != height){
		f2 = fA[f_mem(4 ,x  ,y+1  ,z  ,pitch,height,depth)];//fA[f_mem(2 ,x,y,z,pitch,height,depth)];
		}
		if(z != depth-1){
		f9 = fA[f_mem(14,x  ,y    ,z+1,pitch,height,depth)];//fA[f_mem(9 ,x,y,z,pitch,height,depth)];
		f11= fA[f_mem(18,x  ,y+1  ,z+1,pitch,height,depth)];//fA[f_mem(11,x,y,z,pitch,height,depth)];
		f13= fA[f_mem(16,x  ,y-1  ,z+1,pitch,height,depth)];//fA[f_mem(13,x,y,z,pitch,height,depth)];
		}
		else if(z != 0){
		f14= fA[f_mem(9 ,x  ,y    ,z-1,pitch,height,depth)];//fA[f_mem(14,x,y,z,pitch,height,depth)];
		f16= fA[f_mem(13,x  ,y+1  ,z-1,pitch,height,depth)];//fA[f_mem(16,x,y,z,pitch,height,depth)];
		f18= fA[f_mem(11,x  ,y-1  ,z-1,pitch,height,depth)];//fA[f_mem(18,x,y,z,pitch,height,depth)];
		}

//		f2 = fA[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(2 ,x,y,z,pitch,height,depth)];
//		f4 = fA[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(4 ,x,y,z,pitch,height,depth)];
//		f9 = fA[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(9 ,x,y,z,pitch,height,depth)];
//		f11= fA[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(11,x,y,z,pitch,height,depth)];
//		f13= fA[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(13,x,y,z,pitch,height,depth)];
//		f14= fA[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(14,x,y,z,pitch,height,depth)];
//		f16= fA[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(16,x,y,z,pitch,height,depth)];
//		f18= fA[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(18,x,y,z,pitch,height,depth)];

//		f1 = fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f3 = fA[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];//fA[f_mem(3 ,x,y,z,pitch,height,depth)];
//		f5 = fA[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(5 ,x,y,z,pitch,height,depth)];
//		f7 = fA[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(7 ,x,y,z,pitch,height,depth)];
//		f6 = fA[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(6 ,x,y,z,pitch,height,depth)];
//		f8 = fA[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(8 ,x,y,z,pitch,height,depth)];
//		f10= fA[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(10,x,y,z,pitch,height,depth)];
//		f12= fA[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(12,x,y,z,pitch,height,depth)];
//		f15= fA[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(15,x,y,z,pitch,height,depth)];
//		f17= fA[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(17,x,y,z,pitch,height,depth)];

		if(threadIdx.x != width-1){
		f1 = f3_s[threadIdx.x+1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f5 = f7_s[threadIdx.x+1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f8 = f6_s[threadIdx.x+1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f10=f17_s[threadIdx.x+1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f15=f12_s[threadIdx.x+1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		}
		else if(threadIdx.x != 0){
		f3 = f1_s[threadIdx.x-1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f7 = f5_s[threadIdx.x-1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f6 = f8_s[threadIdx.x-1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f17=f10_s[threadIdx.x-1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f12=f15_s[threadIdx.x-1];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		}


//		f1 = f3_s[dmin(threadIdx.x+1,width)];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f3 = f1_s[dmax(threadIdx.x-1      )];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f5 = f7_s[dmin(threadIdx.x+1,width)];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f7 = f5_s[dmax(threadIdx.x-1      )];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f6 = f8_s[dmax(threadIdx.x-1      )];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f8 = f6_s[dmin(threadIdx.x+1,width)];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f10=f17_s[dmin(threadIdx.x+1,width)];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f17=f10_s[dmax(threadIdx.x-1      )];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f12=f15_s[dmax(threadIdx.x-1      )];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
//		f15=f12_s[dmin(threadIdx.x+1,width)];//fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];

		fB[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fB[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fB[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fB[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fB[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fB[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fB[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fB[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fB[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fB[f_mem(18,x,y,z,pitch,height,depth)] = f18;

//		fB[j+pitch*height*depth*1 ] = f1 ;
//		fB[j+pitch*height*depth*2 ] = f2 ;
//		fB[j+pitch*height*depth*3 ] = f3 ;
//		fB[j+pitch*height*depth*4 ] = f4 ;
//		fB[j+pitch*height*depth*5 ] = f5 ;
//		fB[j+pitch*height*depth*6 ] = f6 ;
//		fB[j+pitch*height*depth*7 ] = f7 ;
//		fB[j+pitch*height*depth*8 ] = f8 ;
//		fB[j+pitch*height*depth*9 ] = f9 ;
//		fB[j+pitch*height*depth*10] = f10;
//		fB[j+pitch*height*depth*11] = f11;
//		fB[j+pitch*height*depth*12] = f12;
//		fB[j+pitch*height*depth*13] = f13;
//		fB[j+pitch*height*depth*14] = f14;
//		fB[j+pitch*height*depth*15] = f15;
//		fB[j+pitch*height*depth*16] = f16;
//		fB[j+pitch*height*depth*17] = f17;
//		fB[j+pitch*height*depth*18] = f18;

	}
	else{


		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

		f0 = fA[j];
		if(y != 0){
		f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
		}
		else if(y != height-1){
		f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
		}
		if(z != depth-1){
		f14= fA[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
		f16= fA[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
		f18= fA[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];
		}
		else if(z != 0){
		f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
		f11= fA[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
		f13= fA[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
		}

		if(threadIdx.x != width-1){
		f3 = f3_s[threadIdx.x+1];//fA[f_mem(3 ,x+1,y  ,z  ,pitch,height,depth)];
		f6 = f6_s[threadIdx.x+1];//fA[f_mem(6 ,x+1,y-1,z  ,pitch,height,depth)];
		f7 = f7_s[threadIdx.x+1];//fA[f_mem(7 ,x+1,y+1,z  ,pitch,height,depth)];
		f12=f12_s[threadIdx.x+1];//fA[f_mem(12,x+1,y  ,z-1,pitch,height,depth)];
		f17=f17_s[threadIdx.x+1];//fA[f_mem(17,x+1,y  ,z+1,pitch,height,depth)];
		}
		else if(threadIdx.x != 0){
		f1 = f1_s[threadIdx.x-1];//fA[f_mem(1 ,x-1,y  ,z  ,pitch,height,depth)];
		f5 = f5_s[threadIdx.x-1];//fA[f_mem(5 ,x-1,y-1,z  ,pitch,height,depth)];
		f8 = f8_s[threadIdx.x-1];//fA[f_mem(8 ,x-1,y+1,z  ,pitch,height,depth)];
		f10=f10_s[threadIdx.x-1];//fA[f_mem(10,x-1,y  ,z-1,pitch,height,depth)];
		f15=f15_s[threadIdx.x-1];//fA[f_mem(15,x-1,y  ,z+1,pitch,height,depth)];
		}


//		f0 = fA[j];
//		f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
//		f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
//		f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
//		f11= fA[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
//		f13= fA[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
//		f14= fA[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
//		f16= fA[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
//		f18= fA[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];
//
//		f1 = f1_s[dmax(threadIdx.x-1      )];//fA[f_mem(1 ,x-1,y  ,z  ,pitch,height,depth)];
//		f3 = f3_s[dmin(threadIdx.x+1,width)];//fA[f_mem(3 ,x+1,y  ,z  ,pitch,height,depth)];
//		f5 = f5_s[dmax(threadIdx.x-1      )];//fA[f_mem(5 ,x-1,y-1,z  ,pitch,height,depth)];
//		f6 = f6_s[dmin(threadIdx.x+1,width)];//fA[f_mem(6 ,x+1,y-1,z  ,pitch,height,depth)];
//		f7 = f7_s[dmin(threadIdx.x+1,width)];//fA[f_mem(7 ,x+1,y+1,z  ,pitch,height,depth)];
//		f8 = f8_s[dmax(threadIdx.x-1      )];//fA[f_mem(8 ,x-1,y+1,z  ,pitch,height,depth)];
//		f10=f10_s[dmax(threadIdx.x-1      )];//fA[f_mem(10,x-1,y  ,z-1,pitch,height,depth)];
//		f12=f12_s[dmin(threadIdx.x+1,width)];//fA[f_mem(12,x+1,y  ,z-1,pitch,height,depth)];
//		f15=f15_s[dmax(threadIdx.x-1      )];//fA[f_mem(15,x-1,y  ,z+1,pitch,height,depth)];
//		f17=f17_s[dmin(threadIdx.x+1,width)];//fA[f_mem(17,x+1,y  ,z+1,pitch,height,depth)];

		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == height-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			if(z == depth-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
//			float fInt1,fInt2;//,fDiff;
			float u,v,w,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = uMax;//0.0;
			w = 0.0f;
	        
//			fInt1 = f0+f2+f4+f9+f11+f13+f14+f16+f18;
//	        fInt2 = f3+f6+f7+f12+f17;
//        	rho = u+(fInt1+2.0f*fInt2); //D2Q9i
			
        	rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			float usqr = u*u+v*v+w*w;
			f1 = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr)+f3-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr)+f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr)+f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr)+f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr)+f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		//mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fB[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fB[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fB[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fB[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fB[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fB[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fB[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fB[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fB[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fB[f_mem(18,x,y,z,pitch,height,depth)] = f18;
	}
}




__global__ void mrt_d_single(float* fA, float* fB,
							int *image, float omega, float uMax,
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
//	int i = x+y*blockDim.x*gridDim.x;
	//float u,v,w,rho;//,usqr;
	int im = image[i];
	if(im == 1){//BB
		float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
		f1 = fA[f_mem(3 ,dmin(x+1,width),y               ,z               ,pitch,height,depth)];//fA[f_mem(1 ,x,y,z,pitch,height,depth)];
		f2 = fA[f_mem(4 ,x              ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(2 ,x,y,z,pitch,height,depth)];
		f3 = fA[f_mem(1 ,dmax(x-1)      ,y               ,z               ,pitch,height,depth)];//fA[f_mem(3 ,x,y,z,pitch,height,depth)];
		f4 = fA[f_mem(2 ,x              ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(4 ,x,y,z,pitch,height,depth)];
		f5 = fA[f_mem(7 ,dmin(x+1,width),dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(5 ,x,y,z,pitch,height,depth)];
		f7 = fA[f_mem(5 ,dmax(x-1)      ,dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(7 ,x,y,z,pitch,height,depth)];
		f6 = fA[f_mem(8 ,dmax(x-1)      ,dmin(y+1,height),z               ,pitch,height,depth)];//fA[f_mem(6 ,x,y,z,pitch,height,depth)];
		f8 = fA[f_mem(6 ,dmin(x+1,width),dmax(y-1)       ,z               ,pitch,height,depth)];//fA[f_mem(8 ,x,y,z,pitch,height,depth)];
		f9 = fA[f_mem(14,x              ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(9 ,x,y,z,pitch,height,depth)];
		f10= fA[f_mem(17,dmin(x+1,width),y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(10,x,y,z,pitch,height,depth)];
		f11= fA[f_mem(18,x              ,dmin(y+1,height),dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(11,x,y,z,pitch,height,depth)];
		f12= fA[f_mem(15,dmax(x-1)      ,y               ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(12,x,y,z,pitch,height,depth)];
		f13= fA[f_mem(16,x              ,dmax(y-1)       ,dmin(z+1,depth) ,pitch,height,depth)];//fA[f_mem(13,x,y,z,pitch,height,depth)];
		f14= fA[f_mem(9 ,x              ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(14,x,y,z,pitch,height,depth)];
		f15= fA[f_mem(12,dmin(x+1,width),y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(15,x,y,z,pitch,height,depth)];
		f16= fA[f_mem(13,x              ,dmin(y+1,height),dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(16,x,y,z,pitch,height,depth)];
		f17= fA[f_mem(10,dmax(x-1)      ,y               ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(17,x,y,z,pitch,height,depth)];
		f18= fA[f_mem(11,x              ,dmax(y-1)       ,dmax(z-1)       ,pitch,height,depth)];//fA[f_mem(18,x,y,z,pitch,height,depth)];


		fB[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fB[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fB[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fB[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fB[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fB[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fB[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fB[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fB[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fB[f_mem(18,x,y,z,pitch,height,depth)] = f18;

//		fB[j+pitch*height*depth*1 ] = f1 ;
//		fB[j+pitch*height*depth*2 ] = f2 ;
//		fB[j+pitch*height*depth*3 ] = f3 ;
//		fB[j+pitch*height*depth*4 ] = f4 ;
//		fB[j+pitch*height*depth*5 ] = f5 ;
//		fB[j+pitch*height*depth*6 ] = f6 ;
//		fB[j+pitch*height*depth*7 ] = f7 ;
//		fB[j+pitch*height*depth*8 ] = f8 ;
//		fB[j+pitch*height*depth*9 ] = f9 ;
//		fB[j+pitch*height*depth*10] = f10;
//		fB[j+pitch*height*depth*11] = f11;
//		fB[j+pitch*height*depth*12] = f12;
//		fB[j+pitch*height*depth*13] = f13;
//		fB[j+pitch*height*depth*14] = f14;
//		fB[j+pitch*height*depth*15] = f15;
//		fB[j+pitch*height*depth*16] = f16;
//		fB[j+pitch*height*depth*17] = f17;
//		fB[j+pitch*height*depth*18] = f18;

	}
	else{


		float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

		f0 = fA[j];
		f1 = fA[f_mem(1 ,x-1,y  ,z  ,pitch,height,depth)];
		f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch,height,depth)];
		f3 = fA[f_mem(3 ,x+1,y  ,z  ,pitch,height,depth)];
		f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch,height,depth)];
		f5 = fA[f_mem(5 ,x-1,y-1,z  ,pitch,height,depth)];
		f6 = fA[f_mem(6 ,x+1,y-1,z  ,pitch,height,depth)];
		f7 = fA[f_mem(7 ,x+1,y+1,z  ,pitch,height,depth)];
		f8 = fA[f_mem(8 ,x-1,y+1,z  ,pitch,height,depth)];
		f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch,height,depth)];
		f10= fA[f_mem(10,x-1,y  ,z-1,pitch,height,depth)];
		f11= fA[f_mem(11,x  ,y-1,z-1,pitch,height,depth)];
		f12= fA[f_mem(12,x+1,y  ,z-1,pitch,height,depth)];
		f13= fA[f_mem(13,x  ,y+1,z-1,pitch,height,depth)];
		f14= fA[f_mem(14,x  ,y  ,z+1,pitch,height,depth)];
		f15= fA[f_mem(15,x-1,y  ,z+1,pitch,height,depth)];
		f16= fA[f_mem(16,x  ,y-1,z+1,pitch,height,depth)];
		f17= fA[f_mem(17,x+1,y  ,z+1,pitch,height,depth)];
		f18= fA[f_mem(18,x  ,y+1,z+1,pitch,height,depth)];

		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == height-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			if(z == depth-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
//			float fInt1,fInt2;//,fDiff;
			float u,v,w,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = uMax;//0.0;
			w = 0.0f;
	        
//			fInt1 = f0+f2+f4+f9+f11+f13+f14+f16+f18;
//	        fInt2 = f3+f6+f7+f12+f17;
//        	rho = u+(fInt1+2.0f*fInt2); //D2Q9i
			
        	rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			float usqr = u*u+v*v+w*w;
			f1 = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr)+f3-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr)+f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr)+f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr)+f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr)+f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		//mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fB[f_mem(0 ,x,y,z,pitch,height,depth)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch,height,depth)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch,height,depth)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch,height,depth)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch,height,depth)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch,height,depth)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch,height,depth)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch,height,depth)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch,height,depth)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch,height,depth)] = f9 ;
		fB[f_mem(10,x,y,z,pitch,height,depth)] = f10;
		fB[f_mem(11,x,y,z,pitch,height,depth)] = f11;
		fB[f_mem(12,x,y,z,pitch,height,depth)] = f12;
		fB[f_mem(13,x,y,z,pitch,height,depth)] = f13;
		fB[f_mem(14,x,y,z,pitch,height,depth)] = f14;
		fB[f_mem(15,x,y,z,pitch,height,depth)] = f15;
		fB[f_mem(16,x,y,z,pitch,height,depth)] = f16;
		fB[f_mem(17,x,y,z,pitch,height,depth)] = f17;
		fB[f_mem(18,x,y,z,pitch,height,depth)] = f18;
	}
}


__global__ void initialize_single(float *f, 
							int width, int height, int depth, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	f[j+0 *pitch*height*depth]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*height*depth]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*height*depth]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*height*depth]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*height*depth]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*height*depth]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f[j+12*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f[j+14*pitch*height*depth]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*height*depth]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}

__global__ void initialize(float* f0, float* f1, float* f2,
							float* f3, float* f4, float* f5,
							float* f6, float* f7, float* f8, float* f9,
							float* f10, float* f11, float* f12,
							float* f13, float* f14, float* f15,
							float* f16, float* f17, float* f18,
							int width, int height, size_t pitch)//pitch in elements
//__global__ void initialize(void** f0in, void** f1in, 
//							int w, int h, int pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int i = x+y*width+z*width*height;//index on linear mem
	int j = x+y*pitch+z*height*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
	float u,v,w,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	feq = 1.0f/3.0f*(rho-1.5f*usqr);
	f0[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[j] = feq;

	feq = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f9[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f10[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f11[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f12[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f13[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f14[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f15[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f16[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f17[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
	f18[j] = feq;
}



int main(int argc, char *argv[])
{

//	float *f0_h, *f1_h, *f2_h, *f3_h, *f4_h, *f5_h, *f6_h, *f7_h, *f8_h, *f9_h;
//	float *f10_h, *f11_h, *f12_h, *f13_h, *f14_h, *f15_h, *f16_h, *f17_h, *f18_h;
//	float *f0_dA, *f1_dA, *f2_dA, *f3_dA, *f4_dA, *f5_dA, *f6_dA, *f7_dA, *f8_dA, *f9_dA;
//	float *f10_dA, *f11_dA, *f12_dA, *f13_dA, *f14_dA, *f15_dA, *f16_dA, *f17_dA, *f18_dA;
//	float *f0_dB, *f1_dB, *f2_dB, *f3_dB, *f4_dB, *f5_dB, *f6_dB, *f7_dB, *f8_dB, *f9_dB;
//	float *f10_dB, *f11_dB, *f12_dB, *f13_dB, *f14_dB, *f15_dB, *f16_dB, *f17_dB, *f18_dB;
	int *image_d, *image_h;



	//hipPitchedPtr f0_d;


	ofstream output;
	output.open ("LBM1_out.dat");

	size_t memsize, memsize_int;
	size_t pitch;
	int i, n, nBlocks, xDim, yDim, zDim,tMax;
	float Re, omega, uMax, CharLength;
	
	int BLOCKSIZEx = 256;
	int BLOCKSIZEy = 1;
	int BLOCKSIZEz = 1;
	xDim = 256;
	yDim = 128;
	zDim = 32;
	tMax = 100;
	Re = 100.f;//100.f;
	uMax = 0.08f;
	CharLength = xDim-2.f;
	omega = 1.0f/(3.0f*(uMax*CharLength/Re)+0.5f);

	cout<<"omega: "<<omega<<endl;
	cout<<"blocksize: "<<BLOCKSIZEx<<"x"<<BLOCKSIZEy<<"x"<<BLOCKSIZEz<<endl;
	cout<<"grid: "<<xDim<<"x"<<yDim<<"x"<<zDim<<endl;
	cout<<"tMax: "<<tMax<<endl;

	nBlocks = (xDim/BLOCKSIZEx+xDim%BLOCKSIZEx)*(yDim/BLOCKSIZEy+yDim%BLOCKSIZEy)
				*(zDim/BLOCKSIZEz+zDim%BLOCKSIZEz);
	int B = BLOCKSIZEx*BLOCKSIZEy*BLOCKSIZEz;
	n = nBlocks*B;//block*dimx*dimy
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEx, BLOCKSIZEy, BLOCKSIZEz);
    dim3 grid(xDim/BLOCKSIZEx,yDim/BLOCKSIZEy,zDim/BLOCKSIZEz);

	memsize = n*sizeof(float);
	memsize_int = n*sizeof(int);

	hipExtent extent = make_hipExtent(xDim*sizeof(float),yDim,zDim);

	image_h = (int *)malloc(memsize_int);

	float *fA_h,*fA_d,*fB_d;
	fA_h = (float *)malloc(memsize*19);
	hipMallocPitch((void **) &fA_d, &pitch, xDim*sizeof(float), yDim*zDim*19);
	hipMallocPitch((void **) &fB_d, &pitch, xDim*sizeof(float), yDim*zDim*19);

	hipMalloc((void **) &image_d, memsize_int);

	cout<<pitch<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = i;
	}
	for (i = 0; i < n; i++)
	{
		int x = i%xDim;
		int y = (i/xDim)%yDim;
		int z = (i/xDim)/yDim;
		fA_h[i] = 0;
		image_h[i] = 0;
		if(x < 1) image_h[i] = 3;//DirichletWest
		if(x > xDim-2) image_h[i] = 1;//BB
		if(y < 1) image_h[i] = 1;//BB
		if(y > yDim-2) image_h[i] = 1;//BB
		if(z < 1) image_h[i] = 1;//DirichletWest
		if(z > zDim-2) image_h[i] = 1;//BB
	}
	hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;
	texRef_f9B.normalized = false;
	texRef_f10B.normalized = false;
	texRef_f11B.normalized = false;
	texRef_f12B.normalized = false;
	texRef_f13B.normalized = false;
	texRef_f14B.normalized = false;
	texRef_f15B.normalized = false;
	texRef_f16B.normalized = false;
	texRef_f17B.normalized = false;
	texRef_f18B.normalized = false;
	texRef_f0B.filterMode = hipFilterModePoint;
	texRef_f1B.filterMode = hipFilterModePoint;
	texRef_f2B.filterMode = hipFilterModePoint;
	texRef_f3B.filterMode = hipFilterModePoint;
	texRef_f4B.filterMode = hipFilterModePoint;
	texRef_f5B.filterMode = hipFilterModePoint;
	texRef_f6B.filterMode = hipFilterModePoint;
	texRef_f7B.filterMode = hipFilterModePoint;
	texRef_f8B.filterMode = hipFilterModePoint;
	texRef_f9B.filterMode = hipFilterModePoint;
	texRef_f10B.filterMode = hipFilterModePoint;
	texRef_f11B.filterMode = hipFilterModePoint;
	texRef_f12B.filterMode = hipFilterModePoint;
	texRef_f13B.filterMode = hipFilterModePoint;
	texRef_f14B.filterMode = hipFilterModePoint;
	texRef_f15B.filterMode = hipFilterModePoint;
	texRef_f16B.filterMode = hipFilterModePoint;
	texRef_f17B.filterMode = hipFilterModePoint;
	texRef_f18B.filterMode = hipFilterModePoint;
	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f9A.normalized = false;
	texRef_f10A.normalized = false;
	texRef_f11A.normalized = false;
	texRef_f12A.normalized = false;
	texRef_f13A.normalized = false;
	texRef_f14A.normalized = false;
	texRef_f15A.normalized = false;
	texRef_f16A.normalized = false;
	texRef_f17A.normalized = false;
	texRef_f18A.normalized = false;
	texRef_f0A.filterMode = hipFilterModePoint;
	texRef_f1A.filterMode = hipFilterModePoint;
	texRef_f2A.filterMode = hipFilterModePoint;
	texRef_f3A.filterMode = hipFilterModePoint;
	texRef_f4A.filterMode = hipFilterModePoint;
	texRef_f5A.filterMode = hipFilterModePoint;
	texRef_f6A.filterMode = hipFilterModePoint;
	texRef_f7A.filterMode = hipFilterModePoint;
	texRef_f8A.filterMode = hipFilterModePoint;
	texRef_f9A.filterMode = hipFilterModePoint;
	texRef_f10A.filterMode = hipFilterModePoint;
	texRef_f11A.filterMode = hipFilterModePoint;
	texRef_f12A.filterMode = hipFilterModePoint;
	texRef_f13A.filterMode = hipFilterModePoint;
	texRef_f14A.filterMode = hipFilterModePoint;
	texRef_f15A.filterMode = hipFilterModePoint;
	texRef_f16A.filterMode = hipFilterModePoint;
	texRef_f17A.filterMode = hipFilterModePoint;
	texRef_f18A.filterMode = hipFilterModePoint;
	}
	
	hipMemcpy2D(fA_d ,pitch,fA_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d ,pitch,fA_h ,xDim*sizeof(float),xDim*sizeof(float),yDim*zDim*19,hipMemcpyHostToDevice);

	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = 0;
	}


	if(true)//bind texture
	{
	hipBindTexture2D(0,&texRef_f0A, fA_d                            ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f1A, fA_d+pitch_elements*yDim*zDim   ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f2A, fA_d+pitch_elements*yDim*zDim*2 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f3A, fA_d+pitch_elements*yDim*zDim*3 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f4A, fA_d+pitch_elements*yDim*zDim*4 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f5A, fA_d+pitch_elements*yDim*zDim*5 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f6A, fA_d+pitch_elements*yDim*zDim*6 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f7A, fA_d+pitch_elements*yDim*zDim*7 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f8A, fA_d+pitch_elements*yDim*zDim*8 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f9A, fA_d+pitch_elements*yDim*zDim*9 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f10A,fA_d+pitch_elements*yDim*zDim*10,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f11A,fA_d+pitch_elements*yDim*zDim*11,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f12A,fA_d+pitch_elements*yDim*zDim*12,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f13A,fA_d+pitch_elements*yDim*zDim*13,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f14A,fA_d+pitch_elements*yDim*zDim*14,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f15A,fA_d+pitch_elements*yDim*zDim*15,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f16A,fA_d+pitch_elements*yDim*zDim*16,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f17A,fA_d+pitch_elements*yDim*zDim*17,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f18A,fA_d+pitch_elements*yDim*zDim*18,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f0B, fB_d                            ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f1B, fB_d+pitch_elements*yDim*zDim   ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f2B, fB_d+pitch_elements*yDim*zDim*2 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f3B, fB_d+pitch_elements*yDim*zDim*3 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f4B, fB_d+pitch_elements*yDim*zDim*4 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f5B, fB_d+pitch_elements*yDim*zDim*5 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f6B, fB_d+pitch_elements*yDim*zDim*6 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f7B, fB_d+pitch_elements*yDim*zDim*7 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f8B, fB_d+pitch_elements*yDim*zDim*8 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f9B, fB_d+pitch_elements*yDim*zDim*9 ,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f10B,fB_d+pitch_elements*yDim*zDim*10,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f11B,fB_d+pitch_elements*yDim*zDim*11,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f12B,fB_d+pitch_elements*yDim*zDim*12,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f13B,fB_d+pitch_elements*yDim*zDim*13,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f14B,fB_d+pitch_elements*yDim*zDim*14,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f15B,fB_d+pitch_elements*yDim*zDim*15,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f16B,fB_d+pitch_elements*yDim*zDim*16,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f17B,fB_d+pitch_elements*yDim*zDim*17,&desc,xDim,yDim*zDim,pitch);
	hipBindTexture2D(0,&texRef_f18B,fB_d+pitch_elements*yDim*zDim*18,&desc,xDim,yDim*zDim,pitch);
	}


//	initialize<<<grid, threads>>>(f0_dA.ptr, f1_dA.ptr, f2_dA.ptr, f3_dA.ptr, f4_dA.ptr, f5_dA.ptr, f6_dA.ptr, f7_dA.ptr, f8_dA.ptr, f9_dA.ptr,
//									f10_dA.ptr, f11_dA.ptr, f12_dA.ptr, f13_dA.ptr, f14_dA.ptr, f15_dA.ptr, f16_dA.ptr, f17_dA.ptr, f18_dA.ptr,
//									xDim,yDim,pitch);
//	initialize<<<grid, threads>>>(f0_dA, f1_dA, f2_dA, f3_dA, f4_dA, f5_dA, f6_dA, f7_dA, f8_dA, f9_dA,
//									f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//									xDim,yDim,pitch_elements);

	initialize_single<<<grid, threads>>>(fA_d,xDim,yDim,zDim,pitch_elements);

//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(mrt_d_single),hipFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<tMax; t=t+2){
	//for(int t = 0; t<tMax; t=t+1){

//		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);
//		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);

//		mrt_d_hybAB<<<grid, threads>>>(fA_d,fB_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);
//		mrt_d_hybBA<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);

//		mrt_d_textAB<<<grid, threads>>>(fA_d,fB_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);
//		mrt_d_textBA<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);

		mrt_d_shared<<<grid, threads>>>(fA_d,fB_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);
		mrt_d_shared<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);

//		simple_copy<<<grid, threads>>>(fA_d,fB_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);
//		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,uMax,xDim,yDim,zDim,pitch_elements);

		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(xDim*yDim*zDim*double(tMax/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<xDim<<","<<yDim<<","<<zDim<<","<<tMax<<","<<restime<<endl;




//	copytest<<<grid, threads>>>(f10_dA,test_d,xDim,yDim,zDim);
	//copytest<<<grid, threads>>>(test_d);
	//copytest<<<grid, threads>>>(image_d);
	hipUnbindTexture(texRef_f0A);
	hipUnbindTexture(texRef_f1A);
	hipUnbindTexture(texRef_f2A);
	hipUnbindTexture(texRef_f3A);
	hipUnbindTexture(texRef_f4A);
	hipUnbindTexture(texRef_f5A);
	hipUnbindTexture(texRef_f6A);
	hipUnbindTexture(texRef_f7A);
	hipUnbindTexture(texRef_f8A);
	hipUnbindTexture(texRef_f9A);
	hipUnbindTexture(texRef_f10A);
	hipUnbindTexture(texRef_f11A);
	hipUnbindTexture(texRef_f12A);
	hipUnbindTexture(texRef_f13A);
	hipUnbindTexture(texRef_f14A);
	hipUnbindTexture(texRef_f15A);
	hipUnbindTexture(texRef_f16A);
	hipUnbindTexture(texRef_f17A);
	hipUnbindTexture(texRef_f18A);
	hipUnbindTexture(texRef_f0B);
	hipUnbindTexture(texRef_f1B);
	hipUnbindTexture(texRef_f2B);
	hipUnbindTexture(texRef_f3B);
	hipUnbindTexture(texRef_f4B);
	hipUnbindTexture(texRef_f5B);
	hipUnbindTexture(texRef_f6B);
	hipUnbindTexture(texRef_f7B);
	hipUnbindTexture(texRef_f8B);
	hipUnbindTexture(texRef_f9B);
	hipUnbindTexture(texRef_f10B);
	hipUnbindTexture(texRef_f11B);
	hipUnbindTexture(texRef_f12B);
	hipUnbindTexture(texRef_f13B);
	hipUnbindTexture(texRef_f14B);
	hipUnbindTexture(texRef_f15B);
	hipUnbindTexture(texRef_f16B);
	hipUnbindTexture(texRef_f17B);
	hipUnbindTexture(texRef_f18B);

//	hipMemcpy2D(f0_h,xDim*sizeof(float) , f0_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f1_h,xDim*sizeof(float) , f1_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f2_h,xDim*sizeof(float) , f2_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f3_h,xDim*sizeof(float) , f3_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f4_h,xDim*sizeof(float) , f4_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f5_h,xDim*sizeof(float) , f5_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f6_h,xDim*sizeof(float) , f6_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f7_h,xDim*sizeof(float) , f7_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f8_h,xDim*sizeof(float) , f8_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f9_h,xDim*sizeof(float) , f9_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f10_h,xDim*sizeof(float),f10_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f11_h,xDim*sizeof(float),f11_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f12_h,xDim*sizeof(float),f12_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f13_h,xDim*sizeof(float),f13_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f14_h,xDim*sizeof(float),f14_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f15_h,xDim*sizeof(float),f15_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f16_h,xDim*sizeof(float),f16_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f17_h,xDim*sizeof(float),f17_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f18_h,xDim*sizeof(float),f18_dA,pitch,xDim*sizeof(float),yDim*zDim,hipMemcpyDeviceToHost);

	hipMemcpy2D(fA_h,xDim*sizeof(float),fA_d,pitch,xDim*sizeof(float),yDim*zDim*19,hipMemcpyDeviceToHost);

//	cout<<"f1_h is "<<f1_h[0]<<endl;
	//hipMemcpy(f0_h, f0_d.ptr, memsize, hipMemcpyDeviceToHost);

	hipMemcpy(image_h, image_d, memsize_int, hipMemcpyDeviceToHost);

//	cout<<image_h[0]<<endl;
//	cout<<"test_d: "<<test_h[0]<<endl;
//	for(i = 0; i<n; i++){
//	cout<<f0_h[i]<<",";
//	}


	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<xDim<<", J="<<yDim<<", K="<<zDim<<"\n";
	
	int row = 0;
	int col = 0;
	int dep = 0;
	i = 0;
	float rho, u, v, w;
	int j;

	for(dep = 0; dep<zDim; dep++){
	for(row = 0; row<yDim; row++){
		for(col = 0; col<xDim; col++){
			i = dep*xDim*yDim+row*xDim+col;
//			rho = 0;
			rho = fA_h[i];
			for(j = 1; j<19; j++)
				rho+=fA_h[i+xDim*yDim*zDim*j];
//			rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i]+f9_h[i]+
//			      f10_h[i]+f11_h[i]+f12_h[i]+f13_h[i]+f14_h[i]+f15_h[i]+f16_h[i]+f17_h[i]+f18_h[i];
			u = fA_h[i+xDim*yDim*zDim*1]-fA_h[i+xDim*yDim*zDim*3]+fA_h[i+xDim*yDim*zDim*5]-fA_h[i+xDim*yDim*zDim*6]-
				fA_h[i+xDim*yDim*zDim*7]+fA_h[i+xDim*yDim*zDim*8]+fA_h[i+xDim*yDim*zDim*10]-fA_h[i+xDim*yDim*zDim*12]
				+fA_h[i+xDim*yDim*zDim*15]-fA_h[i+xDim*yDim*zDim*17];
			v = fA_h[i+xDim*yDim*zDim*2]-fA_h[i+xDim*yDim*zDim*4]+fA_h[i+xDim*yDim*zDim*5]+fA_h[i+xDim*yDim*zDim*6]-fA_h[i+xDim*yDim*zDim*7]-fA_h[i+xDim*yDim*zDim*8]+fA_h[i+xDim*yDim*zDim*11]-fA_h[i+xDim*yDim*zDim*13]+fA_h[i+xDim*yDim*zDim*16]-fA_h[i+xDim*yDim*zDim*18];
			w = fA_h[i+xDim*yDim*zDim*9]+fA_h[i+xDim*yDim*zDim*10]+fA_h[i+xDim*yDim*zDim*11]+fA_h[i+xDim*yDim*zDim*12]+fA_h[i+xDim*yDim*zDim*13]-fA_h[i+xDim*yDim*zDim*14]-fA_h[i+xDim*yDim*zDim*15]-fA_h[i+xDim*yDim*zDim*16]-fA_h[i+xDim*yDim*zDim*17]-fA_h[i+xDim*yDim*zDim*18];
			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;
//			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<fA_h[i+xDim*yDim*zDim*1]<<","<<rho<<endl;
		}
	}
	}
	output.close();




	hipFree(image_d);
//	hipFree(f0_dA);
//	hipFree(f1_dA);
//	hipFree(f2_dA);
//	hipFree(f3_dA);
//	hipFree(f4_dA);
//	hipFree(f5_dA);
//	hipFree(f6_dA);
//	hipFree(f7_dA);
//	hipFree(f8_dA);
//	hipFree(f9_dA);
//	hipFree(f10_dA);
//	hipFree(f11_dA);
//	hipFree(f12_dA);
//	hipFree(f13_dA);
//	hipFree(f14_dA);
//	hipFree(f15_dA);
//	hipFree(f16_dA);
//	hipFree(f17_dA);
//	hipFree(f18_dA);
//	hipFree(f0_dB);
//	hipFree(f1_dB);
//	hipFree(f2_dB);
//	hipFree(f3_dB);
//	hipFree(f4_dB);
//	hipFree(f5_dB);
//	hipFree(f6_dB);
//	hipFree(f7_dB);
//	hipFree(f8_dB);
//	hipFree(f9_dB);
//	hipFree(f10_dB);
//	hipFree(f11_dB);
//	hipFree(f12_dB);
//	hipFree(f13_dB);
//	hipFree(f14_dB);
//	hipFree(f15_dB);
//	hipFree(f16_dB);
//	hipFree(f17_dB);
//	hipFree(f18_dB);

	hipFree(fA_d);
	hipFree(fB_d);
	return(0);

}
