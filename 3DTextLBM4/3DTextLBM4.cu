#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define BLOCKSIZEX 64
#define BLOCKSIZEY 1
#define BLOCKSIZEZ 1
#define XDIM 64
#define YDIM 64
#define ZDIM 64
#define TMAX 100
#define RE 50.f//100.f;
#define UMAX 0.08f
#define METHOD "SINGLE" //SINGLE,HYB,TEXT,SHARED
#define MODEL "BGK" //BGK,MRT,STREAM
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>


inline __device__ int ImageFcn(int x, int y, int z){
	if(y == 0 || z == 0 || x == XDIM-1 || y == YDIM-1 || z == ZDIM-1)
		return 1;
	else if(x == 0)
		return 3;
	else
		return 0;
}

texture<float,2,hipReadModeElementType> texRef_f0A;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;
texture<float,2,hipReadModeElementType> texRef_f9A;
texture<float,2,hipReadModeElementType> texRef_f10A;
texture<float,2,hipReadModeElementType> texRef_f11A;
texture<float,2,hipReadModeElementType> texRef_f12A;
texture<float,2,hipReadModeElementType> texRef_f13A;
texture<float,2,hipReadModeElementType> texRef_f14A;
texture<float,2,hipReadModeElementType> texRef_f15A;
texture<float,2,hipReadModeElementType> texRef_f16A;
texture<float,2,hipReadModeElementType> texRef_f17A;
texture<float,2,hipReadModeElementType> texRef_f18A;

texture<float,2,hipReadModeElementType> texRef_f0B;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;
texture<float,2,hipReadModeElementType> texRef_f9B;
texture<float,2,hipReadModeElementType> texRef_f10B;
texture<float,2,hipReadModeElementType> texRef_f11B;
texture<float,2,hipReadModeElementType> texRef_f12B;
texture<float,2,hipReadModeElementType> texRef_f13B;
texture<float,2,hipReadModeElementType> texRef_f14B;
texture<float,2,hipReadModeElementType> texRef_f15B;
texture<float,2,hipReadModeElementType> texRef_f16B;
texture<float,2,hipReadModeElementType> texRef_f17B;
texture<float,2,hipReadModeElementType> texRef_f18B;


int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//	float usqr = u*u+v*v+w*w;
	float usqr = fma(u,u,fma(v,v,w*w));

//	f0 -= omega*fma(-0.3333333333f,(fma(-1.5f,usqr,rho)),f0);//(f0 -0.3333333333f*(fma(-1.5f,usqr,rho)));//rho-1.5f*usqr));
//	f1 -= omega*fma(-0.0555555556f,fma(3.0f,    u ,rho)+fma(4.5f,u*u,-1.5f*usqr),f1);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f2 -= omega*fma(-0.0555555556f,fma(3.0f,    v ,rho)+fma(4.5f,v*v,-1.5f*usqr),f2);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f3 -= omega*fma(-0.0555555556f,fma(3.0f,    u ,rho)+fma(4.5f,u*u,-1.5f*usqr),f3);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f4 -= omega*fma(-0.0555555556f,fma(3.0f,    v ,rho)+fma(4.5f,v*v,-1.5f*usqr),f4);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f5 -= omega*fma(-0.0555555556f,fma(3.0f,( u+v),rho)+fma(4.5f,( u+v)*( u+v),-1.5f*usqr),f5 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f6 -= omega*fma(-0.0555555556f,fma(3.0f,(-u+v),rho)+fma(4.5f,(-u+v)*(-u+v),-1.5f*usqr),f6 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f7 -= omega*fma(-0.0555555556f,fma(3.0f,(-u-v),rho)+fma(4.5f,(-u-v)*(-u-v),-1.5f*usqr),f7 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f8 -= omega*fma(-0.0555555556f,fma(3.0f,( u-v),rho)+fma(4.5f,( u-v)*( u-v),-1.5f*usqr),f8 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f9 -= omega*fma(-0.0555555556f,fma(3.0f,(   w),rho)+fma(4.5f,(   w)*(   w),-1.5f*usqr),f9 );//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f10-= omega*fma(-0.0277777778f,fma(3.0f,( u+w),rho)+fma(4.5f,( u+w)*( u+w),-1.5f*usqr),f10);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f11-= omega*fma(-0.0277777778f,fma(3.0f,( v+w),rho)+fma(4.5f,( v+w)*( v+w),-1.5f*usqr),f11);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f12-= omega*fma(-0.0277777778f,fma(3.0f,(-u+w),rho)+fma(4.5f,(-u+w)*(-u+w),-1.5f*usqr),f12);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f13-= omega*fma(-0.0277777778f,fma(3.0f,(-v+w),rho)+fma(4.5f,(-v+w)*(-v+w),-1.5f*usqr),f13);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f14-= omega*fma(-0.0555555556f,fma(3.0f,(  -w),rho)+fma(4.5f,(  -w)*(  -w),-1.5f*usqr),f14);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f15-= omega*fma(-0.0277777778f,fma(3.0f,( u-w),rho)+fma(4.5f,( u-w)*( u-w),-1.5f*usqr),f15);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f16-= omega*fma(-0.0277777778f,fma(3.0f,( v-w),rho)+fma(4.5f,( v-w)*( v-w),-1.5f*usqr),f16);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f17-= omega*fma(-0.0277777778f,fma(3.0f,(-u-w),rho)+fma(4.5f,(-u-w)*(-u-w),-1.5f*usqr),f17);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f18-= omega*fma(-0.0277777778f,fma(3.0f,(-v-w),rho)+fma(4.5f,(-v-w)*(-v-w),-1.5f*usqr),f18);//(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	
	f0 = f0 -omega*(f0 -0.3333333333f*(rho-1.5f*usqr));
	f1 = f1 -omega*(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 = f2 -omega*(f2 -0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 = f3 -omega*(f3 -0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 = f4 -omega*(f4 -0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 = f5 -omega*(f5 -0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 = f6 -omega*(f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 = f7 -omega*(f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 = f8 -omega*(f8 -0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 = f9 -omega*(f9 -0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10= f10-omega*(f10-0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11= f11-omega*(f11-0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
	f12= f12-omega*(f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13= f13-omega*(f13-0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
	f14= f14-omega*(f14-0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15= f15-omega*(f15-0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16= f16-omega*(f16-0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17= f17-omega*(f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18= f18-omega*(f18-0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));


}


inline __device__ void mrt_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	//float rho,u,v,w;	
	float u,v,w;	
//	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
//	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	//m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
	//m1  = -30.f*f0+-11.f*(f1+f2+f3+f4+f9+f14)+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m1  = -19.f*f0+ 19.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m4  =           -4.f*f1         +  4.f*f3         +      f5+ -    f6+ -    f7+      f8         +    f10          + -      f12                    +      f15          + -    f17          ;
	//m6  =                    -4.f*f2         +  4.f*f4+      f5+      f6+ -    f7+ -    f8                   +      f11          + -    f13                    +      f16          + -    f18;
	//m8  =                                                                                 + -4.f*f9+    f10+        f11+      f12+      f13+  4.f*f14+ -    f15+ -    f16+ -    f17+ -    f18;
	
	//COMPUTE M-MEQ
	m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
	m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m9  =            2.f*f1+   -  f2+  2.f*f3+   -  f4+      f5+      f6+      f7+      f8+ -    f9+    f10+   -2.f*f11+      f12+ -2.f*f13+ -    f14+      f15+ -2.f*f16+      f17+ -2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+      f5+      f6+      f7+      f8+  2.f*f9+    f10+   -2.f*f11+      f12+ -2.f*f13+  2.f*f14+      f15+ -2.f*f16+      f17+ -2.f*f18;
	m11 =                         f2         +      f4+      f5+      f6+      f7+      f8+ -    f9+ -  f10            + -    f12          + -    f14+ -    f15          + -    f17          -(v*v-w*w);
	m12 =                    -2.f*f2           -2.f*f4+      f5+      f6+      f7+      f8+  2.f*f9+ -  f10            + -    f12          +  2.f*f14+ -    f15          + -    f17          ;
	m13 =                                                    f5+   -  f6+      f7+ -    f8                                                                                                     -u*v;
	m14 =                                                                                                           f11          + -    f13                    + -    f16          +      f18  -v*w;
	m15 =                                                                                               f10          + -      f12                    + -    f15          +      f17            -u*w;  
	m16 =                                                    f5+   -  f6+ -    f7+      f8           -  f10          +        f12                    + -    f15          +      f17          ;  
	m17 =                                                 -  f5+   -  f6+      f7+      f8                   +      f11          + -    f13                    +      f16          + -    f18;  
	m18 =                                                                                               f10+ -      f11+      f12+ -    f13          + -    f15+      f16+ -    f17+      f18;

	//m1 -= -11.f*rho+19.f*(u*u+v*v+w*w);
//	m1 -= 19.f*(u*u+v*v+w*w);
//	m2 -= -7.53968254f*(u*u+v*v+w*w);
	//m4 -= -0.66666667f*u;//qx_eq
	//m6 -= -0.66666667f*v;//qx_eq
	//m8 -= -0.66666667f*w;//qx_eq
//	m9 -= (2.f*u*u-(v*v+w*w));//(2.f*.f*.f-(u1*u1+u2*u2));///3.f;//pxx_eq
//	m11-= (v*v-w*w);//pww_eq
//	m13-= u*v;//pxy_eq
//	m14-= v*w;//pyz_eq
//	m15-= u*w;//pxz_eq

f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
}


inline __device__ int f_mem(int f_num, int x, int y, int z, size_t pitch)
{

	return (x+y*pitch+z*YDIM*pitch)+f_num*pitch*YDIM*ZDIM;
}
__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}

__global__ void simple_copy(float* fA, float* fB,
							int *image, float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)

	fB[j] = fA[j];//+0.01f;
}

__global__ void mrt_d_hybAB(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch)];
	f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f11= fin[f_mem(11,x  ,y-1,z-1,pitch)];
	f13= fin[f_mem(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_mem(14,x  ,y  ,z+1,pitch)];
	f16= fin[f_mem(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fin[f_mem(18,x  ,y+1,z+1,pitch)];
	}
	f1 = tex2D(texRef_f1A ,x-1,y  +YDIM*(z));
	f3 = tex2D(texRef_f3A ,x+1,y  +YDIM*(z));
	f5 = tex2D(texRef_f5A ,x-1,y-1+YDIM*(z));
	f6 = tex2D(texRef_f6A ,x+1,y-1+YDIM*(z));
	f7 = tex2D(texRef_f7A ,x+1,y+1+YDIM*(z));
	f8 = tex2D(texRef_f8A ,x-1,y+1+YDIM*(z));
	f15= tex2D(texRef_f15A,x-1,y  +YDIM*(z+1));
	f17= tex2D(texRef_f17A,x+1,y  +YDIM*(z+1));
	f10= tex2D(texRef_f10A,x-1,y  +YDIM*(z-1));
	f12= tex2D(texRef_f12A,x+1,y  +YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);

		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}
__global__ void mrt_d_hybBA(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch)];
	f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f11= fin[f_mem(11,x  ,y-1,z-1,pitch)];
	f13= fin[f_mem(13,x  ,y+1,z-1,pitch)];
	f14= fin[f_mem(14,x  ,y  ,z+1,pitch)];
	f16= fin[f_mem(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fin[f_mem(18,x  ,y+1,z+1,pitch)];
	}
	f1 = tex2D(texRef_f1B ,x-1,y  +YDIM*(z));
	f3 = tex2D(texRef_f3B ,x+1,y  +YDIM*(z));
	f5 = tex2D(texRef_f5B ,x-1,y-1+YDIM*(z));
	f6 = tex2D(texRef_f6B ,x+1,y-1+YDIM*(z));
	f7 = tex2D(texRef_f7B ,x+1,y+1+YDIM*(z));
	f8 = tex2D(texRef_f8B ,x-1,y+1+YDIM*(z));
	f15= tex2D(texRef_f15B,x-1,y  +YDIM*(z+1));
	f17= tex2D(texRef_f17B,x+1,y  +YDIM*(z+1));
	f10= tex2D(texRef_f10B,x-1,y  +YDIM*(z-1));
	f12= tex2D(texRef_f12B,x+1,y  +YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}


__global__ void mrt_d_textAB(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = tex2D(texRef_f2A ,x  ,y-1+YDIM*(z));
	f4 = tex2D(texRef_f4A ,x  ,y+1+YDIM*(z));
	f9 = tex2D(texRef_f9A ,x  ,y+1+YDIM*(z-1));
	f11= tex2D(texRef_f11A,x  ,y-1+YDIM*(z-1));
	f13= tex2D(texRef_f13A,x  ,y+1+YDIM*(z-1));
	f14= tex2D(texRef_f14A,x  ,y  +YDIM*(z+1));
	f16= tex2D(texRef_f16A,x  ,y-1+YDIM*(z+1));
	f18= tex2D(texRef_f18A,x  ,y+1+YDIM*(z+1));

	f1 = tex2D(texRef_f1A ,x-1,y  +YDIM*(z));
	f3 = tex2D(texRef_f3A ,x+1,y  +YDIM*(z));
	f5 = tex2D(texRef_f5A ,x-1,y-1+YDIM*(z));
	f6 = tex2D(texRef_f6A ,x+1,y-1+YDIM*(z));
	f7 = tex2D(texRef_f7A ,x+1,y+1+YDIM*(z));
	f8 = tex2D(texRef_f8A ,x-1,y+1+YDIM*(z));
	f15= tex2D(texRef_f15A,x-1,y  +YDIM*(z+1));
	f17= tex2D(texRef_f17A,x+1,y  +YDIM*(z+1));
	f10= tex2D(texRef_f10A,x-1,y  +YDIM*(z-1));
	f12= tex2D(texRef_f12A,x+1,y  +YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}

__global__ void mrt_d_textBA(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fin[j];
	f2 = tex2D(texRef_f2B ,x  ,y-1+YDIM*(z));
	f4 = tex2D(texRef_f4B ,x  ,y+1+YDIM*(z));
	f9 = tex2D(texRef_f9B ,x  ,y+1+YDIM*(z-1));
	f11= tex2D(texRef_f11B,x  ,y-1+YDIM*(z-1));
	f13= tex2D(texRef_f13B,x  ,y+1+YDIM*(z-1));
	f14= tex2D(texRef_f14B,x  ,y  +YDIM*(z+1));
	f16= tex2D(texRef_f16B,x  ,y-1+YDIM*(z+1));
	f18= tex2D(texRef_f18B,x  ,y+1+YDIM*(z+1));

	f1 = tex2D(texRef_f1B ,x-1,y  +YDIM*(z));
	f3 = tex2D(texRef_f3B ,x+1,y  +YDIM*(z));
	f5 = tex2D(texRef_f5B ,x-1,y-1+YDIM*(z));
	f6 = tex2D(texRef_f6B ,x+1,y-1+YDIM*(z));
	f7 = tex2D(texRef_f7B ,x+1,y+1+YDIM*(z));
	f8 = tex2D(texRef_f8B ,x-1,y+1+YDIM*(z));
	f15= tex2D(texRef_f15B,x-1,y  +YDIM*(z+1));
	f17= tex2D(texRef_f17B,x+1,y  +YDIM*(z+1));
	f10= tex2D(texRef_f10B,x-1,y  +YDIM*(z-1));
	f12= tex2D(texRef_f12B,x+1,y  +YDIM*(z-1));


	int im = ImageFcn(x,y,z);
	if(im == 1){//BB
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
        	//rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i

			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}
}



__global__ void mrt_d_shared(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = blockIdx.y;//threadIdx.y+blockIdx.y*blockDim.y;
	int z = blockIdx.z;//threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,z);

	__shared__ float  f1_s[BLOCKSIZEX];
	__shared__ float  f3_s[BLOCKSIZEX];
	__shared__ float  f5_s[BLOCKSIZEX];
	__shared__ float  f7_s[BLOCKSIZEX];
	__shared__ float  f6_s[BLOCKSIZEX];
	__shared__ float  f8_s[BLOCKSIZEX];
	__shared__ float f10_s[BLOCKSIZEX];
	__shared__ float f12_s[BLOCKSIZEX];
	__shared__ float f15_s[BLOCKSIZEX];
	__shared__ float f17_s[BLOCKSIZEX];

	 f1_s[threadIdx.x] = fA[f_mem(1 ,x ,y  ,z               ,pitch)];//dmax(x-1)      
	 f3_s[threadIdx.x] = fA[f_mem(3 ,x ,y  ,z               ,pitch)];//dmin(x+1,XDIM)
//	if(y != 0){//takin these out was good
	 f5_s[threadIdx.x] = fA[f_mem(5 ,x ,y-1,z               ,pitch)];//dmax(x-1)      
	 f6_s[threadIdx.x] = fA[f_mem(6 ,x ,y-1,z               ,pitch)];//dmin(x+1,XDIM)
//	}
//	else{
//	f5_s[threadIdx.x] = 0.f;
//	f6_s[threadIdx.x] = 0.f;
//	}
//	if(y != YDIM-1){
	 f7_s[threadIdx.x] = fA[f_mem(7 ,x ,y+1,z               ,pitch)];//dmin(x+1,XDIM)
	 f8_s[threadIdx.x] = fA[f_mem(8 ,x ,y+1,z               ,pitch)];//dmax(x-1)      
//	}
//	else{
//	f7_s[threadIdx.x] = 0.f;
//	f8_s[threadIdx.x] = 0.f;
//	}
//	if(z != 0){
	f10_s[threadIdx.x] = fA[f_mem(10,x ,y  ,z-1,pitch)];//dmax(x-1)      
	f12_s[threadIdx.x] = fA[f_mem(12,x ,y  ,z-1,pitch)];//dmin(x+1,XDIM)
//	}
//	else{
//	f10_s[threadIdx.x] = 0.f;
//	f12_s[threadIdx.x] = 0.f;
//	}
//	if(z != ZDIM-1){
	f15_s[threadIdx.x] = fA[f_mem(15,x ,y  ,z+1,pitch)];//dmax(x-1)      
	f17_s[threadIdx.x] = fA[f_mem(17,x ,y  ,z+1,pitch)];//dmin(x+1,XDIM)
//	}
//	else{
//	f15_s[threadIdx.x] = 0.f;
//	f17_s[threadIdx.x] = 0.f;
//	}

	float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	//f0  = 0.f;
	f1  = 0.f;
	f2  = 0.f;
	f3  = 0.f;
	f4  = 0.f;
	f5  = 0.f;
	f6  = 0.f;
	f7  = 0.f;
	f8  = 0.f;
	f9  = 0.f;
	f10 = 0.f;
	f11 = 0.f;
	f12 = 0.f;
	f13 = 0.f;
	f14 = 0.f;
	f15 = 0.f;
	f16 = 0.f;
	f17 = 0.f;
	f18 = 0.f;

//	if(y != 0){
	f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch)];
//	}
//	if(y != YDIM-1){
	f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch)];
//	}
	f14= fA[f_mem(14,x  ,y  ,z+1,pitch)];
	f16= fA[f_mem(16,x  ,y-1,z+1,pitch)];
	if(z != ZDIM-1){
	f18= fA[f_mem(18,x  ,y+1,z+1,pitch)];
	}
//	else{
//	f18 = 0.0f;
//	}
	
//		if(z != 0){
	f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f11= fA[f_mem(11,x  ,y-1,z-1,pitch)];
	f13= fA[f_mem(13,x  ,y+1,z-1,pitch)];
//		}

	__syncthreads();

	if(threadIdx.x != XDIM-1){
	f3 = f3_s[threadIdx.x+1];//fA[f_mem(3 ,x+1,y  ,z  ,pitch,YDIM,ZDIM)];
	f6 = f6_s[threadIdx.x+1];//fA[f_mem(6 ,x+1,y-1,z  ,pitch,YDIM,ZDIM)];
	f7 = f7_s[threadIdx.x+1];//fA[f_mem(7 ,x+1,y+1,z  ,pitch,YDIM,ZDIM)];
	f12=f12_s[threadIdx.x+1];//fA[f_mem(12,x+1,y  ,z-1,pitch,YDIM,ZDIM)];
	f17=f17_s[threadIdx.x+1];//fA[f_mem(17,x+1,y  ,z+1,pitch,YDIM,ZDIM)];
	}
//	else{
//	f3 =0.f;
//	f6 =0.f;
//	f7 =0.f;
//	f12=0.f;
//	f17=0.f;
//	}
	if(threadIdx.x != 0){
	f1 = f1_s[threadIdx.x-1];//fA[f_mem(1 ,x-1,y  ,z  ,pitch,YDIM,ZDIM)];
	f5 = f5_s[threadIdx.x-1];//fA[f_mem(5 ,x-1,y-1,z  ,pitch,YDIM,ZDIM)];
	f8 = f8_s[threadIdx.x-1];//fA[f_mem(8 ,x-1,y+1,z  ,pitch,YDIM,ZDIM)];
	f10=f10_s[threadIdx.x-1];//fA[f_mem(10,x-1,y  ,z-1,pitch,YDIM,ZDIM)];
	f15=f15_s[threadIdx.x-1];//fA[f_mem(15,x-1,y  ,z+1,pitch,YDIM,ZDIM)];
	}
//	else{
//	f1 =0.f;
//	f5 =0.f;
//	f8 =0.f;
//	f10=0.f;
//	f15=0.f;
//	}




	if(true){//BB
	//if(im == 1){//BB

//	float f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//
//	f4 = fA[f_mem(2 ,x  ,y-1,z  ,pitch)];
//	f2 = fA[f_mem(4 ,x  ,y+1,z  ,pitch)];
//	f9 = fA[f_mem(14,x  ,y  ,z+1,pitch)];
//	f13= fA[f_mem(16,x  ,y-1,z+1,pitch)];
//	if(z != ZDIM-1){
//	f11= fA[f_mem(18,x  ,y+1,z+1,pitch)];
//	}
//	f14= fA[f_mem(9 ,x  ,y  ,z-1,pitch)];
//	f18= fA[f_mem(11,x  ,y-1,z-1,pitch)];
//	f16= fA[f_mem(13,x  ,y+1,z-1,pitch)];
//
//	if(threadIdx.x != XDIM-1){
//	f1 = f3_s[threadIdx.x+1];//fA[f_mem(3 ,x+1,y  ,z  ,pitch,YDIM,ZDIM)];
//	f5 = f7_s[threadIdx.x+1];//fA[f_mem(6 ,x+1,y-1,z  ,pitch,YDIM,ZDIM)];
//	f8 = f6_s[threadIdx.x+1];//fA[f_mem(7 ,x+1,y+1,z  ,pitch,YDIM,ZDIM)];
//	f10=f17_s[threadIdx.x+1];//fA[f_mem(12,x+1,y  ,z-1,pitch,YDIM,ZDIM)];
//	f15=f12_s[threadIdx.x+1];//fA[f_mem(17,x+1,y  ,z+1,pitch,YDIM,ZDIM)];
//	}
//	if(threadIdx.x != 0){
//	f3 = f1_s[threadIdx.x-1];//fA[f_mem(1 ,x-1,y  ,z  ,pitch,YDIM,ZDIM)];
//	f7 = f5_s[threadIdx.x-1];//fA[f_mem(5 ,x-1,y-1,z  ,pitch,YDIM,ZDIM)];
//	f6 = f8_s[threadIdx.x-1];//fA[f_mem(8 ,x-1,y+1,z  ,pitch,YDIM,ZDIM)];
//	f17=f10_s[threadIdx.x-1];//fA[f_mem(10,x-1,y  ,z-1,pitch,YDIM,ZDIM)];
//	f12=f15_s[threadIdx.x-1];//fA[f_mem(15,x-1,y  ,z+1,pitch,YDIM,ZDIM)];
//	}

	//	fB[j] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f14;
		fB[f_mem(10,x,y,z,pitch)] = f17;
		fB[f_mem(11,x,y,z,pitch)] = f18;
		fB[f_mem(12,x,y,z,pitch)] = f15;
		fB[f_mem(13,x,y,z,pitch)] = f16;
		fB[f_mem(14,x,y,z,pitch)] = f9 ;
		fB[f_mem(15,x,y,z,pitch)] = f12;
		fB[f_mem(16,x,y,z,pitch)] = f13;
		fB[f_mem(17,x,y,z,pitch)] = f10;
		fB[f_mem(18,x,y,z,pitch)] = f11;

//		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
//		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
//		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
//		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
//		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
//		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
//		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
//		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
//		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
//		fB[f_mem(10,x,y,z,pitch)] = f10;
//		fB[f_mem(11,x,y,z,pitch)] = f11;
//		fB[f_mem(12,x,y,z,pitch)] = f12;
//		fB[f_mem(13,x,y,z,pitch)] = f13;
//		fB[f_mem(14,x,y,z,pitch)] = f14;
//		fB[f_mem(15,x,y,z,pitch)] = f15;
//		fB[f_mem(16,x,y,z,pitch)] = f16;
//		fB[f_mem(17,x,y,z,pitch)] = f17;
//		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
	else{


	float f0 = fA[j];


		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);

		}
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fB[f_mem(10,x,y,z,pitch)] = f10;
		fB[f_mem(11,x,y,z,pitch)] = f11;
		fB[f_mem(12,x,y,z,pitch)] = f12;
		fB[f_mem(13,x,y,z,pitch)] = f13;
		fB[f_mem(14,x,y,z,pitch)] = f14;
		fB[f_mem(15,x,y,z,pitch)] = f15;
		fB[f_mem(16,x,y,z,pitch)] = f16;
		fB[f_mem(17,x,y,z,pitch)] = f17;
		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
}
//{
//	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
//	int y = threadIdx.y+blockIdx.y*blockDim.y;
//	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
//	int im = ImageFcn(x,y,z);
//
//	__shared__ float  f1_s[BLOCKSIZEX];
//	__shared__ float  f3_s[BLOCKSIZEX];
//	__shared__ float  f5_s[BLOCKSIZEX];
//	__shared__ float  f7_s[BLOCKSIZEX];
//	__shared__ float  f6_s[BLOCKSIZEX];
//	__shared__ float  f8_s[BLOCKSIZEX];
//	__shared__ float f10_s[BLOCKSIZEX];
//	__shared__ float f12_s[BLOCKSIZEX];
//	__shared__ float f15_s[BLOCKSIZEX];
//	__shared__ float f17_s[BLOCKSIZEX];
//
//	 f1_s[threadIdx.x] = fA[f_mem(1 ,x ,y  ,z               ,pitch)];//dmax(x-1)      
//	 f3_s[threadIdx.x] = fA[f_mem(3 ,x ,y  ,z               ,pitch)];//dmin(x+1,XDIM)
////	if(y != 0){//takin these out was good
//	 f5_s[threadIdx.x] = fA[f_mem(5 ,x ,y-1,z               ,pitch)];//dmax(x-1)      
//	 f6_s[threadIdx.x] = fA[f_mem(6 ,x ,y-1,z               ,pitch)];//dmin(x+1,XDIM)
////	}
////	if(y != YDIM-1){
//	 f7_s[threadIdx.x] = fA[f_mem(7 ,x ,y+1,z               ,pitch)];//dmin(x+1,XDIM)
//	 f8_s[threadIdx.x] = fA[f_mem(8 ,x ,y+1,z               ,pitch)];//dmax(x-1)      
////	}
////	if(z != 0){
//	f10_s[threadIdx.x] = fA[f_mem(10,x ,y  ,z-1,pitch)];//dmax(x-1)      
//	f12_s[threadIdx.x] = fA[f_mem(12,x ,y  ,z-1,pitch)];//dmin(x+1,XDIM)
////	}
////	if(z != ZDIM-1){
//	f15_s[threadIdx.x] = fA[f_mem(15,x ,y  ,z+1,pitch)];//dmax(x-1)      
//	f17_s[threadIdx.x] = fA[f_mem(17,x ,y  ,z+1,pitch)];//dmin(x+1,XDIM)
////	}
//	__syncthreads();
//
//	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//
//	f0 = fA[j];
//	f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch)];
//	f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch)];
//	f14= fA[f_mem(14,x  ,y  ,z+1,pitch)];
//	f16= fA[f_mem(16,x  ,y-1,z+1,pitch)];
//	if(z != ZDIM-1){
//	f18= fA[f_mem(18,x  ,y+1,z+1,pitch)];
//	}
////	else
////	f18 = 0.0f;
//	
//	f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch)];
//	f11= fA[f_mem(11,x  ,y-1,z-1,pitch)];
//	f13= fA[f_mem(13,x  ,y+1,z-1,pitch)];
//
//	if(threadIdx.x != XDIM-1){
//	f3 = f3_s[threadIdx.x+1];//fA[f_mem(3 ,x+1,y  ,z  ,pitch,YDIM,ZDIM)];
//	f6 = f6_s[threadIdx.x+1];//fA[f_mem(6 ,x+1,y-1,z  ,pitch,YDIM,ZDIM)];
//	f7 = f7_s[threadIdx.x+1];//fA[f_mem(7 ,x+1,y+1,z  ,pitch,YDIM,ZDIM)];
//	f12=f12_s[threadIdx.x+1];//fA[f_mem(12,x+1,y  ,z-1,pitch,YDIM,ZDIM)];
//	f17=f17_s[threadIdx.x+1];//fA[f_mem(17,x+1,y  ,z+1,pitch,YDIM,ZDIM)];
//	}
//	if(threadIdx.x != 0){
//	f1 = f1_s[threadIdx.x-1];//fA[f_mem(1 ,x-1,y  ,z  ,pitch,YDIM,ZDIM)];
//	f5 = f5_s[threadIdx.x-1];//fA[f_mem(5 ,x-1,y-1,z  ,pitch,YDIM,ZDIM)];
//	f8 = f8_s[threadIdx.x-1];//fA[f_mem(8 ,x-1,y+1,z  ,pitch,YDIM,ZDIM)];
//	f10=f10_s[threadIdx.x-1];//fA[f_mem(10,x-1,y  ,z-1,pitch,YDIM,ZDIM)];
//	f15=f15_s[threadIdx.x-1];//fA[f_mem(15,x-1,y  ,z+1,pitch,YDIM,ZDIM)];
//	}
//
//	if(im == 1){//BB
//		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
//		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
//		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
//		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
//		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
//		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
//		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
//		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
//		fB[f_mem(9 ,x,y,z,pitch)] = f14;
//		fB[f_mem(10,x,y,z,pitch)] = f17;
//		fB[f_mem(11,x,y,z,pitch)] = f18;
//		fB[f_mem(12,x,y,z,pitch)] = f15;
//		fB[f_mem(13,x,y,z,pitch)] = f16;
//		fB[f_mem(14,x,y,z,pitch)] = f9 ;
//		fB[f_mem(15,x,y,z,pitch)] = f12;
//		fB[f_mem(16,x,y,z,pitch)] = f13;
//		fB[f_mem(17,x,y,z,pitch)] = f10;
//		fB[f_mem(18,x,y,z,pitch)] = f11;
//	}
//	else{
//		if(im == 3)//DirichletWest
//		{
//			if(y == 0){
////				f2 = f4;
//				f6 = f7;
////				f11 = f13;
////				f16 = f18;
//			}
//			else if(y == YDIM-1){
////				f4 = f2;
//				f7 = f6;
////				f13 = f11;
////				f18 = f16;
//			}
//			if(z == 0){
////				f9  = f14;
////				f10 = f15;
////				f11 = f16;
//				f12 = f17;
////				f13 = f18;			
//			}    
//			else if(z == ZDIM-1){
////				f14 = f9;
////				f15 = f10;
////				f16 = f11;
//				f17 = f12;
////				f18 = f13;
//			}
//			float u,v,w;//,rho;
//	        u = 0.0f;//*PoisProf(zcoord)*1.5;
//	        v = UMAX;//0.0;
//			w = 0.0f;
//	        
//			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
//			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//
//		}
//
//		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
//		//bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
//
//
//		//fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
//		fB[j] = f0 ;
//		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
//		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
//		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
//		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
//		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
//		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
//		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
//		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
//		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
//		fB[f_mem(10,x,y,z,pitch)] = f10;
//		fB[f_mem(11,x,y,z,pitch)] = f11;
//		fB[f_mem(12,x,y,z,pitch)] = f12;
//		fB[f_mem(13,x,y,z,pitch)] = f13;
//		fB[f_mem(14,x,y,z,pitch)] = f14;
//		fB[f_mem(15,x,y,z,pitch)] = f15;
//		fB[f_mem(16,x,y,z,pitch)] = f16;
//		fB[f_mem(17,x,y,z,pitch)] = f17;
//		fB[f_mem(18,x,y,z,pitch)] = f18;
//	}
//}


__global__ void mrt_d_single(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,z);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	f0 = fA[j];
	f1 = fA[f_mem(1 ,x-1,y  ,z  ,pitch)];
	f3 = fA[f_mem(3 ,x+1,y  ,z  ,pitch)];
	f2 = fA[f_mem(2 ,x  ,y-1,z  ,pitch)];
	f5 = fA[f_mem(5 ,x-1,y-1,z  ,pitch)];
	f6 = fA[f_mem(6 ,x+1,y-1,z  ,pitch)];
	f4 = fA[f_mem(4 ,x  ,y+1,z  ,pitch)];
	f7 = fA[f_mem(7 ,x+1,y+1,z  ,pitch)];
	f8 = fA[f_mem(8 ,x-1,y+1,z  ,pitch)];
	f9 = fA[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f10= fA[f_mem(10,x-1,y  ,z-1,pitch)];
	f11= fA[f_mem(11,x  ,y-1,z-1,pitch)];
	f12= fA[f_mem(12,x+1,y  ,z-1,pitch)];
	f13= fA[f_mem(13,x  ,y+1,z-1,pitch)];
	f14= fA[f_mem(14,x  ,y  ,z+1,pitch)];
	f15= fA[f_mem(15,x-1,y  ,z+1,pitch)];
	f16= fA[f_mem(16,x  ,y-1,z+1,pitch)];
	f17= fA[f_mem(17,x+1,y  ,z+1,pitch)];
	f18= fA[f_mem(18,x  ,y+1,dmin(z+1,ZDIM),pitch)];
	if(im == 1){//BB
		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f14;
		fB[f_mem(10,x,y,z,pitch)] = f17;
		fB[f_mem(11,x,y,z,pitch)] = f18;
		fB[f_mem(12,x,y,z,pitch)] = f15;
		fB[f_mem(13,x,y,z,pitch)] = f16;
		fB[f_mem(14,x,y,z,pitch)] = f9 ;
		fB[f_mem(15,x,y,z,pitch)] = f12;
		fB[f_mem(16,x,y,z,pitch)] = f13;
		fB[f_mem(17,x,y,z,pitch)] = f10;
		fB[f_mem(18,x,y,z,pitch)] = f11;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
				f11 = f13;
				f16 = f18;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
				f13 = f11;
				f18 = f16;
			}
			if(z == 0){
				f9  = f14;
				f10 = f15;
				f11 = f16;
				f12 = f17;
				f13 = f18;			
			}    
			else if(z == ZDIM-1){
				f14 = f9;
				f15 = f10;
				f16 = f11;
				f17 = f12;
				f18 = f13;
			}
			float u,v,w;//,rho;
	        u = 0.0f;//*PoisProf(zcoord)*1.5;
	        v = UMAX;//0.0;
			w = 0.0f;
	        
			f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
			f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
			f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
			f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
			f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);

		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);


		fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fB[f_mem(10,x,y,z,pitch)] = f10;
		fB[f_mem(11,x,y,z,pitch)] = f11;
		fB[f_mem(12,x,y,z,pitch)] = f12;
		fB[f_mem(13,x,y,z,pitch)] = f13;
		fB[f_mem(14,x,y,z,pitch)] = f14;
		fB[f_mem(15,x,y,z,pitch)] = f15;
		fB[f_mem(16,x,y,z,pitch)] = f16;
		fB[f_mem(17,x,y,z,pitch)] = f17;
		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
}


__global__ void initialize_single(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	f[j+0 *pitch*YDIM*ZDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f[j+12*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f[j+14*pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}

__global__ void initialize(float* f0, float* f1, float* f2,
							float* f3, float* f4, float* f5,
							float* f6, float* f7, float* f8, float* f9,
							float* f10, float* f11, float* f12,
							float* f13, float* f14, float* f15,
							float* f16, float* f17, float* f18,
							size_t pitch)//pitch in elements
//__global__ void initialize(void** f0in, void** f1in, 
//							int w, int h, int pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int i = x+y*XDIM+z*XDIM*YDIM;//index on linear mem
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
	float u,v,w,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	feq = 1.0f/3.0f*(rho-1.5f*usqr);
	f0[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[j] = feq;

	feq = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f9[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f10[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f11[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f12[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f13[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f14[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f15[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f16[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f17[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
	f18[j] = feq;
}



int main(int argc, char *argv[])
{

//	float *f0_h, *f1_h, *f2_h, *f3_h, *f4_h, *f5_h, *f6_h, *f7_h, *f8_h, *f9_h;
//	float *f10_h, *f11_h, *f12_h, *f13_h, *f14_h, *f15_h, *f16_h, *f17_h, *f18_h;
//	float *f0_dA, *f1_dA, *f2_dA, *f3_dA, *f4_dA, *f5_dA, *f6_dA, *f7_dA, *f8_dA, *f9_dA;
//	float *f10_dA, *f11_dA, *f12_dA, *f13_dA, *f14_dA, *f15_dA, *f16_dA, *f17_dA, *f18_dA;
//	float *f0_dB, *f1_dB, *f2_dB, *f3_dB, *f4_dB, *f5_dB, *f6_dB, *f7_dB, *f8_dB, *f9_dB;
//	float *f10_dB, *f11_dB, *f12_dB, *f13_dB, *f14_dB, *f15_dB, *f16_dB, *f17_dB, *f18_dB;
	int *image_d, *image_h;



	//hipPitchedPtr f0_d;


	ofstream output;
	output.open ("LBM1_out.dat");

	size_t memsize, memsize_int;
	size_t pitch;
	int i, n, nBlocks;
	float omega, CharLength;

	CharLength = XDIM-2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);

	cout<<"omega: "<<omega<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<"x"<<BLOCKSIZEZ<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<"x"<<ZDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

	nBlocks = (XDIM/BLOCKSIZEX+XDIM%BLOCKSIZEX)*(YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY)
				*(ZDIM/BLOCKSIZEZ+ZDIM%BLOCKSIZEZ);
	int B = BLOCKSIZEX*BLOCKSIZEY*BLOCKSIZEZ;
	n = nBlocks*B;//block*dimx*dimy
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
    dim3 grid(XDIM/BLOCKSIZEX,YDIM/BLOCKSIZEY,ZDIM/BLOCKSIZEZ);

	memsize = n*sizeof(float);
	memsize_int = n*sizeof(int);

	hipExtent extent = make_hipExtent(XDIM*sizeof(float),YDIM,ZDIM);

	image_h = (int *)malloc(memsize_int);

	float *fA_h,*fA_d,*fB_d;
	fA_h = (float *)malloc(memsize*19);
	hipMallocPitch((void **) &fA_d, &pitch, XDIM*sizeof(float), YDIM*ZDIM*19);
	hipMallocPitch((void **) &fB_d, &pitch, XDIM*sizeof(float), YDIM*ZDIM*19);

	hipMalloc((void **) &image_d, memsize_int);

	cout<<pitch<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = i;
	}
	for (i = 0; i < n; i++)
	{
		int x = i%XDIM;
		int y = (i/XDIM)%YDIM;
		int z = (i/XDIM)/YDIM;
		fA_h[i] = 0;
		image_h[i] = 0;
		if(x < 1) image_h[i] = 1;//DirichletWest
		if(x > XDIM-2) image_h[i] = 1;//BB
		if(y < 1) image_h[i] = 1;//BB
		if(y > YDIM-2) image_h[i] = 1;//BB
		if(z < 1) image_h[i] = 1;//DirichletWest
		if(z > ZDIM-2) image_h[i] = 1;//BB
	}
	hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;
	texRef_f9B.normalized = false;
	texRef_f10B.normalized = false;
	texRef_f11B.normalized = false;
	texRef_f12B.normalized = false;
	texRef_f13B.normalized = false;
	texRef_f14B.normalized = false;
	texRef_f15B.normalized = false;
	texRef_f16B.normalized = false;
	texRef_f17B.normalized = false;
	texRef_f18B.normalized = false;
	texRef_f0B.filterMode = hipFilterModePoint;
	texRef_f1B.filterMode = hipFilterModePoint;
	texRef_f2B.filterMode = hipFilterModePoint;
	texRef_f3B.filterMode = hipFilterModePoint;
	texRef_f4B.filterMode = hipFilterModePoint;
	texRef_f5B.filterMode = hipFilterModePoint;
	texRef_f6B.filterMode = hipFilterModePoint;
	texRef_f7B.filterMode = hipFilterModePoint;
	texRef_f8B.filterMode = hipFilterModePoint;
	texRef_f9B.filterMode = hipFilterModePoint;
	texRef_f10B.filterMode = hipFilterModePoint;
	texRef_f11B.filterMode = hipFilterModePoint;
	texRef_f12B.filterMode = hipFilterModePoint;
	texRef_f13B.filterMode = hipFilterModePoint;
	texRef_f14B.filterMode = hipFilterModePoint;
	texRef_f15B.filterMode = hipFilterModePoint;
	texRef_f16B.filterMode = hipFilterModePoint;
	texRef_f17B.filterMode = hipFilterModePoint;
	texRef_f18B.filterMode = hipFilterModePoint;
	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f9A.normalized = false;
	texRef_f10A.normalized = false;
	texRef_f11A.normalized = false;
	texRef_f12A.normalized = false;
	texRef_f13A.normalized = false;
	texRef_f14A.normalized = false;
	texRef_f15A.normalized = false;
	texRef_f16A.normalized = false;
	texRef_f17A.normalized = false;
	texRef_f18A.normalized = false;
	texRef_f0A.filterMode = hipFilterModePoint;
	texRef_f1A.filterMode = hipFilterModePoint;
	texRef_f2A.filterMode = hipFilterModePoint;
	texRef_f3A.filterMode = hipFilterModePoint;
	texRef_f4A.filterMode = hipFilterModePoint;
	texRef_f5A.filterMode = hipFilterModePoint;
	texRef_f6A.filterMode = hipFilterModePoint;
	texRef_f7A.filterMode = hipFilterModePoint;
	texRef_f8A.filterMode = hipFilterModePoint;
	texRef_f9A.filterMode = hipFilterModePoint;
	texRef_f10A.filterMode = hipFilterModePoint;
	texRef_f11A.filterMode = hipFilterModePoint;
	texRef_f12A.filterMode = hipFilterModePoint;
	texRef_f13A.filterMode = hipFilterModePoint;
	texRef_f14A.filterMode = hipFilterModePoint;
	texRef_f15A.filterMode = hipFilterModePoint;
	texRef_f16A.filterMode = hipFilterModePoint;
	texRef_f17A.filterMode = hipFilterModePoint;
	texRef_f18A.filterMode = hipFilterModePoint;
	}
	
	hipMemcpy2D(fA_d ,pitch,fA_h ,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d ,pitch,fA_h ,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyHostToDevice);

	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = 0;
	}


	if(true)//bind texture
	{
	hipBindTexture2D(0,&texRef_f0A, fA_d                            ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f1A, fA_d+pitch_elements*YDIM*ZDIM   ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f2A, fA_d+pitch_elements*YDIM*ZDIM*2 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f3A, fA_d+pitch_elements*YDIM*ZDIM*3 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f4A, fA_d+pitch_elements*YDIM*ZDIM*4 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f5A, fA_d+pitch_elements*YDIM*ZDIM*5 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f6A, fA_d+pitch_elements*YDIM*ZDIM*6 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f7A, fA_d+pitch_elements*YDIM*ZDIM*7 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f8A, fA_d+pitch_elements*YDIM*ZDIM*8 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f9A, fA_d+pitch_elements*YDIM*ZDIM*9 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f10A,fA_d+pitch_elements*YDIM*ZDIM*10,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f11A,fA_d+pitch_elements*YDIM*ZDIM*11,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f12A,fA_d+pitch_elements*YDIM*ZDIM*12,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f13A,fA_d+pitch_elements*YDIM*ZDIM*13,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f14A,fA_d+pitch_elements*YDIM*ZDIM*14,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f15A,fA_d+pitch_elements*YDIM*ZDIM*15,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f16A,fA_d+pitch_elements*YDIM*ZDIM*16,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f17A,fA_d+pitch_elements*YDIM*ZDIM*17,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f18A,fA_d+pitch_elements*YDIM*ZDIM*18,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f0B, fB_d                            ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f1B, fB_d+pitch_elements*YDIM*ZDIM   ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f2B, fB_d+pitch_elements*YDIM*ZDIM*2 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f3B, fB_d+pitch_elements*YDIM*ZDIM*3 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f4B, fB_d+pitch_elements*YDIM*ZDIM*4 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f5B, fB_d+pitch_elements*YDIM*ZDIM*5 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f6B, fB_d+pitch_elements*YDIM*ZDIM*6 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f7B, fB_d+pitch_elements*YDIM*ZDIM*7 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f8B, fB_d+pitch_elements*YDIM*ZDIM*8 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f9B, fB_d+pitch_elements*YDIM*ZDIM*9 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f10B,fB_d+pitch_elements*YDIM*ZDIM*10,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f11B,fB_d+pitch_elements*YDIM*ZDIM*11,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f12B,fB_d+pitch_elements*YDIM*ZDIM*12,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f13B,fB_d+pitch_elements*YDIM*ZDIM*13,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f14B,fB_d+pitch_elements*YDIM*ZDIM*14,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f15B,fB_d+pitch_elements*YDIM*ZDIM*15,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f16B,fB_d+pitch_elements*YDIM*ZDIM*16,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f17B,fB_d+pitch_elements*YDIM*ZDIM*17,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f18B,fB_d+pitch_elements*YDIM*ZDIM*18,&desc,XDIM,YDIM*ZDIM,pitch);
	}


//	initialize<<<grid, threads>>>(f0_dA.ptr, f1_dA.ptr, f2_dA.ptr, f3_dA.ptr, f4_dA.ptr, f5_dA.ptr, f6_dA.ptr, f7_dA.ptr, f8_dA.ptr, f9_dA.ptr,
//									f10_dA.ptr, f11_dA.ptr, f12_dA.ptr, f13_dA.ptr, f14_dA.ptr, f15_dA.ptr, f16_dA.ptr, f17_dA.ptr, f18_dA.ptr,
//									XDIM,YDIM,pitch);
//	initialize<<<grid, threads>>>(f0_dA, f1_dA, f2_dA, f3_dA, f4_dA, f5_dA, f6_dA, f7_dA, f8_dA, f9_dA,
//									f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//									XDIM,YDIM,pitch_elements);

	initialize_single<<<grid, threads>>>(fA_d,pitch_elements);

//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(mrt_d_single),hipFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<TMAX; t=t+2){
	//for(int t = 0; t<TMAX; t=t+1){
		if(METHOD == "SINGLE"){
		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
		}

		else if(METHOD == "HYB"){
		mrt_d_hybAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		mrt_d_hybBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
		}

		else if(METHOD == "TEXT"){
		mrt_d_textAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		mrt_d_textBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
		}

		else if(METHOD == "SHARED"){
		mrt_d_shared<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		mrt_d_shared<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
		}

//		simple_copy<<<grid, threads>>>(fA_d,fB_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);
//		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);

		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(XDIM*YDIM*ZDIM*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<XDIM<<","<<YDIM<<","<<ZDIM<<","<<TMAX<<","<<restime<<endl;




//	copytest<<<grid, threads>>>(f10_dA,test_d,XDIM,YDIM,ZDIM);
	//copytest<<<grid, threads>>>(test_d);
	//copytest<<<grid, threads>>>(image_d);
	hipUnbindTexture(texRef_f0A);
	hipUnbindTexture(texRef_f1A);
	hipUnbindTexture(texRef_f2A);
	hipUnbindTexture(texRef_f3A);
	hipUnbindTexture(texRef_f4A);
	hipUnbindTexture(texRef_f5A);
	hipUnbindTexture(texRef_f6A);
	hipUnbindTexture(texRef_f7A);
	hipUnbindTexture(texRef_f8A);
	hipUnbindTexture(texRef_f9A);
	hipUnbindTexture(texRef_f10A);
	hipUnbindTexture(texRef_f11A);
	hipUnbindTexture(texRef_f12A);
	hipUnbindTexture(texRef_f13A);
	hipUnbindTexture(texRef_f14A);
	hipUnbindTexture(texRef_f15A);
	hipUnbindTexture(texRef_f16A);
	hipUnbindTexture(texRef_f17A);
	hipUnbindTexture(texRef_f18A);
	hipUnbindTexture(texRef_f0B);
	hipUnbindTexture(texRef_f1B);
	hipUnbindTexture(texRef_f2B);
	hipUnbindTexture(texRef_f3B);
	hipUnbindTexture(texRef_f4B);
	hipUnbindTexture(texRef_f5B);
	hipUnbindTexture(texRef_f6B);
	hipUnbindTexture(texRef_f7B);
	hipUnbindTexture(texRef_f8B);
	hipUnbindTexture(texRef_f9B);
	hipUnbindTexture(texRef_f10B);
	hipUnbindTexture(texRef_f11B);
	hipUnbindTexture(texRef_f12B);
	hipUnbindTexture(texRef_f13B);
	hipUnbindTexture(texRef_f14B);
	hipUnbindTexture(texRef_f15B);
	hipUnbindTexture(texRef_f16B);
	hipUnbindTexture(texRef_f17B);
	hipUnbindTexture(texRef_f18B);

//	hipMemcpy2D(f0_h,XDIM*sizeof(float) , f0_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f1_h,XDIM*sizeof(float) , f1_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f2_h,XDIM*sizeof(float) , f2_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f3_h,XDIM*sizeof(float) , f3_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f4_h,XDIM*sizeof(float) , f4_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f5_h,XDIM*sizeof(float) , f5_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f6_h,XDIM*sizeof(float) , f6_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f7_h,XDIM*sizeof(float) , f7_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f8_h,XDIM*sizeof(float) , f8_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f9_h,XDIM*sizeof(float) , f9_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f10_h,XDIM*sizeof(float),f10_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f11_h,XDIM*sizeof(float),f11_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f12_h,XDIM*sizeof(float),f12_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f13_h,XDIM*sizeof(float),f13_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f14_h,XDIM*sizeof(float),f14_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f15_h,XDIM*sizeof(float),f15_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f16_h,XDIM*sizeof(float),f16_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f17_h,XDIM*sizeof(float),f17_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
//	hipMemcpy2D(f18_h,XDIM*sizeof(float),f18_dA,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);

//	hipMemcpy2D(fA_h,XDIM*sizeof(float),fA_d,pitch,XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyDeviceToHost);

//	cout<<"f1_h is "<<f1_h[0]<<endl;
	//hipMemcpy(f0_h, f0_d.ptr, memsize, hipMemcpyDeviceToHost);

	hipMemcpy(image_h, image_d, memsize_int, hipMemcpyDeviceToHost);

//	cout<<image_h[0]<<endl;
//	cout<<"test_d: "<<test_h[0]<<endl;
//	for(i = 0; i<n; i++){
//	cout<<f0_h[i]<<",";
//	}


	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<", K="<<ZDIM<<"\n";
	
	int row = 0;
	int col = 0;
	int dep = 0;
	i = 0;
	float rho, u, v, w;
	int j;

	for(dep = 0; dep<ZDIM; dep++){
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = dep*XDIM*YDIM+row*XDIM+col;
//			rho = 0;
			rho = fA_h[i];
			for(j = 1; j<19; j++)
				rho+=fA_h[i+XDIM*YDIM*ZDIM*j];
//			rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i]+f9_h[i]+
//			      f10_h[i]+f11_h[i]+f12_h[i]+f13_h[i]+f14_h[i]+f15_h[i]+f16_h[i]+f17_h[i]+f18_h[i];
			u = fA_h[i+XDIM*YDIM*ZDIM*1]-fA_h[i+XDIM*YDIM*ZDIM*3]+fA_h[i+XDIM*YDIM*ZDIM*5]-fA_h[i+XDIM*YDIM*ZDIM*6]-
				fA_h[i+XDIM*YDIM*ZDIM*7]+fA_h[i+XDIM*YDIM*ZDIM*8]+fA_h[i+XDIM*YDIM*ZDIM*10]-fA_h[i+XDIM*YDIM*ZDIM*12]
				+fA_h[i+XDIM*YDIM*ZDIM*15]-fA_h[i+XDIM*YDIM*ZDIM*17];
			v = fA_h[i+XDIM*YDIM*ZDIM*2]-fA_h[i+XDIM*YDIM*ZDIM*4]+fA_h[i+XDIM*YDIM*ZDIM*5]+fA_h[i+XDIM*YDIM*ZDIM*6]-fA_h[i+XDIM*YDIM*ZDIM*7]-fA_h[i+XDIM*YDIM*ZDIM*8]+fA_h[i+XDIM*YDIM*ZDIM*11]-fA_h[i+XDIM*YDIM*ZDIM*13]+fA_h[i+XDIM*YDIM*ZDIM*16]-fA_h[i+XDIM*YDIM*ZDIM*18];
			w = fA_h[i+XDIM*YDIM*ZDIM*9]+fA_h[i+XDIM*YDIM*ZDIM*10]+fA_h[i+XDIM*YDIM*ZDIM*11]+fA_h[i+XDIM*YDIM*ZDIM*12]+fA_h[i+XDIM*YDIM*ZDIM*13]-fA_h[i+XDIM*YDIM*ZDIM*14]-fA_h[i+XDIM*YDIM*ZDIM*15]-fA_h[i+XDIM*YDIM*ZDIM*16]-fA_h[i+XDIM*YDIM*ZDIM*17]-fA_h[i+XDIM*YDIM*ZDIM*18];
			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;
//			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<fA_h[i+XDIM*YDIM*ZDIM*1]<<","<<rho<<endl;
		}
	}
	}
	output.close();


//	cout<<endl<<fA_h[1280+81920];
//	cout<<endl<<fA_h[1281+81920];
//	cout<<endl<<fA_h[1282+81920];



	hipFree(image_d);
//	hipFree(f0_dA);
//	hipFree(f1_dA);
//	hipFree(f2_dA);
//	hipFree(f3_dA);
//	hipFree(f4_dA);
//	hipFree(f5_dA);
//	hipFree(f6_dA);
//	hipFree(f7_dA);
//	hipFree(f8_dA);
//	hipFree(f9_dA);
//	hipFree(f10_dA);
//	hipFree(f11_dA);
//	hipFree(f12_dA);
//	hipFree(f13_dA);
//	hipFree(f14_dA);
//	hipFree(f15_dA);
//	hipFree(f16_dA);
//	hipFree(f17_dA);
//	hipFree(f18_dA);
//	hipFree(f0_dB);
//	hipFree(f1_dB);
//	hipFree(f2_dB);
//	hipFree(f3_dB);
//	hipFree(f4_dB);
//	hipFree(f5_dB);
//	hipFree(f6_dB);
//	hipFree(f7_dB);
//	hipFree(f8_dB);
//	hipFree(f9_dB);
//	hipFree(f10_dB);
//	hipFree(f11_dB);
//	hipFree(f12_dB);
//	hipFree(f13_dB);
//	hipFree(f14_dB);
//	hipFree(f15_dB);
//	hipFree(f16_dB);
//	hipFree(f17_dB);
//	hipFree(f18_dB);

	hipFree(fA_d);
	hipFree(fB_d);
	return(0);

}
