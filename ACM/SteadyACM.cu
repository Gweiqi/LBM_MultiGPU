#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>
using namespace std;
	
#define CASENAME "stest2"

#define BLOCKSIZEX 128
#define BLOCKSIZEY 1
#define UBLOCKSIZEX 32
#define UBLOCKSIZEY 10
#define L1BLOCKSIZEX 64
#define L1BLOCKSIZEY 1
#define L1UBLOCKSIZEX 32
#define L1UBLOCKSIZEY 10
#define XDIM 128
#define YDIM 100
#define HALO 3
#define TMAX 1500
#define MAXIT 2000//20000
#define MAXRES 0.0001
#define RE 20
#define UMAX 1.f
#define BETA 0.05f
#define DTAU 0.01f
#define DT 1000000000.f

#define OBSTD 8.f
#define OBSTX 45.f
#define OBSTY 45.f

#define CONV 1  //1:UDS

int L1XDIM;
int L1YDIM;
texture<float,2,hipReadModeElementType> uA0,uB0,vA0,vB0,pA0,pB0,uA1,uB1,vA1,vB1,pA1,pB1;

int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}


void AllocateArray(float ****f,int x,int y)
{
    float ***array = new float **[4];
    for(int i = 0;i<4;i++)
    {
        array[i] = new float *[x];
        for(int j = 0;j<x;j++)
        {
            array[i][j] = new float [y];
            for(int k = 0;k<y;k++)
                array[i][j][k] = 0.f;
        }
    }
    *f = array;
}

void DeallocateArray(float ***f,int x)
{
    for(int i = 0;i<4;i++)
    {
    for(int j = 0;j<x;j++)
        delete [] f[i][j];
    delete [] f[i];
    }
    delete [] f;
}

void WriteResults(ostream &output, float *u, float *v, float *p)
{
    output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"p\"\n";
    output<<"ZONE F=POINT, I="<<XDIM-HALO*2<<", J="<<YDIM-HALO*2<<endl;
    float dx = 1;
    float dy = 1;
    float uval,vval,pval;
    for(int j = HALO; j<YDIM-HALO; j++){
    for(int i = HALO; i<XDIM-HALO; i++)
    {
        float xc = 0.5f*dx+(i)*dx;
        float yc = 0.5f*dy+(j)*dy;
        uval = 0.5f*(u[i+j*XDIM]+u[i-1  +j*XDIM]);
        vval = 0.5f*(v[i+j*XDIM]+v[i+(j-1)*XDIM]);
        pval = p[i+j*XDIM];
        if(xc>OBSTX && xc<OBSTX+OBSTD && yc>OBSTY && yc<OBSTY+OBSTD)
        {
        uval = 0.f; vval = 0.f; pval = 0.f;
        }
        output<<xc<<", "<<yc<<", "<<uval<<", "<<vval<<", "<<pval<<endl;
    }
    }
}

void WriteResultsL1(ostream &output, float *u, float *v, float *p)
{
    output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"p\"\n";
    output<<"ZONE F=POINT, I="<<L1XDIM-HALO*2<<", J="<<L1YDIM-HALO*2<<endl;
    float dx = 2;
    float dy = 2;
    float uval,vval,pval;
    for(int j = HALO; j<L1YDIM-HALO; j++){
    for(int i = HALO; i<L1XDIM-HALO; i++)
    {
        float xc = 0.5f*dx+0.5f+(i-HALO)*dx+HALO;
        float yc = 0.5f*dy+0.5f+(j-HALO)*dy+HALO;
        uval = 0.5f*(u[i+j*L1XDIM]+u[i-1  +j*L1XDIM]);
        vval = 0.5f*(v[i+j*L1XDIM]+v[i+(j-1)*L1XDIM]);
        pval = p[i+j*L1XDIM];
        int xcoord = (i-HALO)*2.f+HALO;
        int ycoord = (j-HALO)*2.f+HALO;
        if(xc>OBSTX+0.5f*dx && xc<OBSTX+OBSTD+0.5f*dx && yc>OBSTY+0.5f*dx && yc<OBSTY+OBSTD+0.5f*dx)
        {
        uval = 0.f; vval = 0.f; pval = 0.f;
        }
        output<<xc<<", "<<yc<<", "<<uval<<", "<<vval<<", "<<pval<<endl;
    }
    }
}


void WriteResiduals(ostream &output, float *Res)
{
    for(int i = 0; i<TMAX; i++)
        output<<i<<", "<<sqrt(Res[i])/((XDIM-2*HALO)*(YDIM-2*HALO))<<endl;
}

void WriteInputs(ostream &output)
{
    output<<"Domain size: \t"<<XDIM<<"x"<<YDIM<<endl;
    output<<"Halo size: \t"<<HALO<<endl;
    output<<"Target residual: \t"<<MAXRES<<endl;
    output<<"Pseudo time step size: \t"<<DTAU<<endl;
    output<<"Maximum iterations: \t"<<MAXIT<<endl;
    output<<"Real time step size: \t"<<DT<<endl;
    output<<"Maximum time steps: \t"<<TMAX<<endl;
    output<<"Re: \t"<<RE<<endl;
    output<<"uMax: \t"<<UMAX<<endl;
    string scheme;
    if(CONV == 0) scheme = "CDS   ";
    if(CONV == 1) scheme = "UDS   ";
    if(CONV == 2) scheme = "Hybrid";
    if(CONV == 3) scheme = "QUICK ";
    output<<"Convective discretization: \t"<<scheme<<endl;
}

__global__ void ACM_U_Shared(float* uA, float* vA, float* Res, float* uB, float* vB, float* pB, float* uC, float* vC, float* uD, float* vD, float nu, int it, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int tx = threadIdx.x+1;
    int ty = threadIdx.y+1;
    __shared__ float u[UBLOCKSIZEX+2][UBLOCKSIZEY+2];
    __shared__ float v[UBLOCKSIZEX+2][UBLOCKSIZEY+2];


    //if(x > HALO-2 && x < XDIM-HALO && y > HALO-2 && y < YDIM-HALO){
    if(x > 0 && x < XDIM-1 && y > 0 && y < YDIM-1){

    if(threadIdx.x == 0){
    u[0][ty] = uB[ x-1+ y   *pitch];
    v[0][ty] = vB[ x-1+ y   *pitch];
    if(threadIdx.y == blockDim.y-1){
    u[0][ty+1] = uB[ x-1+ (y+1)*pitch];
    }
    }
    if(threadIdx.x == blockDim.x-1){
    u[UBLOCKSIZEX+1][ty] = uB[ x+1+ y   *pitch];
    v[UBLOCKSIZEX+1][ty] = vB[ x+1+ y   *pitch];
    }
    if(threadIdx.y == 0){
    u[tx][0] = uB[ x+  (y-1)*pitch];
    v[tx][0] = vB[ x+  (y-1)*pitch];
    if(threadIdx.x == blockDim.x-1){
    v[tx+1][0] = vB[ x+1+(y-1)*pitch];
    }
    }
    if(threadIdx.y == blockDim.y-1){
    u[tx][UBLOCKSIZEY+1] = uB[ x+ (y+1)*pitch];
    v[tx][UBLOCKSIZEY+1] = vB[ x+ (y+1)*pitch];
    }

    u[tx][ty] = uB[ x+ (y  )*pitch];
    v[tx][ty] = vB[ x+ (y  )*pitch];
    }
    syncthreads();
    //if(x > HALO-2 && x < XDIM-HALO && y > HALO-2 && y < YDIM-HALO){
    if(x > 0 && x < XDIM-1 && y > 0 && y < YDIM-1){
    float Ae,Aw,An,As;
    float AP_Ue,AE_Ue,AW_Ue,AN_Ue,AS_Ue; //A coeff for East node on u of east face
    float AP_Vn,AE_Vn,AW_Vn,AN_Vn,AS_Vn;
    float Fe,Fw,Fn,Fs;
    float De,Dw,Dn,Ds;
    float B_Ue,B_Vn;
    float dx = 1.f/OBSTD;
    float dy = 1.f/OBSTD;
    Ae = dy; Aw = dy; An = dx; As = dx;
    Fe = 0.5f*(u[tx  ][ty  ]+u[tx+1][ty  ])*Ae;
    Fw = 0.5f*(u[tx-1][ty  ]+u[tx  ][ty  ])*Aw;
    Fn = 0.5f*(v[tx  ][ty  ]+v[tx+1][ty  ])*An;
    Fs = 0.5f*(v[tx  ][ty-1]+v[tx+1][ty-1])*As;
    De = nu*Ae/dx; Dw = nu*Aw/dx; Dn = nu*An/dy; Ds = nu*As/dy;
    if(CONV == 1){
    AE_Ue = max(-Fe,0.f)+De;
    AW_Ue = max( Fw,0.f)+Dw;
    AN_Ue = max(-Fn,0.f)+Dn;
    AS_Ue = max( Fs,0.f)+Ds;
    AP_Ue = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
    }
    else if(CONV == 2){
    AE_Ue = max(-Fe*0.5f+De,max(-Fe,0.f));
    AW_Ue = max( Fw*0.5f+Dw,max( Fw,0.f));
    AN_Ue = max(-Fn*0.5f+Dn,max(-Fn,0.f));
    AS_Ue = max( Fs*0.5f+Ds,max( Fs,0.f));
    AP_Ue = AE_Ue+AW_Ue+AN_Ue+AS_Ue;
    }
     B_Ue = Ae*(pB[x  +(y  )*pitch]-pB[x+1+(y  )*pitch]);
     //B_Ue-=( 3.f*u[tx  ][ty  ]-4.f*uC[x  +(y  )*pitch]+uD[x  +(y  )*pitch])*0.5f/DT;
    uA[ x  + y   *pitch] =((AE_Ue*u[tx+1][ty  ]+AW_Ue*u[tx-1][ty  ]
                           +AN_Ue*u[tx  ][ty+1]+AS_Ue*u[tx  ][ty-1]
                           -AP_Ue*u[tx  ][ty  ]+ B_Ue)/(dx*dy)
                           -( 3.f*u[tx  ][ty  ]-4.f*uC[x  +(y  )*pitch]+uD[x  +(y  )*pitch])*0.5f/DT
                           //-( u[tx  ][ty  ]-uC[x  +(y  )*pitch])*0.5f/DT
                           )*DTAU
                           + u[tx  ][ty  ];
    Fe = 0.5f*(u[tx  ][ty  ]+u[tx  ][ty+1])*Ae;
    Fw = 0.5f*(u[tx-1][ty  ]+u[tx-1][ty+1])*Aw;
    Fn = 0.5f*(v[tx  ][ty  ]+v[tx  ][ty+1])*An;
    Fs = 0.5f*(v[tx  ][ty  ]+v[tx  ][ty-1])*As;
    De = nu*Ae/dx; Dw = nu*Aw/dx; Dn = nu*An/dy; Ds = nu*As/dy;
    if(CONV == 1){
    AE_Vn = max(-Fe,0.f)+De;
    AW_Vn = max( Fw,0.f)+Dw;
    AN_Vn = max(-Fn,0.f)+Dn;
    AS_Vn = max( Fs,0.f)+Ds;
    AP_Vn = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
    }
    else if(CONV == 2){
    AE_Vn = max(-Fe*0.5f+De,max(-Fe,0.f));
    AW_Vn = max( Fw*0.5f+Dw,max( Fw,0.f));
    AN_Vn = max(-Fn*0.5f+Dn,max(-Fn,0.f));
    AS_Vn = max( Fs*0.5f+Ds,max( Fs,0.f));
    AP_Vn = AE_Vn+AW_Vn+AN_Vn+AS_Vn;
    }
     B_Vn = An*(pB[x  +(y  )*pitch]-pB[x  +(y+1)*pitch]);
     //B_Vn-=( 3.f*v[tx  ][ty  ]-4.f*vC[x  +(y  )*pitch]+vD[x  +(y  )*pitch])*0.5f/DT;
    vA[ x  + y   *pitch] =((AE_Vn*v[tx+1][ty  ]+AW_Vn*v[tx-1][ty  ]
                           +AN_Vn*v[tx  ][ty+1]+AS_Vn*v[tx  ][ty-1]
                           -AP_Vn*v[tx  ][ty  ]+ B_Vn)/(dx*dy)
                           -( 3.f*v[tx  ][ty  ]-4.f*vC[x  +(y  )*pitch]+vD[x  +(y  )*pitch])*0.5f/DT
                           //-( v[tx  ][ty  ]-vC[x  +(y  )*pitch])*0.5f/DT
                           )*DTAU
                           + v[tx  ][ty  ];

    if(y < HALO)
    {
    vA[ x  + y   *pitch] = 0.f;
    uA[ x  + y   *pitch] = u[tx  ][ty+1];
    }
    if(y > YDIM-HALO-2)
    {
    vA[ x  + y   *pitch] = 0.f;
    uA[ x  + y   *pitch] = u[tx  ][ty-1];
    }
    if(x < HALO)
    {
    uA[ x  + y   *pitch] = UMAX;
    vA[ x  + y   *pitch] = 0.f;
    }
    if(x > XDIM-HALO-2)
    {
    uA[ x  + y   *pitch] = u[tx-1][ty  ];
    vA[ x  + y   *pitch] = v[tx-1][ty  ];
    }

    if(x == OBSTX-1 && y>=OBSTY & y<OBSTY+OBSTD)
    uA[ x  + y   *pitch] = 0.f;
    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD)
    uA[ x  + y   *pitch] = 0.f;
    if(y == OBSTY-1 && x>=OBSTX & x<OBSTX+OBSTD)
    vA[ x  + y   *pitch] = 0.f;
    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD)
    vA[ x  + y   *pitch] = 0.f;   

    if(x == OBSTX && y>=OBSTY & y<OBSTY+OBSTD-1)
    vA[ x  + y   *pitch] = -v[tx-1][ty  ];
    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD-1)
    vA[ x  + y   *pitch] = -v[tx+1][ty  ];
    if(y == OBSTY && x>=OBSTX & x<OBSTX+OBSTD-1){
//    if(t < 200) uA[ x  + y   *pitch] = u[tx][ty-1];
//    else 
    uA[ x  + y   *pitch] = -u[tx  ][ty-1];
    }
    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD-1)
    uA[ x  + y   *pitch] = -u[tx  ][ty+1];

    }
    if(x == 0 && y == 0) Res[t] = 0.f;
}

__global__ void ACM_P(float* pA, float* Res, float* uA, float* vA, float* pB, int it, int t, size_t pitch, float *uD)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    __shared__ float sumRes[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();
    float dx = 1.f/OBSTD;
    float dy = 1.f/OBSTD;

    if(x > HALO-1 && x < XDIM-HALO && y > HALO-1 && y < YDIM-HALO){
    float res = (dy*(uA[ x  + y   *pitch]-uA[ x-1+ y   *pitch])
                                 +dx*(vA[ x  + y   *pitch]-vA[ x  +(y-1)*pitch]))/(dx*dy);
    pA[ x  + y   *pitch] = -res*DTAU/BETA+pB[ x  + y   *pitch];
    if(x > HALO && x < XDIM-HALO-2 && y > HALO && y < YDIM-HALO-2 
        && abs(res/UMAX)>MAXRES && !(x>= OBSTX && y>=OBSTY && x<OBSTX+OBSTD && y<OBSTY+OBSTD)){
    check[0] = 1;
    sumRes[threadIdx.x]=1.f;
    }
    else
    sumRes[threadIdx.x]=0.f;

    }
    else{
    sumRes[threadIdx.x]=0.f;
    }
    syncthreads();

    if(check[0] == 1){
    //reduction for residual
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumRes[threadIdx.x] += sumRes[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&Res[t],sumRes[0]);
    }
    }
}

__global__ void L1ACM_CF(float* uA, float* vA, float* pA, int it, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    float xc = 0.5f+(x-HALO)*2.f+HALO+0.5f;
    float yc = 0.5f+(y-HALO)*2.f+HALO+0.5f;
    uA[ x  + y   *pitch] = tex2D(uA0,xc,yc);
    vA[ x  + y   *pitch] = tex2D(vA0,xc,yc);
    pA[ x  + y   *pitch] = tex2D(pA0,xc,yc);
}

__global__ void L1ACM_FC(float* uA, float* vA, float* pA, int it, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    float xc = -0.25f+(x-HALO)*0.5f+HALO+0.5f;
    float yc = -0.25f+(y-HALO)*0.5f+HALO+0.5f;
    uA[ x  + y   *pitch] = tex2D(uA1,xc,yc);
    vA[ x  + y   *pitch] = tex2D(vA1,xc,yc);
    pA[ x  + y   *pitch] = tex2D(pA1,xc,yc);
}

//__global__ void L1ACM_U(float* uA, float* vA, float* uB, float* vB, float* pB, float nu, int it, int t, size_t pitch)
__global__ void L1ACM_U(float* uA, float* vA, float* Res, float* uB, float* vB, float* pB, float* uC, float* vC, float* uD, float* vD, float nu, int it, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int tx = threadIdx.x+1;
    int ty = threadIdx.y+1;
    __shared__ float u[L1UBLOCKSIZEX+2][L1UBLOCKSIZEY+2];
    __shared__ float v[L1UBLOCKSIZEX+2][L1UBLOCKSIZEY+2];
    int L1xDim = (XDIM-2*HALO)/2+2*HALO;
    int L1yDim = (YDIM-2*HALO)/2+2*HALO;
    int xc = (x-HALO)*2.f+HALO;
    int yc = (y-HALO)*2.f+HALO;


    if(x > 0 && x < L1xDim-1 && y > 0 && y < L1yDim-1){
    if(threadIdx.x == 0){
    u[0][ty] = uB[ x-1+ y   *pitch];
    v[0][ty] = vB[ x-1+ y   *pitch];
    if(threadIdx.y == blockDim.y-1){
    u[0][ty+1] = uB[ x-1+ (y+1)*pitch];
    }
    }
    if(threadIdx.x == blockDim.x-1){
    u[L1UBLOCKSIZEX+1][ty] = uB[ x+1+ y   *pitch];
    v[L1UBLOCKSIZEX+1][ty] = vB[ x+1+ y   *pitch];
    }
    if(threadIdx.y == 0){
    u[tx][0] = uB[ x+  (y-1)*pitch];
    v[tx][0] = vB[ x+  (y-1)*pitch];
    if(threadIdx.x == blockDim.x-1){
    v[tx+1][0] = vB[ x+1+(y-1)*pitch];
    }
    }
    if(threadIdx.y == blockDim.y-1){
    u[tx][L1UBLOCKSIZEY+1] = uB[ x+ (y+1)*pitch];
    v[tx][L1UBLOCKSIZEY+1] = vB[ x+ (y+1)*pitch];
    }
    u[tx][ty] = uB[ x+ (y  )*pitch];
    v[tx][ty] = vB[ x+ (y  )*pitch];
    }
    syncthreads();

    if(x > 0 && x < L1xDim-1 && y > 0 && y < L1yDim-1){
    float Ae,Aw,An,As;
    float AP_Ue,AE_Ue,AW_Ue,AN_Ue,AS_Ue; //A coeff for East node on u of east face
    float AP_Vn,AE_Vn,AW_Vn,AN_Vn,AS_Vn;
    float Fe,Fw,Fn,Fs;
    float De,Dw,Dn,Ds;
    float B_Ue,B_Vn;
    float dx = 2.f*1.f/OBSTD;
    float dy = 2.f*1.f/OBSTD;
    Ae = dy; Aw = dy; An = dx; As = dx;
    Fe = 0.5f*(u[tx  ][ty  ]+u[tx+1][ty  ])*Ae;
    Fw = 0.5f*(u[tx-1][ty  ]+u[tx  ][ty  ])*Aw;
    Fn = 0.5f*(v[tx  ][ty  ]+v[tx+1][ty  ])*An;
    Fs = 0.5f*(v[tx  ][ty-1]+v[tx+1][ty-1])*As;
    De = nu*Ae/dx; Dw = nu*Aw/dx; Dn = nu*An/dy; Ds = nu*As/dy;
    if(CONV == 1){
    AE_Ue = max(-Fe,0.f)+De;
    AW_Ue = max( Fw,0.f)+Dw;
    AN_Ue = max(-Fn,0.f)+Dn;
    AS_Ue = max( Fs,0.f)+Ds;
    AP_Ue = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
    }
    else if(CONV == 2){
    AE_Ue = max(-Fe*0.5f+De,max(-Fe,0.f));
    AW_Ue = max( Fw*0.5f+Dw,max( Fw,0.f));
    AN_Ue = max(-Fn*0.5f+Dn,max(-Fn,0.f));
    AS_Ue = max( Fs*0.5f+Ds,max( Fs,0.f));
    AP_Ue = AE_Ue+AW_Ue+AN_Ue+AS_Ue;
    }
     B_Ue = Ae*(pB[x  +(y  )*pitch]-pB[x+1+(y  )*pitch]);
    uA[ x  + y   *pitch] =((AE_Ue*u[tx+1][ty  ]+AW_Ue*u[tx-1][ty  ]
                           +AN_Ue*u[tx  ][ty+1]+AS_Ue*u[tx  ][ty-1]
                           -AP_Ue*u[tx  ][ty  ]+ B_Ue)/(dx*dy)
                           )*DTAU
                           + u[tx  ][ty  ];
    Fe = 0.5f*(u[tx  ][ty  ]+u[tx  ][ty+1])*Ae;
    Fw = 0.5f*(u[tx-1][ty  ]+u[tx-1][ty+1])*Aw;
    Fn = 0.5f*(v[tx  ][ty  ]+v[tx  ][ty+1])*An;
    Fs = 0.5f*(v[tx  ][ty  ]+v[tx  ][ty-1])*As;
    De = nu*Ae/dx; Dw = nu*Aw/dx; Dn = nu*An/dy; Ds = nu*As/dy;
    if(CONV == 1){
    AE_Vn = max(-Fe,0.f)+De;
    AW_Vn = max( Fw,0.f)+Dw;
    AN_Vn = max(-Fn,0.f)+Dn;
    AS_Vn = max( Fs,0.f)+Ds;
    AP_Vn = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
    }
    else if(CONV == 2){
    AE_Vn = max(-Fe*0.5f+De,max(-Fe,0.f));
    AW_Vn = max( Fw*0.5f+Dw,max( Fw,0.f));
    AN_Vn = max(-Fn*0.5f+Dn,max(-Fn,0.f));
    AS_Vn = max( Fs*0.5f+Ds,max( Fs,0.f));
    AP_Vn = AE_Vn+AW_Vn+AN_Vn+AS_Vn;
    }
     B_Vn = An*(pB[x  +(y  )*pitch]-pB[x  +(y+1)*pitch]);
    vA[ x  + y   *pitch] =((AE_Vn*v[tx+1][ty  ]+AW_Vn*v[tx-1][ty  ]
                           +AN_Vn*v[tx  ][ty+1]+AS_Vn*v[tx  ][ty-1]
                           -AP_Vn*v[tx  ][ty  ]+ B_Vn)/(dx*dy)
                           )*DTAU
                           + v[tx  ][ty  ];

    if(y < HALO)
    {
    vA[ x  + y   *pitch] = 0.f;
    uA[ x  + y   *pitch] = u[tx  ][ty+1];
    }
    if(y > L1yDim-HALO-2)
    {
    vA[ x  + y   *pitch] = 0.f;
    uA[ x  + y   *pitch] = u[tx  ][ty-1];
    }
    if(x < HALO)
    {
    uA[ x  + y   *pitch] = UMAX;
    vA[ x  + y   *pitch] = 0.f;
    }
    if(x > L1xDim-HALO-2)
    {
    uA[ x  + y   *pitch] = u[tx-1][ty  ];
    vA[ x  + y   *pitch] = v[tx-1][ty  ];
    }

    if(xc == OBSTX-dx && yc>=OBSTY & yc<OBSTY+OBSTD)
    uA[ x  + y   *pitch] = 0.f;
    if(xc == OBSTX+OBSTD-dx && yc>=OBSTY && yc<OBSTY+OBSTD)
    uA[ x  + y   *pitch] = 0.f;
    if(yc == OBSTY-dy && xc>=OBSTX & xc<OBSTX+OBSTD)
    vA[ x  + y   *pitch] = 0.f;
    if(yc == OBSTY+OBSTD-dy && xc>=OBSTX && xc<OBSTX+OBSTD)
    vA[ x  + y   *pitch] = 0.f;   

    if(xc == OBSTX && yc>=OBSTY & yc<OBSTY+OBSTD-dy)
    vA[ x  + y   *pitch] = -v[tx-1][ty  ];
    if(xc == OBSTX+OBSTD-dx && yc>=OBSTY && yc<OBSTY+OBSTD-dy)
    vA[ x  + y   *pitch] = -v[tx+1][ty  ];
    if(yc == OBSTY && xc>=OBSTX & xc<OBSTX+OBSTD-dy){
//    if(t < 200) uA[ x  + y   *pitch] = u[tx][ty-1];
//    else 
    uA[ x  + y   *pitch] = -u[tx  ][ty-1];
    }
    if(yc == OBSTY+OBSTD-dy && xc>=OBSTX && xc<OBSTX+OBSTD-dy)
    uA[ x  + y   *pitch] = -u[tx  ][ty+1];

    }




}

__global__ void L1ACM_P(float* pA, float* Res, float* uA, float* vA, float* pB, int it, int t, size_t pitch, float *uD)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    __shared__ float sumRes[L1BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();
    float dx = 2.f*1.f/OBSTD;
    float dy = 2.f*1.f/OBSTD;
    int L1xDim = (XDIM-2*HALO)/2+2*HALO;
    int L1yDim = (YDIM-2*HALO)/2+2*HALO;

    if(x > HALO-1 && x < L1xDim-HALO && y > HALO-1 && y < L1yDim-HALO){
    float res = (dy*(uA[ x  + y   *pitch]-uA[ x-1+ y   *pitch])
                                 +dx*(vA[ x  + y   *pitch]-vA[ x  +(y-1)*pitch]))/(dx*dy);
    pA[ x  + y   *pitch] = -res*DTAU/BETA+pB[ x  + y   *pitch];
//    if(x > HALO && x < L1XDIM-HALO-2 && y > HALO && y < L1YDIM-HALO-2 
//        && abs(res/UMAX)>MAXRES && !(2*x>= OBSTX && 2*y>=OBSTY && 2*x<OBSTX+OBSTD && 2*y<OBSTY+OBSTD)){
//    check[0] = 1;
//    sumRes[threadIdx.x]=1.f;
//    }
//    else
//    sumRes[threadIdx.x]=0.f;

    }
    else{
    sumRes[threadIdx.x]=0.f;
    }
    syncthreads();

    if(check[0] == 1){
    //reduction for residual
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumRes[threadIdx.x] += sumRes[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&Res[t],sumRes[0]);
    }
    }
}


__global__ void ACM_Forces(float *FX, float *FY, float* uD, float* vD, float* pA, float nu, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    __shared__ float sumFX[BLOCKSIZEX],sumFY[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    sumFX[threadIdx.x] = 0.f;
    sumFY[threadIdx.x] = 0.f;
    syncthreads();
    float dx = 1.f/OBSTD;
    float dy = 1.f/OBSTD;

    //forces on bottom wall
    if(x >= OBSTX && x<OBSTX+OBSTD && y == OBSTY){
    check[0] = 1;
    sumFY[threadIdx.x] += dx*pA[ x  +(y-1)*pitch];
    if(x<OBSTX+OBSTD-1)
        sumFX[threadIdx.x] -= dx*nu*2.f*uD[ x  +(y  )*pitch]/dy;
    }
    //forces on top wall
    if(x >= OBSTX && x<OBSTX+OBSTD && y == OBSTY+OBSTD-1){
    check[0] = 1;
    sumFY[threadIdx.x] -= dx*pA[ x  +(y+1)*pitch];
    if(x<OBSTX+OBSTD-1)
        sumFX[threadIdx.x] -= dx*nu*2.f*uD[ x  +(y  )*pitch]/dy;
    }

    //forces on left wall
    if(y >= OBSTY && y<OBSTY+OBSTD && x == OBSTX){
    check[0] = 1;
    sumFX[threadIdx.x] += dy*pA[ x-1+(y  )*pitch];
    if(y<OBSTY+OBSTD-1)
        sumFY[threadIdx.x] -= dy*nu*2.f*vD[ x  +(y  )*pitch]/dx;
    }
    //forces on right wall
    if(y >= OBSTY && y<OBSTY+OBSTD && x == OBSTX+OBSTD-1){
    check[0] = 1;
    sumFX[threadIdx.x] -= dy*pA[ x+1+(y  )*pitch];
    if(y<OBSTY+OBSTD-1)
        sumFY[threadIdx.x] -= dy*nu*2.f*vD[ x  +(y  )*pitch]/dx;
    }

    syncthreads();

    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumFX[threadIdx.x] += sumFX[threadIdx.x+halfPoint];
            sumFY[threadIdx.x] += sumFY[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t],sumFX[0]);
        atomicAdd(&FY[t],sumFY[0]);
    }
    }


}

int main()
{
    
    ofstream output_log,output_results,output_residual,output_vel,output_force;
    float nu = UMAX/RE;
    float Ma = UMAX*sqrt(BETA);
    L1XDIM = (XDIM-2*HALO)/2+2*HALO;
    L1YDIM = (YDIM-2*HALO)/2+2*HALO;
    cout<<"Ma = "<<Ma<<endl;
    string FileName = CASENAME;
    output_log.open ((FileName+".log").c_str());
    output_results.open ((FileName+".dat").c_str());
    output_residual.open ((FileName+".res").c_str());
    output_vel.open ((FileName+".vel").c_str());
    output_force.open ((FileName+".frc").c_str());
    //write input parameters to console and log file
    WriteInputs(cout);
    WriteInputs(output_log);

    //allocate and initialize arrays
    float *u[4],*v[4],*p[4],*Res,*FX,*FY;
    float *L1u[4],*L1v[4],*L1p[4];
    for(int i = 0; i<4; i++){
        u[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
        v[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
        p[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
        L1u[i] = (float *)malloc(L1XDIM*L1YDIM*sizeof(float));
        L1v[i] = (float *)malloc(L1XDIM*L1YDIM*sizeof(float));
        L1p[i] = (float *)malloc(L1XDIM*L1YDIM*sizeof(float));
    }
    Res= (float *)malloc(TMAX*sizeof(float));
    FX = (float *)malloc(TMAX*sizeof(float));
    FY = (float *)malloc(TMAX*sizeof(float));
    //initialize host memory
    for(int i = 0; i<4; i++){
    for(int j = 0; j<XDIM*YDIM; j++){
        u[i][j] = UMAX;
        v[i][j] = 0.f;
        p[i][j] = 0.f;
    }
    for(int j = 0; j<L1XDIM*L1YDIM; j++){
        L1u[i][j] = 0.f;
        L1v[i][j] = 0.f;
        L1p[i][j] = 0.f;
    }
    }
    for(int j = 0; j<TMAX; j++){
        Res[j] = 0.f;
        FX[j] = 0.f;
        FY[j] = 0.f;
    }

    //size_t memsize, memsize2;
    size_t pitch = 2;
    while(pitch<XDIM)
        pitch=pitch*2;
    pitch *= sizeof(float);//pitch*sizeof(float);
    size_t pitch_e = pitch/sizeof(float);
    cout<<"Pitch (in elements): "<<pitch/sizeof(float)<<endl;

    size_t L1pitch = 2;
    while(L1pitch<L1XDIM)
        L1pitch=L1pitch*2;
    L1pitch *= sizeof(float);//pitch*sizeof(float);
    size_t L1pitch_e = L1pitch/sizeof(float);

    L1pitch_e = pitch_e;
    L1pitch = pitch;
    cout<<"L1Pitch (in elements): "<<L1pitch/sizeof(float)<<endl;


    dim3 u_threads(UBLOCKSIZEX, UBLOCKSIZEY,1);
    dim3 u_grid  (((XDIM+UBLOCKSIZEX-1)/UBLOCKSIZEX),((YDIM+UBLOCKSIZEY-1)/UBLOCKSIZEY),1);
    dim3 threads(BLOCKSIZEX, BLOCKSIZEY,1);
    dim3 grid  (((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);

    dim3 L1u_threads(L1UBLOCKSIZEX, L1UBLOCKSIZEY,1);
    dim3 L1u_grid  (((L1XDIM+L1UBLOCKSIZEX-1)/L1UBLOCKSIZEX),((L1YDIM+L1UBLOCKSIZEY-1)/L1UBLOCKSIZEY),1);
    dim3 L1threads(L1BLOCKSIZEX, L1BLOCKSIZEY,1);
    dim3 L1grid  (((L1XDIM+L1BLOCKSIZEX-1)/L1BLOCKSIZEX),((L1YDIM+L1BLOCKSIZEY-1)/L1BLOCKSIZEY),1);

    hipStream_t compute;
    hipStream_t transfer;
    hipStreamCreate(&compute);
    hipStreamCreate(&transfer);

    float *u_d[4],*v_d[4],*p_d[4], *Res_d, *FX_d, *FY_d;
    float *L1u_d[4],*L1v_d[4],*L1p_d[4];
    for(int i = 0; i<4; i++){
        hipMalloc((void **) &u_d[i], pitch_e*YDIM*sizeof(float));
        hipMalloc((void **) &v_d[i], pitch_e*YDIM*sizeof(float));
        hipMalloc((void **) &p_d[i], pitch_e*YDIM*sizeof(float));
        hipMalloc((void **) &L1u_d[i], L1pitch_e*L1YDIM*sizeof(float));
        hipMalloc((void **) &L1v_d[i], L1pitch_e*L1YDIM*sizeof(float));
        hipMalloc((void **) &L1p_d[i], L1pitch_e*L1YDIM*sizeof(float));
    }
    hipMalloc((void **) &Res_d, TMAX*sizeof(float));
    hipMalloc((void **) &FX_d, TMAX*sizeof(float));
    hipMalloc((void **) &FY_d, TMAX*sizeof(float));
    for(int i = 0; i<4; i++){
        hipMemcpy2D(u_d[i],pitch,u[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(v_d[i],pitch,v[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(p_d[i],pitch,p[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(L1u_d[i],L1pitch,L1u[i],L1XDIM*sizeof(float),L1XDIM*sizeof(float),L1YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(L1v_d[i],L1pitch,L1v[i],L1XDIM*sizeof(float),L1XDIM*sizeof(float),L1YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(L1p_d[i],L1pitch,L1p[i],L1XDIM*sizeof(float),L1XDIM*sizeof(float),L1YDIM,hipMemcpyHostToDevice);
    }
    hipMemcpy(Res_d,Res,TMAX*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(FX_d,FX,TMAX*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(FY_d,FY,TMAX*sizeof(float),hipMemcpyHostToDevice);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	uA0.normalized = false; uA0.filterMode = hipFilterModeLinear;
	uB0.normalized = false; uB0.filterMode = hipFilterModeLinear;
	vA0.normalized = false; vA0.filterMode = hipFilterModeLinear;
	vB0.normalized = false; vB0.filterMode = hipFilterModeLinear;
	pA0.normalized = false; pA0.filterMode = hipFilterModeLinear;
	pB0.normalized = false; pB0.filterMode = hipFilterModeLinear;
	uA1.normalized = false; uA1.filterMode = hipFilterModeLinear;
	uB1.normalized = false; uB1.filterMode = hipFilterModeLinear;
	vA1.normalized = false; vA1.filterMode = hipFilterModeLinear;
	vB1.normalized = false; vB1.filterMode = hipFilterModeLinear;
	pA1.normalized = false; pA1.filterMode = hipFilterModeLinear;
	pB1.normalized = false; pB1.filterMode = hipFilterModeLinear;
	for(int i = 0; i<2; i++){
	uA0.addressMode[i]  = hipAddressModeClamp;
	uB0.addressMode[i]  = hipAddressModeClamp;
	vA0.addressMode[i]  = hipAddressModeClamp;
	vB0.addressMode[i]  = hipAddressModeClamp;
	pA0.addressMode[i]  = hipAddressModeClamp;
	pB0.addressMode[i]  = hipAddressModeClamp;
	uA1.addressMode[i]  = hipAddressModeClamp;
	uB1.addressMode[i]  = hipAddressModeClamp;
	vA1.addressMode[i]  = hipAddressModeClamp;
	vB1.addressMode[i]  = hipAddressModeClamp;
	pA1.addressMode[i]  = hipAddressModeClamp;
	pB1.addressMode[i]  = hipAddressModeClamp;
    }

	hipBindTexture2D(0,&uA0,   u_d[0],&desc,  XDIM,  YDIM,  pitch);
	hipBindTexture2D(0,&uB0,   u_d[1],&desc,  XDIM,  YDIM,  pitch);
	hipBindTexture2D(0,&vA0,   v_d[0],&desc,  XDIM,  YDIM,  pitch);
	hipBindTexture2D(0,&vB0,   v_d[1],&desc,  XDIM,  YDIM,  pitch);
	hipBindTexture2D(0,&pA0,   p_d[0],&desc,  XDIM,  YDIM,  pitch);
	hipBindTexture2D(0,&pB0,   p_d[1],&desc,  XDIM,  YDIM,  pitch);
	hipBindTexture2D(0,&uA1, L1u_d[0],&desc,L1XDIM,L1YDIM,L1pitch);
	hipBindTexture2D(0,&uB1, L1u_d[1],&desc,L1XDIM,L1YDIM,L1pitch);
	hipBindTexture2D(0,&vA1, L1v_d[0],&desc,L1XDIM,L1YDIM,L1pitch);
	hipBindTexture2D(0,&vB1, L1v_d[1],&desc,L1XDIM,L1YDIM,L1pitch);
	hipBindTexture2D(0,&pA1, L1p_d[0],&desc,L1XDIM,L1YDIM,L1pitch);
	hipBindTexture2D(0,&pB1, L1p_d[1],&desc,L1XDIM,L1YDIM,L1pitch);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ACM_U_Shared),hipFuncCachePreferShared);

    int A,B,C,D,A1,B1;
    int its;
    A = 0; B = 1; C = 2; D = 3;
    A1= 0; B1= 1;
    struct timeval tdr0,tdr1;
    double restime;
    hipDeviceSynchronize();
    gettimeofday (&tdr0,NULL);

    //time loop
    for(int t = 0; t<1; t++){
    for(int it = 0; it<MAXIT; it++){

        L1ACM_CF<<<L1grid,L1threads,0,compute>>>(L1u_d[B1],L1v_d[B1],L1p_d[B1],0,L1pitch_e);
        for(int i = 0; i<20; i++)
        {
        L1ACM_U<<<L1u_grid,L1u_threads,0,compute>>>(L1u_d[A1],L1v_d[A1],Res_d,L1u_d[B1],L1v_d[B1],L1p_d[B1],L1u_d[C],L1v_d[C],L1u_d[D],L1v_d[D],nu,0,t,L1pitch_e);
        L1ACM_P<<<L1grid,L1threads,0,compute>>>(L1p_d[A1],Res_d,L1u_d[A1],L1v_d[A1],L1p_d[B1],0,t,L1pitch_e,L1u_d[D]);
      swap(A1,B1);
        }
        L1ACM_FC<<<grid,threads,0,compute>>>(u_d[A],v_d[A],p_d[A],0,pitch_e);
        hipDeviceSynchronize();

        for(int i = 0; i<20; i++)
        {
        swap(A,B);
        ACM_U_Shared<<<u_grid,u_threads,0,compute>>>(u_d[A],v_d[A],Res_d,u_d[B],v_d[B],p_d[B],u_d[C],v_d[C],u_d[D],v_d[D],nu,it,t,pitch_e);
        ACM_P<<<grid,threads,0,compute>>>(p_d[A],Res_d,u_d[A],v_d[A],p_d[B],it,t,pitch_e,u_d[D]);
        hipDeviceSynchronize();
        hipMemcpyAsync(&Res[t],&Res_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
        output_residual<<it<<", "<<Res[t]<<endl;
        if(it > 0){
        if(Res[t]<1 || it == MAXIT-1){
            its = it; it = MAXIT;
        }
        }
        }

    }//end iteration 
    ACM_Forces<<<grid,threads,0,compute>>>(FX_d,FY_d,u_d[A],v_d[A],p_d[A],nu,t,pitch_e);


    hipMemcpyAsync(&FX[t],&FX_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
    hipMemcpyAsync(&FY[t],&FY_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
    if(t%1000==0 && t>0) cout<<"finished time step "<<t<<endl;
    hipDeviceSynchronize();
    output_force<<t<<", "<<FX[t]<<", "<<FY[t]<<endl;
    swap(C,D);
    swap(C,A);
    }//end time loop


    hipDeviceSynchronize();
    gettimeofday (&tdr1,NULL);
    timeval_subtract (&restime, &tdr1, &tdr0);
    int Nodes;
    Nodes = XDIM*YDIM;
    cout<<"Time taken for main kernel: "<<restime<<" ("
            <<double(Nodes*double(TMAX/1000000.f))/restime<<"MNUPS)\n";

    for(int i = 0; i<4; i++){
    for(int j = 0; j<XDIM*YDIM; j++){
        u[i][j] = 1000.f;
        v[i][j] = 1000.f;
        p[i][j] = 1000.f;
    }
    }

    //Copy results from device to host
    for(int i = 0; i<4; i++){
        hipMemcpy2D(u[i],XDIM*sizeof(float),u_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(v[i],XDIM*sizeof(float),v_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(p[i],XDIM*sizeof(float),p_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(L1u[i],L1XDIM*sizeof(float),L1u_d[i],L1pitch,L1XDIM*sizeof(float),L1YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(L1v[i],L1XDIM*sizeof(float),L1v_d[i],L1pitch,L1XDIM*sizeof(float),L1YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(L1p[i],L1XDIM*sizeof(float),L1p_d[i],L1pitch,L1XDIM*sizeof(float),L1YDIM,hipMemcpyDeviceToHost);
    }
    hipMemcpy(Res,Res_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(FX,FX_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(FY,FY_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);


    WriteResults(output_results,u[0],v[0],p[0]);
    WriteResultsL1(output_results,L1u[0],L1v[0],L1p[0]);
//    WriteResults(output_results,u[3],v[3],p[3]);
    //WriteForces(output_force,FX,FY);


    output_log.close();
    output_results.close();
    output_residual.close();
    output_vel.close();

    return 0;

}
