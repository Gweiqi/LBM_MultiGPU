#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>
using namespace std;
	
#define CASENAME "speed3"

#define BLOCKSIZEX 192
#define BLOCKSIZEY 1
#define UBLOCKSIZEX 32
#define UBLOCKSIZEY 10
#define HALO 2
#define XDIM 576
#define YDIM 165
#define OBSTD 20 //.f
#define OBSTX 122 //.f
#define OBSTY 72 //.f

#define TMAX 200
#define MAXIT 5000
#define MAXRES 0.0001
#define RE 100
#define UMAX 20.f
#define BETA 0.000025f //beta = 1/c^2
#define DTAU 0.002f
#define DT 1.f

#define CONV 2  //1:UDS 2:Hybrid
#define TIMEMARCH 2  //1:Explicit Euler 2:RK2

int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ float PoisProf (float x){
	float radius = (YDIM-1-2*HALO)*0.5f;
	float result = -1.0f*(((1.0f-(x-HALO)/radius))*((1.0f-(x-HALO)/radius))-1.0f);
	return (result);
//	return 1.f;
}

void AllocateArray(float ****f,int x,int y)
{
    float ***array = new float **[4];
    for(int i = 0;i<4;i++)
    {
        array[i] = new float *[x];
        for(int j = 0;j<x;j++)
        {
            array[i][j] = new float [y];
            for(int k = 0;k<y;k++)
                array[i][j][k] = 0.f;
        }
    }
    *f = array;
}

void DeallocateArray(float ***f,int x)
{
    for(int i = 0;i<4;i++)
    {
    for(int j = 0;j<x;j++)
        delete [] f[i][j];
    delete [] f[i];
    }
    delete [] f;
}

void WriteResults(ostream &output, float *u, float *v, float *p)
{
    output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"p\"\n";
    output<<"ZONE F=POINT, I="<<XDIM-HALO*2<<", J="<<YDIM-HALO*2<<endl;
    float dx = 1;
    float dy = 1;
    float uval,vval,pval;
    for(int j = HALO; j<YDIM-HALO; j++){
    for(int i = HALO; i<XDIM-HALO; i++)
    {
        float xc = 0.5f*dx+(i)*dx;
        float yc = 0.5f*dy+(j)*dy;
        uval = 0.5f*(u[i+j*XDIM]+u[i-1  +j*XDIM]);
        vval = 0.5f*(v[i+j*XDIM]+v[i+(j-1)*XDIM]);
        pval = p[i+j*XDIM];
        if(xc>OBSTX && xc<OBSTX+OBSTD && yc>OBSTY && yc<OBSTY+OBSTD)
        {
        uval = 0.f; vval = 0.f; pval = 0.f;
        }
        output<<xc<<", "<<yc<<", "<<uval<<", "<<vval<<", "<<pval<<endl;
    }
    }
}

void WriteResiduals(ostream &output, float *Res)
{
    for(int i = 0; i<TMAX; i++)
        output<<i<<", "<<sqrt(Res[i])/((XDIM-2*HALO)*(YDIM-2*HALO))<<endl;
}

void WriteInputs(ostream &output)
{
    output<<"Domain size: \t"<<XDIM<<"x"<<YDIM<<endl;
    output<<"Halo size: \t"<<HALO<<endl;
    output<<"Target residual: \t"<<MAXRES<<endl;
    output<<"Pseudo time step size: \t"<<DTAU<<endl;
    output<<"Maximum iterations: \t"<<MAXIT<<endl;
    output<<"Real time step size: \t"<<DT<<endl;
    output<<"Maximum time steps: \t"<<TMAX<<endl;
    output<<"Re: \t"<<RE<<endl;
    output<<"uMax: \t"<<UMAX<<endl;
    string scheme;
    if(CONV == 0) scheme = "CDS   ";
    if(CONV == 1) scheme = "UDS   ";
    if(CONV == 2) scheme = "Hybrid";
    if(CONV == 3) scheme = "QUICK ";
    output<<"Convective discretization: \t"<<scheme<<endl;
}

__global__ void ACM_U_Shared_Single2(float* uA, float* vA, float* Res, float* uB, float* vB, float* pB, float* uC, float* vC, float nu, float dtau, int it, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int tx = threadIdx.x+1;
    int ty = threadIdx.y+1;
    __shared__ float u[UBLOCKSIZEX+2][UBLOCKSIZEY+2];
    __shared__ float v[UBLOCKSIZEX+2][UBLOCKSIZEY+2];


    //if(x > HALO-2 && x < XDIM-HALO && y > HALO-2 && y < YDIM-HALO){
    if(x > 0 && x < XDIM-1 && y > 0 && y < YDIM-1){

    if(threadIdx.x == 0){
    u[0][ty] = uB[ x-1+ y   *pitch];
    v[0][ty] = vB[ x-1+ y   *pitch];
    if(threadIdx.y == blockDim.y-1){
    u[0][ty+1] = uB[ x-1+ (y+1)*pitch];
    }
    }
    if(threadIdx.x == blockDim.x-1){
    u[UBLOCKSIZEX+1][ty] = uB[ x+1+ y   *pitch];
    v[UBLOCKSIZEX+1][ty] = vB[ x+1+ y   *pitch];
    }
    if(threadIdx.y == 0){
    u[tx][0] = uB[ x+  (y-1)*pitch];
    v[tx][0] = vB[ x+  (y-1)*pitch];
    if(threadIdx.x == blockDim.x-1){
    v[tx+1][0] = vB[ x+1+(y-1)*pitch];
    }
    }
    if(threadIdx.y == blockDim.y-1){
    u[tx][UBLOCKSIZEY+1] = uB[ x+ (y+1)*pitch];
    v[tx][UBLOCKSIZEY+1] = vB[ x+ (y+1)*pitch];
    }

    u[tx][ty] = uB[ x+ (y  )*pitch];
    v[tx][ty] = vB[ x+ (y  )*pitch];
    }
    syncthreads();
    //if(x > HALO-2 && x < XDIM-HALO && y > HALO-2 && y < YDIM-HALO){
    if(x > 0 && x < XDIM-1 && y > 0 && y < YDIM-1){
    float Ae,Aw,An,As;
    float AP_Ue,AE_Ue,AW_Ue,AN_Ue,AS_Ue; //A coeff for East node on u of east face
    float AP_Vn,AE_Vn,AW_Vn,AN_Vn,AS_Vn;
    float Fe,Fw,Fn,Fs;
    float De,Dw,Dn,Ds;
    float B_Ue,B_Vn;
    float dx = 1.f;///OBSTD;
    float dy = 1.f;///OBSTD;
    Ae = dy; Aw = dy; An = dx; As = dx;
    Fe = 0.5f*(u[tx  ][ty  ]+u[tx+1][ty  ])*Ae;
    Fw = 0.5f*(u[tx-1][ty  ]+u[tx  ][ty  ])*Aw;
    Fn = 0.5f*(v[tx  ][ty  ]+v[tx+1][ty  ])*An;
    Fs = 0.5f*(v[tx  ][ty-1]+v[tx+1][ty-1])*As;
    De = nu*Ae/dx; Dw = nu*Aw/dx; Dn = nu*An/dy; Ds = nu*As/dy;
    if(CONV == 1){
    AE_Ue = max(-Fe,0.f)+De;
    AW_Ue = max( Fw,0.f)+Dw;
    AN_Ue = max(-Fn,0.f)+Dn;
    AS_Ue = max( Fs,0.f)+Ds;
    AP_Ue = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
    }
    else if(CONV == 2){
    AE_Ue = max(-Fe*0.5f+De,max(-Fe,0.f));
    AW_Ue = max( Fw*0.5f+Dw,max( Fw,0.f));
    AN_Ue = max(-Fn*0.5f+Dn,max(-Fn,0.f));
    AS_Ue = max( Fs*0.5f+Ds,max( Fs,0.f));
    AP_Ue = AE_Ue+AW_Ue+AN_Ue+AS_Ue;
    }
     B_Ue = Ae*(pB[x  +(y  )*pitch]-pB[x+1+(y  )*pitch]);
     //B_Ue-=( 3.f*u[tx  ][ty  ]-4.f*uC[x  +(y  )*pitch]+uD[x  +(y  )*pitch])*0.5f/DT;
    uA[ x  + y   *pitch] =((AE_Ue*u[tx+1][ty  ]+AW_Ue*u[tx-1][ty  ]
                           +AN_Ue*u[tx  ][ty+1]+AS_Ue*u[tx  ][ty-1]
                           -AP_Ue*u[tx  ][ty  ]+ B_Ue)/(dx*dy)
                           //-( 3.f*u[tx  ][ty  ]-4.f*uC[x  +(y  )*pitch]+uD[x  +(y  )*pitch])*0.5f/DT
                           //-( u[tx  ][ty  ]-uC[x  +(y  )*pitch])*0.5f/DT
                           )*dtau
                           + uC[x+y*pitch];
    Fe = 0.5f*(u[tx  ][ty  ]+u[tx  ][ty+1])*Ae;
    Fw = 0.5f*(u[tx-1][ty  ]+u[tx-1][ty+1])*Aw;
    Fn = 0.5f*(v[tx  ][ty  ]+v[tx  ][ty+1])*An;
    Fs = 0.5f*(v[tx  ][ty  ]+v[tx  ][ty-1])*As;
    De = nu*Ae/dx; Dw = nu*Aw/dx; Dn = nu*An/dy; Ds = nu*As/dy;
    if(CONV == 1){
    AE_Vn = max(-Fe,0.f)+De;
    AW_Vn = max( Fw,0.f)+Dw;
    AN_Vn = max(-Fn,0.f)+Dn;
    AS_Vn = max( Fs,0.f)+Ds;
    AP_Vn = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
    }
    else if(CONV == 2){
    AE_Vn = max(-Fe*0.5f+De,max(-Fe,0.f));
    AW_Vn = max( Fw*0.5f+Dw,max( Fw,0.f));
    AN_Vn = max(-Fn*0.5f+Dn,max(-Fn,0.f));
    AS_Vn = max( Fs*0.5f+Ds,max( Fs,0.f));
    AP_Vn = AE_Vn+AW_Vn+AN_Vn+AS_Vn;
    }
     B_Vn = An*(pB[x  +(y  )*pitch]-pB[x  +(y+1)*pitch]);
     //B_Vn-=( 3.f*v[tx  ][ty  ]-4.f*vC[x  +(y  )*pitch]+vD[x  +(y  )*pitch])*0.5f/DT;
    vA[ x  + y   *pitch] =((AE_Vn*v[tx+1][ty  ]+AW_Vn*v[tx-1][ty  ]
                           +AN_Vn*v[tx  ][ty+1]+AS_Vn*v[tx  ][ty-1]
                           -AP_Vn*v[tx  ][ty  ]+ B_Vn)/(dx*dy)
                           //-( 3.f*v[tx  ][ty  ]-4.f*vC[x  +(y  )*pitch]+vD[x  +(y  )*pitch])*0.5f/DT
                           //-( v[tx  ][ty  ]-vC[x  +(y  )*pitch])*0.5f/DT
                           )*dtau
                           + vC[x+y*pitch];

//    if(y < HALO)
//    {
//    vA[ x  + y   *pitch] = 0.f;
//    uA[ x  + y   *pitch] = -u[tx  ][ty+1];
//    }
//    if(y > YDIM-HALO-2)
//    {
//    vA[ x  + y   *pitch] = 0.f;
//    uA[ x  + y   *pitch] = -u[tx  ][ty-1];
//    }
//    if(y > YDIM-HALO-3)
//    vA[ x  + y   *pitch] = 0.f;
//    if(x < HALO)
//    {
//    uA[ x  + y   *pitch] = UMAX*PoisProf(y);
//    vA[ x  + y   *pitch] = 0.f;
//    }
//    if(x > XDIM-HALO-2)
//    {
//    uA[ x  + y   *pitch] = u[tx-1][ty  ];
//    vA[ x  + y   *pitch] = v[tx-1][ty  ];
//    }
//    if(x > XDIM-HALO-3)
//    uA[ x  + y   *pitch] = u[tx-1][ty  ];
//
//    if(x == OBSTX-1 && y>=OBSTY & y<OBSTY+OBSTD)
//    uA[ x  + y   *pitch] = 0.f;
//    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD)
//    uA[ x  + y   *pitch] = 0.f;
//    if(y == OBSTY-1 && x>=OBSTX & x<OBSTX+OBSTD)
//    vA[ x  + y   *pitch] = 0.f;
//    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD)
//    vA[ x  + y   *pitch] = 0.f;   
//
//    if(x == OBSTX && y>=OBSTY & y<OBSTY+OBSTD-1)
//    vA[ x  + y   *pitch] = -v[tx-1][ty  ];
//    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD-1)
//    vA[ x  + y   *pitch] = -v[tx+1][ty  ];
//    if(y == OBSTY && x>=OBSTX & x<OBSTX+OBSTD-1){
//    if(t < 500) uA[ x  + y   *pitch] = u[tx][ty-1];
//    else 
//    uA[ x  + y   *pitch] = -u[tx  ][ty-1];
//    }
//    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD-1)
//    uA[ x  + y   *pitch] = -u[tx  ][ty+1];

    }
    if(x == 0 && y == 0) Res[t] = 0.f;
}

__global__ void ACM_BC(float* u, float* v, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    if(y < HALO)
    {
    v[ x  + y   *pitch] = 0.f;
    u[ x  + y   *pitch] = -u[x +(y+1)*pitch];
    }
    if(y > YDIM-HALO-2)
    {
    v[ x  + y   *pitch] = 0.f;
    u[ x  + y   *pitch] = -u[x +(y-1)*pitch];
    }
    if(y > YDIM-HALO-3)
    v[ x  + y   *pitch] = 0.f;
    if(x < HALO)
    {
    u[ x  + y   *pitch] = UMAX*PoisProf(y);
    v[ x  + y   *pitch] = 0.f;
    }
    if(x > XDIM-HALO-2)
    {
    u[ x  + y   *pitch] = u[XDIM-HALO-2+y*pitch];
    v[ x  + y   *pitch] = v[XDIM-HALO-2+y*pitch];
    }
//    if(x > XDIM-HALO-3)
//    u[ x  + y   *pitch] = u[x-1+y*pitch];

    if(x == OBSTX-1 && y>=OBSTY & y<OBSTY+OBSTD)
    u[ x  + y   *pitch] = 0.f;
    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD)
    u[ x  + y   *pitch] = 0.f;
    if(y == OBSTY-1 && x>=OBSTX & x<OBSTX+OBSTD)
    v[ x  + y   *pitch] = 0.f;
    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD)
    v[ x  + y   *pitch] = 0.f;   

    if(x == OBSTX && y>=OBSTY & y<OBSTY+OBSTD-1)
    v[ x  + y   *pitch] = -v[x-1+y*pitch];
    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD-1)
    v[ x  + y   *pitch] = -v[x+1+y*pitch];
    if(y == OBSTY && x>=OBSTX & x<OBSTX+OBSTD-1){
    if(t < 500) u[ x  + y   *pitch] = u[x+(y-1)*pitch];
    else 
    u[ x  + y   *pitch] = -u[x+(y-1)*pitch];
    }
    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD-1)
    u[ x  + y   *pitch] = -u[x+(y+1)*pitch];

}

__global__ void ACM_P(float* pA, float* Res, float* uA, float* vA, float* pB, float dtau, int it, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    __shared__ float sumRes[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();
    float dx = 1.f;///OBSTD;
    float dy = 1.f;///OBSTD;

    if(x > HALO-1 && x < XDIM-HALO && y > HALO-1 && y < YDIM-HALO){
    float res = (dy*(uA[ x  + y   *pitch]-uA[ x-1+ y   *pitch])
                                 +dx*(vA[ x  + y   *pitch]-vA[ x  +(y-1)*pitch]))/(dx*dy);
    pA[ x  + y   *pitch] = -res*dtau/BETA+pB[ x  + y   *pitch];
    if(x > HALO && x < XDIM-HALO-2 && y > HALO && y < YDIM-HALO-2 
        && abs(res)>MAXRES && !(x>= OBSTX && y>=OBSTY && x<OBSTX+OBSTD && y<OBSTY+OBSTD)){
    check[0] = 1;
    sumRes[threadIdx.x]=1.f;
    }
    else
    sumRes[threadIdx.x]=0.f;

    }
    else{
    sumRes[threadIdx.x]=0.f;
    }
    syncthreads();

    if(check[0] == 1){
    //reduction for residual
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumRes[threadIdx.x] += sumRes[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&Res[t],sumRes[0]);
    }
    }
}

__global__ void ACM_Forces1(float *FX_intm, float *FY_intm, float* uD, float* vD, float* pA, float nu, int t, size_t pitch, float *test)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int xcoord = x+OBSTX;
    int ycoord = y+OBSTY;
    __shared__ float sumFX[OBSTD],sumFY[OBSTD];
    sumFX[threadIdx.x] = 0.f;
    sumFY[threadIdx.x] = 0.f;
    syncthreads();
    float dx = 1.f;///OBSTD;
    float dy = 1.f;///OBSTD;

    //forces on bottom wall
    if(y == 0){
    sumFY[threadIdx.x] += dx*pA[ xcoord  +(ycoord-1)*pitch];
    sumFX[threadIdx.x] -= dx*nu*2.f*uD[ xcoord  +(ycoord  )*pitch]/dy;
    }
    //forces on top wall
    if(y == OBSTD-1){
    sumFY[threadIdx.x] -= dx*pA[ xcoord  +(ycoord+1)*pitch];
    sumFX[threadIdx.x] -= dx*nu*2.f*uD[ xcoord  +(ycoord  )*pitch]/dy;
    }
    //forces on left wall
    if(x == 0){
    sumFX[threadIdx.x] += dy*pA[ xcoord-1+(ycoord  )*pitch];
    sumFY[threadIdx.x] -= dy*nu*2.f*vD[ xcoord  +(ycoord  )*pitch]/dy;
    }
    //forces on right wall
    if(x == OBSTD-1){
    sumFX[threadIdx.x] -= dy*pA[ xcoord+1+(ycoord  )*pitch];
    sumFY[threadIdx.x] -= dy*nu*2.f*vD[ xcoord  +(ycoord  )*pitch]/dx;
    }

    syncthreads();

    //reduction for force
//    int nTotalThreads = blockDim.x;
//    while(nTotalThreads > 1){
//        int halfPoint = (nTotalThreads >> 1);
//        if(threadIdx.x < halfPoint){
//            sumFX[threadIdx.x] += sumFX[threadIdx.x+halfPoint];
//            sumFY[threadIdx.x] += sumFY[threadIdx.x+halfPoint];
//        }
//        syncthreads();
//        nTotalThreads = halfPoint;
//    }
    float sum = 0;
    if(threadIdx.x == 0){
    for(int i = 0; i<blockDim.x; i++)
        sum += sumFX[i];
    }
    sumFX[0] = sum;
    sum = 0;
    if(threadIdx.x == 0){
    for(int i = 0; i<blockDim.x; i++)
        sum += sumFY[i];
    }
    sumFY[0] = sum;
    if(threadIdx.x == 0){
        FX_intm[y] = sumFX[0];
        FY_intm[y] = sumFY[0];
    }


}

__global__ void ACM_Forces2(float *FX, float *FY, float *FX_intm, float *FY_intm, int t, size_t pitch, float *test)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int xcoord = x+OBSTX;
    int ycoord = y+OBSTY;
    __shared__ float sumFX[OBSTD],sumFY[OBSTD];
    sumFX[threadIdx.y] = 0.f;
    sumFY[threadIdx.y] = 0.f;
    syncthreads();
    
    sumFX[threadIdx.y] = FX_intm[threadIdx.y];
    sumFY[threadIdx.y] = FY_intm[threadIdx.y];
    //test[xcoord+ycoord*pitch] = 1.f;
    
    syncthreads();

    float sum = 0;
    if(threadIdx.y == 0){
    for(int i = 0; i<blockDim.y; i++)
        sum += sumFX[i];
    }
        sumFX[0] = sum;
    sum = 0;
    if(threadIdx.y == 0){
    for(int i = 0; i<blockDim.y; i++)
        sum += sumFY[i];
    }
        sumFY[0] = sum;
    if(threadIdx.y == 0){
        FX[t] = sumFX[0];
        FY[t] = sumFY[0];
    }


}

int main()
{
    
    ofstream output_log,output_results,output_residual,output_vel,output_force;
    float nu = UMAX*OBSTD/RE;
    float Ma = UMAX*sqrt(BETA);
    cout<<"Ma = "<<Ma<<endl;
    string FileName = CASENAME;
    output_log.open ((FileName+".log").c_str());
    output_results.open ((FileName+".dat").c_str());
    output_residual.open ((FileName+".res").c_str());
    output_vel.open ((FileName+".vel").c_str());
    output_force.open ((FileName+".frc").c_str());
    //write input parameters to console and log file
    WriteInputs(cout);
    WriteInputs(output_log);

    //allocate and initialize arrays
    float *u[4],*v[4],*p[4],*Res,*FX,*FY;
    for(int i = 0; i<4; i++){
        u[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
        v[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
        p[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
    }
    Res = (float *)malloc(TMAX*sizeof(float));
    FX = (float *)malloc(TMAX*sizeof(float));
    FY = (float *)malloc(TMAX*sizeof(float));
    //initialize host memory
    for(int i = 0; i<4; i++){
    for(int j = 0; j<XDIM*YDIM; j++){
        u[i][j] = UMAX;
        v[i][j] = 0.f;
        p[i][j] = 0.f;
    }
    }
    for(int j = 0; j<TMAX; j++){
        Res[j] = 0.f;
        FX[j] = 0.f;
        FY[j] = 0.f;
    }

    //size_t memsize, memsize2;
    size_t pitch = 2;
    while(pitch<XDIM)
        pitch=pitch*2;
    pitch *= sizeof(float);//pitch*sizeof(float);
    size_t pitch_e = pitch/sizeof(float);
    cout<<"Pitch (in elements): "<<pitch/sizeof(float)<<endl;

    dim3 u_threads(UBLOCKSIZEX, UBLOCKSIZEY,1);
    dim3 u_grid  (((XDIM+UBLOCKSIZEX-1)/UBLOCKSIZEX),((YDIM+UBLOCKSIZEY-1)/UBLOCKSIZEY),1);
    dim3 threads(BLOCKSIZEX, BLOCKSIZEY,1);
    dim3 grid  (((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);
    dim3 f1_threads (OBSTD,1,1);
    dim3 f1_grid    (1,OBSTD,1);
    dim3 f2_threads (1,OBSTD,1);
    dim3 f2_grid    (1,1,1);

    hipStream_t compute;
    hipStream_t transfer;
    hipStreamCreate(&compute);
    hipStreamCreate(&transfer);

    float *u_d[4],*v_d[4],*p_d[4], *Res_d, *FX_d, *FY_d;
    float *FX_intm, *FY_intm;
    float *test;
    for(int i = 0; i<4; i++){
        hipMalloc((void **) &u_d[i], pitch_e*YDIM*sizeof(float));
        hipMalloc((void **) &v_d[i], pitch_e*YDIM*sizeof(float));
        hipMalloc((void **) &p_d[i], pitch_e*YDIM*sizeof(float));
    }
    hipMalloc((void **) &test, pitch_e*YDIM*sizeof(float));
    hipMalloc((void **) &FX_intm, int(OBSTD)*sizeof(float));
    hipMalloc((void **) &FY_intm, int(OBSTD)*sizeof(float));
    hipMalloc((void **) &Res_d,TMAX*sizeof(float));
    hipMalloc((void **) &FX_d, TMAX*sizeof(float));
    hipMalloc((void **) &FY_d, TMAX*sizeof(float));
    for(int i = 0; i<4; i++){
        hipMemcpy2D(u_d[i],pitch,u[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(v_d[i],pitch,v[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(p_d[i],pitch,p[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
    }
    hipMemcpy(Res_d,Res,TMAX*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(FX_d,FX,TMAX*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(FY_d,FY,TMAX*sizeof(float),hipMemcpyHostToDevice);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ACM_U_Shared_Single2),hipFuncCachePreferShared);

    int A,B,C,D;
    int its;
    A = 0; B = 1; C = 2; D = 3;
    struct timeval tdr0,tdr1;
    double restime;
    hipDeviceSynchronize();
    gettimeofday (&tdr0,NULL);

    //time loop
    for(int t = 0; t<TMAX; t++){
    //for(int it = 0; it<MAXIT; it++){
        swap(A,B);
        int it = 10;
        //if(it > 0)
        if(TIMEMARCH == 1){
        //ACM_U_Shared_Single<<<u_grid,u_threads,0,compute>>>(u_d[B],v_d[B],Res_d,u_d[A],v_d[A],p_d[A],nu,it,t,pitch_e);
        ACM_U_Shared_Single2<<<u_grid,u_threads,0,compute>>>(u_d[B],v_d[B],Res_d,u_d[A],v_d[A],p_d[A],u_d[A],v_d[A],nu,DTAU,it,t,pitch_e);
        ACM_BC<<<grid,threads,0,compute>>>(u_d[B],v_d[B],t,pitch_e);
        ACM_P<<<grid,threads,0,compute>>>(p_d[B],Res_d,u_d[B],v_d[B],p_d[A],DTAU,it,t,pitch_e);
        }
        else if(TIMEMARCH == 2){
        ACM_U_Shared_Single2<<<u_grid,u_threads,0,compute>>>(u_d[C],v_d[C],Res_d,u_d[A],v_d[A],p_d[A],u_d[A],v_d[A],nu,DTAU*0.5f,it,t,pitch_e);
        ACM_BC<<<grid,threads,0,compute>>>(u_d[C],v_d[C],t,pitch_e);
        ACM_P<<<grid,threads,0,compute>>>(p_d[C],Res_d,u_d[C],v_d[C],p_d[A],DTAU*0.5f,it,t,pitch_e);
        ACM_U_Shared_Single2<<<u_grid,u_threads,0,compute>>>(u_d[B],v_d[B],Res_d,u_d[C],v_d[C],p_d[C],u_d[A],v_d[A],nu,DTAU,it,t,pitch_e);
        ACM_BC<<<grid,threads,0,compute>>>(u_d[B],v_d[B],t,pitch_e);
        ACM_P<<<grid,threads,0,compute>>>(p_d[B],Res_d,u_d[B],v_d[B],p_d[A],DTAU,it,t,pitch_e);
        }
        hipDeviceSynchronize();
        hipMemcpyAsync(&Res[t],&Res_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
    //    if(it > 0){
    //    //if(sqrt(Res[t])/float((XDIM-2*HALO)*(YDIM-2*HALO)) < MAXRES || it == MAXIT-1){
    //    if(Res[t]<1 || it == MAXIT-1){
    //        its = it; it = MAXIT;
    //    }
    //    }
    //}//end iteration 

    ACM_Forces1<<<f1_grid,f1_threads,0,compute>>>(FX_intm,FY_intm,u_d[B],v_d[B],p_d[B],nu,t,pitch_e,test);
    ACM_Forces2<<<f2_grid,f2_threads,0,compute>>>(FX_d,FY_d,FX_intm,FY_intm,t,pitch_e,test);


    hipMemcpyAsync(&FX[t],&FX_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
    hipMemcpyAsync(&FY[t],&FY_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
    if(t%1000==0 && t>0) cout<<"finished time step "<<t<<endl;
    hipDeviceSynchronize();
    //output_residual<<t<<", "<<its<<", "<<sqrt(Res[t])/((XDIM-2*HALO)*(YDIM-2*HALO))<<endl;
    output_residual<<t<<", "<<its<<", "<<Res[t]<<endl;
    output_force<<t<<", "<<FX[t]/0.5f<<", "<<FY[t]/0.5f<<endl;
    //swap(C,D);
    //swap(C,A);
    //cout<<A<<", "<<B<<" "<<C<<", "<<D<<endl;
    }//end time loop


    hipDeviceSynchronize();
    gettimeofday (&tdr1,NULL);
    timeval_subtract (&restime, &tdr1, &tdr0);
    int Nodes;
    Nodes = XDIM*YDIM;
    cout<<"Time taken for main kernel: "<<restime<<" ("
            <<double(Nodes*double(TMAX/1000000.f))/restime<<"MNUPS)\n";

    for(int i = 0; i<4; i++){
    for(int j = 0; j<XDIM*YDIM; j++){
        u[i][j] = 1000.f;
        v[i][j] = 1000.f;
        p[i][j] = 1000.f;
    }
    }

    //Copy results from device to host
    for(int i = 0; i<4; i++){
        hipMemcpy2D(u[i],XDIM*sizeof(float),u_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(v[i],XDIM*sizeof(float),v_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(p[i],XDIM*sizeof(float),p_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
    }
    //hipMemcpy2D(p[0],XDIM*sizeof(float),test,pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
    hipMemcpy(Res,Res_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(FX,FX_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(FY,FY_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);


    WriteResults(output_results,u[0],v[0],p[0]);
//    WriteResults(output_results,u[3],v[3],p[3]);
    //WriteForces(output_force,FX,FY);


    output_log.close();
    output_results.close();
    output_residual.close();
    output_vel.close();

    return 0;

}
