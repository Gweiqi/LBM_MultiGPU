#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>
using namespace std;
	
#define CASENAME "test5"

#define BLOCKSIZEX 128
#define BLOCKSIZEY 1
#define XDIM 256
#define YDIM 200
#define HALO 3
#define TMAX 50000
#define MAXIT 500
#define MAXRES 0.0001
#define RE 20000
#define UMAX 1.f
#define BETA 0.05f
#define DTAU 0.05f
#define DT 0.1f

#define OBSTD 20.f
#define OBSTX 90.f
#define OBSTY 90.f

#define CONV 1  //1:UDS

int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}


void AllocateArray(float ****f,int x,int y)
{
    float ***array = new float **[4];
    for(int i = 0;i<4;i++)
    {
        array[i] = new float *[x];
        for(int j = 0;j<x;j++)
        {
            array[i][j] = new float [y];
            for(int k = 0;k<y;k++)
                array[i][j][k] = 0.f;
        }
    }
    *f = array;
}

void DeallocateArray(float ***f,int x)
{
    for(int i = 0;i<4;i++)
    {
    for(int j = 0;j<x;j++)
        delete [] f[i][j];
    delete [] f[i];
    }
    delete [] f;
}

void WriteResults(ostream &output, float *u, float *v, float *p)
{
    output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"p\"\n";
    output<<"ZONE F=POINT, I="<<XDIM-HALO*2<<", J="<<YDIM-HALO*2<<endl;
    float dx = 1;
    float dy = 1;
    float uval,vval,pval;
    for(int j = HALO; j<YDIM-HALO; j++){
    for(int i = HALO; i<XDIM-HALO; i++)
    {
        float xc = 0.5f*dx+(i)*dx;
        float yc = 0.5f*dy+(j)*dy;
        uval = 0.5f*(u[i+j*XDIM]+u[i-1  +j*XDIM]);
        vval = 0.5f*(v[i+j*XDIM]+v[i+(j-1)*XDIM]);
        pval = p[i+j*XDIM];
        if(xc>OBSTX && xc<OBSTX+OBSTD && yc>OBSTY && yc<OBSTY+OBSTD)
        {
        uval = 0.f; vval = 0.f; pval = 0.f;
        }
        output<<xc<<", "<<yc<<", "<<uval<<", "<<vval<<", "<<pval<<endl;
    }
    }
}

void WriteResiduals(ostream &output, float *Res)
{
    for(int i = 0; i<TMAX; i++)
        output<<i<<", "<<sqrt(Res[i])/((XDIM-2*HALO)*(YDIM-2*HALO))<<endl;
}

void WriteInputs(ostream &output)
{
    output<<"Domain size: \t"<<XDIM<<"x"<<YDIM<<endl;
    output<<"Halo size: \t"<<HALO<<endl;
    output<<"Target residual: \t"<<MAXRES<<endl;
    output<<"Maximum iterations: \t"<<MAXIT<<endl;
    output<<"Re: \t"<<RE<<endl;
    output<<"uMax: \t"<<UMAX<<endl;
    string scheme;
    if(CONV == 0) scheme = "CDS   ";
    if(CONV == 1) scheme = "UDS   ";
    if(CONV == 2) scheme = "Hybrid";
    if(CONV == 3) scheme = "QUICK ";
    output<<"Convective discretization: \t"<<scheme<<endl;
}


__global__ void ACM_U(float* uA, float* vA, float* Res, float* uB, float* vB, float* pB, float* uC, float* vC, float* uD, float* vD, float nu, int it, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;

    if(x > HALO-2 && x < XDIM-HALO && y > HALO-2 && y < YDIM-HALO){
    float AP_Ue,AE_Ue,AW_Ue,AN_Ue,AS_Ue; //A coeff for East node on u of east face
    float AP_Vn,AE_Vn,AW_Vn,AN_Vn,AS_Vn;
    //float Ae,Aw,An,As;
    float Fe,Fw,Fn,Fs;
    float De,Dw,Dn,Ds;
    float B_Ue,B_Vn;
    Fe = 0.5f*(uB[ x  + y   *pitch]+uB[ x+1+ y   *pitch]);
    Fw = 0.5f*(uB[ x-1+ y   *pitch]+uB[ x  + y   *pitch]);
    Fn = 0.5f*(vB[ x  + y   *pitch]+vB[ x+1+ y   *pitch]);
    Fs = 0.5f*(vB[ x  +(y-1)*pitch]+vB[ x+1+(y-1)*pitch]);
    De = nu; Dw = nu; Dn = nu; Ds = nu;
    AE_Ue = max(-Fe,0.f)+De;
    AW_Ue = max( Fw,0.f)+Dw;
    AN_Ue = max(-Fn,0.f)+Dn;
    AS_Ue = max( Fs,0.f)+Ds;
    AP_Ue = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
     B_Ue = (pB[x  +(y  )*pitch]-pB[x+1+(y  )*pitch]);
    uA[ x  + y   *pitch] =( AE_Ue*uB[x+1+(y  )*pitch]+AW_Ue*uB[x-1+(y  )*pitch]
                           +AN_Ue*uB[x  +(y+1)*pitch]+AS_Ue*uB[x  +(y-1)*pitch]
                           -AP_Ue*uB[x  +(y  )*pitch]+ B_Ue
                           -(3.f*uB[x  +(y  )*pitch]-4.f*uC[x  +(y  )*pitch]+uD[x  +(y  )*pitch])*0.5f/DT)*DTAU
                           + uB[x  +(y  )*pitch];
    Fe = 0.5f*(uB[ x  + y   *pitch]+uB[ x  +(y+1)*pitch]);
    Fw = 0.5f*(uB[ x-1+ y   *pitch]+uB[ x-1+(y+1)*pitch]);
    Fn = 0.5f*(vB[ x  + y   *pitch]+vB[ x  +(y+1)*pitch]);
    Fs = 0.5f*(vB[ x  +(y  )*pitch]+vB[ x  +(y-1)*pitch]);
    De = nu; Dw = nu; Dn = nu; Ds = nu;
    AE_Vn = max(-Fe,0.f)+De;
    AW_Vn = max( Fw,0.f)+Dw;
    AN_Vn = max(-Fn,0.f)+Dn;
    AS_Vn = max( Fs,0.f)+Ds;
    AP_Vn = max( Fe,0.f)+max(-Fw,0.f)+max( Fn,0.f)+max(-Fs,0.f)+Dw+De+Dn+Ds;
     B_Vn = (pB[x  +(y  )*pitch]-pB[x  +(y+1)*pitch]);
    vA[ x  + y   *pitch] =( AE_Vn*vB[x+1+(y  )*pitch]+AW_Vn*vB[x-1+(y  )*pitch]
                           +AN_Vn*vB[x  +(y+1)*pitch]+AS_Vn*vB[x  +(y-1)*pitch]
                           -AP_Vn*vB[x  +(y  )*pitch]+ B_Vn
                           -(3.f*vB[x  +(y  )*pitch]-4.f*vC[x  +(y  )*pitch]+vD[x  +(y  )*pitch])*0.5f/DT)*DTAU
                           + vB[x  +(y  )*pitch];
    if(y < HALO)
    {
    vA[ x  + y   *pitch] = 0.f;
    uA[ x  + y   *pitch] = uB[ x  + (y+1)*pitch];
    }
    //if(j == yDim+halo-1)
    if(y > YDIM-HALO-2)
    {
    vA[ x  + y   *pitch] = 0.f;
    uA[ x  + (y  )*pitch] = uB[ x  + (y-1)*pitch];
    }
    if(x < HALO)
    {
    uA[ x  + y   *pitch] = UMAX;
    vA[ x  + y   *pitch] = 0.f;
    }
    if(x > XDIM-HALO-2)
    {
    uA[ x  + y   *pitch] = uB[ x-1+ y   *pitch];
    vA[ x  + y   *pitch] = vB[ x-1+ y   *pitch];//x+1?? probly not
    }


    //if(x == 20 && y == 20)
    //uA[ x  + y   *pitch] = x;
    //vA[ x  + y   *pitch] = y;

    if(x == OBSTX-1 && y>=OBSTY & y<OBSTY+OBSTD)
    uA[ x  + y   *pitch] = 0.f;
    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD)
    uA[ x  + y   *pitch] = 0.f;
    if(y == OBSTY-1 && x>=OBSTX & x<OBSTX+OBSTD)
    vA[ x  + y   *pitch] = 0.f;
    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD)
    vA[ x  + y   *pitch] = 0.f;   

    if(x == OBSTX && y>=OBSTY & y<OBSTY+OBSTD-1)
    vA[ x  + y   *pitch] = -vB[ x-1+ y   *pitch];
    if(x == OBSTX+OBSTD-1 && y>=OBSTY && y<OBSTY+OBSTD-1)
    vA[ x  + y   *pitch] = -vB[ x+1+ y   *pitch];
    if(y == OBSTY && x>=OBSTX & x<OBSTX+OBSTD-1){
    if(t < 200) uA[ x  + y   *pitch] = uB[ x  +(y-1)*pitch];
    else 
    uA[ x  + y   *pitch] = -uB[ x  +(y-1)*pitch];
    }
    if(y == OBSTY+OBSTD-1 && x>=OBSTX && x<OBSTX+OBSTD-1)
    uA[ x  + y   *pitch] = -uB[ x  +(y+1)*pitch];






    }
    if(x == 0 && y == 0) Res[t] = 0.f;
}


__global__ void ACM_P(float* pA, float* Res, float* uA, float* vA, float* pB, int it, int t, size_t pitch, float *uD)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    __shared__ float sumRes[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();

    if(x > HALO-2 && x < XDIM-HALO && y > HALO-2 && y < YDIM-HALO){
    float res = ((uA[ x  + y   *pitch]-uA[ x-1+ y   *pitch])
                                 +(vA[ x  + y   *pitch]-vA[ x  +(y-1)*pitch]));
    pA[ x  + y   *pitch] = -DTAU*res/BETA+pB[ x  + y   *pitch];
    if(x > HALO && x < XDIM-HALO-2 && y > HALO && y < YDIM-HALO-2 
        && abs(res)>MAXRES && !(x>= OBSTX && y>=OBSTY && x<OBSTX+OBSTD && y<OBSTY+OBSTD)){
    check[0] = 1;
    sumRes[threadIdx.x]=1.f;
    }
    else
    sumRes[threadIdx.x]=0.f;

    }
    else{
    sumRes[threadIdx.x]=0.f;
    }
    syncthreads();

    if(check[0] == 1){
    //reduction for residual
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumRes[threadIdx.x] += sumRes[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&Res[t],sumRes[0]);
    }
    }
}

__global__ void ACM_VelTransfer(float* uD, float* vD, float* uA, float* vA, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    uD[ x  + y   *pitch] = uA[ x  + y   *pitch];
    vD[ x  + y   *pitch] = vA[ x  + y   *pitch];
}

__global__ void ACM_Forces(float *FX, float *FY, float* uD, float* vD, float* pA, float nu, int t, size_t pitch)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    __shared__ float sumFX[BLOCKSIZEX],sumFY[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    sumFX[threadIdx.x] = 0.f;
    sumFY[threadIdx.x] = 0.f;
    syncthreads();

    if(x >= OBSTX && x<OBSTX+OBSTD && y == OBSTY){
    check[0] = 1;
    sumFY[threadIdx.x] += pA[ x  +(y-1)*pitch];
    if(x<OBSTX+OBSTD-1)
        sumFX[threadIdx.x] -= nu*2.f*uD[ x  +(y  )*pitch];
    }
    if(x >= OBSTX && x<OBSTX+OBSTD && y == OBSTY+OBSTD-1){
    check[0] = 1;
    sumFY[threadIdx.x] -= pA[ x  +(y+1)*pitch];
    if(x<OBSTX+OBSTD-1)
        sumFX[threadIdx.x] -= nu*2.f*uD[ x  +(y  )*pitch];
    }

    if(y >= OBSTY && y<OBSTY+OBSTD && x == OBSTX){
    check[0] = 1;
    sumFX[threadIdx.x] += pA[ x-1+(y  )*pitch];
    if(y<OBSTY+OBSTD-1)
        sumFY[threadIdx.x] -= nu*2.f*vD[ x  +(y  )*pitch];
    }
    if(y >= OBSTY && y<OBSTY+OBSTD && x == OBSTX+OBSTD-1){
    check[0] = 1;
    sumFX[threadIdx.x] -= pA[ x+1+(y  )*pitch];
    if(y<OBSTX+OBSTD-1)
        sumFY[threadIdx.x] -= nu*2.f*vD[ x  +(y  )*pitch];
    }
    syncthreads();

    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumFX[threadIdx.x] += sumFX[threadIdx.x+halfPoint];
            sumFY[threadIdx.x] += sumFY[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t],sumFX[0]);
        atomicAdd(&FY[t],sumFY[0]);
    }
    }


}

int main()
{
    
    ofstream output_log,output_results,output_residual,output_vel,output_force;
    float nu = UMAX*OBSTD/RE;
    float Ma = UMAX*BETA*BETA;
    cout<<"Ma = "<<Ma<<endl;
    string FileName = CASENAME;
    output_log.open ((FileName+".log").c_str());
    output_results.open ((FileName+".dat").c_str());
    output_residual.open ((FileName+".res").c_str());
    output_vel.open ((FileName+".vel").c_str());
    output_force.open ((FileName+".frc").c_str());
    //write input parameters to console and log file
    WriteInputs(cout);
    WriteInputs(output_log);

    //allocate and initialize arrays
    float *u[4],*v[4],*p[4],*Res,*FX,*FY;
    for(int i = 0; i<4; i++){
        u[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
        v[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
        p[i] = (float *)malloc(XDIM*YDIM*sizeof(float));
    }
    Res = (float *)malloc(TMAX*sizeof(float));
    FX = (float *)malloc(TMAX*sizeof(float));
    FY = (float *)malloc(TMAX*sizeof(float));
    //initialize host memory
    for(int i = 0; i<4; i++){
    for(int j = 0; j<XDIM*YDIM; j++){
        u[i][j] = 0.f;
        v[i][j] = 0.f;
        p[i][j] = 0.f;
    }
    }
    for(int j = 0; j<TMAX; j++){
        Res[j] = 0.f;
        FX[j] = 0.f;
        FY[j] = 0.f;
    }

    //size_t memsize, memsize2;
    size_t pitch = 2;
    while(pitch<XDIM)
        pitch=pitch*2;
    pitch *= sizeof(float);//pitch*sizeof(float);
    size_t pitch_e = pitch/sizeof(float);
    cout<<"Pitch (in elements): "<<pitch/sizeof(float)<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY,1);
    dim3 grid  (((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);
    hipStream_t compute;
    hipStream_t transfer;
    hipStreamCreate(&compute);
    hipStreamCreate(&transfer);

    float *u_d[4],*v_d[4],*p_d[4], *Res_d, *FX_d, *FY_d;
    for(int i = 0; i<4; i++){
        hipMalloc((void **) &u_d[i], pitch_e*YDIM*sizeof(float));
        hipMalloc((void **) &v_d[i], pitch_e*YDIM*sizeof(float));
        hipMalloc((void **) &p_d[i], pitch_e*YDIM*sizeof(float));
    }
    hipMalloc((void **) &Res_d, TMAX*sizeof(float));
    hipMalloc((void **) &FX_d, TMAX*sizeof(float));
    hipMalloc((void **) &FY_d, TMAX*sizeof(float));
    for(int i = 0; i<4; i++){
        hipMemcpy2D(u_d[i],pitch,u[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(v_d[i],pitch,v[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
        hipMemcpy2D(p_d[i],pitch,p[i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM,hipMemcpyHostToDevice);
    }
    hipMemcpy(Res_d,Res,TMAX*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(FX_d,FX,TMAX*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(FY_d,FY,TMAX*sizeof(float),hipMemcpyHostToDevice);

    int A,B,C,D;
    int its;
    A = 0; B = 1; C = 2; D = 3;
    struct timeval tdr0,tdr1;
    double restime;
    hipDeviceSynchronize();
    gettimeofday (&tdr0,NULL);

    //time loop
    for(int t = 0; t<TMAX; t++){
    for(int it = 0; it<MAXIT; it++){
        if(it > 0)
        hipMemcpyAsync(&Res[t],&Res_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
        ACM_U<<<grid,threads,0,compute>>>(u_d[A],v_d[A],Res_d,u_d[B],v_d[B],p_d[B],u_d[C],v_d[C],u_d[D],v_d[D],nu,it,t,pitch_e);
        hipStreamSynchronize(compute);
        ACM_P<<<grid,threads,0,compute>>>(p_d[A],Res_d,u_d[A],v_d[A],p_d[B],it,t,pitch_e,u_d[D]);
        hipDeviceSynchronize();
        if(it > 0){
        //if(sqrt(Res[t])/float((XDIM-2*HALO)*(YDIM-2*HALO)) < MAXRES || it == MAXIT-1){
        if(Res[t]<1 || it == MAXIT-1){
            its = it; it = MAXIT;
        }
        }
        swap(A,B);
    }//end iteration 
    //ACM_VelTransfer<<<grid,threads,0,compute>>>(u_d[C],v_d[C],u_d[A],v_d[A],pitch_e);
    swap(C,D);
    swap(C,A);
    ACM_Forces<<<grid,threads,0,compute>>>(FX_d,FY_d,u_d[C],v_d[C],p_d[A],nu,t,pitch_e);
    hipMemcpyAsync(&FX[t],&FX_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
    hipMemcpyAsync(&FY[t],&FY_d[t],sizeof(float),hipMemcpyDeviceToHost,compute);
    if(t%1000==0 && t>0) cout<<"finished time step "<<t<<endl;
    hipDeviceSynchronize();
    //output_residual<<t<<", "<<its<<", "<<sqrt(Res[t])/((XDIM-2*HALO)*(YDIM-2*HALO))<<endl;
    output_residual<<t<<", "<<its<<", "<<Res[t]<<endl;
    output_force<<t<<", "<<FX[t]<<", "<<FY[t]<<endl;
    }//end time loop
    hipDeviceSynchronize();
    gettimeofday (&tdr1,NULL);
    timeval_subtract (&restime, &tdr1, &tdr0);
    int Nodes;
    Nodes = XDIM*YDIM;
    cout<<"Time taken for main kernel: "<<restime<<" ("
            <<double(Nodes*double(TMAX/1000000.f))/restime<<"MNUPS)\n";

    for(int i = 0; i<4; i++){
    for(int j = 0; j<XDIM*YDIM; j++){
        u[i][j] = 1000.f;
        v[i][j] = 1000.f;
        p[i][j] = 1000.f;
    }
    }

    //Copy results from device to host
    for(int i = 0; i<4; i++){
        hipMemcpy2D(u[i],XDIM*sizeof(float),u_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(v[i],XDIM*sizeof(float),v_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
        hipMemcpy2D(p[i],XDIM*sizeof(float),p_d[i],pitch,XDIM*sizeof(float),YDIM,hipMemcpyDeviceToHost);
    }
    hipMemcpy(Res,Res_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(FX,FX_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(FY,FY_d,TMAX*sizeof(float),hipMemcpyDeviceToHost);


    WriteResults(output_results,u[0],v[0],p[0]);
    WriteResults(output_results,u[3],v[3],p[3]);
    //WriteForces(output_force,FX,FY);


    output_log.close();
    output_results.close();
    output_residual.close();
    output_vel.close();

    return 0;

}
