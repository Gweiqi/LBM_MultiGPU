#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define CASENAME "Test31"

#define BLOCKSIZEX 4
#define BLOCKSIZEY 1
#define BLOCKSIZEZ 1
#define BLOCKSIZELRX 64
#define BLOCKSIZELRY 1
#define BLOCKSIZELRZ 1
#define XDIM 4
#define YDIM 4
#define ZDIM 10
#define TMAX 10
#define STARTF 0

#define OBSTR1 8.f
#define OBSTX1 127.5f
#define OBSTY1 95.5f
#define OBSTZ1 15.5f

#define OBSTR2 4.f
#define OBSTX2 63.5f
#define OBSTY2 31.5f
#define OBSTZ2 31.5f

#define RE 100.f//2000.f//100.f;
#define UMAX 0.04f
#define METHOD "SINGLE" //SINGLE,HYB,TEXT,SHARED,CACHE
#define SmagLES "YES" //YES,NO
#define MODEL "MRT" //BGK,MRT,STREAM
#define ZPERIODIC "NO"
#define CS 0.04f
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>

/*
Image List:
0  fluid
1  BB
2
3  DirichletWest(simple)
10 BB(force)
13 DirichletWest_Reg
14 NeumannEast_Reg
15 DirichletNorth_Reg
16 DirichletSouth_Reg
21 ysymmetry_top
22 ysymmetry_bot
23 zsymmetry_top
24 zsymmetry_bot
25 xsymmetry_top
26 xsymmetry_bot
*/
inline __device__ int ImageFcn(float x, float y, float z){
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//		return 10;
//	else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//		return 10;
	//if(((x-OBSTX)*(x-OBSTX)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1)+(z-OBSTZ1)*(z-OBSTZ1))<OBSTR1*OBSTR1)
//	{
//		return 10;
//	}
//	else
//	//if(y < 0.1f || z < 0.1f || (XDIM-x) < 0.1f || (YDIM-y) < 0.1f || (ZDIM-z) < 0.1f)
//	if(y < 17.5f || z < 17.5f || y > 46.5f || z > 46.5f)
//		return 1;
//	else if(x < 17.5f)
//		return 13;
//	else if(x > 78.5f)
//		return 14;
//	else
    
    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
        return 10;
    else
		return 0;
}

inline __device__ int ImageFcn(int x, int y, int z){
//    int value = 0;
//Cylinder
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//		value = 10;
//	else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//		value = 10;
//Sphere
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1)+(z-OBSTZ1)*(z-OBSTZ1))<OBSTR1*OBSTR1)
//	{
////		if(z == 0 || z == ZDIM-1)
////		return 1;
////		else
//		return 10;
//	}
//	if(z == 0)
//		value = 0;
//	else if(z == ZDIM-1)
//		value = 0;

//    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
//        value = 10;
//	else if(y == 0)
//		value = 200;//22;
//	else if(y == YDIM-1)
//		value = 100;
//	else if(x == 0)
//		value = 26;
//	else if(x == XDIM-1)
//		value = 25;
//	else if(z == 0)
//		value = 0;
//	else if(z == ZDIM-1)
//		value = 0;

    //return value;

//Lid Driven Cavity
	if(x == 0 || x == XDIM-1 || y == 0 || y == YDIM-1 || z == 0 || z == ZDIM-1)
		return 1;
//	else if(x == XDIM-2 || y == 1 || y == YDIM-2 || z == 1 || z == ZDIM-2)
//		return 1;
//	else if(x == 0)
//		return 1;
//	else if(x == 1)
//		return 53;
	else
		return 0;
}

inline __device__ float PoisProf (float x){
	float radius = (YDIM-1-1)*0.5f;
	float result = -1.0f*(((1.0f-(x-0.5f)/radius))*((1.0f-(x-0.5f)/radius))-1.0f);
	return (result);
//	return 1.f;
}

__device__ void DirichletWest(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(y == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(y == YDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
//	if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
    }
	float u,v,w;//,rho;
    u = UMAX;//*PoisProf(zcoord)*1.5;
    v = 0.0f;
	w = 0.0f;
    
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//    float usqr = u*u+v*v+w*w;

	f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
	f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
////	f0 = 1.0f/3.0f*(rho-1.5f*usqr);
//	f1 = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
////	f2 = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
////	f3 = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
////	f4 = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//	f5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
////	f6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
////	f7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//	f8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
////	f9 = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//	f10= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
////	f11= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
////	f12= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
////	f13= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
////	f14= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//	f15= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
////	f16= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
////	f17= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
////	f18= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}

__device__ void DirichletWest_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0){
		f2 = f4;
		f6 = f7;
		f11 = f13;
		f16 = f18;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f7 = f6;
		f13 = f11;
		f18 = f16;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;//*PoisProf(y)*1.5;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

//	f1 =(0.166666667f*u)+
//		(f3-(-(0.166666667f*u)));
	f1 = f3+0.33333333f*u;
//	f5 =(0.0833333333f*( u+v))+
//		(f7-(0.0833333333f*(-u-v)));
	f5 = f7+0.166666667f*(u+v);
//	f8 =(0.0833333333f*( u-v  ))+
//		(f6-(0.0833333333f*(-u+v  )));
	f8 = f6+0.166666667f*(u-v);
//	f10=(0.0833333333f*( u+w))+
//		(f17-(0.0833333333f*(-u-w)));
	f10= f17+0.166666667f*(u+w);
//	f15=(0.0833333333f*( u-w))+
//		(f12-(0.0833333333f*(-u+w)));
	f15= f12+0.166666667f*(u-w);
		
//	f1 =(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2)+
//		(f3-(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2));
//	f5 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f7-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v));
//	f8 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f6-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v));
//	f10=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f17-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w));
//	f15=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f12-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w));

//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}


void __device__ DirichletWest_Regularized(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
    }

	float PI11 = 0;
	float PI12 = 0;
	float PI22 = 0;
	float PI33 = 0;
	float PI13 = 0;
	float PI23 = 0;
	float u;//,v;//,w;//,rho;
    u = UMAX;//*PoisProf(z)*1.5;
    //v = 0.0f;
	//w = 0.0f;
    float usqr = u*u;//+v*v+w*w;
	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
    
    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq9  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq14 = 0.0555555556f*(rho                -1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho                         -1.5f*usqr);

//    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
//    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq5  = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
//    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//    float feq8  = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
//    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq10 = 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
//    float feq11 = 0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
//    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
//    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq15 = 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
//    float feq16 = 0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
//    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

	f1 = feq1 +f3 -feq3 ;
	f5 = feq5 +f7 -feq7 ;
	f8 = feq8 +f6 -feq6 ;
	f10= feq10+f17-feq17;
	f15= feq15+f12-feq12;

    PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
    PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
    PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
    PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

    f0  = feq0 +1.5f  *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f1  = feq1 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f2  = feq2 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f3  = feq3 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f4  = feq4 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f5  = feq5 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f6  = feq6 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f7  = feq7 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f8  = feq8 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f9  = feq9 +0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f10 = feq10+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f11 = feq11+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f12 = feq12+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f13 = feq13+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f14 = feq14+0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f15 = feq15+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f16 = feq16+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f17 = feq17+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f18 = feq18+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
		
}



void __device__ NeumannEast_Regularized(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
		f11 = f13;
		f16 = f18;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f13 = f11;
		f18 = f16;
		f8 = f5;
	}
	else if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
	}

	float PI11 = 0;
	float PI12 = 0;
	float PI22 = 0;
	float PI33 = 0;
	float PI13 = 0;
	float PI23 = 0;

	float u;//,v;//,w;//,rho;
	float rho = 1.0f;
    //v = 0.0f;
	//w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
    float usqr = u*u;//+v*v+w*w;
    
    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq9  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq14 = 0.0555555556f*(rho                -1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho                         -1.5f*usqr);
    
//    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
//    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq5  = 0.0277777778f*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
//    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//    float feq8  = 0.0277777778f*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
//    float feq10 = 0.0277777778f*(rho+3.0f*( u+w)+4.5f*( u+w)*( u+w)-1.5f*usqr);
//    float feq11 = 0.0277777778f*(rho+3.0f*( v+w)+4.5f*( v+w)*( v+w)-1.5f*usqr);
//    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
//    float feq15 = 0.0277777778f*(rho+3.0f*( u-w)+4.5f*( u-w)*( u-w)-1.5f*usqr);
//    float feq16 = 0.0277777778f*(rho+3.0f*( v-w)+4.5f*( v-w)*( v-w)-1.5f*usqr);
//    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

	f3 = feq3 +f1 -feq1 ;
	f7 = feq7 +f5 -feq5 ;
	f6 = feq6 +f8 -feq8 ;
	f17= feq17+f10-feq10;
	f12= feq12+f15-feq15;

    PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
    PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
    PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
    PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

    f0  = feq0 +1.5f  *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f1  = feq1 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f2  = feq2 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f3  = feq3 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f4  = feq4 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f5  = feq5 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f6  = feq6 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f7  = feq7 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f8  = feq8 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f9  = feq9 +0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f10 = feq10+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f11 = feq11+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f12 = feq12+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f13 = feq13+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f14 = feq14+0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f15 = feq15+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f16 = feq16+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f17 = feq17+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f18 = feq18+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
			
}


__device__ void NeumannEast(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
//		f6 = f7;
		f11 = f13;
		f16 = f18;

		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
//		f7 = f6;
		f13 = f11;
		f18 = f16;

		f8 = f5;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
//		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
//		f17 = f12;
		f18 = f13;
	}
	}

	float u,v,w;//,rho;
	float rho = 1.0f;
    v = 0.0f;
	w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
	float u2 = u*u;
	float v2 = v*v;
	float w2 = w*w;
	float usqr = u2+v2+w2;

//	f3 = f1 -0.333333333f*u;
//	f7 = f5 -0.166666667f*(u+v);
//	f6 = f8 -0.166666667f*(u-v);
//	f17= f10-0.166666667f*(u+w);
//	f12= f15-0.166666667f*(u-w);
	f0 = 1.0f/3.0f*(rho-1.5f*usqr);
	f1 = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f2 = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f3 = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f4 = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f9 = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f10= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f11= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f12= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f13= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f14= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f15= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f16= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f17= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f18= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);


}


__device__ void NeumannEast_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
//		f6 = f7;
		f11 = f13;
		f16 = f18;

		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
//		f7 = f6;
		f13 = f11;
		f18 = f16;

		f8 = f5;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
//		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
//		f17 = f12;
		f18 = f13;
	}
	}

	float u,v,w;//,rho;
	float rho = 1.0f;
    v = 0.0f;
	w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

	f3 = f1 -0.333333333f*u;
	f7 = f5 -0.166666667f*(u+v);
	f6 = f8 -0.166666667f*(u-v);
	f17= f10-0.166666667f*(u+w);
	f12= f15-0.166666667f*(u-w);


//	f3 =(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2)+
//		(f1-(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2));
//	f7 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f5-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v));
//	f6 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f8-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v));
//	f17=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f10-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w));
//	f12=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f15-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w));


//	f1 =(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2)+
//		(f3-(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2));
//	f5 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f7-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v));
//	f8 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f6-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v));
//	f10=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f17-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w));
//	f15=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f12-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w));

//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}

__device__ void DirichletNorth_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(x == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(x == XDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

//	f1 =(0.166666667f*u)+
//		(f3-(-(0.166666667f*u)));
	f4 = f2-0.33333333f*v;
//	f5 =(0.0833333333f*( u+v))+
//		(f7-(0.0833333333f*(-u-v)));
	f7 = f5-0.166666667f*(u+v);
//	f8 =(0.0833333333f*( u-v  ))+
//		(f6-(0.0833333333f*(-u+v  )));
	f8 = f6+0.166666667f*(u-v);
//	f10=(0.0833333333f*( u+w))+
//		(f17-(0.0833333333f*(-u-w)));
	f13= f16-0.166666667f*(v-w);
//	f15=(0.0833333333f*( u-w))+
//		(f12-(0.0833333333f*(-u+w)));
	f18= f11-0.166666667f*(v+w);
	
//
//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                     
//float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;   +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
//float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
//float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w    +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
//float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;
//



//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}
__device__ void DirichletSouth_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(x == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(x == XDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

	f2 = f4 +0.33333333f*v;
	f5 = f7 +0.166666667f*(u+v);
	f6 = f8 -0.166666667f*(u-v);
	f16= f13+0.166666667f*(v-w);
	f11= f18+0.166666667f*(v+w);
	
//
//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                     
//float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;   +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
//float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
//float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w    +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
//float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;
//



//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}

__device__ void xsymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
//	if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
    }
	f1 = f3 ;
	f5 = f6 ;
	f8 = f7 ;
	f10= f12;
	f15= f17;
}
__device__ void xsymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
		f11 = f13;
		f16 = f18;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f13 = f11;
		f18 = f16;
		f8 = f5;
	}
//	else if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
	}
	f3 = f1 ;
	f6 = f5 ;
	f7 = f8 ;
	f12= f10;
	f17= f15;
}

__device__ void ysymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int z)
{
	if(z == 0){
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
	}
	if(z == ZDIM-1){
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
	}
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
}

__device__ void ysymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int z)
{
	if(z == 0){
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
	}
	if(z == ZDIM-1){
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
	}
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
}

__device__ void zsymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y)
{
	if(y == 0){
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
	}
	if(y == YDIM-1){
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
	}
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
}

__device__ void zsymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y)
{
	if(y == 0){
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
	}
	if(y == YDIM-1){
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
	}
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
}

inline __device__ void boundaries(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z, int im)
{
//	if(im == 3)//DirichletWest
//	{
//		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 53)//DirichletWest
	{
		//DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		DirichletWest_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 54)//DirichletWest
	{
		//NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		NeumannEast_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
//	if(im == 4)//DirichletWest
//	{
//		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	if(im == 13)//DirichletWest
//	{
//		DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 14)//DirichletWest
//	{
//		NeumannEast_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 15)//DirichletNorth
//	{
//		DirichletNorth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	if(im == 16)//DirichletSouth
//	{
//		DirichletSouth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 21)//ysymm top
	{
		ysymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 22)//ysymm bot
	{
		ysymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 23)//zsymm top
	{
		zsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 24)//zsymm bot
	{
		zsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
}
inline __device__ void boundaries_force(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z, int im)
{
//	if(im == 3)//DirichletWest
//	{
//		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 53)//DirichletWest
	{
		DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		//DirichletWest_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 54)//DirichletWest
	{
		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		//NeumannEast_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
//	else if(im == 15)//DirichletNorth
//	{
//		DirichletNorth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 16)//DirichletSouth
//	{
//		DirichletSouth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	else if(im == 21)//ysymm top
	{
		ysymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 22)//ysymm bot
	{
		ysymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 23)//zsymm top
	{
		zsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 24)//zsymm bot
	{
		zsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 25)//zsymm top
	{
		xsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 26)//zsymm bot
	{
		xsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
}




int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

//	f0 =(1.f-omega)*f0 +omega*(0.3333333333f*(rho-1.5f*usqr));
//	f1 =(1.f-omega)*f1 +omega*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f2 =(1.f-omega)*f2 +omega*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =(1.f-omega)*f3 +omega*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =(1.f-omega)*f4 +omega*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =(1.f-omega)*f5 +omega*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
//	f6 =(1.f-omega)*f6 +omega*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =(1.f-omega)*f7 +omega*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =(1.f-omega)*f8 +omega*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
//	f9 =(1.f-omega)*f9 +omega*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
//	f10=(1.f-omega)*f10+omega*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
//	f11=(1.f-omega)*f11+omega*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
//	f12=(1.f-omega)*f12+omega*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
//	f13=(1.f-omega)*f13+omega*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
//	f14=(1.f-omega)*f14+omega*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
//	f15=(1.f-omega)*f15+omega*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
//	f16=(1.f-omega)*f16+omega*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
//	f17=(1.f-omega)*f17+omega*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
//	f18=(1.f-omega)*f18+omega*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));

	f0 -=omega*(f0 -0.3333333333f*(rho-1.5f*usqr));
	f1 -=omega*(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 -=omega*(f2 -0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 -=omega*(f3 -0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 -=omega*(f4 -0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 -=omega*(f5 -0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 -=omega*(f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 -=omega*(f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 -=omega*(f8 -0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 -=omega*(f9 -0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10-=omega*(f10-0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11-=omega*(f11-0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr));
	f12-=omega*(f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13-=omega*(f13-0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr));
	f14-=omega*(f14-0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15-=omega*(f15-0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16-=omega*(f16-0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17-=omega*(f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18-=omega*(f18-0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));

}

inline __device__ void mrt_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float u,v,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
    float usqr = u*u+v*v+w*w;
//	u = rho*u;
//	v = rho*v;
//	w = rho*w;


	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	//COMPUTE M-MEQ
	//m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	//m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	//m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
//	m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//	m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//	m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
	m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
	m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
	m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
	m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
	m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
	m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
	m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
	m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
	m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;

	if(SmagLES == "YES"){
////		float PI11 = -1.0f/38.0f*(     (m1)+19.0f*omega* (m9));
////		float PI22 = -1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)-3.0f*omega*(m11)));
////		float PI33 = -1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)+3.0f*omega*(m11)));
//		float PI11 = LRLEVEL*-0.026315789f*m1-0.5f *omega*m9;
//		float PI22 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//		float PI33 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//		float PI12 = LRLEVEL*-1.5f*omega*m13;
//		float PI23 = LRLEVEL*-1.5f*omega*m14;
//		float PI13 = LRLEVEL*-1.5f*omega*m15;
//		float nu0 = ((1.0f/omega)-0.5f)*LRFACTOR/3.0f;
//		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//		//float Cs = 0.01f;
//		omega = 1.0f/(3.0f*(nu0+CS*Smag*LRFACTOR*LRFACTOR)*LRLEVEL+0.5f);
//		//omega = 1.0f/(1.0f/omega+3.f*CS*Smag*LRFACTOR*LRFACTOR);
//        //omega = 1.0f/(1.0f*LRLEVEL/1.99983f-1.f+0.5f+3.f*CS*Smag*LRFACTOR);

//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)*LRFACTOR/3.0f;
float tau0 = 1.f/omega;

//float Smag = (sqrt(nu0*nu0+18.f*CS*LRFACTOR*LRFACTOR*Q)-nu0)/(6.f*CS*LRFACTOR*LRFACTOR);
//float Smag = LRFACTOR*(sqrt(4.f/9.f*tau0*tau0+8.f*CS*LRFACTOR*Q)-2.f/3.f*tau0)/(4.f*CS*LRFACTOR*LRFACTOR);

//omega = 1.0f/(3.0f*(nu0+CS*Smag*LRFACTOR*LRFACTOR)*LRLEVEL+0.5f);

//float tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*LRFACTOR*Q));
float tau = tau0+0.5f*(-tau0+sqrt(tau0*tau0+18.f*CS*sqrt(2.f)*Q));
omega = 1.f/tau;

//float tau = 3.f*nu0*LRFACTOR+0.5f+(sqrt(tau0*tau0+18.f*CS*CS*LRFACTOR*LRFACTOR*Q)-tau0)*0.5f;
//omega = 1.f/tau;



	}


f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));



}

inline __device__ void mrt_collide_LES(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega, float Cs)
{
	float u,v,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
    float usqr = u*u+v*v+w*w;
//	u = rho*u;
//	v = rho*v;
//	w = rho*w;


	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	//COMPUTE M-MEQ
	//m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	//m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	//m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
//	m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//	m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//	m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
	m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
	m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
	m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
	m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
	m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
	m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
	m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
	m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
	m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;

	if(SmagLES == "YES"){
//		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
//		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//
//		float PI12 = -1.5f*omega*m13;
//		float PI23 = -1.5f*omega*m14;
//		float PI13 = -1.5f*omega*m15;
//		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//		omega = 1.0f/(1.0f/omega+3.f*CS*Smag);

//		float PI11 = LRLEVEL*-1.0f/38.0f*(     (m1)+19.0f*omega* (m9));
//		float PI22 = LRLEVEL*-1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)-3.0f*omega*(m11)));
//		float PI33 = LRLEVEL*-1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)+3.0f*omega*(m11)));
//		float PI12 = LRLEVEL*-1.5f*omega*m13;
//		float PI23 = LRLEVEL*-1.5f*omega*m14;
//		float PI13 = LRLEVEL*-1.5f*omega*m15;
//		float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//		float Smag = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+PI12*PI12+PI23*PI23+PI13*PI13);
//		omega = 1.0f/(3.0f*(nu0+Cs*Smag*LRLEVEL*LRLEVEL)+0.5f);


//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//
//float Smag = (sqrt(nu0*nu0+18.f*CS*Q)-nu0)/(6.f*CS);
//
////omega = 1.0f/(1.0f/omega+3.f*CS*Smag);
//
//float tau0 = 1.f/omega;
//float tau = 3.f*nu0+0.5f+(sqrt(tau0*tau0+18.f*CS*CS*Q)-tau0)*0.5f;
//omega = 1.f/tau;

float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
float tau0 = 1.f/omega;

//float Smag = (sqrt(nu0*nu0+18.f*CS*LRFACTOR*LRFACTOR*Q)-nu0)/(6.f*CS*LRFACTOR*LRFACTOR);
//float Smag = (sqrt(4.f/9.f*tau0*tau0+8.f*CS*Q)-2.f/3.f*tau0)/(4.f*CS);

//omega = 1.0f/(3.0f*(nu0+CS*Smag)+0.5f);

float tau = tau0+0.5f*(-tau0+sqrt(tau0*tau0+18.f*sqrt(2.f)*CS*Q));
omega = 1.f/tau;




	}


f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));



}

inline __device__ void bgk_scale_cf(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float SF)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

    f0 =SF*f0 +(1.0f-SF)*feq0 ;
    f1 =SF*f1 +(1.0f-SF)*feq1 ;
    f2 =SF*f2 +(1.0f-SF)*feq2 ;
    f3 =SF*f3 +(1.0f-SF)*feq3 ;
    f4 =SF*f4 +(1.0f-SF)*feq4 ;
    f5 =SF*f5 +(1.0f-SF)*feq5 ;
    f6 =SF*f6 +(1.0f-SF)*feq6 ;
    f7 =SF*f7 +(1.0f-SF)*feq7 ;
    f8 =SF*f8 +(1.0f-SF)*feq8 ;
    f9 =SF*f9 +(1.0f-SF)*feq9 ;
    f10=SF*f10+(1.0f-SF)*feq10;
    f11=SF*f11+(1.0f-SF)*feq11;
    f12=SF*f12+(1.0f-SF)*feq12;
    f13=SF*f13+(1.0f-SF)*feq13;
    f14=SF*f14+(1.0f-SF)*feq14;
    f15=SF*f15+(1.0f-SF)*feq15;
    f16=SF*f16+(1.0f-SF)*feq16;
    f17=SF*f17+(1.0f-SF)*feq17;
    f18=SF*f18+(1.0f-SF)*feq18;

}

inline __device__ void mrt_scale_cf(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float SF)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;
                                                                                                                
float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

//float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18)   -19.f*(u*u+v*v+w*w);
//float m2  = 12.f*f0+-4.f*f1+-4.f*f2+-4.f*f3+-4.f*f4+f5+f6+f7+f8+-4.f*f9+f10+f11+f12+f13+-4.f*f14+f15+f16+f17+f18 +7.53968254f*(u*u+v*v+w*w);
//float m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//float m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//float m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
//float m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
//float m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
//float m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
//float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
//float m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
//float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
//float m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
//float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
//float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
//float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
//float m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
//float m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
//float m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;





float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;

float omega = 1.0f/(3.0f*(UMAX*OBSTR1*2.f/RE)+0.5f);
float omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));

float PI11 = -0.026315789f*m1-0.5f *omega*m9;
float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
float PI12 = -1.5f*omega*m13;
float PI23 = -1.5f*omega*m14;
float PI13 = -1.5f*omega*m15;
//we know Smag on coarse mesh
float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//omega = 1.0f/(3.0f*(nu0+Cs*Smag*sqrt(2.f))+0.5f);
//omega  = 1.0f/(1.0f/omega+3.f*CS*Smag);
//omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*sqrt(2.f)*LRFACTOR*LRFACTOR);
//omega  = 1.0f/(1.0f/omega +3.f*CS*Smag);
//omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*sqrt(2.f)*LRFACTOR*LRFACTOR);
//omega  = 1.0f/(1.0f/omega +3.f*CS*Smag);
//omega2 = 1.0f/(1.0f*LRLEVEL/omega2-1.f+0.5f+3.f*CS*Smag*sqrt(2.f)*LRFACTOR);

//float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
//float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
//float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
//float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);
//
//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//float tau0c = 1.f/omega;
//float tau = tau0c+0.5*(-tau0c+sqrt(tau0c*tau0c+18.f*CS*Q));//tau_total of coarse mesh
//omega = 1.f/tau;//total omega on coarse mesh
//tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*LRFACTOR*Q));
//omega2= 1.f/tau;

SF = (omega*(1.0f-omega2))/((1.0f-omega)*omega2);//for post-collision 
//SF = omega*0.5f/omega2;//for post-streaming, pre-collision?





f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;

}
inline __device__ void mrt_scale_fc_LES(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega, float omega2)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;
                                                                                                                
float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

//float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
//float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
//float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
//float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
//float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
//float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;

//float PI11 = -0.026315789f*m1-0.5f *omega*m9;
//float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//float PI12 = -1.5f*omega*m13;
//float PI23 = -1.5f*omega*m14;
//float PI13 = -1.5f*omega*m15;
////we know Smag on fine mesh. Smag_c=Smag_f*sqrt(2)
//float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
////omega = 1.0f/(3.0f*(nu0+CS*Smag*sqrt(2.f))+0.5f);
////omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*LRFACTOR);
////omega  = 1.0f/(1.0f/omega+3.f*CS*Smag/sqrt(2.f));
////omega2 = 1.0f/(1.0f*LRLEVEL/omega2-1.f+0.5f+3.f*CS*Smag*LRFACTOR);
////omega  = 1.0f/(1.0f/omega+3.f*CS*Smag/sqrt(2.f));

//float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
//float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
//float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
//float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);
//
//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//float tau0f = 1.f/omega2;
//float tau0c = 1.f/omega;
//float tau = tau0f+0.5*(-tau0f+sqrt(tau0f*tau0f+18.f*CS*sqrt(2.f)*Q));//tau_total of fine
//omega2 = 1.f/tau;//total omega on fine mesh
//tau = LRLEVEL*(tau-tau0f)+tau0c;
//omega= 1.f/tau;

//tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*Q));

float SF = (omega*(1.0f-omega2))/((1.0f-omega)*omega2);
//float SF = omega2*2.f/omega;



//float SF = ((1.0f-omega)*omega2/LRFACTOR)/(omega*(1.0f-omega2));
//SF = omega*2.f/omega2;

f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;

}

inline __device__ int f_mem(int f_num, int x, int y, int z, size_t pitch)
{

	return (x+y*pitch+z*YDIM*pitch)+f_num*pitch*YDIM*(ZDIM/2-2);
}

inline __device__ int buff_mem(int f_num, int x, int y, size_t pitch)
{

	return (x+y*pitch)+f_num*pitch*YDIM;
}


__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}
__device__ int dmin_p(int a, int b)
{
	if (a<b) return a;
	else return 0;
}
__device__ int dmax_p(int a, int b)
{
	if (a>-1) return a;
	else return b-1;
}

//__global__ void mrt_d_single_force(float* fin, float* fout,
//							float omega, size_t pitch, float *FX, float *FY, float *FZ, 
//                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc, int GPU)
//{
//	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
//	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
//	int z = threadIdx.z+blockIdx.z*blockDim.z+ZDIM/2*GPU;
//	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
//	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
//	__shared__ int check[1];
//	check[0] = 0;
//	syncthreads();
//    float u_Av, v_Av, u_fluc, v_fluc;
//
//
////	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && 
////						y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1 &&
////						z > LRZ0+1 && z < LRZ0+(ZLRDIM-1)*LRFACTOR-1 ||
////						(x>XDIM-1)))
////	{
////	}
////	else{
//	int im = ImageFcn(x,y,z);
//
//	f0 = fin[j];
//	f1 = fin[f_mem(1 ,x-1,y  ,z  ,pitch)];
//	f3 = fin[f_mem(3 ,x+1,y  ,z  ,pitch)];
//	f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch)];
//	f5 = fin[f_mem(5 ,x-1,y-1,z  ,pitch)];
//	f6 = fin[f_mem(6 ,x+1,y-1,z  ,pitch)];
//	f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch)];
//	f7 = fin[f_mem(7 ,x+1,y+1,z  ,pitch)];
//	f8 = fin[f_mem(8 ,x-1,y+1,z  ,pitch)];
//    if(z != 0){
//	f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch)];
//	f10= fin[f_mem(10,x-1,y  ,z-1,pitch)];
//	f11= fin[f_mem(11,x  ,y-1,z-1,pitch)];
//	f12= fin[f_mem(12,x+1,y  ,z-1,pitch)];
//	f13= fin[f_mem(13,x  ,y+1,z-1,pitch)];
//    }
//    else{
//	f9 = fin[f_mem(9 ,x  ,y  ,ZDIM-1,pitch)];
//	f10= fin[f_mem(10,dmax_p(x-1,XDIM),y  ,ZDIM-1,pitch)];
//	f11= fin[f_mem(11,x  ,dmax_p(y-1,YDIM),ZDIM-1,pitch)];
//	f12= fin[f_mem(12,dmin_p(x+1,XDIM),y  ,ZDIM-1,pitch)];
//	f13= fin[f_mem(13,x  ,dmin_p(y+1,YDIM),ZDIM-1,pitch)];
//    }
//    if(z != ZDIM-1){
//	f14= fin[f_mem(14,x  ,y  ,z+1,pitch)];
//	f15= fin[f_mem(15,x-1,y  ,z+1,pitch)];
//	f16= fin[f_mem(16,x  ,y-1,z+1,pitch)];
//	f17= fin[f_mem(17,x+1,y  ,z+1,pitch)];
//	f18= fin[f_mem(18,x  ,y+1,z+1,pitch)];
//    }
//    else{
//	f14= fin[f_mem(14,x  ,y  ,0  ,pitch)];
//	f15= fin[f_mem(15,dmax_p(x-1,XDIM),y  ,0  ,pitch)];
//	f16= fin[f_mem(16,x  ,dmax_p(y-1,YDIM),0  ,pitch)];
//	f17= fin[f_mem(17,dmin_p(x+1,XDIM),y  ,0  ,pitch)];
//	f18= fin[f_mem(18,x  ,dmin_p(y+1,YDIM),0  ,pitch)];    
//    }
//
//	if(im == 1 || im == 10){//BB
//		if(im == 10){
//		check[0] = 1;
//		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
//		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
//		sumZ[threadIdx.x]=2.f*f9+2.f*f10+2.f*f11+2.f*f12+2.f*f13-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
//		}
//		else{
//		sumX[threadIdx.x]=0.f;
//		sumY[threadIdx.x]=0.f;
//		sumZ[threadIdx.x]=0.f;
//		}
//		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
//		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
//		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
//		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
//		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
//		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
//		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
//		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
//		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
//		fout[j+pitch*YDIM*ZDIM*10] = f17;
//		fout[j+pitch*YDIM*ZDIM*11] = f18;
//		fout[j+pitch*YDIM*ZDIM*12] = f15;
//		fout[j+pitch*YDIM*ZDIM*13] = f16;
//		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
//		fout[j+pitch*YDIM*ZDIM*15] = f12;
//		fout[j+pitch*YDIM*ZDIM*16] = f13;
//		fout[j+pitch*YDIM*ZDIM*17] = f10;
//		fout[j+pitch*YDIM*ZDIM*18] = f11;
//	}
//	else{
//		sumX[threadIdx.x]=0.f;
//		sumY[threadIdx.x]=0.f;
//		sumZ[threadIdx.x]=0.f;
//		boundaries_force(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);
//
//        if(im == 100)//north outlet
//        {
//           	f0 = fin[f_mem(0 ,x,y-1,z,pitch)];
//        	f1 = fin[f_mem(1 ,x,y-1,z,pitch)];
//        	f3 = fin[f_mem(3 ,x,y-1,z,pitch)];
//        	f2 = fin[f_mem(2 ,x,y-1,z,pitch)];
//        	f5 = fin[f_mem(5 ,x,y-1,z,pitch)];
//        	f6 = fin[f_mem(6 ,x,y-1,z,pitch)];
//        	f4 = fin[f_mem(4 ,x,y-1,z,pitch)];
//        	f7 = fin[f_mem(7 ,x,y-1,z,pitch)];
//        	f8 = fin[f_mem(8 ,x,y-1,z,pitch)];
//        	f9 = fin[f_mem(9 ,x,y-1,z,pitch)];
//        	f10= fin[f_mem(10,x,y-1,z,pitch)];
//        	f11= fin[f_mem(11,x,y-1,z,pitch)];
//        	f12= fin[f_mem(12,x,y-1,z,pitch)];
//        	f13= fin[f_mem(13,x,y-1,z,pitch)];
//        	f14= fin[f_mem(14,x,y-1,z,pitch)];
//        	f15= fin[f_mem(15,x,y-1,z,pitch)];
//        	f16= fin[f_mem(16,x,y-1,z,pitch)];
//        	f17= fin[f_mem(17,x,y-1,z,pitch)];
//        	f18= fin[f_mem(18,x,y-1,z,pitch)];
//
//	float rho,u,v,w;	
//	rho = 1.0f;
//	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
//	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
//	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//
//	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//
//	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
//	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+    f8+ -4.f*f9+    f10+      f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
//	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
//	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
//	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
//	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
//	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
//	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
//	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
//	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
//	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
//	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
//	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
//	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
//	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;
//
//f0 =(0.052631579f*rho                           +- 0.012531328f*(m1)+ 0.047619048f*(m2));
//f1 =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)-m10));
//f2 =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
//f3 =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)-m10));                                                                                         
//f4 =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
//f5 =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
//f6 =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
//f7 =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
//f8 =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
//f9 =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                       
//f10=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
//f11=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
//f12=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
//f13=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
//f14=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                      
//f15=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
//f16=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
//f17=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
//f18=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
//
//        }
//        if(im == 200)//south inlet
//        {
//           	f0 = fin[f_mem(0 ,x,y+1,z,pitch)];
//        	f1 = fin[f_mem(1 ,x,y+1,z,pitch)];
//        	f3 = fin[f_mem(3 ,x,y+1,z,pitch)];
//        	f2 = fin[f_mem(2 ,x,y+1,z,pitch)];
//        	f5 = fin[f_mem(5 ,x,y+1,z,pitch)];
//        	f6 = fin[f_mem(6 ,x,y+1,z,pitch)];
//        	f4 = fin[f_mem(4 ,x,y+1,z,pitch)];
//        	f7 = fin[f_mem(7 ,x,y+1,z,pitch)];
//        	f8 = fin[f_mem(8 ,x,y+1,z,pitch)];
//        	f9 = fin[f_mem(9 ,x,y+1,z,pitch)];
//        	f10= fin[f_mem(10,x,y+1,z,pitch)];
//        	f11= fin[f_mem(11,x,y+1,z,pitch)];
//        	f12= fin[f_mem(12,x,y+1,z,pitch)];
//        	f13= fin[f_mem(13,x,y+1,z,pitch)];
//        	f14= fin[f_mem(14,x,y+1,z,pitch)];
//        	f15= fin[f_mem(15,x,y+1,z,pitch)];
//        	f16= fin[f_mem(16,x,y+1,z,pitch)];
//        	f17= fin[f_mem(17,x,y+1,z,pitch)];
//        	f18= fin[f_mem(18,x,y+1,z,pitch)];
//
//	float rho,u,v,w;	
//	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+f10+f11+f12+f13+f14+f15+f16+f17+f18;
//	u = 0.f;//f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
//	v = UMAX;//f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
//	w = 0.f;//f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//
//	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;
//
//	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
//	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+    f8+ -4.f*f9+    f10+      f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
//	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
//	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
//	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
//	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
//	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
//	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
//	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
//	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
//	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
//	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
//	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
//	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
//	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;
//
//f0 =(0.052631579f*rho                           +- 0.012531328f*(m1)+ 0.047619048f*(m2));
//f1 =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)-m10));
//f2 =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
//f3 =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)-m10));                                                                                         
//f4 =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
//f5 =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
//f6 =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
//f7 =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
//f8 =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
//f9 =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                       
//f10=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
//f11=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
//f12=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
//f13=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
//f14=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                      
//f15=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
//f16=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
//f17=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
//f18=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
//
//        }
//
//
//        //float Cs = 0.01f;
//        //if(XDIM-x < 64.f)
//        ////Cs = 0.01f+(x-64.f)/64.f*(x-64.f)/64.f*0.1f;
//        //Cs = 0.01f*pow(2.f,((x-448.f)/16.f));
//
//		if(MODEL == "MRT")
//		mrt_collide_LES(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega,CS);
//		else if(MODEL == "BGK")
//		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
//
//        if(VELAV == "YES"){
//            if(t>=START_VELAV && t<START_VELFLUC){
//                u_Av = uAv[j];
//                v_Av = vAv[j];
//        		vel_av(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t);
//                uAv[j] = u_Av;
//                vAv[j] = v_Av;
//            }
//            else if(t>=START_VELFLUC){
//                u_Av = uAv[j];
//                v_Av = vAv[j];
//                u_fluc = ufluc[j];
//                v_fluc = vfluc[j];
//        		vel_fluc(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t);
//                ufluc[j] = u_fluc;
//                vfluc[j] = v_fluc;
//                
//            }
//        }
//
//
//
//		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
//		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
//		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
//		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
//		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
//		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
//		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
//		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
//		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
//		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
//		fout[f_mem(10,x,y,z,pitch)] = f10;
//		fout[f_mem(11,x,y,z,pitch)] = f11;
//		fout[f_mem(12,x,y,z,pitch)] = f12;
//		fout[f_mem(13,x,y,z,pitch)] = f13;
//		fout[f_mem(14,x,y,z,pitch)] = f14;
//		fout[f_mem(15,x,y,z,pitch)] = f15;
//		fout[f_mem(16,x,y,z,pitch)] = f16;
//		fout[f_mem(17,x,y,z,pitch)] = f17;
//		fout[f_mem(18,x,y,z,pitch)] = f18;
//	}
//
//	syncthreads();
//	if(check[0] == 1 && t>=STARTF && REFINEMENT == "NO"){
//	//reduction for force
//	int nTotalThreads = blockDim.x;
//	while(nTotalThreads > 1){
//		int halfPoint = (nTotalThreads >> 1);
//		if(threadIdx.x < halfPoint){
//			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
//			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
//			sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
//		}
//		syncthreads();
//		nTotalThreads = halfPoint;
//	}
//	if(threadIdx.x == 0){
//		atomicAdd(&FX[t],sumX[0]);
//		atomicAdd(&FY[t],sumY[0]);
//		atomicAdd(&FZ[t],sumZ[0]);
//	}
//	}
//
//
////	}
//}

__global__ void mrt_d_single(float* fA, float* fB,
							float omega, size_t pitch, int GPU)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z+ZDIM/2*GPU;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,z);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

//	if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 
//		&& y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1 && z > LRZ0+1 && z < LRZ0+(ZLRDIM-1)*LRFACTOR-1 ||
//		(x>XDIM-1)){
//	}
//	else{

	f0 = fA[j];
	f1 = fA[f_mem(1 ,x-1,y       ,z             ,pitch)];
	f3 = fA[f_mem(3 ,x+1,y       ,z             ,pitch)];
	f2 = fA[f_mem(2 ,x  ,y-1,z             ,pitch)];
	f5 = fA[f_mem(5 ,x-1,y-1,z             ,pitch)];
	f6 = fA[f_mem(6 ,x+1,y-1,z             ,pitch)];
	f4 = fA[f_mem(4 ,x  ,y+1,z             ,pitch)];
	f7 = fA[f_mem(7 ,x+1,y+1,z             ,pitch)];
	f8 = fA[f_mem(8 ,x-1,y+1,z             ,pitch)];
	f9 = fA[f_mem(9 ,x  ,y       ,z-1,pitch)];
	f10= fA[f_mem(10,x-1,y       ,z-1,pitch)];
	f11= fA[f_mem(11,x  ,y-1,z-1,pitch)];
	f12= fA[f_mem(12,x+1,y       ,z-1,pitch)];
	f13= fA[f_mem(13,x  ,y+1,z-1,pitch)];
	f14= fA[f_mem(14,x  ,y       ,z+1,pitch)];
	f15= fA[f_mem(15,x-1,y       ,z+1,pitch)];
	f16= fA[f_mem(16,x  ,y-1,z+1,pitch)];
	f17= fA[f_mem(17,x+1,y       ,z+1,pitch)];
	//f18= fA[f_mem(18,x  ,y+1,dmin(z+1,ZDIM),pitch)];
    if(z != ZDIM-1)
	f18= fA[f_mem(18,x  ,y+1,z+1,pitch)];

	if(im == 1 || im ==10){//BB
		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f14;
		fB[f_mem(10,x,y,z,pitch)] = f17;
		fB[f_mem(11,x,y,z,pitch)] = f18;
		fB[f_mem(12,x,y,z,pitch)] = f15;
		fB[f_mem(13,x,y,z,pitch)] = f16;
		fB[f_mem(14,x,y,z,pitch)] = f9 ;
		fB[f_mem(15,x,y,z,pitch)] = f12;
		fB[f_mem(16,x,y,z,pitch)] = f13;
		fB[f_mem(17,x,y,z,pitch)] = f10;
		fB[f_mem(18,x,y,z,pitch)] = f11;
	}
	else{

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fB[f_mem(10,x,y,z,pitch)] = f10;
		fB[f_mem(11,x,y,z,pitch)] = f11;
		fB[f_mem(12,x,y,z,pitch)] = f12;
		fB[f_mem(13,x,y,z,pitch)] = f13;
		fB[f_mem(14,x,y,z,pitch)] = f14;
		fB[f_mem(15,x,y,z,pitch)] = f15;
		fB[f_mem(16,x,y,z,pitch)] = f16;
		fB[f_mem(17,x,y,z,pitch)] = f17;
		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
//	}
}

__global__ void update_inner(float* fA, float* fB, float* g, float* h,
							float omega, size_t pitch, int GPU)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,GPU*ZDIM/2+1+z);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

//	if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 
//		&& y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1 && z > LRZ0+1 && z < LRZ0+(ZLRDIM-1)*LRFACTOR-1 ||
//		(x>XDIM-1)){
//	}
//	else{
    if(z==ZDIM/2-1-1){//top nodes need info from h
    f0 = fA[j];
	f1 = fA[f_mem   (1 ,dmax(x-1)     ,y             ,z  ,pitch)];
	f3 = fA[f_mem   (3 ,dmin(x+1,XDIM),y             ,z  ,pitch)];
	f2 = fA[f_mem   (2 ,x             ,dmax(y-1)     ,z  ,pitch)];
	f5 = fA[f_mem   (5 ,dmax(x-1)     ,dmax(y-1)     ,z  ,pitch)];
	f6 = fA[f_mem   (6 ,dmin(x+1,XDIM),dmax(y-1)     ,z  ,pitch)];
	f4 = fA[f_mem   (4 ,x             ,dmin(y+1,YDIM),z  ,pitch)];
	f7 = fA[f_mem   (7 ,dmin(x+1,XDIM),dmin(y+1,YDIM),z  ,pitch)];
	f8 = fA[f_mem   (8 ,dmax(x-1)     ,dmin(y+1,YDIM),z  ,pitch)];
	f9 = fA[f_mem   (9 ,x             ,y             ,z-1,pitch)];
	f10= fA[f_mem   (10,dmax(x-1)     ,y             ,z-1,pitch)];
	f11= fA[f_mem   (11,x             ,dmax(y-1)     ,z-1,pitch)];
	f12= fA[f_mem   (12,dmin(x+1,XDIM),y             ,z-1,pitch)];
	f13= fA[f_mem   (13,x             ,dmin(y+1,YDIM),z-1,pitch)];
	f14= h [buff_mem(14,x             ,y             ,pitch)];
	f15= h [buff_mem(15,dmax(x-1)     ,y             ,pitch)];
	f16= h [buff_mem(16,x             ,dmax(y-1)     ,pitch)];
	f17= h [buff_mem(17,dmin(x+1,XDIM),y             ,pitch)];
	f18= h [buff_mem(18,x             ,dmin(y+1,YDIM),pitch)];
    }
    else if(z==0){//bottom nodes need info from g
	f0 = fA[j];
	f1 = fA[f_mem   (1 ,dmax(x-1)     ,y             ,z  ,pitch)];
	f3 = fA[f_mem   (3 ,dmin(x+1,XDIM),y             ,z  ,pitch)];
	f2 = fA[f_mem   (2 ,x             ,dmax(y-1)     ,z  ,pitch)];
	f5 = fA[f_mem   (5 ,dmax(x-1)     ,dmax(y-1)     ,z  ,pitch)];
	f6 = fA[f_mem   (6 ,dmin(x+1,XDIM),dmax(y-1)     ,z  ,pitch)];
	f4 = fA[f_mem   (4 ,x             ,dmin(y+1,YDIM),z  ,pitch)];
	f7 = fA[f_mem   (7 ,dmin(x+1,XDIM),dmin(y+1,YDIM),z  ,pitch)];
	f8 = fA[f_mem   (8 ,dmax(x-1)     ,dmin(y+1,YDIM),z  ,pitch)];
	f9 = h [buff_mem(9 ,x             ,y             ,pitch)];
	f10= h [buff_mem(10,dmax(x-1)     ,y             ,pitch)];
	f11= h [buff_mem(11,x             ,dmax(y-1)     ,pitch)];
	f12= h [buff_mem(12,dmin(x+1,XDIM),y             ,pitch)];
	f13= h [buff_mem(13,x             ,dmin(y+1,YDIM),pitch)];
	f14= fA[f_mem   (14,x             ,y             ,z+1,pitch)];
	f15= fA[f_mem   (15,dmax(x-1)     ,y             ,z+1,pitch)];
	f16= fA[f_mem   (16,x             ,dmax(y-1)     ,z+1,pitch)];
	f17= fA[f_mem   (17,dmin(x+1,XDIM),y             ,z+1,pitch)];
	f18= fA[f_mem   (18,x             ,dmin(y+1,YDIM),z+1,pitch)];
    }
    else{//normal nodes
	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,z,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,z,pitch)];
	f2 = fA[f_mem(2 ,x             ,dmax(y-1)     ,z,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,dmax(y-1)     ,z,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),dmax(y-1)     ,z,pitch)];
	f4 = fA[f_mem(4 ,x             ,dmin(y+1,YDIM),z,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),dmin(y+1,YDIM),z,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),z,pitch)];
	f9 = fA[f_mem(9 ,x             ,y             ,z,pitch)];
	f10= fA[f_mem(10,dmax(x-1)     ,y             ,z,pitch)];
	f11= fA[f_mem(11,x             ,dmax(y-1)     ,z,pitch)];
	f12= fA[f_mem(12,dmin(x+1,XDIM),y             ,z,pitch)];
	f13= fA[f_mem(13,x             ,dmin(y+1,YDIM),z,pitch)];
	f14= fA[f_mem(14,x             ,y             ,z,pitch)];
	f15= fA[f_mem(15,dmax(x-1)     ,y             ,z,pitch)];
	f16= fA[f_mem(16,x             ,dmax(y-1)     ,z,pitch)];
	f17= fA[f_mem(17,dmin(x+1,XDIM),y             ,z,pitch)];
	f18= fA[f_mem(18,x             ,dmin(y+1,YDIM),z,pitch)];

    }//end normal nodes

	if(im == 1 || im ==10){//BB
		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f14;
		fB[f_mem(10,x,y,z,pitch)] = f17;
		fB[f_mem(11,x,y,z,pitch)] = f18;
		fB[f_mem(12,x,y,z,pitch)] = f15;
		fB[f_mem(13,x,y,z,pitch)] = f16;
		fB[f_mem(14,x,y,z,pitch)] = f9 ;
		fB[f_mem(15,x,y,z,pitch)] = f12;
		fB[f_mem(16,x,y,z,pitch)] = f13;
		fB[f_mem(17,x,y,z,pitch)] = f10;
		fB[f_mem(18,x,y,z,pitch)] = f11;
	}
	else{

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fB[f_mem(10,x,y,z,pitch)] = f10;
		fB[f_mem(11,x,y,z,pitch)] = f11;
		fB[f_mem(12,x,y,z,pitch)] = f12;
		fB[f_mem(13,x,y,z,pitch)] = f13;
		fB[f_mem(14,x,y,z,pitch)] = f14;
		fB[f_mem(15,x,y,z,pitch)] = f15;
		fB[f_mem(16,x,y,z,pitch)] = f16;
		fB[f_mem(17,x,y,z,pitch)] = f17;
		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
//	}
}

__global__ void update_bottom(float* gA, float* gB, float* f, float* temp,
							float omega, size_t pitch, int GPU)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,GPU*ZDIM/2);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	f0 = gA  [j];
	f1 = gA  [buff_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = gA  [buff_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = gA  [buff_mem(2 ,x             ,dmax(y-1)     ,pitch)];
	f5 = gA  [buff_mem(5 ,dmax(x-1)     ,dmax(y-1)     ,pitch)];
	f6 = gA  [buff_mem(6 ,dmin(x+1,XDIM),dmax(y-1)     ,pitch)];
	f4 = gA  [buff_mem(4 ,x             ,dmin(y+1,YDIM),pitch)];
	f7 = gA  [buff_mem(7 ,dmin(x+1,XDIM),dmin(y+1,YDIM),pitch)];
	f8 = gA  [buff_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	f9 = temp[buff_mem(9 ,x             ,y             ,pitch)];
	f10= temp[buff_mem(10,dmax(x-1)     ,y             ,pitch)];
	f11= temp[buff_mem(11,x             ,dmax(y-1)     ,pitch)];
	f12= temp[buff_mem(12,dmin(x+1,XDIM),y             ,pitch)];
	f13= temp[buff_mem(13,x             ,dmin(y+1,YDIM),pitch)];
	f14= f   [f_mem   (14,x             ,y             ,0,pitch)];
	f15= f   [f_mem   (15,dmax(x-1)     ,y             ,0,pitch)];
	f16= f   [f_mem   (16,x             ,dmax(y-1)     ,0,pitch)];
	f17= f   [f_mem   (17,dmin(x+1,XDIM),y             ,0,pitch)];
	f18= f   [f_mem   (18,x             ,dmin(y+1,YDIM),0,pitch)];

	if(im == 1 || im ==10){//BB
		gB[buff_mem(0 ,x,y,pitch)] = f0 ;
		gB[buff_mem(1 ,x,y,pitch)] = f3 ;
		gB[buff_mem(2 ,x,y,pitch)] = f4 ;
		gB[buff_mem(3 ,x,y,pitch)] = f1 ;
		gB[buff_mem(4 ,x,y,pitch)] = f2 ;
		gB[buff_mem(5 ,x,y,pitch)] = f7 ;
		gB[buff_mem(6 ,x,y,pitch)] = f8 ;
		gB[buff_mem(7 ,x,y,pitch)] = f5 ;
		gB[buff_mem(8 ,x,y,pitch)] = f6 ;
		gB[buff_mem(9 ,x,y,pitch)] = f14;
		gB[buff_mem(10,x,y,pitch)] = f17;
		gB[buff_mem(11,x,y,pitch)] = f18;
		gB[buff_mem(12,x,y,pitch)] = f15;
		gB[buff_mem(13,x,y,pitch)] = f16;
		gB[buff_mem(14,x,y,pitch)] = f9 ;
		gB[buff_mem(15,x,y,pitch)] = f12;
		gB[buff_mem(16,x,y,pitch)] = f13;
		gB[buff_mem(17,x,y,pitch)] = f10;
		gB[buff_mem(18,x,y,pitch)] = f11;
	}
	else{

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,GPU*ZDIM/2,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		gB[buff_mem(0 ,x,y,pitch)] = f0 ;
		gB[buff_mem(1 ,x,y,pitch)] = f1 ;
		gB[buff_mem(2 ,x,y,pitch)] = f2 ;
		gB[buff_mem(3 ,x,y,pitch)] = f3 ;
		gB[buff_mem(4 ,x,y,pitch)] = f4 ;
		gB[buff_mem(5 ,x,y,pitch)] = f5 ;
		gB[buff_mem(6 ,x,y,pitch)] = f6 ;
		gB[buff_mem(7 ,x,y,pitch)] = f7 ;
		gB[buff_mem(8 ,x,y,pitch)] = f8 ;
		gB[buff_mem(9 ,x,y,pitch)] = f9 ;
		gB[buff_mem(10,x,y,pitch)] = f10;
		gB[buff_mem(11,x,y,pitch)] = f11;
		gB[buff_mem(12,x,y,pitch)] = f12;
		gB[buff_mem(13,x,y,pitch)] = f13;
		gB[buff_mem(14,x,y,pitch)] = f14;
		gB[buff_mem(15,x,y,pitch)] = f15;
		gB[buff_mem(16,x,y,pitch)] = f16;
		gB[buff_mem(17,x,y,pitch)] = f17;
		gB[buff_mem(18,x,y,pitch)] = f18;
	}
//	}
}

__global__ void update_top(float* hA, float* hB, float* f, float* temp,
							float omega, size_t pitch, int GPU)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,(GPU+1)*ZDIM/2-1);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	f0 = hA[j];
	f1 = hA  [buff_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = hA  [buff_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = hA  [buff_mem(2 ,x             ,dmax(y-1)     ,pitch)];
	f5 = hA  [buff_mem(5 ,dmax(x-1)     ,dmax(y-1)     ,pitch)];
	f6 = hA  [buff_mem(6 ,dmin(x+1,XDIM),dmax(y-1)     ,pitch)];
	f4 = hA  [buff_mem(4 ,x             ,dmin(y+1,YDIM),pitch)];
	f7 = hA  [buff_mem(7 ,dmin(x+1,XDIM),dmin(y+1,YDIM),pitch)];
	f8 = hA  [buff_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	f9 = f   [f_mem   (9 ,x             ,y             ,ZDIM/2-2-1,pitch)];
	f10= f   [f_mem   (10,dmax(x-1)     ,y             ,ZDIM/2-2-1,pitch)];
	f11= f   [f_mem   (11,x             ,dmax(y-1)     ,ZDIM/2-2-1,pitch)];
	f12= f   [f_mem   (12,dmin(x+1,XDIM),y             ,ZDIM/2-2-1,pitch)];
	f13= f   [f_mem   (13,x             ,dmin(y+1,YDIM),ZDIM/2-2-1,pitch)];
	f14= temp[buff_mem(14,x             ,y             ,pitch)];
	f15= temp[buff_mem(15,dmax(x-1)     ,y             ,pitch)];
	f16= temp[buff_mem(16,x             ,dmax(y-1)     ,pitch)];
	f17= temp[buff_mem(17,dmin(x+1,XDIM),y             ,pitch)];
	f18= temp[buff_mem(18,x             ,dmin(y+1,YDIM),pitch)];

	if(im == 1 || im ==10){//BB
		hB[buff_mem(0 ,x,y,pitch)] = f0 ;
		hB[buff_mem(1 ,x,y,pitch)] = f3 ;
		hB[buff_mem(2 ,x,y,pitch)] = f4 ;
		hB[buff_mem(3 ,x,y,pitch)] = f1 ;
		hB[buff_mem(4 ,x,y,pitch)] = f2 ;
		hB[buff_mem(5 ,x,y,pitch)] = f7 ;
		hB[buff_mem(6 ,x,y,pitch)] = f8 ;
		hB[buff_mem(7 ,x,y,pitch)] = f5 ;
		hB[buff_mem(8 ,x,y,pitch)] = f6 ;
		hB[buff_mem(9 ,x,y,pitch)] = f14;
		hB[buff_mem(10,x,y,pitch)] = f17;
		hB[buff_mem(11,x,y,pitch)] = f18;
		hB[buff_mem(12,x,y,pitch)] = f15;
		hB[buff_mem(13,x,y,pitch)] = f16;
		hB[buff_mem(14,x,y,pitch)] = f9 ;
		hB[buff_mem(15,x,y,pitch)] = f12;
		hB[buff_mem(16,x,y,pitch)] = f13;
		hB[buff_mem(17,x,y,pitch)] = f10;
		hB[buff_mem(18,x,y,pitch)] = f11;
	}
	else{

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,GPU*ZDIM/2,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		hB[buff_mem(0 ,x,y,pitch)] = f0 ;
		hB[buff_mem(1 ,x,y,pitch)] = f1 ;
		hB[buff_mem(2 ,x,y,pitch)] = f2 ;
		hB[buff_mem(3 ,x,y,pitch)] = f3 ;
		hB[buff_mem(4 ,x,y,pitch)] = f4 ;
		hB[buff_mem(5 ,x,y,pitch)] = f5 ;
		hB[buff_mem(6 ,x,y,pitch)] = f6 ;
		hB[buff_mem(7 ,x,y,pitch)] = f7 ;
		hB[buff_mem(8 ,x,y,pitch)] = f8 ;
		hB[buff_mem(9 ,x,y,pitch)] = f9 ;
		hB[buff_mem(10,x,y,pitch)] = f10;
		hB[buff_mem(11,x,y,pitch)] = f11;
		hB[buff_mem(12,x,y,pitch)] = f12;
		hB[buff_mem(13,x,y,pitch)] = f13;
		hB[buff_mem(14,x,y,pitch)] = f14;
		hB[buff_mem(15,x,y,pitch)] = f15;
		hB[buff_mem(16,x,y,pitch)] = f16;
		hB[buff_mem(17,x,y,pitch)] = f17;
		hB[buff_mem(18,x,y,pitch)] = f18;
	}
//	}
}




__device__ __inline__ float ld_gb1_cg(const float *addr)
{
    float return_value;
    asm("ld.global.cg.f32 %0, [%1];" : "=f"(return_value) : "l"(addr));
    return return_value;
}

__global__ void initialize_single(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	
	int im = ImageFcn(x,y,z);
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = UMAX;
	w = 0.0f;

    if(im == 10 || im == 1){
    u = 0.0f;
    v = 0.0f;
    w = 0.0f;
    }
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

    if(MODEL == "BGK"){ 
	f[j+0 *pitch*YDIM*ZDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
	f[j+12*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
	f[j+14*pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);   
    }
    else{
                                                                                                                
float f0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float f1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float f2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float f3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float f4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float f5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float f6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float f7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float f8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float f9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float f10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float f11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float f12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float f13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float f14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float f15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float f16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float f17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float f18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

f1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
f12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
f14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
f17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

f[j+0 *pitch*YDIM*(ZDIM/2-2)]=f0 ;
f[j+1 *pitch*YDIM*(ZDIM/2-2)]=f1 ;
f[j+2 *pitch*YDIM*(ZDIM/2-2)]=f2 ;
f[j+3 *pitch*YDIM*(ZDIM/2-2)]=f3 ;
f[j+4 *pitch*YDIM*(ZDIM/2-2)]=f4 ;
f[j+5 *pitch*YDIM*(ZDIM/2-2)]=f5 ;
f[j+6 *pitch*YDIM*(ZDIM/2-2)]=f6 ;
f[j+7 *pitch*YDIM*(ZDIM/2-2)]=f7 ;
f[j+8 *pitch*YDIM*(ZDIM/2-2)]=f8 ;
f[j+9 *pitch*YDIM*(ZDIM/2-2)]=f9 ;
f[j+10*pitch*YDIM*(ZDIM/2-2)]=f10;
f[j+11*pitch*YDIM*(ZDIM/2-2)]=f11;
f[j+12*pitch*YDIM*(ZDIM/2-2)]=f12;
f[j+13*pitch*YDIM*(ZDIM/2-2)]=f13;
f[j+14*pitch*YDIM*(ZDIM/2-2)]=f14;
f[j+15*pitch*YDIM*(ZDIM/2-2)]=f15;
f[j+16*pitch*YDIM*(ZDIM/2-2)]=f16;
f[j+17*pitch*YDIM*(ZDIM/2-2)]=f17;
f[j+18*pitch*YDIM*(ZDIM/2-2)]=f18;

    }


	if(x == XDIM-1){
	for(int i = XDIM; i<pitch; i++){
		j = i+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
		f[j+0 *pitch*YDIM*ZDIM]=0.f;
		f[j+1 *pitch*YDIM*ZDIM]=0.f;
		f[j+2 *pitch*YDIM*ZDIM]=0.f;
		f[j+3 *pitch*YDIM*ZDIM]=0.f;
		f[j+4 *pitch*YDIM*ZDIM]=0.f;
		f[j+5 *pitch*YDIM*ZDIM]=0.f;
		f[j+6 *pitch*YDIM*ZDIM]=0.f;
		f[j+7 *pitch*YDIM*ZDIM]=0.f;
		f[j+8 *pitch*YDIM*ZDIM]=0.f;
		f[j+9 *pitch*YDIM*ZDIM]=0.f;
		f[j+10*pitch*YDIM*ZDIM]=0.f;
		f[j+11*pitch*YDIM*ZDIM]=0.f;
		f[j+12*pitch*YDIM*ZDIM]=0.f;
		f[j+13*pitch*YDIM*ZDIM]=0.f;
		f[j+14*pitch*YDIM*ZDIM]=0.f;
		f[j+15*pitch*YDIM*ZDIM]=0.f;
		f[j+16*pitch*YDIM*ZDIM]=0.f;
		f[j+17*pitch*YDIM*ZDIM]=0.f;
		f[j+18*pitch*YDIM*ZDIM]=0.f;
	}
	}
}

__global__ void initialize_buffer(float *g, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = UMAX;
	w = 0.0f;

	usqr = u*u+v*v+w*w;

    if(MODEL == "BGK"){ 
	g[j+0 *pitch*YDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	g[j+1 *pitch*YDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	g[j+2 *pitch*YDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	g[j+3 *pitch*YDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	g[j+4 *pitch*YDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	g[j+5 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	g[j+6 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	g[j+7 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	g[j+8 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	g[j+9 *pitch*YDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	g[j+10*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	g[j+11*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
	g[j+12*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	g[j+13*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
	g[j+14*pitch*YDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	g[j+15*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	g[j+16*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	g[j+17*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	g[j+18*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);   
    }
    else{
                                                                                                                
float f0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float f1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float f2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float f3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float f4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float f5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float f6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float f7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float f8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float f9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float f10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float f11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float f12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float f13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float f14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float f15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float f16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float f17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float f18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

f1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
f12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
f14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
f17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

g[j+0 *pitch*YDIM]=f0 ;
g[j+1 *pitch*YDIM]=f1 ;
g[j+2 *pitch*YDIM]=f2 ;
g[j+3 *pitch*YDIM]=f3 ;
g[j+4 *pitch*YDIM]=f4 ;
g[j+5 *pitch*YDIM]=f5 ;
g[j+6 *pitch*YDIM]=f6 ;
g[j+7 *pitch*YDIM]=f7 ;
g[j+8 *pitch*YDIM]=f8 ;
g[j+9 *pitch*YDIM]=f9 ;
g[j+10*pitch*YDIM]=f10;
g[j+11*pitch*YDIM]=f11;
g[j+12*pitch*YDIM]=f12;
g[j+13*pitch*YDIM]=f13;
g[j+14*pitch*YDIM]=f14;
g[j+15*pitch*YDIM]=f15;
g[j+16*pitch*YDIM]=f16;
g[j+17*pitch*YDIM]=f17;
g[j+18*pitch*YDIM]=f18;

    }
}


int main(int argc, char *argv[])
{
	int GPU_N;
	hipGetDeviceCount(&GPU_N);
    GPU_N = 2;
	cout<<"number of GPUs: "<<GPU_N<<endl;


	//int *image_d, *image_h;

	ofstream output;
	ofstream output2;
	string FileName = CASENAME;
	//output.open ("LBM1_out.dat");
	output.open ((FileName+".dat").c_str());
	output2.open ((FileName+".force").c_str());

	//size_t memsize, memsize2;
	size_t pitch = 0;
	int i, n, nBlocks;
	float omega, CharLength;

	CharLength = OBSTR1*2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);

	cout<<"omega : "<<omega<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<"x"<<BLOCKSIZEZ<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<"x"<<ZDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

    int zInner = ZDIM/GPU_N-2;

	//nBlocks does not include the halo layers
	nBlocks = ((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX)*((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY)
				*((zInner+BLOCKSIZEZ-1)/BLOCKSIZEZ);
	int B = BLOCKSIZEX*BLOCKSIZEY*BLOCKSIZEZ;
	n = nBlocks*B;
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	//2 halo layers per GPU (for 2 GPUs)
    dim3 grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),(zInner)/BLOCKSIZEZ);
    dim3 g_grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);
    dim3 h_grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);

	hipStream_t stream[2];

	hipSetDevice(0);
	
    //buffers g,h
	float *gA_h;
	float *gA_d ,*gB_d;
	float *gA_temp_d;
	float *hA_h;
	float *hA_d ,*hB_d;
	float *hA_temp_d;

	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	//image_h = (int *)malloc(memsize_int);
	float *fA_h,*fA_d,*fB_d;
	float *FX_h,*FY_h,*FZ_h,*FX_d,*FY_d,*FZ_d;
	gA_h = (float *)malloc(XDIM*YDIM*19*sizeof(float));
	hA_h = (float *)malloc(XDIM*YDIM*19*sizeof(float));
	fA_h = (float *)malloc(XDIM*YDIM*zInner*sizeof(float)*19);//(float *)malloc(memsize *19);
	FX_h = (float *)malloc(TMAX*sizeof(float));
	FY_h = (float *)malloc(TMAX*sizeof(float));
	FZ_h = (float *)malloc(TMAX*sizeof(float));
	hipMallocPitch((void **) &fA_d, &pitch, XDIM*sizeof(float), YDIM*zInner*19);
	hipMallocPitch((void **) &fB_d, &pitch, XDIM*sizeof(float), YDIM*zInner*19);
	hipMallocPitch((void **) &gA_d, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &gB_d, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &hA_d, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &hB_d, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &gA_temp_d, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &hA_temp_d, &pitch, XDIM*sizeof(float), YDIM*19);

	hipMalloc((void **) &FX_d, TMAX*sizeof(float));
	hipMalloc((void **) &FY_d, TMAX*sizeof(float));
	hipMalloc((void **) &FZ_d, TMAX*sizeof(float));

	size_t pitch_elements = pitch/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	
	//initialize host f
	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = 0;
	}
	//initialize host g,h
	for (i = 0; i < XDIM*YDIM*19; i++)
	{
		gA_h[i] = 0;
		hA_h[i] = 0;
	}
	//initialize host FX
	for (i = 0; i < TMAX; i++){
		FX_h[i] = 0.f;
		FY_h[i] = 0.f;
		FZ_h[i] = 0.f;
	}

	//memcpy FX
	hipMemcpy(FX_d, FX_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FY_d, FY_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FZ_d, FZ_h, TMAX*sizeof(float), hipMemcpyHostToDevice);


	hipMemcpy2D(fA_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyHostToDevice);
	hipMemcpy2D(gA_d,pitch ,gA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(gB_d,pitch ,gA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(hA_d,pitch ,hA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(hB_d,pitch ,hA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);


	hipSetDevice(1);
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	
    float *gA_d2,*gB_d2;
	float *gA_h2;
	float *gA_temp_d2;
	float *hA_d2,*hB_d2;
	float *hA_h2;
	float *hA_temp_d2;

	float *fA_h2,*fA_d2,*fB_d2;
	float *FX_h2,*FY_h2,*FZ_h2,*FX_d2,*FY_d2,*FZ_d2;
	gA_h2 = (float *)malloc(XDIM*YDIM*19*sizeof(float));
	hA_h2 = (float *)malloc(XDIM*YDIM*19*sizeof(float));
	fA_h2 = (float *)malloc(XDIM*YDIM*zInner*sizeof(float)*19);//(float *)malloc(memsize*19);
	FX_h2 = (float *)malloc(TMAX*sizeof(float));
	FY_h2 = (float *)malloc(TMAX*sizeof(float));
	FZ_h2 = (float *)malloc(TMAX*sizeof(float));
	hipMallocPitch((void **) &fA_d2, &pitch, XDIM*sizeof(float), YDIM*zInner*19);
	hipMallocPitch((void **) &fB_d2, &pitch, XDIM*sizeof(float), YDIM*zInner*19);
	hipMallocPitch((void **) &gA_d2, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &gB_d2, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &hA_d2, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &hB_d2, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &gA_temp_d2, &pitch, XDIM*sizeof(float), YDIM*19);
	hipMallocPitch((void **) &hA_temp_d2, &pitch, XDIM*sizeof(float), YDIM*19);

	hipMalloc((void **) &FX_d2, TMAX*sizeof(float));
	hipMalloc((void **) &FY_d2, TMAX*sizeof(float));
	hipMalloc((void **) &FZ_d2, TMAX*sizeof(float));

	for (i = 0; i < n*19; i++)
	{
		fA_h2[i] = 0;
	}
	//initialize host g,h
	for (i = 0; i < XDIM*YDIM*19; i++)
	{
		gA_h2[i] = 0;
		hA_h2[i] = 0;
	}

	for (i = 0; i < TMAX; i++){
		FX_h2[i] = 0.f;
		FY_h2[i] = 0.f;
		FZ_h2[i] = 0.f;
	}

	hipMemcpy(FX_d2, FX_h2, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FY_d2, FY_h2, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FZ_d2, FZ_h2, TMAX*sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy2D(fA_d2,pitch ,fA_h2,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d2,pitch ,fA_h2,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyHostToDevice);
	hipMemcpy2D(gA_d2,pitch ,gA_h2,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(gB_d2,pitch ,gA_h2,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(hA_d2,pitch ,hA_h2,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(hB_d2,pitch ,hA_h2,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*19,hipMemcpyHostToDevice);

	hipSetDevice(0);
	initialize_single<<<grid, threads>>>(fA_d,pitch_elements);
	initialize_single<<<grid, threads>>>(fB_d,pitch_elements);
	initialize_buffer<<<g_grid, threads>>>(gA_d,pitch_elements);
	initialize_buffer<<<h_grid, threads>>>(hA_d,pitch_elements);
	initialize_buffer<<<g_grid, threads>>>(gB_d,pitch_elements);
	initialize_buffer<<<h_grid, threads>>>(hB_d,pitch_elements);
	hipSetDevice(1);
	initialize_single<<<grid, threads>>>(fA_d2,pitch_elements);
	initialize_single<<<grid, threads>>>(fB_d2,pitch_elements);
	initialize_buffer<<<g_grid, threads>>>(gA_d2,pitch_elements);
	initialize_buffer<<<h_grid, threads>>>(hA_d2,pitch_elements);
	initialize_buffer<<<g_grid, threads>>>(gB_d2,pitch_elements);
	initialize_buffer<<<h_grid, threads>>>(hB_d2,pitch_elements);
	hipSetDevice(0);


	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
    //time loop
	for(int t = 0; t<1; t=t++){

		//memcpy from dev0 to dev1
		//cudaMemcpyPeerAsync(*dst,dstdev#,*src,srcdev#,size_t(bytes),cudastream#);

        //send g from dev1 to h of dev0
		hipMemcpyPeerAsync(&hA_temp_d[pitch_elements*YDIM*14],0,&gA_d2[pitch_elements*YDIM*14],1,pitch_elements*YDIM*sizeof(float)*5,stream[0]);
        //send g from dev0 to h of dev1
		hipMemcpyPeerAsync(&hA_temp_d2[pitch_elements*YDIM*14],1,&gA_d[pitch_elements*YDIM*14],0,pitch_elements*YDIM*sizeof(float)*5,stream[0]);

        //send h from dev1 to g of dev0
		hipMemcpyPeerAsync(&gA_temp_d[pitch_elements*YDIM*9],0,&hA_d2[pitch_elements*YDIM*9],1,pitch_elements*YDIM*sizeof(float)*5,stream[0]);
        //send h from dev0 to g of dev1
		hipMemcpyPeerAsync(&gA_temp_d2[pitch_elements*YDIM*9],1,&hA_d[pitch_elements*YDIM*9],0,pitch_elements*YDIM*sizeof(float)*5,stream[0]);
	
    hipDeviceSynchronize();

		hipSetDevice(0);
		update_inner <<<grid, threads>>>(fA_d,fB_d,gA_d,hA_d,omega,pitch_elements,0);
		update_top   <<<h_grid, threads>>>(hA_d,hB_d,fA_d,hA_temp_d,omega,pitch_elements,0);
		update_bottom<<<h_grid, threads>>>(gA_d,gB_d,fA_d,gA_temp_d,omega,pitch_elements,0);

        hipSetDevice(1);
		update_inner <<<grid, threads>>>(fA_d2,fB_d2,gA_d2,hA_d2,omega,pitch_elements,1);
		update_top   <<<h_grid, threads>>>(hA_d2,hB_d2,fA_d2,hA_temp_d2,omega,pitch_elements,1);
		update_bottom<<<h_grid, threads>>>(gA_d2,gB_d2,fA_d2,gA_temp_d2,omega,pitch_elements,1);


//		if(METHOD == "SINGLE"){
//		if(t >= STARTF)
//		mrt_d_single_force<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,FX_d,FY_d,FZ_d,t,uAv_d,vAv_d,ufluc_d,vfluc_d,0);
//		else
//		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,0);
//		
//
//		if(t >= STARTF)
//		mrt_d_single_force<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,FX_d,FY_d,FZ_d,t+1,uAv_d,vAv_d,ufluc_d,vfluc_d,0);
//		else
//		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,0);
//		}
//
//		cudaSetDevice(1);
//
//		if(METHOD == "SINGLE"){
//		if(t >= STARTF)
//		mrt_d_single_force<<<grid, threads>>>(fA_d2,fB_d2,omega,pitch_elements,FX_d2,FY_d2,FZ_d2,t,uAv_d2,vAv_d2,ufluc_d2,vfluc_d2,1);
//		else
//		mrt_d_single<<<grid, threads>>>(fA_d2,fB_d2,omega,pitch_elements,1);
//		
//
//		if(t >= STARTF)
//		mrt_d_single_force<<<grid, threads>>>(fB_d2,fA_d2,omega,pitch_elements,FX_d2,FY_d2,FZ_d2,t+1,uAv_d2,vAv_d2,ufluc_d2,vfluc_d2,1);
//		else
//		mrt_d_single<<<grid, threads>>>(fB_d2,fA_d2,omega,pitch_elements,1);
//		}
//		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();
	hipSetDevice(0);
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	int Nodes;
	Nodes = XDIM*YDIM*ZDIM;
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)";
	cout<<endl;
	cout<<XDIM<<","<<YDIM<<","<<ZDIM<<","<<TMAX<<","<<restime<<endl;
	

	hipMemcpy2D(fA_h,XDIM*sizeof(float),fB_d,pitch,XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyDeviceToHost);
	hipMemcpy2D(gA_h,XDIM*sizeof(float),gB_d,pitch,XDIM*sizeof(float),YDIM*19,hipMemcpyDeviceToHost);
	hipMemcpy2D(hA_h,XDIM*sizeof(float),hB_d,pitch,XDIM*sizeof(float),YDIM*19,hipMemcpyDeviceToHost);

	hipMemcpy(FX_h, FX_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FY_h, FY_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FZ_h, FZ_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);

	hipSetDevice(1);

	hipMemcpy2D(fA_h2,XDIM*sizeof(float),fB_d2,pitch,XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyDeviceToHost);
	hipMemcpy2D(gA_h2,XDIM*sizeof(float),gB_d2,pitch,XDIM*sizeof(float),YDIM*19,hipMemcpyDeviceToHost);
	hipMemcpy2D(hA_h2,XDIM*sizeof(float),hB_d2,pitch,XDIM*sizeof(float),YDIM*19,hipMemcpyDeviceToHost);


	hipMemcpy(FX_h2, FX_d2, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FY_h2, FY_d2, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FZ_h2, FZ_d2, TMAX*sizeof(float), hipMemcpyDeviceToHost);

	hipSetDevice(0);

	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\",\"uAv\",\"vAv\",\"ufluc\",\"vfluc\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<", K="<<ZDIM<<"\n";
	
	int row = 0;
	int col = 0;
	int dep = 0;
	i = 0;
	float rho, u, v, w;//, usqr;
	//int j;
    int check = 0;
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
 			i = row*XDIM+col;
            f0 = gA_h[i+XDIM*YDIM*0 ];
            f1 = gA_h[i+XDIM*YDIM*1 ];
            f2 = gA_h[i+XDIM*YDIM*2 ];
            f3 = gA_h[i+XDIM*YDIM*3 ];
            f4 = gA_h[i+XDIM*YDIM*4 ];
            f5 = gA_h[i+XDIM*YDIM*5 ];
            f6 = gA_h[i+XDIM*YDIM*6 ];
            f7 = gA_h[i+XDIM*YDIM*7 ];
            f8 = gA_h[i+XDIM*YDIM*8 ];
            f9 = gA_h[i+XDIM*YDIM*9 ];
            f10= gA_h[i+XDIM*YDIM*10];
            f11= gA_h[i+XDIM*YDIM*11];
            f12= gA_h[i+XDIM*YDIM*12];
            f13= gA_h[i+XDIM*YDIM*13];
            f14= gA_h[i+XDIM*YDIM*14];
            f15= gA_h[i+XDIM*YDIM*15];
            f16= gA_h[i+XDIM*YDIM*16];
            f17= gA_h[i+XDIM*YDIM*17];
            f18= gA_h[i+XDIM*YDIM*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));           
			output<<col<<", "<<row<<", "<<0<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
                  //<<uAv_h[i]<<","<<vAv_h[i]<<", "<<ufluc_h[i]<<","<<vfluc_h[i]<<endl;
                  <<f0<<","<<f1<<", "<<f9<<","<<f18<<endl;
        }
    }


	for(dep = 0; dep<zInner; dep++){
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = dep*XDIM*YDIM+row*XDIM+col;
            f0 = fA_h[i+XDIM*YDIM*zInner*0 ];
            f1 = fA_h[i+XDIM*YDIM*zInner*1 ];
            f2 = fA_h[i+XDIM*YDIM*zInner*2 ];
            f3 = fA_h[i+XDIM*YDIM*zInner*3 ];
            f4 = fA_h[i+XDIM*YDIM*zInner*4 ];
            f5 = fA_h[i+XDIM*YDIM*zInner*5 ];
            f6 = fA_h[i+XDIM*YDIM*zInner*6 ];
            f7 = fA_h[i+XDIM*YDIM*zInner*7 ];
            f8 = fA_h[i+XDIM*YDIM*zInner*8 ];
            f9 = fA_h[i+XDIM*YDIM*zInner*9 ];
            f10= fA_h[i+XDIM*YDIM*zInner*10];
            f11= fA_h[i+XDIM*YDIM*zInner*11];
            f12= fA_h[i+XDIM*YDIM*zInner*12];
            f13= fA_h[i+XDIM*YDIM*zInner*13];
            f14= fA_h[i+XDIM*YDIM*zInner*14];
            f15= fA_h[i+XDIM*YDIM*zInner*15];
            f16= fA_h[i+XDIM*YDIM*zInner*16];
            f17= fA_h[i+XDIM*YDIM*zInner*17];
            f18= fA_h[i+XDIM*YDIM*zInner*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));

			output<<col<<", "<<row<<", "<<dep+1<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
                  //<<uAv_h[i]<<","<<vAv_h[i]<<", "<<ufluc_h[i]<<","<<vfluc_h[i]<<endl;
                  <<f0<<","<<f1<<", "<<f9<<","<<f18<<endl;
            if(rho>-0.001f && rho<2.f){
            }
            else{
                check = 1;
            }
		}
	}
	}
    //top of GPU1
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
 			i = row*XDIM+col;
            f0 = hA_h[i+XDIM*YDIM*0 ];
            f1 = hA_h[i+XDIM*YDIM*1 ];
            f2 = hA_h[i+XDIM*YDIM*2 ];
            f3 = hA_h[i+XDIM*YDIM*3 ];
            f4 = hA_h[i+XDIM*YDIM*4 ];
            f5 = hA_h[i+XDIM*YDIM*5 ];
            f6 = hA_h[i+XDIM*YDIM*6 ];
            f7 = hA_h[i+XDIM*YDIM*7 ];
            f8 = hA_h[i+XDIM*YDIM*8 ];
            f9 = hA_h[i+XDIM*YDIM*9 ];
            f10= hA_h[i+XDIM*YDIM*10];
            f11= hA_h[i+XDIM*YDIM*11];
            f12= hA_h[i+XDIM*YDIM*12];
            f13= hA_h[i+XDIM*YDIM*13];
            f14= hA_h[i+XDIM*YDIM*14];
            f15= hA_h[i+XDIM*YDIM*15];
            f16= hA_h[i+XDIM*YDIM*16];
            f17= hA_h[i+XDIM*YDIM*17];
            f18= hA_h[i+XDIM*YDIM*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));           
			output<<col<<", "<<row<<", "<<ZDIM/2-1<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
                  //<<uAv_h[i]<<","<<vAv_h[i]<<", "<<ufluc_h[i]<<","<<vfluc_h[i]<<endl;
                  <<f0<<","<<f1<<", "<<f9<<","<<f18<<endl;
        }
    }


    //bottom of GPU2
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
 			i = row*XDIM+col;
            f0 = gA_h2[i+XDIM*YDIM*0 ];
            f1 = gA_h2[i+XDIM*YDIM*1 ];
            f2 = gA_h2[i+XDIM*YDIM*2 ];
            f3 = gA_h2[i+XDIM*YDIM*3 ];
            f4 = gA_h2[i+XDIM*YDIM*4 ];
            f5 = gA_h2[i+XDIM*YDIM*5 ];
            f6 = gA_h2[i+XDIM*YDIM*6 ];
            f7 = gA_h2[i+XDIM*YDIM*7 ];
            f8 = gA_h2[i+XDIM*YDIM*8 ];
            f9 = gA_h2[i+XDIM*YDIM*9 ];
            f10= gA_h2[i+XDIM*YDIM*10];
            f11= gA_h2[i+XDIM*YDIM*11];
            f12= gA_h2[i+XDIM*YDIM*12];
            f13= gA_h2[i+XDIM*YDIM*13];
            f14= gA_h2[i+XDIM*YDIM*14];
            f15= gA_h2[i+XDIM*YDIM*15];
            f16= gA_h2[i+XDIM*YDIM*16];
            f17= gA_h2[i+XDIM*YDIM*17];
            f18= gA_h2[i+XDIM*YDIM*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));           
			output<<col<<", "<<row<<", "<<ZDIM/2<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
                  //<<uAv_h[i]<<","<<vAv_h[i]<<", "<<ufluc_h[i]<<","<<vfluc_h[i]<<endl;
                  <<f0<<","<<f1<<", "<<f9<<","<<f18<<endl;
        }
    }

	for(dep = 0; dep<zInner; dep++){
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = dep*XDIM*YDIM+row*XDIM+col;
            f0 = fA_h2[i+XDIM*YDIM*zInner*0 ];
            f1 = fA_h2[i+XDIM*YDIM*zInner*1 ];
            f2 = fA_h2[i+XDIM*YDIM*zInner*2 ];
            f3 = fA_h2[i+XDIM*YDIM*zInner*3 ];
            f4 = fA_h2[i+XDIM*YDIM*zInner*4 ];
            f5 = fA_h2[i+XDIM*YDIM*zInner*5 ];
            f6 = fA_h2[i+XDIM*YDIM*zInner*6 ];
            f7 = fA_h2[i+XDIM*YDIM*zInner*7 ];
            f8 = fA_h2[i+XDIM*YDIM*zInner*8 ];
            f9 = fA_h2[i+XDIM*YDIM*zInner*9 ];
            f10= fA_h2[i+XDIM*YDIM*zInner*10];
            f11= fA_h2[i+XDIM*YDIM*zInner*11];
            f12= fA_h2[i+XDIM*YDIM*zInner*12];
            f13= fA_h2[i+XDIM*YDIM*zInner*13];
            f14= fA_h2[i+XDIM*YDIM*zInner*14];
            f15= fA_h2[i+XDIM*YDIM*zInner*15];
            f16= fA_h2[i+XDIM*YDIM*zInner*16];
            f17= fA_h2[i+XDIM*YDIM*zInner*17];
            f18= fA_h2[i+XDIM*YDIM*zInner*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));

			output<<col<<", "<<row<<", "<<ZDIM/2+dep+1<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
                  //<<uAv_h2[i]<<","<<vAv_h2[i]<<", "<<ufluc_h2[i]<<","<<vfluc_h2[i]<<endl;
                  <<f0<<","<<f1<<", "<<f9<<","<<f18<<endl;
            if(rho>0.f && rho<2.f){
            }
            else{
                check = 1;
            }
		}
	}
	}
    //top of GPU2
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
 			i = row*XDIM+col;
            f0 = hA_h2[i+XDIM*YDIM*0 ];
            f1 = hA_h2[i+XDIM*YDIM*1 ];
            f2 = hA_h2[i+XDIM*YDIM*2 ];
            f3 = hA_h2[i+XDIM*YDIM*3 ];
            f4 = hA_h2[i+XDIM*YDIM*4 ];
            f5 = hA_h2[i+XDIM*YDIM*5 ];
            f6 = hA_h2[i+XDIM*YDIM*6 ];
            f7 = hA_h2[i+XDIM*YDIM*7 ];
            f8 = hA_h2[i+XDIM*YDIM*8 ];
            f9 = hA_h2[i+XDIM*YDIM*9 ];
            f10= hA_h2[i+XDIM*YDIM*10];
            f11= hA_h2[i+XDIM*YDIM*11];
            f12= hA_h2[i+XDIM*YDIM*12];
            f13= hA_h2[i+XDIM*YDIM*13];
            f14= hA_h2[i+XDIM*YDIM*14];
            f15= hA_h2[i+XDIM*YDIM*15];
            f16= hA_h2[i+XDIM*YDIM*16];
            f17= hA_h2[i+XDIM*YDIM*17];
            f18= hA_h2[i+XDIM*YDIM*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));           
			output<<col<<", "<<row<<", "<<ZDIM-1<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
                  //<<uAv_h[i]<<","<<vAv_h[i]<<", "<<ufluc_h[i]<<","<<vfluc_h[i]<<endl;
                  <<f0<<","<<f1<<", "<<f9<<","<<f18<<endl;
        }
    }

    if(check == 1) cout<<"error!"<<endl;

	output.close();

	//for(int t = STARTF-1; t<TMAX; t++){
	for(int t = 0; t<TMAX; t++){
		output2<<t<<", "<<FX_h[t]/(0.5f*UMAX*UMAX*2.f*OBSTR1*ZDIM)<<", "
						<<FY_h[t]/(0.5f*UMAX*UMAX*2.f*OBSTR1*ZDIM)<<", "
						<<FZ_h[t]/(0.5f*UMAX*UMAX*2.f*OBSTR1*ZDIM)<<endl;
//		output2<<t<<", "<<FX_h[t]/(0.5f*UMAX*UMAX*OBSTR1*OBSTR1*3.14158f)<<", "
//						<<FY_h[t]/(0.5f*UMAX*UMAX*OBSTR1*OBSTR1*3.14158f)<<", "
//						<<FZ_h[t]/(0.5f*UMAX*UMAX*OBSTR1*OBSTR1*3.14158f)<<endl;
	}
	output2.close();



	//cudaFree(image_d);


	hipFree(fA_d);
	hipFree(fB_d);
	return(0);

}
