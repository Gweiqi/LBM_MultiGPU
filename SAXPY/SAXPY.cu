#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;


void saxpy_cpu(float *vecY, float *vecX, float alpha, int n)
{
	int i;

	for(i = 0; i<n; i++)
		vecY[i] = alpha*vecX[i]+vecY[i];
}


__global__ void saxpy_gpu(float *vecY, float *vecX, float alpha, int n)
{
	int i;
	i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n)
		vecY[i] = alpha*vecX[i]+vecY[i];
}



int main(int argc, char *argv[])
{
	float *x_host, *y_host;
	float *x_dev, *y_dev;
	float *y_shadow;

	ofstream output;
	output.open ("saxpy_out.dat");

	int n = 16*1024*1024;
	float alpha = 0.5f;

	size_t memsize;
	int i, blockSize, nBlocks;

	CUT_DEVICE_INIT(argc,argv);

	memsize = n*sizeof(float);

	x_host = (float *)malloc(memsize);
	y_host = (float *)malloc(memsize);
	y_shadow = (float *)malloc(memsize);

	CUDA_SAFE_CALL(hipMalloc((void **) &x_dev, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &y_dev, memsize));

	for (i = 0; i < n; i++)
	{
		x_host[i] = rand()/(float)RAND_MAX;
		y_host[i] = rand()/(float)RAND_MAX;
	}

	CUDA_SAFE_CALL(hipMemcpy(x_dev, x_host, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(y_dev, y_host, memsize, hipMemcpyHostToDevice));

	blockSize = 512;
	nBlocks = n/blockSize+(n%blockSize > 0);

	saxpy_gpu<<<nBlocks, blockSize>>>(y_dev, x_dev, alpha, n);

	CUT_CHECK_ERROR("Kernel execution failed");

	saxpy_cpu(y_host,x_host,alpha,n);

	CUDA_SAFE_CALL(hipMemcpy(y_shadow, y_dev, memsize, hipMemcpyDeviceToHost));

	if(cutComparef(y_shadow,y_host,n))
		printf("Passed!\n");
	else
		printf("Failed!\n");
		
	for(i = 0; i<20; i++)
	{
		output<<x_host[i]<<endl;
	}
	cout<<y_host[0]<<endl;

	free(x_host);
	free(y_host);
	free(y_shadow);

	output.close();

	CUDA_SAFE_CALL(hipFree(x_dev));
	CUDA_SAFE_CALL(hipFree(y_dev));

	return(0);

}
