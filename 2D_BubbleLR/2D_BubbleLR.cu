#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define CASENAME "test"

#define BLOCKSIZEX 128
#define BLOCKSIZEY 1
#define BLOCKSIZELRX 64
#define BLOCKSIZELRY 1
#define XDIM 256
#define YDIM 64
#define TMAX 30000
#define OBSTR 2.f
#define OBSTX 63.5f
#define OBSTY 31.5f

#define STARTF 0

//#define LRFACTOR 0.5f
//#define LRLEVEL  2 //inverse of LRFACTOR as an integer
//#define LRX0 47.75	//minimum x coord of LR
//#define XLRDIM 64	//number of nodes in x
//#define LRY0 15.75
//#define YLRDIM 64

#define LRFACTOR 0.25f
#define LRLEVEL  4 //inverse of LRFACTOR as an integer
#define LRX0 47.625	//minimum x coord of LR
#define XLRDIM 128	//number of nodes in x
#define LRY0 15.625
#define YLRDIM 128


#define RE 20.f
#define UMAX 0.08f
#define METHOD "SINGLE" //LR,SINGLE,HYB,TEXT,SHARED,SINGLEF,TEST
#define REFINEMENT "YES" //YES,NO
#define MODEL "MRT" //BGK,MRT,STREAM
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>


/*
Image List:
0  fluid
1  BB
2  Neumann East
3  Dirichlet West
10 BB(force)
11 xsymmetry(y=max)
12 xsymmetry(y=min)

*/
inline __device__ int ImageFcn(float x, float y){
	//if(y < 0.1f || (XDIM-x) < 1.1f || (YDIM-y) < 1.1f)
//	if(y < 0.1f || (YDIM-y) < 1.1f)
//		return 1;
//	else if(x < 0.1f)
//		return 3;//west
//	else if((XDIM-x) < 1.1f)
//		return 2;//east
//	else if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
	//if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
	if(x<(OBSTX+OBSTR) && x>(OBSTX-OBSTR) && y<(OBSTY+OBSTR) && y>(OBSTY-OBSTR))
		return 10;
	else
		return 0;
}

inline __device__ int ImageFcn(int x, int y){
	//if(y == 0 || x == XDIM-1 || y == YDIM-1)
	if(x < 0.1f)
		return 3;//west
	else if((XDIM-x) < 1.1f)
		return 2;//east
	else if((YDIM-y) < 1.1f)
		return 1;//11;//xsymmetry top
	else if(y < 0.1f)
		return 1;//12;//xsymmetry bottom
	//else if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
	else if(x<(OBSTX+OBSTR) && x>(OBSTX-OBSTR) && y<(OBSTY+OBSTR) && y>(OBSTY-OBSTR))
		return 10;
	else
		return 0;
//	return 1;
}

inline __device__ float PoisProf (float x){
	float radius = (YDIM-1-1)*0.5f;
	float result = -1.0f*(((1.0f-(x-0.5f)/radius))*((1.0f-(x-0.5f)/radius))-1.0f);
	return (result);
//	return 1.f;
}



texture<float,2,hipReadModeElementType> texRef_f0A;texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;
texture<float,2,hipReadModeElementType> texRef_f0B;texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;
texture<float,2,hipReadModeElementType> texRef_f0C;texture<float,2,hipReadModeElementType> texRef_f1C;
texture<float,2,hipReadModeElementType> texRef_f2C;texture<float,2,hipReadModeElementType> texRef_f3C;
texture<float,2,hipReadModeElementType> texRef_f4C;texture<float,2,hipReadModeElementType> texRef_f5C;
texture<float,2,hipReadModeElementType> texRef_f6C;texture<float,2,hipReadModeElementType> texRef_f7C;
texture<float,2,hipReadModeElementType> texRef_f8C;
texture<float,2,hipReadModeElementType> texRef_f0D;texture<float,2,hipReadModeElementType> texRef_f1D;
texture<float,2,hipReadModeElementType> texRef_f2D;texture<float,2,hipReadModeElementType> texRef_f3D;
texture<float,2,hipReadModeElementType> texRef_f4D;texture<float,2,hipReadModeElementType> texRef_f5D;
texture<float,2,hipReadModeElementType> texRef_f6D;texture<float,2,hipReadModeElementType> texRef_f7D;
texture<float,2,hipReadModeElementType> texRef_f8D;


int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 ,  float omega)
{
//	float rho,u,v;	
//	//rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
//	u = f1-f3+f5-f6-f7+f8;
//	v = f2-f4+f5+f6-f7-f8;
//	//float usqr = u*u+v*v;
//	f0 = f0 -omega*(f0 -0.4444444444f*(f0+f1+f2+f3+f4+f5+f6+f7+f8                                    -1.5f*(u*u)-1.5f*(v*v)));
//	f1 = f1 -omega*(f1 -0.1111111111f*(f0+f1+f2+f3+f4+f5+f6+f7+f8+3.0f*(f1-f3+f5-f6-f7+f8)     +3.0f*u*u                          -1.5f*(v*v)));                
//	f2 = f2 -omega*(f2 -0.1111111111f*(f0+f1+f2+f3+f4+f5+f6+f7+f8+3.0f*(f2-f4+f5+f6-f7-f8)     +3.0f*v*v               -1.5f*(u*u)           ));
//	f3 = f3 -omega*(f3 -0.1111111111f*(f0+f1+f2+f3+f4+f5+f6+f7+f8-3.0f*(f1-f3+f5-f6-f7+f8)     +3.0f*u*u                          -1.5f*(v*v)));
//	f4 = f4 -omega*(f4 -0.1111111111f*(f0+f1+f2+f3+f4+f5+f6+f7+f8-3.0f*(f2-f4+f5+f6-f7-f8)     +3.0f*v*v               -1.5f*(u*u)           ));
//	f5 = f5 -omega*(f5 -0.02777777778*(f0+f1+f2+f3+f4+f5+f6+f7+f8+3.0f*( (f1-f3+f5   -f7   )+(f2-f4+f5   -f7   ))+3.0f*u*u+9.f*u*v+3.0f*v*v));
//	f6 = f6 -omega*(f6 -0.02777777778*(f0+f1+f2+f3+f4+f5+f6+f7+f8+3.0f*(-(f1-f3   -f6   +f8)+(f2-f4   +f6   -f8))+3.0f*u*u-9.f*u*v+3.0f*v*v));
//	f7 = f7 -omega*(f7 -0.02777777778*(f0+f1+f2+f3+f4+f5+f6+f7+f8+3.0f*(-(f1-f3+f5   -f7   )-(f2-f4+f5   -f7   ))+3.0f*u*u-9.f*u*v+3.0f*v*v));
//	f8 = f8 -omega*(f8 -0.02777777778*(f0+f1+f2+f3+f4+f5+f6+f7+f8+3.0f*( (f1-f3   -f6   +f8)-(f2-f4   +f6   -f8))+3.0f*u*u-9.f*u*v+3.0f*v*v));
	                                                                             
    float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	f0 = f0 -omega*(f0 -0.4444444444f*(rho-1.5f*usqr));
	f1 = f1 -omega*(f1 -0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 = f2 -omega*(f2 -0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 = f3 -omega*(f3 -0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 = f4 -omega*(f4 -0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 = f5 -omega*(f5 -0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	f6 = f6 -omega*(f6 -0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 = f7 -omega*(f7 -0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 = f8 -omega*(f8 -0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
}

__device__ void mrt_collide(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, float omega)
{
	//float rho,u,v;	
	float u,v;	
	//rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float m1,m2,m4,m6,m7,m8;

//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	m1 =-2.f*f0 +    f1 +    f2 +    f3 +    f4+ 4.f*f5+ 4.f*f6+ 4.f*f7+ 4.f*f8-3.0f*(u*u+v*v);
	//m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	m2 = 3.f*f0 -3.f*f1 -3.f*f2 -3.f*f3 -3.f*f4                                +3.0f*(u*u+v*v); //ep
	//m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	m4 =        -    f1        +     f3        + 2.f*f5 -2.f*f6 -2.f*f7+ 2.f*f8;//-(-u);//qx_eq
	m6 =                -    f2        +     f4+ 2.f*f5+ 2.f*f6 -2.f*f7 -2.f*f8;//-(-v);//qy_eq
	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq

//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
//	m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
//	m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
//	m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
//	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
//	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq

	f0=f0-(-m1+m2)*0.11111111f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
	//f1=f1-(-m1-2.0f*(m2+m4)+m7*omega*9.0f)*0.027777777f;
	f1=f1-(-m1*0.027777777f-0.05555555556f*m2-0.16666666667f*m4+m7*omega*0.25f);
	f2=f2-(-m1*0.027777777f-0.05555555556f*m2-0.16666666667f*m6-m7*omega*0.25f);
	f3=f3-(-m1*0.027777777f-0.05555555556f*m2+0.16666666667f*m4+m7*omega*0.25f);
	f4=f4-(-m1*0.027777777f-0.05555555556f*m2+0.16666666667f*m6-m7*omega*0.25f);
	f5=f5-(0.05555555556f*m1+m2*0.027777777f+0.08333333333f*m4+0.08333333333f*m6+m8*omega*0.25f);
	f6=f6-(0.05555555556f*m1+m2*0.027777777f-0.08333333333f*m4+0.08333333333f*m6-m8*omega*0.25f);
	f7=f7-(0.05555555556f*m1+m2*0.027777777f-0.08333333333f*m4-0.08333333333f*m6+m8*omega*0.25f);
	f8=f8-(0.05555555556f*m1+m2*0.027777777f+0.08333333333f*m4-0.08333333333f*m6-m8*omega*0.25f);
//	f1=f1-(-m1-2.f*m2-6.f*m4+m7*omega*9.0f)*0.027777777f;
//	f2=f2-(-m1-2.f*m2-6.f*m6-m7*omega*9.0f)*0.027777777f;
//	f3=f3-(-m1-2.f*m2+6.f*m4+m7*omega*9.0f)*0.027777777f;
//	f4=f4-(-m1-2.f*m2+6.f*m6-m7*omega*9.0f)*0.027777777f;
//	f5=f5-(2.f*m1+m2+3.f*m4+3.f*m6+m8*omega*9.0f)*0.027777777f;
//	f6=f6-(2.f*m1+m2-3.f*m4+3.f*m6-m8*omega*9.0f)*0.027777777f;
//	f7=f7-(2.f*m1+m2-3.f*m4-3.f*m6+m8*omega*9.0f)*0.027777777f;
//	f8=f8-(2.f*m1+m2+3.f*m4-3.f*m6-m8*omega*9.0f)*0.027777777f;
}

__device__ void NeumannEast(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, int y)
{
	if(y == 0){
		f2 = f4;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f8 = f5;
	}
	float u,rho;
    //float v = 0.0;
    rho = 1.0;
	u = -rho+((f0+f2+f4)+2.0f*f1+2.0f*f5+2.0f*f8);

    f3 = f1-u*0.66666667f;
    f7 = f5+0.5f*(f2-f4)-u*0.16666667f;
    f6 = f8-0.5f*(f2-f4)-u*0.16666667f;
    //f7 = f5+0.5f*(f2-f4)-0.5f*v-u*0.16666667f;
    //f6 = f8-0.5f*(f2-f4)+0.5f*v-u*0.16666667f;
}
__device__ void DirichletWest(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, int y)
{
	if(y == 0){
		f2 = f4;
		f6 = f7;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f7 = f6;
	}
	float u;//,v;//,rho;
	u = UMAX*PoisProf(float(y));
	//v = 0.0f;//0.0;
	f1 = f3+u*0.66666667f;
	f5 = f7-0.5f*(f2-f4)+u*0.166666667f;
	f8 = f6+0.5f*(f2-f4)+u*0.166666667f;	
	//f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	//f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;	
}




__device__ void boundaries(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 ,
					int y, int im)
{
	if(im == 2)//NeumannEast
	{
		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	else if(im == 3)//DirichletWest
	{
		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	else if(im == 11)//xsymmetry
	{
		f4 = f2;
		f7 = f6;
		f8 = f5;
	}
	else if(im == 12)//xsymmetry
	{
		f2 = f4;
		f6 = f7;
		f5 = f8;
	}
}


__device__ void boundaries_force(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 ,
					int y, int im)
{
	if(im == 2)//NeumannEast
	{
		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	else if(im == 3)//DirichletWest
	{
		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	if(im == 11)//xsymmetry
	{
		f4 = f2;
		f7 = f6;
		f8 = f5;
	}
	else if(im == 12)//xsymmetry
	{
		f2 = f4;
		f6 = f7;
		f5 = f8;
	}
}


inline __device__ int f_mem(int f_num, int x, int y, size_t pitch)
{

	return (x+y*pitch)+f_num*pitch*YDIM;
}

inline __device__ int f_memLR(int f_num, int x, int y, size_t pitch)
{

	return (x+y*pitch)+f_num*pitch*YLRDIM;
}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}

__global__ void simple_copy(float* fA, float* fB,
							int *image, float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)

	fB[j] = fA[j];//+0.01f;
}

__global__ void ExtractFromC_d(float* fout,float* f_fine,
							size_t pitch, size_t pitch2, float SF, float omega)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if(x < LRX0+1 || x > LRX0+XLRDIM*LRFACTOR-1 || y < LRY0+1 || y > LRY0+YLRDIM*LRFACTOR-1)
////	if(x < LRX0+2 || x > LRX0+XLRDIM-3 || y < LRY0+2 || y > LRY0+YLRDIM-3)
//	{
//	//do nothing
//	}
//	else
	//if(x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1)
	if( (x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1) && 
	(x == int(LRX0+2) || x == int(LRX0+XLRDIM*LRFACTOR-1) || y == int(LRY0+2) || y == int(LRY0+YLRDIM*LRFACTOR-1)) )
	{
	float xcoord = (x-LRX0)/LRFACTOR;
	float ycoord = (y-LRY0)/LRFACTOR;
    float u_00,u_01,u_10,u_11;
    float v_00,v_01,v_10,v_11;
    float m0_00,m0_01,m0_10,m0_11;
    float m1_00,m1_01,m1_10,m1_11;
    float m2_00,m2_01,m2_10,m2_11;
    float m4_00,m4_01,m4_10,m4_11;
    float m6_00,m6_01,m6_10,m6_11;
    float m7_00,m7_01,m7_10,m7_11;
    float m8_00,m8_01,m8_10,m8_11;
    float Sxx_00,Sxx_01,Sxx_10,Sxx_11;
    float Syy_00,Syy_01,Syy_10,Syy_11;
    float Sxy_00,Sxy_01,Sxy_10,Sxy_11;
    
    //moments for 00
	f0 = f_fine[f_memLR(0 ,int(xcoord),int(ycoord),pitch2)];
	f1 = f_fine[f_memLR(1 ,int(xcoord),int(ycoord),pitch2)];
	f2 = f_fine[f_memLR(2 ,int(xcoord),int(ycoord),pitch2)];
	f3 = f_fine[f_memLR(3 ,int(xcoord),int(ycoord),pitch2)];
	f4 = f_fine[f_memLR(4 ,int(xcoord),int(ycoord),pitch2)];
	f5 = f_fine[f_memLR(5 ,int(xcoord),int(ycoord),pitch2)];
	f6 = f_fine[f_memLR(6 ,int(xcoord),int(ycoord),pitch2)];
	f7 = f_fine[f_memLR(7 ,int(xcoord),int(ycoord),pitch2)];
	f8 = f_fine[f_memLR(8 ,int(xcoord),int(ycoord),pitch2)];
//	f0 = tex2D(texRef_f0C,int(xcoord)+0.5f,int(ycoord)+0.5f);//YDIM and not YLRDIM
//	f1 = tex2D(texRef_f1C,int(xcoord)+0.5f,int(ycoord)+0.5f);
//	f2 = tex2D(texRef_f2C,int(xcoord)+0.5f,int(ycoord)+0.5f);
//	f3 = tex2D(texRef_f3C,int(xcoord)+0.5f,int(ycoord)+0.5f);
//	f4 = tex2D(texRef_f4C,int(xcoord)+0.5f,int(ycoord)+0.5f);
//	f5 = tex2D(texRef_f5C,int(xcoord)+0.5f,int(ycoord)+0.5f);
//	f6 = tex2D(texRef_f6C,int(xcoord)+0.5f,int(ycoord)+0.5f);
//	f7 = tex2D(texRef_f7C,int(xcoord)+0.5f,int(ycoord)+0.5f);
//	f8 = tex2D(texRef_f8C,int(xcoord)+0.5f,int(ycoord)+0.5f);

	m0_00 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_00 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_00 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_00 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_00 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_00 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_00 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_00  = f1-f3+f5-f6-f7+f8;
	v_00  = f2-f4+f5+f6-f7-f8;
    Sxx_00=LRLEVEL*omega*0.75f*( u_00*u_00-v_00*v_00-m7_00);
    Syy_00=LRLEVEL*omega*0.75f*(-u_00*u_00+v_00*v_00+m7_00);
    Sxy_00=LRLEVEL*omega*1.5f*(u_00*v_00-m8_00);

    //moments for 10
	f0 = f_fine[f_memLR(0 ,int(xcoord)+1,int(ycoord),pitch2)];
	f1 = f_fine[f_memLR(1 ,int(xcoord)+1,int(ycoord),pitch2)];
	f2 = f_fine[f_memLR(2 ,int(xcoord)+1,int(ycoord),pitch2)];
	f3 = f_fine[f_memLR(3 ,int(xcoord)+1,int(ycoord),pitch2)];
	f4 = f_fine[f_memLR(4 ,int(xcoord)+1,int(ycoord),pitch2)];
	f5 = f_fine[f_memLR(5 ,int(xcoord)+1,int(ycoord),pitch2)];
	f6 = f_fine[f_memLR(6 ,int(xcoord)+1,int(ycoord),pitch2)];
	f7 = f_fine[f_memLR(7 ,int(xcoord)+1,int(ycoord),pitch2)];
	f8 = f_fine[f_memLR(8 ,int(xcoord)+1,int(ycoord),pitch2)];
//	f0 = tex2D(texRef_f0C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);//YDIM and not YLRDIM
//	f1 = tex2D(texRef_f1C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
//	f2 = tex2D(texRef_f2C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
//	f3 = tex2D(texRef_f3C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
//	f4 = tex2D(texRef_f4C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
//	f5 = tex2D(texRef_f5C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
//	f6 = tex2D(texRef_f6C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
//	f7 = tex2D(texRef_f7C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
//	f8 = tex2D(texRef_f8C,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	m0_10 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_10 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_10 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_10 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_10 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_10 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_10 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_10  = f1-f3+f5-f6-f7+f8;
	v_10  = f2-f4+f5+f6-f7-f8;
    Sxx_10=LRLEVEL*omega*0.75f*(u_10*u_10-v_10*v_10-m7_10);
    Syy_10=LRLEVEL*omega*0.75f*(-u_10*u_10+v_10*v_10+m7_10);
    Sxy_10=LRLEVEL*omega*1.5f*(u_10*v_10-m8_10);

    //moments for 01
	f0 = f_fine[f_memLR(0 ,int(xcoord),int(ycoord)+1,pitch2)];
	f1 = f_fine[f_memLR(1 ,int(xcoord),int(ycoord)+1,pitch2)];
	f2 = f_fine[f_memLR(2 ,int(xcoord),int(ycoord)+1,pitch2)];
	f3 = f_fine[f_memLR(3 ,int(xcoord),int(ycoord)+1,pitch2)];
	f4 = f_fine[f_memLR(4 ,int(xcoord),int(ycoord)+1,pitch2)];
	f5 = f_fine[f_memLR(5 ,int(xcoord),int(ycoord)+1,pitch2)];
	f6 = f_fine[f_memLR(6 ,int(xcoord),int(ycoord)+1,pitch2)];
	f7 = f_fine[f_memLR(7 ,int(xcoord),int(ycoord)+1,pitch2)];
	f8 = f_fine[f_memLR(8 ,int(xcoord),int(ycoord)+1,pitch2)];
//	f0 = tex2D(texRef_f0C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);//YDIM and not YLRDIM
//	f1 = tex2D(texRef_f1C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
//	f2 = tex2D(texRef_f2C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
//	f3 = tex2D(texRef_f3C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
//	f4 = tex2D(texRef_f4C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
//	f5 = tex2D(texRef_f5C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
//	f6 = tex2D(texRef_f6C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
//	f7 = tex2D(texRef_f7C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
//	f8 = tex2D(texRef_f8C,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	m0_01 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_01 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_01 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_01 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_01 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_01 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_01 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_01  = f1-f3+f5-f6-f7+f8;
	v_01  = f2-f4+f5+f6-f7-f8;
    Sxx_01=LRLEVEL*omega*0.75f*(u_01*u_01-v_01*v_01-m7_01);
    Syy_01=LRLEVEL*omega*0.75f*(-u_01*u_01+v_01*v_01+m7_01);
    Sxy_01=LRLEVEL*omega*1.5f*(u_01*v_01-m8_01);

    //moments for 11
	f0 = f_fine[f_memLR(0 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f1 = f_fine[f_memLR(1 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f2 = f_fine[f_memLR(2 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f3 = f_fine[f_memLR(3 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f4 = f_fine[f_memLR(4 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f5 = f_fine[f_memLR(5 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f6 = f_fine[f_memLR(6 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f7 = f_fine[f_memLR(7 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
	f8 = f_fine[f_memLR(8 ,int(xcoord)+1,int(ycoord)+1,pitch2)];
//	f0 = tex2D(texRef_f0C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);//YDIM and not YLRDIM
//	f1 = tex2D(texRef_f1C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
//	f2 = tex2D(texRef_f2C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
//	f3 = tex2D(texRef_f3C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
//	f4 = tex2D(texRef_f4C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
//	f5 = tex2D(texRef_f5C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
//	f6 = tex2D(texRef_f6C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
//	f7 = tex2D(texRef_f7C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
//	f8 = tex2D(texRef_f8C,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	m0_11 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_11 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_11 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_11 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_11 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_11 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_11 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
 	u_11  = f1-f3+f5-f6-f7+f8;
	v_11  = f2-f4+f5+f6-f7-f8;
    Sxx_11=LRLEVEL*omega*0.75f*(u_11*u_11-v_11*v_11-m7_11);
    Syy_11=LRLEVEL*omega*0.75f*(-u_11*u_11+v_11*v_11+m7_11);
    Sxy_11=LRLEVEL*omega*1.5f*(u_11*v_11-m8_11);

    m0_00 = (m0_00*(ycoord-int(ycoord))+m0_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m0_10*(ycoord-int(ycoord))+m0_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m1_00 = (m1_00*(ycoord-int(ycoord))+m1_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m1_10*(ycoord-int(ycoord))+m1_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m2_00 = (m2_00*(ycoord-int(ycoord))+m2_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m2_10*(ycoord-int(ycoord))+m2_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m4_00 = (m4_00*(ycoord-int(ycoord))+m4_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m4_10*(ycoord-int(ycoord))+m4_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m6_00 = (m6_00*(ycoord-int(ycoord))+m6_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m6_10*(ycoord-int(ycoord))+m6_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m7_00 = (m7_00*(ycoord-int(ycoord))+m7_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m7_10*(ycoord-int(ycoord))+m7_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m8_00 = (m8_00*(ycoord-int(ycoord))+m8_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m8_10*(ycoord-int(ycoord))+m8_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
//    m0_00 = 0.25f*(m0_00+m0_01+m0_10+m0_11);
//    m1_00 = 0.25f*(m1_00+m1_01+m1_10+m1_11);
//    m2_00 = 0.25f*(m2_00+m2_01+m2_10+m2_11);
//    m4_00 = 0.25f*(m4_00+m4_01+m4_10+m4_11);
//    m6_00 = 0.25f*(m6_00+m6_01+m6_10+m6_11);
//    m7_00 = 0.25f*(m7_00+m7_01+m7_10+m7_11);
//    m8_00 = 0.25f*(m8_00+m8_01+m8_10+m8_11);


    float cx,cy,dx,dy;
    cx = 0.0f;//0.0625f*(Sxx_00+Sxx_01-Sxx_10-Sxx_11);
    dy = 0.0f;//0.0625f*(Syy_00-Syy_01+Syy_10-Syy_11);
    cy = 0.0f;//0.125f*(Sxy_00+Sxy_10-Sxy_01-Sxy_11)+0.125f*(v_00+v_11-v_01-v_10);
    dx = 0.0f;//0.125f*(Sxy_00+Sxy_01-Sxy_10-Sxy_11)+0.125f*(u_00+u_11-u_01-u_10);

    float a0,a1,a2,a3,b0,b1,b2,b3;
    float xp,yp;
    xp = 2.f*(xcoord-int(xcoord))-1.f;
    yp = 2.f*(ycoord-int(ycoord))-1.f;

    a0 =0.25f*( u_00+u_10+u_11+u_01);
    a1 =0.25f*(-u_00+u_10+u_11-u_01);
    a2 =0.25f*(-u_00-u_10+u_11+u_01);
    a3 =0.25f*( u_00-u_10+u_11-u_01);
    b0 =0.25f*( v_00+v_10+v_11+v_01);
    b1 =0.25f*(-v_00+v_10+v_11-v_01);
    b2 =0.25f*(-v_00-v_10+v_11+v_01);
    b3 =0.25f*( v_00-v_10+v_11-v_01);

	float rho,u,v;	
    u = a0+a1*xp+a2*yp+a3*xp*yp+cx*(1.f-xp*xp)+cy*(1.f-yp*yp);
    v = b0+b1*xp+b2*yp+b3*xp*yp+dx*(1.f-xp*xp)+dy*(1.f-yp*yp);
    rho = m0_00;

//    u = u_00+u_01+u_10+u_11;
//    v = v_00+v_01+v_11+v_11;
//    u *= 0.25f;
//    v *= 0.25f;

    m1_00 = m1_00*SF+(1.f-SF)*(-2.0f*rho+3.0f*(u*u+v*v));
    m2_00 = m2_00*SF+(1.f-SF)*(rho-3.0f*(u*u+v*v));
    m4_00 = m4_00*SF+(1.f-SF)*(-u);
    m6_00 = m6_00*SF+(1.f-SF)*(-v);
    m7_00 = m7_00*SF+(1.f-SF)*(u*u-v*v);
    m8_00 = m8_00*SF+(1.f-SF)*(u*v);

    //use rho u v and m's and Minv to get f's. can rescale during this step
f0 = 1.f/9.f*m0_00+-4.f/36.f*m1_00+ 4.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 0.f/4.f*m7_00+ 0.f/4.f*m8_00;
f1 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 1.f/6.f*u+-2.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f2 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+ 1.f/6.f*v+-2.f/12.f*m6_00+-1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f3 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+-1.f/6.f*u+ 2.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f4 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+-1.f/6.f*v+ 2.f/12.f*m6_00+-1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f5 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+ 1.f/6.f*u+ 1.f/12.f*m4_00+ 1.f/6.f*v+ 1.f/12.f*m6_00+ 0.f/4.f*m7_00+ 1.f/4.f*m8_00;
f6 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+-1.f/6.f*u+-1.f/12.f*m4_00+ 1.f/6.f*v+ 1.f/12.f*m6_00+ 0.f/4.f*m7_00+-1.f/4.f*m8_00;
f7 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+-1.f/6.f*u+-1.f/12.f*m4_00+-1.f/6.f*v+-1.f/12.f*m6_00+ 0.f/4.f*m7_00+ 1.f/4.f*m8_00;
f8 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+ 1.f/6.f*u+ 1.f/12.f*m4_00+-1.f/6.f*v+-1.f/12.f*m6_00+ 0.f/4.f*m7_00+-1.f/4.f*m8_00;


	fout[f_mem(0,x,y,pitch)] = f0;
	fout[f_mem(1,x,y,pitch)] = f1;
	fout[f_mem(2,x,y,pitch)] = f2;
	fout[f_mem(3,x,y,pitch)] = f3;
	fout[f_mem(4,x,y,pitch)] = f4;
	fout[f_mem(5,x,y,pitch)] = f5;
	fout[f_mem(6,x,y,pitch)] = f6;
	fout[f_mem(7,x,y,pitch)] = f7;
	fout[f_mem(8,x,y,pitch)] = f8;













////	float xcoord = 2.0f*x-2.0f*LRX0+0.5f;
////	float ycoord = 2.0f*y-2.0f*LRY0+0.5f;
//	float xcoord = (x-LRX0)/LRFACTOR+0.5f;
//	float ycoord = (y-LRY0)/LRFACTOR+0.5f;
//	f0 = tex2D(texRef_f0C ,xcoord,ycoord);
//	f1 = tex2D(texRef_f1C ,xcoord,ycoord);
//	f2 = tex2D(texRef_f2C ,xcoord,ycoord);
//	f3 = tex2D(texRef_f3C ,xcoord,ycoord);
//	f4 = tex2D(texRef_f4C ,xcoord,ycoord);
//	f5 = tex2D(texRef_f5C ,xcoord,ycoord);
//	f6 = tex2D(texRef_f6C ,xcoord,ycoord);
//	f7 = tex2D(texRef_f7C ,xcoord,ycoord);
//	f8 = tex2D(texRef_f8C ,xcoord,ycoord);
//
//	float rho,u,v;	
//	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
//	u = f1-f3+f5-f6-f7+f8;
//	v = f2-f4+f5+f6-f7-f8;
//	float usqr = u*u+v*v;
//	//scale
////	f0 =SF*f0 +(1.0f-SF)*(0.4444444444f*rho-0.6666666667f*usqr);
////	f1 =SF*f1 +(1.0f-SF)*(0.1111111111f*rho+0.3333333333f*u+0.5f*u*u-0.1666666667f*usqr);                 
////	f2 =SF*f2 +(1.0f-SF)*(0.1111111111f*rho+0.3333333333f*v+0.5f*v*v-0.1666666667f*usqr);
////	f3 =SF*f3 +(1.0f-SF)*(0.1111111111f*rho-0.3333333333f*u+0.5f*u*u-0.1666666667f*usqr);
////	f4 =SF*f4 +(1.0f-SF)*(0.1111111111f*rho-0.3333333333f*v+0.5f*v*v-0.1666666667f*usqr);
////	f5 =SF*f5 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*( u+v)+0.125f*( u+v)*( u+v)-0.04166666667f*usqr);
////	f6 =SF*f6 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*(-u+v)+0.125f*(-u+v)*(-u+v)-0.04166666667f*usqr);
////	f7 =SF*f7 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*(-u-v)+0.125f*(-u-v)*(-u-v)-0.04166666667f*usqr);
////	f8 =SF*f8 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*( u-v)+0.125f*( u-v)*( u-v)-0.04166666667f*usqr);
//
//	fout[f_mem(0 ,x,y,pitch)] =SF*f0 +(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	fout[f_mem(1 ,x,y,pitch)] =SF*f1 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	fout[f_mem(2 ,x,y,pitch)] =SF*f2 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	fout[f_mem(3 ,x,y,pitch)] =SF*f3 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	fout[f_mem(4 ,x,y,pitch)] =SF*f4 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	fout[f_mem(5 ,x,y,pitch)] =SF*f5 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	fout[f_mem(6 ,x,y,pitch)] =SF*f6 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	fout[f_mem(7 ,x,y,pitch)] =SF*f7 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	fout[f_mem(8 ,x,y,pitch)] =SF*f8 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
//	
////	fout[f_mem(0 ,x,y,pitch)] = f0 ;
////	fout[f_mem(1 ,x,y,pitch)] = f1 ;
////	fout[f_mem(2 ,x,y,pitch)] = f2 ;
////	fout[f_mem(3 ,x,y,pitch)] = f3 ;
////	fout[f_mem(4 ,x,y,pitch)] = f4 ;
////	fout[f_mem(5 ,x,y,pitch)] = f5 ;
////	fout[f_mem(6 ,x,y,pitch)] = f6 ;
////	fout[f_mem(7 ,x,y,pitch)] = f7 ;
////	fout[f_mem(8 ,x,y,pitch)] = f8 ;
	}
}

__global__ void LR_d_ABDC2(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

//	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n)
//	{
//	//no interp
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];

	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC_Interp(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL)
	//if(x < 2 || x > XLRDIM-1-2 || y < 2 || y > YLRDIM-1-2)
	{
	//interpolate for next time step. from B
	f0 = tex2D(texRef_f0B,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2B,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3B,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4B,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5B,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6B,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7B,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8B,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];

	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC_Interp2(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL)
	//if(x < 2 || x > XLRDIM-1-2 || y < 2 || y > YLRDIM-1-2)
	{
    float u_00,u_01,u_10,u_11;
    float v_00,v_01,v_10,v_11;
    float m0_00,m0_01,m0_10,m0_11;
    float m1_00,m1_01,m1_10,m1_11;
    float m2_00,m2_01,m2_10,m2_11;
    float m4_00,m4_01,m4_10,m4_11;
    float m6_00,m6_01,m6_10,m6_11;
    float m7_00,m7_01,m7_10,m7_11;
    float m8_00,m8_01,m8_10,m8_11;
    float Sxx_00,Sxx_01,Sxx_10,Sxx_11;
    float Syy_00,Syy_01,Syy_10,Syy_11;
    float Sxy_00,Sxy_01,Sxy_10,Sxy_11;
    
    //moments for 00
	f0 = tex2D(texRef_f0B,int(xcoord)+0.5f,int(ycoord)+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f2 = tex2D(texRef_f2B,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f3 = tex2D(texRef_f3B,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f4 = tex2D(texRef_f4B,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f5 = tex2D(texRef_f5B,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f6 = tex2D(texRef_f6B,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f7 = tex2D(texRef_f7B,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f8 = tex2D(texRef_f8B,int(xcoord)+0.5f,int(ycoord)+0.5f);

	m0_00 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_00 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_00 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_00 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_00 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_00 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_00 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_00  = f1-f3+f5-f6-f7+f8;
	v_00  = f2-f4+f5+f6-f7-f8;
    Sxx_00=omega*0.75f*( u_00*u_00-v_00*v_00-m7_00);
    Syy_00=omega*0.75f*(-u_00*u_00+v_00*v_00+m7_00);
    Sxy_00=omega*1.5f*(u_00*v_00-m8_00);

    //moments for 10
	f0 = tex2D(texRef_f0B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f2 = tex2D(texRef_f2B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f3 = tex2D(texRef_f3B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f4 = tex2D(texRef_f4B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f5 = tex2D(texRef_f5B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f6 = tex2D(texRef_f6B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f7 = tex2D(texRef_f7B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f8 = tex2D(texRef_f8B,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	m0_10 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_10 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_10 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_10 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_10 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_10 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_10 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_10  = f1-f3+f5-f6-f7+f8;
	v_10  = f2-f4+f5+f6-f7-f8;
    Sxx_10=omega*0.75f*(u_10*u_10-v_10*v_10-m7_10);
    Syy_10=omega*0.75f*(-u_10*u_10+v_10*v_10+m7_10);
    Sxy_10=omega*1.5f*(u_10*v_10-m8_10);

    //moments for 01
	f0 = tex2D(texRef_f0B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f2 = tex2D(texRef_f2B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f3 = tex2D(texRef_f3B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f4 = tex2D(texRef_f4B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f5 = tex2D(texRef_f5B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f6 = tex2D(texRef_f6B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f7 = tex2D(texRef_f7B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f8 = tex2D(texRef_f8B,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	m0_01 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_01 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_01 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_01 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_01 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_01 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_01 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_01  = f1-f3+f5-f6-f7+f8;
	v_01  = f2-f4+f5+f6-f7-f8;
    Sxx_01=omega*0.75f*(u_01*u_01-v_01*v_01-m7_01);
    Syy_01=omega*0.75f*(-u_01*u_01+v_01*v_01+m7_01);
    Sxy_01=omega*1.5f*(u_01*v_01-m8_01);

    //moments for 11
	f0 = tex2D(texRef_f0B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f2 = tex2D(texRef_f2B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f3 = tex2D(texRef_f3B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f4 = tex2D(texRef_f4B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f5 = tex2D(texRef_f5B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f6 = tex2D(texRef_f6B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f7 = tex2D(texRef_f7B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f8 = tex2D(texRef_f8B,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	m0_11 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_11 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_11 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_11 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_11 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_11 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_11 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
 	u_11  = f1-f3+f5-f6-f7+f8;
	v_11  = f2-f4+f5+f6-f7-f8;
    Sxx_11=omega*0.75f*(u_11*u_11-v_11*v_11-m7_11);
    Syy_11=omega*0.75f*(-u_11*u_11+v_11*v_11+m7_11);
    Sxy_11=omega*1.5f*(u_11*v_11-m8_11);

    //this needs to be fixed. needs to be bilinear interp. missing weighting 
    m0_00 = (m0_00*(ycoord-int(ycoord))+m0_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m0_10*(ycoord-int(ycoord))+m0_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m1_00 = (m1_00*(ycoord-int(ycoord))+m1_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m1_10*(ycoord-int(ycoord))+m1_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m2_00 = (m2_00*(ycoord-int(ycoord))+m2_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m2_10*(ycoord-int(ycoord))+m2_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m4_00 = (m4_00*(ycoord-int(ycoord))+m4_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m4_10*(ycoord-int(ycoord))+m4_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m6_00 = (m6_00*(ycoord-int(ycoord))+m6_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m6_10*(ycoord-int(ycoord))+m6_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m7_00 = (m7_00*(ycoord-int(ycoord))+m7_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m7_10*(ycoord-int(ycoord))+m7_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m8_00 = (m8_00*(ycoord-int(ycoord))+m8_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m8_10*(ycoord-int(ycoord))+m8_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
//    m0_00 = 0.25f*(m0_00+m0_01+m0_10+m0_11);
//    m1_00 = 0.25f*(m1_00+m1_01+m1_10+m1_11);
//    m2_00 = 0.25f*(m2_00+m2_01+m2_10+m2_11);
//    m4_00 = 0.25f*(m4_00+m4_01+m4_10+m4_11);
//    m6_00 = 0.25f*(m6_00+m6_01+m6_10+m6_11);
//    m7_00 = 0.25f*(m7_00+m7_01+m7_10+m7_11);
//    m8_00 = 0.25f*(m8_00+m8_01+m8_10+m8_11);


    float cx,cy,dx,dy;
    cx = 0.f;//0.0625f*(Sxx_00+Sxx_01-Sxx_10-Sxx_11);
    dy = 0.f;//0.0625f*(Syy_00-Syy_01+Syy_10-Syy_11);
    cy = 0.f;//0.125f*(Sxy_00+Sxy_10-Sxy_01-Sxy_11)+0.125f*(v_00+v_11-v_01-v_10);
    dx = 0.f;//0.125f*(Sxy_00+Sxy_01-Sxy_10-Sxy_11)+0.125f*(u_00+u_11-u_01-u_10);

    float a0,a1,a2,a3,b0,b1,b2,b3;
    float xp,yp;
    xp = 2.f*(xcoord-int(xcoord))-1.f;
    yp = 2.f*(ycoord-int(ycoord))-1.f;

    a0 =0.25f*( u_00+u_10+u_11+u_01);
    a1 =0.25f*(-u_00+u_10+u_11-u_01);
    a2 =0.25f*(-u_00-u_10+u_11+u_01);
    a3 =0.25f*( u_00-u_10+u_11-u_01);
    b0 =0.25f*( v_00+v_10+v_11+v_01);
    b1 =0.25f*(-v_00+v_10+v_11-v_01);
    b2 =0.25f*(-v_00-v_10+v_11+v_01);
    b3 =0.25f*( v_00-v_10+v_11-v_01);

	float rho,u,v;	
    u = a0+a1*xp+a2*yp+a3*xp*yp+cx*(1.f-xp*xp)+cy*(1.f-yp*yp);
    v = b0+b1*xp+b2*yp+b3*xp*yp+dx*(1.f-xp*xp)+dy*(1.f-yp*yp);
    rho = m0_00;

    m1_00 = m1_00*SF+(1.f-SF)*(-2.0f*rho+3.0f*(u*u+v*v));
    m2_00 = m2_00*SF+(1.f-SF)*(rho-3.0f*(u*u+v*v));
    m4_00 = m4_00*SF+(1.f-SF)*(-u);
    m6_00 = m6_00*SF+(1.f-SF)*(-v);
    m7_00 = m7_00*SF+(1.f-SF)*(u*u-v*v);
    m8_00 = m8_00*SF+(1.f-SF)*(u*v);

    //use rho u v and m's and Minv to get f's. can rescale during this step
f0 = 1.f/9.f*m0_00+-4.f/36.f*m1_00+ 4.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 0.f/4.f*m7_00+ 0.f/4.f*m8_00;
f1 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 1.f/6.f*u+-2.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f2 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+ 1.f/6.f*v+-2.f/12.f*m6_00+-1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f3 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+-1.f/6.f*u+ 2.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f4 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+-1.f/6.f*v+ 2.f/12.f*m6_00+-1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f5 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+ 1.f/6.f*u+ 1.f/12.f*m4_00+ 1.f/6.f*v+ 1.f/12.f*m6_00+ 0.f/4.f*m7_00+ 1.f/4.f*m8_00;
f6 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+-1.f/6.f*u+-1.f/12.f*m4_00+ 1.f/6.f*v+ 1.f/12.f*m6_00+ 0.f/4.f*m7_00+-1.f/4.f*m8_00;
f7 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+-1.f/6.f*u+-1.f/12.f*m4_00+-1.f/6.f*v+-1.f/12.f*m6_00+ 0.f/4.f*m7_00+ 1.f/4.f*m8_00;
f8 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+ 1.f/6.f*u+ 1.f/12.f*m4_00+-1.f/6.f*v+-1.f/12.f*m6_00+ 0.f/4.f*m7_00+-1.f/4.f*m8_00;


	fout[f_memLR(0,x,y,pitch)] = f0;
	fout[f_memLR(1,x,y,pitch)] = f1;
	fout[f_memLR(2,x,y,pitch)] = f2;
	fout[f_memLR(3,x,y,pitch)] = f3;
	fout[f_memLR(4,x,y,pitch)] = f4;
	fout[f_memLR(5,x,y,pitch)] = f5;
	fout[f_memLR(6,x,y,pitch)] = f6;
	fout[f_memLR(7,x,y,pitch)] = f7;
	fout[f_memLR(8,x,y,pitch)] = f8;

//	float rho,u,v;	
//	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
//	u = f1-f3+f5-f6-f7+f8;
//	v = f2-f4+f5+f6-f7-f8;
//	float usqr = u*u+v*v;
//	//scale
//	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];

	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BACD2(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
//second time through in inner loop. don't need to calculate for the 3 outer most cells
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
//	}//end else (not at edge of LR)
}
__global__ void LR_d_BADC2(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

//	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n)
//	{
//	//no interp
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
//	}//end else (not at edge of LR)
}
__global__ void LR_d_ABCD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_ABCD_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();
//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		//check = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;

    	syncthreads();
    	if(check[0] == 1 && t>=STARTF){
    	//reduction for force
    	int nTotalThreads = blockDim.x;
    	while(nTotalThreads > 1){
    		int halfPoint = (nTotalThreads >> 1);
    		if(threadIdx.x < halfPoint){
    			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
    			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
    		}
    		syncthreads();
    		nTotalThreads = halfPoint;
    	}
    	if(threadIdx.x == 0){
    		atomicAdd(&FX[t],sumX[0]);
    		atomicAdd(&FY[t],sumY[0]);
    	}
    	}
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_ABCD2(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if(x < 3 || x > XLRDIM-4 || y < 3 || y > YLRDIM-4)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//interpolate for next time step. from B
	f0 = tex2D(texRef_f0B,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2B,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3B,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4B,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5B,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6B,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7B,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8B,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

//	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
//	fout[f_memLR(0,x,y,pitch)] = f0 ;
//	fout[f_memLR(1,x,y,pitch)] = f1 ;
//	fout[f_memLR(2,x,y,pitch)] = f2 ;
//	fout[f_memLR(3,x,y,pitch)] = f3 ;
//	fout[f_memLR(4,x,y,pitch)] = f4 ;
//	fout[f_memLR(5,x,y,pitch)] = f5 ;
//	fout[f_memLR(6,x,y,pitch)] = f6 ;
//	fout[f_memLR(7,x,y,pitch)] = f7 ;
//	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fin[f_memLR(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fin[f_memLR(2 ,x             ,y-1           ,pitch)];
	f5 = fin[f_memLR(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fin[f_memLR(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fin[f_memLR(4 ,x             ,y+1           ,pitch)];
	f7 = fin[f_memLR(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fin[f_memLR(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BACD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_BACD_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();
//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		//check = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;

 	    syncthreads();
    	if(check[0] == 1 && t>=STARTF){
    	//reduction for force
    	int nTotalThreads = blockDim.x;
    	while(nTotalThreads > 1){
    		int halfPoint = (nTotalThreads >> 1);
    		if(threadIdx.x < halfPoint){
    			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
    			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
    		}
    		syncthreads();
    		nTotalThreads = halfPoint;
    	}
    	if(threadIdx.x == 0){
    		atomicAdd(&FX[t],sumX[0]);
    		atomicAdd(&FY[t],sumY[0]);
    	}
    	}
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_BADC_Interp(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

	//if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	if(x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL)
	{
	//interpolate for next time step. from A
	f0 = tex2D(texRef_f0A,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1A,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2A,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3A,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4A,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5A,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6A,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7A,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8A,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

//	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
//	fout[f_memLR(0,x,y,pitch)] = f0 ;
//	fout[f_memLR(1,x,y,pitch)] = f1 ;
//	fout[f_memLR(2,x,y,pitch)] = f2 ;
//	fout[f_memLR(3,x,y,pitch)] = f3 ;
//	fout[f_memLR(4,x,y,pitch)] = f4 ;
//	fout[f_memLR(5,x,y,pitch)] = f5 ;
//	fout[f_memLR(6,x,y,pitch)] = f6 ;
//	fout[f_memLR(7,x,y,pitch)] = f7 ;
//	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BADC_Interp2(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

	//if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	if(x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL)
	{
    float u_00,u_01,u_10,u_11;
    float v_00,v_01,v_10,v_11;
    float m0_00,m0_01,m0_10,m0_11;
    float m1_00,m1_01,m1_10,m1_11;
    float m2_00,m2_01,m2_10,m2_11;
    float m4_00,m4_01,m4_10,m4_11;
    float m6_00,m6_01,m6_10,m6_11;
    float m7_00,m7_01,m7_10,m7_11;
    float m8_00,m8_01,m8_10,m8_11;
    float Sxx_00,Sxx_01,Sxx_10,Sxx_11;
    float Syy_00,Syy_01,Syy_10,Syy_11;
    float Sxy_00,Sxy_01,Sxy_10,Sxy_11;
    
    //moments for 00
	f0 = tex2D(texRef_f0A,int(xcoord)+0.5f,int(ycoord)+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1A,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f2 = tex2D(texRef_f2A,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f3 = tex2D(texRef_f3A,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f4 = tex2D(texRef_f4A,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f5 = tex2D(texRef_f5A,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f6 = tex2D(texRef_f6A,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f7 = tex2D(texRef_f7A,int(xcoord)+0.5f,int(ycoord)+0.5f);
	f8 = tex2D(texRef_f8A,int(xcoord)+0.5f,int(ycoord)+0.5f);

	m0_00 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_00 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_00 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_00 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_00 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_00 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_00 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_00  = f1-f3+f5-f6-f7+f8;
	v_00  = f2-f4+f5+f6-f7-f8;
    Sxx_00=omega*0.75f*( u_00*u_00-v_00*v_00-m7_00);
    Syy_00=omega*0.75f*(-u_00*u_00+v_00*v_00+m7_00);
    Sxy_00=omega*1.5f*(u_00*v_00-m8_00);

    //moments for 10
	f0 = tex2D(texRef_f0A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f2 = tex2D(texRef_f2A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f3 = tex2D(texRef_f3A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f4 = tex2D(texRef_f4A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f5 = tex2D(texRef_f5A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f6 = tex2D(texRef_f6A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f7 = tex2D(texRef_f7A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	f8 = tex2D(texRef_f8A,int(xcoord)+0.5f+1,int(ycoord)+0.5f);
	m0_10 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_10 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_10 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_10 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_10 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_10 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_10 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_10  = f1-f3+f5-f6-f7+f8;
	v_10  = f2-f4+f5+f6-f7-f8;
    Sxx_10=omega*0.75f*(u_10*u_10-v_10*v_10-m7_10);
    Syy_10=omega*0.75f*(-u_10*u_10+v_10*v_10+m7_10);
    Sxy_10=omega*1.5f*(u_10*v_10-m8_10);

    //moments for 01
	f0 = tex2D(texRef_f0A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f2 = tex2D(texRef_f2A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f3 = tex2D(texRef_f3A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f4 = tex2D(texRef_f4A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f5 = tex2D(texRef_f5A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f6 = tex2D(texRef_f6A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f7 = tex2D(texRef_f7A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	f8 = tex2D(texRef_f8A,int(xcoord)+0.5f,int(ycoord)+0.5f+1);
	m0_01 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_01 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_01 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_01 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_01 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_01 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_01 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
	u_01  = f1-f3+f5-f6-f7+f8;
	v_01  = f2-f4+f5+f6-f7-f8;
    Sxx_01=omega*0.75f*(u_01*u_01-v_01*v_01-m7_01);
    Syy_01=omega*0.75f*(-u_01*u_01+v_01*v_01+m7_01);
    Sxy_01=omega*1.5f*(u_01*v_01-m8_01);

    //moments for 11
	f0 = tex2D(texRef_f0A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f2 = tex2D(texRef_f2A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f3 = tex2D(texRef_f3A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f4 = tex2D(texRef_f4A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f5 = tex2D(texRef_f5A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f6 = tex2D(texRef_f6A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f7 = tex2D(texRef_f7A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	f8 = tex2D(texRef_f8A,int(xcoord)+0.5f+1,int(ycoord)+0.5f+1);
	m0_11 = f0+f1+f2+f3+f4+f5+f6+f7+f8;
    m1_11 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8;
	m2_11 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8; //ep
	m4_11 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8;//qx_eq
	m6_11 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8;//qy_eq
	m7_11 =             f1 -    f2+     f3 -    f4                                ;//pxx_eq
	m8_11 =                                             f5 -    f6+     f7 -    f8;//pxy_eq
 	u_11  = f1-f3+f5-f6-f7+f8;
	v_11  = f2-f4+f5+f6-f7-f8;
    Sxx_11=omega*0.75f*(u_11*u_11-v_11*v_11-m7_11);
    Syy_11=omega*0.75f*(-u_11*u_11+v_11*v_11+m7_11);
    Sxy_11=omega*1.5f*(u_11*v_11-m8_11);

    m0_00 = (m0_00*(ycoord-int(ycoord))+m0_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m0_10*(ycoord-int(ycoord))+m0_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m1_00 = (m1_00*(ycoord-int(ycoord))+m1_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m1_10*(ycoord-int(ycoord))+m1_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m2_00 = (m2_00*(ycoord-int(ycoord))+m2_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m2_10*(ycoord-int(ycoord))+m2_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m4_00 = (m4_00*(ycoord-int(ycoord))+m4_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m4_10*(ycoord-int(ycoord))+m4_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m6_00 = (m6_00*(ycoord-int(ycoord))+m6_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m6_10*(ycoord-int(ycoord))+m6_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m7_00 = (m7_00*(ycoord-int(ycoord))+m7_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m7_10*(ycoord-int(ycoord))+m7_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);
    m8_00 = (m8_00*(ycoord-int(ycoord))+m8_01*(1.f+int(ycoord)-ycoord))*(xcoord-int(xcoord))+(m8_10*(ycoord-int(ycoord))+m8_11*(1.f+int(ycoord)-ycoord))*(1.f+int(xcoord)-xcoord);

    float cx,cy,dx,dy;
    cx = 0.f;//0.0625f*(Sxx_00+Sxx_01-Sxx_10-Sxx_11);
    dy = 0.f;//0.0625f*(Syy_00-Syy_01+Syy_10-Syy_11);
    cy = 0.f;//0.125f*(Sxy_00+Sxy_10-Sxy_01-Sxy_11)+0.125f*(v_00+v_11-v_01-v_10);
    dx = 0.f;//0.125f*(Sxy_00+Sxy_01-Sxy_10-Sxy_11)+0.125f*(u_00+u_11-u_01-u_10);

    float a0,a1,a2,a3,b0,b1,b2,b3;
    float xp,yp;
    xp = 2.f*(xcoord-int(xcoord))-1.f;
    yp = 2.f*(ycoord-int(ycoord))-1.f;

    a0 =0.25f*( u_00+u_10+u_11+u_01);
    a1 =0.25f*(-u_00+u_10+u_11-u_01);
    a2 =0.25f*(-u_00-u_10+u_11+u_01);
    a3 =0.25f*( u_00-u_10+u_11-u_01);
    b0 =0.25f*( v_00+v_10+v_11+v_01);
    b1 =0.25f*(-v_00+v_10+v_11-v_01);
    b2 =0.25f*(-v_00-v_10+v_11+v_01);
    b3 =0.25f*( v_00-v_10+v_11-v_01);

	float rho,u,v;	
    u = a0+a1*xp+a2*yp+a3*xp*yp+cx*(1.f-xp*xp)+cy*(1.f-yp*yp);
    v = b0+b1*xp+b2*yp+b3*xp*yp+dx*(1.f-xp*xp)+dy*(1.f-yp*yp);
    rho = m0_00;

    m1_00 = m1_00*SF+(1.f-SF)*(-2.0f*rho+3.0f*(u*u+v*v));
    m2_00 = m2_00*SF+(1.f-SF)*(rho-3.0f*(u*u+v*v));
    m4_00 = m4_00*SF+(1.f-SF)*(-u);
    m6_00 = m6_00*SF+(1.f-SF)*(-v);
    m7_00 = m7_00*SF+(1.f-SF)*(u*u-v*v);
    m8_00 = m8_00*SF+(1.f-SF)*(u*v);

    //use rho u v and m's and Minv to get f's. can rescale during this step
f0 = 1.f/9.f*m0_00+-4.f/36.f*m1_00+ 4.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 0.f/4.f*m7_00+ 0.f/4.f*m8_00;
f1 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 1.f/6.f*u+-2.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f2 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+ 1.f/6.f*v+-2.f/12.f*m6_00+-1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f3 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+-1.f/6.f*u+ 2.f/12.f*m4_00+ 0.f/6.f*v+ 0.f/12.f*m6_00+ 1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f4 = 1.f/9.f*m0_00+-1.f/36.f*m1_00+-2.f/36.f*m2_00+ 0.f/6.f*u+ 0.f/12.f*m4_00+-1.f/6.f*v+ 2.f/12.f*m6_00+-1.f/4.f*m7_00+ 0.f/4.f*m8_00;
f5 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+ 1.f/6.f*u+ 1.f/12.f*m4_00+ 1.f/6.f*v+ 1.f/12.f*m6_00+ 0.f/4.f*m7_00+ 1.f/4.f*m8_00;
f6 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+-1.f/6.f*u+-1.f/12.f*m4_00+ 1.f/6.f*v+ 1.f/12.f*m6_00+ 0.f/4.f*m7_00+-1.f/4.f*m8_00;
f7 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+-1.f/6.f*u+-1.f/12.f*m4_00+-1.f/6.f*v+-1.f/12.f*m6_00+ 0.f/4.f*m7_00+ 1.f/4.f*m8_00;
f8 = 1.f/9.f*m0_00+ 2.f/36.f*m1_00+ 1.f/36.f*m2_00+ 1.f/6.f*u+ 1.f/12.f*m4_00+-1.f/6.f*v+-1.f/12.f*m6_00+ 0.f/4.f*m7_00+-1.f/4.f*m8_00;


	fout[f_memLR(0,x,y,pitch)] = f0;
	fout[f_memLR(1,x,y,pitch)] = f1;
	fout[f_memLR(2,x,y,pitch)] = f2;
	fout[f_memLR(3,x,y,pitch)] = f3;
	fout[f_memLR(4,x,y,pitch)] = f4;
	fout[f_memLR(5,x,y,pitch)] = f5;
	fout[f_memLR(6,x,y,pitch)] = f6;
	fout[f_memLR(7,x,y,pitch)] = f7;
	fout[f_memLR(8,x,y,pitch)] = f8;





//	//interpolate for next time step. from A
//	f0 = tex2D(texRef_f0A,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
//	f1 = tex2D(texRef_f1A,xcoord+0.5f,ycoord+0.5f);
//	f2 = tex2D(texRef_f2A,xcoord+0.5f,ycoord+0.5f);
//	f3 = tex2D(texRef_f3A,xcoord+0.5f,ycoord+0.5f);
//	f4 = tex2D(texRef_f4A,xcoord+0.5f,ycoord+0.5f);
//	f5 = tex2D(texRef_f5A,xcoord+0.5f,ycoord+0.5f);
//	f6 = tex2D(texRef_f6A,xcoord+0.5f,ycoord+0.5f);
//	f7 = tex2D(texRef_f7A,xcoord+0.5f,ycoord+0.5f);
//	f8 = tex2D(texRef_f8A,xcoord+0.5f,ycoord+0.5f);
//
//	float rho,u,v;	
//	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
//	u = f1-f3+f5-f6-f7+f8;
//	v = f2-f4+f5+f6-f7-f8;
//	float usqr = u*u+v*v;
//	//scale
//	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

//	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
//	fout[f_memLR(0,x,y,pitch)] = f0 ;
//	fout[f_memLR(1,x,y,pitch)] = f1 ;
//	fout[f_memLR(2,x,y,pitch)] = f2 ;
//	fout[f_memLR(3,x,y,pitch)] = f3 ;
//	fout[f_memLR(4,x,y,pitch)] = f4 ;
//	fout[f_memLR(5,x,y,pitch)] = f5 ;
//	fout[f_memLR(6,x,y,pitch)] = f6 ;
//	fout[f_memLR(7,x,y,pitch)] = f7 ;
//	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void mrt_d_single_LR(float* fA, float* fB,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

    if( (x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1) && 
	(x == int(LRX0+2) || x == int(LRX0+XLRDIM*LRFACTOR-1) || y == int(LRY0+2) || y == int(LRY0+YLRDIM*LRFACTOR-1)) )
	{
	float xcoord = (x-LRX0)/LRFACTOR+0.5f;
	float ycoord = (y-LRY0)/LRFACTOR+0.5f;
	f0 = tex2D(texRef_f0C ,xcoord,ycoord);
	f1 = tex2D(texRef_f1C ,xcoord,ycoord);
	f2 = tex2D(texRef_f2C ,xcoord,ycoord);
	f3 = tex2D(texRef_f3C ,xcoord,ycoord);
	f4 = tex2D(texRef_f4C ,xcoord,ycoord);
	f5 = tex2D(texRef_f5C ,xcoord,ycoord);
	f6 = tex2D(texRef_f6C ,xcoord,ycoord);
	f7 = tex2D(texRef_f7C ,xcoord,ycoord);
	f8 = tex2D(texRef_f8C ,xcoord,ycoord);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	f0 = SF*f0 +(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	f1 = SF*f1 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	f2 = SF*f2 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 = SF*f3 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 = SF*f4 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 = SF*f5 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	f6 = SF*f6 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 = SF*f7 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 = SF*f8 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
	}





	//if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1){
//	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
//	}
//	else{
	f0 = fA[j];
	f1 = fA[f_mem(1 ,x-1,y  ,pitch)];
	f3 = fA[f_mem(3 ,x+1,y  ,pitch)];
	f2 = fA[f_mem(2 ,x  ,y-1,pitch)];
	f5 = fA[f_mem(5 ,x-1,y-1,pitch)];
	f6 = fA[f_mem(6 ,x+1,y-1,pitch)];
	f4 = fA[f_mem(4 ,x  ,y+1,pitch)];
	f7 = fA[f_mem(7 ,x+1,y+1,pitch)];
    if(y != YDIM-1)
	f8 = fA[f_mem(8 ,x-1,y+1,pitch)];
//	f0 = fA[j];
//	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
//	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
//	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
//	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
//	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
//	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
//	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
//	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		//atomicAdd();
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}
//	}

}
__global__ void mrt_d_single(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	//if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1){
//	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
//	}
//	else{
	f0 = fA[j];
	f1 = fA[f_mem(1 ,x-1,y  ,pitch)];
	f3 = fA[f_mem(3 ,x+1,y  ,pitch)];
	f2 = fA[f_mem(2 ,x  ,y-1,pitch)];
	f5 = fA[f_mem(5 ,x-1,y-1,pitch)];
	f6 = fA[f_mem(6 ,x+1,y-1,pitch)];
	f4 = fA[f_mem(4 ,x  ,y+1,pitch)];
	f7 = fA[f_mem(7 ,x+1,y+1,pitch)];
    if(y != YDIM-1)
	f8 = fA[f_mem(8 ,x-1,y+1,pitch)];
//	f0 = fA[j];
//	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
//	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
//	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
//	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
//	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
//	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
//	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
//	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		//atomicAdd();
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}
//	}

}

__global__ void mrt_d_single_force(float* fA, float* fB,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	//int check = 0; //check if block contains force calculation nodes
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();

	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
	}
	else{
	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		//check = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		}
		//atomicAdd();
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
//		if(im == 2)//NeumannEast
//		{
//			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 3)//DirichletWest
//		{
//			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}

		boundaries_force(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}

	syncthreads();
	if(check[0] == 1 && t>=STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
	}
	}
	}
}
__global__ void mrt_d_LR(float* fA, float* fB,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
			}
			float u,v;//,rho;
			u = 0.0f;
	        v = UMAX;//0.0;
			f1 = f3+u*0.66666667f;
	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;			
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}
}


__global__ void initialize_single(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	if(x<XDIM && y<YDIM){
	float u,v,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;

	f[j+0 *pitch*YDIM]= 0.4444444444f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM]= 0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);               
	f[j+2 *pitch*YDIM]= 0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM]= 0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM]= 0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM]= 0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM]= 0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM]= 0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM]= 0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
	}
}


__global__ void initialize_LR(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	if(x<XLRDIM && y<YLRDIM){
	float u,v,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;

	f[j+0 *pitch*YLRDIM]= 0.4444444444f*(rho-1.5f*usqr);
	f[j+1 *pitch*YLRDIM]= 0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);               
	f[j+2 *pitch*YLRDIM]= 0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YLRDIM]= 0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YLRDIM]= 0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
	f[j+6 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
	}
}

int main(int argc, char *argv[])
{

	//int *image_d, *image_h;

	ofstream output;
	ofstream output2;
	string FileName = CASENAME;
	output.open ((FileName+".dat").c_str());
	output2.open ((FileName+".force").c_str());

	size_t memsize, memsize2;
	size_t pitch = 0;
	size_t pitch2 = 0;
	int i, n, nBlocks, nBlocks2, n2;
	float omega, CharLength, omega2;

	if(abs(LRFACTOR-1.f/LRLEVEL)>0.001f){
		cout<<"LRLEVEL and LRFACTOR don't match! Exiting..."<<endl;
		return 0;
	}

	CharLength = OBSTR*2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));
	if(LRFACTOR == 0.25f){
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	}
	else if(LRFACTOR == 0.125f){
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	}
	else if(LRFACTOR == 0.0625){
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	}
	float SF_cf = omega*(1.0f-omega2)/((1.0f-omega)*omega2/LRFACTOR);
	float SF_fc = 1.f/SF_cf;

//	float SF_cf3 = omega*(1.0f-omega3)/((1.0f-omega)*omega3/LRFACTOR);
//	float SF_fc3 = 1.f/SF_cf3;

	cout<<"omega:  "<<omega<<endl;
	cout<<"omega2: "<<omega2<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<endl;
	cout<<"LRblocksize: "<<BLOCKSIZELRX<<"x"<<BLOCKSIZELRY<<endl;
	cout<<"LRgrid: "<<XLRDIM<<"x"<<YLRDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

	nBlocks = ((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX)*(YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY);
	nBlocks2 = (XLRDIM/BLOCKSIZELRX+XLRDIM%BLOCKSIZELRX)*(YLRDIM/BLOCKSIZELRY+YLRDIM%BLOCKSIZELRY);
	int B = BLOCKSIZEX*BLOCKSIZELRY;
	int B2 = BLOCKSIZELRX*BLOCKSIZELRY;
	n = nBlocks*B;
	n2 = nBlocks2*B2;
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
    //dim3 grid(XDIM/BLOCKSIZEX,YDIM/BLOCKSIZEY);
    dim3 grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY);

    dim3 threads2(BLOCKSIZELRX, BLOCKSIZELRY);
    //dim3 grid2(XLRDIM/BLOCKSIZELRX,YLRDIM/BLOCKSIZELRY);
    dim3 grid2(XLRDIM/BLOCKSIZELRX+XLRDIM%BLOCKSIZELRX,YLRDIM/BLOCKSIZELRY+YLRDIM%BLOCKSIZELRY);

	memsize = n*sizeof(float);
	//memsize_int = n*sizeof(int);
	memsize2 = n2*sizeof(float);

	//hipExtent extent = make_hipExtent(XDIM*sizeof(float),YDIM,ZDIM);

	//image_h = (int *)malloc(memsize_int);

	float *fA_h,*fA_d,*fB_d,*fC_h,*fC_d,*fD_d;
	float *FX_h,*FY_h,*FX_d,*FY_d;
	fA_h = (float *)malloc(memsize*9);
	fC_h = (float *)malloc(memsize2*9);
	FX_h = (float *)malloc(TMAX*sizeof(float));
	FY_h = (float *)malloc(TMAX*sizeof(float));
	hipMallocPitch((void **) &fA_d, &pitch, XDIM*sizeof(float), YDIM*9);
	hipMallocPitch((void **) &fB_d, &pitch, XDIM*sizeof(float), YDIM*9);
//	hipMalloc((void **) &fA_d, XDIM*sizeof(float)*YDIM*9);
//	hipMalloc((void **) &fB_d, XDIM*sizeof(float)*YDIM*9);
//	pitch = 16384;

	if(REFINEMENT == "YES"){
	hipMallocPitch((void **) &fC_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*9);
	hipMallocPitch((void **) &fD_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*9);
	}

	hipMalloc((void **) &FX_d, TMAX*sizeof(float));
	hipMalloc((void **) &FY_d, TMAX*sizeof(float));

	//hipMalloc((void **) &image_d, memsize_int);

	cout<<pitch<<", "<<pitch2<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);
	size_t pitch_elements2 = pitch2/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*9; i++)
		fA_h[i] = i;
	for (i = 0; i < n2*9; i++)
		fC_h[i] = 0;
	for (i = 0; i < TMAX; i++){
		FX_h[i] = 0.0f;
		FY_h[i] = 0.0f;
	}
	hipMemcpy(FX_d, FX_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FY_d, FY_h, TMAX*sizeof(float), hipMemcpyHostToDevice);

//	for (i = 0; i < n; i++)
//	{
//		int x = i%XDIM;
//		int y = (i/XDIM)%YDIM;
//		int z = (i/XDIM)/YDIM;
////		image_h[i] = 0;
////		if(x < 1) image_h[i] = 1;//DirichletWest
////		if(x > XDIM-2) image_h[i] = 1;//BB
////		if(y < 1) image_h[i] = 1;//BB
////		if(y > YDIM-2) image_h[i] = 1;//BB
////		if(z < 1) image_h[i] = 1;//DirichletWest
////		if(z > ZDIM-2) image_h[i] = 1;//BB
//	}
	//hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;

	texRef_f0B.filterMode = hipFilterModeLinear;
	texRef_f1B.filterMode = hipFilterModeLinear;
	texRef_f2B.filterMode = hipFilterModeLinear;
	texRef_f3B.filterMode = hipFilterModeLinear;
	texRef_f4B.filterMode = hipFilterModeLinear;
	texRef_f5B.filterMode = hipFilterModeLinear;
	texRef_f6B.filterMode = hipFilterModeLinear;
	texRef_f7B.filterMode = hipFilterModeLinear;
	texRef_f8B.filterMode = hipFilterModeLinear;
	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f0A.filterMode = hipFilterModeLinear;
	texRef_f1A.filterMode = hipFilterModeLinear;
	texRef_f2A.filterMode = hipFilterModeLinear;
	texRef_f3A.filterMode = hipFilterModeLinear;
	texRef_f4A.filterMode = hipFilterModeLinear;
	texRef_f5A.filterMode = hipFilterModeLinear;
	texRef_f6A.filterMode = hipFilterModeLinear;
	texRef_f7A.filterMode = hipFilterModeLinear;
	texRef_f8A.filterMode = hipFilterModeLinear;
	
	texRef_f0C.normalized = false;
	texRef_f1C.normalized = false;
	texRef_f2C.normalized = false;
	texRef_f3C.normalized = false;
	texRef_f4C.normalized = false;
	texRef_f5C.normalized = false;
	texRef_f6C.normalized = false;
	texRef_f7C.normalized = false;
	texRef_f8C.normalized = false;

	texRef_f0C.filterMode = hipFilterModeLinear;
	texRef_f1C.filterMode = hipFilterModeLinear;
	texRef_f2C.filterMode = hipFilterModeLinear;
	texRef_f3C.filterMode = hipFilterModeLinear;
	texRef_f4C.filterMode = hipFilterModeLinear;
	texRef_f5C.filterMode = hipFilterModeLinear;
	texRef_f6C.filterMode = hipFilterModeLinear;
	texRef_f7C.filterMode = hipFilterModeLinear;
	texRef_f8C.filterMode = hipFilterModeLinear;

	texRef_f0D.normalized = false;
	texRef_f1D.normalized = false;
	texRef_f2D.normalized = false;
	texRef_f3D.normalized = false;
	texRef_f4D.normalized = false;
	texRef_f5D.normalized = false;
	texRef_f6D.normalized = false;
	texRef_f7D.normalized = false;
	texRef_f8D.normalized = false;

	texRef_f0D.filterMode = hipFilterModeLinear;
	texRef_f1D.filterMode = hipFilterModeLinear;
	texRef_f2D.filterMode = hipFilterModeLinear;
	texRef_f3D.filterMode = hipFilterModeLinear;
	texRef_f4D.filterMode = hipFilterModeLinear;
	texRef_f5D.filterMode = hipFilterModeLinear;
	texRef_f6D.filterMode = hipFilterModeLinear;
	texRef_f7D.filterMode = hipFilterModeLinear;
	texRef_f8D.filterMode = hipFilterModeLinear;

	for(int i = 0; i<2; i++){
	texRef_f0A.addressMode[i]  = hipAddressModeClamp;
	texRef_f1A.addressMode[i]  = hipAddressModeClamp;
	texRef_f2A.addressMode[i]  = hipAddressModeClamp;
	texRef_f3A.addressMode[i]  = hipAddressModeClamp;
	texRef_f4A.addressMode[i]  = hipAddressModeClamp;
	texRef_f5A.addressMode[i]  = hipAddressModeClamp;
	texRef_f6A.addressMode[i]  = hipAddressModeClamp;
	texRef_f7A.addressMode[i]  = hipAddressModeClamp;
	texRef_f8A.addressMode[i]  = hipAddressModeClamp;

	texRef_f0B.addressMode[i]  = hipAddressModeClamp;
	texRef_f1B.addressMode[i]  = hipAddressModeClamp;
	texRef_f2B.addressMode[i]  = hipAddressModeClamp;
	texRef_f3B.addressMode[i]  = hipAddressModeClamp;
	texRef_f4B.addressMode[i]  = hipAddressModeClamp;
	texRef_f5B.addressMode[i]  = hipAddressModeClamp;
	texRef_f6B.addressMode[i]  = hipAddressModeClamp;
	texRef_f7B.addressMode[i]  = hipAddressModeClamp;
	texRef_f8B.addressMode[i]  = hipAddressModeClamp;
	}
	}
	
	hipMemcpy2D(fA_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*9,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*9,hipMemcpyHostToDevice);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyHostToDevice);
	hipMemcpy2D(fD_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyHostToDevice);
	}
	for (i = 0; i < n*9; i++)
		fA_h[i] = 0;
	for (i = 0; i < n2*9; i++)
		fC_h[i] = 0;


	if(true)//bind texture
	{
	hipBindTexture2D(0,&texRef_f0A, fA_d                       ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f1A, fA_d+pitch_elements*YDIM   ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f2A, fA_d+pitch_elements*YDIM*2 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f3A, fA_d+pitch_elements*YDIM*3 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f4A, fA_d+pitch_elements*YDIM*4 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f5A, fA_d+pitch_elements*YDIM*5 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f6A, fA_d+pitch_elements*YDIM*6 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f7A, fA_d+pitch_elements*YDIM*7 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f8A, fA_d+pitch_elements*YDIM*8 ,&desc,XDIM,YDIM,pitch);

	hipBindTexture2D(0,&texRef_f0B, fB_d                       ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f1B, fB_d+pitch_elements*YDIM   ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f2B, fB_d+pitch_elements*YDIM*2 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f3B, fB_d+pitch_elements*YDIM*3 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f4B, fB_d+pitch_elements*YDIM*4 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f5B, fB_d+pitch_elements*YDIM*5 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f6B, fB_d+pitch_elements*YDIM*6 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f7B, fB_d+pitch_elements*YDIM*7 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f8B, fB_d+pitch_elements*YDIM*8 ,&desc,XDIM,YDIM,pitch);

	hipBindTexture2D(0,&texRef_f0C, fC_d                          ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1C, fC_d+pitch_elements2*YLRDIM   ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2C, fC_d+pitch_elements2*YLRDIM*2 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3C, fC_d+pitch_elements2*YLRDIM*3 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4C, fC_d+pitch_elements2*YLRDIM*4 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5C, fC_d+pitch_elements2*YLRDIM*5 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6C, fC_d+pitch_elements2*YLRDIM*6 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7C, fC_d+pitch_elements2*YLRDIM*7 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8C, fC_d+pitch_elements2*YLRDIM*8 ,&desc,XLRDIM,YLRDIM,pitch2);

	hipBindTexture2D(0,&texRef_f0D, fD_d                          ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1D, fD_d+pitch_elements2*YLRDIM   ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2D, fD_d+pitch_elements2*YLRDIM*2 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3D, fD_d+pitch_elements2*YLRDIM*3 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4D, fD_d+pitch_elements2*YLRDIM*4 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5D, fD_d+pitch_elements2*YLRDIM*5 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6D, fD_d+pitch_elements2*YLRDIM*6 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7D, fD_d+pitch_elements2*YLRDIM*7 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8D, fD_d+pitch_elements2*YLRDIM*8 ,&desc,XLRDIM,YLRDIM,pitch2);
	}
	


	initialize_single<<<grid, threads>>>(fA_d,pitch_elements);
	initialize_single<<<grid, threads>>>(fB_d,pitch_elements);

	if(REFINEMENT == "YES"){
	initialize_LR<<<grid2, threads2>>>(fC_d,pitch_elements2);
	}

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(mrt_d_single),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ExtractFromC_d),hipFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<TMAX; t=t+2){
		if(REFINEMENT == "YES"){
			if(METHOD == "SINGLE")
			mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			
			if(METHOD == "SINGLE"){
				if(LRFACTOR == 0.5f){
				//LR_d_ABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
				LR_d_ABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t);
				LR_d_ABDC_Interp2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.25f){
				//LR_d_ABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
				LR_d_ABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t);
				LR_d_ABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				LR_d_ABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
				LR_d_ABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
			}

			
			ExtractFromC_d<<<grid, threads>>>(fB_d,fC_d,pitch_elements,pitch_elements2,SF_fc,omega2);

	
			if(METHOD == "SINGLE")
			mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);

			if(METHOD == "SINGLE"){
				if(LRFACTOR == 0.5f){
			    //LR_d_BACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
			    LR_d_BACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t+1);
			    LR_d_BADC_Interp2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.25f){
				//LR_d_BACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
			    LR_d_BACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t+1);
				LR_d_BADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				LR_d_BACD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
				LR_d_BADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
			}
			
			ExtractFromC_d<<<grid, threads>>>(fA_d,fC_d,pitch_elements,pitch_elements2,SF_fc,omega2);
		}
		else{
			if(METHOD == "SINGLE"){
			mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
			}
	
			else if(METHOD == "SINGLEF"){
			mrt_d_single_force<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,FX_d,FY_d,t);
			mrt_d_single_force<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,FX_d,FY_d,t+1);
			}
		}

//		else if(METHOD == "SHARED"){
//		mrt_d_shared<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//		mrt_d_shared<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//		}

//		simple_copy<<<grid, threads>>>(fA_d,fB_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);
//		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);

		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	int Nodes;
	if(REFINEMENT == "YES"){
		Nodes = (XDIM*YDIM+XLRDIM*YLRDIM/LRFACTOR);
	}
	else{
		Nodes = XDIM*YDIM;
	}
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
//			<<double((XDIM*YDIM*ZDIM)*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<XDIM<<","<<YDIM<<","<<TMAX<<","<<restime<<endl;

	if(true){
	hipUnbindTexture(texRef_f0A);
	hipUnbindTexture(texRef_f1A);
	hipUnbindTexture(texRef_f2A);
	hipUnbindTexture(texRef_f3A);
	hipUnbindTexture(texRef_f4A);
	hipUnbindTexture(texRef_f5A);
	hipUnbindTexture(texRef_f6A);
	hipUnbindTexture(texRef_f7A);
	hipUnbindTexture(texRef_f8A);

	hipUnbindTexture(texRef_f0B);
	hipUnbindTexture(texRef_f1B);
	hipUnbindTexture(texRef_f2B);
	hipUnbindTexture(texRef_f3B);
	hipUnbindTexture(texRef_f4B);
	hipUnbindTexture(texRef_f5B);
	hipUnbindTexture(texRef_f6B);
	hipUnbindTexture(texRef_f7B);
	hipUnbindTexture(texRef_f8B);

	hipUnbindTexture(texRef_f0C);
	hipUnbindTexture(texRef_f1C);
	hipUnbindTexture(texRef_f2C);
	hipUnbindTexture(texRef_f3C);
	hipUnbindTexture(texRef_f4C);
	hipUnbindTexture(texRef_f5C);
	hipUnbindTexture(texRef_f6C);
	hipUnbindTexture(texRef_f7C);
	hipUnbindTexture(texRef_f8C);

	hipUnbindTexture(texRef_f0D);
	hipUnbindTexture(texRef_f1D);
	hipUnbindTexture(texRef_f2D);
	hipUnbindTexture(texRef_f3D);
	hipUnbindTexture(texRef_f4D);
	hipUnbindTexture(texRef_f5D);
	hipUnbindTexture(texRef_f6D);
	hipUnbindTexture(texRef_f7D);
	hipUnbindTexture(texRef_f8D);
	}

	hipMemcpy2D(fA_h,XDIM*sizeof(float),fA_d,pitch,XDIM*sizeof(float),YDIM*9,hipMemcpyDeviceToHost);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_h,XLRDIM*sizeof(float),fC_d,pitch2,XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyDeviceToHost);
	}	
	hipMemcpy(FX_h, FX_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FY_h, FY_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);

	output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<"\n";
	
	int row = 0;
	int col = 0;
	i = 0;
	float rho, u, v;
	int j;

	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = row*XDIM+col;
			rho = fA_h[i];
			for(j = 1; j<9; j++)
				rho+=fA_h[i+XDIM*YDIM*j];
			u = fA_h[i+XDIM*YDIM*1]-fA_h[i+XDIM*YDIM*3]+fA_h[i+XDIM*YDIM*5]-fA_h[i+XDIM*YDIM*6]-
				fA_h[i+XDIM*YDIM*7]+fA_h[i+XDIM*YDIM*8];
			v = fA_h[i+XDIM*YDIM*2]-fA_h[i+XDIM*YDIM*4]+fA_h[i+XDIM*YDIM*5]+fA_h[i+XDIM*YDIM*6]-fA_h[i+XDIM*YDIM*7]-fA_h[i+XDIM*YDIM*8];
			output<<col<<", "<<row<<", "<<u<<","<<v<<","<<rho<<endl;
		}
	}

	if(REFINEMENT == "YES"){
	output<<endl<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
	//output<<"ZONE F=POINT, I="<<XLRDIM-LRLEVEL*2<<", J="<<YLRDIM-LRLEVEL*2<<"\n";
	output<<"ZONE F=POINT, I="<<XLRDIM<<", J="<<YLRDIM<<"\n";
	
//	for(row = LRLEVEL; row<YLRDIM-LRLEVEL; row++){
//		for(col = LRLEVEL; col<XLRDIM-LRLEVEL; col++){
	for(row = 0; row<YLRDIM-0; row++){
		for(col = 0; col<XLRDIM-0; col++){
			i = row*XLRDIM+col;
			rho = fC_h[i];
			for(j = 1; j<9; j++)
				rho+=fC_h[i+XLRDIM*YLRDIM*j];
			u = fC_h[i+XLRDIM*YLRDIM*1]-fC_h[i+XLRDIM*YLRDIM*3]+fC_h[i+XLRDIM*YLRDIM*5]-fC_h[i+XLRDIM*YLRDIM*6]-
				fC_h[i+XLRDIM*YLRDIM*7]+fC_h[i+XLRDIM*YLRDIM*8];
			v = fC_h[i+XLRDIM*YLRDIM*2]-fC_h[i+XLRDIM*YLRDIM*4]+fC_h[i+XLRDIM*YLRDIM*5]+fC_h[i+XLRDIM*YLRDIM*6]-fC_h[i+XLRDIM*YLRDIM*7]-fC_h[i+XLRDIM*YLRDIM*8];
			output<<LRX0+LRFACTOR*col<<", "<<LRY0+LRFACTOR*row<<", "<<u<<","<<v<<","<<rho<<endl;
		}
	}
	}

	output.close();

	//for(int t = STARTF-1; t<TMAX; t++){
	for(int t = 0; t<TMAX; t++){
		output2<<t<<", "<<FX_h[t]/(0.5f*UMAX*UMAX*OBSTR*2.0f)<<", "<<FY_h[t]/(0.5f*UMAX*UMAX*OBSTR*2.0f)<<endl;
	}
	output2.close();


	//hipFree(image_d);


	hipFree(fA_d);
	hipFree(fB_d);
	hipFree(fC_d);
	hipFree(fD_d);
	hipFree(FX_d);
	hipFree(FY_d);
	return(0);

}
