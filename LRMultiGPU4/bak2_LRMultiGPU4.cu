#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>

using namespace std;

#define CASENAME "test_new"

#define BLOCKSIZEX 64
#define BLOCKSIZEY 1
#define BLOCKSIZEZ 1
#define BLOCKSIZELRX 64
#define BLOCKSIZELRY 1
#define BLOCKSIZELRZ 1
#define XDIM 64
#define YDIM 64
#define ZDIM 8
#define TMAX  1
#define STARTF 0

#define OBSTR1 4.f
#define OBSTX1 23.5f
#define OBSTY1 23.5f
#define OBSTZ1 32.5f

#define OBSTR2 32.f
#define OBSTX2 319.5f
#define OBSTY2 511.5f
#define OBSTZ2 31.5f

#define LRFACTOR 0.5f
#define LRLEVEL 2
#define LRX0 11.75f    //minimum x coord of LR
#define XLRDIM 64    //number of nodes in x
#define LRY0 11.75f
#define YLRDIM 92
#define LRZ0 -0.25f
#define ZLRDIM 16

#define RE 20.f//2000.f//100.f;
#define UMAX 0.08f
#define SmagLES 0 //YES,NO
#define MODEL "MRT" //BGK,MRT,STREAM
#define REFINEMENT 1 //1,0
#define CS 0.1f

#define VELAV 0
#define START_VELAV   400000
#define START_VELFLUC 700000

inline __device__ int ImageFcn(float x, float y, float z){
    int value = 0;
//    if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//        value = 10;
//    else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//        value = 10;
    
    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
        value = 10;

    return value;
}

inline __device__ int ImageFcn(int x, int y, int z){
    int value = 0;
//Cylinder
//    if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//        value = 10;
//    else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//        value = 10;
    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
        value = 10;
//Lid Driven Cavity
//    if(y == 0 || y == YDIM-1 || z == 0 || z == ZDIM-1)
//        value = 1;
//    else if(x == XDIM-2 || y == 1 || y == YDIM-2 || z == 1 || z == ZDIM-2)
//        return 1;
//    else if(x == 0)
//        return 1;
    
//    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
//        value = 10;
//    if(z == 1)
//        value = 1;
//    if(z == ZDIM-2)
//        value = 1;
    else if(y == 0)
        value = 200;//22;
    else if(y == YDIM-1)
        value = 100;
    else if(x == 0)
        value = 26;
    else if(x == XDIM-1)
        value = 25;
    return value;
}

inline __device__ float PoisProf (float x){
    float radius = (YDIM-1-1)*0.5f;
    float result = -1.0f*(((1.0f-(x-0.5f)/radius))*((1.0f-(x-0.5f)/radius))-1.0f);
    return (result);
}

int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

__device__ int dmin(int a, int b)
{
    if (a<b) return a;
    else return b-1;
}
__device__ int dmax(int a)
{
    if (a>-1) return a;
    else return 0;
}
__device__ int dmin_p(int a, int b)
{
    if (a<b) return a;
    else return 0;
}
__device__ int dmax_p(int a, int b)
{
    if (a>-1) return a;
    else return b-1;
}

inline __device__ float trilinear_interp (float v000, float v001, float v010, float v011,
                                          float v100, float v101, float v110, float v111, float x, float y, float z){
    return v000*(1.f-x)*(1.f-y)*(1.f-z)+
           v001*(    x)*(1.f-y)*(1.f-z)+
           v010*(1.f-x)*(    y)*(1.f-z)+
           v011*(    x)*(    y)*(1.f-z)+
           v100*(1.f-x)*(1.f-y)*(    z)+
           v101*(    x)*(1.f-y)*(    z)+
           v110*(1.f-x)*(    y)*(    z)+
           v111*(    x)*(    y)*(    z);
}

inline __device__ int f_mem(int f_num, int x, int y, int z, size_t pitch, int zInner)
{
    int index = (x+y*pitch+z*YDIM*pitch)+f_num*pitch*YDIM*(zInner);
    index = dmax(index);
    index = dmin(index,19*pitch*YDIM*(zInner));
    return index;
}

inline __device__ int f_memLR(int f_num, int x, int y, int z, size_t pitch, int zInner)
{
    int index = (x+y*pitch+z*YLRDIM*pitch)+f_num*pitch*YLRDIM*(zInner);
    index = dmax(index);
    index = dmin(index,19*pitch*YLRDIM*(zInner));
    return index;
}

inline __device__ int buff_mem(int f_num, int x, int y, size_t pitch)
{
    int index = (x+y*pitch)+f_num*pitch*YDIM;
    index = dmax(index);
    index = dmin(index,19*pitch*YDIM);
    return index;
}

inline __device__ int buff_memLR(int f_num, int x, int y, size_t pitch)
{
    int index = (x+y*pitch)+f_num*pitch*YLRDIM;
    index = dmax(index);
    index = dmin(index,19*pitch*YLRDIM);
    return index;
}

inline __device__ void mrt_meq(float* meq, float rho, float u, float v, float w)
{
    meq[ 0] = rho;
    meq[ 1] = -11.f*rho+19.f*(u*u+v*v+w*w);
    meq[ 2] = 7.53968254f*(u*u+v*v+w*w);;
    meq[ 3] = u;
    meq[ 4] = -0.666666667f*u;
    meq[ 5] = v;
    meq[ 6] = -0.666666667f*v;
    meq[ 7] = w;
    meq[ 8] = -0.666666667f*w;
    meq[ 9] = 2.f*u*u-(v*v+w*w);
    meq[11] = v*v-w*w;
}

inline __device__ void InvertMoments(float* f, float* m)
{
float u = m[3];
float v = m[5];
float w = m[7];
f[0 ]=(0.052631579f*m[0]                           +- 0.012531328f*(m[1])+ 0.047619048f*(m[2]));
f[1 ]=(0.052631579f*m[0]+  0.1f*u                  +-0.0045948204f*(m[1])+-0.015873016f*(m[2])+  -0.1f*(m[4])                 + 0.055555556f*((m[9])-m[10]));
f[2 ]=(0.052631579f*m[0]         +  0.1f*v         +-0.0045948204f*(m[1])+-0.015873016f*(m[2])             +   -0.1f*(m[6])   +-0.027777778f*((m[9])-m[10])+ 0.083333333f*((m[11])-m[12]));
f[3 ]=(0.052631579f*m[0]+ -0.1f*u                  +-0.0045948204f*(m[1])+-0.015873016f*(m[2])+   0.1f*(m[4])                 + 0.055555556f*((m[9])-m[10]));                                                      
f[4 ]=(0.052631579f*m[0]         + -0.1f*v         +-0.0045948204f*(m[1])+-0.015873016f*(m[2])             +    0.1f*(m[6])   +-0.027777778f*((m[9])-m[10])+ 0.083333333f*((m[11])-m[12]));
f[5 ]=(0.052631579f*m[0]+  0.1f*u+  0.1f*v         + 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+ 0.025f*(m[4]+m[6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]-m[17])+ (0.027777778f*(m[9]) +0.083333333f*(m[11])+( 0.25f*(m[13]))));
f[6 ]=(0.052631579f*m[0]+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+-0.025f*(m[4]-m[6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]-m[17])+ (0.027777778f*(m[9]) +0.083333333f*(m[11])+(-0.25f*(m[13]))));
f[7 ]=(0.052631579f*m[0]+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+-0.025f*(m[4]+m[6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]+m[17])+ (0.027777778f*(m[9]) +0.083333333f*(m[11])+( 0.25f*(m[13]))));
f[8 ]=(0.052631579f*m[0]+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+ 0.025f*(m[4]-m[6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]+m[17])+ (0.027777778f*(m[9]) +0.083333333f*(m[11])+(-0.25f*(m[13]))));
f[9 ]=(0.052631579f*m[0]                  +  0.1f*w+-0.0045948204f*(m[1])+-0.015873016f*(m[2])                +   -0.1f*(m[8])+-0.027777778f*((m[9])-m[10])+-0.083333333f*((m[11])-m[12]));                                       
f[10]=(0.052631579f*m[0]+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+ 0.025f*(m[4]+m[8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]+m[18])+ (0.027777778f*(m[9]) -0.083333333f*(m[11])+( 0.25f*(m[15]))));
f[11]=(0.052631579f*m[0]         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])             +  0.025f*(m[6]+m[8])+0.125f*( m[17]-m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[9]) +( 0.25f*(m[14]))));
f[12]=(0.052631579f*m[0]+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+-0.025f*(m[4]-m[8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]+m[18])+ (0.027777778f*(m[9]) -0.083333333f*(m[11])+(-0.25f*(m[15]))));
f[13]=(0.052631579f*m[0]         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])             + -0.025f*(m[6]-m[8])+0.125f*(-m[17]-m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[9]) +(-0.25f*(m[14]))));
f[14]=(0.052631579f*m[0]                  + -0.1f*w+-0.0045948204f*(m[1])+-0.015873016f*(m[2])                +    0.1f*(m[8])+-0.027777778f*((m[9])-m[10])+-0.083333333f*((m[11])-m[12]));                                      
f[15]=(0.052631579f*m[0]+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+ 0.025f*(m[4]-m[8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]-m[18])+ (0.027777778f*(m[9]) -0.083333333f*(m[11])+(-0.25f*(m[15]))));
f[16]=(0.052631579f*m[0]         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])             +  0.025f*(m[6]-m[8])+0.125f*( m[17]+m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[9]) +(-0.25f*(m[14]))));
f[17]=(0.052631579f*m[0]+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])+-0.025f*(m[4]+m[8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]-m[18])+ (0.027777778f*(m[9]) -0.083333333f*(m[11])+( 0.25f*(m[15]))));
f[18]=(0.052631579f*m[0]         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m[1])+ 0.003968254f*(m[2])             + -0.025f*(m[6]+m[8])+0.125f*(-m[17]+m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[9]) +( 0.25f*(m[14]))));


}

inline __device__ void mrt_collide(float* f, float omega)
{
    float m[19];
    float u,v,w;    
    u = f[ 1]-f[ 3]+f[ 5]-f[ 6]-f[ 7]+f[ 8]+f[10]-f[12]+f[15]-f[17];
    v = f[ 2]-f[ 4]+f[ 5]+f[ 6]-f[ 7]-f[ 8]+f[11]-f[13]+f[16]-f[18];
    w = f[ 9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
    float rho = f[ 0]+f[ 1]+f[ 2]+f[ 3]+f[ 4]+f[ 5]+f[ 6]+f[ 7]+f[ 8]+f[ 9]+
          f[10]+f[11]+f[12]+f[13]+f[14]+f[15]+f[16]+f[17]+f[18];

    m[ 1]  = 19.f*(-f[ 0]+ f[ 5]+f[ 6]+f[ 7]+f[ 8]+f[10]+f[11]+f[12]+f[13]+f[15]+f[16]+f[17]+f[18]   -(u*u+v*v+w*w));//+8.f*(f[ 5]+f[ 6]+f[ 7]+f[ 8]+f[10]+f[11]+f[12]+f[13]+f[15]+f[16]+f[17]+f[18]);
    m[ 2]  =  12.f*f[ 0]+ -4.f*f[ 1]+ -4.f*f[ 2]+ -4.f*f[ 3]+ -4.f*f[ 4]+      f[ 5]+      f[ 6]+      f[ 7]+      f[ 8]+ -4.f*f[ 9]+    f[10]+        f[11]+      f[12]+      f[13]+ -4.f*f[14]+      f[15]+      f[16]+      f[17]+      f[18] +7.53968254f*(u*u+v*v+w*w);
    m[ 4]  = 1.666666667f*(-3.f*f[1]+3.f*f[ 3]+u);
    m[ 6]  = 1.666666667f*(-3.f*f[2]+3.f*f[ 4]+v);
    m[ 8]  = 1.666666667f*(-3.f*f[9]+3.f*f[14]+w);
    m[ 9]  = 2.f*f[ 1]+  -  f[ 2]+  2.f*f[ 3]+  -  f[ 4]+ f[ 5]+ f[ 6]+ f[ 7]+ f[ 8]+-    f[ 9]+ f[10]+ -2.f*f[11]+ f[12]+-2.f*f[13]+-    f[14]+ f[15]+ -2.f*f[16]+ f[17]+-2.f*f[18]  -(2.f*u*u-(v*v+w*w));
    m[10] =-4.f*f[ 1]+ 2.f*f[ 2]+ -4.f*f[ 3]+ 2.f*f[ 4]+ f[ 5]+ f[ 6]+ f[ 7]+ f[ 8]+ 2.f*f[ 9]+ f[10]+ -2.f*f[11]+ f[12]+-2.f*f[13]+ 2.f*f[14]+ f[15]+ -2.f*f[16]+ f[17]+-2.f*f[18];
    m[11] =             f[ 2]         +     f[ 4]+ f[ 5]+ f[ 6]+ f[ 7]+ f[ 8]+-    f[ 9]+-f[10]          +-f[12]         +-    f[14]+-f[15]          +-f[17]         -(v*v-w*w);
    m[12] =        -2.f*f[ 2]          -2.f*f[ 4]+ f[ 5]+ f[ 6]+ f[ 7]+ f[ 8]+ 2.f*f[ 9]+-f[10]          +-f[12]         + 2.f*f[14]+-f[15]          +-f[17]         ;
    m[13] =                                  f[ 5]+-f[ 6]+ f[ 7]+-f[ 8]                                                                             -u*v;
    m[14] =                                                                    f[11]     +-    f[13]              + -    f[16]     +     f[18]  -v*w;
    m[15] =                                                          f[10]        + - f[12]                  +-f[15]          + f[17]           -u*w;  
    m[16] =                                  f[ 5]+-f[ 6]+-f[ 7]+ f[ 8]         -f[10]        +   f[12]                  +-f[15]          + f[17]         ;  
    m[17] =                                 -f[ 5]+-f[ 6]+ f[ 7]+ f[ 8]              +     f[11]     +-    f[13]              +      f[16]     +-    f[18];  
    m[18] =                                                          f[10]+-     f[11]+ f[12]+-    f[13]         +-f[15]+      f[16]+-f[17]+     f[18];

f[ 0] -=- 0.012531328f*(m[ 1])+ 0.047619048f*(m[ 2]);
f[ 1] -=-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])+  -0.1f*(m[ 4])                 + 0.055555556f*((m[ 9])*omega-m[10]);
f[ 2] -=-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])             +   -0.1f*(m[ 6])   +-0.027777778f*((m[ 9])*omega-m[10])+ 0.083333333f*((m[11])*omega-m[12]);
f[ 3] -=-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])+   0.1f*(m[ 4])                 + 0.055555556f*((m[ 9])*omega-m[10]);                                                                                         
f[ 4] -=-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])             +    0.1f*(m[ 6])   +-0.027777778f*((m[ 9])*omega-m[10])+ 0.083333333f*((m[11])*omega-m[12]);
f[ 5] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]+m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]-m[17])+ omega*(0.027777778f*(m[ 9]) +0.083333333f*(m[11])+( 0.25f*(m[13])));
f[ 6] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]-m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]-m[17])+ omega*(0.027777778f*(m[ 9]) +0.083333333f*(m[11])+(-0.25f*(m[13])));
f[ 7] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]+m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]+m[17])+ omega*(0.027777778f*(m[ 9]) +0.083333333f*(m[11])+( 0.25f*(m[13])));
f[ 8] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]-m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]+m[17])+ omega*(0.027777778f*(m[ 9]) +0.083333333f*(m[11])+(-0.25f*(m[13])));
f[ 9] -=-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])                +   -0.1f*(m[ 8])+-0.027777778f*((m[ 9])*omega-m[10])+-0.083333333f*((m[11])*omega-m[12]);                                       
f[10] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]+m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]+m[18])+ omega*(0.027777778f*(m[ 9]) -0.083333333f*(m[11])+( 0.25f*(m[15])));
f[11] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             +  0.025f*(m[ 6]+m[ 8])+0.125f*( m[17]-m[18])-0.027777778f*(m[10])+omega*(-0.055555556f*(m[ 9]) +( 0.25f*(m[14])));
f[12] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]-m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]+m[18])+ omega*(0.027777778f*(m[ 9]) -0.083333333f*(m[11])+(-0.25f*(m[15])));
f[13] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             + -0.025f*(m[ 6]-m[ 8])+0.125f*(-m[17]-m[18])-0.027777778f*(m[10])+omega*(-0.055555556f*(m[ 9]) +(-0.25f*(m[14])));
f[14] -=-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])                +    0.1f*(m[ 8])+-0.027777778f*((m[ 9])*omega-m[10])+-0.083333333f*((m[11])*omega-m[12]);                                      
f[15] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]-m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]-m[18])+ omega*(0.027777778f*(m[ 9]) -0.083333333f*(m[11])+(-0.25f*(m[15])));
f[16] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             +  0.025f*(m[ 6]-m[ 8])+0.125f*( m[17]+m[18])-0.027777778f*(m[10])+omega*(-0.055555556f*(m[ 9]) +(-0.25f*(m[14])));
f[17] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]+m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]-m[18])+ omega*(0.027777778f*(m[ 9]) -0.083333333f*(m[11])+( 0.25f*(m[15])));
f[18] -= 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             + -0.025f*(m[ 6]+m[ 8])+0.125f*(-m[17]+m[18])-0.027777778f*(m[10])+omega*(-0.055555556f*(m[ 9]) +( 0.25f*(m[14])));

}

inline __device__ void North_Extrap(float* f, float rho)
{
    float m[19];
    rho = 1.0f;
    float u = f[ 1]-f[ 3]+f[ 5]-f[ 6]-f[ 7]+f[ 8]+f[10]-f[12]+f[15]-f[17];
    float v = f[ 2]-f[ 4]+f[ 5]+f[ 6]-f[ 7]-f[ 8]+f[11]-f[13]+f[16]-f[18];
    float w = f[ 9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];


    m[ 1]  = -30.f*f[ 0]+-11.f*f[ 1]+-11.f*f[ 2]+-11.f*f[ 3]+-11.f*f[ 4]+  8.f*f[ 5]+  8.f*f[ 6]+  8.f*f[ 7]+  8.f*f[ 8]+-11.f*f[ 9]+  8.f*f[10]+  8.f*f[11]+  8.f*f[12]+  8.f*f[13]+-11.f*f[14]+  8.f*f[15]+  8.f*f[16]+  8.f*f[17]+  8.f*f[18];
    m[ 2]  =  12.f*f[ 0]+ -4.f*f[ 1]+ -4.f*f[ 2]+ -4.f*f[ 3]+ -4.f*f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+ -4.f*f[ 9]+    f[10]+      f[11]+    f[12]+    f[13]+ -4.f*f[14]+    f[15]+    f[16]+    f[17]+    f[18];
    m[ 4]  =           -4.f*f[ 1]         +  4.f*f[ 3]         +    f[ 5]+ -  f[ 6]+ -  f[ 7]+    f[ 8]         +    f[10]          + -  f[12]                    +    f[15]          + -  f[17]          ;
    m[ 6]  =                    -4.f*f[ 2]         +  4.f*f[ 4]+    f[ 5]+    f[ 6]+ -  f[ 7]+ -  f[ 8]                   +    f[11]          + -  f[13]                    +    f[16]          + -  f[18];
    m[ 8]  =                                                                                 + -4.f*f[ 9]+    f[10]+    f[11]+    f[12]+    f[13]+  4.f*f[14]+ -  f[15]+ -  f[16]+ -  f[17]+ -  f[18];
    m[ 9]  =            2.f*f[ 1]+ -  f[ 2]+  2.f*f[ 3]+ -  f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+ -  f[ 9]+    f[10]+ -2.f*f[11]+    f[12]+ -2.f*f[13]+ -  f[14]+    f[15]+ -2.f*f[16]+    f[17]+ -2.f*f[18];
    m[10] =           -4.f*f[ 1]+  2.f*f[ 2]+ -4.f*f[ 3]+  2.f*f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+  2.f*f[ 9]+    f[10]+ -2.f*f[11]+    f[12]+ -2.f*f[13]+  2.f*f[14]+    f[15]+ -2.f*f[16]+    f[17]+ -2.f*f[18];
    m[11] =                       f[ 2]         +    f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+ -  f[ 9]+ -  f[10]          + -  f[12]          + -  f[14]+ -  f[15]          + -  f[17]          ;
    m[12] =                    -2.f*f[ 2]           -2.f*f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+  2.f*f[ 9]+ -  f[10]          + -  f[12]          +  2.f*f[14]+ -  f[15]          + -  f[17]          ;
    m[13] =                                                  f[ 5]+ -  f[ 6]+    f[ 7]+ -  f[ 8]                                                                                                   ;
    m[14] =                                                                                                         f[11]          + -  f[13]                    + -  f[16]          +    f[18];
    m[15] =                                                                                               f[10]          + -  f[12]                    + -  f[15]          +    f[17]          ;  
    m[16] =                                                  f[ 5]+ -  f[ 6]+ -  f[ 7]+    f[ 8]           -  f[10]          +    f[12]                    + -  f[15]          +    f[17]          ;  
    m[17] =                                               -  f[ 5]+ -  f[ 6]+    f[ 7]+    f[ 8]                   +    f[11]          + -  f[13]                    +    f[16]          + -  f[18];  
    m[18] =                                                                                               f[10]+ -  f[11]+    f[12]+ -  f[13]          + -  f[15]+    f[16]+ -  f[17]+    f[18];

f[ 0] =(0.052631579f*rho                           +- 0.012531328f*(m[ 1])+ 0.047619048f*(m[ 2]));
f[ 1] =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])+  -0.1f*(m[ 4])                 + 0.055555556f*((m[ 9])-m[10]));
f[ 2] =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])             +   -0.1f*(m[ 6])   +-0.027777778f*((m[ 9])-m[10])+ 0.083333333f*((m[11])-m[12]));
f[ 3] =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])+   0.1f*(m[ 4])                 + 0.055555556f*((m[ 9])-m[10]));        
f[ 4] =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])             +    0.1f*(m[ 6])   +-0.027777778f*((m[ 9])-m[10])+ 0.083333333f*((m[11])-m[12]));
f[ 5] =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]+m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]-m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+( 0.25f*(m[13]))));
f[ 6] =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]-m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]-m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+(-0.25f*(m[13]))));
f[ 7] =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]+m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]+m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+( 0.25f*(m[13]))));
f[ 8] =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]-m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]+m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+(-0.25f*(m[13]))));
f[ 9] =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])                +   -0.1f*(m[ 8])+-0.027777778f*((m[ 9])-m[10])+-0.083333333f*((m[11])-m[12]));                                       
f[10]=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]+m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]+m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+( 0.25f*(m[15]))));
f[11]=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             +  0.025f*(m[ 6]+m[ 8])+0.125f*( m[17]-m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +( 0.25f*(m[14]))));
f[12]=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]-m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]+m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+(-0.25f*(m[15]))));
f[13]=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             + -0.025f*(m[ 6]-m[ 8])+0.125f*(-m[17]-m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +(-0.25f*(m[14]))));
f[14]=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])                +    0.1f*(m[ 8])+-0.027777778f*((m[ 9])-m[10])+-0.083333333f*((m[11])-m[12]));                                      
f[15]=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]-m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]-m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+(-0.25f*(m[15]))));
f[16]=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             +  0.025f*(m[ 6]-m[ 8])+0.125f*( m[17]+m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +(-0.25f*(m[14]))));
f[17]=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]+m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]-m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+( 0.25f*(m[15]))));
f[18]=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             + -0.025f*(m[ 6]+m[ 8])+0.125f*(-m[17]+m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +( 0.25f*(m[14]))));

}

inline __device__ void South_Extrap(float* f, float v)
{
    float m[19];
    float u = 0.f;//f[ 1]-f[ 3]+f[ 5]-f[ 6]-f[ 7]+f[ 8]+f[10]-f[12]+f[15]-f[17];
    float w = 0.f;//f[ 9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
    float rho = f[0]+f[1]+f[2]+f[3]+f[4]+f[5]+f[6]+f[7]+f[8]+f[9]+f[10]+f[11]+f[12]+f[13]+f[14]+f[15]+f[16]+f[17]+f[18];

    m[ 1]  = -30.f*f[ 0]+-11.f*f[ 1]+-11.f*f[ 2]+-11.f*f[ 3]+-11.f*f[ 4]+  8.f*f[ 5]+  8.f*f[ 6]+  8.f*f[ 7]+  8.f*f[ 8]+-11.f*f[ 9]+  8.f*f[10]+  8.f*f[11]+  8.f*f[12]+  8.f*f[13]+-11.f*f[14]+  8.f*f[15]+  8.f*f[16]+  8.f*f[17]+  8.f*f[18];
    m[ 2]  =  12.f*f[ 0]+ -4.f*f[ 1]+ -4.f*f[ 2]+ -4.f*f[ 3]+ -4.f*f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+ -4.f*f[ 9]+    f[10]+      f[11]+    f[12]+    f[13]+ -4.f*f[14]+    f[15]+    f[16]+    f[17]+    f[18];
    m[ 4]  =           -4.f*f[ 1]         +  4.f*f[ 3]         +    f[ 5]+ -  f[ 6]+ -  f[ 7]+    f[ 8]         +    f[10]          + -  f[12]                    +    f[15]          + -  f[17]          ;
    m[ 6]  =                    -4.f*f[ 2]         +  4.f*f[ 4]+    f[ 5]+    f[ 6]+ -  f[ 7]+ -  f[ 8]                   +    f[11]          + -  f[13]                    +    f[16]          + -  f[18];
    m[ 8]  =                                                                                 + -4.f*f[ 9]+    f[10]+    f[11]+    f[12]+    f[13]+  4.f*f[14]+ -  f[15]+ -  f[16]+ -  f[17]+ -  f[18];
    m[ 9]  =            2.f*f[ 1]+ -  f[ 2]+  2.f*f[ 3]+ -  f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+ -  f[ 9]+    f[10]+ -2.f*f[11]+    f[12]+ -2.f*f[13]+ -  f[14]+    f[15]+ -2.f*f[16]+    f[17]+ -2.f*f[18];
    m[10] =           -4.f*f[ 1]+  2.f*f[ 2]+ -4.f*f[ 3]+  2.f*f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+  2.f*f[ 9]+    f[10]+ -2.f*f[11]+    f[12]+ -2.f*f[13]+  2.f*f[14]+    f[15]+ -2.f*f[16]+    f[17]+ -2.f*f[18];
    m[11] =                       f[ 2]         +    f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+ -  f[ 9]+ -  f[10]          + -  f[12]          + -  f[14]+ -  f[15]          + -  f[17]          ;
    m[12] =                    -2.f*f[ 2]           -2.f*f[ 4]+    f[ 5]+    f[ 6]+    f[ 7]+    f[ 8]+  2.f*f[ 9]+ -  f[10]          + -  f[12]          +  2.f*f[14]+ -  f[15]          + -  f[17]          ;
    m[13] =                                                  f[ 5]+ -  f[ 6]+    f[ 7]+ -  f[ 8]                                                                                                   ;
    m[14] =                                                                                                         f[11]          + -  f[13]                    + -  f[16]          +    f[18];
    m[15] =                                                                                               f[10]          + -  f[12]                    + -  f[15]          +    f[17]          ;  
    m[16] =                                                  f[ 5]+ -  f[ 6]+ -  f[ 7]+    f[ 8]           -  f[10]          +    f[12]                    + -  f[15]          +    f[17]          ;  
    m[17] =                                               -  f[ 5]+ -  f[ 6]+    f[ 7]+    f[ 8]                   +    f[11]          + -  f[13]                    +    f[16]          + -  f[18];  
    m[18] =                                                                                               f[10]+ -  f[11]+    f[12]+ -  f[13]          + -  f[15]+    f[16]+ -  f[17]+    f[18];

f[ 0] =(0.052631579f*rho                           +- 0.012531328f*(m[ 1])+ 0.047619048f*(m[ 2]));
f[ 1] =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])+  -0.1f*(m[ 4])                 + 0.055555556f*((m[ 9])-m[10]));
f[ 2] =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])             +   -0.1f*(m[ 6])   +-0.027777778f*((m[ 9])-m[10])+ 0.083333333f*((m[11])-m[12]));
f[ 3] =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])+   0.1f*(m[ 4])                 + 0.055555556f*((m[ 9])-m[10]));        
f[ 4] =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])             +    0.1f*(m[ 6])   +-0.027777778f*((m[ 9])-m[10])+ 0.083333333f*((m[11])-m[12]));
f[ 5] =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]+m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]-m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+( 0.25f*(m[13]))));
f[ 6] =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]-m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]-m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+(-0.25f*(m[13]))));
f[ 7] =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]+m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*(-m[16]+m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+( 0.25f*(m[13]))));
f[ 8] =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]-m[ 6])              +0.013888889f*(m[10])+0.041666667f*(m[12])+0.125f*( m[16]+m[17])+ (0.027777778f*(m[ 9]) +0.083333333f*(m[11])+(-0.25f*(m[13]))));
f[ 9] =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])                +   -0.1f*(m[ 8])+-0.027777778f*((m[ 9])-m[10])+-0.083333333f*((m[11])-m[12]));                                       
f[10]=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]+m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]+m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+( 0.25f*(m[15]))));
f[11]=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             +  0.025f*(m[ 6]+m[ 8])+0.125f*( m[17]-m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +( 0.25f*(m[14]))));
f[12]=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]-m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]+m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+(-0.25f*(m[15]))));
f[13]=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             + -0.025f*(m[ 6]-m[ 8])+0.125f*(-m[17]-m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +(-0.25f*(m[14]))));
f[14]=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m[ 1])+-0.015873016f*(m[ 2])                +    0.1f*(m[ 8])+-0.027777778f*((m[ 9])-m[10])+-0.083333333f*((m[11])-m[12]));                                      
f[15]=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+ 0.025f*(m[ 4]-m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*(-m[16]-m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+(-0.25f*(m[15]))));
f[16]=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             +  0.025f*(m[ 6]-m[ 8])+0.125f*( m[17]+m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +(-0.25f*(m[14]))));
f[17]=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])+-0.025f*(m[ 4]+m[ 8])              +0.013888889f*(m[10])-0.041666667f*(m[12])+0.125f*( m[16]-m[18])+ (0.027777778f*(m[ 9]) -0.083333333f*(m[11])+( 0.25f*(m[15]))));
f[18]=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m[ 1])+ 0.003968254f*(m[ 2])             + -0.025f*(m[ 6]+m[ 8])+0.125f*(-m[17]+m[18])-0.027777778f*(m[10])+(-0.055555556f*(m[ 9]) +( 0.25f*(m[14]))));

}

__device__ void xsymmetry_bot(float* f, int y, int z)
{
    if(y == 0 && z == 0){
        f[ 2] = f[ 4];
        f[13]=f[18];
        f[11]=f[18];
        f[16]=f[18];
        f[ 6] =f[ 7];
        f[ 9] =f[14];
        f[12]=f[17];
    }
    else if(y == 0 && z == ZDIM-1){
        f[ 4] = f[ 2];
        f[11]=f[13];
        f[18]=f[13];
        f[16]=f[13];
        f[ 6] =f[ 7];
        f[14]=f[ 9];
        f[17]=f[12];
    }
    else if(y == YDIM-1 && z == 0){
        f[ 4] = f[ 2];
        f[11]=f[16];
        f[18]=f[16];
        f[13]=f[16];
        f[ 7] =f[ 6];
        f[ 9] =f[14];
        f[12]=f[17];
    }
    else if(y == YDIM-1 && z == ZDIM-1){
        f[ 4] = f[ 2];
        f[16]=f[11];
        f[18]=f[11];
        f[13]=f[11];
        f[ 7] =f[ 6];
        f[14]=f[ 9];
        f[17]=f[12];
    }
    else{
    if(y == 0){
        f[ 2] = f[ 4];
        f[11]=f[13];
        f[16]=f[18];
        f[ 8] = f[ 5];
    }
    else if(y == YDIM-1){
         f[ 4]=f[ 2] ;
        f[13]=f[11];
        f[18]=f[16];
         f[ 5]=f[ 8] ;
    }
    }
    f[ 1] = f[ 3] ;
    f[ 5] = f[ 6] ;
    f[ 8] = f[ 7] ;
    f[10]= f[12];
    f[15]= f[17];
}

__device__ void xsymmetry_top(float* f, int y, int z)
{
    if(y == 0 && z == 0){
        f[ 2]  = f[ 4];
        f[13] = f[18];            
        f[11] = f[18];
        f[16] = f[18];
        f[ 5]  = f[ 8];
        f[ 9]  = f[14];
        f[10] = f[15];
    }
    else if(y == 0 && z == ZDIM-1){
        f[ 2]  = f[ 4];
        f[11] = f[13];
        f[18] = f[13];
        f[16] = f[13];
        f[ 5]  = f[ 8];
        f[14] = f[ 9];
        f[15] = f[10];
    }
    else if(y == YDIM-1 && z == 0){
        f[ 4]  = f[ 2];
        f[18] = f[16];
        f[11] = f[16];
        f[13] = f[16];    
        f[ 8]  = f[ 5];
        f[ 9]  = f[14];
        f[10] = f[15];
    }
    else if(y == YDIM-1 && z == ZDIM-1){
        f[ 4]  = f[ 2];
        f[13] = f[11];
        f[16] = f[11];
        f[18] = f[11];
        f[ 8]  = f[ 5];
        f[14] = f[ 9];
        f[15] = f[10];
    }

    else{
    if(y == 0){
        f[ 2] = f[ 4];
        f[11] = f[13];
        f[16] = f[18];
        f[ 5] = f[ 8];
    }
    else if(y == YDIM-1){
        f[ 4] = f[ 2];
        f[13] = f[11];
        f[18] = f[16];
        f[ 8] = f[ 5];
    }
    }
    f[ 3] = f[ 1] ;
    f[ 6] = f[ 5] ;
    f[ 7] = f[ 8] ;
    f[12]= f[10];
    f[17]= f[15];
}

inline __device__ void vel_av(float* f, float& uAv, float& vAv, int t)
{
    float u,v;//,w;    
    u = f[ 1]-f[ 3]+f[ 5]-f[ 6]-f[ 7]+f[ 8]+f[10]-f[12]+f[15]-f[17];
    v = f[ 2]-f[ 4]+f[ 5]+f[ 6]-f[ 7]-f[ 8]+f[11]-f[13]+f[16]-f[18];
    uAv = (uAv*(t-START_VELAV)+u)/((t-START_VELAV)+1);
    vAv = (vAv*(t-START_VELAV)+v)/((t-START_VELAV)+1);
}
inline __device__ void vel_avLR(float* f, float& uAv, float& vAv, float t)
{
    float u,v;//,w;    
    u = f[ 1]-f[ 3]+f[ 5]-f[ 6]-f[ 7]+f[ 8]+f[10]-f[12]+f[15]-f[17];
    v = f[ 2]-f[ 4]+f[ 5]+f[ 6]-f[ 7]-f[ 8]+f[11]-f[13]+f[16]-f[18];
    uAv = (uAv*(t-START_VELAV)+u*LRFACTOR)/((t-START_VELAV)+LRFACTOR);
    vAv = (vAv*(t-START_VELAV)+v*LRFACTOR)/((t-START_VELAV)+LRFACTOR);
}

inline __device__ void vel_fluc(float* f, float& uAv, 
                    float& vAv, float& ufluc, float& vfluc, int t)
{
    float u,v;//,w;    
    u = f[ 1]-f[ 3]+f[ 5]-f[ 6]-f[ 7]+f[ 8]+f[10]-f[12]+f[15]-f[17];
    v = f[ 2]-f[ 4]+f[ 5]+f[ 6]-f[ 7]-f[ 8]+f[11]-f[13]+f[16]-f[18];
    u = (u-uAv)*(u-uAv);
    v = (v-vAv)*(v-vAv);
    ufluc = (ufluc*(t-START_VELFLUC)+u)/((t-START_VELFLUC)+1);
    vfluc = (vfluc*(t-START_VELFLUC)+v)/((t-START_VELFLUC)+1);
}
inline __device__ void vel_flucLR(float* f, float& uAv, 
                    float& vAv, float& ufluc, float& vfluc, float t)
{
    float u,v;//,w;    
    u = f[ 1]-f[ 3]+f[ 5]-f[ 6]-f[ 7]+f[ 8]+f[10]-f[12]+f[15]-f[17];
    v = f[ 2]-f[ 4]+f[ 5]+f[ 6]-f[ 7]-f[ 8]+f[11]-f[13]+f[16]-f[18];
    u = (u-uAv)*(u-uAv);
    v = (v-vAv)*(v-vAv);
    ufluc = (ufluc*(t-START_VELFLUC)+u*LRFACTOR)/((t-START_VELFLUC)+LRFACTOR);
    vfluc = (vfluc*(t-START_VELFLUC)+v*LRFACTOR)/((t-START_VELFLUC)+LRFACTOR);
}

__global__ void initialize(float *fout, size_t pitch, int zInner, int GPU_N)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = threadIdx.z+blockIdx.z*blockDim.z;
    float xcoord = x;
    float ycoord = y;
    float zcoord = z+1+GPU_N*ZDIM;
    int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
    float f[19] = {0};
    float m[19] = {0};
    
    int im = ImageFcn(xcoord,ycoord,zcoord);
    float u,v,w,rho;
    rho = 1.f;
    u = 0.01f;
    v = UMAX;
    w = 0.0;

    if(im == 10 || im == 1){
    u = 0.0f;
    v = 0.0f;
    w = 0.0f;
    }
    
    mrt_meq(m,rho,u,v,w);
    InvertMoments(f,m);
    
    for(int i = 0; i<19; i++)
    fout[j+i *pitch*YDIM*zInner]=f[ i];
}

__global__ void initializeLR(float *fout, size_t pitch, int zInner, int GPU_N)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = threadIdx.z+blockIdx.z*blockDim.z;
    float xcoord = x;
    float ycoord = y;
    float zcoord = z+1+GPU_N*(zInner+2);
    xcoord = LRX0+x*LRFACTOR;
    ycoord = LRY0+y*LRFACTOR;
    zcoord = LRZ0+z*LRFACTOR;
    int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
    float f[19] = {0};
    float m[19] = {0};
    
    int im = ImageFcn(xcoord,ycoord,zcoord);
    float u,v,w,rho;
    rho = 1.f;
    u = 0.01f;
    v = UMAX;
    w = 0.0f;

    if(im == 10 || im == 1){
    u = 0.0f;
    v = 0.0f;
    w = 0.0f;
    }
    
    mrt_meq(m,rho,u,v,w);
    InvertMoments(f,m);
    
    for(int i = 0; i<19; i++)
    fout[j+i *pitch*YLRDIM*zInner]=f[ i];
}

__global__ void update_top(float* hB, float* hA, float* fA, float* temp,
                            float omega, size_t pitch, int GPU, int zInner, float* FX, float* FY, float* FZ, int t, int flag_F)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int j = x+y*pitch;//index on padded mem (pitch in elements)
    int im = ImageFcn(x,y,(GPU+1)*(zInner+2)-1);
    float f[19];
    __shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();

    f[0 ]= hA  [j];
    f[1 ]= hA  [buff_mem(1 ,x-1,y  ,pitch)];
    f[3 ]= hA  [buff_mem(3 ,x+1,y  ,pitch)];
    f[2 ]= hA  [buff_mem(2 ,x  ,y-1,pitch)];
    f[5 ]= hA  [buff_mem(5 ,x-1,y-1,pitch)];
    f[6 ]= hA  [buff_mem(6 ,x+1,y-1,pitch)];
    f[4 ]= hA  [buff_mem(4 ,x  ,y+1,pitch)];
    f[7 ]= hA  [buff_mem(7 ,x+1,y+1,pitch)];
    f[8 ]= hA  [buff_mem(8 ,x-1,y+1,pitch)];
    f[9 ]= fA  [f_mem   (9 ,x  ,y  ,zInner-1,pitch, zInner)];
    f[10]= fA  [f_mem   (10,x-1,y  ,zInner-1,pitch, zInner)];
    f[11]= fA  [f_mem   (11,x  ,y-1,zInner-1,pitch, zInner)];
    f[12]= fA  [f_mem   (12,x+1,y  ,zInner-1,pitch, zInner)];
    f[13]= fA  [f_mem   (13,x  ,y+1,zInner-1,pitch, zInner)];
    f[14]= temp[buff_mem(14,x  ,y  ,pitch)];
    f[15]= temp[buff_mem(15,x-1,y  ,pitch)];
    f[16]= temp[buff_mem(16,x  ,y-1,pitch)];
    f[17]= temp[buff_mem(17,x+1,y  ,pitch)];
    f[18]= temp[buff_mem(18,x  ,y+1,pitch)];

    if(im == 1 || im ==10){//BB
        if(im == 10 && flag_F == 1){
        check[0] = 1;
        sumX[threadIdx.x]=2.f*f[ 1]-2.f*f[ 3]+2.f*f[ 5]+2.f*f[ 8]-2.f*f[ 6];
        sumX[threadIdx.x]+=-2.f*f[ 7]+2.f*f[10]-2.f*f[12]+2.f*f[15]-2.f*f[17];
        sumY[threadIdx.x]=2.f*f[ 2]-2.f*f[ 4]+2.f*f[ 5]-2.f*f[ 8]+2.f*f[ 6];
        sumY[threadIdx.x]+=-2.f*f[ 7]+2.f*f[11]-2.f*f[13]+2.f*f[16]-2.f*f[18];
        sumZ[threadIdx.x]=2.f*f[ 9]+2.f*f[10]+2.f*f[11]+2.f*f[12]+2.f*f[13];
        sumZ[threadIdx.x]+=-2.f*f[14]-2.f*f[15]-2.f*f[16]-2.f*f[17]-2.f*f[18];
        }
        else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        }
        hB[buff_mem(0 ,x,y,pitch)] = f[0 ];
        hB[buff_mem(1 ,x,y,pitch)] = f[3 ];
        hB[buff_mem(2 ,x,y,pitch)] = f[4 ];
        hB[buff_mem(3 ,x,y,pitch)] = f[1 ];
        hB[buff_mem(4 ,x,y,pitch)] = f[2 ];
        hB[buff_mem(5 ,x,y,pitch)] = f[7 ];
        hB[buff_mem(6 ,x,y,pitch)] = f[8 ];
        hB[buff_mem(7 ,x,y,pitch)] = f[5 ];
        hB[buff_mem(8 ,x,y,pitch)] = f[6 ];
        hB[buff_mem(9 ,x,y,pitch)] = f[14];
        hB[buff_mem(10,x,y,pitch)] = f[17];
        hB[buff_mem(11,x,y,pitch)] = f[18];
        hB[buff_mem(12,x,y,pitch)] = f[15];
        hB[buff_mem(13,x,y,pitch)] = f[16];
        hB[buff_mem(14,x,y,pitch)] = f[9 ];
        hB[buff_mem(15,x,y,pitch)] = f[12];
        hB[buff_mem(16,x,y,pitch)] = f[13];
        hB[buff_mem(17,x,y,pitch)] = f[10];
        hB[buff_mem(18,x,y,pitch)] = f[11];
    }
    else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        if(im == 100)//north outlet
        {
            for(int i = 0; i<19; i++)
                   f[i ]= hA[buff_mem(i ,x,y-1,pitch)];
            North_Extrap(f,1.0f);
        }
        if(im == 200)//south inlet
        {
            for(int i = 0; i<19; i++)
                   f[i ]= hA[buff_mem(i ,x,y+1,pitch)];
            South_Extrap(f,UMAX);
        }
        if(im == 25)
            xsymmetry_top(f,y,(GPU+1)*(zInner+2)-1);
        if(im == 26)
            xsymmetry_bot(f,y,(GPU+1)*(zInner+2)-1);

        mrt_collide(f,omega);

        for(int i = 0; i<19; i++)
            hB[buff_mem(i ,x,y,pitch)] = f[i ];
    }
    syncthreads();
    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
            sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
            sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t-STARTF],sumX[0]);
        atomicAdd(&FY[t-STARTF],sumY[0]);
        atomicAdd(&FZ[t-STARTF],sumZ[0]);
    }
    }

}

__global__ void update_bot(float* gB, float* gA, float* fA, float* temp,
                            float omega, size_t pitch, int GPU, int zInner, float* FX, float* FY, float* FZ, int t, int flag_F)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int j = x+y*pitch;//index on padded mem (pitch in elements)
    int im = ImageFcn(x,y,(GPU+1)*(zInner+2)-1);
    float f[19];
    __shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();

    f[0 ]= gA  [j];
    f[1 ]= gA  [buff_mem(1 ,x-1,y  ,pitch)];
    f[3 ]= gA  [buff_mem(3 ,x+1,y  ,pitch)];
    f[2 ]= gA  [buff_mem(2 ,x  ,y-1,pitch)];
    f[5 ]= gA  [buff_mem(5 ,x-1,y-1,pitch)];
    f[6 ]= gA  [buff_mem(6 ,x+1,y-1,pitch)];
    f[4 ]= gA  [buff_mem(4 ,x  ,y+1,pitch)];
    f[7 ]= gA  [buff_mem(7 ,x+1,y+1,pitch)];
    f[8 ]= gA  [buff_mem(8 ,x-1,y+1,pitch)];
    f[9 ]= temp[buff_mem(9 ,x  ,y  ,pitch)];
    f[10]= temp[buff_mem(10,x-1,y  ,pitch)];
    f[11]= temp[buff_mem(11,x  ,y-1,pitch)];
    f[12]= temp[buff_mem(12,x+1,y  ,pitch)];
    f[13]= temp[buff_mem(13,x  ,y+1,pitch)];
    f[14]= fA  [f_mem   (14,x  ,y  ,0,pitch, zInner)];
    f[15]= fA  [f_mem   (15,x-1,y  ,0,pitch, zInner)];
    f[16]= fA  [f_mem   (16,x  ,y-1,0,pitch, zInner)];
    f[17]= fA  [f_mem   (17,x+1,y  ,0,pitch, zInner)];
    f[18]= fA  [f_mem   (18,x  ,y+1,0,pitch, zInner)];

    if(im == 1 || im ==10){//BB
        if(im == 10 && flag_F == 1){
        check[0] = 1;
        sumX[threadIdx.x]=2.f*f[ 1]-2.f*f[ 3]+2.f*f[ 5]+2.f*f[ 8]-2.f*f[ 6];
        sumX[threadIdx.x]+=-2.f*f[ 7]+2.f*f[10]-2.f*f[12]+2.f*f[15]-2.f*f[17];
        sumY[threadIdx.x]=2.f*f[ 2]-2.f*f[ 4]+2.f*f[ 5]-2.f*f[ 8]+2.f*f[ 6];
        sumY[threadIdx.x]+=-2.f*f[ 7]+2.f*f[11]-2.f*f[13]+2.f*f[16]-2.f*f[18];
        sumZ[threadIdx.x]=2.f*f[ 9]+2.f*f[10]+2.f*f[11]+2.f*f[12]+2.f*f[13];
        sumZ[threadIdx.x]+=-2.f*f[14]-2.f*f[15]-2.f*f[16]-2.f*f[17]-2.f*f[18];
        }
        else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        }
        gB[buff_mem(0 ,x,y,pitch)] = f[0 ];
        gB[buff_mem(1 ,x,y,pitch)] = f[3 ];
        gB[buff_mem(2 ,x,y,pitch)] = f[4 ];
        gB[buff_mem(3 ,x,y,pitch)] = f[1 ];
        gB[buff_mem(4 ,x,y,pitch)] = f[2 ];
        gB[buff_mem(5 ,x,y,pitch)] = f[7 ];
        gB[buff_mem(6 ,x,y,pitch)] = f[8 ];
        gB[buff_mem(7 ,x,y,pitch)] = f[5 ];
        gB[buff_mem(8 ,x,y,pitch)] = f[6 ];
        gB[buff_mem(9 ,x,y,pitch)] = f[14];
        gB[buff_mem(10,x,y,pitch)] = f[17];
        gB[buff_mem(11,x,y,pitch)] = f[18];
        gB[buff_mem(12,x,y,pitch)] = f[15];
        gB[buff_mem(13,x,y,pitch)] = f[16];
        gB[buff_mem(14,x,y,pitch)] = f[9 ];
        gB[buff_mem(15,x,y,pitch)] = f[12];
        gB[buff_mem(16,x,y,pitch)] = f[13];
        gB[buff_mem(17,x,y,pitch)] = f[10];
        gB[buff_mem(18,x,y,pitch)] = f[11];
    }
    else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        if(im == 100)//north outlet
        {
            for(int i = 0; i<19; i++)
                   f[i ]= gA[buff_mem(i ,x,y-1,pitch)];
            North_Extrap(f,1.0f);
        }
        if(im == 200)//south inlet
        {
            for(int i = 0; i<19; i++)
                   f[i ]= gA[buff_mem(i ,x,y+1,pitch)];
            South_Extrap(f,UMAX);
        }
        if(im == 25)
            xsymmetry_top(f,y,GPU*(zInner+2));
        if(im == 26)
            xsymmetry_bot(f,y,GPU*(zInner+2));

        mrt_collide(f,omega);

        for(int i = 0; i<19; i++)
            gB[buff_mem(i ,x,y,pitch)] = f[i ];
    }
    syncthreads();
    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
            sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
            sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t-STARTF],sumX[0]);
        atomicAdd(&FY[t-STARTF],sumY[0]);
        atomicAdd(&FZ[t-STARTF],sumZ[0]);
    }
    }
}

__global__ void update_inn(float* fB, float* fA, float* g, float* h, float omega, size_t pitch, int GPU, int zInner, float** velAv, float** velFluc, float* FX, float* FY, float* FZ, int t, int flag_F)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = threadIdx.z+blockIdx.z*blockDim.z;
    int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
    int im = ImageFcn(x,y,GPU*(zInner+2)+1+z);
    float f[19];
    __shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();

    f[ 0] = fA[j];
    f[ 1] = fA[f_mem   (1 ,x-1,y  ,z  ,pitch, zInner)];
    f[ 3] = fA[f_mem   (3 ,x+1,y  ,z  ,pitch, zInner)];
    f[ 2] = fA[f_mem   (2 ,x  ,y-1,z  ,pitch, zInner)];
    f[ 5] = fA[f_mem   (5 ,x-1,y-1,z  ,pitch, zInner)];
    f[ 6] = fA[f_mem   (6 ,x+1,y-1,z  ,pitch, zInner)];
    f[ 4] = fA[f_mem   (4 ,x  ,y+1,z  ,pitch, zInner)];
    f[ 7] = fA[f_mem   (7 ,x+1,y+1,z  ,pitch, zInner)];
    f[ 8] = fA[f_mem   (8 ,x-1,y+1,z  ,pitch, zInner)];

    if(z==zInner-1){//top nodes need info from h
    f[ 9] = fA[f_mem   (9 ,x  ,y  ,z-1,pitch, zInner)];
    f[10]= fA[f_mem   (10,x-1,y  ,z-1,pitch, zInner)];
    f[11]= fA[f_mem   (11,x  ,y-1,z-1,pitch, zInner)];
    f[12]= fA[f_mem   (12,x+1,y  ,z-1,pitch, zInner)];
    f[13]= fA[f_mem   (13,x  ,y+1,z-1,pitch, zInner)];
    f[14]= h [buff_mem(14,x  ,y  ,pitch)];
    f[15]= h [buff_mem(15,x-1,y  ,pitch)];
    f[16]= h [buff_mem(16,x  ,y-1,pitch)];
    f[17]= h [buff_mem(17,x+1,y  ,pitch)];
    f[18]= h [buff_mem(18,x  ,y+1,pitch)];
    }
    else if(z==0){//bottom nodes need info from g
    f[ 9] =g [buff_mem(9 ,x  ,y  ,pitch)];
    f[10]= g [buff_mem(10,x-1,y  ,pitch)];
    f[11]= g [buff_mem(11,x  ,y-1,pitch)];
    f[12]= g [buff_mem(12,x+1,y  ,pitch)];
    f[13]= g [buff_mem(13,x  ,y+1,pitch)];
    f[14]= fA[f_mem   (14,x  ,y  ,z+1,pitch, zInner)];
    f[15]= fA[f_mem   (15,x-1,y  ,z+1,pitch, zInner)];
    f[16]= fA[f_mem   (16,x  ,y-1,z+1,pitch, zInner)];
    f[17]= fA[f_mem   (17,x+1,y  ,z+1,pitch, zInner)];
    f[18]= fA[f_mem   (18,x  ,y+1,z+1,pitch, zInner)];
    }
    else{//normal nodes
    f[ 9] = fA[f_mem(9 ,x  ,y  ,z-1,pitch,zInner)];
    f[10]= fA[f_mem(10,x-1,y  ,z-1,pitch,zInner)];
    f[11]= fA[f_mem(11,x  ,y-1,z-1,pitch,zInner)];
    f[12]= fA[f_mem(12,x+1,y  ,z-1,pitch,zInner)];
    f[13]= fA[f_mem(13,x  ,y+1,z-1,pitch,zInner)];
    f[14]= fA[f_mem(14,x  ,y  ,z+1,pitch,zInner)];
    f[15]= fA[f_mem(15,x-1,y  ,z+1,pitch,zInner)];
    f[16]= fA[f_mem(16,x  ,y-1,z+1,pitch,zInner)];
    f[17]= fA[f_mem(17,x+1,y  ,z+1,pitch,zInner)];
    f[18]= fA[f_mem(18,x  ,y+1,z+1,pitch,zInner)];

    }//end normal nodes

    if(im == 1 || im ==10){//BB
        if(im == 10 && flag_F == 1){
        check[0] = 1;
        sumX[threadIdx.x]=2.f*f[ 1]-2.f*f[ 3]+2.f*f[ 5]+2.f*f[ 8]-2.f*f[ 6];
        sumX[threadIdx.x]+=-2.f*f[ 7]+2.f*f[10]-2.f*f[12]+2.f*f[15]-2.f*f[17];
        sumY[threadIdx.x]=2.f*f[ 2]-2.f*f[ 4]+2.f*f[ 5]-2.f*f[ 8]+2.f*f[ 6];
        sumY[threadIdx.x]+=-2.f*f[ 7]+2.f*f[11]-2.f*f[13]+2.f*f[16]-2.f*f[18];
        sumZ[threadIdx.x]=2.f*f[ 9]+2.f*f[10]+2.f*f[11]+2.f*f[12]+2.f*f[13];
        sumZ[threadIdx.x]+=-2.f*f[14]-2.f*f[15]-2.f*f[16]-2.f*f[17]-2.f*f[18];
        }
        else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        }
        fB[f_mem(1 ,x,y,z,pitch,zInner)] = f[ 3] ;
        fB[f_mem(2 ,x,y,z,pitch,zInner)] = f[ 4] ;
        fB[f_mem(3 ,x,y,z,pitch,zInner)] = f[ 1] ;
        fB[f_mem(4 ,x,y,z,pitch,zInner)] = f[ 2] ;
        fB[f_mem(5 ,x,y,z,pitch,zInner)] = f[ 7] ;
        fB[f_mem(6 ,x,y,z,pitch,zInner)] = f[ 8] ;
        fB[f_mem(7 ,x,y,z,pitch,zInner)] = f[ 5] ;
        fB[f_mem(8 ,x,y,z,pitch,zInner)] = f[ 6] ;
        fB[f_mem(9 ,x,y,z,pitch,zInner)] = f[14];
        fB[f_mem(10,x,y,z,pitch,zInner)] = f[17];
        fB[f_mem(11,x,y,z,pitch,zInner)] = f[18];
        fB[f_mem(12,x,y,z,pitch,zInner)] = f[15];
        fB[f_mem(13,x,y,z,pitch,zInner)] = f[16];
        fB[f_mem(14,x,y,z,pitch,zInner)] = f[ 9] ;
        fB[f_mem(15,x,y,z,pitch,zInner)] = f[12];
        fB[f_mem(16,x,y,z,pitch,zInner)] = f[13];
        fB[f_mem(17,x,y,z,pitch,zInner)] = f[10];
        fB[f_mem(18,x,y,z,pitch,zInner)] = f[11];
    }
    else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        if(im == 100)//north outlet
        {
            for(int i = 0; i<19; i++)
                   f[i ]= fA[f_mem(i ,x,y-1,z,pitch,zInner)];
            North_Extrap(f,1.0f);
        }
        if(im == 200)//south inlet
        {
            for(int i = 0; i<19; i++)
                   f[i ]= fA[f_mem(i ,x,y+1,z,pitch,zInner)];
            South_Extrap(f,UMAX);
        }
        if(im == 25)
            xsymmetry_top(f,y,GPU*(zInner+2)+1+z);
        if(im == 26)
            xsymmetry_bot(f,y,GPU*(zInner+2)+1+z);

        mrt_collide(f,omega);

        if(VELAV == 1){
            if(t>=START_VELAV && t<START_VELFLUC){
                float u_Av = velAv[0][x+y*pitch+(z+1)*pitch*YDIM];
                float v_Av = velAv[1][x+y*pitch+(z+1)*pitch*YDIM];
                vel_av(f,u_Av,v_Av,t);
                velAv[0][x+y*pitch+(z+1)*pitch*YDIM] = u_Av;
                velAv[1][x+y*pitch+(z+1)*pitch*YDIM] = v_Av;
            }
            else if(t>=START_VELFLUC){
                float u_Av = velAv[0][x+y*pitch+(z+1)*pitch*YDIM];
                float v_Av = velAv[1][x+y*pitch+(z+1)*pitch*YDIM];
                float u_fluc = velFluc[0][x+y*pitch+(z+1)*pitch*YDIM];
                float v_fluc = velFluc[1][x+y*pitch+(z+1)*pitch*YDIM];
                vel_fluc(f,u_Av,v_Av,u_fluc,v_fluc,t);
                velFluc[0][x+y*pitch+(z+1)*pitch*YDIM] = u_fluc;
                velFluc[1][x+y*pitch+(z+1)*pitch*YDIM] = v_fluc;
            }
        }

        for(int i = 0; i<19; i++)
            fB[f_mem(i ,x,y,z,pitch,zInner)] = f[ i] ;
    }

    syncthreads();
    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
            sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
            sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t-STARTF],sumX[0]);
        atomicAdd(&FY[t-STARTF],sumY[0]);
        atomicAdd(&FZ[t-STARTF],sumZ[0]);
    }
    }

}

__global__ void update_top_LR(float* hB, float* hA, float* fA, float* temp,
                            float omega, size_t pitch, int GPU, int zInner, float* FX, float* FY, float* FZ, int t, int flag_F)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = (GPU+1)*(zInner+2)-1;//physical coord in LR region
    int j = x+y*pitch;//index on padded mem (pitch in elements)
    float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+LRFACTOR*z;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    float f[19];
    __shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();

    f[0 ]= hA  [j];
    f[1 ]= hA  [buff_memLR(1 ,x-1,y  ,pitch)];
    f[3 ]= hA  [buff_memLR(3 ,x+1,y  ,pitch)];
    f[2 ]= hA  [buff_memLR(2 ,x  ,y-1,pitch)];
    f[5 ]= hA  [buff_memLR(5 ,x-1,y-1,pitch)];
    f[6 ]= hA  [buff_memLR(6 ,x+1,y-1,pitch)];
    f[4 ]= hA  [buff_memLR(4 ,x  ,y+1,pitch)];
    f[7 ]= hA  [buff_memLR(7 ,x+1,y+1,pitch)];
    f[8 ]= hA  [buff_memLR(8 ,x-1,y+1,pitch)];
    f[9 ]= fA  [   f_memLR(9 ,x  ,y  ,zInner-1,pitch, zInner)];
    f[10]= fA  [   f_memLR(10,x-1,y  ,zInner-1,pitch, zInner)];
    f[11]= fA  [   f_memLR(11,x  ,y-1,zInner-1,pitch, zInner)];
    f[12]= fA  [   f_memLR(12,x+1,y  ,zInner-1,pitch, zInner)];
    f[13]= fA  [   f_memLR(13,x  ,y+1,zInner-1,pitch, zInner)];
    f[14]= temp[buff_memLR(14,x  ,y  ,pitch)];
    f[15]= temp[buff_memLR(15,x-1,y  ,pitch)];
    f[16]= temp[buff_memLR(16,x  ,y-1,pitch)];
    f[17]= temp[buff_memLR(17,x+1,y  ,pitch)];
    f[18]= temp[buff_memLR(18,x  ,y+1,pitch)];

    if(im == 1 || im ==10){//BB
        if(im == 10 && flag_F == 1){
        check[0] = 1;
        sumX[threadIdx.x]=2.f*f[ 1]-2.f*f[ 3]+2.f*f[ 5]+2.f*f[ 8]-2.f*f[ 6];
        sumX[threadIdx.x]+=-2.f*f[ 7]+2.f*f[10]-2.f*f[12]+2.f*f[15]-2.f*f[17];
        sumY[threadIdx.x]=2.f*f[ 2]-2.f*f[ 4]+2.f*f[ 5]-2.f*f[ 8]+2.f*f[ 6];
        sumY[threadIdx.x]+=-2.f*f[ 7]+2.f*f[11]-2.f*f[13]+2.f*f[16]-2.f*f[18];
        sumZ[threadIdx.x]=2.f*f[ 9]+2.f*f[10]+2.f*f[11]+2.f*f[12]+2.f*f[13];
        sumZ[threadIdx.x]+=-2.f*f[14]-2.f*f[15]-2.f*f[16]-2.f*f[17]-2.f*f[18];
        }
        else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        }
        hB[buff_memLR(0 ,x,y,pitch)] = f[0 ];
        hB[buff_memLR(1 ,x,y,pitch)] = f[3 ];
        hB[buff_memLR(2 ,x,y,pitch)] = f[4 ];
        hB[buff_memLR(3 ,x,y,pitch)] = f[1 ];
        hB[buff_memLR(4 ,x,y,pitch)] = f[2 ];
        hB[buff_memLR(5 ,x,y,pitch)] = f[7 ];
        hB[buff_memLR(6 ,x,y,pitch)] = f[8 ];
        hB[buff_memLR(7 ,x,y,pitch)] = f[5 ];
        hB[buff_memLR(8 ,x,y,pitch)] = f[6 ];
        hB[buff_memLR(9 ,x,y,pitch)] = f[14];
        hB[buff_memLR(10,x,y,pitch)] = f[17];
        hB[buff_memLR(11,x,y,pitch)] = f[18];
        hB[buff_memLR(12,x,y,pitch)] = f[15];
        hB[buff_memLR(13,x,y,pitch)] = f[16];
        hB[buff_memLR(14,x,y,pitch)] = f[9 ];
        hB[buff_memLR(15,x,y,pitch)] = f[12];
        hB[buff_memLR(16,x,y,pitch)] = f[13];
        hB[buff_memLR(17,x,y,pitch)] = f[10];
        hB[buff_memLR(18,x,y,pitch)] = f[11];
    }
    else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;

        mrt_collide(f,omega);

        for(int i = 0; i<19; i++)
            hB[buff_memLR(i ,x,y,pitch)] = f[i ];
    }
    syncthreads();
    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
            sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
            sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t-STARTF],sumX[0]);
        atomicAdd(&FY[t-STARTF],sumY[0]);
        atomicAdd(&FZ[t-STARTF],sumZ[0]);
    }
    }

}

__global__ void update_bot_LR(float* gB, float* gA, float* fA, float* temp,
                            float omega, size_t pitch, int GPU, int zInner, float* FX, float* FY, float* FZ, int t, int flag_F)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = (zInner+2)-1;
    int j = x+y*pitch;//index on padded mem (pitch in elements)
    float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+GPU*LRFACTOR*z;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    float f[19];
    __shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();

    f[0 ]= gA  [j];
    f[1 ]= gA  [buff_memLR(1 ,x-1,y  ,pitch)];
    f[3 ]= gA  [buff_memLR(3 ,x+1,y  ,pitch)];
    f[2 ]= gA  [buff_memLR(2 ,x  ,y-1,pitch)];
    f[5 ]= gA  [buff_memLR(5 ,x-1,y-1,pitch)];
    f[6 ]= gA  [buff_memLR(6 ,x+1,y-1,pitch)];
    f[4 ]= gA  [buff_memLR(4 ,x  ,y+1,pitch)];
    f[7 ]= gA  [buff_memLR(7 ,x+1,y+1,pitch)];
    f[8 ]= gA  [buff_memLR(8 ,x-1,y+1,pitch)];
    f[9 ]= temp[buff_memLR(9 ,x  ,y  ,pitch)];
    f[10]= temp[buff_memLR(10,x-1,y  ,pitch)];
    f[11]= temp[buff_memLR(11,x  ,y-1,pitch)];
    f[12]= temp[buff_memLR(12,x+1,y  ,pitch)];
    f[13]= temp[buff_memLR(13,x  ,y+1,pitch)];
    f[14]= fA  [   f_memLR(14,x  ,y  ,0,pitch, zInner)];
    f[15]= fA  [   f_memLR(15,x-1,y  ,0,pitch, zInner)];
    f[16]= fA  [   f_memLR(16,x  ,y-1,0,pitch, zInner)];
    f[17]= fA  [   f_memLR(17,x+1,y  ,0,pitch, zInner)];
    f[18]= fA  [   f_memLR(18,x  ,y+1,0,pitch, zInner)];

    if(im == 1 || im ==10){//BB
        if(im == 10 && flag_F == 1){
        check[0] = 1;
        sumX[threadIdx.x]=2.f*f[ 1]-2.f*f[ 3]+2.f*f[ 5]+2.f*f[ 8]-2.f*f[ 6];
        sumX[threadIdx.x]+=-2.f*f[ 7]+2.f*f[10]-2.f*f[12]+2.f*f[15]-2.f*f[17];
        sumY[threadIdx.x]=2.f*f[ 2]-2.f*f[ 4]+2.f*f[ 5]-2.f*f[ 8]+2.f*f[ 6];
        sumY[threadIdx.x]+=-2.f*f[ 7]+2.f*f[11]-2.f*f[13]+2.f*f[16]-2.f*f[18];
        sumZ[threadIdx.x]=2.f*f[ 9]+2.f*f[10]+2.f*f[11]+2.f*f[12]+2.f*f[13];
        sumZ[threadIdx.x]+=-2.f*f[14]-2.f*f[15]-2.f*f[16]-2.f*f[17]-2.f*f[18];
        }
        else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        }
        gB[buff_memLR(0 ,x,y,pitch)] = f[0 ];
        gB[buff_memLR(1 ,x,y,pitch)] = f[3 ];
        gB[buff_memLR(2 ,x,y,pitch)] = f[4 ];
        gB[buff_memLR(3 ,x,y,pitch)] = f[1 ];
        gB[buff_memLR(4 ,x,y,pitch)] = f[2 ];
        gB[buff_memLR(5 ,x,y,pitch)] = f[7 ];
        gB[buff_memLR(6 ,x,y,pitch)] = f[8 ];
        gB[buff_memLR(7 ,x,y,pitch)] = f[5 ];
        gB[buff_memLR(8 ,x,y,pitch)] = f[6 ];
        gB[buff_memLR(9 ,x,y,pitch)] = f[14];
        gB[buff_memLR(10,x,y,pitch)] = f[17];
        gB[buff_memLR(11,x,y,pitch)] = f[18];
        gB[buff_memLR(12,x,y,pitch)] = f[15];
        gB[buff_memLR(13,x,y,pitch)] = f[16];
        gB[buff_memLR(14,x,y,pitch)] = f[9 ];
        gB[buff_memLR(15,x,y,pitch)] = f[12];
        gB[buff_memLR(16,x,y,pitch)] = f[13];
        gB[buff_memLR(17,x,y,pitch)] = f[10];
        gB[buff_memLR(18,x,y,pitch)] = f[11];
    }
    else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;

        mrt_collide(f,omega);

        for(int i = 0; i<19; i++)
            gB[buff_memLR(i ,x,y,pitch)] = f[i ];
    }
    syncthreads();
    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
            sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
            sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t-STARTF],sumX[0]);
        atomicAdd(&FY[t-STARTF],sumY[0]);
        atomicAdd(&FZ[t-STARTF],sumZ[0]);
    }
    }
}

__global__ void update_inn_LR(float* fB, float* fA, float* g, float* h, float omega, size_t pitch, int GPU, int zInner, float** velAv, float** velFluc, float* FX, float* FY, float* FZ, int t, int flag_F)
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
    int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(LRX0+LRFACTOR*x,LRY0+LRFACTOR*y,LRZ0+LRFACTOR*(GPU*(zInner+2)+1+z));
    float f[19];
    __shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
    __shared__ int check[1];
    check[0] = 0;
    syncthreads();

    f[ 0] = fA[j];
    f[ 1] = fA[f_memLR   (1 ,x-1,y  ,z  ,pitch, zInner)];
    f[ 3] = fA[f_memLR   (3 ,x+1,y  ,z  ,pitch, zInner)];
    f[ 2] = fA[f_memLR   (2 ,x  ,y-1,z  ,pitch, zInner)];
    f[ 5] = fA[f_memLR   (5 ,x-1,y-1,z  ,pitch, zInner)];
    f[ 6] = fA[f_memLR   (6 ,x+1,y-1,z  ,pitch, zInner)];
    f[ 4] = fA[f_memLR   (4 ,x  ,y+1,z  ,pitch, zInner)];
    f[ 7] = fA[f_memLR   (7 ,x+1,y+1,z  ,pitch, zInner)];
    f[ 8] = fA[f_memLR   (8 ,x-1,y+1,z  ,pitch, zInner)];

    if(z==zInner-1){//top nodes need info from h
    f[ 9] =fA[   f_memLR(9 ,x  ,y  ,z-1,pitch, zInner)];
    f[10]= fA[   f_memLR(10,x-1,y  ,z-1,pitch, zInner)];
    f[11]= fA[   f_memLR(11,x  ,y-1,z-1,pitch, zInner)];
    f[12]= fA[   f_memLR(12,x+1,y  ,z-1,pitch, zInner)];
    f[13]= fA[   f_memLR(13,x  ,y+1,z-1,pitch, zInner)];
    f[14]= h [buff_memLR(14,x  ,y  ,pitch)];
    f[15]= h [buff_memLR(15,x-1,y  ,pitch)];
    f[16]= h [buff_memLR(16,x  ,y-1,pitch)];
    f[17]= h [buff_memLR(17,x+1,y  ,pitch)];
    f[18]= h [buff_memLR(18,x  ,y+1,pitch)];
    }
    else if(z==0){//bottom nodes need info from g
    f[ 9] =g [buff_memLR(9 ,x  ,y  ,pitch)];
    f[10]= g [buff_memLR(10,x-1,y  ,pitch)];
    f[11]= g [buff_memLR(11,x  ,y-1,pitch)];
    f[12]= g [buff_memLR(12,x+1,y  ,pitch)];
    f[13]= g [buff_memLR(13,x  ,y+1,pitch)];
    f[14]= fA[   f_memLR(14,x  ,y  ,z+1,pitch, zInner)];
    f[15]= fA[   f_memLR(15,x-1,y  ,z+1,pitch, zInner)];
    f[16]= fA[   f_memLR(16,x  ,y-1,z+1,pitch, zInner)];
    f[17]= fA[   f_memLR(17,x+1,y  ,z+1,pitch, zInner)];
    f[18]= fA[   f_memLR(18,x  ,y+1,z+1,pitch, zInner)];
    }
    else{//normal nodes
    f[ 9] =fA[f_memLR(9 ,x  ,y  ,z-1,pitch,zInner)];
    f[10]= fA[f_memLR(10,x-1,y  ,z-1,pitch,zInner)];
    f[11]= fA[f_memLR(11,x  ,y-1,z-1,pitch,zInner)];
    f[12]= fA[f_memLR(12,x+1,y  ,z-1,pitch,zInner)];
    f[13]= fA[f_memLR(13,x  ,y+1,z-1,pitch,zInner)];
    f[14]= fA[f_memLR(14,x  ,y  ,z+1,pitch,zInner)];
    f[15]= fA[f_memLR(15,x-1,y  ,z+1,pitch,zInner)];
    f[16]= fA[f_memLR(16,x  ,y-1,z+1,pitch,zInner)];
    f[17]= fA[f_memLR(17,x+1,y  ,z+1,pitch,zInner)];
    f[18]= fA[f_memLR(18,x  ,y+1,z+1,pitch,zInner)];

    }//end normal nodes

    if(im == 1 || im ==10){//BB
        if(im == 10 && flag_F == 1){
        check[0] = 1;
        sumX[threadIdx.x]=2.f*f[ 1]-2.f*f[ 3]+2.f*f[ 5]+2.f*f[ 8]-2.f*f[ 6];
        sumX[threadIdx.x]+=-2.f*f[ 7]+2.f*f[10]-2.f*f[12]+2.f*f[15]-2.f*f[17];
        sumY[threadIdx.x]=2.f*f[ 2]-2.f*f[ 4]+2.f*f[ 5]-2.f*f[ 8]+2.f*f[ 6];
        sumY[threadIdx.x]+=-2.f*f[ 7]+2.f*f[11]-2.f*f[13]+2.f*f[16]-2.f*f[18];
        sumZ[threadIdx.x]=2.f*f[ 9]+2.f*f[10]+2.f*f[11]+2.f*f[12]+2.f*f[13];
        sumZ[threadIdx.x]+=-2.f*f[14]-2.f*f[15]-2.f*f[16]-2.f*f[17]-2.f*f[18];
        }
        else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;
        }
        fB[f_memLR(1 ,x,y,z,pitch,zInner)] = f[ 3] ;
        fB[f_memLR(2 ,x,y,z,pitch,zInner)] = f[ 4] ;
        fB[f_memLR(3 ,x,y,z,pitch,zInner)] = f[ 1] ;
        fB[f_memLR(4 ,x,y,z,pitch,zInner)] = f[ 2] ;
        fB[f_memLR(5 ,x,y,z,pitch,zInner)] = f[ 7] ;
        fB[f_memLR(6 ,x,y,z,pitch,zInner)] = f[ 8] ;
        fB[f_memLR(7 ,x,y,z,pitch,zInner)] = f[ 5] ;
        fB[f_memLR(8 ,x,y,z,pitch,zInner)] = f[ 6] ;
        fB[f_memLR(9 ,x,y,z,pitch,zInner)] = f[14];
        fB[f_memLR(10,x,y,z,pitch,zInner)] = f[17];
        fB[f_memLR(11,x,y,z,pitch,zInner)] = f[18];
        fB[f_memLR(12,x,y,z,pitch,zInner)] = f[15];
        fB[f_memLR(13,x,y,z,pitch,zInner)] = f[16];
        fB[f_memLR(14,x,y,z,pitch,zInner)] = f[ 9] ;
        fB[f_memLR(15,x,y,z,pitch,zInner)] = f[12];
        fB[f_memLR(16,x,y,z,pitch,zInner)] = f[13];
        fB[f_memLR(17,x,y,z,pitch,zInner)] = f[10];
        fB[f_memLR(18,x,y,z,pitch,zInner)] = f[11];
    }
    else{
        sumX[threadIdx.x]=0.f;
        sumY[threadIdx.x]=0.f;
        sumZ[threadIdx.x]=0.f;

        mrt_collide(f,omega);

        if(VELAV == 1){
            if(t>=START_VELAV && t<START_VELFLUC){
                float u_Av = velAv[0][x+y*pitch+(z+1)*pitch*YLRDIM];
                float v_Av = velAv[1][x+y*pitch+(z+1)*pitch*YLRDIM];
                vel_avLR(f,u_Av,v_Av,t);
                velAv[0][x+y*pitch+(z+1)*pitch*YLRDIM] = u_Av;
                velAv[1][x+y*pitch+(z+1)*pitch*YLRDIM] = v_Av;
            }
            else if(t>=START_VELFLUC){
                float u_Av = velAv[0][x+y*pitch+(z+1)*pitch*YLRDIM];
                float v_Av = velAv[1][x+y*pitch+(z+1)*pitch*YLRDIM];
                float u_fluc = velFluc[0][x+y*pitch+(z+1)*pitch*YLRDIM];
                float v_fluc = velFluc[1][x+y*pitch+(z+1)*pitch*YLRDIM];
                vel_flucLR(f,u_Av,v_Av,u_fluc,v_fluc,t);
                velFluc[0][x+y*pitch+(z+1)*pitch*YLRDIM] = u_fluc;
                velFluc[1][x+y*pitch+(z+1)*pitch*YLRDIM] = v_fluc;
            }
        }

        for(int i = 0; i<19; i++)
            fB[f_memLR(i ,x,y,z,pitch,zInner)] = f[ i] ;
    }

    syncthreads();
    if(check[0] == 1){
    //reduction for force
    int nTotalThreads = blockDim.x;
    while(nTotalThreads > 1){
        int halfPoint = (nTotalThreads >> 1);
        if(threadIdx.x < halfPoint){
            sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
            sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
            sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
        }
        syncthreads();
        nTotalThreads = halfPoint;
    }
    if(threadIdx.x == 0){
        atomicAdd(&FX[t-STARTF],sumX[0]);
        atomicAdd(&FY[t-STARTF],sumY[0]);
        atomicAdd(&FZ[t-STARTF],sumZ[0]);
    }
    }

}



void WriteResults(ostream &output, float *fin, float *gin, float *hin, float **velAv,  
                  float **velFluc, float omega, int GPU_N, int GPU)
{
    float f[19];
    output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\",\"velAv[0]\",\"velAv[1]\",\"ufluc\",\"vfluc\"\n";
    output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<", K="<<ZDIM/GPU_N<<"\n";

    for(int j = 0; j<YDIM; j++){
    for(int i = 0; i<XDIM; i++){
        float rho = 0;
        for(int l = 0; l<19; l++){
            f[l] = gin[(i+j*XDIM)+l *XDIM*YDIM];
            rho += f[l];
        }
        float u = f[1]-f[3 ]+f[5 ]-f[6 ]-f[7 ]+f[8 ]+f[10]-f[12]+f[15]-f[17];
        float v = f[2]-f[4 ]+f[5 ]+f[6 ]-f[7 ]-f[8 ]+f[11]-f[13]+f[16]-f[18];
        float w = f[9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
        output<<i<<", "<<j<<", "<<(ZDIM/GPU_N*GPU)<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
              <<velAv[0][i+j*XDIM]<<","<<velAv[1][i+j*XDIM]<<", "<<velFluc[0][i+j*XDIM]<<","<<velFluc[1][i+j*XDIM]<<endl;
    }}

    for(int k = 1; k<ZDIM/GPU_N-1; k++){
    for(int j = 0; j<YDIM; j++){
    for(int i = 0; i<XDIM; i++){
        float rho = 0;
        for(int l = 0; l<19; l++){
            f[l] = fin[(i+j*XDIM)+(k-1)*XDIM*YDIM+l*XDIM*YDIM*(ZDIM/GPU_N-2)];
            rho += f[l];
        }
        float u = f[1]-f[3 ]+f[5 ]-f[6 ]-f[7 ]+f[8 ]+f[10]-f[12]+f[15]-f[17];
        float v = f[2]-f[4 ]+f[5 ]+f[6 ]-f[7 ]-f[8 ]+f[11]-f[13]+f[16]-f[18];
        float w = f[9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
        int z = (ZDIM/GPU_N*GPU+k);
        output<<i<<", "<<j<<", "<<z<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
              <<velAv[0][i+j*XDIM+k*XDIM*YDIM]<<","<<velAv[1][i+j*XDIM+k*XDIM*YDIM]<<", "
              <<velFluc[0][i+j*XDIM+k*XDIM*YDIM]<<","<<velFluc[1][i+j*XDIM+k*XDIM*YDIM]<<endl;
    }}}

    for(int j = 0; j<YDIM; j++){
    for(int i = 0; i<XDIM; i++){
        float rho = 0;
        for(int l = 0; l<19; l++){
            f[l] = hin[(i+j*XDIM)+l *XDIM*YDIM];
            rho += f[l];
        }
        float u = f[1]-f[3 ]+f[5 ]-f[6 ]-f[7 ]+f[8 ]+f[10]-f[12]+f[15]-f[17];
        float v = f[2]-f[4 ]+f[5 ]+f[6 ]-f[7 ]-f[8 ]+f[11]-f[13]+f[16]-f[18];
        float w = f[9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
        output<<i<<", "<<j<<", "<<(ZDIM/GPU_N*(GPU+1)-1)<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
              <<velAv[0][i+j*XDIM+(ZDIM-1)*XDIM*YDIM]<<","<<velAv[1][i+j*XDIM+(ZDIM/GPU_N-1)*XDIM*YDIM]<<", "
              <<velFluc[0][i+j*XDIM+(ZDIM-1)*XDIM*YDIM]<<","<<velFluc[1][i+j*XDIM+(ZDIM/GPU_N-1)*XDIM*YDIM]<<endl;
    }}
}

void WriteResultsLR(ofstream &output, float *fin, float *gin, float *hin, float **velAv,  
                  float **velFluc, float omega, int GPU_N, int GPU)
{
    float f[19];

    for(int j = 0; j<YLRDIM; j++){
    for(int i = 0; i<XLRDIM; i++){
        float rho = 0;
        for(int l = 0; l<19; l++){
            f[l] = gin[(i+j*XLRDIM)+l *XLRDIM*YLRDIM];
            rho += f[l];
        }
        float u = f[1]-f[3 ]+f[5 ]-f[6 ]-f[7 ]+f[8 ]+f[10]-f[12]+f[15]-f[17];
        float v = f[2]-f[4 ]+f[5 ]+f[6 ]-f[7 ]-f[8 ]+f[11]-f[13]+f[16]-f[18];
        float w = f[9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
        float x = LRX0+LRFACTOR*i;
        float y = LRY0+LRFACTOR*j;
        float z = LRZ0+LRFACTOR*(ZLRDIM/GPU_N*GPU);
        output<<x<<", "<<y<<", "<<z<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
              <<velAv[0][i+j*XLRDIM]<<","<<velAv[1][i+j*XLRDIM]<<", "<<velFluc[0][i+j*XLRDIM]<<","<<velFluc[1][i+j*XLRDIM]<<endl;
    }}

    for(int k = 1; k<ZLRDIM/GPU_N-1; k++){
    for(int j = 0; j<YLRDIM; j++){
    for(int i = 0; i<XLRDIM; i++){
        float rho = 0;
        for(int l = 0; l<19; l++){
            f[l] = fin[(i+j*XLRDIM)+(k-1)*XLRDIM*YLRDIM+l*XLRDIM*YLRDIM*(ZLRDIM/GPU_N-2)];
            rho += f[l];
        }
        float u = f[1]-f[3 ]+f[5 ]-f[6 ]-f[7 ]+f[8 ]+f[10]-f[12]+f[15]-f[17];
        float v = f[2]-f[4 ]+f[5 ]+f[6 ]-f[7 ]-f[8 ]+f[11]-f[13]+f[16]-f[18];
        float w = f[9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
        float x = LRX0+LRFACTOR*i;
        float y = LRY0+LRFACTOR*j;
        float z = LRZ0+LRFACTOR*(ZLRDIM/GPU_N*GPU+k);
        output<<x<<", "<<y<<", "<<z<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
              <<velAv  [0][i+j*XLRDIM+k*XLRDIM*YLRDIM]<<","<<velAv  [1][i+j*XLRDIM+k*XLRDIM*YLRDIM]<<", "
              <<velFluc[0][i+j*XLRDIM+k*XLRDIM*YLRDIM]<<","<<velFluc[1][i+j*XLRDIM+k*XLRDIM*YLRDIM]<<endl;
    }}}

    for(int j = 0; j<YLRDIM; j++){
    for(int i = 0; i<XLRDIM; i++){
        float rho = 0;
        for(int l = 0; l<19; l++){
            f[l] = hin[(i+j*XLRDIM)+l *XLRDIM*YLRDIM];
            rho += f[l];
        }
        float u = f[1]-f[3 ]+f[5 ]-f[6 ]-f[7 ]+f[8 ]+f[10]-f[12]+f[15]-f[17];
        float v = f[2]-f[4 ]+f[5 ]+f[6 ]-f[7 ]-f[8 ]+f[11]-f[13]+f[16]-f[18];
        float w = f[9]+f[10]+f[11]+f[12]+f[13]-f[14]-f[15]-f[16]-f[17]-f[18];
        float x = LRX0+LRFACTOR*i;
        float y = LRY0+LRFACTOR*j;
        float z = LRZ0+LRFACTOR*(ZLRDIM/GPU_N*(GPU+1)-1);
        output<<x<<", "<<y<<", "<<z<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
              <<velAv[0][i+j*XLRDIM+(ZLRDIM/GPU_N-1)*XLRDIM*YLRDIM]<<","<<velAv[1][i+j*XLRDIM+(ZLRDIM/GPU_N-1)*XLRDIM*YLRDIM]<<", "
              <<velFluc[0][i+j*XLRDIM+(ZLRDIM/GPU_N-1)*XLRDIM*YLRDIM]<<","<<velFluc[1][i+j*XLRDIM+(ZLRDIM/GPU_N-1)*XLRDIM*YLRDIM]<<endl;
    }}
}



void WriteForces(float **F, ofstream &output, int ForceTime, int level)
{
    float ref = UMAX*UMAX*ZDIM*OBSTR1;
    if(level > 0)
    ref *= LRLEVEL*LRLEVEL;
    for(int i = 0; i<ForceTime; i++){
        output<<i+STARTF<<", "<<F[0][i]/ref<<", "<<F[1][i]/ref<<", "<<F[2][i]/ref<<endl;
    }
}




void WriteInputs(ostream &output, float omega, float omegaLR, int GPU_per_node)
{
    output<<"Base domain size \t"<<XDIM<<"x"<<YDIM<<"x"<<ZDIM<<endl;
    output<<"Base blocksize: \t"<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<"x"<<BLOCKSIZEZ<<endl;
    output<<"Obst1 location: \t("<<OBSTX1<<","<<OBSTY1<<","<<OBSTZ1<<")"<<endl;
    output<<"Obst1 radius: \t"<<OBSTR1<<endl;
    output<<"Obst2 location: \t("<<OBSTX2<<","<<OBSTY2<<","<<OBSTZ2<<")"<<endl;
    output<<"Obst2 radius: \t"<<OBSTR2<<endl;
    output<<"RE: \t"<<RE<<endl;
    output<<"UMAX: \t"<<UMAX<<endl;
    output<<"omega \t: "<<omega<<endl;
    output<<"TMAX: \t"<<TMAX<<endl;
    output<<"STARTF: \t"<<STARTF<<endl;
    output<<"START_VELAV: \t"<<START_VELAV<<endl;
    output<<"START_VELFLUC: \t"<<START_VELFLUC<<endl;
    output<<"REFINEMENT: \t"<<REFINEMENT<<endl;
    output<<"MODEL: \t"<<MODEL<<endl;
    output<<"Smagorinski LES: \t"<<SmagLES<<endl;
    output<<"CS: \t"<<CS<<endl;
    output<<"LR domain size \t"<<XLRDIM<<"x"<<YLRDIM<<"x"<<ZLRDIM<<endl;
    output<<"LR factor \t"<<LRFACTOR<<endl;
    output<<"LR location \t"<<LRX0<<"x"<<LRY0<<"x"<<LRZ0<<endl;
    output<<"LR blocksize: \t"<<BLOCKSIZELRX<<"x"<<BLOCKSIZELRY<<"x"<<BLOCKSIZELRZ<<endl;   
    output<<"omega in LR \t: "<<omegaLR<<endl;
    output<<"GPUs per node \t: "<<GPU_per_node<<endl;
}





int main(int argc, char *argv[])
{
    
    int GPU_N; hipGetDeviceCount(&GPU_N);
    cout<<"number of GPUs: "<<GPU_N<<endl;

    ofstream output; ofstream outputForce; ofstream outputInputs;
    string FileName = CASENAME;
    output.open ((FileName+".dat").c_str());
    outputForce.open ((FileName+".force").c_str());
    outputInputs.open ((FileName+".inputs").c_str());

    //size_t memsize, memsize2;
    size_t pitch = 2;
    while(pitch<XDIM)
        pitch=pitch*2;
    pitch *= sizeof(float);//pitch*sizeof(float);
    size_t pitch_e = pitch/sizeof(float);
    cout<<"Pitch (in elements): "<<pitch/sizeof(float)<<endl;

    float CharLength = OBSTR1*2.f;

    float omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);
    float omegaLR = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));
    if(LRFACTOR == 0.25f){
    omegaLR = 2.0f/(1.0f+2.0f*(2.0f/omegaLR-1.0f));
    }
    float SF_cf = omega*(1.0f-omegaLR)/((1.0f-omega)*omegaLR/LRFACTOR);
    float SF_fc = 1.f/SF_cf;

    WriteInputs(outputInputs,omega,omegaLR,GPU_N);
    WriteInputs(cout,omega,omegaLR,GPU_N);

    if(abs(LRFACTOR-1.f/LRLEVEL)>0.001f && REFINEMENT == 1){
        cout<<"LRLEVEL and LRFACTOR don't match! Exiting..."<<endl;
        return 0;
    }

    int zInner = ZDIM/GPU_N-2; //excluding halo
    int ForceTime = max(0,TMAX-STARTF);

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
    //2 halo layers per GPU (for 2 GPUs)
    dim3 grid  (((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),(zInner)/BLOCKSIZEZ);
    dim3 g_grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);

    hipStream_t stream_halo[GPU_N];
    hipStream_t stream_inner[GPU_N];

    //data pointers as 3D array (GPUxCoord)
    float   *f_h[GPU_N],   *g_h[GPU_N],   *h_h[GPU_N];
    float *f_d[GPU_N][2], *g_d[GPU_N][2], *h_d[GPU_N][2];
    float *g_temp[GPU_N], *h_temp[GPU_N];
    float *F_h[GPU_N][3];
    float *F_d[GPU_N][3];
    float *F_total[3];
    float *velAv_h[GPU_N][3],*velFluc_h[GPU_N][3];
    float *velAv_d[GPU_N][3],*velFluc_d[GPU_N][3];

    for(int i = 0; i<3; i++)
    F_total[i] = (float *)malloc(ForceTime*sizeof(float));
    
    for(int i=0;i<3;i++)
    for(int j=0;j<(ForceTime);j++)
        F_total[i][j] = 0;

    //Malloc and Initialize for each GPU
    for(int n = 0; n<GPU_N; n++){
        f_h    [n] = (float *)malloc(XDIM*YDIM*zInner*19*sizeof(float));
        g_h    [n] = (float *)malloc(XDIM*YDIM*       19*sizeof(float));
        h_h    [n] = (float *)malloc(XDIM*YDIM*       19*sizeof(float));
        for(int i = 0; i<3; i++){
        F_h      [n][i] = (float *)malloc(ForceTime*sizeof(float));
        velAv_h  [n][i] = (float *)malloc(XDIM*YDIM*ZDIM/GPU_N*sizeof(float));
        velFluc_h[n][i] = (float *)malloc(XDIM*YDIM*ZDIM/GPU_N*sizeof(float));
        }
        hipSetDevice(n);
        hipStreamCreate(&stream_halo[n]);
        hipStreamCreate(&stream_inner[n]);
        for(int m = 0; m<GPU_N; m++)
            if(m != n) hipDeviceEnablePeerAccess(m,0);

        for(int i = 0; i<2; i++){
           hipMalloc((void **) &f_d[n][i], pitch_e*YDIM*zInner*19*sizeof(float));
        hipMalloc((void **) &g_d[n][i], pitch_e*YDIM*       19*sizeof(float));
        hipMalloc((void **) &h_d[n][i], pitch_e*YDIM*       19*sizeof(float));
        }
        hipMalloc((void **) & g_temp[n], pitch_e*YDIM*       19*sizeof(float));
        hipMalloc((void **) & h_temp[n], pitch_e*YDIM*       19*sizeof(float));
        for(int i = 0; i<3; i++){
        hipMalloc((void **) & F_d      [n][i], (ForceTime)*sizeof(float));
        hipMalloc((void **) & velAv_d  [n][i], pitch_e*YDIM*ZDIM/GPU_N*sizeof(float));
        hipMalloc((void **) & velFluc_d[n][i], pitch_e*YDIM*ZDIM/GPU_N*sizeof(float));
        }

        //initialize host f_inner
        for (int i = 0; i < XDIM*YDIM*zInner*19; i++)
            f_h[n][i] = 0;
        //initialize host g,h
        for (int i = 0; i < XDIM*YDIM*19; i++){
            g_h[n][i] = 0;
            h_h[n][i] = 0;
        }
        for(int i=0;i<3;i++){
            for(int j=0;j<(ForceTime);j++)
                F_h[n][i][j] = 0;
            for (int j = 0; j < XDIM*YDIM*ZDIM/GPU_N; j++){
            velAv_h  [n][i][j] = 0;
            velFluc_h[n][i][j] = 0;
            }
        }
        for(int i = 0; i<2; i++){
        hipMemcpy2D(f_d[n][i],pitch,f_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyHostToDevice);
        hipMemcpy2D(g_d[n][i],pitch,g_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM       *19,hipMemcpyHostToDevice);
        hipMemcpy2D(h_d[n][i],pitch,h_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM       *19,hipMemcpyHostToDevice);
        }

        for(int i = 0; i<3; i++){
        hipMemcpy2D(velAv_d  [n][i],pitch,velAv_h  [n][i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM/GPU_N,hipMemcpyHostToDevice);
        hipMemcpy2D(velFluc_d[n][i],pitch,velFluc_h[n][i],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM/GPU_N,hipMemcpyHostToDevice);
        hipMemcpy(F_d[n][i],F_h[n][i],sizeof(float)*(ForceTime),hipMemcpyHostToDevice);
        }

        //initialization kernels
        for(int i = 0; i<2; i++){
        initialize<<<  grid,threads>>>(f_d[n][i],pitch_e,zInner,GPU_N);
        initialize<<<g_grid,threads>>>(g_d[n][i],pitch_e,     1,GPU_N);
        initialize<<<g_grid,threads>>>(h_d[n][i],pitch_e,     1,GPU_N);
        }
        initialize<<<g_grid,threads>>>(g_temp[n],pitch_e,     1,GPU_N);
        initialize<<<g_grid,threads>>>(h_temp[n],pitch_e,     1,GPU_N);

    }//end Malloc and Initialize
    

    //data pointers as 3D array (GPUxCoord)
    float   *f_LR_h[GPU_N],   *g_LR_h[GPU_N],   *h_LR_h[GPU_N];
    float *f_LR_d[GPU_N][2], *g_LR_d[GPU_N][2], *h_LR_d[GPU_N][2];
    float *g_LR_temp[GPU_N], *h_LR_temp[GPU_N];
    float *velAv_LR_h[GPU_N][3],*velFluc_LR_h[GPU_N][3];
    float *velAv_LR_d[GPU_N][3],*velFluc_LR_d[GPU_N][3];

    size_t pitchLR = 2;
    while(pitchLR<XLRDIM)
        pitchLR=pitchLR*2;
    pitchLR = pitchLR*sizeof(float);
    size_t pitchLR_e = pitchLR/sizeof(float);
    cout<<"LR Pitch (in elements): "<<pitchLR_e<<endl;

    int zLRInner = ZLRDIM/GPU_N-2;
    dim3 LR_threads(BLOCKSIZELRX, BLOCKSIZELRY, BLOCKSIZELRZ);
    dim3 LR_grid(((XLRDIM+BLOCKSIZELRX-1)/BLOCKSIZELRX),((YLRDIM+BLOCKSIZELRY-1)/BLOCKSIZELRY),(zLRInner)/BLOCKSIZELRZ);
    dim3 g_LR_grid(((XLRDIM+BLOCKSIZELRX-1)/BLOCKSIZELRX),((YLRDIM+BLOCKSIZELRY-1)/BLOCKSIZELRY),1);    
    
    //setup LR
    if(REFINEMENT == 1){

    for(int n = 0; n<GPU_N; n++){
        f_LR_h    [n] = (float *)malloc(XLRDIM*YLRDIM*zLRInner*19*sizeof(float));
        g_LR_h    [n] = (float *)malloc(XLRDIM*YLRDIM*         19*sizeof(float));
        h_LR_h    [n] = (float *)malloc(XLRDIM*YLRDIM*         19*sizeof(float));
        for(int i = 0; i<3; i++){
        velAv_LR_h  [n][i] = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM/GPU_N*sizeof(float));
        velFluc_LR_h[n][i] = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM/GPU_N*sizeof(float));
        }
        hipSetDevice(n);
        for(int i = 0; i<2; i++){
        hipMalloc((void **) &f_LR_d[n][i], pitchLR_e*YLRDIM*zLRInner*19*sizeof(float));
        hipMalloc((void **) &g_LR_d[n][i], pitchLR_e*YLRDIM*         19*sizeof(float));
        hipMalloc((void **) &h_LR_d[n][i], pitchLR_e*YLRDIM*         19*sizeof(float));
        }
        hipMalloc((void **) & g_LR_temp[n], pitchLR_e*YLRDIM*       19*sizeof(float));
        hipMalloc((void **) & h_LR_temp[n], pitchLR_e*YLRDIM*       19*sizeof(float));
        for(int i = 0; i<3; i++){
        hipMalloc((void **) & velAv_LR_d  [n][i], pitchLR_e*YLRDIM*ZLRDIM/GPU_N*sizeof(float));
        hipMalloc((void **) & velFluc_LR_d[n][i], pitchLR_e*YLRDIM*ZLRDIM/GPU_N*sizeof(float));
        }
        for (int i = 0; i < XLRDIM*YLRDIM*zLRInner*19; i++)
            f_LR_h[n][i] = 0;
        //initialize host g,h
        for (int i = 0; i < XLRDIM*YLRDIM*19; i++){
            g_LR_h[n][i] = 0;
            h_LR_h[n][i] = 0;
        }
        for(int i=0;i<3;i++){
            for (int j = 0; j < XLRDIM*YLRDIM*ZLRDIM/GPU_N; j++){
            velAv_LR_h  [n][i][j] = 0;
            velFluc_LR_h[n][i][j] = 0;
            }
        }
        for(int i = 0; i<2; i++){
        hipMemcpy2D(f_LR_d[n][i],pitchLR,f_LR_h[n],XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*zLRInner*19,hipMemcpyHostToDevice);
        hipMemcpy2D(g_LR_d[n][i],pitchLR,g_LR_h[n],XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM         *19,hipMemcpyHostToDevice);
        hipMemcpy2D(h_LR_d[n][i],pitchLR,h_LR_h[n],XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM         *19,hipMemcpyHostToDevice);
        }
        for(int i = 0; i<3; i++){
        hipMemcpy2D(velAv_LR_d  [n][i],pitchLR,velAv_LR_h  [n][i],XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*ZLRDIM/GPU_N,hipMemcpyHostToDevice);
        hipMemcpy2D(velFluc_LR_d[n][i],pitchLR,velFluc_LR_h[n][i],XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*ZLRDIM/GPU_N,hipMemcpyHostToDevice);
        }
        //initialization kernels
        for(int i = 0; i<2; i++){
        initializeLR<<<  LR_grid,LR_threads>>>(f_LR_d[n][i],pitchLR_e,zLRInner,GPU_N);
        initializeLR<<<g_LR_grid,LR_threads>>>(g_LR_d[n][i],pitchLR_e,       1,GPU_N);
        initializeLR<<<g_LR_grid,LR_threads>>>(h_LR_d[n][i],pitchLR_e,       1,GPU_N);
        }
        initializeLR<<<g_LR_grid,LR_threads>>>(g_LR_temp[n],pitchLR_e,       1,GPU_N);
        initializeLR<<<g_LR_grid,LR_threads>>>(h_LR_temp[n],pitchLR_e,       1,GPU_N);



    }//end of GPU loop for malloc and initialize for LR
    }//end of LR malloc and initialize


    int A = 0; int B = 1; int C = 0; int D = 1;
    
    struct timeval tdr0,tdr1;
    double restime;
    hipDeviceSynchronize();
    gettimeofday (&tdr0,NULL);

    for(int n = 0; n<GPU_N; n++){
    hipSetDevice(n);
    size_t mem_avail, mem_total;
    hipMemGetInfo(&mem_avail,&mem_total);
    cout<<"Device memory used for dev"<<n<<" : "<<(mem_total-mem_avail)*pow(10,-9)<<" GB\n";
    cout<<"Device memory available for dev"<<n<<" : "<<(mem_avail)*pow(10,-9)<<" GB\n";
    }
    
    //time loop
    for(int t = 0; t<TMAX; t++)
    {
        for(int n = 0; n<GPU_N; n++)
        {
              hipSetDevice(n);
            update_top<<<g_grid, threads, 0, stream_halo [n]>>>(h_d[n][B],h_d[n][A],f_d[n][A],h_temp[n],omega,pitch_e,n,zInner,F_d[n][0],F_d[n][1],F_d[n][2],t,!REFINEMENT);
            update_bot<<<g_grid, threads, 0, stream_halo [n]>>>(g_d[n][B],g_d[n][A],f_d[n][A],g_temp[n],omega,pitch_e,n,zInner,F_d[n][0],F_d[n][1],F_d[n][2],t,!REFINEMENT);
        }
        for(int n = 0; n<GPU_N; n++)
        hipStreamSynchronize(stream_halo[n]);
        for(int n = 0; n<GPU_N; n++){
            hipSetDevice(n);
            update_inn<<<grid,threads,0,stream_inner[n]>>>(f_d[n][B],f_d[n][A],g_d[n][A], h_d[n][A],omega,pitch_e,n,zInner,velAv_d[n],velFluc_d[n],F_d[n][0],F_d[n][1],F_d[n][2],t,!REFINEMENT);
        }

        for(int n = 0; n<GPU_N; n++)
            hipMemcpyPeerAsync(&h_temp[n][0],n,&g_d[   (n+1)%GPU_N][B][0],   (n+1)%GPU_N,pitch_e*YDIM*sizeof(float)*19,stream_halo[n]);
        for(int n = 0; n<GPU_N; n++)
            hipMemcpyPeerAsync(&g_temp[n][0],n,&h_d[abs(n-1)%GPU_N][B][0],abs(n-1)%GPU_N,pitch_e*YDIM*sizeof(float)*19,stream_halo[n]);

        hipDeviceSynchronize();
        swap(A,B);



        if(REFINEMENT == 1){
        int flag_F = 0;
        if(t>STARTF) flag_F = 1;
        for(int i = 0; i<LRLEVEL; i++){
            for(int n = 0; n<GPU_N; n++){
            hipSetDevice(n);
            update_top_LR<<<g_LR_grid,LR_threads,0,stream_halo[n]>>>(h_LR_d[n][D],h_LR_d[n][C],f_LR_d[n][C],h_LR_temp[n],omegaLR,pitchLR_e,n,zLRInner,F_d[n][0],F_d[n][1],F_d[n][2],t,flag_F);
            update_bot_LR<<<g_LR_grid,LR_threads,0,stream_halo[n]>>>(g_LR_d[n][D],g_LR_d[n][C],f_LR_d[n][C],g_LR_temp[n],omegaLR,pitchLR_e,n,zLRInner,F_d[n][0],F_d[n][1],F_d[n][2],t,flag_F);
            }
            for(int n = 0; n<GPU_N; n++)
    		hipStreamSynchronize(stream_halo[n]);
        
            for(int n = 0; n<GPU_N; n++){
    		hipSetDevice(n);
            update_inn_LR<<<LR_grid,LR_threads,0,stream_inner[n]>>>(f_LR_d[n][D],f_LR_d[n][C],g_LR_d[n][C], h_LR_d[n][C],omegaLR,pitchLR_e,n,zLRInner,velAv_LR_d[n],velFluc_LR_d[n],F_d[n][0],F_d[n][1],F_d[n][2],t,flag_F);
            }

            for(int n = 0; n<GPU_N; n++){
            hipMemcpyPeerAsync(&h_LR_temp[n][pitchLR_e*YLRDIM*14],n,&g_LR_d[   (n+1)%GPU_N][D][pitchLR_e*YLRDIM*14],   (n+1)%GPU_N,pitchLR_e*YLRDIM*sizeof(float)*5,stream_halo[n]);
            hipMemcpyPeerAsync(&g_LR_temp[n][pitchLR_e*YLRDIM*9 ],n,&h_LR_d[abs(n-1)%GPU_N][D][pitchLR_e*YLRDIM*9 ],abs(n-1)%GPU_N,pitchLR_e*YLRDIM*sizeof(float)*5,stream_halo[n]);
            }
   
            for(int n = 0; n<GPU_N; n++){
    		hipSetDevice(n);
    		hipDeviceSynchronize();
            }
            flag_F = 0;
            swap(C,D);
        }
        }








    }


    hipDeviceSynchronize();
    gettimeofday (&tdr1,NULL);
    timeval_subtract (&restime, &tdr1, &tdr0);
    int Nodes;
    Nodes = XDIM*YDIM*ZDIM;
    if (REFINEMENT == 1)
        Nodes += XLRDIM*YLRDIM*ZLRDIM*LRLEVEL;
    cout<<"Time taken for main kernel: "<<restime<<" ("
            <<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)\n";

    //D2H Memcpy and write results
    for(int n = 0; n<GPU_N; n++){
        hipSetDevice(n);
        hipMemcpy2D(f_h[n],XDIM*sizeof(float),f_d[n][A],pitch,XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyDeviceToHost);
        hipMemcpy2D(g_h[n],XDIM*sizeof(float),g_d[n][A],pitch,XDIM*sizeof(float),YDIM       *19,hipMemcpyDeviceToHost);
        hipMemcpy2D(h_h[n],XDIM*sizeof(float),h_d[n][A],pitch,XDIM*sizeof(float),YDIM       *19,hipMemcpyDeviceToHost);
        for(int i = 0; i<3; i++){
        hipMemcpy2D(  velAv_h[n][i],XDIM*sizeof(float),velAv_d[n][i],pitch,XDIM*sizeof(float),YDIM*ZDIM/GPU_N,hipMemcpyDeviceToHost);
        hipMemcpy2D(velFluc_h[n][i],XDIM*sizeof(float),velAv_d[n][i],pitch,XDIM*sizeof(float),YDIM*ZDIM/GPU_N,hipMemcpyDeviceToHost);
        hipMemcpy(F_h[n][i],F_d[n][i],sizeof(float)*ForceTime,hipMemcpyDeviceToHost);
        }
        WriteResults(output,f_h[n],g_h[n],h_h[n],velAv_h[n],velFluc_h[n],omega,GPU_N,n);
        output<<endl;

        for(int i=0;i<3;i++)
        for(int j=0;j<ForceTime;j++)
            F_total[i][j] += F_h[n][i][j];
        
        for(int i = 0; i<2; i++){
            hipFree(f_d[n][i]);
            hipFree(g_d[n][i]);
            hipFree(h_d[n][i]);
        }
        hipFree(f_d[n]);
        hipFree(g_d[n]);
        hipFree(h_d[n]);
        hipFree(g_temp[n]);
        hipFree(h_temp[n]);
        for(int i=0;i<3;i++)
            hipFree(F_d[n][i]);
        hipFree(F_d[n]);
    }//end Memcpy and write results
    WriteForces(F_total,outputForce,ForceTime,REFINEMENT*LRLEVEL);


    if(REFINEMENT == 1){
    output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\",\"uAv\",\"vAv\",\"ufluc\",\"vfluc\"\n";
    output<<"ZONE F=POINT, I="<<XLRDIM<<", J="<<YLRDIM<<", K="<<ZLRDIM<<"\n";
    for(int n = 0; n<GPU_N; n++){
        hipSetDevice(n);
        hipMemcpy2D(f_LR_h[n],XLRDIM*sizeof(float),f_LR_d[n][C],pitchLR,XLRDIM*sizeof(float),YLRDIM*zLRInner*19,hipMemcpyDeviceToHost);
        hipMemcpy2D(g_LR_h[n],XLRDIM*sizeof(float),g_LR_d[n][C],pitchLR,XLRDIM*sizeof(float),YLRDIM         *19,hipMemcpyDeviceToHost);
        hipMemcpy2D(h_LR_h[n],XLRDIM*sizeof(float),h_LR_d[n][C],pitchLR,XLRDIM*sizeof(float),YLRDIM         *19,hipMemcpyDeviceToHost);
        for(int i = 0; i<3; i++){
        hipMemcpy2D(  velAv_LR_h[n][i],XLRDIM*sizeof(float),velAv_LR_d[n][i],pitchLR,XLRDIM*sizeof(float),YLRDIM*ZLRDIM/GPU_N,hipMemcpyDeviceToHost);
        hipMemcpy2D(velFluc_LR_h[n][i],XLRDIM*sizeof(float),velAv_LR_d[n][i],pitchLR,XLRDIM*sizeof(float),YLRDIM*ZLRDIM/GPU_N,hipMemcpyDeviceToHost);
        }
        WriteResultsLR(output,f_LR_h[n],g_LR_h[n],h_LR_h[n],velAv_LR_h[n],velFluc_LR_h[n],omegaLR,GPU_N,n);
        output<<endl;

        for(int i = 0; i<2; i++){
            hipFree(f_LR_d[n][i]);
            hipFree(g_LR_d[n][i]);
            hipFree(h_LR_d[n][i]);
        }
        hipFree(f_LR_d[n]);
        hipFree(g_LR_d[n]);
        hipFree(h_LR_d[n]);
        hipFree(g_LR_temp[n]);
        hipFree(h_LR_temp[n]);
    }
    }


    return 0;
}
