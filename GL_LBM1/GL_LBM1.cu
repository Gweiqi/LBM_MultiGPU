#include "hip/hip_runtime.h"
// OpenGL Graphics includes
#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// Includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// CUDA standard includes
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

// CUDA FFT Libraries
#include <hipfft/hipfft.h>

// CUDA helper functions
#include <sdkHelper.h>
#include <rendercheck_gl.h>
#include <cudaHelper.h>
#include <cudaGLHelper.h>

// Shared Library Test Functions
#include <shrQATest.h>

#ifdef WIN32
     bool IsOpenGLAvailable(const char *appName) { return true; }
#else
  #if (defined(__APPLE__) || defined(MACOSX))
     bool IsOpenGLAvailable(const char *appName) { return true; }
  #else
     // check if this is a linux machine
     #include <X11/Xlib.h>

     bool IsOpenGLAvailable(const char *appName)
     {
        Display *Xdisplay = XOpenDisplay(NULL);
        if (Xdisplay == NULL) {
           return false;
        } else {
           XCloseDisplay(Xdisplay);
           return true;
        }
     }
  #endif
#endif

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

//#include "defines.h"
//#include "fluidsGL_kernels.h"

#define MAX_EPSILON_ERROR 1.0f

const char *sSDKname = "GL_LBM1";

// Define the files that are to be save and the reference images for validation
const char *sOriginal[] =
{
    "GL_LBM1.ppm",
    NULL
};

const char *sReference[] =
{
    "ref_GL_LBM1.ppm",
    NULL
};

#define getmin(a,b) (a < b ? a : b)
#define getmax(a,b) (a > b ? a : b)






#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <fstream>
#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;






//#define BLOCKSIZE 16;
int const BLOCKSIZE = 16;
//int const XDIM = 32;
//int const YDIM = 32;

int Xcoord(int i, int XDim){
	int B = BLOCKSIZE*BLOCKSIZE;
	return (i%B)%BLOCKSIZE+((i/B)%(XDim/BLOCKSIZE))*BLOCKSIZE;
}
int Ycoord(int i, int XDim){
	int B = BLOCKSIZE*BLOCKSIZE;
	return (i%B)/BLOCKSIZE+((i/B)/(XDim/BLOCKSIZE))*BLOCKSIZE;
}

__global__ void mrt_d(float *f0, float *f1, float *f2,
							float *f3, float *f4, float *f5,
							float *f6, float *f7, float *f8,
							int n, int *image, float omega, float uMax)
{
	int i,m,j,k;
	j = threadIdx.y*blockDim.x+threadIdx.x;//local block index (for shared mem)
	i = j+blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	i += blockDim.x*blockDim.y*blockIdx.x;//global memory index
	m = blockDim.x*blockDim.y;
	//m = BLOCKSIZE*BLOCKSIZE;
	
	__shared__ float f0_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f1_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f2_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f3_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f4_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f5_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f6_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f7_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f8_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ int image_s[(BLOCKSIZE)*(BLOCKSIZE)+1];

	f0_s[j] = f0[i];
	f1_s[j] = f1[i];
	f2_s[j] = f2[i];
	f3_s[j] = f3[i];
	f4_s[j] = f4[i];
	f5_s[j] = f5[i];
	f6_s[j] = f6[i];
	f7_s[j] = f7[i];
	f8_s[j] = f8[i];
	image_s[j] = image[i];
	
	//1
	if(threadIdx.x == blockDim.x-1 && blockIdx.x != 0){
		f1_s[j] = f1[i-m];
		f5_s[j] = f5[i-m];
		f8_s[j] = f8[i-m];
	}
	//3
	if(threadIdx.x == 0 && blockIdx.x != gridDim.x-1){
		f3_s[j] = f3[i+m];
		f7_s[j] = f7[i+m];
		f6_s[j] = f6[i+m];
	}
	//2
	if(threadIdx.y == blockDim.y-1 && blockIdx.y != 0){
		f2_s[j] = f2[i-m*gridDim.x];
		f5_s[j] = f5[i-m*gridDim.x];
		f6_s[j] = f6[i-m*gridDim.x];
	}
	//4
	if(threadIdx.y == 0 && blockIdx.y != gridDim.y-1){
		f4_s[j] = f4[i+m*gridDim.x];
		f7_s[j] = f7[i+m*gridDim.x];
		f8_s[j] = f8[i+m*gridDim.x];
	}
	//5
	if(threadIdx.y == blockDim.y-1 && threadIdx.x == blockDim.x-1 &&
		blockIdx.x != 0 && blockIdx.y != 0)//upper corner
		f5_s[j] = f5[i-m*(1+gridDim.x)];
	//7
	if(threadIdx.y == 0 && threadIdx.x == 0 &&
		blockIdx.x != gridDim.x-1 && blockIdx.y != gridDim.y-1)
		f7_s[j] = f7[i+m*(1+gridDim.x)];
	//6
	if(threadIdx.y == blockDim.y-1 && threadIdx.x == 0 &&
		blockIdx.x != gridDim.x-1 && blockIdx.y != 0)
		f6_s[j] = f6[i-m*(gridDim.x-1)];
	//8
	if(threadIdx.y == 0 && threadIdx.x == blockDim.x-1 &&
		blockIdx.x != 0 && blockIdx.y != gridDim.y-1)
		f8_s[j] = f8[i+m*(gridDim.x-1)];


	__syncthreads();

	//STREAM
	//j = (i+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	//j = (i+1)-blockDim.x*(threadIdx.x==blockDim.x-1);//equiv to (i+1)%(m), if m is pow of 2;
	if((j+1)%(blockDim.x)==0) k=j-blockDim.x+1;
	else k=j+1;
	float temp = f3_s[k];
	f3_s[k]=f1_s[j];
	f1_s[j]=temp;
	k = (j+blockDim.x) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f4_s[k];
	f4_s[k]=f2_s[j];
	f2_s[j]=temp;
	if((j+1)%(blockDim.x)==0) k=(j+1) & (m-1);
	else k = (j+blockDim.x+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f7_s[k];
	f7_s[k]=f5_s[j];
	f5_s[j]=temp;
	if((j)%(blockDim.x)==0) k=(j+2*blockDim.x-1) & (m-1);
	else k = (j+blockDim.x-1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f8_s[k];
	f8_s[k]=f6_s[j];
	f6_s[j]=temp;

	__syncthreads();

	if(image_s[j] != 1){
	temp = f3_s[j];
	f3_s[j]=f1_s[j];
	f1_s[j]=temp;
	temp = f4_s[j];
	f4_s[j]=f2_s[j];
	f2_s[j]=temp;
	temp = f7_s[j];
	f7_s[j]=f5_s[j];
	f5_s[j]=temp;
	temp = f8_s[j];
	f8_s[j]=f6_s[j];
	f6_s[j]=temp;

	
	//float u,v,rho,feq,usqr;
	float u,v,rho,usqr;
	float m1,m2,m4,m6,m7,m8;
	float meq1,meq2,meq4,meq6,meq7,meq8;
	rho = f0_s[j]+f1_s[j]+f2_s[j]+f3_s[j]+f4_s[j]+f5_s[j]+f6_s[j]+f7_s[j]+f8_s[j];
	u = f1_s[j]-f3_s[j]+f5_s[j]-f6_s[j]-f7_s[j]+f8_s[j];
	v = f2_s[j]-f4_s[j]+f5_s[j]+f6_s[j]-f7_s[j]-f8_s[j];
	if(image_s[j] == 2){
		u = 0.0f;
		v = uMax;
		rho = u+(f0_s[j]+f2_s[j]+f4_s[j]+2.0f*f3_s[j]+2.0f*f6_s[j]+2.0f*f7_s[j]);
		f1_s[j] = f3_s[j]+4.0f*u/6.0f;
        f5_s[j] = f7_s[j]-0.5f*(f2_s[j]-f4_s[j])+v*0.5f+u/6.0f;
        f8_s[j] = f6_s[j]+0.5f*(f2_s[j]-f4_s[j])-v*0.5f+u/6.0f;	
	}
	else if(image_s[j] == 3){//north
		u = uMax;
		v = 0.0f;
		rho = -v+(f0_s[j]+f1_s[j]+f3_s[j]+2.0f*f6_s[j]+2.0f*f2_s[j]+2.0f*f5_s[j]);
		f4_s[j] = f2_s[j]-4.0f*v/6.0f;
        f7_s[j] = f5_s[j]+0.5f*(f1_s[j]-f3_s[j])-u*0.5f+v/6.0f;
        f8_s[j] = f6_s[j]-0.5f*(f1_s[j]-f3_s[j])+u*0.5f+v/6.0f;	
	}
	usqr = u*u+v*v;

		meq1 = -2.0f*rho+3.0f*usqr;//e_eq (uses rho, Yu)
		meq2 = rho-3.0f*usqr; //epsilon_eq (uses rho, Yu)
		meq4 = -u;//qx_eq
		meq6 = -v;//qy_eq
		meq7 = u*u-v*v;//pxx_eq
		meq8 = u*v;//pxy_eq

		m1 =-4.f*f0_s[j]+-1.f*f1_s[j]+-1.f*f2_s[j]+-1.f*f3_s[j]+-1.f*f4_s[j]+ 2.f*f5_s[j]+ 2.f*f6_s[j]+ 2.f*f7_s[j]+ 2.f*f8_s[j];
		m2 = 4.f*f0_s[j]+-2.f*f1_s[j]+-2.f*f2_s[j]+-2.f*f3_s[j]+-2.f*f4_s[j]+ 1.f*f5_s[j]+ 1.f*f6_s[j]+ 1.f*f7_s[j]+ 1.f*f8_s[j];
		m4 =             -2.f*f1_s[j]             + 2.f*f3_s[j]             + 1.f*f5_s[j]+-1.f*f6_s[j]+-1.f*f7_s[j]+ 1.f*f8_s[j];
		m6 =                          -2.f*f2_s[j]             + 2.f*f4_s[j]+ 1.f*f5_s[j]+ 1.f*f6_s[j]+-1.f*f7_s[j]+-1.f*f8_s[j];
		m7 =              1.f*f1_s[j]+-1.f*f2_s[j]+ 1.f*f3_s[j]+-1.f*f4_s[j]                                                    ;
		m8 =                                                                  1.f*f5_s[j]+-1.f*f6_s[j]+ 1.f*f7_s[j]+-1.f*f8_s[j];

		float Q11 = (m7-meq7)*0.5f;
		float Q22 = -Q11;
		float Q12 = m8-meq8;
		float Q21 = Q12;
		float Q = Q11*Q11+Q12*Q12+Q21*Q21+Q22*Q22;
	
		float Cs = 0.01f;
		float nu0 = ((1.0f/omega)-0.5f)/3.0f;
		float Smag = (sqrt((2.0f*rho/(3.0f*omega))*(2.0f*rho/(3.0f*omega))
				+4.0f*2.0f*rho*Cs*sqrt(Q))-2.0f*rho/(3.0f*omega))/(4.0f*rho*Cs);
		float OmegaStar = 1.0f/(3.0f*(nu0+Cs*Smag)+0.5f);

f0_s[j]-=-4.f*(m1-meq1)/36.0f+4.f *(m2-meq2)/36.0f;//+0   *(m4-meq4)/12.0f+0   *(m6-meq6)/12.0f;
f1_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+-2.f*(m4-meq4)/12.0f                     +   (m7-meq7)*OmegaStar/4.0f;
f2_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +-2.f*(m6-meq6)/12.0f+-  (m7-meq7)*OmegaStar/4.0f;
f3_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+2.f *(m4-meq4)/12.0f                     +   (m7-meq7)*OmegaStar/4.0f;
f4_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +2.f *(m6-meq6)/12.0f+-  (m7-meq7)*OmegaStar/4.0f;
f5_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+     (m6-meq6)/12.0f+   (m8-meq8)*OmegaStar/4.0f;
f6_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+     (m6-meq6)/12.0f+-  (m8-meq8)*OmegaStar/4.0f;
f7_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+   (m8-meq8)*OmegaStar/4.0f;
f8_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+-  (m8-meq8)*OmegaStar/4.0f;	

//f0_s[j]-=-4.f*(m1-meq1)/36.0f+4.f *(m2-meq2)/36.0f;//+0   *(m4-meq4)/12.0f+0   *(m6-meq6)/12.0f;
//f1_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+-2.f*(m4-meq4)/12.0f                     +   (m7-meq7)*omega/4.0f;
//f2_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +-2.f*(m6-meq6)/12.0f+-  (m7-meq7)*omega/4.0f;
//f3_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+2.f *(m4-meq4)/12.0f                     +   (m7-meq7)*omega/4.0f;
//f4_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +2.f *(m6-meq6)/12.0f+-  (m7-meq7)*omega/4.0f;
//f5_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+     (m6-meq6)/12.0f+   (m8-meq8)*omega/4.0f;
//f6_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+     (m6-meq6)/12.0f+-  (m8-meq8)*omega/4.0f;
//f7_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+   (m8-meq8)*omega/4.0f;
//f8_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+-  (m8-meq8)*omega/4.0f;	

	}

	__syncthreads();
	f0[i] = f0_s[j];
	f1[i] = f1_s[j];
	f2[i] = f2_s[j];
	f3[i] = f3_s[j];
	f4[i] = f4_s[j];
	f5[i] = f5_s[j];
	f6[i] = f6_s[j];
	f7[i] = f7_s[j];
	f8[i] = f8_s[j];


}

__global__ void stream1_d(float *f0, float *f1, float *f2,
							float *f3, float *f4, float *f5,
							float *f6, float *f7, float *f8,
							int n, int *image, float omega, float uMax)
{
	int i,m,j,k;
	j = threadIdx.y*blockDim.x+threadIdx.x;//local block index (for shared mem)
	i = j+blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	i += blockDim.x*blockDim.y*blockIdx.x;//global memory index
	m = blockDim.x*blockDim.y;
	//m = BLOCKSIZE*BLOCKSIZE;
	
	__shared__ float f0_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f1_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f2_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f3_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f4_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f5_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f6_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f7_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f8_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ int image_s[(BLOCKSIZE)*(BLOCKSIZE)+1];

	f0_s[j] = f0[i];
	f1_s[j] = f1[i];
	f2_s[j] = f2[i];
	f3_s[j] = f3[i];
	f4_s[j] = f4[i];
	f5_s[j] = f5[i];
	f6_s[j] = f6[i];
	f7_s[j] = f7[i];
	f8_s[j] = f8[i];
	image_s[j] = image[i];
	
	//1
	if(threadIdx.x == blockDim.x-1 && blockIdx.x != 0){
		f1_s[j] = f1[i-m];
		f5_s[j] = f5[i-m];
		f8_s[j] = f8[i-m];
	}
	//3
	if(threadIdx.x == 0 && blockIdx.x != gridDim.x-1){
		f3_s[j] = f3[i+m];
		f7_s[j] = f7[i+m];
		f6_s[j] = f6[i+m];
	}
	//2
	if(threadIdx.y == blockDim.y-1 && blockIdx.y != 0){
		f2_s[j] = f2[i-m*gridDim.x];
		f5_s[j] = f5[i-m*gridDim.x];
		f6_s[j] = f6[i-m*gridDim.x];
	}
	//4
	if(threadIdx.y == 0 && blockIdx.y != gridDim.y-1){
		f4_s[j] = f4[i+m*gridDim.x];
		f7_s[j] = f7[i+m*gridDim.x];
		f8_s[j] = f8[i+m*gridDim.x];
	}
	//5
	if(threadIdx.y == blockDim.y-1 && threadIdx.x == blockDim.x-1 &&
		blockIdx.x != 0 && blockIdx.y != 0)//upper corner
		f5_s[j] = f5[i-m*(1+gridDim.x)];
	//7
	if(threadIdx.y == 0 && threadIdx.x == 0 &&
		blockIdx.x != gridDim.x-1 && blockIdx.y != gridDim.y-1)
		f7_s[j] = f7[i+m*(1+gridDim.x)];
	//6
	if(threadIdx.y == blockDim.y-1 && threadIdx.x == 0 &&
		blockIdx.x != gridDim.x-1 && blockIdx.y != 0)
		f6_s[j] = f6[i-m*(gridDim.x-1)];
	//8
	if(threadIdx.y == 0 && threadIdx.x == blockDim.x-1 &&
		blockIdx.x != 0 && blockIdx.y != gridDim.y-1)
		f8_s[j] = f8[i+m*(gridDim.x-1)];


	__syncthreads();

	//STREAM
	//j = (i+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	//j = (i+1)-blockDim.x*(threadIdx.x==blockDim.x-1);//equiv to (i+1)%(m), if m is pow of 2;
	if((j+1)%(blockDim.x)==0) k=j-blockDim.x+1;
	else k=j+1;
	float temp = f3_s[k];
	f3_s[k]=f1_s[j];
	f1_s[j]=temp;
	k = (j+blockDim.x) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f4_s[k];
	f4_s[k]=f2_s[j];
	f2_s[j]=temp;
	if((j+1)%(blockDim.x)==0) k=(j+1) & (m-1);
	else k = (j+blockDim.x+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f7_s[k];
	f7_s[k]=f5_s[j];
	f5_s[j]=temp;
	if((j)%(blockDim.x)==0) k=(j+2*blockDim.x-1) & (m-1);
	else k = (j+blockDim.x-1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f8_s[k];
	f8_s[k]=f6_s[j];
	f6_s[j]=temp;

	__syncthreads();

	if(image_s[j] != 1){
	temp = f3_s[j];
	f3_s[j]=f1_s[j];
	f1_s[j]=temp;
	temp = f4_s[j];
	f4_s[j]=f2_s[j];
	f2_s[j]=temp;
	temp = f7_s[j];
	f7_s[j]=f5_s[j];
	f5_s[j]=temp;
	temp = f8_s[j];
	f8_s[j]=f6_s[j];
	f6_s[j]=temp;

	
	float u,v,rho,feq,usqr;
	rho = f0_s[j]+f1_s[j]+f2_s[j]+f3_s[j]+f4_s[j]+f5_s[j]+f6_s[j]+f7_s[j]+f8_s[j];
	u = f1_s[j]-f3_s[j]+f5_s[j]-f6_s[j]-f7_s[j]+f8_s[j];
	v = f2_s[j]-f4_s[j]+f5_s[j]+f6_s[j]-f7_s[j]-f8_s[j];
	if(image_s[j] == 2){
		u = 0.0f;
		v = uMax;
		rho = u+(f0_s[j]+f2_s[j]+f4_s[j]+2.0f*f3_s[j]+2.0f*f6_s[j]+2.0f*f7_s[j]);
		f1_s[j] = f3_s[j]+4.0f*u/6.0f;
        f5_s[j] = f7_s[j]-0.5f*(f2_s[j]-f4_s[j])+v*0.5f+u/6.0f;
        f8_s[j] = f6_s[j]+0.5f*(f2_s[j]-f4_s[j])-v*0.5f+u/6.0f;	
	}
	else if(image_s[j] == 3){//north
		u = uMax;
		v = 0.0f;
		rho = -v+(f0_s[j]+f1_s[j]+f3_s[j]+2.0f*f6_s[j]+2.0f*f2_s[j]+2.0f*f5_s[j]);
		f4_s[j] = f2_s[j]-4.0f*v/6.0f;
        f7_s[j] = f5_s[j]+0.5f*(f1_s[j]-f3_s[j])-u*0.5f+v/6.0f;
        f8_s[j] = f6_s[j]-0.5f*(f1_s[j]-f3_s[j])+u*0.5f+v/6.0f;	
	}
	usqr = u*u+v*v;
	feq = 4.0f/9.0f*(rho-1.5f*usqr);
	f0_s[j] = f0_s[j]-omega*(f0_s[j]-feq);
	feq = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1_s[j] = f1_s[j]-omega*(f1_s[j]-feq);
	feq = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2_s[j] = f2_s[j]-omega*(f2_s[j]-feq);
	feq = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3_s[j] = f3_s[j]-omega*(f3_s[j]-feq);
	feq = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4_s[j] = f4_s[j]-omega*(f4_s[j]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5_s[j] = f5_s[j]-omega*(f5_s[j]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6_s[j] = f6_s[j]-omega*(f6_s[j]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7_s[j] = f7_s[j]-omega*(f7_s[j]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8_s[j] = f8_s[j]-omega*(f8_s[j]-feq);
	}

	__syncthreads();
	f0[i] = f0_s[j];
	f1[i] = f1_s[j];
	f2[i] = f2_s[j];
	f3[i] = f3_s[j];
	f4[i] = f4_s[j];
	f5[i] = f5_s[j];
	f6[i] = f6_s[j];
	f7[i] = f7_s[j];
	f8[i] = f8_s[j];


}

__global__ void stream1_d_orig(float *f0, float *f1, float *f2,
							float *f3, float *f4, float *f5,
							float *f6, float *f7, float *f8,
							int n, int *image, float omega, float uMax)
{
	int i,m,j;
	j = threadIdx.y*blockDim.x+threadIdx.x;
	i = j+blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	i += blockDim.x*blockDim.y*blockIdx.x;
	m = blockDim.x*blockDim.y;
	
//	__shared__ float f0_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f1_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f2_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f3_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f4_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f5_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f6_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f7_s[BLOCKSIZE*BLOCKSIZE];
//	__shared__ float f8_s[BLOCKSIZE*BLOCKSIZE];
//
//	f0_s[j] = f0[i];

	//STREAM
	//j = (i+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	//j = (i+1)-blockDim.x*(threadIdx.x==blockDim.x-1);//equiv to (i+1)%(m), if m is pow of 2;
	if((i+1)%(blockDim.x)==0) j=i-blockDim.x+1;
	else j=i+1;
	float temp = f3[j];
	f3[j]=f1[i];
	f1[i]=temp;
	j = (i+blockDim.x) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f4[j];
	f4[j]=f2[i];
	f2[i]=temp;
	if((i+1)%(blockDim.x)==0) j=i+1;
	else j = (i+blockDim.x+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f7[j];
	f7[j]=f5[i];
	f5[i]=temp;
	if((i)%(blockDim.x)==0) j=i+2*blockDim.x-1;
	else j = (i+blockDim.x-1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f8[j];
	f8[j]=f6[i];
	f6[i]=temp;

	__syncthreads();

	if(image[i] != 1){
	temp = f3[i];
	f3[i]=f1[i];
	f1[i]=temp;
	temp = f4[i];
	f4[i]=f2[i];
	f2[i]=temp;
	temp = f7[i];
	f7[i]=f5[i];
	f5[i]=temp;
	temp = f8[i];
	f8[i]=f6[i];
	f6[i]=temp;

	
	float u,v,rho,feq,usqr;
	rho = f0[i]+f1[i]+f2[i]+f3[i]+f4[i]+f5[i]+f6[i]+f7[i]+f8[i];
	u = f1[i]-f3[i]+f5[i]-f6[i]-f7[i]+f8[i];
	v = f2[i]-f4[i]+f5[i]+f6[i]-f7[i]-f8[i];
	if(image[i] == 2){
		u = 0.0f;
		v = uMax;
		rho = u+(f0[i]+f2[i]+f4[i]+2.0f*f3[i]+2.0f*f6[i]+2.0f*f7[i]);
		f1[i] = f3[i]+4.0f*u/6.0f;
        f5[i] = f7[i]-0.5f*(f2[i]-f4[i])+v*0.5f+u/6.0f;
        f8[i] = f6[i]+0.5f*(f2[i]-f4[i])-v*0.5f+u/6.0f;	
	}
	usqr = u*u+v*v;
	feq = 4.0f/9.0f*(rho-1.5f*usqr);
	f0[i] = f0[i]-omega*(f0[i]-feq);
	feq = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[i] = f1[i]-omega*(f1[i]-feq);
	feq = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[i] = f2[i]-omega*(f2[i]-feq);
	feq = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[i] = f3[i]-omega*(f3[i]-feq);
	feq = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[i] = f4[i]-omega*(f4[i]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[i] = f5[i]-omega*(f5[i]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[i] = f6[i]-omega*(f6[i]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[i] = f7[i]-omega*(f7[i]-feq);
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[i] = f8[i]-omega*(f8[i]-feq);

	}

}

__global__ void initialize(float *f0, float *f1, float *f2,
							float *f3, float *f4, float *f5,
							float *f6, float *f7, float *f8,
							int n)
{
	int i;
	//x = blockIdx.x*blockDim.x+threadIdx.x;
	//y = blockIdx.y*blockDim.y+threadIdx.y;
	i = threadIdx.y*blockDim.x+threadIdx.x;
	i += blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	i += blockDim.x*blockDim.y*blockIdx.x;
	//i = y*blockDim.x+x;
	float u,v,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;
	feq = 4.0f/9.0f*(rho-1.5f*usqr);
	f0[i] = feq;
	feq = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[i] = feq;
	feq = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[i] = feq;
	feq = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[i] = feq;
	feq = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[i] = feq;
}


int main(int argc, char *argv[])
{

	float *f0_h, *f1_h, *f2_h, *f3_h, *f4_h, *f5_h, *f6_h, *f7_h, *f8_h;
	float *f0_d, *f1_d, *f2_d, *f3_d, *f4_d, *f5_d, *f6_d, *f7_d, *f8_d;
	int *image_d, *image_h;

	ofstream output;
	output.open ("LBM1_out.dat");


	size_t memsize, memsize_int;
	int i, tMax, n, nBlocks, xDim, yDim;
	float Re, omega, uMax, CharLength;
	
	xDim = 256;//512;//128;//32;
	yDim = 256;//512;//128;//32;
	tMax = 100000;
	Re = 10000.f;//100.f;
	uMax = 0.1f;
	CharLength = xDim-2.f;
	omega = 1.0f/(3.0f*(uMax*CharLength/Re)+0.5f);

	cout<<"omega: "<<omega<<endl;



	nBlocks = (xDim/BLOCKSIZE+xDim%BLOCKSIZE)*(yDim/BLOCKSIZE+yDim%BLOCKSIZE);
	int B = BLOCKSIZE*BLOCKSIZE;
	n = nBlocks*B;//block*dimx*dimy



	CUT_DEVICE_INIT(argc,argv);

	memsize = n*sizeof(float);
	memsize_int = n*sizeof(int);

	f0_h = (float *)malloc(memsize);
	f1_h = (float *)malloc(memsize);
	f2_h = (float *)malloc(memsize);
	f3_h = (float *)malloc(memsize);
	f4_h = (float *)malloc(memsize);
	f5_h = (float *)malloc(memsize);
	f6_h = (float *)malloc(memsize);
	f7_h = (float *)malloc(memsize);
	f8_h = (float *)malloc(memsize);
	image_h = (int *)malloc(memsize_int);

	CUDA_SAFE_CALL(hipMalloc((void **) &f0_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f1_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f2_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f3_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f4_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f5_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f6_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f7_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &f8_d, memsize));
	CUDA_SAFE_CALL(hipMalloc((void **) &image_d, memsize_int));

	for (i = 0; i < n; i++)
	{
		f0_h[i] = i;
		f1_h[i] = n-i;
		f2_h[i] = i;
		f3_h[i] = i;
		f4_h[i] = i;
		f5_h[i] = i;
		f6_h[i] = i;
		f7_h[i] = i;
		f8_h[i] = i;
		image_h[i] = 0;
		if(Xcoord(i,xDim) < 1) image_h[i] = 1;//DirichletWest
		if(Xcoord(i,xDim) > xDim-2) image_h[i] = 1;//BB
		if(Ycoord(i,xDim) < 1) image_h[i] = 1;//BB
		if(Ycoord(i,xDim) > yDim-2) image_h[i] = 3;//BB

	}
	CUDA_SAFE_CALL(hipMemcpy(f0_d, f0_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f1_d, f1_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f2_d, f2_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f3_d, f3_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f4_d, f4_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f5_d, f5_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f6_d, f6_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f7_d, f7_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(f8_d, f8_h, memsize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice));


    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 grid(xDim/BLOCKSIZE,yDim/BLOCKSIZE);
	cout<<"nBlocks:"<<nBlocks<<endl;


	initialize<<<grid, threads>>>(f0_d, f1_d, f2_d, f3_d, f4_d, f5_d, f6_d, f7_d, f8_d,n);
	for(int t = 0; t<tMax; t++){
		//stream1_d_orig<<<grid, threads>>>(f0_d,f1_d,f2_d,f3_d,f4_d,f5_d,f6_d,f7_d,f8_d,n,image_d,omega,uMax);
		//stream1_d<<<grid, threads>>>(f0_d,f1_d,f2_d,f3_d,f4_d,f5_d,f6_d,f7_d,f8_d,n,image_d,omega,uMax);
		mrt_d<<<grid, threads>>>(f0_d,f1_d,f2_d,f3_d,f4_d,f5_d,f6_d,f7_d,f8_d,n,image_d,omega,uMax);
		if(t%10000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	CUT_CHECK_ERROR("Kernel execution failed");

		CUDA_SAFE_CALL(hipMemcpy(f0_h, f0_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f1_h, f1_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f2_h, f2_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f3_h, f3_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f4_h, f4_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f5_h, f5_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f6_h, f6_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f7_h, f7_d, memsize, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(f8_h, f8_d, memsize, hipMemcpyDeviceToHost));
		output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
		output<<"ZONE F=POINT, I="<<xDim<<", J="<<yDim<<"\n";
		
//		for(i = 0; i<n; i++)
//		{
		int row = 0;
		int col = 0;
		i = 0;
		int rowB, colB;
		float xcoord, ycoord;
		float rho, u, v;
		rho = 0;
		u = 0;
		v = 0;

		for(rowB = 0; rowB<(yDim/BLOCKSIZE); rowB++){
			//i = rowB+B*(xDim/BLOCKSIZE);
			for(row = 0; row<BLOCKSIZE; row++){
				i = row*BLOCKSIZE+rowB*B*(xDim/BLOCKSIZE);
				//ycoord = row+rowB*BLOCKSIZE;//(i%B)/BLOCKSIZE+(i/B)/(xDim/BLOCKSIZE)*BLOCKSIZE;
				ycoord = Ycoord(i,xDim);
				for(colB = 0; colB<(xDim/BLOCKSIZE); colB++){
					for(col = 0; col<BLOCKSIZE; col++){
						rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i];
						u = f1_h[i]-f3_h[i]+f5_h[i]-f6_h[i]-f7_h[i]+f8_h[i];
						v = f2_h[i]-f4_h[i]+f5_h[i]+f6_h[i]-f7_h[i]-f8_h[i];
						//xcoord = (i%B)%BLOCKSIZE+((i/B)%(xDim/BLOCKSIZE))*BLOCKSIZE;
						xcoord = Xcoord(i,xDim);
						//output<<xcoord<<", "<<ycoord<<", "<<"0"<<","<<"0"<<","<<"0"<<endl;
						output<<xcoord<<", "<<ycoord<<", "<<u<<","<<v<<","<<rho<<endl;
						i++;
					}
					i += (BLOCKSIZE-1)*BLOCKSIZE;
				}
			}
			//i += B*(xDim/BLOCKSIZE-1)+BLOCKSIZE;
		}





//		for(row = 0; row<yDim; row++){
//			i = row*xDim;
//			for(col = 0; col<xDim; col++){
//				float rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i];
//				float u = f1_h[i]-f3_h[i]+f5_h[i]-f6_h[i]-f7_h[i]+f8_h[i];
//				float v = f2_h[i]-f4_h[i]+f5_h[i]+f6_h[i]-f7_h[i]-f8_h[i];
////				output<<i%16<<", "<<i/16<<", "<<f5_h[i]<<","<<f6_h[i]<<","<<f7_h[i]<<endl;
////				output<<i%xDim<<", "<<i/xDim<<", "<<u<<","<<v<<","<<rho<<endl;
//				float xcoord = (i%B)%BLOCKSIZE+(i/B)%(xDim/BLOCKSIZE);
//				float ycoord = (i%B)/BLOCKSIZE+(i/B)/(xDim/BLOCKSIZE);
//				output<<xcoord<<", "<<ycoord<<", "<<u<<","<<v<<","<<rho<<endl;
//				i += 2*BLOCKSIZE+1;
//			}
//		}

	free(f0_h);
	free(f1_h);
	free(f2_h);
	free(f3_h);
	free(f4_h);
	free(f5_h);
	free(f6_h);
	free(f7_h);
	free(f8_h);

	output.close();

	CUDA_SAFE_CALL(hipFree(f0_d));
	CUDA_SAFE_CALL(hipFree(f1_d));
	CUDA_SAFE_CALL(hipFree(f2_d));
	CUDA_SAFE_CALL(hipFree(f3_d));
	CUDA_SAFE_CALL(hipFree(f4_d));
	CUDA_SAFE_CALL(hipFree(f5_d));
	CUDA_SAFE_CALL(hipFree(f6_d));
	CUDA_SAFE_CALL(hipFree(f7_d));
	CUDA_SAFE_CALL(hipFree(f8_d));

	return(0);

}
