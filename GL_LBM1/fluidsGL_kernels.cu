#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include "cutil_inline.h"

// CUDA FFT Libraries
#include <hipfft/hipfft.h>

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// FluidsGL CUDA kernel definitions
#include "fluidsGL_kernels.cuh"

// Texture reference for reading velocity field
texture<float2, 2> texref;
static hipArray *array = NULL;

// Particle data
extern GLuint vbo;                 // OpenGL vertex buffer object
extern struct hipGraphicsResource *cuda_vbo_resource; // handles OpenGL-CUDA exchange

// Texture pitch
extern size_t tPitch;
extern hipfftHandle planr2c;
extern hipfftHandle planc2r;
extern cData *vxfield = NULL;
extern cData *vyfield = NULL;

void setupTexture(int x, int y) {
    // Wrap mode appears to be the new default
    texref.filterMode = hipFilterModeLinear;
    hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

    hipMallocArray(&array, &desc, y, x);
    cutilCheckMsg("hipMalloc failed");
}

void bindTexture(void) {
    hipBindTextureToArray(texref, array);
    cutilCheckMsg("hipBindTexture failed");
}

void unbindTexture(void) {
    hipUnbindTexture(texref);
}
    
void updateTexture(cData *data, size_t wib, size_t h, size_t pitch) {
    hipMemcpy2DToArray(array, 0, 0, data, pitch, wib, h, hipMemcpyDeviceToDevice);
    cutilCheckMsg("hipMemcpy failed"); 
}

void deleteTexture(void) {
    hipFreeArray(array);
}

// Note that these kernels are designed to work with arbitrary 
// domain sizes, not just domains that are multiples of the tile
// size. Therefore, we have extra code that checks to make sure
// a given thread location falls within the domain boundaries in
// both X and Y. Also, the domain is covered by looping over
// multiple elements in the Y direction, while there is a one-to-one
// mapping between threads in X and the tile size in X.
// Nolan Goodnight 9/22/06

// This method adds constant force vectors to the velocity field 
// stored in 'v' according to v(x,t+1) = v(x,t) + dt * f.
__global__ void 
addForces_k(cData *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    cData *fj = (cData*)((char*)v + (ty + spy) * pitch) + tx + spx;

    cData vterm = *fj;
    tx -= r; ty -= r;
    float s = 1.f / (1.f + tx*tx*tx*tx + ty*ty*ty*ty);
    vterm.x += s * fx;
    vterm.y += s * fy;
    *fj = vterm;
}

// This method performs the velocity advection step, where we
// trace velocity vectors back in time to update each grid cell.
// That is, v(x,t+1) = v(p(x,-dt),t). Here we perform bilinear
// interpolation in the velocity space.
__global__ void 
advectVelocity_k(cData *v, float *vx, float *vy,
                 int dx, int pdx, int dy, float dt, int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    cData vterm, ploc;
    float vxterm, vyterm;
    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * pdx + gtidx;
                vterm = tex2D(texref, (float)gtidx, (float)fi);
                ploc.x = (gtidx + 0.5f) - (dt * vterm.x * dx);
                ploc.y = (fi + 0.5f) - (dt * vterm.y * dy);
                vterm = tex2D(texref, ploc.x, ploc.y);
                vxterm = vterm.x; vyterm = vterm.y; 
                vx[fj] = vxterm;
                vy[fj] = vyterm; 
            }
        }
    }
}

// This method performs velocity diffusion and forces mass conservation 
// in the frequency domain. The inputs 'vx' and 'vy' are complex-valued 
// arrays holding the Fourier coefficients of the velocity field in
// X and Y. Diffusion in this space takes a simple form described as:
// v(k,t) = v(k,t) / (1 + visc * dt * k^2), where visc is the viscosity,
// and k is the wavenumber. The projection step forces the Fourier
// velocity vectors to be orthogonal to the vectors for each
// wavenumber: v(k,t) = v(k,t) - ((k dot v(k,t) * k) / k^2.
__global__ void 
diffuseProject_k(cData *vx, cData *vy, int dx, int dy, float dt, 
                 float visc, int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    cData xterm, yterm;
    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                xterm = vx[fj];
                yterm = vy[fj];

                // Compute the index of the wavenumber based on the
                // data order produced by a standard NN FFT.
                int iix = gtidx;
                int iiy = (fi>dy/2)?(fi-(dy)):fi;

                // Velocity diffusion
                float kk = (float)(iix * iix + iiy * iiy); // k^2 
                float diff = 1.f / (1.f + visc * dt * kk);
                xterm.x *= diff; xterm.y *= diff;
                yterm.x *= diff; yterm.y *= diff;

                // Velocity projection
                if (kk > 0.f) {
                    float rkk = 1.f / kk;
                    // Real portion of velocity projection
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    // Imaginary portion of velocity projection
                    float ikp = (iix * xterm.y + iiy * yterm.y);
                    xterm.x -= rkk * rkp * iix;
                    xterm.y -= rkk * ikp * iix;
                    yterm.x -= rkk * rkp * iiy;
                    yterm.y -= rkk * ikp * iiy;
                }
                
                vx[fj] = xterm;
                vy[fj] = yterm;
            }
        }
    }
}

// This method updates the velocity field 'v' using the two complex 
// arrays from the previous step: 'vx' and 'vy'. Here we scale the 
// real components by 1/(dx*dy) to account for an unnormalized FFT. 
__global__ void 
updateVelocity_k(cData *v, float *vx, float *vy, 
                 int dx, int pdx, int dy, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float vxterm, vyterm;
    cData nvterm;
    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fjr = fi * pdx + gtidx; 
                vxterm = vx[fjr];
                vyterm = vy[fjr];

                // Normalize the result of the inverse FFT
                float scale = 1.f / (dx * dy);
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;
               
                cData *fj = (cData*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}

// This method updates the particles by moving particle positions
// according to the velocity field and time step. That is, for each
// particle: p(t+1) = p(t) + dt * v(p(t)).  
__global__ void 
advectParticles_k(cData *part, cData *v, int dx, int dy, 
                  float dt, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    // gtidx is the domain location in x for this thread
    cData pterm, vterm;
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                pterm = part[fj];
                
                int xvi = ((int)(pterm.x * dx));
                int yvi = ((int)(pterm.y * dy));
                vterm = *((cData*)((char*)v + yvi * pitch) + xvi);   
 
                pterm.x += dt * vterm.x;
                pterm.x = pterm.x - (int)pterm.x;            
                pterm.x += 1.f; 
                pterm.x = pterm.x - (int)pterm.x;              
                pterm.y += dt * vterm.y;
                pterm.y = pterm.y - (int)pterm.y;            
                pterm.y += 1.f; 
                pterm.y = pterm.y - (int)pterm.y;                  

                part[fj] = pterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}


// These are the external function calls necessary for launching fluid simuation
extern "C"
void addForces(cData *v, int dx, int dy, int spx, int spy, float fx, float fy, int r) { 

    dim3 tids(2*r+1, 2*r+1);
    
    addForces_k<<<1, tids>>>(v, dx, dy, spx, spy, fx, fy, r, tPitch);
    cutilCheckMsg("addForces_k failed.");
}

extern "C"
void advectVelocity(cData *v, float *vx, float *vy, int dx, int pdx, int dy, float dt) 
{ 
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));

    dim3 tids(TIDSX, TIDSY);

    updateTexture(v, DIM*sizeof(cData), DIM, tPitch);
    advectVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, dt, TILEY/TIDSY);

    cutilCheckMsg("advectVelocity_k failed.");
}

extern "C"
void diffuseProject(cData *vx, cData *vy, int dx, int dy, float dt, float visc) 
{ 
    // Forward FFT
    hipfftExecR2C(planr2c, (hipfftReal*)vx, (hipfftComplex*)vx); 
    hipfftExecR2C(planr2c, (hipfftReal*)vy, (hipfftComplex*)vy);

    uint3 grid = make_uint3((dx/TILEX)+(!(dx%TILEX)?0:1), 
                            (dy/TILEY)+(!(dy%TILEY)?0:1), 1);
    uint3 tids = make_uint3(TIDSX, TIDSY, 1);
    
    diffuseProject_k<<<grid, tids>>>(vx, vy, dx, dy, dt, visc, TILEY/TIDSY);
    cutilCheckMsg("diffuseProject_k failed.");

    // Inverse FFT
    hipfftExecC2R(planc2r, (hipfftComplex*)vx, (hipfftReal*)vx); 
    hipfftExecC2R(planc2r, (hipfftComplex*)vy, (hipfftReal*)vy);
}

extern "C"
void updateVelocity(cData *v, float *vx, float *vy, int dx, int pdx, int dy) 
{ 
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));
    dim3 tids(TIDSX, TIDSY);

    updateVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, TILEY/TIDSY, tPitch);
    cutilCheckMsg("updateVelocity_k failed.");
}

extern "C"
void advectParticles(GLuint vbo, cData *v, int dx, int dy, float dt) 
{
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));
    dim3 tids(TIDSX, TIDSY);

    cData *p;
    cutilSafeCall(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
    size_t num_bytes; 
    cutilSafeCall(hipGraphicsResourceGetMappedPointer((void **)&p, &num_bytes,  
						       cuda_vbo_resource));
    cutilCheckMsg("hipGraphicsResourceGetMappedPointer failed");
   
    advectParticles_k<<<grid, tids>>>(p, v, dx, dy, dt, TILEY/TIDSY, tPitch);
    cutilCheckMsg("advectParticles_k failed.");
    
    cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

    cutilCheckMsg("hipGraphicsUnmapResources failed");
}





////////

int Xcoord(int i, int XDim){
	int B = BLOCKSIZE*BLOCKSIZE;
	return (i%B)%BLOCKSIZE+((i/B)%(XDim/BLOCKSIZE))*BLOCKSIZE;
}
int Ycoord(int i, int XDim){
	int B = BLOCKSIZE*BLOCKSIZE;
	return (i%B)/BLOCKSIZE+((i/B)/(XDim/BLOCKSIZE))*BLOCKSIZE;
}

__global__ void mrt_d(float *f0, float *f1, float *f2,
							float *f3, float *f4, float *f5,
							float *f6, float *f7, float *f8,
							int n, int *image, float omega, float uMax)
{
	int i,m,j,k;
	j = threadIdx.y*blockDim.x+threadIdx.x;//local block index (for shared mem)
	i = j+blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	i += blockDim.x*blockDim.y*blockIdx.x;//global memory index
	m = blockDim.x*blockDim.y;
	//m = BLOCKSIZE*BLOCKSIZE;
	
	__shared__ float f0_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f1_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f2_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f3_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f4_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f5_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f6_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f7_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ float f8_s[(BLOCKSIZE)*(BLOCKSIZE)+1];
	__shared__ int image_s[(BLOCKSIZE)*(BLOCKSIZE)+1];

	f0_s[j] = f0[i];
	f1_s[j] = f1[i];
	f2_s[j] = f2[i];
	f3_s[j] = f3[i];
	f4_s[j] = f4[i];
	f5_s[j] = f5[i];
	f6_s[j] = f6[i];
	f7_s[j] = f7[i];
	f8_s[j] = f8[i];
	image_s[j] = image[i];
	
	//1
	if(threadIdx.x == blockDim.x-1 && blockIdx.x != 0){
		f1_s[j] = f1[i-m];
		f5_s[j] = f5[i-m];
		f8_s[j] = f8[i-m];
	}
	//3
	if(threadIdx.x == 0 && blockIdx.x != gridDim.x-1){
		f3_s[j] = f3[i+m];
		f7_s[j] = f7[i+m];
		f6_s[j] = f6[i+m];
	}
	//2
	if(threadIdx.y == blockDim.y-1 && blockIdx.y != 0){
		f2_s[j] = f2[i-m*gridDim.x];
		f5_s[j] = f5[i-m*gridDim.x];
		f6_s[j] = f6[i-m*gridDim.x];
	}
	//4
	if(threadIdx.y == 0 && blockIdx.y != gridDim.y-1){
		f4_s[j] = f4[i+m*gridDim.x];
		f7_s[j] = f7[i+m*gridDim.x];
		f8_s[j] = f8[i+m*gridDim.x];
	}
	//5
	if(threadIdx.y == blockDim.y-1 && threadIdx.x == blockDim.x-1 &&
		blockIdx.x != 0 && blockIdx.y != 0)//upper corner
		f5_s[j] = f5[i-m*(1+gridDim.x)];
	//7
	if(threadIdx.y == 0 && threadIdx.x == 0 &&
		blockIdx.x != gridDim.x-1 && blockIdx.y != gridDim.y-1)
		f7_s[j] = f7[i+m*(1+gridDim.x)];
	//6
	if(threadIdx.y == blockDim.y-1 && threadIdx.x == 0 &&
		blockIdx.x != gridDim.x-1 && blockIdx.y != 0)
		f6_s[j] = f6[i-m*(gridDim.x-1)];
	//8
	if(threadIdx.y == 0 && threadIdx.x == blockDim.x-1 &&
		blockIdx.x != 0 && blockIdx.y != gridDim.y-1)
		f8_s[j] = f8[i+m*(gridDim.x-1)];


	__syncthreads();

	//STREAM
	//j = (i+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	//j = (i+1)-blockDim.x*(threadIdx.x==blockDim.x-1);//equiv to (i+1)%(m), if m is pow of 2;
	if((j+1)%(blockDim.x)==0) k=j-blockDim.x+1;
	else k=j+1;
	float temp = f3_s[k];
	f3_s[k]=f1_s[j];
	f1_s[j]=temp;
	k = (j+blockDim.x) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f4_s[k];
	f4_s[k]=f2_s[j];
	f2_s[j]=temp;
	if((j+1)%(blockDim.x)==0) k=(j+1) & (m-1);
	else k = (j+blockDim.x+1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f7_s[k];
	f7_s[k]=f5_s[j];
	f5_s[j]=temp;
	if((j)%(blockDim.x)==0) k=(j+2*blockDim.x-1) & (m-1);
	else k = (j+blockDim.x-1) & (m-1);//equiv to (i+1)%(m), if m is pow of 2;
	temp = f8_s[k];
	f8_s[k]=f6_s[j];
	f6_s[j]=temp;

	__syncthreads();

	if(image_s[j] != 1){
	temp = f3_s[j];
	f3_s[j]=f1_s[j];
	f1_s[j]=temp;
	temp = f4_s[j];
	f4_s[j]=f2_s[j];
	f2_s[j]=temp;
	temp = f7_s[j];
	f7_s[j]=f5_s[j];
	f5_s[j]=temp;
	temp = f8_s[j];
	f8_s[j]=f6_s[j];
	f6_s[j]=temp;

	
	//float u,v,rho,feq,usqr;
	float u,v,rho,usqr;
	float m1,m2,m4,m6,m7,m8;
	float meq1,meq2,meq4,meq6,meq7,meq8;
	rho = f0_s[j]+f1_s[j]+f2_s[j]+f3_s[j]+f4_s[j]+f5_s[j]+f6_s[j]+f7_s[j]+f8_s[j];
	u = f1_s[j]-f3_s[j]+f5_s[j]-f6_s[j]-f7_s[j]+f8_s[j];
	v = f2_s[j]-f4_s[j]+f5_s[j]+f6_s[j]-f7_s[j]-f8_s[j];
	if(image_s[j] == 2){
		u = 0.0f;
		v = uMax;
		rho = u+(f0_s[j]+f2_s[j]+f4_s[j]+2.0f*f3_s[j]+2.0f*f6_s[j]+2.0f*f7_s[j]);
		f1_s[j] = f3_s[j]+4.0f*u/6.0f;
        f5_s[j] = f7_s[j]-0.5f*(f2_s[j]-f4_s[j])+v*0.5f+u/6.0f;
        f8_s[j] = f6_s[j]+0.5f*(f2_s[j]-f4_s[j])-v*0.5f+u/6.0f;	
	}
	else if(image_s[j] == 3){//north
		u = uMax;
		v = 0.0f;
		rho = -v+(f0_s[j]+f1_s[j]+f3_s[j]+2.0f*f6_s[j]+2.0f*f2_s[j]+2.0f*f5_s[j]);
		f4_s[j] = f2_s[j]-4.0f*v/6.0f;
        f7_s[j] = f5_s[j]+0.5f*(f1_s[j]-f3_s[j])-u*0.5f+v/6.0f;
        f8_s[j] = f6_s[j]-0.5f*(f1_s[j]-f3_s[j])+u*0.5f+v/6.0f;	
	}
	usqr = u*u+v*v;

		meq1 = -2.0f*rho+3.0f*usqr;//e_eq (uses rho, Yu)
		meq2 = rho-3.0f*usqr; //epsilon_eq (uses rho, Yu)
		meq4 = -u;//qx_eq
		meq6 = -v;//qy_eq
		meq7 = u*u-v*v;//pxx_eq
		meq8 = u*v;//pxy_eq

		m1 =-4.f*f0_s[j]+-1.f*f1_s[j]+-1.f*f2_s[j]+-1.f*f3_s[j]+-1.f*f4_s[j]+ 2.f*f5_s[j]+ 2.f*f6_s[j]+ 2.f*f7_s[j]+ 2.f*f8_s[j];
		m2 = 4.f*f0_s[j]+-2.f*f1_s[j]+-2.f*f2_s[j]+-2.f*f3_s[j]+-2.f*f4_s[j]+ 1.f*f5_s[j]+ 1.f*f6_s[j]+ 1.f*f7_s[j]+ 1.f*f8_s[j];
		m4 =             -2.f*f1_s[j]             + 2.f*f3_s[j]             + 1.f*f5_s[j]+-1.f*f6_s[j]+-1.f*f7_s[j]+ 1.f*f8_s[j];
		m6 =                          -2.f*f2_s[j]             + 2.f*f4_s[j]+ 1.f*f5_s[j]+ 1.f*f6_s[j]+-1.f*f7_s[j]+-1.f*f8_s[j];
		m7 =              1.f*f1_s[j]+-1.f*f2_s[j]+ 1.f*f3_s[j]+-1.f*f4_s[j]                                                    ;
		m8 =                                                                  1.f*f5_s[j]+-1.f*f6_s[j]+ 1.f*f7_s[j]+-1.f*f8_s[j];

		float Q11 = (m7-meq7)*0.5f;
		float Q22 = -Q11;
		float Q12 = m8-meq8;
		float Q21 = Q12;
		float Q = Q11*Q11+Q12*Q12+Q21*Q21+Q22*Q22;
	
		float Cs = 0.01f;
		float nu0 = ((1.0f/omega)-0.5f)/3.0f;
		float Smag = (sqrt((2.0f*rho/(3.0f*omega))*(2.0f*rho/(3.0f*omega))
				+4.0f*2.0f*rho*Cs*sqrt(Q))-2.0f*rho/(3.0f*omega))/(4.0f*rho*Cs);
		float OmegaStar = 1.0f/(3.0f*(nu0+Cs*Smag)+0.5f);

f0_s[j]-=-4.f*(m1-meq1)/36.0f+4.f *(m2-meq2)/36.0f;//+0   *(m4-meq4)/12.0f+0   *(m6-meq6)/12.0f;
f1_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+-2.f*(m4-meq4)/12.0f                     +   (m7-meq7)*OmegaStar/4.0f;
f2_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +-2.f*(m6-meq6)/12.0f+-  (m7-meq7)*OmegaStar/4.0f;
f3_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+2.f *(m4-meq4)/12.0f                     +   (m7-meq7)*OmegaStar/4.0f;
f4_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +2.f *(m6-meq6)/12.0f+-  (m7-meq7)*OmegaStar/4.0f;
f5_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+     (m6-meq6)/12.0f+   (m8-meq8)*OmegaStar/4.0f;
f6_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+     (m6-meq6)/12.0f+-  (m8-meq8)*OmegaStar/4.0f;
f7_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+   (m8-meq8)*OmegaStar/4.0f;
f8_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+-  (m8-meq8)*OmegaStar/4.0f;	

//f0_s[j]-=-4.f*(m1-meq1)/36.0f+4.f *(m2-meq2)/36.0f;//+0   *(m4-meq4)/12.0f+0   *(m6-meq6)/12.0f;
//f1_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+-2.f*(m4-meq4)/12.0f                     +   (m7-meq7)*omega/4.0f;
//f2_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +-2.f*(m6-meq6)/12.0f+-  (m7-meq7)*omega/4.0f;
//f3_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f+2.f *(m4-meq4)/12.0f                     +   (m7-meq7)*omega/4.0f;
//f4_s[j]-=-    (m1-meq1)/36.0f+-2.f*(m2-meq2)/36.0f                     +2.f *(m6-meq6)/12.0f+-  (m7-meq7)*omega/4.0f;
//f5_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+     (m6-meq6)/12.0f+   (m8-meq8)*omega/4.0f;
//f6_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+     (m6-meq6)/12.0f+-  (m8-meq8)*omega/4.0f;
//f7_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+-    (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+   (m8-meq8)*omega/4.0f;
//f8_s[j]-=2.f *(m1-meq1)/36.0f+     (m2-meq2)/36.0f+     (m4-meq4)/12.0f+-    (m6-meq6)/12.0f+-  (m8-meq8)*omega/4.0f;	

	}

	__syncthreads();
	f0[i] = f0_s[j];
	f1[i] = f1_s[j];
	f2[i] = f2_s[j];
	f3[i] = f3_s[j];
	f4[i] = f4_s[j];
	f5[i] = f5_s[j];
	f6[i] = f6_s[j];
	f7[i] = f7_s[j];
	f8[i] = f8_s[j];


}

__global__ void initialize(float *f0, float *f1, float *f2,
							float *f3, float *f4, float *f5,
							float *f6, float *f7, float *f8,
							int n)
{
	int i;
	//x = blockIdx.x*blockDim.x+threadIdx.x;
	//y = blockIdx.y*blockDim.y+threadIdx.y;
	i = threadIdx.y*blockDim.x+threadIdx.x;
	i += blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	i += blockDim.x*blockDim.y*blockIdx.x;
	//i = y*blockDim.x+x;
	float u,v,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;
	feq = 4.0f/9.0f*(rho-1.5f*usqr);
	f0[i] = feq;
	feq = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[i] = feq;
	feq = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[i] = feq;
	feq = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[i] = feq;
	feq = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[i] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[i] = feq;
}
