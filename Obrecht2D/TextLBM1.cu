#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;

//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

//int Xcoord(int i, int XDim){
//	int B = BLOCKSIZE*BLOCKSIZE;
//	return (i%B)%BLOCKSIZE+((i/B)%(XDim/BLOCKSIZE))*BLOCKSIZE;
//}
//int Ycoord(int i, int XDim){
//	int B = BLOCKSIZE*BLOCKSIZE;
//	return (i%B)/BLOCKSIZE+((i/B)/(XDim/BLOCKSIZE))*BLOCKSIZE;
//}

//texture
//texture<float,1,hipReadModeElementType> texRef_f1;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;

#include <sys/time.h>
#include <time.h>

int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}


__global__ void test(float *f0A, float *f1A, float *f2A,
						   float *f3A, float *f4A, float *f5A,
						   float *f6A, float *f7A, float *f8A,
						               float *f1B, float *f2B,
						   float *f3B, float *f4B, float *f5B,
						   float *f6B, float *f7B, float *f8B, //int pitch)
						   int n, int *image, float omega, float uMax, int pitch)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
//	int i = x+y*blockDim.x*gridDim.x;
//	j = threadIdx.y*blockDim.x+threadIdx.x;//local block index (for shared mem)
//	i = j+blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
//	i += blockDim.x*blockDim.y*blockIdx.x;//global memory index

		//f0A[i] = f;
		float f1,f2,f3,f4,f5,f6,f7,f8;
//		f1=tex2D(texRef_f1,x+1,y  );
//		f2=tex2D(texRef_f2,x-1,y  );
//		f3=tex2D(texRef_f3,x  ,y+1);
//		f4=tex2D(texRef_f4,x  ,y-1);
//		f5=tex2D(texRef_f5,x+1,y+1);
//		f7=tex2D(texRef_f7,x-1,y-1);
//		f6=tex2D(texRef_f6,x-1,y+1);
//		f8=tex2D(texRef_f8,x+1,y-1);
		//if(image[i] == 0){

//		f1 = tex2D(texRef_f1,x-1,y  );
//		f2 = tex2D(texRef_f2,x  ,y-1);
//		f3 = tex2D(texRef_f3,x+1,y  );
//		f4 = tex2D(texRef_f4,x  ,y+1);
//		f5 = tex2D(texRef_f5,x-1,y-1);
//		f6 = tex2D(texRef_f6,x+1,y-1);
//		f7 = tex2D(texRef_f7,x+1,y+1);
//		f8 = tex2D(texRef_f8,x-1,y+1);

//		f1 = f1A[x-1+y*pitch];
//		f2 = f2A[x  +(y-1)*pitch];
//		f3 = f3A[x+1+y*pitch];
//		f4 = f4A[x  +(y+1)*pitch];
//		f5 = f5A[x-1+(y-1)*pitch];
//		f6 = f6A[x+1+(y-1)*pitch];
//		f7 = f7A[x+1+(y+1)*pitch];
//		f8 = f8A[x-1+(y+1)*pitch];

//		f1=f1A[x+1+(y  )*pitch];
//		f2=f2A[x-1+(y  )*pitch];
//		f3=f3A[x  +(y+1)*pitch];
//		f4=f4A[x  +(y-1)*pitch];
//		f5=f5A[x+1+(y+1)*pitch];
//		f7=f7A[x-1+(y-1)*pitch];
//		f6=f6A[x-1+(y+1)*pitch];
//		f8=f8A[x+1+(y-1)*pitch];
		f1=f1A[x+y*pitch];
		f2=f2A[x+y*pitch];
		f3=f3A[x+y*pitch];
		f4=f4A[x+y*pitch];
		f5=f5A[x+y*pitch];
		f6=f6A[x+y*pitch];
		f7=f7A[x+y*pitch];
		f8=f8A[x+y*pitch];
//		f1B[x+y*pitch]=f1+1;
//		f3B[x+y*pitch]=f3+1;
//		f2B[x+y*pitch]=f2+1;
//		f4B[x+y*pitch]=f4+1;
//		f5B[x+y*pitch]=f5+1;
//		f6B[x+y*pitch]=f6+1;
//		f7B[x+y*pitch]=f7+1;
//		f8B[x+y*pitch]=f8+1;
		//}
//		f1=f1A[x+y*pitch];
//		f3=f3A[x+y*pitch];
//		f2=f2A[x+y*pitch];
//		f4=f4A[x+y*pitch];
//		f5=f5A[x+y*pitch];
//		f7=f7A[x+y*pitch];
//		f6=f6A[x+y*pitch];
//		f8=f8A[x+y*pitch];
		f1B[x+y*pitch]=f1;
		f2B[x+y*pitch]=f2;
		f3B[x+y*pitch]=f3;
		f4B[x+y*pitch]=f4;
		f5B[x+y*pitch]=f5;
		f6B[x+y*pitch]=f6;
		f7B[x+y*pitch]=f7;
		f8B[x+y*pitch]=f8;


}

__device__ void collide(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, float rho, float u, float v, float omega)
{
	float m1,m2,m4,m6,m7,m8;
	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq

	f0=f0-(-m1+m2)*0.11111111f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
	f1=f1-(-m1-2.0f*(m2+m4)+m7*omega*9.0f)*0.027777777f;
	f2=f2-(-m1-2.f*m2-6.f*m6-m7*omega*9.0f)*0.027777777f;
	f3=f3-(-m1-2.f*m2+6.f*m4+m7*omega*9.0f)*0.027777777f;
	f4=f4-(-m1-2.f*m2+6.f*m6-m7*omega*9.0f)*0.027777777f;
	f5=f5-(2.f*m1+m2+3.f*m4+3.f*m6+m8*omega*9.0f)*0.027777777f;
	f6=f6-(2.f*m1+m2-3.f*m4+3.f*m6-m8*omega*9.0f)*0.027777777f;
	f7=f7-(2.f*m1+m2-3.f*m4-3.f*m6+m8*omega*9.0f)*0.027777777f;
	f8=f8-(2.f*m1+m2+3.f*m4-3.f*m6-m8*omega*9.0f)*0.027777777f;


//	f0=f0-(-m1+m2)/9.0f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
//	f1=f1-(-m1-2.0f*(m2+m4)+m7*omega*9.0f)/36.0f;
//	f2=f2-(-m1-2.f*m2-6.f*m6-m7*omega*9.0f)/36.0f;
//	f3=f3-(-m1-2.f*m2+6.f*m4+m7*omega*9.0f)/36.0f;
//	f4=f4-(-m1-2.f*m2+6.f*m6-m7*omega*9.0f)/36.0f;
//	f5=f5-(2.f*m1+m2+3.f*m4+3.f*m6+m8*omega*9.0f)/36.0f;
//	f6=f6-(2.f*m1+m2-3.f*m4+3.f*m6-m8*omega*9.0f)/36.0f;
//	f7=f7-(2.f*m1+m2-3.f*m4-3.f*m6+m8*omega*9.0f)/36.0f;
//	f8=f8-(2.f*m1+m2+3.f*m4-3.f*m6-m8*omega*9.0f)/36.0f;

//	float feq;
//	float usqr = u*u+v*v;
//	feq = 4.0f/9.0f*(rho-1.5f*usqr);
//	f0 = f0-omega*(f0-feq);
//	feq = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//	f1 = f1-omega*(f1-feq);
//	feq = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//	f2 = f2-omega*(f2-feq);
//	feq = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//	f3 = f3-omega*(f3-feq);
//	feq = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//	f4 = f4-omega*(f4-feq);
//	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
//	f5 = f5-omega*(f5-feq);
//	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//	f6 = f6-omega*(f6-feq);
//	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//	f7 = f7-omega*(f7-feq);
//	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
//	f8 = f8-omega*(f8-feq);


}


__global__ void mrt_d_textAB(float *f0A, float *f1A, float *f2A,
						   float *f3A, float *f4A, float *f5A,
						   float *f6A, float *f7A, float *f8A,
						               float *f1B, float *f2B,
						   float *f3B, float *f4B, float *f5B,
						   float *f6B, float *f7B, float *f8B,
						   int n, int *image, float omega, float uMax, int pitch)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int i = x+y*blockDim.x*gridDim.x;
	int im = image[i];
	if(im == 1){//BB
		float f0,f1,f2,f3,f4,f5,f6,f7,f8;

//		//f0 = f0A[x+y*pitch];
//		f2 = f4A[x  +(y+1)*pitch];
//		f4 = f2A[x  +(y-1)*pitch];
//		f1 = tex2D(texRef_f3A,x+1,y  );
//		f3 = tex2D(texRef_f1A,x-1,y  );
//		f5 = tex2D(texRef_f7A,x+1,y+1);
//		f7 = tex2D(texRef_f5A,x-1,y-1);
//		f6 = tex2D(texRef_f8A,x-1,y+1);
//		f8 = tex2D(texRef_f6A,x+1,y-1);
//
//		//f0A[x+y*pitch] = f0;
//		f2B[x+y*pitch] = f2;
//		f4B[x+y*pitch] = f4;
//		f1B[x+y*pitch] = f1;
//		f3B[x+y*pitch] = f3;
//		f5B[x+y*pitch] = f5;
//		f7B[x+y*pitch] = f7;
//		f6B[x+y*pitch] = f6;
//		f8B[x+y*pitch] = f8;


		f0A[x+y*pitch]=f0A[x+y*pitch];
		f1B[x+y*pitch]=tex2D(texRef_f3A,x+1,y  );
		f3B[x+y*pitch]=tex2D(texRef_f1A,x-1,y  );
		f2B[x+y*pitch]=tex2D(texRef_f4A,x  ,y+1);
		f4B[x+y*pitch]=tex2D(texRef_f2A,x  ,y-1);
		f5B[x+y*pitch]=tex2D(texRef_f7A,x+1,y+1);
		f7B[x+y*pitch]=tex2D(texRef_f5A,x-1,y-1);
		f6B[x+y*pitch]=tex2D(texRef_f8A,x-1,y+1);
		f8B[x+y*pitch]=tex2D(texRef_f6A,x+1,y-1);

//		f0A[x+y*pitch] = f0A[x+y*pitch];
//		f1B[x+y*pitch] = f3A[x+1+(y  )*pitch];
//		f3B[x+y*pitch] = f1A[x-1+(y  )*pitch];
//		f2B[x+y*pitch] = f4A[x  +(y+1)*pitch];
//		f4B[x+y*pitch] = f2A[x  +(y-1)*pitch];
//		f5B[x+y*pitch] = f7A[x+1+(y+1)*pitch];
//		f7B[x+y*pitch] = f5A[x-1+(y-1)*pitch];
//		f6B[x+y*pitch] = f8A[x-1+(y+1)*pitch];
//		f8B[x+y*pitch] = f6A[x+1+(y-1)*pitch];

//		f0A[x+y*pitch] = f0A[x+y*pitch];
//		f1B[x+y*pitch] = f1A[x+(y)*pitch];
//		f3B[x+y*pitch] = f2A[x+(y)*pitch];
//		f2B[x+y*pitch] = f3A[x+(y)*pitch];
//		f4B[x+y*pitch] = f4A[x+(y)*pitch];
//		f5B[x+y*pitch] = f5A[x+(y)*pitch];
//		f7B[x+y*pitch] = f6A[x+(y)*pitch];
//		f6B[x+y*pitch] = f7A[x+(y)*pitch];
//		f8B[x+y*pitch] = f8A[x+(y)*pitch];


	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8;
		float u,v,rho;//,feq,usqr;
		
//		f0 = f0A[x  +y*pitch];
//		f2 = f2A[x  +(y-1)*pitch];
//		f4 = f4A[x  +(y+1)*pitch];
//		f1 = tex2D(texRef_f1A,x-1,y  );
//		f3 = tex2D(texRef_f3A,x+1,y  );
//		f5 = tex2D(texRef_f5A,x-1,y-1);
//		f6 = tex2D(texRef_f6A,x+1,y-1);
//		f7 = tex2D(texRef_f7A,x+1,y+1);
//		f8 = tex2D(texRef_f8A,x-1,y+1);


		f1 = tex2D(texRef_f1A,x-1,y  );
		f2 = tex2D(texRef_f2A,x  ,y-1);
		f3 = tex2D(texRef_f3A,x+1,y  );
		f4 = tex2D(texRef_f4A,x  ,y+1);
		f5 = tex2D(texRef_f5A,x-1,y-1);
		f6 = tex2D(texRef_f6A,x+1,y-1);
		f7 = tex2D(texRef_f7A,x+1,y+1);
		f8 = tex2D(texRef_f8A,x-1,y+1);
		f0 = f0A[x  +y*pitch];

//		f0 = f0A[x  +y*pitch];
//		f1 = f1A[x-1+y*pitch];
//		f2 = f2A[x  +(y-1)*pitch];
//		f3 = f3A[x+1+y*pitch];
//		f4 = f4A[x  +(y+1)*pitch];
//		f5 = f5A[x-1+(y-1)*pitch];
//		f6 = f6A[x+1+(y-1)*pitch];
//		f7 = f7A[x+1+(y+1)*pitch];
//		f8 = f8A[x-1+(y+1)*pitch];
	
//		f0 = f0A[x  +y*pitch];
//		f1 = f1A[x+y*pitch];
//		f2 = f2A[x+y*pitch];
//		f3 = f3A[x+y*pitch];
//		f4 = f4A[x+y*pitch];
//		f5 = f5A[x+y*pitch];
//		f6 = f6A[x+y*pitch];
//		f7 = f7A[x+y*pitch];
//		f8 = f8A[x+y*pitch];
		
		rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
		u = f1-f3+f5-f6-f7+f8;
		v = f2-f4+f5+f6-f7-f8;

		if(im == 2){
			u = 0.0f;
			v = uMax;
			rho = u+(f0+f2+f4+2.0f*f3+2.0f*f6+2.0f*f7);
			//f1 = f3+4.0f*u/6.0f;
			f1 = f3+u*0.66666667f;
	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;	
		}
		else if(im == 3){//north
			u = uMax;
			v = 0.0f;
			rho = -v+(f0+f1+f3+2.0f*f6+2.0f*f2+2.0f*f5);
			//f4 = f2-4.0f*v/6.0f;
			f4 = f2-v*0.66666667f;
	        f7 = f5+0.5f*(f1-f3)-u*0.5f+v*0.166666667f;
	        f8 = f6-0.5f*(f1-f3)+u*0.5f+v*0.166666667f;	
		}
		collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,rho,u,v,omega);

	f0A[y*pitch+x]=f0;
	f1B[y*pitch+x]=f1;
	f2B[y*pitch+x]=f2;
	f3B[y*pitch+x]=f3;
	f4B[y*pitch+x]=f4;
	f5B[y*pitch+x]=f5;
	f6B[y*pitch+x]=f6;
	f7B[y*pitch+x]=f7;
	f8B[y*pitch+x]=f8;

	}
}

__global__ void mrt_d_textBA(float *f0A, float *f1A, float *f2A,
						   float *f3A, float *f4A, float *f5A,
						   float *f6A, float *f7A, float *f8A,
						               float *f1B, float *f2B,
						   float *f3B, float *f4B, float *f5B,
						   float *f6B, float *f7B, float *f8B,
						   int n, int *image, float omega, float uMax, int pitch)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int i = x+y*blockDim.x*gridDim.x;
	float u,v,rho;//,usqr;
	int im = image[i];
	if(im == 1){//BB
		float f1,f2,f3,f4,f5,f6,f7,f8;

//		//f0 = f0A[x+y*pitch];
//		f2 = f4A[x  +(y+1)*pitch];
//		f4 = f2A[x  +(y-1)*pitch];
//		f1 = tex2D(texRef_f3B,x+1,y  );
//		f3 = tex2D(texRef_f1B,x-1,y  );
//		f5 = tex2D(texRef_f7B,x+1,y+1);
//		f7 = tex2D(texRef_f5B,x-1,y-1);
//		f6 = tex2D(texRef_f8B,x-1,y+1);
//		f8 = tex2D(texRef_f6B,x+1,y-1);
//
//		//f0A[x+y*pitch] = f0;
//		f2B[x+y*pitch] = f2;
//		f4B[x+y*pitch] = f4;
//		f1B[x+y*pitch] = f1;
//		f3B[x+y*pitch] = f3;
//		f5B[x+y*pitch] = f5;
//		f7B[x+y*pitch] = f7;
//		f6B[x+y*pitch] = f6;
//		f8B[x+y*pitch] = f8;


		//f0A[i] = f0A[i];
//		f0A[x+y*pitch]=f0A[x+y*pitch];
		f1B[x+y*pitch]=tex2D(texRef_f3B,x+1,y  );
		f3B[x+y*pitch]=tex2D(texRef_f1B,x-1,y  );
		f2B[x+y*pitch]=tex2D(texRef_f4B,x  ,y+1);
		f4B[x+y*pitch]=tex2D(texRef_f2B,x  ,y-1);
		f5B[x+y*pitch]=tex2D(texRef_f7B,x+1,y+1);
		f7B[x+y*pitch]=tex2D(texRef_f5B,x-1,y-1);
		f6B[x+y*pitch]=tex2D(texRef_f8B,x-1,y+1);
		f8B[x+y*pitch]=tex2D(texRef_f6B,x+1,y-1);
//
//		f0A[x+y*pitch] = f0A[x+y*pitch];
//		f1B[x+y*pitch] = f3A[x+1+(y  )*pitch];
//		f3B[x+y*pitch] = f1A[x-1+(y  )*pitch];
//		f2B[x+y*pitch] = f4A[x  +(y+1)*pitch];
//		f4B[x+y*pitch] = f2A[x  +(y-1)*pitch];
//		f5B[x+y*pitch] = f7A[x+1+(y+1)*pitch];
//		f7B[x+y*pitch] = f5A[x-1+(y-1)*pitch];
//		f6B[x+y*pitch] = f8A[x-1+(y+1)*pitch];
//		f8B[x+y*pitch] = f6A[x+1+(y-1)*pitch];
//
//		f0A[x+y*pitch] = f0A[x+y*pitch];
//		f1B[x+y*pitch] = f1A[x+(y)*pitch];
//		f3B[x+y*pitch] = f2A[x+(y)*pitch];
//		f2B[x+y*pitch] = f3A[x+(y)*pitch];
//		f4B[x+y*pitch] = f4A[x+(y)*pitch];
//		f5B[x+y*pitch] = f5A[x+(y)*pitch];
//		f7B[x+y*pitch] = f6A[x+(y)*pitch];
//		f6B[x+y*pitch] = f7A[x+(y)*pitch];
//		f8B[x+y*pitch] = f8A[x+(y)*pitch];


	}
	else{
		float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//		f0 = f0A[x  +y*pitch];
//		f2 = f2B[x  +(y-1)*pitch];
//		f4 = f4B[x  +(y+1)*pitch];
//		f1 = tex2D(texRef_f1B,x-1,y  );
//		f3 = tex2D(texRef_f3B,x+1,y  );
//		f5 = tex2D(texRef_f5B,x-1,y-1);
//		f6 = tex2D(texRef_f6B,x+1,y-1);
//		f7 = tex2D(texRef_f7B,x+1,y+1);
//		f8 = tex2D(texRef_f8B,x-1,y+1);


		f1 = tex2D(texRef_f1B,x-1,y  );
		f2 = tex2D(texRef_f2B,x  ,y-1);
		f3 = tex2D(texRef_f3B,x+1,y  );
		f4 = tex2D(texRef_f4B,x  ,y+1);
		f5 = tex2D(texRef_f5B,x-1,y-1);
		f6 = tex2D(texRef_f6B,x+1,y-1);
		f7 = tex2D(texRef_f7B,x+1,y+1);
		f8 = tex2D(texRef_f8B,x-1,y+1);
		f0 = f0A[x  +y*pitch];

//		f0 = f0A[x  +y*pitch];
//		f1 = f1A[x-1+y*pitch];
//		f2 = f2A[x  +(y-1)*pitch];
//		f3 = f3A[x+1+y*pitch];
//		f4 = f4A[x  +(y+1)*pitch];
//		f5 = f5A[x-1+(y-1)*pitch];
//		f6 = f6A[x+1+(y-1)*pitch];
//		f7 = f7A[x+1+(y+1)*pitch];
//		f8 = f8A[x-1+(y+1)*pitch];

//		f0 = f0A[x  +y*pitch];
//		f1 = f1A[x+y*pitch];
//		f2 = f2A[x+y*pitch];
//		f3 = f3A[x+y*pitch];
//		f4 = f4A[x+y*pitch];
//		f5 = f5A[x+y*pitch];
//		f6 = f6A[x+y*pitch];
//		f7 = f7A[x+y*pitch];
//		f8 = f8A[x+y*pitch];
	
	
		rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
		u = f1-f3+f5-f6-f7+f8;
		v = f2-f4+f5+f6-f7-f8;

		if(im == 2){
			u = 0.0f;
			v = uMax;
			rho = u+(f0+f2+f4+2.0f*f3+2.0f*f6+2.0f*f7);
			//f1 = f3+4.0f*u/6.0f;
			f1 = f3+u*0.66666667f;
	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;	
		}
		else if(im == 3){//north
			u = uMax;
			v = 0.0f;
			rho = -v+(f0+f1+f3+2.0f*f6+2.0f*f2+2.0f*f5);
			//f4 = f2-4.0f*v/6.0f;
			f4 = f2-v*0.66666667f;
	        f7 = f5+0.5f*(f1-f3)-u*0.5f+v*0.166666667f;
	        f8 = f6-0.5f*(f1-f3)+u*0.5f+v*0.166666667f;	
		}
		collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,rho,u,v,omega);

	f0A[y*pitch+x]=f0;
	f1B[y*pitch+x]=f1;
	f2B[y*pitch+x]=f2;
	f3B[y*pitch+x]=f3;
	f4B[y*pitch+x]=f4;
	f5B[y*pitch+x]=f5;
	f6B[y*pitch+x]=f6;
	f7B[y*pitch+x]=f7;
	f8B[y*pitch+x]=f8;


	}
}

__global__ void initialize(float *f0, float *f1, float *f2,
							float *f3, float *f4, float *f5,
							float *f6, float *f7, float *f8,
							int n, int pitch)
{
	int i;
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	i = threadIdx.y*blockDim.x+threadIdx.x;
	i += blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	i += blockDim.x*blockDim.y*blockIdx.x;
	//i = y*blockDim.x+x;
	//f1[y*pitch+x] = tex2D(texRef_f1,x,y);
	float u,v,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;
	feq = 4.0f/9.0f*(rho-1.5f*usqr);
	f0[i] = feq;
	feq = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[y*pitch+x] = feq;
	feq = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[y*pitch+x] = feq;
	feq = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[y*pitch+x] = feq;
	feq = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[y*pitch+x] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[y*pitch+x] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[y*pitch+x] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[y*pitch+x] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[y*pitch+x] = feq;
}




int main(int argc, char *argv[])
{

	float *f0_h, *f1_h, *f2_h, *f3_h, *f4_h, *f5_h, *f6_h, *f7_h, *f8_h;
	float *f0_d, *f1_dA, *f2_dA, *f3_dA, *f4_dA, *f5_dA, *f6_dA, *f7_dA, *f8_dA;
	float        *f1_dB, *f2_dB, *f3_dB, *f4_dB, *f5_dB, *f6_dB, *f7_dB, *f8_dB;
	int *image_d, *image_h;

	ofstream output;
	output.open ("LBM1_out.dat");


	size_t memsize, memsize_int;
	size_t pitch;
	int i, tMax, n, nBlocks, xDim, yDim;
	float Re, omega, uMax, CharLength;
	
	int BLOCKSIZEx = 16;
	int BLOCKSIZEy = 16;
	xDim = 1024;//32;
	yDim = 1024;//32;
	tMax = 500;
	Re = 500.f;//100.f;
	uMax = 0.08f;
	CharLength = xDim-2.f;
	omega = 1.0f/(3.0f*(uMax*CharLength/Re)+0.5f);

	cout<<"omega: "<<omega<<endl;

	nBlocks = (xDim/BLOCKSIZEx+xDim%BLOCKSIZEx)*(yDim/BLOCKSIZEy+yDim%BLOCKSIZEy);
	int B = BLOCKSIZEx*BLOCKSIZEy;
	n = nBlocks*B;//block*dimx*dimy

	//CUT_DEVICE_INIT(argc,argv);

	memsize = n*sizeof(float);
	memsize_int = n*sizeof(int);

	f0_h = (float *)malloc(memsize);
	f1_h = (float *)malloc(memsize);
	f2_h = (float *)malloc(memsize);
	f3_h = (float *)malloc(memsize);
	f4_h = (float *)malloc(memsize);
	f5_h = (float *)malloc(memsize);
	f6_h = (float *)malloc(memsize);
	f7_h = (float *)malloc(memsize);
	f8_h = (float *)malloc(memsize);
	image_h = (int *)malloc(memsize_int);

	//hipMalloc((void **) &f0_d, memsize);
	//CUDA_SAFE_CALL(hipMalloc((void **) &f1_d, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f1_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f2_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f3_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f4_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f5_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f6_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f7_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f8_dA, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f1_dB, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f2_dB, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f3_dB, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f4_dB, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f5_dB, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f6_dB, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f7_dB, memsize));
//	CUDA_SAFE_CALL(hipMalloc((void **) &f8_dB, memsize));
//	pitch = xDim*sizeof(float);
	hipMallocPitch((void **) &f0_d , &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f1_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f2_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f3_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f4_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f5_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f6_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f7_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f8_dA, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f1_dB, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f2_dB, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f3_dB, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f4_dB, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f5_dB, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f6_dB, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f7_dB, &pitch, xDim*sizeof(float), yDim);
	hipMallocPitch((void **) &f8_dB, &pitch, xDim*sizeof(float), yDim);
	
	//CUDA_SAFE_CALL(hipMalloc((void **) &f2_d, memsize));
	//CUDA_SAFE_CALL(hipMalloc((void **) &f3_d, memsize));
	//CUDA_SAFE_CALL(hipMalloc((void **) &f4_d, memsize));
	//CUDA_SAFE_CALL(hipMalloc((void **) &f5_d, memsize));
	//CUDA_SAFE_CALL(hipMalloc((void **) &f6_d, memsize));
	//CUDA_SAFE_CALL(hipMalloc((void **) &f7_d, memsize));
	//CUDA_SAFE_CALL(hipMalloc((void **) &f8_d, memsize));
	hipMalloc((void **) &image_d, memsize_int);

	for (i = 0; i < n; i++)
	{
		int x = i%xDim;
		int y = i/xDim;
		f0_h[i] = i;
		f1_h[i] = n-i;
		f2_h[i] = i;
		f3_h[i] = i;
		f4_h[i] = i;
		f5_h[i] = i;
		f6_h[i] = i;
		f7_h[i] = i;
		f8_h[i] = i;
		image_h[i] = 0;
		if(x < 1) image_h[i] = 1;//DirichletWest
		if(x > xDim-2) image_h[i] = 1;//BB
		if(y < 1) image_h[i] = 1;//BB
		if(y > yDim-2) image_h[i] = 3;//BB

	}

	
	//hipMemcpy(f0_d, f0_h, memsize, hipMemcpyHostToDevice);
	//CUDA_SAFE_CALL(hipMemcpy(f1_d, f1_h, memsize, hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f1_dA,f1_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f2_dA,f2_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f3_dA,f3_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f4_dA,f4_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f5_dA,f5_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f6_dA,f6_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f7_dA,f7_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f8_dA,f8_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f1_dB,f1_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f2_dB,f2_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f3_dB,f3_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f4_dB,f4_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f5_dB,f5_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f6_dB,f6_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f7_dB,f7_h,memsize,hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(f8_dB,f8_h,memsize,hipMemcpyHostToDevice));
	hipMemcpy2D(f0_d ,pitch,f1_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f1_dA,pitch,f1_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f2_dA,pitch,f2_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f3_dA,pitch,f3_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f4_dA,pitch,f4_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f5_dA,pitch,f5_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f6_dA,pitch,f6_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f7_dA,pitch,f7_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f8_dA,pitch,f8_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f1_dB,pitch,f1_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f2_dB,pitch,f2_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f3_dB,pitch,f3_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f4_dB,pitch,f4_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f5_dB,pitch,f5_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f6_dB,pitch,f6_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f7_dB,pitch,f7_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	hipMemcpy2D(f8_dB,pitch,f8_h,xDim*sizeof(float),xDim*sizeof(float),yDim,hipMemcpyHostToDevice);
	
	//CUDA_SAFE_CALL(hipMemcpy(f2_d, f2_h, memsize, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(f3_d, f3_h, memsize, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(f4_d, f4_h, memsize, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(f5_d, f5_h, memsize, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(f6_d, f6_h, memsize, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(f7_d, f7_h, memsize, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(f8_d, f8_h, memsize, hipMemcpyHostToDevice));
	hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	//cout<<(int)(pitch/sizeof(float))<<endl;


    dim3 threads(BLOCKSIZEx, BLOCKSIZEy);
    dim3 grid(xDim/BLOCKSIZEx,yDim/BLOCKSIZEy);

	cout<<"nBlocks:"<<nBlocks<<endl;
	
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f1A.filterMode = hipFilterModePoint;
	texRef_f2A.filterMode = hipFilterModePoint;
	texRef_f3A.filterMode = hipFilterModePoint;
	texRef_f4A.filterMode = hipFilterModePoint;
	texRef_f5A.filterMode = hipFilterModePoint;
	texRef_f6A.filterMode = hipFilterModePoint;
	texRef_f7A.filterMode = hipFilterModePoint;
	texRef_f8A.filterMode = hipFilterModePoint;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;
	texRef_f1B.filterMode = hipFilterModePoint;
	texRef_f2B.filterMode = hipFilterModePoint;
	texRef_f3B.filterMode = hipFilterModePoint;
	texRef_f4B.filterMode = hipFilterModePoint;
	texRef_f5B.filterMode = hipFilterModePoint;
	texRef_f6B.filterMode = hipFilterModePoint;
	texRef_f7B.filterMode = hipFilterModePoint;
	texRef_f8B.filterMode = hipFilterModePoint;
	
	initialize<<<grid, threads>>>(f0_d, f1_dA, f2_dA, f3_dA, f4_dA, f5_dA, f6_dA, f7_dA, f8_dA,
									n,(int)(pitch/sizeof(float)));

	hipBindTexture2D(0,&texRef_f1A, f1_dA,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f2A, f2_dA,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f3A, f3_dA,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f4A, f4_dA,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f5A, f5_dA,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f6A, f6_dA,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f7A, f7_dA,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f8A, f8_dA,&desc,xDim,yDim,pitch);

	hipBindTexture2D(0,&texRef_f1B, f1_dB,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f2B, f2_dB,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f3B, f3_dB,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f4B, f4_dB,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f5B, f5_dB,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f6B, f6_dB,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f7B, f7_dB,&desc,xDim,yDim,pitch);
	hipBindTexture2D(0,&texRef_f8B, f8_dB,&desc,xDim,yDim,pitch);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<tMax; t=t+2){
	//for(int t = 0; t<tMax; t=t+1){
		//mrt_d<<<grid, threads>>>(f0_d,f1_d,f2_d,f3_d,f4_d,f5_d,f6_d,f7_d,f8_d,n,image_d,omega,uMax);
		//test<<<grid, threads>>>(f0_d,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,
		mrt_d_textAB<<<grid, threads>>>(f0_d,f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,
									        f1_dB,f2_dB,f3_dB,f4_dB,f5_dB,f6_dB,f7_dB,f8_dB,
										    n,image_d,omega,uMax,(int)(pitch/sizeof(float)));


		//test<<<grid, threads>>>(f0_d,f1_dB,f2_dB,f3_dB,f4_dB,f5_dB,f6_dB,f7_dB,f8_dB,
		mrt_d_textBA<<<grid, threads>>>(f0_d,f1_dB,f2_dB,f3_dB,f4_dB,f5_dB,f6_dB,f7_dB,f8_dB,
										   f1_dA,f2_dA,f3_dA,f4_dA,f5_dA,f6_dA,f7_dA,f8_dA,
										   n,image_d,omega,uMax,(int)(pitch/sizeof(float)));

		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();
	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	cout<<"Time taken for main kernel: "<<restime<<" ("<<double(xDim*yDim*double(tMax/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<xDim<<","<<yDim<<","<<tMax<<","<<restime<<endl;


		hipUnbindTexture(texRef_f1A);
		hipUnbindTexture(texRef_f2A);
		hipUnbindTexture(texRef_f3A);
		hipUnbindTexture(texRef_f4A);
		hipUnbindTexture(texRef_f5A);
		hipUnbindTexture(texRef_f6A);
		hipUnbindTexture(texRef_f7A);
		hipUnbindTexture(texRef_f8A);
		hipUnbindTexture(texRef_f1B);
		hipUnbindTexture(texRef_f2B);
		hipUnbindTexture(texRef_f3B);
		hipUnbindTexture(texRef_f4B);
		hipUnbindTexture(texRef_f5B);
		hipUnbindTexture(texRef_f6B);
		hipUnbindTexture(texRef_f7B);
		hipUnbindTexture(texRef_f8B);


	//CUT_CHECK_ERROR("Kernel execution failed");

		//hipMemcpy(f0_h, f0_d, memsize, hipMemcpyDeviceToHost);
		//CUDA_SAFE_CALL(hipMemcpy(f1_h, f1_d, memsize, hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f1_h,f1_dB,memsize,hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f2_h,f2_dB,memsize,hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f3_h,f3_dB,memsize,hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f4_h,f4_dB,memsize,hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f5_h,f5_dB,memsize,hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f6_h,f6_dB,memsize,hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f7_h,f7_dB,memsize,hipMemcpyDeviceToHost));
//		CUDA_SAFE_CALL(hipMemcpy(f8_h,f8_dB,memsize,hipMemcpyDeviceToHost));
		hipMemcpy2D(f0_h,xDim*sizeof(float),f0_d ,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f1_h,xDim*sizeof(float),f1_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f2_h,xDim*sizeof(float),f2_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f3_h,xDim*sizeof(float),f3_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f4_h,xDim*sizeof(float),f4_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f5_h,xDim*sizeof(float),f5_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f6_h,xDim*sizeof(float),f6_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f7_h,xDim*sizeof(float),f7_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		hipMemcpy2D(f8_h,xDim*sizeof(float),f8_dB,pitch,xDim*sizeof(float),yDim,hipMemcpyDeviceToHost);
		//CUDA_SAFE_CALL(hipMemcpy(f2_h, f2_d, memsize, hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(f3_h, f3_d, memsize, hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(f4_h, f4_d, memsize, hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(f5_h, f5_d, memsize, hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(f6_h, f6_d, memsize, hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(f7_h, f7_d, memsize, hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(f8_h, f8_d, memsize, hipMemcpyDeviceToHost));
		output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
		output<<"ZONE F=POINT, I="<<xDim<<", J="<<yDim<<"\n";
		
//		for(i = 0; i<n; i++)
//		{
		int row = 0;
		int col = 0;
		i = 0;
		//int rowB, colB;
		//float xcoord, ycoord;
		float rho, u, v;
		rho = 0;
		u = 0;
		v = 0;

			for(row = 0; row<yDim; row++){
				for(col = 0; col<xDim; col++){
					i = row*xDim+col;
					rho = f0_h[i]+f1_h[i]+f2_h[i]+f3_h[i]+f4_h[i]+f5_h[i]+f6_h[i]+f7_h[i]+f8_h[i];
					u = f1_h[i]-f3_h[i]+f5_h[i]-f6_h[i]-f7_h[i]+f8_h[i];
					v = f2_h[i]-f4_h[i]+f5_h[i]+f6_h[i]-f7_h[i]-f8_h[i];
					output<<col<<", "<<row<<", "<<u<<","<<v<<","<<rho<<endl;
				}
			}


	free(f0_h);
	free(f1_h);
	free(f2_h);
	free(f3_h);
	free(f4_h);
	free(f5_h);
	free(f6_h);
	free(f7_h);
	free(f8_h);

	output.close();

	hipFree(f0_d);
	hipFree(f1_dA);
	hipFree(f2_dA);
	hipFree(f3_dA);
	hipFree(f4_dA);
	hipFree(f5_dA);
	hipFree(f6_dA);
	hipFree(f7_dA);
	hipFree(f8_dA);
	hipFree(f1_dB);
	hipFree(f2_dB);
	hipFree(f3_dB);
	hipFree(f4_dB);
	hipFree(f5_dB);
	hipFree(f6_dB);
	hipFree(f7_dB);
	hipFree(f8_dB);
	hipFree(image_d);

	return(0);

}
