#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;

//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

//int Xcoord(int i, int XDim){
//	int B = BLOCKSIZE*BLOCKSIZE;
//	return (i%B)%BLOCKSIZE+((i/B)%(XDim/BLOCKSIZE))*BLOCKSIZE;
//}
//int Ycoord(int i, int XDim){
//	int B = BLOCKSIZE*BLOCKSIZE;
//	return (i%B)/BLOCKSIZE+((i/B)/(XDim/BLOCKSIZE))*BLOCKSIZE;
//}

//texture
//texture<float,1,hipReadModeElementType> texRef_f1;
texture<float,3,hipReadModeElementType> texRef_f1A;
texture<float,3,hipReadModeElementType> texRef_f2A;
texture<float,3,hipReadModeElementType> texRef_f3A;
texture<float,3,hipReadModeElementType> texRef_f4A;
texture<float,3,hipReadModeElementType> texRef_f5A;
texture<float,3,hipReadModeElementType> texRef_f6A;
texture<float,3,hipReadModeElementType> texRef_f7A;
texture<float,3,hipReadModeElementType> texRef_f8A;
texture<float,3,hipReadModeElementType> texRef_f9A;
texture<float,3,hipReadModeElementType> texRef_f10A;
texture<float,3,hipReadModeElementType> texRef_f11A;
texture<float,3,hipReadModeElementType> texRef_f12A;
texture<float,3,hipReadModeElementType> texRef_f13A;
texture<float,3,hipReadModeElementType> texRef_f14A;
texture<float,3,hipReadModeElementType> texRef_f15A;
texture<float,3,hipReadModeElementType> texRef_f16A;
texture<float,3,hipReadModeElementType> texRef_f17A;
texture<float,3,hipReadModeElementType> texRef_f18A;


surface<void, 3> surfRef;

texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;

#include <sys/time.h>
#include <time.h>



int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}


//__global__ void test(float *f0A, float *f1A, float *f2A,
//						   float *f3A, float *f4A, float *f5A,
//						   float *f6A, float *f7A, float *f8A,
//						               float *f1B, float *f2B,
//						   float *f3B, float *f4B, float *f5B,
//						   float *f6B, float *f7B, float *f8B, //int pitch)
//						   int n, int *image, float omega, float uMax, int pitch)
//{
//	int x = blockIdx.x*blockDim.x+threadIdx.x;
//	int y = blockIdx.y*blockDim.y+threadIdx.y;
////	int i = x+y*blockDim.x*gridDim.x;
////	j = threadIdx.y*blockDim.x+threadIdx.x;//local block index (for shared mem)
////	i = j+blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
////	i += blockDim.x*blockDim.y*blockIdx.x;//global memory index
//
//		//f0A[i] = f;
//		float f1,f2,f3,f4,f5,f6,f7,f8;
////		f1=tex2D(texRef_f1,x+1,y  );
////		f2=tex2D(texRef_f2,x-1,y  );
////		f3=tex2D(texRef_f3,x  ,y+1);
////		f4=tex2D(texRef_f4,x  ,y-1);
////		f5=tex2D(texRef_f5,x+1,y+1);
////		f7=tex2D(texRef_f7,x-1,y-1);
////		f6=tex2D(texRef_f6,x-1,y+1);
////		f8=tex2D(texRef_f8,x+1,y-1);
//		//if(image[i] == 0){
//
////		f1 = tex2D(texRef_f1,x-1,y  );
////		f2 = tex2D(texRef_f2,x  ,y-1);
////		f3 = tex2D(texRef_f3,x+1,y  );
////		f4 = tex2D(texRef_f4,x  ,y+1);
////		f5 = tex2D(texRef_f5,x-1,y-1);
////		f6 = tex2D(texRef_f6,x+1,y-1);
////		f7 = tex2D(texRef_f7,x+1,y+1);
////		f8 = tex2D(texRef_f8,x-1,y+1);
//
////		f1 = f1A[x-1+y*pitch];
////		f2 = f2A[x  +(y-1)*pitch];
////		f3 = f3A[x+1+y*pitch];
////		f4 = f4A[x  +(y+1)*pitch];
////		f5 = f5A[x-1+(y-1)*pitch];
////		f6 = f6A[x+1+(y-1)*pitch];
////		f7 = f7A[x+1+(y+1)*pitch];
////		f8 = f8A[x-1+(y+1)*pitch];
//
////		f1=f1A[x+1+(y  )*pitch];
////		f2=f2A[x-1+(y  )*pitch];
////		f3=f3A[x  +(y+1)*pitch];
////		f4=f4A[x  +(y-1)*pitch];
////		f5=f5A[x+1+(y+1)*pitch];
////		f7=f7A[x-1+(y-1)*pitch];
////		f6=f6A[x-1+(y+1)*pitch];
////		f8=f8A[x+1+(y-1)*pitch];
//		f1=f1A[x+y*pitch];
//		f2=f2A[x+y*pitch];
//		f3=f3A[x+y*pitch];
//		f4=f4A[x+y*pitch];
//		f5=f5A[x+y*pitch];
//		f6=f6A[x+y*pitch];
//		f7=f7A[x+y*pitch];
//		f8=f8A[x+y*pitch];
////		f1B[x+y*pitch]=f1+1;
////		f3B[x+y*pitch]=f3+1;
////		f2B[x+y*pitch]=f2+1;
////		f4B[x+y*pitch]=f4+1;
////		f5B[x+y*pitch]=f5+1;
////		f6B[x+y*pitch]=f6+1;
////		f7B[x+y*pitch]=f7+1;
////		f8B[x+y*pitch]=f8+1;
//		//}
////		f1=f1A[x+y*pitch];
////		f3=f3A[x+y*pitch];
////		f2=f2A[x+y*pitch];
////		f4=f4A[x+y*pitch];
////		f5=f5A[x+y*pitch];
////		f7=f7A[x+y*pitch];
////		f6=f6A[x+y*pitch];
////		f8=f8A[x+y*pitch];
//		f1B[x+y*pitch]=f1;
//		f2B[x+y*pitch]=f2;
//		f3B[x+y*pitch]=f3;
//		f4B[x+y*pitch]=f4;
//		f5B[x+y*pitch]=f5;
//		f6B[x+y*pitch]=f6;
//		f7B[x+y*pitch]=f7;
//		f8B[x+y*pitch]=f8;
//
//
//}
//__global__ void mrt_d_textAB(float *f0A, float *f1A, float *f2A,
//						   float *f3A, float *f4A, float *f5A,
//						   float *f6A, float *f7A, float *f8A,
//						               float *f1B, float *f2B,
//						   float *f3B, float *f4B, float *f5B,
//						   float *f6B, float *f7B, float *f8B,
//						   int n, int *image, float omega, float uMax, int pitch)
//{
//	int x = blockIdx.x*blockDim.x+threadIdx.x;
//	int y = blockIdx.y*blockDim.y+threadIdx.y;
//	int i = x+y*blockDim.x*gridDim.x;
//	float u,v,rho;//,usqr;
//	int im = image[i];
//	if(im == 1){//BB
//		//f0A[i] = f0A[i];
//		f0A[x+y*pitch]=f0A[x+y*pitch];
//		f1B[x+y*pitch]=tex2D(texRef_f3A,x+1,y  );
//		f3B[x+y*pitch]=tex2D(texRef_f1A,x-1,y  );
//		f2B[x+y*pitch]=tex2D(texRef_f4A,x  ,y+1);
//		f4B[x+y*pitch]=tex2D(texRef_f2A,x  ,y-1);
//		f5B[x+y*pitch]=tex2D(texRef_f7A,x+1,y+1);
//		f7B[x+y*pitch]=tex2D(texRef_f5A,x-1,y-1);
//		f6B[x+y*pitch]=tex2D(texRef_f8A,x-1,y+1);
//		f8B[x+y*pitch]=tex2D(texRef_f6A,x+1,y-1);
//
////		f0A[x+y*pitch] = f0A[x+y*pitch];
////		f1B[x+y*pitch] = f1A[x+1+(y  )*pitch];
////		f3B[x+y*pitch] = f2A[x-1+(y  )*pitch];
////		f2B[x+y*pitch] = f3A[x  +(y+1)*pitch];
////		f4B[x+y*pitch] = f4A[x  +(y-1)*pitch];
////		f5B[x+y*pitch] = f5A[x+1+(y+1)*pitch];
////		f7B[x+y*pitch] = f6A[x-1+(y-1)*pitch];
////		f6B[x+y*pitch] = f7A[x-1+(y+1)*pitch];
////		f8B[x+y*pitch] = f8A[x+1+(y-1)*pitch];
//
////		f0A[x+y*pitch] = f0A[x+y*pitch];
////		f1B[x+y*pitch] = f1A[x+(y)*pitch];
////		f3B[x+y*pitch] = f2A[x+(y)*pitch];
////		f2B[x+y*pitch] = f3A[x+(y)*pitch];
////		f4B[x+y*pitch] = f4A[x+(y)*pitch];
////		f5B[x+y*pitch] = f5A[x+(y)*pitch];
////		f7B[x+y*pitch] = f6A[x+(y)*pitch];
////		f6B[x+y*pitch] = f7A[x+(y)*pitch];
////		f8B[x+y*pitch] = f8A[x+(y)*pitch];
//
//
//	}
//	else{
//		float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//		//float u,v,rho,feq,usqr;
////		float meq1,meq2,meq4,meq6,meq7,meq8;
////		f0 = f0A[i];
//
//		f1 = tex2D(texRef_f1A,x-1,y  );
//		f2 = tex2D(texRef_f2A,x  ,y-1);
//		f3 = tex2D(texRef_f3A,x+1,y  );
//		f4 = tex2D(texRef_f4A,x  ,y+1);
//		f5 = tex2D(texRef_f5A,x-1,y-1);
//		f6 = tex2D(texRef_f6A,x+1,y-1);
//		f7 = tex2D(texRef_f7A,x+1,y+1);
//		f8 = tex2D(texRef_f8A,x-1,y+1);
//		f0 = f0A[x  +y*pitch];
//
////		f0 = f0A[x  +y*pitch];
////		f1 = f1A[x+y*pitch];
////		f2 = f2A[x+y*pitch];
////		f3 = f3A[x+y*pitch];
////		f4 = f4A[x+y*pitch];
////		f5 = f5A[x+y*pitch];
////		f6 = f6A[x+y*pitch];
////		f7 = f7A[x+y*pitch];
////		f8 = f8A[x+y*pitch];
//	
////		f0 = f0A[x  +y*pitch];
////		f1 = f1A[x-1+y*pitch];
////		f2 = f2A[x  +(y-1)*pitch];
////		f3 = f3A[x+1+y*pitch];
////		f4 = f4A[x  +(y+1)*pitch];
////		f5 = f5A[x-1+(y-1)*pitch];
////		f6 = f6A[x+1+(y-1)*pitch];
////		f7 = f7A[x+1+(y+1)*pitch];
////		f8 = f8A[x-1+(y+1)*pitch];
//	
//		rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
//		u = f1-f3+f5-f6-f7+f8;
//		v = f2-f4+f5+f6-f7-f8;
//		if(im == 2){
//			u = 0.0f;
//			v = uMax;
//			rho = u+(f0+f2+f4+2.0f*f3+2.0f*f6+2.0f*f7);
//			//f1 = f3+4.0f*u/6.0f;
//			f1 = f3+u/1.5f;
//	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u/6.0f;
//	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u/6.0f;	
//		}
//		else if(im == 3){//north
//			u = uMax;
//			v = 0.0f;
//			rho = -v+(f0+f1+f3+2.0f*f6+2.0f*f2+2.0f*f5);
//			//f4 = f2-4.0f*v/6.0f;
//			f4 = f2-v/1.5f;
//	        f7 = f5+0.5f*(f1-f3)-u*0.5f+v/6.0f;
//	        f8 = f6-0.5f*(f1-f3)+u*0.5f+v/6.0f;	
//		}
////		usqr = u*u+v*v;
//		float m1,m2,m4,m6,m7,m8;
//		m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
//		m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
//		m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
//		m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
//		m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
//		m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq
//
////		float Q11 = (m7-meq7)*0.5f;
////		float Q22 = -Q11;
////		float Q12 = m8-meq8;
////		float Q21 = Q12;
////		float Q = Q11*Q11+Q12*Q12+Q21*Q21+Q22*Q22;
////	
////		float Cs = 0.01f;
////		float nu0 = ((1.0f/omega)-0.5f)/3.0f;
////		float Smag = (sqrt((2.0f*rho/(3.0f*omega))*(2.0f*rho/(3.0f*omega))
////				+4.0f*2.0f*rho*Cs*sqrt(Q))-2.0f*rho/(3.0f*omega))/(4.0f*rho*Cs);
////		float OmegaStar = omega;//1.0f/(3.0f*(nu0+Cs*Smag)+0.5f);
//
//f0A[y*pitch+x]=f0-(-m1+m2)/9.0f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
//f1B[y*pitch+x]=f1-(-m1-2.0f*(m2+m4)+m7*omega*9.0f)/36.0f;
//f2B[y*pitch+x]=f2-(-m1-2.f*m2-6.f*m6-m7*omega*9.0f)/36.0f;
//f3B[y*pitch+x]=f3-(-m1-2.f*m2+6.f*m4+m7*omega*9.0f)/36.0f;
//f4B[y*pitch+x]=f4-(-m1-2.f*m2+6.f*m6-m7*omega*9.0f)/36.0f;
//f5B[y*pitch+x]=f5-(2.f*m1+m2+3.f*m4+3.f*m6+m8*omega*9.0f)/36.0f;
//f6B[y*pitch+x]=f6-(2.f*m1+m2-3.f*m4+3.f*m6-m8*omega*9.0f)/36.0f;
//f7B[y*pitch+x]=f7-(2.f*m1+m2-3.f*m4-3.f*m6+m8*omega*9.0f)/36.0f;
//f8B[y*pitch+x]=f8-(2.f*m1+m2+3.f*m4-3.f*m6-m8*omega*9.0f)/36.0f;
////f0A[y*pitch+x]=f0;
////f1B[y*pitch+x]=f1;
////f2B[y*pitch+x]=f2;
////f3B[y*pitch+x]=f3;
////f4B[y*pitch+x]=f4;
////f5B[y*pitch+x]=f5;
////f6B[y*pitch+x]=f6;
////f7B[y*pitch+x]=f7;
////f8B[y*pitch+x]=f8;
//	}
//}
//
//__global__ void mrt_d_textBA(float *f0A, float *f1A, float *f2A,
//						   float *f3A, float *f4A, float *f5A,
//						   float *f6A, float *f7A, float *f8A,
//						               float *f1B, float *f2B,
//						   float *f3B, float *f4B, float *f5B,
//						   float *f6B, float *f7B, float *f8B,
//						   int n, int *image, float omega, float uMax, int pitch)
//{
//	int x = blockIdx.x*blockDim.x+threadIdx.x;
//	int y = blockIdx.y*blockDim.y+threadIdx.y;
//	int i = x+y*blockDim.x*gridDim.x;
//	float u,v,rho;//,usqr;
//	int im = image[i];
//	if(im == 1){//BB
//		//f0A[i] = f0A[i];
//		f0A[x+y*pitch]=f0A[x+y*pitch];
//		f1B[x+y*pitch]=tex2D(texRef_f3B,x+1,y  );
//		f3B[x+y*pitch]=tex2D(texRef_f1B,x-1,y  );
//		f2B[x+y*pitch]=tex2D(texRef_f4B,x  ,y+1);
//		f4B[x+y*pitch]=tex2D(texRef_f2B,x  ,y-1);
//		f5B[x+y*pitch]=tex2D(texRef_f7B,x+1,y+1);
//		f7B[x+y*pitch]=tex2D(texRef_f5B,x-1,y-1);
//		f6B[x+y*pitch]=tex2D(texRef_f8B,x-1,y+1);
//		f8B[x+y*pitch]=tex2D(texRef_f6B,x+1,y-1);
//
////		f0A[x+y*pitch] = f0A[x+y*pitch];
////		f1B[x+y*pitch] = f1A[x+1+(y  )*pitch];
////		f3B[x+y*pitch] = f2A[x-1+(y  )*pitch];
////		f2B[x+y*pitch] = f3A[x  +(y+1)*pitch];
////		f4B[x+y*pitch] = f4A[x  +(y-1)*pitch];
////		f5B[x+y*pitch] = f5A[x+1+(y+1)*pitch];
////		f7B[x+y*pitch] = f6A[x-1+(y-1)*pitch];
////		f6B[x+y*pitch] = f7A[x-1+(y+1)*pitch];
////		f8B[x+y*pitch] = f8A[x+1+(y-1)*pitch];
//
////		f0A[x+y*pitch] = f0A[x+y*pitch];
////		f1B[x+y*pitch] = f1A[x+(y)*pitch];
////		f3B[x+y*pitch] = f2A[x+(y)*pitch];
////		f2B[x+y*pitch] = f3A[x+(y)*pitch];
////		f4B[x+y*pitch] = f4A[x+(y)*pitch];
////		f5B[x+y*pitch] = f5A[x+(y)*pitch];
////		f7B[x+y*pitch] = f6A[x+(y)*pitch];
////		f6B[x+y*pitch] = f7A[x+(y)*pitch];
////		f8B[x+y*pitch] = f8A[x+(y)*pitch];
//
//
//	}
//	else{
//		float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//		//float u,v,rho,feq,usqr;
////		float meq1,meq2,meq4,meq6,meq7,meq8;
////		f0 = f0A[i];
//
//		f1 = tex2D(texRef_f1B,x-1,y  );
//		f2 = tex2D(texRef_f2B,x  ,y-1);
//		f3 = tex2D(texRef_f3B,x+1,y  );
//		f4 = tex2D(texRef_f4B,x  ,y+1);
//		f5 = tex2D(texRef_f5B,x-1,y-1);
//		f6 = tex2D(texRef_f6B,x+1,y-1);
//		f7 = tex2D(texRef_f7B,x+1,y+1);
//		f8 = tex2D(texRef_f8B,x-1,y+1);
//		f0 = f0A[x  +y*pitch];
//
////		f0 = f0A[x  +y*pitch];
////		f1 = f1A[x+y*pitch];
////		f2 = f2A[x+y*pitch];
////		f3 = f3A[x+y*pitch];
////		f4 = f4A[x+y*pitch];
////		f5 = f5A[x+y*pitch];
////		f6 = f6A[x+y*pitch];
////		f7 = f7A[x+y*pitch];
////		f8 = f8A[x+y*pitch];
//	
////		f0 = f0A[x  +y*pitch];
////		f1 = f1A[x-1+y*pitch];
////		f2 = f2A[x  +(y-1)*pitch];
////		f3 = f3A[x+1+y*pitch];
////		f4 = f4A[x  +(y+1)*pitch];
////		f5 = f5A[x-1+(y-1)*pitch];
////		f6 = f6A[x+1+(y-1)*pitch];
////		f7 = f7A[x+1+(y+1)*pitch];
////		f8 = f8A[x-1+(y+1)*pitch];
//	
//		rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
//		u = f1-f3+f5-f6-f7+f8;
//		v = f2-f4+f5+f6-f7-f8;
//		if(im == 2){
//			u = 0.0f;
//			v = uMax;
//			rho = u+(f0+f2+f4+2.0f*f3+2.0f*f6+2.0f*f7);
//			//f1 = f3+4.0f*u/6.0f;
//			f1 = f3+u/1.5f;
//	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u/6.0f;
//	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u/6.0f;	
//		}
//		else if(im == 3){//north
//			u = uMax;
//			v = 0.0f;
//			rho = -v+(f0+f1+f3+2.0f*f6+2.0f*f2+2.0f*f5);
//			//f4 = f2-4.0f*v/6.0f;
//			f4 = f2-v/1.5f;
//	        f7 = f5+0.5f*(f1-f3)-u*0.5f+v/6.0f;
//	        f8 = f6-0.5f*(f1-f3)+u*0.5f+v/6.0f;	
//		}
////		usqr = u*u+v*v;
//		float m1,m2,m4,m6,m7,m8;
//		m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
//		m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
//		m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
//		m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
//		m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
//		m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq
//
////		float Q11 = (m7-meq7)*0.5f;
////		float Q22 = -Q11;
////		float Q12 = m8-meq8;
////		float Q21 = Q12;
////		float Q = Q11*Q11+Q12*Q12+Q21*Q21+Q22*Q22;
////	
////		float Cs = 0.01f;
////		float nu0 = ((1.0f/omega)-0.5f)/3.0f;
////		float Smag = (sqrt((2.0f*rho/(3.0f*omega))*(2.0f*rho/(3.0f*omega))
////				+4.0f*2.0f*rho*Cs*sqrt(Q))-2.0f*rho/(3.0f*omega))/(4.0f*rho*Cs);
////		float OmegaStar = omega;//1.0f/(3.0f*(nu0+Cs*Smag)+0.5f);
//
//f0A[y*pitch+x]=f0-(-m1+m2)/9.0f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
//f1B[y*pitch+x]=f1-(-m1-2.0f*(m2+m4)+m7*omega*9.0f)/36.0f;
//f2B[y*pitch+x]=f2-(-m1-2.f*m2-6.f*m6-m7*omega*9.0f)/36.0f;
//f3B[y*pitch+x]=f3-(-m1-2.f*m2+6.f*m4+m7*omega*9.0f)/36.0f;
//f4B[y*pitch+x]=f4-(-m1-2.f*m2+6.f*m6-m7*omega*9.0f)/36.0f;
//f5B[y*pitch+x]=f5-(2.f*m1+m2+3.f*m4+3.f*m6+m8*omega*9.0f)/36.0f;
//f6B[y*pitch+x]=f6-(2.f*m1+m2-3.f*m4+3.f*m6-m8*omega*9.0f)/36.0f;
//f7B[y*pitch+x]=f7-(2.f*m1+m2-3.f*m4-3.f*m6+m8*omega*9.0f)/36.0f;
//f8B[y*pitch+x]=f8-(2.f*m1+m2+3.f*m4-3.f*m6-m8*omega*9.0f)/36.0f;
////f0A[y*pitch+x]=f0;
////f1B[y*pitch+x]=f1;
////f2B[y*pitch+x]=f2;
////f3B[y*pitch+x]=f3;
////f4B[y*pitch+x]=f4;
////f5B[y*pitch+x]=f5;
////f6B[y*pitch+x]=f6;
////f7B[y*pitch+x]=f7;
////f8B[y*pitch+x]=f8;
//	}
//}
//

__global__ void initialize(void* f0in, void* f1out, void* f2out,
							void* f3out, void* f4out, void* f5out,
							void* f6out, void* f7out, void* f8out, void* f9out,
							void* f10out, void* f11out, void* f12out,
							void* f13out, void* f14out, void* f15out,
							void* f16out, void* f17out, void* f18out,
							int w, int h, int pitch)//pitch in elements
//__global__ void initialize(void** f0in, void** f1in, 
//							int w, int h, int pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	i = threadIdx.y*blockDim.x+threadIdx.x;
//	i += blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
//	i += blockDim.x*blockDim.y*blockIdx.x;
	float* f0 = (float *)(f0in);
	float* f1 = (float *)(f1out);
	int i = x+y*w+z*w*h;//index on linear mem
	int j = x+y*pitch+z*h*pitch;//index on padded mem (pitch in elements)
	float data = 0;
	surf3Dread(&data,surfRef,x,y,z);
	if(data == 10) data = 1;
	f1[j] = data;
	
	//i = y*blockDim.x+x;
	//f1[y*pitch+x] = tex2D(texRef_f1,x,y);
//	float u,v,rho,feq,usqr;
//	rho = 1.0f;
//	u = 0.0f;
//	v = 0.0f;
//	usqr = u*u+v*v;
//	feq = 4.0f/9.0f*(rho-1.5f*usqr);
//	f0[i] = feq;
//	feq = 1.0f/9.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//	f1[y*pitch+x] = feq;
//	feq = 1.0f/9.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//	f2[y*pitch+x] = feq;
//	feq = 1.0f/9.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//	f3[y*pitch+x] = feq;
//	feq = 1.0f/9.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//	f4[y*pitch+x] = feq;
//	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
//	f5[y*pitch+x] = feq;
//	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//	f6[y*pitch+x] = feq;
//	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//	f7[y*pitch+x] = feq;
//	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
//	f8[y*pitch+x] = feq;
}
__global__ void copytest(hipPitchedPtr devPitchedPtr, float * test_d, int w, int h, int d)
//__global__ void copytest(float *test)//, int w, int h, int d)
//__global__ void copytest(int * image)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	char* devPtr = (char*)devPitchedPtr.ptr;
	int pitch = devPitchedPtr.pitch;
//	int slicepitch = pitch*height;
////	int pitch = devPitchedPtr.pitch;
//	char *slice = devPtr + blockIdx.x*slicepitch;
	float* test = (float *)(devPtr);
//	//int slicePitch = pitch*extent.height;
	//int i = threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	int i = x+y*w+z*w*h;//index on linear mem
	//int j = threadIdx.x+threadIdx.y*pitch+threadIdx.z*blockDim.y;
	int j = x+y*pitch/sizeof(float)+z*h*pitch/sizeof(float);//index on padded mem
	//if(test[i] == 2)
	//test[0] = 2.f;//test[i];
	test_d[i] = test[j];
	test[j] += 100;

}



int main(int argc, char *argv[])
{

	float *f0_h, *f1_h, *f2_h, *f3_h, *f4_h, *f5_h, *f6_h, *f7_h, *f8_h, *f9_h;
	float *f10_h, *f11_h, *f12_h, *f13_h, *f14_h, *f15_h, *f16_h, *f17_h, *f18_h;
	hipPitchedPtr f0_dA, f1_dA, f2_dA, f3_dA, f4_dA, f5_dA, f6_dA, f7_dA, f8_dA, f9_dA;
	hipPitchedPtr f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA;
	int *image_d, *image_h;

	//hipPitchedPtr f0_d;

	float *test_h,*test_d;

	ofstream output;
	output.open ("LBM1_out.dat");

	size_t memsize, memsize_int;
	//size_t pitch;
	int i, n, nBlocks, xDim, yDim, zDim;
//	float Re, omega, uMax, CharLength;
	
	int BLOCKSIZEx = 8;
	int BLOCKSIZEy = 1;
	int BLOCKSIZEz = 1;
	xDim = 16;
	yDim = 8;
	zDim = 8;
//	tMax = 1;
//	Re = 10.f;//100.f;
//	uMax = 0.08f;
//	CharLength = xDim-2.f;
//	omega = 1.0f/(3.0f*(uMax*CharLength/Re)+0.5f);
//
//	cout<<"omega: "<<omega<<endl;

	nBlocks = (xDim/BLOCKSIZEx+xDim%BLOCKSIZEx)*(yDim/BLOCKSIZEy+yDim%BLOCKSIZEy)
				*(zDim/BLOCKSIZEz+zDim%BLOCKSIZEz);
	int B = BLOCKSIZEx*BLOCKSIZEy*BLOCKSIZEz;
	n = nBlocks*B;//block*dimx*dimy
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEx, BLOCKSIZEy, BLOCKSIZEz);
    dim3 grid(xDim/BLOCKSIZEx,yDim/BLOCKSIZEy,zDim/BLOCKSIZEz);

	memsize = n*sizeof(float);
	memsize_int = n*sizeof(int);

	hipExtent extent = make_hipExtent(xDim*sizeof(float),yDim,zDim);

	f0_h   = (float *)malloc(memsize);
	f1_h   = (float *)malloc(memsize);
	f2_h   = (float *)malloc(memsize);
	f3_h   = (float *)malloc(memsize);
	f4_h   = (float *)malloc(memsize);
	f5_h   = (float *)malloc(memsize);
	f6_h   = (float *)malloc(memsize);
	f7_h   = (float *)malloc(memsize);
	f8_h   = (float *)malloc(memsize);
	f9_h   = (float *)malloc(memsize);
	f10_h  = (float *)malloc(memsize);
	f11_h  = (float *)malloc(memsize);
	f12_h  = (float *)malloc(memsize);
	f13_h  = (float *)malloc(memsize);
	f14_h  = (float *)malloc(memsize);
	f15_h  = (float *)malloc(memsize);
	f16_h  = (float *)malloc(memsize);
	f17_h  = (float *)malloc(memsize);
	f18_h  = (float *)malloc(memsize);

	image_h = (int *)malloc(memsize_int);
	test_h = (float *)malloc(memsize);

	hipMalloc((void **) &test_d, memsize);
	hipMalloc((void **) &image_d, memsize_int);

	hipMalloc3D(&f0_dA ,  extent);
	hipMalloc3D(&f1_dA ,  extent);
	hipMalloc3D(&f2_dA ,  extent);
	hipMalloc3D(&f3_dA ,  extent);
	hipMalloc3D(&f4_dA ,  extent);
	hipMalloc3D(&f5_dA ,  extent);
	hipMalloc3D(&f6_dA ,  extent);
	hipMalloc3D(&f7_dA ,  extent);
	hipMalloc3D(&f8_dA ,  extent);
	hipMalloc3D(&f9_dA ,  extent);
	hipMalloc3D(&f10_dA ,  extent);
	hipMalloc3D(&f11_dA ,  extent);
	hipMalloc3D(&f12_dA ,  extent);
	hipMalloc3D(&f13_dA ,  extent);
	hipMalloc3D(&f14_dA ,  extent);
	hipMalloc3D(&f15_dA ,  extent);
	hipMalloc3D(&f16_dA ,  extent);
	hipMalloc3D(&f17_dA ,  extent);
	hipMalloc3D(&f18_dA ,  extent);
	int pitch = f0_dA.pitch/sizeof(float);//pitch in elementsa

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipArray * testarray;
	hipMalloc3DArray(&testarray, &desc, extent, hipArraySurfaceLoadStore);
	



	for (i = 0; i < n; i++)
	{
//		int x = i%xDim;
//		int y = (i/xDim)%zDim;
//		int z = (i/xDim)/yDim;
		f0_h[i]  = 5;
		f1_h[i]  = 0;
		f2_h[i]  = 2;
		f3_h[i]  = i;
		f4_h[i]  = i;
		f5_h[i]  = i;
		f6_h[i]  = i;
		f7_h[i]  = i;
		f8_h[i]  = i;
		f9_h[i]  = i;
		f10_h[i] = i;
		f11_h[i] = i;
		f12_h[i] = i;
		f13_h[i] = i;
		f14_h[i] = i;
		f15_h[i] = i;
		f16_h[i] = i;
		f17_h[i] = i;
		f18_h[i] = i;
		image_h[i] = 0;
		test_h[i] = 0;
//		if(x < 1) image_h[i] = 1;//DirichletWest
//		if(x > xDim-2) image_h[i] = 1;//BB
//		if(y < 1) image_h[i] = 1;//BB
//		if(y > yDim-2) image_h[i] = 3;//BB
	}

	hipMemcpy3DParms pfB = {0};
	pfB.srcPtr.ptr = f0_h;
	pfB.srcPtr.pitch = xDim*sizeof(float);
	pfB.srcPtr.xsize = xDim;
	pfB.srcPtr.ysize = yDim;
	pfB.dstArray = testarray;
	pfB.extent = extent;
	pfB.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&pfB);
	surfRef.channelDesc = desc;
	cudaBindSurfaceToArray(surfRef, testarray);



	hipMemcpy3DParms pfA = {0};
	pfA.srcPtr.pitch = xDim*sizeof(float);
	pfA.srcPtr.xsize = xDim;
	pfA.srcPtr.ysize = yDim;
	pfA.dstPtr.pitch = f0_dA.pitch;
	cout<<"pitch on dev is "<<f0_dA.pitch;
	pfA.dstPtr.xsize = xDim;
	pfA.dstPtr.ysize = yDim;
	pfA.extent.width = xDim*sizeof(float);
	pfA.extent.height = yDim;
	pfA.extent.depth = zDim;
	pfA.kind = hipMemcpyHostToDevice;
	
	if(true)//host to dev memcpy
	{
	pfA.srcPtr.ptr = f0_h;
	pfA.dstPtr.ptr = f0_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f1_h;
	pfA.dstPtr.ptr = f1_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f2_h;
	pfA.dstPtr.ptr = f2_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f3_h;
	pfA.dstPtr.ptr = f3_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f4_h;
	pfA.dstPtr.ptr = f4_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f5_h;
	pfA.dstPtr.ptr = f5_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f6_h;
	pfA.dstPtr.ptr = f6_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f7_h;
	pfA.dstPtr.ptr = f7_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f8_h;
	pfA.dstPtr.ptr = f8_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f9_h;
	pfA.dstPtr.ptr = f9_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f10_h;
	pfA.dstPtr.ptr = f10_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f11_h;
	pfA.dstPtr.ptr = f11_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f12_h;
	pfA.dstPtr.ptr = f12_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f13_h;
	pfA.dstPtr.ptr = f13_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f14_h;
	pfA.dstPtr.ptr = f14_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f15_h;
	pfA.dstPtr.ptr = f15_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f16_h;
	pfA.dstPtr.ptr = f16_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f17_h;
	pfA.dstPtr.ptr = f17_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.srcPtr.ptr = f18_h;
	pfA.dstPtr.ptr = f18_dA.ptr;
	hipMemcpy3D(&pfA);
	}


	if(true)//texture settings
	{
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f9A.normalized = false;
	texRef_f10A.normalized = false;
	texRef_f11A.normalized = false;
	texRef_f12A.normalized = false;
	texRef_f13A.normalized = false;
	texRef_f14A.normalized = false;
	texRef_f15A.normalized = false;
	texRef_f16A.normalized = false;
	texRef_f17A.normalized = false;
	texRef_f18A.normalized = false;
	texRef_f1A.filterMode = hipFilterModePoint;
	texRef_f2A.filterMode = hipFilterModePoint;
	texRef_f3A.filterMode = hipFilterModePoint;
	texRef_f4A.filterMode = hipFilterModePoint;
	texRef_f5A.filterMode = hipFilterModePoint;
	texRef_f6A.filterMode = hipFilterModePoint;
	texRef_f7A.filterMode = hipFilterModePoint;
	texRef_f8A.filterMode = hipFilterModePoint;
	texRef_f9A.filterMode = hipFilterModePoint;
	texRef_f10A.filterMode = hipFilterModePoint;
	texRef_f11A.filterMode = hipFilterModePoint;
	texRef_f12A.filterMode = hipFilterModePoint;
	texRef_f13A.filterMode = hipFilterModePoint;
	texRef_f14A.filterMode = hipFilterModePoint;
	texRef_f15A.filterMode = hipFilterModePoint;
	texRef_f16A.filterMode = hipFilterModePoint;
	texRef_f17A.filterMode = hipFilterModePoint;
	texRef_f18A.filterMode = hipFilterModePoint;
	}

//	cudaBindTexture3D(0,&texRef_f1A, f1_dA,&desc,xDim,yDim,pitch);
//	cudaBindTexture3D(0,&texRef_f2A, f2_dA,&desc,xDim,yDim,pitch);
//	cudaBindTexture3D(0,&texRef_f3A, f3_dA,&desc,xDim,yDim,pitch);
//	cudaBindTexture3D(0,&texRef_f4A, f4_dA,&desc,xDim,yDim,pitch);
//	cudaBindTexture3D(0,&texRef_f5A, f5_dA,&desc,xDim,yDim,pitch);
//	cudaBindTexture3D(0,&texRef_f6A, f6_dA,&desc,xDim,yDim,pitch);
//	cudaBindTexture3D(0,&texRef_f7A, f7_dA,&desc,xDim,yDim,pitch);
//	cudaBindTexture3D(0,&texRef_f8A, f8_dA,&desc,xDim,yDim,pitch);



	initialize<<<grid, threads>>>(f0_dA.ptr, f1_dA.ptr, f2_dA.ptr, f3_dA.ptr, f4_dA.ptr, f5_dA.ptr, f6_dA.ptr, f7_dA.ptr, f8_dA.ptr, f9_dA.ptr,
									f10_dA.ptr, f11_dA.ptr, f12_dA.ptr, f13_dA.ptr, f14_dA.ptr, f15_dA.ptr, f16_dA.ptr, f17_dA.ptr, f18_dA.ptr,
									xDim,yDim,pitch);
//	initialize<<<grid, threads>>>(f0_dA, f1_dA, f2_dA, f3_dA, f4_dA, f5_dA, f6_dA, f7_dA, f8_dA, f9_dA,
//									f10_dA, f11_dA, f12_dA, f13_dA, f14_dA, f15_dA, f16_dA, f17_dA, f18_dA,
//									xDim,yDim,pitch);

//	hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	hipMemcpy(test_d, test_h, memsize, hipMemcpyHostToDevice);
	f1_h[0] = 10;
	image_h[0] = 10;
	test_h[0] = 10;


//	copytest<<<grid, threads>>>(f10_dA,test_d,xDim,yDim,zDim);
	//copytest<<<grid, threads>>>(test_d);
	//copytest<<<grid, threads>>>(image_d);





	//COPY FROM DEV TO HOST
	pfA.srcPtr.pitch = f0_dA.pitch;
	pfA.srcPtr.xsize = f0_dA.xsize;//xDim;
	pfA.srcPtr.ysize = f0_dA.ysize;//yDim;
	pfA.dstPtr.pitch = xDim*sizeof(float);
	pfA.dstPtr.xsize = xDim;
	pfA.dstPtr.ysize = yDim;
	pfA.extent.width = xDim*sizeof(float);
	pfA.extent.height = yDim;
	pfA.extent.depth = zDim;
	pfA.kind = hipMemcpyDeviceToHost;

	if(true)//dev to host memcpy
	{
	pfA.dstPtr.ptr = f0_h;
	pfA.srcPtr.ptr = f0_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f1_h;
	pfA.srcPtr.ptr = f1_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f2_h;
	pfA.srcPtr.ptr = f2_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f3_h;
	pfA.srcPtr.ptr = f3_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f4_h;
	pfA.srcPtr.ptr = f4_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f5_h;
	pfA.srcPtr.ptr = f5_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f6_h;
	pfA.srcPtr.ptr = f6_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f7_h;
	pfA.srcPtr.ptr = f7_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f8_h;
	pfA.srcPtr.ptr = f8_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f9_h;
	pfA.srcPtr.ptr = f9_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f10_h;
	pfA.srcPtr.ptr = f10_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f11_h;
	pfA.srcPtr.ptr = f11_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f12_h;
	pfA.srcPtr.ptr = f12_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f13_h;
	pfA.srcPtr.ptr = f13_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f14_h;
	pfA.srcPtr.ptr = f14_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f15_h;
	pfA.srcPtr.ptr = f15_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f16_h;
	pfA.srcPtr.ptr = f16_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f17_h;
	pfA.srcPtr.ptr = f17_dA.ptr;
	hipMemcpy3D(&pfA);
	pfA.dstPtr.ptr = f18_h;
	pfA.srcPtr.ptr = f18_dA.ptr;
	hipMemcpy3D(&pfA);
	}

	cout<<"f1_h is "<<f1_h[0]<<endl;
	//hipMemcpy(f0_h, f0_d.ptr, memsize, hipMemcpyDeviceToHost);

	hipMemcpy(image_h, image_d, memsize_int, hipMemcpyDeviceToHost);
	hipMemcpy(test_h, test_d, memsize, hipMemcpyDeviceToHost);

//	cout<<image_h[0]<<endl;
//	cout<<"test_d: "<<test_h[0]<<endl;
//	for(i = 0; i<n; i++){
//	cout<<f0_h[i]<<",";
//	}


	hipFree(image_d);
	hipFree(test_d);
	hipFree(testarray);
	hipFree(f0_dA.ptr);
	hipFree(f1_dA.ptr);
	hipFree(f2_dA.ptr);
	hipFree(f3_dA.ptr);
	hipFree(f4_dA.ptr);
	hipFree(f5_dA.ptr);
	hipFree(f6_dA.ptr);
	hipFree(f7_dA.ptr);
	hipFree(f8_dA.ptr);
	hipFree(f9_dA.ptr);
	hipFree(f10_dA.ptr);
	hipFree(f11_dA.ptr);
	hipFree(f12_dA.ptr);
	hipFree(f13_dA.ptr);
	hipFree(f14_dA.ptr);
	hipFree(f15_dA.ptr);
	hipFree(f16_dA.ptr);
	hipFree(f17_dA.ptr);
	hipFree(f18_dA.ptr);
	return(0);

}
