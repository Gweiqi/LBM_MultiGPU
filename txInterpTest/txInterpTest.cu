#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define CASENAME "test"

#define BLOCKSIZEX 64
#define BLOCKSIZEY 1
#define BLOCKSIZELRX 64
#define BLOCKSIZELRY 1
#define XDIM 256
#define YDIM 128
#define TMAX 1000
#define OBSTR 8.f
#define OBSTX 128.f
#define OBSTY 64.f

#define STARTF 10

#define LRFACTOR 0.5f
#define LRLEVEL  2 //inverse of LRFACTOR as an integer
#define LRX0 111.75	//minimum x coord of LR
#define XLRDIM 128	//number of nodes in x
#define LRY0 111.75
#define YLRDIM 128

#define RE 20.f
#define UMAX 0.06f
#define METHOD "HYB" //LR,SINGLE,HYB,TEXT,SHARED,SINGLEF,TEST
#define REFINEMENT "NO" //YES,NO
#define MODEL "MRT" //BGK,MRT,STREAM
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>


/*
Image List:
0  fluid
1  BB
2  Neumann East
3  Dirichlet West
10 BB(force)
11 xsymmetry(y=max)
12 xsymmetry(y=min)

*/
inline __device__ int ImageFcn(float x, float y){
	//if(y < 0.1f || (XDIM-x) < 1.1f || (YDIM-y) < 1.1f)
//	if(y < 0.1f || (YDIM-y) < 1.1f)
//		return 1;
//	else if(x < 0.1f)
//		return 3;//west
//	else if((XDIM-x) < 1.1f)
//		return 2;//east
//	else if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
	//if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
	if(abs(x-OBSTX)<OBSTR && abs(y-OBSTY)<OBSTR)
		return 10;
	else
		return 0;
}

inline __device__ int ImageFcn(int x, int y){
	//if(y == 0 || x == XDIM-1 || y == YDIM-1)
	if(x < 0.1f)
		return 1;//3;//west
	else if((XDIM-x) < 1.1f)
		return 1;//2;//east
	else if((YDIM-y) < 1.1f)
		return 1;//11;//xsymmetry top
	else if(y < 0.1f)
		return 1;//12;//xsymmetry bottom
	//else if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
	if(abs(x-OBSTX)<OBSTR && abs(y-OBSTY)<OBSTR)
		return 1;//10;
	else
		return 0;
}

inline __device__ float PoisProf (float x){
	float radius = (YDIM-1-1)*0.5f;
	float result = -1.0f*(((1.0f-(x-0.5f)/radius))*((1.0f-(x-0.5f)/radius))-1.0f);
	return (result);
//	return 1.f;
}



texture<float,2,hipReadModeElementType> texRef_f0A;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;

texture<float,2,hipReadModeElementType> texRef_f0B;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;

texture<float,2,hipReadModeElementType> texRef_f0C;
texture<float,2,hipReadModeElementType> texRef_f1C;
texture<float,2,hipReadModeElementType> texRef_f2C;
texture<float,2,hipReadModeElementType> texRef_f3C;
texture<float,2,hipReadModeElementType> texRef_f4C;
texture<float,2,hipReadModeElementType> texRef_f5C;
texture<float,2,hipReadModeElementType> texRef_f6C;
texture<float,2,hipReadModeElementType> texRef_f7C;
texture<float,2,hipReadModeElementType> texRef_f8C;

texture<float,2,hipReadModeElementType> texRef_f0D;
texture<float,2,hipReadModeElementType> texRef_f1D;
texture<float,2,hipReadModeElementType> texRef_f2D;
texture<float,2,hipReadModeElementType> texRef_f3D;
texture<float,2,hipReadModeElementType> texRef_f4D;
texture<float,2,hipReadModeElementType> texRef_f5D;
texture<float,2,hipReadModeElementType> texRef_f6D;
texture<float,2,hipReadModeElementType> texRef_f7D;
texture<float,2,hipReadModeElementType> texRef_f8D;


int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 ,  float omega)
{
	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	f0 = f0 -omega*(f0 -0.4444444444f*(rho-1.5f*usqr));
	f1 = f1 -omega*(f1 -0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 = f2 -omega*(f2 -0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 = f3 -omega*(f3 -0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 = f4 -omega*(f4 -0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 = f5 -omega*(f5 -0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	f6 = f6 -omega*(f6 -0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 = f7 -omega*(f7 -0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 = f8 -omega*(f8 -0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
}

__device__ void mrt_collide(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, float omega)
{
	//float rho,u,v;	
	float u,v;	
	//rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float m1,m2,m4,m6,m7,m8;

//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	m1 =-2.f*f0 +    f1 +    f2 +    f3 +    f4+ 4.f*f5+ 4.f*f6+ 4.f*f7+ 4.f*f8-3.0f*(u*u+v*v);
	//m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	m2 = 3.f*f0 -3.f*f1 -3.f*f2 -3.f*f3 -3.f*f4                                +3.0f*(u*u+v*v); //ep
	//m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	m4 =        -    f1        +     f3        + 2.f*f5 -2.f*f6 -2.f*f7+ 2.f*f8;//-(-u);//qx_eq
	m6 =                -    f2        +     f4+ 2.f*f5+ 2.f*f6 -2.f*f7 -2.f*f8;//-(-v);//qy_eq
	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq

//	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
//	m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
//	m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
//	m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
//	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
//	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq

	f0=f0-(-m1+m2)*0.11111111f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
	//f1=f1-(-m1-2.0f*(m2+m4)+m7*omega*9.0f)*0.027777777f;
	f1=f1-(-m1*0.027777777f-0.05555555556f*m2-0.16666666667f*m4+m7*omega*0.25f);
	f2=f2-(-m1*0.027777777f-0.05555555556f*m2-0.16666666667f*m6-m7*omega*0.25f);
	f3=f3-(-m1*0.027777777f-0.05555555556f*m2+0.16666666667f*m4+m7*omega*0.25f);
	f4=f4-(-m1*0.027777777f-0.05555555556f*m2+0.16666666667f*m6-m7*omega*0.25f);
	f5=f5-(0.05555555556f*m1+m2*0.027777777f+0.08333333333f*m4+0.08333333333f*m6+m8*omega*0.25f);
	f6=f6-(0.05555555556f*m1+m2*0.027777777f-0.08333333333f*m4+0.08333333333f*m6-m8*omega*0.25f);
	f7=f7-(0.05555555556f*m1+m2*0.027777777f-0.08333333333f*m4-0.08333333333f*m6+m8*omega*0.25f);
	f8=f8-(0.05555555556f*m1+m2*0.027777777f+0.08333333333f*m4-0.08333333333f*m6-m8*omega*0.25f);
//	f1=f1-(-m1-2.f*m2-6.f*m4+m7*omega*9.0f)*0.027777777f;
//	f2=f2-(-m1-2.f*m2-6.f*m6-m7*omega*9.0f)*0.027777777f;
//	f3=f3-(-m1-2.f*m2+6.f*m4+m7*omega*9.0f)*0.027777777f;
//	f4=f4-(-m1-2.f*m2+6.f*m6-m7*omega*9.0f)*0.027777777f;
//	f5=f5-(2.f*m1+m2+3.f*m4+3.f*m6+m8*omega*9.0f)*0.027777777f;
//	f6=f6-(2.f*m1+m2-3.f*m4+3.f*m6-m8*omega*9.0f)*0.027777777f;
//	f7=f7-(2.f*m1+m2-3.f*m4-3.f*m6+m8*omega*9.0f)*0.027777777f;
//	f8=f8-(2.f*m1+m2+3.f*m4-3.f*m6-m8*omega*9.0f)*0.027777777f;
}

__device__ void NeumannEast(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, int y)
{
	if(y == 0){
		f2 = f4;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f8 = f5;
	}
	float u,v,rho;
    v = 0.0;
    rho = 1.0;
	u = -rho+((f0+f2+f4)+2.0f*f1+2.0f*f5+2.0f*f8);

    f3 = f1-u*0.66666667f;
    f7 = f5+0.5f*(f2-f4)-0.5f*v-u*0.16666667f;
    f6 = f8-0.5f*(f2-f4)+0.5f*v-u*0.16666667f;
}
__device__ void DirichletWest(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, int y)
{
	if(y == 0){
		f2 = f4;
		f6 = f7;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f7 = f6;
	}
	float u,v;//,rho;
	u = UMAX;//*PoisProf(float(y));
	v = 0.0f;//0.0;
	f1 = f3+u*0.66666667f;
	f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;	
}




__device__ void boundaries(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 ,
					int y, int im)
{
	if(im == 2)//NeumannEast
	{
		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	else if(im == 3)//DirichletWest
	{
		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	else if(im == 11)//xsymmetry
	{
		f4 = f2;
		f7 = f6;
		f8 = f5;
	}
	else if(im == 12)//xsymmetry
	{
		f2 = f4;
		f6 = f7;
		f5 = f8;
	}
}


__device__ void boundaries_force(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 ,
					int y, int im)
{
	if(im == 2)//NeumannEast
	{
		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	else if(im == 3)//DirichletWest
	{
		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
	}
	if(im == 11)//xsymmetry
	{
		f4 = f2;
		f7 = f6;
		f8 = f5;
	}
	else if(im == 12)//xsymmetry
	{
		f2 = f4;
		f6 = f7;
		f5 = f8;
	}
}


inline __device__ int f_mem(int f_num, int x, int y, size_t pitch)
{

	return (x+y*pitch)+f_num*pitch*YDIM;
}

inline __device__ int f_memLR(int f_num, int x, int y, size_t pitch)
{

	return (x+y*pitch)+f_num*pitch*YLRDIM;
}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}

__global__ void simple_copy(float* fA, float* fB,
							int *image, float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)

	fB[j] = fA[j];//+0.01f;
}

__global__ void ExtractFromC_d(float* fout,
							size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if(x < LRX0+1 || x > LRX0+XLRDIM*LRFACTOR-1 || y < LRY0+1 || y > LRY0+YLRDIM*LRFACTOR-1)
////	if(x < LRX0+2 || x > LRX0+XLRDIM-3 || y < LRY0+2 || y > LRY0+YLRDIM-3)
//	{
//	//do nothing
//	}
//	else
	//if(x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1)
	if( (x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1) && 
	(x == int(LRX0+2) || x == int(LRX0+XLRDIM*LRFACTOR-1) || y == int(LRY0+2) || y == int(LRY0+YLRDIM*LRFACTOR-1)) )
	{
//	float xcoord = 2.0f*x-2.0f*LRX0+0.5f;
//	float ycoord = 2.0f*y-2.0f*LRY0+0.5f;
	float xcoord = (x-LRX0)/LRFACTOR+0.5f;
	float ycoord = (y-LRY0)/LRFACTOR+0.5f;
	f0 = tex2D(texRef_f0C ,xcoord,ycoord);
	f1 = tex2D(texRef_f1C ,xcoord,ycoord);
	f2 = tex2D(texRef_f2C ,xcoord,ycoord);
	f3 = tex2D(texRef_f3C ,xcoord,ycoord);
	f4 = tex2D(texRef_f4C ,xcoord,ycoord);
	f5 = tex2D(texRef_f5C ,xcoord,ycoord);
	f6 = tex2D(texRef_f6C ,xcoord,ycoord);
	f7 = tex2D(texRef_f7C ,xcoord,ycoord);
	f8 = tex2D(texRef_f8C ,xcoord,ycoord);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
//	f0 =SF*f0 +(1.0f-SF)*(0.4444444444f*rho-0.6666666667f*usqr);
//	f1 =SF*f1 +(1.0f-SF)*(0.1111111111f*rho+0.3333333333f*u+0.5f*u*u-0.1666666667f*usqr);                 
//	f2 =SF*f2 +(1.0f-SF)*(0.1111111111f*rho+0.3333333333f*v+0.5f*v*v-0.1666666667f*usqr);
//	f3 =SF*f3 +(1.0f-SF)*(0.1111111111f*rho-0.3333333333f*u+0.5f*u*u-0.1666666667f*usqr);
//	f4 =SF*f4 +(1.0f-SF)*(0.1111111111f*rho-0.3333333333f*v+0.5f*v*v-0.1666666667f*usqr);
//	f5 =SF*f5 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*( u+v)+0.125f*( u+v)*( u+v)-0.04166666667f*usqr);
//	f6 =SF*f6 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*(-u+v)+0.125f*(-u+v)*(-u+v)-0.04166666667f*usqr);
//	f7 =SF*f7 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*(-u-v)+0.125f*(-u-v)*(-u-v)-0.04166666667f*usqr);
//	f8 =SF*f8 +(1.0f-SF)*(0.02777777778*rho+0.08333333333f*( u-v)+0.125f*( u-v)*( u-v)-0.04166666667f*usqr);

	fout[f_mem(0 ,x,y,pitch)] =SF*f0 +(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_mem(1 ,x,y,pitch)] =SF*f1 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_mem(2 ,x,y,pitch)] =SF*f2 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_mem(3 ,x,y,pitch)] =SF*f3 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_mem(4 ,x,y,pitch)] =SF*f4 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_mem(5 ,x,y,pitch)] =SF*f5 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_mem(6 ,x,y,pitch)] =SF*f6 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_mem(7 ,x,y,pitch)] =SF*f7 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_mem(8 ,x,y,pitch)] =SF*f8 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
	
//	fout[f_mem(0 ,x,y,pitch)] = f0 ;
//	fout[f_mem(1 ,x,y,pitch)] = f1 ;
//	fout[f_mem(2 ,x,y,pitch)] = f2 ;
//	fout[f_mem(3 ,x,y,pitch)] = f3 ;
//	fout[f_mem(4 ,x,y,pitch)] = f4 ;
//	fout[f_mem(5 ,x,y,pitch)] = f5 ;
//	fout[f_mem(6 ,x,y,pitch)] = f6 ;
//	fout[f_mem(7 ,x,y,pitch)] = f7 ;
//	fout[f_mem(8 ,x,y,pitch)] = f8 ;
	}
}

__global__ void LR_d_hybABCD_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();

	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	sumX[threadIdx.x]=0.f;
	sumY[threadIdx.x]=0.f;
	//dont do anything
	}
	else{
//	if(x != 0 && x != XLRDIM-1 && y != 0 && y != YLRDIM-1){
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1C ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3C ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5C ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6C ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7C ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8C ,x-1+0.5f,y+1+0.5f);

	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		}

		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}

	}//end else (not at edge of LR)

	syncthreads();
	if(check[0] == 1 && t>STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
	}
	}

}

__global__ void LR_d_hybBACD_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();

	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	sumX[threadIdx.x]=0.f;
	sumY[threadIdx.x]=0.f;
	//dont do anything
	}
	else{
//	if(x != 0 && x != XLRDIM-1 && y != 0 && y != YLRDIM-1){
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1C ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3C ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5C ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6C ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7C ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8C ,x-1+0.5f,y+1+0.5f);

	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		}

		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)

	syncthreads();
	if(check[0] == 1 && t>=STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
	}
	}

}



__global__ void LR_d_hybABCD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1C ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3C ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5C ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6C ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7C ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8C ,x-1+0.5f,y+1+0.5f);

	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}

	}//end else (not at edge of LR)
}

__global__ void LR_d_hybABCD2(float* fin, float* fout,
							float omega, size_t pitch, int n)//pitch in elements
//second time through in inner loop. don't need to calculate for the 3 outer most cells
//n=number of outer elements to ignore
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	//if(x < 3 || x > XLRDIM-4 || y < 3 || y > YLRDIM-4)
	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1C ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3C ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5C ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6C ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7C ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8C ,x-1+0.5f,y+1+0.5f);

	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}

	}//end else (not at edge of LR)
}

__global__ void LR_d_hybABDC2(float* fin, float* fout,
							float omega, size_t pitch, float SF, int n)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n)
	{
	//no interp
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1D ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3D ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5D ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6D ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7D ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8D ,x-1+0.5f,y+1+0.5f);

	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_hybABDC_Interp(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL)
	//if(x < 2 || x > XLRDIM-1-2 || y < 2 || y > YLRDIM-1-2)
	{
	//interpolate for next time step. from B
	f0 = tex2D(texRef_f0B,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2B,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3B,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4B,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5B,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6B,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7B,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8B,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1D ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3D ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5D ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6D ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7D ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8D ,x-1+0.5f,y+1+0.5f);

	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_hybBACD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1C ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3C ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5C ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6C ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7C ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8C ,x-1+0.5f,y+1+0.5f);
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_hybBACD2(float* fin, float* fout,
							float omega, size_t pitch, int n)//pitch in elements
//second time through in inner loop. don't need to calculate for the 3 outer most cells
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1C ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3C ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5C ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6C ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7C ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8C ,x-1+0.5f,y+1+0.5f);
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_hybBADC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//no interp
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1D ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3D ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5D ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6D ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7D ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8D ,x-1+0.5f,y+1+0.5f);
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}
__global__ void LR_d_hybBADC2(float* fin, float* fout,
							float omega, size_t pitch, float SF, int n)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n)
	{
	//no interp
	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1D ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3D ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5D ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6D ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7D ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8D ,x-1+0.5f,y+1+0.5f);
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_hybBADC_Interp(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

	if(x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL)
	//if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//interpolate for next time step. from A
	f0 = tex2D(texRef_f0A,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1A,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2A,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3A,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4A,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5A,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6A,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7A,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8A,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

	}
	else{
	f0 = fin[j];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1D ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3D ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5D ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6D ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7D ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8D ,x-1+0.5f,y+1+0.5f);
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABCD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}
__global__ void LR_d_ABCD2(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	if(x < 3 || x > XLRDIM-4 || y < 3 || y > YLRDIM-4)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//interpolate for next time step. from B
	f0 = tex2D(texRef_f0B,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2B,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3B,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4B,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5B,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6B,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7B,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8B,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

//	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
//	fout[f_memLR(0,x,y,pitch)] = f0 ;
//	fout[f_memLR(1,x,y,pitch)] = f1 ;
//	fout[f_memLR(2,x,y,pitch)] = f2 ;
//	fout[f_memLR(3,x,y,pitch)] = f3 ;
//	fout[f_memLR(4,x,y,pitch)] = f4 ;
//	fout[f_memLR(5,x,y,pitch)] = f5 ;
//	fout[f_memLR(6,x,y,pitch)] = f6 ;
//	fout[f_memLR(7,x,y,pitch)] = f7 ;
//	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fin[f_memLR(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fin[f_memLR(2 ,x             ,y-1           ,pitch)];
	f5 = fin[f_memLR(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fin[f_memLR(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fin[f_memLR(4 ,x             ,y+1           ,pitch)];
	f7 = fin[f_memLR(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fin[f_memLR(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC2(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//interpolate for next time step. from B
	f0 = tex2D(texRef_f0B,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1B,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2B,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3B,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4B,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5B,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6B,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7B,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8B,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

//	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
//	fout[f_memLR(0,x,y,pitch)] = f0 ;
//	fout[f_memLR(1,x,y,pitch)] = f1 ;
//	fout[f_memLR(2,x,y,pitch)] = f2 ;
//	fout[f_memLR(3,x,y,pitch)] = f3 ;
//	fout[f_memLR(4,x,y,pitch)] = f4 ;
//	fout[f_memLR(5,x,y,pitch)] = f5 ;
//	fout[f_memLR(6,x,y,pitch)] = f6 ;
//	fout[f_memLR(7,x,y,pitch)] = f7 ;
//	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fin[f_memLR(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fin[f_memLR(2 ,x             ,y-1           ,pitch)];
	f5 = fin[f_memLR(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fin[f_memLR(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fin[f_memLR(4 ,x             ,y+1           ,pitch)];
	f7 = fin[f_memLR(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fin[f_memLR(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}
__global__ void LR_d_BACD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(xcoord,ycoord);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BADC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//interpolate for next time step. from A
	f0 = tex2D(texRef_f0A,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f1 = tex2D(texRef_f1A,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2A,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3A,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4A,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5A,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6A,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7A,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8A,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	fout[f_memLR(0,x,y,pitch)] =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	fout[f_memLR(1,x,y,pitch)] =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	fout[f_memLR(2,x,y,pitch)] =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(3,x,y,pitch)] =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	fout[f_memLR(4,x,y,pitch)] =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	fout[f_memLR(5,x,y,pitch)] =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	fout[f_memLR(6,x,y,pitch)] =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	fout[f_memLR(7,x,y,pitch)] =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	fout[f_memLR(8,x,y,pitch)] =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));

//	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
//	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
//	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
//	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
//	fout[f_memLR(0,x,y,pitch)] = f0 ;
//	fout[f_memLR(1,x,y,pitch)] = f1 ;
//	fout[f_memLR(2,x,y,pitch)] = f2 ;
//	fout[f_memLR(3,x,y,pitch)] = f3 ;
//	fout[f_memLR(4,x,y,pitch)] = f4 ;
//	fout[f_memLR(5,x,y,pitch)] = f5 ;
//	fout[f_memLR(6,x,y,pitch)] = f6 ;
//	fout[f_memLR(7,x,y,pitch)] = f7 ;
//	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fin[f_memLR(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fin[f_memLR(2 ,x             ,y-1           ,pitch)];
	f5 = fin[f_memLR(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fin[f_memLR(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fin[f_memLR(4 ,x             ,y+1           ,pitch)];
	f7 = fin[f_memLR(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fin[f_memLR(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}
__global__ void mrt_d_single(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	//if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1){
	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
	}
	else{
	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		//atomicAdd();
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
//		if(im == 2)//NeumannEast
//		{
//			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 3)//DirichletWest
//		{
//			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}
	}

}

__global__ void mrt_d_single_force(float* fA, float* fB,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	//int check = 0; //check if block contains force calculation nodes
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();

	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
	}
	else{
	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		//check = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		}
		//atomicAdd();
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
//		if(im == 2)//NeumannEast
//		{
//			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 3)//DirichletWest
//		{
//			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}

		boundaries_force(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}

	syncthreads();
	if(check[0] == 1 && t>=STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
	}
	}
	}
}
__global__ void mrt_d_hybAB_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();

	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
	}
	else{

	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,pitch)];
//	f1 = tex2D(texRef_f1A ,xcoord-1,ycoord  );
//	f3 = tex2D(texRef_f3A ,xcoord+1,ycoord  );
//	f5 = tex2D(texRef_f5A ,xcoord-1,ycoord-1);
//	f6 = tex2D(texRef_f6A ,xcoord+1,ycoord-1);
//	f7 = tex2D(texRef_f7A ,xcoord+1,ycoord+1);
//	f8 = tex2D(texRef_f8A ,xcoord-1,ycoord+1);

	f1 = tex2D(texRef_f1A ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3A ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5A ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6A ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7A ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8A ,x-1+0.5f,y+1+0.5f);

	int im = ImageFcn(x,y);
	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		}
		fout[f_mem(1 ,x,y,pitch)] = f3 ;
		fout[f_mem(2 ,x,y,pitch)] = f4 ;
		fout[f_mem(3 ,x,y,pitch)] = f1 ;
		fout[f_mem(4 ,x,y,pitch)] = f2 ;
		fout[f_mem(5 ,x,y,pitch)] = f7 ;
		fout[f_mem(6 ,x,y,pitch)] = f8 ;
		fout[f_mem(7 ,x,y,pitch)] = f5 ;
		fout[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
//		if(im == 2)//NeumannEast
//		{
//			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 3)//DirichletWest
//		{
//			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 11)//xsymmetry
//		{
//			f4 = f2;
//			f7 = f6;
//			f8 = f5;
//		}
//		else if(im == 12)//xsymmetry
//		{
//			f2 = f4;
//			f6 = f7;
//			f5 = f8;
//		}

		boundaries_force(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_mem(0 ,x,y,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,pitch)] = f8 ;

	}

	syncthreads();
	if(check[0] == 1 && t>=STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
	}
	}

	}//outside the refinement area
}

__global__ void mrt_d_hybBA_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();

	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
	}
	else{

	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,pitch)];
//	f1 = tex2D(texRef_f1A ,xcoord-1,ycoord  );
//	f3 = tex2D(texRef_f3A ,xcoord+1,ycoord  );
//	f5 = tex2D(texRef_f5A ,xcoord-1,ycoord-1);
//	f6 = tex2D(texRef_f6A ,xcoord+1,ycoord-1);
//	f7 = tex2D(texRef_f7A ,xcoord+1,ycoord+1);
//	f8 = tex2D(texRef_f8A ,xcoord-1,ycoord+1);

	f1 = tex2D(texRef_f1B ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3B ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5B ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6B ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7B ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8B ,x-1+0.5f,y+1+0.5f);

	int im = ImageFcn(x,y);
	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		}
		fout[f_mem(1 ,x,y,pitch)] = f3 ;
		fout[f_mem(2 ,x,y,pitch)] = f4 ;
		fout[f_mem(3 ,x,y,pitch)] = f1 ;
		fout[f_mem(4 ,x,y,pitch)] = f2 ;
		fout[f_mem(5 ,x,y,pitch)] = f7 ;
		fout[f_mem(6 ,x,y,pitch)] = f8 ;
		fout[f_mem(7 ,x,y,pitch)] = f5 ;
		fout[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
//		if(im == 2)//NeumannEast
//		{
//			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 3)//DirichletWest
//		{
//			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 11)//xsymmetry
//		{
//			f4 = f2;
//			f7 = f6;
//			f8 = f5;
//		}
//		else if(im == 12)//xsymmetry
//		{
//			f2 = f4;
//			f6 = f7;
//			f5 = f8;
//		}

		boundaries_force(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_mem(0 ,x,y,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,pitch)] = f8 ;

	}

	syncthreads();
	if(check[0] == 1 && t>=STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
	}
	}

	}//outside the refinement area
}

__global__ void mrt_d_hybAB(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
	}
	else{

	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1A ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3A ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5A ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6A ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7A ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8A ,x-1+0.5f,y+1+0.5f);

	int im = ImageFcn(x,y);
	if(im == 1 || im == 10){//BB
		fout[f_mem(1 ,x,y,pitch)] = f3 ;
		fout[f_mem(2 ,x,y,pitch)] = f4 ;
		fout[f_mem(3 ,x,y,pitch)] = f1 ;
		fout[f_mem(4 ,x,y,pitch)] = f2 ;
		fout[f_mem(5 ,x,y,pitch)] = f7 ;
		fout[f_mem(6 ,x,y,pitch)] = f8 ;
		fout[f_mem(7 ,x,y,pitch)] = f5 ;
		fout[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
//		if(im == 2)//NeumannEast
//		{
//			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 3)//DirichletWest
//		{
//			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 11)//xsymmetry
//		{
//			f4 = f2;
//			f7 = f6;
//			f8 = f5;
//		}
//		else if(im == 12)//xsymmetry
//		{
//			f2 = f4;
//			f6 = f7;
//			f5 = f8;
//		}
	
		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_mem(0 ,x,y,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,pitch)] = f8 ;
	}
	}
}
__global__ void mrt_d_hybBA(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1) || (x>XDIM-1)){
//	}
//	else{
	f0 = fin[j];
	f2 = fin[f_mem(2 ,x  ,y-1,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,pitch)];
	f1 = tex2D(texRef_f1B ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3B ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5B ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6B ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7B ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8B ,x-1+0.5f,y+1+0.5f);

	int im = ImageFcn(x,y);
	if(im == 1 || im == 10){//BB
		fout[f_mem(1 ,x,y,pitch)] = f3 ;
		fout[f_mem(2 ,x,y,pitch)] = f4 ;
		fout[f_mem(3 ,x,y,pitch)] = f1 ;
		fout[f_mem(4 ,x,y,pitch)] = f2 ;
		fout[f_mem(5 ,x,y,pitch)] = f7 ;
		fout[f_mem(6 ,x,y,pitch)] = f8 ;
		fout[f_mem(7 ,x,y,pitch)] = f5 ;
		fout[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
//		if(im == 2)//NeumannEast
//		{
//			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 3)//DirichletWest
//		{
//			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
//		}
//		else if(im == 11)//xsymmetry
//		{
//			f4 = f2;
//			f7 = f6;
//			f8 = f5;
//		}
//		else if(im == 12)//xsymmetry
//		{
//			f2 = f4;
//			f6 = f7;
//			f5 = f8;
//		}

		boundaries(f1,f1,f2,f3,f4,f5,f6,f7,f8,y,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_mem(0 ,x,y,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,pitch)] = f8 ;
	}
//	}
}
__global__ void mrt_d_textAB(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	f0 = fin[j];
	f2 = tex2D(texRef_f2A ,x  +0.5f,y-1+0.5f);
	f4 = tex2D(texRef_f4A ,x  +0.5f,y+1+0.5f);
	f1 = tex2D(texRef_f1A ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3A ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5A ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6A ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7A ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8A ,x-1+0.5f,y+1+0.5f);

	int im = ImageFcn(x,y);
	if(im == 1 || im == 10){//BB
		fout[f_mem(1 ,x,y,pitch)] = f3 ;
		fout[f_mem(2 ,x,y,pitch)] = f4 ;
		fout[f_mem(3 ,x,y,pitch)] = f1 ;
		fout[f_mem(4 ,x,y,pitch)] = f2 ;
		fout[f_mem(5 ,x,y,pitch)] = f7 ;
		fout[f_mem(6 ,x,y,pitch)] = f8 ;
		fout[f_mem(7 ,x,y,pitch)] = f5 ;
		fout[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(im == 2)//NeumannEast
		{
			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
		}
		else if(im == 3)//DirichletWest
		{
			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_mem(0 ,x,y,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,pitch)] = f8 ;

	}
}
__global__ void mrt_d_textBA(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	f0 = fin[j];
	f2 = tex2D(texRef_f2B ,x  +0.5f,y-1+0.5f);
	f4 = tex2D(texRef_f4B ,x  +0.5f,y+1+0.5f);
	f1 = tex2D(texRef_f1B ,x-1+0.5f,y  +0.5f);
	f3 = tex2D(texRef_f3B ,x+1+0.5f,y  +0.5f);
	f5 = tex2D(texRef_f5B ,x-1+0.5f,y-1+0.5f);
	f6 = tex2D(texRef_f6B ,x+1+0.5f,y-1+0.5f);
	f7 = tex2D(texRef_f7B ,x+1+0.5f,y+1+0.5f);
	f8 = tex2D(texRef_f8B ,x-1+0.5f,y+1+0.5f);

	int im = ImageFcn(x,y);
	if(im == 1 || im == 10){//BB
		fout[f_mem(1 ,x,y,pitch)] = f3 ;
		fout[f_mem(2 ,x,y,pitch)] = f4 ;
		fout[f_mem(3 ,x,y,pitch)] = f1 ;
		fout[f_mem(4 ,x,y,pitch)] = f2 ;
		fout[f_mem(5 ,x,y,pitch)] = f7 ;
		fout[f_mem(6 ,x,y,pitch)] = f8 ;
		fout[f_mem(7 ,x,y,pitch)] = f5 ;
		fout[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(im == 2)//NeumannEast
		{
			NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
		}
		else if(im == 3)//DirichletWest
		{
			DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,y);
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_mem(0 ,x,y,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,pitch)] = f8 ;

	}
}


__global__ void mrt_d_LR(float* fA, float* fB,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
			}
			float u,v;//,rho;
			u = 0.0f;
	        v = UMAX;//0.0;
			f1 = f3+u*0.66666667f;
	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;			
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}
}


__global__ void initialize_single(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	if(x<XDIM && y<YDIM){
	float u,v,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;

	f[j+0 *pitch*YDIM]= 0.4444444444f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM]= 0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);               
	f[j+2 *pitch*YDIM]= 0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM]= 0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM]= 0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM]= 0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM]= 0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM]= 0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM]= 0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
	}
}


__global__ void initialize_LR(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	if(x<XLRDIM && y<YLRDIM){
	float u,v,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;

	f[j+0 *pitch*YLRDIM]= 0.4444444444f*(rho-1.5f*usqr);
	f[j+1 *pitch*YLRDIM]= 0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);               
	f[j+2 *pitch*YLRDIM]= 0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YLRDIM]= 0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YLRDIM]= 0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
	f[j+6 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YLRDIM]= 0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
	}
}

int main(int argc, char *argv[])
{

	//int *image_d, *image_h;

	ofstream output;
	ofstream output2;
	string FileName = CASENAME;
	output.open ((FileName+".dat").c_str());
	output2.open ((FileName+".force").c_str());

	size_t memsize, memsize2;
	size_t pitch = 0;
	size_t pitch2 = 0;
	int i, n, nBlocks, nBlocks2, n2;
	float omega, CharLength, omega2;

	if(abs(LRFACTOR-1.f/LRLEVEL)>0.001f){
		cout<<"LRLEVEL and LRFACTOR don't match! Exiting..."<<endl;
		return 0;
	}

	CharLength = OBSTR*2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));
	if(LRFACTOR == 0.25f){
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	}
	else if(LRFACTOR == 0.125f){
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	}
	else if(LRFACTOR == 0.0625){
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	}
	float SF_cf = omega*(1.0f-omega2)/((1.0f-omega)*omega2/LRFACTOR);
	float SF_fc = 1.f/SF_cf;

//	float SF_cf3 = omega*(1.0f-omega3)/((1.0f-omega)*omega3/LRFACTOR);
//	float SF_fc3 = 1.f/SF_cf3;

	cout<<"omega:  "<<omega<<endl;
	cout<<"omega2: "<<omega2<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<endl;
	cout<<"LRblocksize: "<<BLOCKSIZELRX<<"x"<<BLOCKSIZELRY<<endl;
	cout<<"LRgrid: "<<XLRDIM<<"x"<<YLRDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

	nBlocks = ((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX)*(YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY);
	nBlocks2 = (XLRDIM/BLOCKSIZELRX+XLRDIM%BLOCKSIZELRX)*(YLRDIM/BLOCKSIZELRY+YLRDIM%BLOCKSIZELRY);
	int B = BLOCKSIZEX*BLOCKSIZELRY;
	int B2 = BLOCKSIZELRX*BLOCKSIZELRY;
	n = nBlocks*B;
	n2 = nBlocks2*B2;
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
    //dim3 grid(XDIM/BLOCKSIZEX,YDIM/BLOCKSIZEY);
    dim3 grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY);

    dim3 threads2(BLOCKSIZELRX, BLOCKSIZELRY);
    //dim3 grid2(XLRDIM/BLOCKSIZELRX,YLRDIM/BLOCKSIZELRY);
    dim3 grid2(XLRDIM/BLOCKSIZELRX+XLRDIM%BLOCKSIZELRX,YLRDIM/BLOCKSIZELRY+YLRDIM%BLOCKSIZELRY);

	memsize = n*sizeof(float);
	//memsize_int = n*sizeof(int);
	memsize2 = n2*sizeof(float);

	//hipExtent extent = make_hipExtent(XDIM*sizeof(float),YDIM,ZDIM);

	//image_h = (int *)malloc(memsize_int);

	float *fA_h,*fA_d,*fB_d,*fC_h,*fC_d,*fD_d;
	float *FX_h,*FY_h,*FX_d,*FY_d;
	fA_h = (float *)malloc(memsize*9);
	fC_h = (float *)malloc(memsize2*9);
	FX_h = (float *)malloc(TMAX*sizeof(float));
	FY_h = (float *)malloc(TMAX*sizeof(float));
	hipMallocPitch((void **) &fA_d, &pitch, XDIM*sizeof(float), YDIM*9);
	hipMallocPitch((void **) &fB_d, &pitch, XDIM*sizeof(float), YDIM*9);
//	hipMalloc((void **) &fA_d, XDIM*sizeof(float)*YDIM*9);
//	hipMalloc((void **) &fB_d, XDIM*sizeof(float)*YDIM*9);
//	pitch = 16384;

	if(REFINEMENT == "YES"){
	hipMallocPitch((void **) &fC_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*9);
	hipMallocPitch((void **) &fD_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*9);
	}

	hipMalloc((void **) &FX_d, TMAX*sizeof(float));
	hipMalloc((void **) &FY_d, TMAX*sizeof(float));

	//hipMalloc((void **) &image_d, memsize_int);

	cout<<pitch<<", "<<pitch2<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);
	size_t pitch_elements2 = pitch2/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*9; i++)
		fA_h[i] = i;
	for (i = 0; i < n2*9; i++)
		fC_h[i] = 0;
	for (i = 0; i < TMAX; i++){
		FX_h[i] = 0.0f;
		FY_h[i] = 0.0f;
	}
	hipMemcpy(FX_d, FX_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FY_d, FY_h, TMAX*sizeof(float), hipMemcpyHostToDevice);

//	for (i = 0; i < n; i++)
//	{
//		int x = i%XDIM;
//		int y = (i/XDIM)%YDIM;
//		int z = (i/XDIM)/YDIM;
////		image_h[i] = 0;
////		if(x < 1) image_h[i] = 1;//DirichletWest
////		if(x > XDIM-2) image_h[i] = 1;//BB
////		if(y < 1) image_h[i] = 1;//BB
////		if(y > YDIM-2) image_h[i] = 1;//BB
////		if(z < 1) image_h[i] = 1;//DirichletWest
////		if(z > ZDIM-2) image_h[i] = 1;//BB
//	}
	//hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;

	texRef_f0B.filterMode = hipFilterModeLinear;
	texRef_f1B.filterMode = hipFilterModeLinear;
	texRef_f2B.filterMode = hipFilterModeLinear;
	texRef_f3B.filterMode = hipFilterModeLinear;
	texRef_f4B.filterMode = hipFilterModeLinear;
	texRef_f5B.filterMode = hipFilterModeLinear;
	texRef_f6B.filterMode = hipFilterModeLinear;
	texRef_f7B.filterMode = hipFilterModeLinear;
	texRef_f8B.filterMode = hipFilterModeLinear;
	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f0A.filterMode = hipFilterModeLinear;
	texRef_f1A.filterMode = hipFilterModeLinear;
	texRef_f2A.filterMode = hipFilterModeLinear;
	texRef_f3A.filterMode = hipFilterModeLinear;
	texRef_f4A.filterMode = hipFilterModeLinear;
	texRef_f5A.filterMode = hipFilterModeLinear;
	texRef_f6A.filterMode = hipFilterModeLinear;
	texRef_f7A.filterMode = hipFilterModeLinear;
	texRef_f8A.filterMode = hipFilterModeLinear;
	
	texRef_f0C.normalized = false;
	texRef_f1C.normalized = false;
	texRef_f2C.normalized = false;
	texRef_f3C.normalized = false;
	texRef_f4C.normalized = false;
	texRef_f5C.normalized = false;
	texRef_f6C.normalized = false;
	texRef_f7C.normalized = false;
	texRef_f8C.normalized = false;

	texRef_f0C.filterMode = hipFilterModeLinear;
	texRef_f1C.filterMode = hipFilterModeLinear;
	texRef_f2C.filterMode = hipFilterModeLinear;
	texRef_f3C.filterMode = hipFilterModeLinear;
	texRef_f4C.filterMode = hipFilterModeLinear;
	texRef_f5C.filterMode = hipFilterModeLinear;
	texRef_f6C.filterMode = hipFilterModeLinear;
	texRef_f7C.filterMode = hipFilterModeLinear;
	texRef_f8C.filterMode = hipFilterModeLinear;

	texRef_f0D.normalized = false;
	texRef_f1D.normalized = false;
	texRef_f2D.normalized = false;
	texRef_f3D.normalized = false;
	texRef_f4D.normalized = false;
	texRef_f5D.normalized = false;
	texRef_f6D.normalized = false;
	texRef_f7D.normalized = false;
	texRef_f8D.normalized = false;

	texRef_f0D.filterMode = hipFilterModeLinear;
	texRef_f1D.filterMode = hipFilterModeLinear;
	texRef_f2D.filterMode = hipFilterModeLinear;
	texRef_f3D.filterMode = hipFilterModeLinear;
	texRef_f4D.filterMode = hipFilterModeLinear;
	texRef_f5D.filterMode = hipFilterModeLinear;
	texRef_f6D.filterMode = hipFilterModeLinear;
	texRef_f7D.filterMode = hipFilterModeLinear;
	texRef_f8D.filterMode = hipFilterModeLinear;

	for(int i = 0; i<2; i++){
	texRef_f0A.addressMode[i]  = hipAddressModeClamp;
	texRef_f1A.addressMode[i]  = hipAddressModeClamp;
	texRef_f2A.addressMode[i]  = hipAddressModeClamp;
	texRef_f3A.addressMode[i]  = hipAddressModeClamp;
	texRef_f4A.addressMode[i]  = hipAddressModeClamp;
	texRef_f5A.addressMode[i]  = hipAddressModeClamp;
	texRef_f6A.addressMode[i]  = hipAddressModeClamp;
	texRef_f7A.addressMode[i]  = hipAddressModeClamp;
	texRef_f8A.addressMode[i]  = hipAddressModeClamp;

	texRef_f0B.addressMode[i]  = hipAddressModeClamp;
	texRef_f1B.addressMode[i]  = hipAddressModeClamp;
	texRef_f2B.addressMode[i]  = hipAddressModeClamp;
	texRef_f3B.addressMode[i]  = hipAddressModeClamp;
	texRef_f4B.addressMode[i]  = hipAddressModeClamp;
	texRef_f5B.addressMode[i]  = hipAddressModeClamp;
	texRef_f6B.addressMode[i]  = hipAddressModeClamp;
	texRef_f7B.addressMode[i]  = hipAddressModeClamp;
	texRef_f8B.addressMode[i]  = hipAddressModeClamp;
	}
	}
	
	hipMemcpy2D(fA_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*9,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*9,hipMemcpyHostToDevice);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyHostToDevice);
	hipMemcpy2D(fD_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyHostToDevice);
	}
	for (i = 0; i < n*9; i++)
		fA_h[i] = 0;
	for (i = 0; i < n2*9; i++)
		fC_h[i] = 0;


	if(true)//bind texture
	{
	hipBindTexture2D(0,&texRef_f0A, fA_d                       ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f1A, fA_d+pitch_elements*YDIM   ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f2A, fA_d+pitch_elements*YDIM*2 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f3A, fA_d+pitch_elements*YDIM*3 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f4A, fA_d+pitch_elements*YDIM*4 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f5A, fA_d+pitch_elements*YDIM*5 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f6A, fA_d+pitch_elements*YDIM*6 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f7A, fA_d+pitch_elements*YDIM*7 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f8A, fA_d+pitch_elements*YDIM*8 ,&desc,XDIM,YDIM,pitch);

	hipBindTexture2D(0,&texRef_f0B, fB_d                       ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f1B, fB_d+pitch_elements*YDIM   ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f2B, fB_d+pitch_elements*YDIM*2 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f3B, fB_d+pitch_elements*YDIM*3 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f4B, fB_d+pitch_elements*YDIM*4 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f5B, fB_d+pitch_elements*YDIM*5 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f6B, fB_d+pitch_elements*YDIM*6 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f7B, fB_d+pitch_elements*YDIM*7 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f8B, fB_d+pitch_elements*YDIM*8 ,&desc,XDIM,YDIM,pitch);

	hipBindTexture2D(0,&texRef_f0C, fC_d                          ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1C, fC_d+pitch_elements2*YLRDIM   ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2C, fC_d+pitch_elements2*YLRDIM*2 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3C, fC_d+pitch_elements2*YLRDIM*3 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4C, fC_d+pitch_elements2*YLRDIM*4 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5C, fC_d+pitch_elements2*YLRDIM*5 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6C, fC_d+pitch_elements2*YLRDIM*6 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7C, fC_d+pitch_elements2*YLRDIM*7 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8C, fC_d+pitch_elements2*YLRDIM*8 ,&desc,XLRDIM,YLRDIM,pitch2);

	hipBindTexture2D(0,&texRef_f0D, fD_d                          ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1D, fD_d+pitch_elements2*YLRDIM   ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2D, fD_d+pitch_elements2*YLRDIM*2 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3D, fD_d+pitch_elements2*YLRDIM*3 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4D, fD_d+pitch_elements2*YLRDIM*4 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5D, fD_d+pitch_elements2*YLRDIM*5 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6D, fD_d+pitch_elements2*YLRDIM*6 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7D, fD_d+pitch_elements2*YLRDIM*7 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8D, fD_d+pitch_elements2*YLRDIM*8 ,&desc,XLRDIM,YLRDIM,pitch2);
	}
	


	initialize_single<<<grid, threads>>>(fA_d,pitch_elements);
	initialize_single<<<grid, threads>>>(fB_d,pitch_elements);

	if(REFINEMENT == "YES"){
	initialize_LR<<<grid2, threads2>>>(fC_d,pitch_elements2);
	}

//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(mrt_d_single),hipFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<TMAX; t=t+2){
		if(REFINEMENT == "YES"){
			if(METHOD == "SINGLE")
			mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			else if(METHOD == "HYB")
			mrt_d_hybAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			else if(METHOD == "TEXT")
			mrt_d_textAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			

			if(METHOD == "SINGLE"){
				LR_d_ABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
				LR_d_ABDC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
			}
			else if(METHOD == "HYB"){
				if(LRFACTOR == 0.5f){
					if(t >= STARTF)
					LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t);
					else
					LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
					LR_d_hybABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.25f){
					if(t >= STARTF)
					LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t);
					else
					LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
					LR_d_hybABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);

					LR_d_hybABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3);
					LR_d_hybABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.125f){
					if(t >= STARTF)
					LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t);
					else
					LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
					LR_d_hybABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);

					for(int k = 0; k<2;k++){
					LR_d_hybABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3+k);
					LR_d_hybABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,4+k);
					}

					LR_d_hybABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,7);
					LR_d_hybABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.0625f){
					if(t >= STARTF)
					LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t);
					else
					LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
					LR_d_hybABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);

					for(int k = 0; k<6;k++){
					LR_d_hybABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3+k);
					LR_d_hybABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,4+k);
					}

					LR_d_hybABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,15);
					LR_d_hybABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
			}
			
			ExtractFromC_d<<<grid, threads>>>(fB_d,pitch_elements,SF_fc);

	
			if(METHOD == "SINGLE")
			mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
			else if(METHOD == "HYB")
			mrt_d_hybBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
			else if(METHOD == "TEXT")
			mrt_d_textBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
	

			if(METHOD == "SINGLE"){
			LR_d_BACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
			LR_d_BADC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
			}
			else if(METHOD == "HYB"){
				if(LRFACTOR == 0.5f){
				if(t >= STARTF)
				LR_d_hybBACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t+1);
				else
				LR_d_hybBACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
				LR_d_hybBADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.25f){
				if(t >= STARTF)
				LR_d_hybBACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t+1);
				else
				LR_d_hybBACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
				LR_d_hybBADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);

				LR_d_hybBACD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3);
				LR_d_hybBADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.125f){
					if(t >= STARTF)
					LR_d_hybBACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t+1);
					else
					LR_d_hybBACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
					LR_d_hybBADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);

					for(int k = 0; k<2;k++){
					LR_d_hybBACD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3+k);
					LR_d_hybBADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,4+k);
					}

					LR_d_hybBACD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,7);
					LR_d_hybBADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
				else if(LRFACTOR == 0.0625f){
					if(t >= STARTF)
					LR_d_hybBACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,t+1);
					else
					LR_d_hybBACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
					LR_d_hybBADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);

					for(int k = 0; k<6;k++){
					LR_d_hybBACD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3+k);
					LR_d_hybBADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,4+k);
					}

					LR_d_hybBACD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,15);
					LR_d_hybBADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
				}
			}
			
			ExtractFromC_d<<<grid, threads>>>(fA_d,pitch_elements,SF_fc);
		}
		else{
			if(METHOD == "SINGLE"){
			mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
			}
	
			else if(METHOD == "SINGLEF"){
			mrt_d_single_force<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,FX_d,FY_d,t);
			mrt_d_single_force<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,FX_d,FY_d,t+1);
			}
	
			else if(METHOD == "HYB"){
			if(t >= STARTF){
			mrt_d_hybAB_force<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,FX_d,FY_d,t);
			mrt_d_hybBA_force<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,FX_d,FY_d,t+1);
			}
			else{
			mrt_d_hybAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			mrt_d_hybBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
			}
			}
	
			else if(METHOD == "TEXT"){
			mrt_d_textAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
			mrt_d_textBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
			}
		}

//		else if(METHOD == "SHARED"){
//		mrt_d_shared<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//		mrt_d_shared<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//		}

//		simple_copy<<<grid, threads>>>(fA_d,fB_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);
//		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);

		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	int Nodes;
	if(REFINEMENT == "YES"){
		Nodes = (XDIM*YDIM+XLRDIM*YLRDIM/LRFACTOR);
	}
	else{
		Nodes = XDIM*YDIM;
	}
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
//			<<double((XDIM*YDIM*ZDIM)*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<XDIM<<","<<YDIM<<","<<TMAX<<","<<restime<<endl;

	if(true){
	hipUnbindTexture(texRef_f0A);
	hipUnbindTexture(texRef_f1A);
	hipUnbindTexture(texRef_f2A);
	hipUnbindTexture(texRef_f3A);
	hipUnbindTexture(texRef_f4A);
	hipUnbindTexture(texRef_f5A);
	hipUnbindTexture(texRef_f6A);
	hipUnbindTexture(texRef_f7A);
	hipUnbindTexture(texRef_f8A);

	hipUnbindTexture(texRef_f0B);
	hipUnbindTexture(texRef_f1B);
	hipUnbindTexture(texRef_f2B);
	hipUnbindTexture(texRef_f3B);
	hipUnbindTexture(texRef_f4B);
	hipUnbindTexture(texRef_f5B);
	hipUnbindTexture(texRef_f6B);
	hipUnbindTexture(texRef_f7B);
	hipUnbindTexture(texRef_f8B);

	hipUnbindTexture(texRef_f0C);
	hipUnbindTexture(texRef_f1C);
	hipUnbindTexture(texRef_f2C);
	hipUnbindTexture(texRef_f3C);
	hipUnbindTexture(texRef_f4C);
	hipUnbindTexture(texRef_f5C);
	hipUnbindTexture(texRef_f6C);
	hipUnbindTexture(texRef_f7C);
	hipUnbindTexture(texRef_f8C);

	hipUnbindTexture(texRef_f0D);
	hipUnbindTexture(texRef_f1D);
	hipUnbindTexture(texRef_f2D);
	hipUnbindTexture(texRef_f3D);
	hipUnbindTexture(texRef_f4D);
	hipUnbindTexture(texRef_f5D);
	hipUnbindTexture(texRef_f6D);
	hipUnbindTexture(texRef_f7D);
	hipUnbindTexture(texRef_f8D);
	}

	hipMemcpy2D(fA_h,XDIM*sizeof(float),fA_d,pitch,XDIM*sizeof(float),YDIM*9,hipMemcpyDeviceToHost);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_h,XLRDIM*sizeof(float),fC_d,pitch2,XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyDeviceToHost);
	}	
	hipMemcpy(FX_h, FX_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FY_h, FY_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);

	output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<"\n";
	
	int row = 0;
	int col = 0;
	i = 0;
	float rho, u, v;
	int j;

	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = row*XDIM+col;
			rho = fA_h[i];
			for(j = 1; j<9; j++)
				rho+=fA_h[i+XDIM*YDIM*j];
			u = fA_h[i+XDIM*YDIM*1]-fA_h[i+XDIM*YDIM*3]+fA_h[i+XDIM*YDIM*5]-fA_h[i+XDIM*YDIM*6]-
				fA_h[i+XDIM*YDIM*7]+fA_h[i+XDIM*YDIM*8];
			v = fA_h[i+XDIM*YDIM*2]-fA_h[i+XDIM*YDIM*4]+fA_h[i+XDIM*YDIM*5]+fA_h[i+XDIM*YDIM*6]-fA_h[i+XDIM*YDIM*7]-fA_h[i+XDIM*YDIM*8];
			output<<col<<", "<<row<<", "<<u<<","<<v<<","<<rho<<endl;
		}
	}

	if(REFINEMENT == "YES"){
	output<<endl<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
	//output<<"ZONE F=POINT, I="<<XLRDIM-LRLEVEL*2<<", J="<<YLRDIM-LRLEVEL*2<<"\n";
	output<<"ZONE F=POINT, I="<<XLRDIM<<", J="<<YLRDIM<<"\n";
	
//	for(row = LRLEVEL; row<YLRDIM-LRLEVEL; row++){
//		for(col = LRLEVEL; col<XLRDIM-LRLEVEL; col++){
	for(row = 0; row<YLRDIM-0; row++){
		for(col = 0; col<XLRDIM-0; col++){
			i = row*XLRDIM+col;
			rho = fC_h[i];
			for(j = 1; j<9; j++)
				rho+=fC_h[i+XLRDIM*YLRDIM*j];
			u = fC_h[i+XLRDIM*YLRDIM*1]-fC_h[i+XLRDIM*YLRDIM*3]+fC_h[i+XLRDIM*YLRDIM*5]-fC_h[i+XLRDIM*YLRDIM*6]-
				fC_h[i+XLRDIM*YLRDIM*7]+fC_h[i+XLRDIM*YLRDIM*8];
			v = fC_h[i+XLRDIM*YLRDIM*2]-fC_h[i+XLRDIM*YLRDIM*4]+fC_h[i+XLRDIM*YLRDIM*5]+fC_h[i+XLRDIM*YLRDIM*6]-fC_h[i+XLRDIM*YLRDIM*7]-fC_h[i+XLRDIM*YLRDIM*8];
			output<<LRX0+LRFACTOR*col<<", "<<LRY0+LRFACTOR*row<<", "<<u<<","<<v<<","<<rho<<endl;
		}
	}
	}

	output.close();

	//for(int t = STARTF-1; t<TMAX; t++){
	for(int t = 0; t<TMAX; t++){
		output2<<t<<", "<<FX_h[t]/(0.5f*UMAX*UMAX*OBSTR*2.0f)<<", "<<FY_h[t]/(0.5f*UMAX*UMAX*OBSTR*2.0f)<<endl;
	}
	output2.close();


	//hipFree(image_d);


	hipFree(fA_d);
	hipFree(fB_d);
	hipFree(fC_d);
	hipFree(fD_d);
	hipFree(FX_d);
	hipFree(FY_d);
	return(0);

}
