#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define CASENAME "test"

#define BLOCKSIZEX 128
#define BLOCKSIZEY 1
#define BLOCKSIZEZ 1
#define BLOCKSIZELRX 64
#define BLOCKSIZELRY 1
#define BLOCKSIZELRZ 1
#define XDIM 256
#define YDIM 320
#define ZDIM 1
#define TMAX 2000
#define STARTF 0

#define OBSTR1 8.f
#define OBSTX1 127.5f
#define OBSTY1 95.5f
#define OBSTZ1 15.5f

#define OBSTR2 4.f
#define OBSTX2 63.5f
#define OBSTY2 31.5f
#define OBSTZ2 31.5f

#define LRFACTOR 0.5f
#define LRLEVEL 2
#define LRX0 63.75f	//minimum x coord of LR
#define XLRDIM 256	//number of nodes in x
#define LRY0 47.75f
#define YLRDIM 256
#define LRZ0 -0.25f
#define ZLRDIM 128

//#define LRFACTOR 0.25f
//#define LRLEVEL 4
//#define LRX0 63.625f	//minimum x coord of LR
//#define XLRDIM 512	//number of nodes in x
//#define LRY0 47.625f
//#define YLRDIM 512
//#define LRZ0 -0.375f
//#define ZLRDIM 64

#define RE 22000.f//2000.f//100.f;
#define UMAX 0.04f
#define METHOD "SINGLE" //SINGLE,HYB,TEXT,SHARED,CACHE
#define REFINEMENT "NO" //YES,NO
#define SmagLES "YES" //YES,NO
#define MODEL "MRT" //BGK,MRT,STREAM
#define ZPERIODIC "YES"
#define VELAV "YES"
#define START_VELAV 40000
#define START_VELFLUC 400000
#define CS 0.04f
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>

/*
Image List:
0  fluid
1  BB
2
3  DirichletWest(simple)
10 BB(force)
13 DirichletWest_Reg
14 NeumannEast_Reg
15 DirichletNorth_Reg
16 DirichletSouth_Reg
21 ysymmetry_top
22 ysymmetry_bot
23 zsymmetry_top
24 zsymmetry_bot
25 xsymmetry_top
26 xsymmetry_bot
*/
inline __device__ int ImageFcn(float x, float y, float z){
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//		return 10;
//	else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//		return 10;
	//if(((x-OBSTX)*(x-OBSTX)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1)+(z-OBSTZ1)*(z-OBSTZ1))<OBSTR1*OBSTR1)
//	{
//		return 10;
//	}
//	else
//	//if(y < 0.1f || z < 0.1f || (XDIM-x) < 0.1f || (YDIM-y) < 0.1f || (ZDIM-z) < 0.1f)
//	if(y < 17.5f || z < 17.5f || y > 46.5f || z > 46.5f)
//		return 1;
//	else if(x < 17.5f)
//		return 13;
//	else if(x > 78.5f)
//		return 14;
//	else
    
    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
        return 10;
    else
		return 0;
}

inline __device__ int ImageFcn(int x, int y, int z){
    int value = 0;
//Cylinder
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//		value = 10;
//	else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//		value = 10;
//Sphere
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1)+(z-OBSTZ1)*(z-OBSTZ1))<OBSTR1*OBSTR1)
//	{
////		if(z == 0 || z == ZDIM-1)
////		return 1;
////		else
//		return 10;
//	}
//	if(z == 0)
//		value = 0;
//	else if(z == ZDIM-1)
//		value = 0;
    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
        value = 10;
	else if(y == 0)
		value = 200;//22;
	else if(y == YDIM-1)
		value = 100;
	else if(x == 0)
		value = 26;
	else if(x == XDIM-1)
		value = 25;
	else if(z == 0)
		value = 0;
	else if(z == ZDIM-1)
		value = 0;


    return value;

//Lid Driven Cavity
//	if(x == XDIM-1 || y == 0 || y == YDIM-1 || z == 0 || z == ZDIM-1)
//		return 1;
//	else if(x == XDIM-2 || y == 1 || y == YDIM-2 || z == 1 || z == ZDIM-2)
//		return 1;
//	else if(x == 0)
//		return 1;
//	else if(x == 1)
//		return 53;
//	else
//		return 0;
}

inline __device__ float PoisProf (float x){
	float radius = (YDIM-1-1)*0.5f;
	float result = -1.0f*(((1.0f-(x-0.5f)/radius))*((1.0f-(x-0.5f)/radius))-1.0f);
	return (result);
//	return 1.f;
}

__device__ void DirichletWest(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(y == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(y == YDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
//	if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
    }
	float u,v,w;//,rho;
    u = UMAX;//*PoisProf(zcoord)*1.5;
    v = 0.0f;
	w = 0.0f;
    
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//    float usqr = u*u+v*v+w*w;

	f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
	f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
////	f0 = 1.0f/3.0f*(rho-1.5f*usqr);
//	f1 = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
////	f2 = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
////	f3 = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
////	f4 = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//	f5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
////	f6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
////	f7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//	f8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
////	f9 = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//	f10= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
////	f11= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
////	f12= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
////	f13= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
////	f14= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//	f15= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
////	f16= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
////	f17= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
////	f18= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}

__device__ void DirichletWest_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0){
		f2 = f4;
		f6 = f7;
		f11 = f13;
		f16 = f18;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f7 = f6;
		f13 = f11;
		f18 = f16;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;//*PoisProf(y)*1.5;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

//	f1 =(0.166666667f*u)+
//		(f3-(-(0.166666667f*u)));
	f1 = f3+0.33333333f*u;
//	f5 =(0.0833333333f*( u+v))+
//		(f7-(0.0833333333f*(-u-v)));
	f5 = f7+0.166666667f*(u+v);
//	f8 =(0.0833333333f*( u-v  ))+
//		(f6-(0.0833333333f*(-u+v  )));
	f8 = f6+0.166666667f*(u-v);
//	f10=(0.0833333333f*( u+w))+
//		(f17-(0.0833333333f*(-u-w)));
	f10= f17+0.166666667f*(u+w);
//	f15=(0.0833333333f*( u-w))+
//		(f12-(0.0833333333f*(-u+w)));
	f15= f12+0.166666667f*(u-w);
		
//	f1 =(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2)+
//		(f3-(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2));
//	f5 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f7-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v));
//	f8 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f6-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v));
//	f10=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f17-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w));
//	f15=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f12-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w));

//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}


void __device__ DirichletWest_Regularized(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
    }

	float PI11 = 0;
	float PI12 = 0;
	float PI22 = 0;
	float PI33 = 0;
	float PI13 = 0;
	float PI23 = 0;
	float u;//,v;//,w;//,rho;
    u = UMAX;//*PoisProf(z)*1.5;
    //v = 0.0f;
	//w = 0.0f;
    float usqr = u*u;//+v*v+w*w;
	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
    
    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq9  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq14 = 0.0555555556f*(rho                -1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho                         -1.5f*usqr);

//    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
//    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq5  = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
//    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//    float feq8  = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
//    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq10 = 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
//    float feq11 = 0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
//    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
//    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq15 = 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
//    float feq16 = 0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
//    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

	f1 = feq1 +f3 -feq3 ;
	f5 = feq5 +f7 -feq7 ;
	f8 = feq8 +f6 -feq6 ;
	f10= feq10+f17-feq17;
	f15= feq15+f12-feq12;

    PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
    PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
    PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
    PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

    f0  = feq0 +1.5f  *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f1  = feq1 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f2  = feq2 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f3  = feq3 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f4  = feq4 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f5  = feq5 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f6  = feq6 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f7  = feq7 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f8  = feq8 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f9  = feq9 +0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f10 = feq10+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f11 = feq11+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f12 = feq12+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f13 = feq13+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f14 = feq14+0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f15 = feq15+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f16 = feq16+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f17 = feq17+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f18 = feq18+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
		
}



void __device__ NeumannEast_Regularized(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
		f11 = f13;
		f16 = f18;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f13 = f11;
		f18 = f16;
		f8 = f5;
	}
	else if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
	}

	float PI11 = 0;
	float PI12 = 0;
	float PI22 = 0;
	float PI33 = 0;
	float PI13 = 0;
	float PI23 = 0;

	float u;//,v;//,w;//,rho;
	float rho = 1.0f;
    //v = 0.0f;
	//w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
    float usqr = u*u;//+v*v+w*w;
    
    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq9  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq14 = 0.0555555556f*(rho                -1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho                         -1.5f*usqr);
    
//    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
//    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq5  = 0.0277777778f*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
//    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//    float feq8  = 0.0277777778f*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
//    float feq10 = 0.0277777778f*(rho+3.0f*( u+w)+4.5f*( u+w)*( u+w)-1.5f*usqr);
//    float feq11 = 0.0277777778f*(rho+3.0f*( v+w)+4.5f*( v+w)*( v+w)-1.5f*usqr);
//    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
//    float feq15 = 0.0277777778f*(rho+3.0f*( u-w)+4.5f*( u-w)*( u-w)-1.5f*usqr);
//    float feq16 = 0.0277777778f*(rho+3.0f*( v-w)+4.5f*( v-w)*( v-w)-1.5f*usqr);
//    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

	f3 = feq3 +f1 -feq1 ;
	f7 = feq7 +f5 -feq5 ;
	f6 = feq6 +f8 -feq8 ;
	f17= feq17+f10-feq10;
	f12= feq12+f15-feq15;

    PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
    PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
    PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
    PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

    f0  = feq0 +1.5f  *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f1  = feq1 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f2  = feq2 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f3  = feq3 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f4  = feq4 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f5  = feq5 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f6  = feq6 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f7  = feq7 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f8  = feq8 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f9  = feq9 +0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f10 = feq10+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f11 = feq11+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f12 = feq12+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f13 = feq13+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f14 = feq14+0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f15 = feq15+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f16 = feq16+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f17 = feq17+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f18 = feq18+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
			
}


__device__ void NeumannEast(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
//		f6 = f7;
		f11 = f13;
		f16 = f18;

		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
//		f7 = f6;
		f13 = f11;
		f18 = f16;

		f8 = f5;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
//		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
//		f17 = f12;
		f18 = f13;
	}
	}

	float u,v,w;//,rho;
	float rho = 1.0f;
    v = 0.0f;
	w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
	float u2 = u*u;
	float v2 = v*v;
	float w2 = w*w;
	float usqr = u2+v2+w2;

//	f3 = f1 -0.333333333f*u;
//	f7 = f5 -0.166666667f*(u+v);
//	f6 = f8 -0.166666667f*(u-v);
//	f17= f10-0.166666667f*(u+w);
//	f12= f15-0.166666667f*(u-w);
	f0 = 1.0f/3.0f*(rho-1.5f*usqr);
	f1 = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f2 = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f3 = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f4 = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f9 = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f10= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f11= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f12= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f13= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f14= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f15= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f16= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f17= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f18= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);


}


__device__ void NeumannEast_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
//		f6 = f7;
		f11 = f13;
		f16 = f18;

		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
//		f7 = f6;
		f13 = f11;
		f18 = f16;

		f8 = f5;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
//		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
//		f17 = f12;
		f18 = f13;
	}
	}

	float u,v,w;//,rho;
	float rho = 1.0f;
    v = 0.0f;
	w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

	f3 = f1 -0.333333333f*u;
	f7 = f5 -0.166666667f*(u+v);
	f6 = f8 -0.166666667f*(u-v);
	f17= f10-0.166666667f*(u+w);
	f12= f15-0.166666667f*(u-w);


//	f3 =(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2)+
//		(f1-(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2));
//	f7 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f5-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v));
//	f6 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f8-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v));
//	f17=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f10-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w));
//	f12=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f15-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w));


//	f1 =(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2)+
//		(f3-(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2));
//	f5 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f7-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v));
//	f8 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f6-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v));
//	f10=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f17-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w));
//	f15=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f12-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w));

//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}

__device__ void DirichletNorth_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(x == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(x == XDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

//	f1 =(0.166666667f*u)+
//		(f3-(-(0.166666667f*u)));
	f4 = f2-0.33333333f*v;
//	f5 =(0.0833333333f*( u+v))+
//		(f7-(0.0833333333f*(-u-v)));
	f7 = f5-0.166666667f*(u+v);
//	f8 =(0.0833333333f*( u-v  ))+
//		(f6-(0.0833333333f*(-u+v  )));
	f8 = f6+0.166666667f*(u-v);
//	f10=(0.0833333333f*( u+w))+
//		(f17-(0.0833333333f*(-u-w)));
	f13= f16-0.166666667f*(v-w);
//	f15=(0.0833333333f*( u-w))+
//		(f12-(0.0833333333f*(-u+w)));
	f18= f11-0.166666667f*(v+w);
	
//
//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                     
//float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;   +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
//float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
//float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w    +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
//float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;
//



//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}
__device__ void DirichletSouth_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(x == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(x == XDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

	f2 = f4 +0.33333333f*v;
	f5 = f7 +0.166666667f*(u+v);
	f6 = f8 -0.166666667f*(u-v);
	f16= f13+0.166666667f*(v-w);
	f11= f18+0.166666667f*(v+w);
	
//
//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                     
//float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;   +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
//float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
//float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w    +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
//float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;
//



//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}

__device__ void xsymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
//	if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
    }
	f1 = f3 ;
	f5 = f6 ;
	f8 = f7 ;
	f10= f12;
	f15= f17;
}
__device__ void xsymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
		f11 = f13;
		f16 = f18;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f13 = f11;
		f18 = f16;
		f8 = f5;
	}
//	else if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
	}
	f3 = f1 ;
	f6 = f5 ;
	f7 = f8 ;
	f12= f10;
	f17= f15;
}

__device__ void ysymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int z)
{
	if(z == 0){
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
	}
	if(z == ZDIM-1){
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
	}
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
}

__device__ void ysymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int z)
{
	if(z == 0){
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
	}
	if(z == ZDIM-1){
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
	}
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
}

__device__ void zsymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y)
{
	if(y == 0){
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
	}
	if(y == YDIM-1){
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
	}
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
}

__device__ void zsymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y)
{
	if(y == 0){
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
	}
	if(y == YDIM-1){
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
	}
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
}

inline __device__ void boundaries(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z, int im)
{
//	if(im == 3)//DirichletWest
//	{
//		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 53)//DirichletWest
	{
		//DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		DirichletWest_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 54)//DirichletWest
	{
		//NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		NeumannEast_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
//	if(im == 4)//DirichletWest
//	{
//		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	if(im == 13)//DirichletWest
//	{
//		DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 14)//DirichletWest
//	{
//		NeumannEast_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 15)//DirichletNorth
//	{
//		DirichletNorth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	if(im == 16)//DirichletSouth
//	{
//		DirichletSouth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 21)//ysymm top
	{
		ysymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 22)//ysymm bot
	{
		ysymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 23)//zsymm top
	{
		zsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 24)//zsymm bot
	{
		zsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
}
inline __device__ void boundaries_force(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z, int im)
{
//	if(im == 3)//DirichletWest
//	{
//		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 53)//DirichletWest
	{
		DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		//DirichletWest_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 54)//DirichletWest
	{
		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		//NeumannEast_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
//	else if(im == 15)//DirichletNorth
//	{
//		DirichletNorth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 16)//DirichletSouth
//	{
//		DirichletSouth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	else if(im == 21)//ysymm top
	{
		ysymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 22)//ysymm bot
	{
		ysymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 23)//zsymm top
	{
		zsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 24)//zsymm bot
	{
		zsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 25)//zsymm top
	{
		xsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 26)//zsymm bot
	{
		xsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
}



texture<float,2,hipReadModeElementType> texRef_f0A;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;
texture<float,2,hipReadModeElementType> texRef_f9A;
texture<float,2,hipReadModeElementType> texRef_f10A;
texture<float,2,hipReadModeElementType> texRef_f11A;
texture<float,2,hipReadModeElementType> texRef_f12A;
texture<float,2,hipReadModeElementType> texRef_f13A;
texture<float,2,hipReadModeElementType> texRef_f14A;
texture<float,2,hipReadModeElementType> texRef_f15A;
texture<float,2,hipReadModeElementType> texRef_f16A;
texture<float,2,hipReadModeElementType> texRef_f17A;
texture<float,2,hipReadModeElementType> texRef_f18A;

texture<float,2,hipReadModeElementType> texRef_f0B;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;
texture<float,2,hipReadModeElementType> texRef_f9B;
texture<float,2,hipReadModeElementType> texRef_f10B;
texture<float,2,hipReadModeElementType> texRef_f11B;
texture<float,2,hipReadModeElementType> texRef_f12B;
texture<float,2,hipReadModeElementType> texRef_f13B;
texture<float,2,hipReadModeElementType> texRef_f14B;
texture<float,2,hipReadModeElementType> texRef_f15B;
texture<float,2,hipReadModeElementType> texRef_f16B;
texture<float,2,hipReadModeElementType> texRef_f17B;
texture<float,2,hipReadModeElementType> texRef_f18B;


texture<float,2,hipReadModeElementType> texRef_f0C;
texture<float,2,hipReadModeElementType> texRef_f1C;
texture<float,2,hipReadModeElementType> texRef_f2C;
texture<float,2,hipReadModeElementType> texRef_f3C;
texture<float,2,hipReadModeElementType> texRef_f4C;
texture<float,2,hipReadModeElementType> texRef_f5C;
texture<float,2,hipReadModeElementType> texRef_f6C;
texture<float,2,hipReadModeElementType> texRef_f7C;
texture<float,2,hipReadModeElementType> texRef_f8C;
texture<float,2,hipReadModeElementType> texRef_f9C;
texture<float,2,hipReadModeElementType> texRef_f10C;
texture<float,2,hipReadModeElementType> texRef_f11C;
texture<float,2,hipReadModeElementType> texRef_f12C;
texture<float,2,hipReadModeElementType> texRef_f13C;
texture<float,2,hipReadModeElementType> texRef_f14C;
texture<float,2,hipReadModeElementType> texRef_f15C;
texture<float,2,hipReadModeElementType> texRef_f16C;
texture<float,2,hipReadModeElementType> texRef_f17C;
texture<float,2,hipReadModeElementType> texRef_f18C;

texture<float,2,hipReadModeElementType> texRef_f0D;
texture<float,2,hipReadModeElementType> texRef_f1D;
texture<float,2,hipReadModeElementType> texRef_f2D;
texture<float,2,hipReadModeElementType> texRef_f3D;
texture<float,2,hipReadModeElementType> texRef_f4D;
texture<float,2,hipReadModeElementType> texRef_f5D;
texture<float,2,hipReadModeElementType> texRef_f6D;
texture<float,2,hipReadModeElementType> texRef_f7D;
texture<float,2,hipReadModeElementType> texRef_f8D;
texture<float,2,hipReadModeElementType> texRef_f9D;
texture<float,2,hipReadModeElementType> texRef_f10D;
texture<float,2,hipReadModeElementType> texRef_f11D;
texture<float,2,hipReadModeElementType> texRef_f12D;
texture<float,2,hipReadModeElementType> texRef_f13D;
texture<float,2,hipReadModeElementType> texRef_f14D;
texture<float,2,hipReadModeElementType> texRef_f15D;
texture<float,2,hipReadModeElementType> texRef_f16D;
texture<float,2,hipReadModeElementType> texRef_f17D;
texture<float,2,hipReadModeElementType> texRef_f18D;


int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

//	f0 =(1.f-omega)*f0 +omega*(0.3333333333f*(rho-1.5f*usqr));
//	f1 =(1.f-omega)*f1 +omega*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f2 =(1.f-omega)*f2 +omega*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =(1.f-omega)*f3 +omega*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =(1.f-omega)*f4 +omega*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =(1.f-omega)*f5 +omega*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
//	f6 =(1.f-omega)*f6 +omega*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =(1.f-omega)*f7 +omega*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =(1.f-omega)*f8 +omega*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
//	f9 =(1.f-omega)*f9 +omega*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
//	f10=(1.f-omega)*f10+omega*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
//	f11=(1.f-omega)*f11+omega*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
//	f12=(1.f-omega)*f12+omega*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
//	f13=(1.f-omega)*f13+omega*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
//	f14=(1.f-omega)*f14+omega*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
//	f15=(1.f-omega)*f15+omega*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
//	f16=(1.f-omega)*f16+omega*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
//	f17=(1.f-omega)*f17+omega*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
//	f18=(1.f-omega)*f18+omega*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));

	f0 -=omega*(f0 -0.3333333333f*(rho-1.5f*usqr));
	f1 -=omega*(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 -=omega*(f2 -0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 -=omega*(f3 -0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 -=omega*(f4 -0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 -=omega*(f5 -0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 -=omega*(f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 -=omega*(f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 -=omega*(f8 -0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 -=omega*(f9 -0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10-=omega*(f10-0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11-=omega*(f11-0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr));
	f12-=omega*(f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13-=omega*(f13-0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr));
	f14-=omega*(f14-0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15-=omega*(f15-0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16-=omega*(f16-0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17-=omega*(f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18-=omega*(f18-0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));

}

inline __device__ void mrt_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float u,v,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
    float usqr = u*u+v*v+w*w;
//	u = rho*u;
//	v = rho*v;
//	w = rho*w;


	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	//COMPUTE M-MEQ
	//m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	//m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	//m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
//	m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//	m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//	m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
	m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
	m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
	m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
	m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
	m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
	m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
	m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
	m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
	m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;

	if(SmagLES == "YES"){
////		float PI11 = -1.0f/38.0f*(     (m1)+19.0f*omega* (m9));
////		float PI22 = -1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)-3.0f*omega*(m11)));
////		float PI33 = -1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)+3.0f*omega*(m11)));
//		float PI11 = LRLEVEL*-0.026315789f*m1-0.5f *omega*m9;
//		float PI22 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//		float PI33 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//		float PI12 = LRLEVEL*-1.5f*omega*m13;
//		float PI23 = LRLEVEL*-1.5f*omega*m14;
//		float PI13 = LRLEVEL*-1.5f*omega*m15;
//		float nu0 = ((1.0f/omega)-0.5f)*LRFACTOR/3.0f;
//		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//		//float Cs = 0.01f;
//		omega = 1.0f/(3.0f*(nu0+CS*Smag*LRFACTOR*LRFACTOR)*LRLEVEL+0.5f);
//		//omega = 1.0f/(1.0f/omega+3.f*CS*Smag*LRFACTOR*LRFACTOR);
//        //omega = 1.0f/(1.0f*LRLEVEL/1.99983f-1.f+0.5f+3.f*CS*Smag*LRFACTOR);

//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)*LRFACTOR/3.0f;
float tau0 = 1.f/omega;

//float Smag = (sqrt(nu0*nu0+18.f*CS*LRFACTOR*LRFACTOR*Q)-nu0)/(6.f*CS*LRFACTOR*LRFACTOR);
//float Smag = LRFACTOR*(sqrt(4.f/9.f*tau0*tau0+8.f*CS*LRFACTOR*Q)-2.f/3.f*tau0)/(4.f*CS*LRFACTOR*LRFACTOR);

//omega = 1.0f/(3.0f*(nu0+CS*Smag*LRFACTOR*LRFACTOR)*LRLEVEL+0.5f);

//float tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*LRFACTOR*Q));
float tau = tau0+0.5f*(-tau0+sqrt(tau0*tau0+18.f*CS*sqrt(2.f)*Q));
omega = 1.f/tau;

//float tau = 3.f*nu0*LRFACTOR+0.5f+(sqrt(tau0*tau0+18.f*CS*CS*LRFACTOR*LRFACTOR*Q)-tau0)*0.5f;
//omega = 1.f/tau;



	}


f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));



}

inline __device__ void mrt_collide_LES(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega, float Cs)
{
	float u,v,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
    float usqr = u*u+v*v+w*w;
//	u = rho*u;
//	v = rho*v;
//	w = rho*w;


	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	//COMPUTE M-MEQ
	//m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	//m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	//m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
//	m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//	m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//	m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
	m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
	m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
	m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
	m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
	m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
	m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
	m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
	m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
	m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;

	if(SmagLES == "YES"){
//		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
//		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//
//		float PI12 = -1.5f*omega*m13;
//		float PI23 = -1.5f*omega*m14;
//		float PI13 = -1.5f*omega*m15;
//		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//		omega = 1.0f/(1.0f/omega+3.f*CS*Smag);

//		float PI11 = LRLEVEL*-1.0f/38.0f*(     (m1)+19.0f*omega* (m9));
//		float PI22 = LRLEVEL*-1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)-3.0f*omega*(m11)));
//		float PI33 = LRLEVEL*-1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)+3.0f*omega*(m11)));
//		float PI12 = LRLEVEL*-1.5f*omega*m13;
//		float PI23 = LRLEVEL*-1.5f*omega*m14;
//		float PI13 = LRLEVEL*-1.5f*omega*m15;
//		float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//		float Smag = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+PI12*PI12+PI23*PI23+PI13*PI13);
//		omega = 1.0f/(3.0f*(nu0+Cs*Smag*LRLEVEL*LRLEVEL)+0.5f);


//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//
//float Smag = (sqrt(nu0*nu0+18.f*CS*Q)-nu0)/(6.f*CS);
//
////omega = 1.0f/(1.0f/omega+3.f*CS*Smag);
//
//float tau0 = 1.f/omega;
//float tau = 3.f*nu0+0.5f+(sqrt(tau0*tau0+18.f*CS*CS*Q)-tau0)*0.5f;
//omega = 1.f/tau;

float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
float tau0 = 1.f/omega;

//float Smag = (sqrt(nu0*nu0+18.f*CS*LRFACTOR*LRFACTOR*Q)-nu0)/(6.f*CS*LRFACTOR*LRFACTOR);
//float Smag = (sqrt(4.f/9.f*tau0*tau0+8.f*CS*Q)-2.f/3.f*tau0)/(4.f*CS);

//omega = 1.0f/(3.0f*(nu0+CS*Smag)+0.5f);

float tau = tau0+0.5f*(-tau0+sqrt(tau0*tau0+18.f*sqrt(2.f)*CS*Q));
omega = 1.f/tau;




	}


f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));



}

inline __device__ void vel_av(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float& uAv, float& vAv, int t)
{
	float u,v;//,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
    uAv = (uAv*(t-START_VELAV)+u)/((t-START_VELAV)+1);
    vAv = (vAv*(t-START_VELAV)+v)/((t-START_VELAV)+1);
}
inline __device__ void vel_avLR(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float& uAv, float& vAv, float t)
{
	float u,v;//,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
    uAv = (uAv*(t-START_VELAV)+u*LRFACTOR)/((t-START_VELAV)+LRFACTOR);
    vAv = (vAv*(t-START_VELAV)+v*LRFACTOR)/((t-START_VELAV)+LRFACTOR);
}

inline __device__ void vel_fluc(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float& uAv, 
                    float& vAv, float& ufluc, float& vfluc, int t)
{
	float u,v;//,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
    u = (u-uAv)*(u-uAv);
    v = (v-vAv)*(v-vAv);
    ufluc = (ufluc*(t-START_VELFLUC)+u)/((t-START_VELFLUC)+1);
    vfluc = (vfluc*(t-START_VELFLUC)+v)/((t-START_VELFLUC)+1);
}
inline __device__ void vel_flucLR(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float& uAv, 
                    float& vAv, float& ufluc, float& vfluc, float t)
{
	float u,v;//,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
    u = (u-uAv)*(u-uAv);
    v = (v-vAv)*(v-vAv);
    ufluc = (ufluc*(t-START_VELFLUC)+u*LRFACTOR)/((t-START_VELFLUC)+LRFACTOR);
    vfluc = (vfluc*(t-START_VELFLUC)+v*LRFACTOR)/((t-START_VELFLUC)+LRFACTOR);
}

inline __device__ void bgk_scale_cf(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float SF)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

    f0 =SF*f0 +(1.0f-SF)*feq0 ;
    f1 =SF*f1 +(1.0f-SF)*feq1 ;
    f2 =SF*f2 +(1.0f-SF)*feq2 ;
    f3 =SF*f3 +(1.0f-SF)*feq3 ;
    f4 =SF*f4 +(1.0f-SF)*feq4 ;
    f5 =SF*f5 +(1.0f-SF)*feq5 ;
    f6 =SF*f6 +(1.0f-SF)*feq6 ;
    f7 =SF*f7 +(1.0f-SF)*feq7 ;
    f8 =SF*f8 +(1.0f-SF)*feq8 ;
    f9 =SF*f9 +(1.0f-SF)*feq9 ;
    f10=SF*f10+(1.0f-SF)*feq10;
    f11=SF*f11+(1.0f-SF)*feq11;
    f12=SF*f12+(1.0f-SF)*feq12;
    f13=SF*f13+(1.0f-SF)*feq13;
    f14=SF*f14+(1.0f-SF)*feq14;
    f15=SF*f15+(1.0f-SF)*feq15;
    f16=SF*f16+(1.0f-SF)*feq16;
    f17=SF*f17+(1.0f-SF)*feq17;
    f18=SF*f18+(1.0f-SF)*feq18;

}

inline __device__ void mrt_scale_cf(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float SF)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;
                                                                                                                
float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

//float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18)   -19.f*(u*u+v*v+w*w);
//float m2  = 12.f*f0+-4.f*f1+-4.f*f2+-4.f*f3+-4.f*f4+f5+f6+f7+f8+-4.f*f9+f10+f11+f12+f13+-4.f*f14+f15+f16+f17+f18 +7.53968254f*(u*u+v*v+w*w);
//float m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//float m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//float m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
//float m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
//float m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
//float m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
//float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
//float m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
//float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
//float m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
//float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
//float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
//float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
//float m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
//float m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
//float m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;





float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;

float omega = 1.0f/(3.0f*(UMAX*OBSTR1*2.f/RE)+0.5f);
float omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));

float PI11 = -0.026315789f*m1-0.5f *omega*m9;
float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
float PI12 = -1.5f*omega*m13;
float PI23 = -1.5f*omega*m14;
float PI13 = -1.5f*omega*m15;
//we know Smag on coarse mesh
float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//omega = 1.0f/(3.0f*(nu0+Cs*Smag*sqrt(2.f))+0.5f);
//omega  = 1.0f/(1.0f/omega+3.f*CS*Smag);
//omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*sqrt(2.f)*LRFACTOR*LRFACTOR);
//omega  = 1.0f/(1.0f/omega +3.f*CS*Smag);
//omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*sqrt(2.f)*LRFACTOR*LRFACTOR);
//omega  = 1.0f/(1.0f/omega +3.f*CS*Smag);
//omega2 = 1.0f/(1.0f*LRLEVEL/omega2-1.f+0.5f+3.f*CS*Smag*sqrt(2.f)*LRFACTOR);

//float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
//float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
//float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
//float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);
//
//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//float tau0c = 1.f/omega;
//float tau = tau0c+0.5*(-tau0c+sqrt(tau0c*tau0c+18.f*CS*Q));//tau_total of coarse mesh
//omega = 1.f/tau;//total omega on coarse mesh
//tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*LRFACTOR*Q));
//omega2= 1.f/tau;

SF = (omega*(1.0f-omega2))/((1.0f-omega)*omega2/LRFACTOR);//for post-collision 
//SF = omega*0.5f/omega2;//for post-streaming, pre-collision?





f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;

}
inline __device__ void mrt_scale_fc_LES(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega, float omega2)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;
                                                                                                                
float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

//float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
//float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
//float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
//float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
//float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
//float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;

//float PI11 = -0.026315789f*m1-0.5f *omega*m9;
//float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//float PI12 = -1.5f*omega*m13;
//float PI23 = -1.5f*omega*m14;
//float PI13 = -1.5f*omega*m15;
////we know Smag on fine mesh. Smag_c=Smag_f*sqrt(2)
//float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
////omega = 1.0f/(3.0f*(nu0+CS*Smag*sqrt(2.f))+0.5f);
////omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*LRFACTOR);
////omega  = 1.0f/(1.0f/omega+3.f*CS*Smag/sqrt(2.f));
////omega2 = 1.0f/(1.0f*LRLEVEL/omega2-1.f+0.5f+3.f*CS*Smag*LRFACTOR);
////omega  = 1.0f/(1.0f/omega+3.f*CS*Smag/sqrt(2.f));

//float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
//float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
//float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
//float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);
//
//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//float tau0f = 1.f/omega2;
//float tau0c = 1.f/omega;
//float tau = tau0f+0.5*(-tau0f+sqrt(tau0f*tau0f+18.f*CS*sqrt(2.f)*Q));//tau_total of fine
//omega2 = 1.f/tau;//total omega on fine mesh
//tau = LRLEVEL*(tau-tau0f)+tau0c;
//omega= 1.f/tau;

//tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*Q));

float SF = (omega*(1.0f-omega2))/((1.0f-omega)*omega2/LRFACTOR);
//float SF = omega2*2.f/omega;



//float SF = ((1.0f-omega)*omega2/LRFACTOR)/(omega*(1.0f-omega2));
//SF = omega*2.f/omega2;

f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;

}




inline __device__ int f_mem(int f_num, int x, int y, int z, size_t pitch)
{

	return (x+y*pitch+z*YDIM*pitch)+f_num*pitch*YDIM*ZDIM;
}

inline __device__ int f_memLR(int f_num, int x, int y, int z, size_t pitch)
{

	return (x+y*pitch+z*YLRDIM*pitch)+f_num*pitch*YLRDIM*ZLRDIM;
}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}
__device__ int dmin_p(int a, int b)
{
	if (a<b) return a;
	else return 0;
}
__device__ int dmax_p(int a, int b)
{
	if (a>-1) return a;
	else return b-1;
}

__global__ void simple_copy(float* fA, float* fB,
							size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int k = dmin(x+1,XDIM)+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)

	fB[j] = fA[k];//+0.01f;
}
__global__ void simple_text(float* fA, float* fB,
							size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	fB[j] = tex2D(texRef_f0A,x+1,y);//+0.01f;
}

__global__ void ExtractFromC_d(float* fout,
							size_t pitch, float omega, float omega2)//pitch in elements
							//size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//	if(x < LRX0+1 || x > LRX0+XLRDIM-2 || y < LRY0+1 || y > LRY0+YLRDIM-2 || z < LRZ0+1 || z > LRZ0+ZLRDIM-2)
//	//if(x < LRX0+2 || x > LRX0+XLRDIM-3 || y < LRY0+2 || y > LRY0+YLRDIM-3 || z < LRZ0+2 || z > LRZ0+ZLRDIM-3)
//	{
//	//do nothing
//	}
//	else{
//	if( (x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1 && z > LRZ0+1 && z < LRZ0+ZLRDIM*LRFACTOR-1) && 
//	(x == int(LRX0+2) || x == int(LRX0+XLRDIM*LRFACTOR-1) || y == int(LRY0+2) || y == int(LRY0+YLRDIM*LRFACTOR-1) || z == int(LRZ0+2) || z == int(LRY0+ZLRDIM*LRFACTOR-1)) )
	//if( (x > LRX0+3 && x < LRX0+XLRDIM*LRFACTOR-3 && y > LRY0+3 && y < LRY0+YLRDIM*LRFACTOR-3))// && 
	if( (x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1))// && 
	//(x == int(LRX0+2) || x == int(LRX0+XLRDIM*LRFACTOR-1) || y == int(LRY0+2) || y == int(LRY0+YLRDIM*LRFACTOR-1)) )
	{
//	if(x > 10 && y > 10 && z > 10 && x < 20 && y < 20 && z < 20)
//	{
	float xcoord = LRLEVEL*(x-LRX0)+0.5f;
	float ycoord = LRLEVEL*(y-LRY0)+0.5f;
	float zcoord = LRLEVEL*(z-LRZ0);
	int zminus = int(zcoord);
	int zplus = zminus+1;
	f0 = (zplus-zcoord)*tex2D(texRef_f0C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0C ,xcoord,ycoord+YLRDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2C ,xcoord,ycoord+YLRDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4C ,xcoord,ycoord+YLRDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9C ,xcoord,ycoord+YLRDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11C,xcoord,ycoord+YLRDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13C,xcoord,ycoord+YLRDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14C,xcoord,ycoord+YLRDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16C,xcoord,ycoord+YLRDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18C,xcoord,ycoord+YLRDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1C ,xcoord,ycoord+YLRDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3C ,xcoord,ycoord+YLRDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5C ,xcoord,ycoord+YLRDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6C ,xcoord,ycoord+YLRDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7C ,xcoord,ycoord+YLRDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8C ,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8C ,xcoord,ycoord+YLRDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15C,xcoord,ycoord+YLRDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17C,xcoord,ycoord+YLRDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10C,xcoord,ycoord+YLRDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12C,xcoord,ycoord+YLRDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12C,xcoord,ycoord+YLRDIM*(zplus));

	if(MODEL == "MRT")
	mrt_scale_fc_LES(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega,omega2);
	//mrt_scale_cf(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,SF);
//    else
//	bgk_scale_cf(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,SF);

	fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
	fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
	fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
	fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
	fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
	fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
	fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
	fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
	fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
	fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
	fout[f_mem(10,x,y,z,pitch)] = f10;
	fout[f_mem(11,x,y,z,pitch)] = f11;
	fout[f_mem(12,x,y,z,pitch)] = f12;
	fout[f_mem(13,x,y,z,pitch)] = f13;
	fout[f_mem(14,x,y,z,pitch)] = f14;
	fout[f_mem(15,x,y,z,pitch)] = f15;
	fout[f_mem(16,x,y,z,pitch)] = f16;
	fout[f_mem(17,x,y,z,pitch)] = f17;
	fout[f_mem(18,x,y,z,pitch)] = f18;

	}
}

__global__ void LR_d_ABCD_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, float *FZ, int t,
                            float *uAv, float *vAv, float *ufluc, float *vfluc)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    float u_Av, v_Av, u_fluc, v_fluc;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();
//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
//	{
//	//dont do anything
//		sumX[threadIdx.x]=0.f;
//		sumY[threadIdx.x]=0.f;
//		sumZ[threadIdx.x]=0.f;
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y       ,z             ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y       ,z             ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z             ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z             ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z             ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z             ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z             ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z             ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }

    if(im == 1 || im ==10){//BB
		if(im == 10){
		check[0] = 1;
		//sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
		//sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
		//sumZ[threadIdx.x]=2.f*f9+2.f*f10+2.f*f11+2.f*f12+2.f*f13-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6;//-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
		sumX[threadIdx.x]+=-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6;//-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
		sumY[threadIdx.x]+=-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
		sumZ[threadIdx.x]=2.f*f9+2.f*f10+2.f*f11+2.f*f12+2.f*f13;//-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
		sumZ[threadIdx.x]+=-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		sumZ[threadIdx.x]=0.f;
		}
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		sumZ[threadIdx.x]=0.f;

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);


		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }



		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}

//	}//end else (not at edge of LR)
	syncthreads();
	if(check[0] == 1 && t>=STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
			sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
		atomicAdd(&FZ[t],sumZ[0]);
	}
	}

}
__global__ void LR_d_BACD_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, float *FZ,
                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();
//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
//	{
//	//dont do anything
//		sumX[threadIdx.x]=0.f;
//		sumY[threadIdx.x]=0.f;
//		sumZ[threadIdx.x]=0.f;
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y       ,z             ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y       ,z             ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z             ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z             ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z             ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z             ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z             ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z             ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }

    if(im == 1 || im ==10){//BB
		if(im == 10){
		check[0] = 1;
		//sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
		//sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
		//sumZ[threadIdx.x]=2.f*f9+2.f*f10+2.f*f11+2.f*f12+2.f*f13-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6;//-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
		sumX[threadIdx.x]+=-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6;//-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
		sumY[threadIdx.x]+=-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
		sumZ[threadIdx.x]=2.f*f9+2.f*f10+2.f*f11+2.f*f12+2.f*f13;//-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
		sumZ[threadIdx.x]+=-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		sumZ[threadIdx.x]=0.f;
		}
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		sumZ[threadIdx.x]=0.f;

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);


		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
        float u_Av, v_Av, u_fluc, v_fluc;
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }



		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}

//	}//end else (not at edge of LR)
	syncthreads();
	if(check[0] == 1 && t>=STARTF){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
			sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
		atomicAdd(&FZ[t],sumZ[0]);
	}
	}

}


__global__ void LR_d_ABCD2(float* fin, float* fout,
							float omega, size_t pitch, int n, int t,//pitch in elements
                            float *uAv, float *vAv, float *ufluc, float *vfluc)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    float u_Av, v_Av, u_fluc, v_fluc;
//	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n || z < n || z > ZLRDIM-1-n)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y       ,z             ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y       ,z             ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z             ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z             ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z             ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z             ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z             ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z             ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }

	if(im == 1 || im ==10){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t+LRFACTOR*n);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t+LRFACTOR*n);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }



		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC2(float* fin, float* fout,
							float omega, size_t pitch, float SF, int n,
                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;//dont need to +0.5f because z is not using texture interpolation
//	int zminus = int(zcoord);
//	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    float u_Av, v_Av, u_fluc, v_fluc;
//	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n || z < n || z > ZLRDIM-1-n)
//	{
//	//no interp
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y       ,z             ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y       ,z             ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z             ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z             ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z             ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z             ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z             ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z             ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }

	//else f18 = 0.1f;
	if(im == 1 || im ==10){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t+LRFACTOR*n);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t+LRFACTOR*n);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }


		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC_Interp(float* fin, float* fout,
							float omega, size_t pitch, float SF,
                            int t, float *uAv, float *vAv, float *ufluc, float *vfluc)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;//dont need to +0.5f because z is not using texture interpolation
	int zminus = int(zcoord);
	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    float u_Av, v_Av, u_fluc, v_fluc;

    if(((x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL || z < LRLEVEL || z > ZLRDIM-1-LRLEVEL) && ZPERIODIC == "NO")
        ||((x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL) && ZPERIODIC == "YES"))
	{
    if(ZPERIODIC == "YES"){
    if(zcoord < 0){ //if zcoord=-0.25f, 1+zcoord=0.75f
	f0 = (1.f+zcoord)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f2 = (1.f+zcoord)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f4 = (1.f+zcoord)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f9 = (1.f+zcoord)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f11= (1.f+zcoord)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f13= (1.f+zcoord)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f14= (1.f+zcoord)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f16= (1.f+zcoord)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f18= (1.f+zcoord)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f1 = (1.f+zcoord)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f3 = (1.f+zcoord)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f5 = (1.f+zcoord)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f6 = (1.f+zcoord)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f7 = (1.f+zcoord)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f8 = (1.f+zcoord)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f15= (1.f+zcoord)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f17= (1.f+zcoord)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f10= (1.f+zcoord)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f12= (1.f+zcoord)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
    }
    else if(zcoord > ZDIM-1){
	f0 = (zplus-zcoord)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f2 = (zplus-zcoord)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f4 = (zplus-zcoord)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f9 = (zplus-zcoord)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f11= (zplus-zcoord)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f13= (zplus-zcoord)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f14= (zplus-zcoord)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f16= (zplus-zcoord)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f18= (zplus-zcoord)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f1 = (zplus-zcoord)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f3 = (zplus-zcoord)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f5 = (zplus-zcoord)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f6 = (zplus-zcoord)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f7 = (zplus-zcoord)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f8 = (zplus-zcoord)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f15= (zplus-zcoord)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f17= (zplus-zcoord)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f10= (zplus-zcoord)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f12= (zplus-zcoord)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
    }
    else{
	//interpolate for next time step. from B //YDIM and not YLRDIM
	f0 = (zplus-zcoord)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
    }
    }
    else{
	//interpolate for next time step. from B //YDIM and not YLRDIM
	f0 = (zplus-zcoord)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8B ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12B,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
    }


	if(MODEL == "MRT")
	mrt_scale_cf(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,SF);
    else
	bgk_scale_cf(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,SF);

	fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
	fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
	fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
	fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
	fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
	fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
	fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
	fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
	fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
	fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
	fout[f_memLR(10,x,y,z,pitch)] = f10;
	fout[f_memLR(11,x,y,z,pitch)] = f11;
	fout[f_memLR(12,x,y,z,pitch)] = f12;
	fout[f_memLR(13,x,y,z,pitch)] = f13;
	fout[f_memLR(14,x,y,z,pitch)] = f14;
	fout[f_memLR(15,x,y,z,pitch)] = f15;
	fout[f_memLR(16,x,y,z,pitch)] = f16;
	fout[f_memLR(17,x,y,z,pitch)] = f17;
	fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y       ,z             ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y       ,z             ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z             ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z             ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z             ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z             ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z             ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z             ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }

	//else f18 = 0.1f;
	if(im == 1 || im ==10){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t+1-LRFACTOR);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t+1-LRFACTOR);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BADC2(float* fin, float* fout,
							float omega, size_t pitch, float SF, int n,
                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;//dont need to +0.5f because z is not using texture interpolation
//	int zminus = int(zcoord);
//	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
//	if(x < n || x > XLRDIM-1-n || y < n || y > YLRDIM-1-n || z < n || z > ZLRDIM-1-n)
//	{
//	//no interp
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y       ,z             ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y       ,z             ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z             ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z             ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z             ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z             ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z             ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z             ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }
	if(im == 1 || im ==10){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
        float u_Av, v_Av, u_fluc, v_fluc;
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t+LRFACTOR*n);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t+LRFACTOR*n);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }


		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
//	}//end else (not at edge of LR)
}

__global__ void LR_d_BADC_Interp(float* fin, float* fout,
							float omega, size_t pitch, float SF,
                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;//dont need to +0.5f because z is not using texture interpolation
	int zminus = int(zcoord);
	int zplus = zminus+1;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    if(((x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL || z < LRLEVEL || z > ZLRDIM-1-LRLEVEL) && ZPERIODIC == "NO")
        ||((x < LRLEVEL || x > XLRDIM-1-LRLEVEL || y < LRLEVEL || y > YLRDIM-1-LRLEVEL) && ZPERIODIC == "YES"))
	{
    if(ZPERIODIC == "YES"){
    if(zcoord < 0){ //if zcoord=-0.25f, 1+zcoord=0.75f
	f0 = (1.f+zcoord)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f2 = (1.f+zcoord)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f4 = (1.f+zcoord)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f9 = (1.f+zcoord)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f11= (1.f+zcoord)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f13= (1.f+zcoord)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f14= (1.f+zcoord)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f16= (1.f+zcoord)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f18= (1.f+zcoord)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f1 = (1.f+zcoord)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f3 = (1.f+zcoord)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f5 = (1.f+zcoord)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f6 = (1.f+zcoord)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f7 = (1.f+zcoord)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f8 = (1.f+zcoord)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f15= (1.f+zcoord)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f17= (1.f+zcoord)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f10= (1.f+zcoord)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
	f12= (1.f+zcoord)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(-zcoord)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(ZDIM-1));
    }
    else if(zcoord > ZDIM-1){
	f0 = (zplus-zcoord)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f2 = (zplus-zcoord)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f4 = (zplus-zcoord)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f9 = (zplus-zcoord)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f11= (zplus-zcoord)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f13= (zplus-zcoord)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f14= (zplus-zcoord)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f16= (zplus-zcoord)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f18= (zplus-zcoord)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f1 = (zplus-zcoord)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f3 = (zplus-zcoord)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f5 = (zplus-zcoord)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f6 = (zplus-zcoord)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f7 = (zplus-zcoord)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f8 = (zplus-zcoord)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f15= (zplus-zcoord)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f17= (zplus-zcoord)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f10= (zplus-zcoord)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
	f12= (zplus-zcoord)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(0));
    }
    else{
	//interpolate for next time step. from A //YDIM and not YLRDIM
	f0 = (zplus-zcoord)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
    }
    }
    else{
	//interpolate for next time step. from A //YDIM and not YLRDIM
	f0 = (zplus-zcoord)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f2 = (zplus-zcoord)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f4 = (zplus-zcoord)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f9 = (zplus-zcoord)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f11= (zplus-zcoord)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f13= (zplus-zcoord)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f14= (zplus-zcoord)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f16= (zplus-zcoord)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f18= (zplus-zcoord)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f1 = (zplus-zcoord)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f3 = (zplus-zcoord)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f5 = (zplus-zcoord)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f6 = (zplus-zcoord)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f7 = (zplus-zcoord)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f8 = (zplus-zcoord)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f15= (zplus-zcoord)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f17= (zplus-zcoord)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f10= (zplus-zcoord)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
	f12= (zplus-zcoord)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12A,xcoord+0.5f,ycoord+0.5f+YDIM*(zplus));
    }


	
	if(MODEL == "MRT")
	mrt_scale_cf(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,SF);
    else
	bgk_scale_cf(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,SF);

	fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
	fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
	fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
	fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
	fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
	fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
	fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
	fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
	fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
	fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
	fout[f_memLR(10,x,y,z,pitch)] = f10;
	fout[f_memLR(11,x,y,z,pitch)] = f11;
	fout[f_memLR(12,x,y,z,pitch)] = f12;
	fout[f_memLR(13,x,y,z,pitch)] = f13;
	fout[f_memLR(14,x,y,z,pitch)] = f14;
	fout[f_memLR(15,x,y,z,pitch)] = f15;
	fout[f_memLR(16,x,y,z,pitch)] = f16;
	fout[f_memLR(17,x,y,z,pitch)] = f17;
	fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y       ,z             ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y       ,z             ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z             ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z             ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z             ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z             ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z             ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z             ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }

	if(im == 1 || im ==10){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
        float u_Av, v_Av, u_fluc, v_fluc;
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t+1-LRFACTOR);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t+1-LRFACTOR);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }

		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABCD(float* fin, float* fout,
							float omega, size_t pitch,//pitch in elements
                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
    float u_Av, v_Av, u_fluc, v_fluc;
//	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
//	im = -1;//not valid for extraction
//	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
//	{
//	im = -2;//not valid for second TS
//	}

//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }
	if(im == 1 || im ==10){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t+LRFACTOR);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t+LRFACTOR);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }



		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
//	}//end else (not at edge of LR)
}



__global__ void LR_d_BACD(float* fin, float* fout,
							float omega, size_t pitch,
                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	int im = ImageFcn(xcoord,ycoord,zcoord);
//	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
//	im = -1;//not valid for extraction
//	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
//	{
//	im = -2;//not valid for second TS
//	}

//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
//	{
//	//dont do anything
//	}
//	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
    if(z != 0){
	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_memLR(9 ,x                 ,y                 ,ZLRDIM-1,pitch)];
	f10= fin[f_memLR(10,dmax_p(x-1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f11= fin[f_memLR(11,x                 ,dmax_p(y-1,YLRDIM),ZLRDIM-1,pitch)];
	f12= fin[f_memLR(12,dmin_p(x+1,XLRDIM),y                 ,ZLRDIM-1,pitch)];
	f13= fin[f_memLR(13,x                 ,dmin_p(y+1,YLRDIM),ZLRDIM-1,pitch)];
    }
    if(z != ZLRDIM-1){
	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_memLR(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_memLR(14,x                 ,y                 ,0  ,pitch)];
	f15= fin[f_memLR(15,dmax_p(x-1,XLRDIM),y                 ,0  ,pitch)];
	f16= fin[f_memLR(16,x                 ,dmax_p(y-1,YLRDIM),0  ,pitch)];
	f17= fin[f_memLR(17,dmin_p(x+1,XLRDIM),y                 ,0  ,pitch)];
	f18= fin[f_memLR(18,x                 ,dmin_p(y+1,YLRDIM),0  ,pitch)];    
    }
	if(im == 1 || im ==10){//BB
		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
		fout[f_memLR(10,x,y,z,pitch)] = f17;
		fout[f_memLR(11,x,y,z,pitch)] = f18;
		fout[f_memLR(12,x,y,z,pitch)] = f15;
		fout[f_memLR(13,x,y,z,pitch)] = f16;
		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
		fout[f_memLR(15,x,y,z,pitch)] = f12;
		fout[f_memLR(16,x,y,z,pitch)] = f13;
		fout[f_memLR(17,x,y,z,pitch)] = f10;
		fout[f_memLR(18,x,y,z,pitch)] = f11;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
        if(VELAV == "YES"){
        float u_Av, v_Av, u_fluc, v_fluc;
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_avLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t+LRFACTOR);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_flucLR(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t+LRFACTOR);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
            }
        }


		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
		fout[f_memLR(10,x,y,z,pitch)] = f10;
		fout[f_memLR(11,x,y,z,pitch)] = f11;
		fout[f_memLR(12,x,y,z,pitch)] = f12;
		fout[f_memLR(13,x,y,z,pitch)] = f13;
		fout[f_memLR(14,x,y,z,pitch)] = f14;
		fout[f_memLR(15,x,y,z,pitch)] = f15;
		fout[f_memLR(16,x,y,z,pitch)] = f16;
		fout[f_memLR(17,x,y,z,pitch)] = f17;
		fout[f_memLR(18,x,y,z,pitch)] = f18;
	}
//	}//end else (not at edge of LR)
}

__global__ void mrt_d_single_force(float* fin, float* fout,
							float omega, size_t pitch, float *FX, float *FY, float *FZ, 
                            int t,float *uAv, float *vAv, float *ufluc, float *vfluc)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;//;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX], sumZ[BLOCKSIZEX];
	__shared__ int check[1];
	check[0] = 0;
	syncthreads();
    float u_Av, v_Av, u_fluc, v_fluc;


//	if((REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 && 
//						y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1 &&
//						z > LRZ0+1 && z < LRZ0+(ZLRDIM-1)*LRFACTOR-1 ||
//						(x>XDIM-1)))
//	{
//	}
//	else{
	int im = ImageFcn(x,y,z);

	f0 = fin[j];
	f1 = fin[f_mem(1 ,x-1,y  ,z  ,pitch)];
	f3 = fin[f_mem(3 ,x+1,y  ,z  ,pitch)];
	f2 = fin[f_mem(2 ,x  ,y-1,z  ,pitch)];
	f5 = fin[f_mem(5 ,x-1,y-1,z  ,pitch)];
	f6 = fin[f_mem(6 ,x+1,y-1,z  ,pitch)];
	f4 = fin[f_mem(4 ,x  ,y+1,z  ,pitch)];
	f7 = fin[f_mem(7 ,x+1,y+1,z  ,pitch)];
	f8 = fin[f_mem(8 ,x-1,y+1,z  ,pitch)];
    if(z != 0){
	f9 = fin[f_mem(9 ,x  ,y  ,z-1,pitch)];
	f10= fin[f_mem(10,x-1,y  ,z-1,pitch)];
	f11= fin[f_mem(11,x  ,y-1,z-1,pitch)];
	f12= fin[f_mem(12,x+1,y  ,z-1,pitch)];
	f13= fin[f_mem(13,x  ,y+1,z-1,pitch)];
    }
    else{
	f9 = fin[f_mem(9 ,x  ,y  ,ZDIM-1,pitch)];
	f10= fin[f_mem(10,dmax_p(x-1,XDIM),y  ,ZDIM-1,pitch)];
	f11= fin[f_mem(11,x  ,dmax_p(y-1,YDIM),ZDIM-1,pitch)];
	f12= fin[f_mem(12,dmin_p(x+1,XDIM),y  ,ZDIM-1,pitch)];
	f13= fin[f_mem(13,x  ,dmin_p(y+1,YDIM),ZDIM-1,pitch)];
    }
    if(z != ZDIM-1){
	f14= fin[f_mem(14,x  ,y  ,z+1,pitch)];
	f15= fin[f_mem(15,x-1,y  ,z+1,pitch)];
	f16= fin[f_mem(16,x  ,y-1,z+1,pitch)];
	f17= fin[f_mem(17,x+1,y  ,z+1,pitch)];
	f18= fin[f_mem(18,x  ,y+1,z+1,pitch)];
    }
    else{
	f14= fin[f_mem(14,x  ,y  ,0  ,pitch)];
	f15= fin[f_mem(15,dmax_p(x-1,XDIM),y  ,0  ,pitch)];
	f16= fin[f_mem(16,x  ,dmax_p(y-1,YDIM),0  ,pitch)];
	f17= fin[f_mem(17,dmin_p(x+1,XDIM),y  ,0  ,pitch)];
	f18= fin[f_mem(18,x  ,dmin_p(y+1,YDIM),0  ,pitch)];    
    }

	if(im == 1 || im == 10){//BB
		if(im == 10){
		check[0] = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7+2.f*f10-2.f*f12+2.f*f15-2.f*f17;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7+2.f*f11-2.f*f13+2.f*f16-2.f*f18;
		sumZ[threadIdx.x]=2.f*f9+2.f*f10+2.f*f11+2.f*f12+2.f*f13-2.f*f14-2.f*f15-2.f*f16-2.f*f17-2.f*f18;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		sumZ[threadIdx.x]=0.f;
		}
		fout[j+pitch*YDIM*ZDIM*1 ] = f3 ;
		fout[j+pitch*YDIM*ZDIM*2 ] = f4 ;
		fout[j+pitch*YDIM*ZDIM*3 ] = f1 ;
		fout[j+pitch*YDIM*ZDIM*4 ] = f2 ;
		fout[j+pitch*YDIM*ZDIM*5 ] = f7 ;
		fout[j+pitch*YDIM*ZDIM*6 ] = f8 ;
		fout[j+pitch*YDIM*ZDIM*7 ] = f5 ;
		fout[j+pitch*YDIM*ZDIM*8 ] = f6 ;
		fout[j+pitch*YDIM*ZDIM*9 ] = f14;
		fout[j+pitch*YDIM*ZDIM*10] = f17;
		fout[j+pitch*YDIM*ZDIM*11] = f18;
		fout[j+pitch*YDIM*ZDIM*12] = f15;
		fout[j+pitch*YDIM*ZDIM*13] = f16;
		fout[j+pitch*YDIM*ZDIM*14] = f9 ;
		fout[j+pitch*YDIM*ZDIM*15] = f12;
		fout[j+pitch*YDIM*ZDIM*16] = f13;
		fout[j+pitch*YDIM*ZDIM*17] = f10;
		fout[j+pitch*YDIM*ZDIM*18] = f11;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		sumZ[threadIdx.x]=0.f;
		boundaries_force(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);

        if(im == 100)//north outlet
        {
           	f0 = fin[f_mem(0 ,x,y-1,z,pitch)];
        	f1 = fin[f_mem(1 ,x,y-1,z,pitch)];
        	f3 = fin[f_mem(3 ,x,y-1,z,pitch)];
        	f2 = fin[f_mem(2 ,x,y-1,z,pitch)];
        	f5 = fin[f_mem(5 ,x,y-1,z,pitch)];
        	f6 = fin[f_mem(6 ,x,y-1,z,pitch)];
        	f4 = fin[f_mem(4 ,x,y-1,z,pitch)];
        	f7 = fin[f_mem(7 ,x,y-1,z,pitch)];
        	f8 = fin[f_mem(8 ,x,y-1,z,pitch)];
        	f9 = fin[f_mem(9 ,x,y-1,z,pitch)];
        	f10= fin[f_mem(10,x,y-1,z,pitch)];
        	f11= fin[f_mem(11,x,y-1,z,pitch)];
        	f12= fin[f_mem(12,x,y-1,z,pitch)];
        	f13= fin[f_mem(13,x,y-1,z,pitch)];
        	f14= fin[f_mem(14,x,y-1,z,pitch)];
        	f15= fin[f_mem(15,x,y-1,z,pitch)];
        	f16= fin[f_mem(16,x,y-1,z,pitch)];
        	f17= fin[f_mem(17,x,y-1,z,pitch)];
        	f18= fin[f_mem(18,x,y-1,z,pitch)];

	float rho,u,v,w;	
	rho = 1.0f;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+    f8+ -4.f*f9+    f10+      f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;

f0 =(0.052631579f*rho                           +- 0.012531328f*(m1)+ 0.047619048f*(m2));
f1 =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)-m10));
f2 =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f3 =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)-m10));                                                                                         
f4 =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f5 =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f6 =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f7 =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f8 =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f9 =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                       
f10=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f11=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
f12=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f13=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f14=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                      
f15=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f16=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f17=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f18=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));

        }
        if(im == 200)//south inlet
        {
           	f0 = fin[f_mem(0 ,x,y+1,z,pitch)];
        	f1 = fin[f_mem(1 ,x,y+1,z,pitch)];
        	f3 = fin[f_mem(3 ,x,y+1,z,pitch)];
        	f2 = fin[f_mem(2 ,x,y+1,z,pitch)];
        	f5 = fin[f_mem(5 ,x,y+1,z,pitch)];
        	f6 = fin[f_mem(6 ,x,y+1,z,pitch)];
        	f4 = fin[f_mem(4 ,x,y+1,z,pitch)];
        	f7 = fin[f_mem(7 ,x,y+1,z,pitch)];
        	f8 = fin[f_mem(8 ,x,y+1,z,pitch)];
        	f9 = fin[f_mem(9 ,x,y+1,z,pitch)];
        	f10= fin[f_mem(10,x,y+1,z,pitch)];
        	f11= fin[f_mem(11,x,y+1,z,pitch)];
        	f12= fin[f_mem(12,x,y+1,z,pitch)];
        	f13= fin[f_mem(13,x,y+1,z,pitch)];
        	f14= fin[f_mem(14,x,y+1,z,pitch)];
        	f15= fin[f_mem(15,x,y+1,z,pitch)];
        	f16= fin[f_mem(16,x,y+1,z,pitch)];
        	f17= fin[f_mem(17,x,y+1,z,pitch)];
        	f18= fin[f_mem(18,x,y+1,z,pitch)];

	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = 0.f;//f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = UMAX;//f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = 0.f;//f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+    f8+ -4.f*f9+    f10+      f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;

f0 =(0.052631579f*rho                           +- 0.012531328f*(m1)+ 0.047619048f*(m2));
f1 =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)-m10));
f2 =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f3 =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)-m10));                                                                                         
f4 =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f5 =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f6 =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f7 =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f8 =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f9 =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                       
f10=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f11=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
f12=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f13=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f14=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                      
f15=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f16=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f17=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f18=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));

        }


        //float Cs = 0.01f;
        //if(XDIM-x < 64.f)
        ////Cs = 0.01f+(x-64.f)/64.f*(x-64.f)/64.f*0.1f;
        //Cs = 0.01f*pow(2.f,((x-448.f)/16.f));

		if(MODEL == "MRT")
		mrt_collide_LES(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega,CS);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

        if(VELAV == "YES"){
            if(t>=START_VELAV && t<START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
        		vel_av(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,t);
                uAv[j] = u_Av;
                vAv[j] = v_Av;
            }
            else if(t>=START_VELFLUC){
                u_Av = uAv[j];
                v_Av = vAv[j];
                u_fluc = ufluc[j];
                v_fluc = vfluc[j];
        		vel_fluc(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,u_Av,v_Av,u_fluc,v_fluc,t);
                ufluc[j] = u_fluc;
                vfluc[j] = v_fluc;
                
            }
        }



		fout[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fout[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fout[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fout[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fout[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fout[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fout[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fout[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fout[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fout[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fout[f_mem(10,x,y,z,pitch)] = f10;
		fout[f_mem(11,x,y,z,pitch)] = f11;
		fout[f_mem(12,x,y,z,pitch)] = f12;
		fout[f_mem(13,x,y,z,pitch)] = f13;
		fout[f_mem(14,x,y,z,pitch)] = f14;
		fout[f_mem(15,x,y,z,pitch)] = f15;
		fout[f_mem(16,x,y,z,pitch)] = f16;
		fout[f_mem(17,x,y,z,pitch)] = f17;
		fout[f_mem(18,x,y,z,pitch)] = f18;
	}

	syncthreads();
	if(check[0] == 1 && t>=STARTF && REFINEMENT == "NO"){
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
			sumZ[threadIdx.x] += sumZ[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
		atomicAdd(&FZ[t],sumZ[0]);
	}
	}


//	}
}


__global__ void mrt_d_single(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,z);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

//	if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 
//		&& y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1 && z > LRZ0+1 && z < LRZ0+(ZLRDIM-1)*LRFACTOR-1 ||
//		(x>XDIM-1)){
//	}
//	else{

	f0 = fA[j];
	f1 = fA[f_mem(1 ,x-1,y       ,z             ,pitch)];
	f3 = fA[f_mem(3 ,x+1,y       ,z             ,pitch)];
	f2 = fA[f_mem(2 ,x  ,y-1,z             ,pitch)];
	f5 = fA[f_mem(5 ,x-1,y-1,z             ,pitch)];
	f6 = fA[f_mem(6 ,x+1,y-1,z             ,pitch)];
	f4 = fA[f_mem(4 ,x  ,y+1,z             ,pitch)];
	f7 = fA[f_mem(7 ,x+1,y+1,z             ,pitch)];
	f8 = fA[f_mem(8 ,x-1,y+1,z             ,pitch)];
	f9 = fA[f_mem(9 ,x  ,y       ,z-1,pitch)];
	f10= fA[f_mem(10,x-1,y       ,z-1,pitch)];
	f11= fA[f_mem(11,x  ,y-1,z-1,pitch)];
	f12= fA[f_mem(12,x+1,y       ,z-1,pitch)];
	f13= fA[f_mem(13,x  ,y+1,z-1,pitch)];
	f14= fA[f_mem(14,x  ,y       ,z+1,pitch)];
	f15= fA[f_mem(15,x-1,y       ,z+1,pitch)];
	f16= fA[f_mem(16,x  ,y-1,z+1,pitch)];
	f17= fA[f_mem(17,x+1,y       ,z+1,pitch)];
	//f18= fA[f_mem(18,x  ,y+1,dmin(z+1,ZDIM),pitch)];
    if(z != ZDIM-1)
	f18= fA[f_mem(18,x  ,y+1,z+1,pitch)];

	if(im == 1 || im ==10){//BB
		fB[f_mem(1 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f14;
		fB[f_mem(10,x,y,z,pitch)] = f17;
		fB[f_mem(11,x,y,z,pitch)] = f18;
		fB[f_mem(12,x,y,z,pitch)] = f15;
		fB[f_mem(13,x,y,z,pitch)] = f16;
		fB[f_mem(14,x,y,z,pitch)] = f9 ;
		fB[f_mem(15,x,y,z,pitch)] = f12;
		fB[f_mem(16,x,y,z,pitch)] = f13;
		fB[f_mem(17,x,y,z,pitch)] = f10;
		fB[f_mem(18,x,y,z,pitch)] = f11;
	}
	else{

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fB[f_mem(0 ,x,y,z,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch)] = f9 ;
		fB[f_mem(10,x,y,z,pitch)] = f10;
		fB[f_mem(11,x,y,z,pitch)] = f11;
		fB[f_mem(12,x,y,z,pitch)] = f12;
		fB[f_mem(13,x,y,z,pitch)] = f13;
		fB[f_mem(14,x,y,z,pitch)] = f14;
		fB[f_mem(15,x,y,z,pitch)] = f15;
		fB[f_mem(16,x,y,z,pitch)] = f16;
		fB[f_mem(17,x,y,z,pitch)] = f17;
		fB[f_mem(18,x,y,z,pitch)] = f18;
	}
//	}
}

__device__ __inline__ float ld_gb1_cg(const float *addr){
    float return_value;
    asm("ld.global.cg.f32 %0, [%1];" : "=f"(return_value) : "l"(addr));
    return return_value;
}




__global__ void initialize_single(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	
	int im = ImageFcn(x,y,z);
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = UMAX;
	w = 0.0f;

    if(im == 10 || im == 1){
    u = 0.0f;
    v = 0.0f;
    w = 0.0f;
    }
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

    if(MODEL == "BGK"){ 
	f[j+0 *pitch*YDIM*ZDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
	f[j+12*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
	f[j+14*pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);   
    }
    else{
                                                                                                                
float f0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float f1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float f2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float f3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float f4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float f5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float f6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float f7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float f8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float f9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float f10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float f11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float f12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float f13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float f14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float f15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float f16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float f17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float f18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

f1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
f12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
f14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
f17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

f[j+0 *pitch*YDIM*ZDIM]=f0 ;
f[j+1 *pitch*YDIM*ZDIM]=f1 ;
f[j+2 *pitch*YDIM*ZDIM]=f2 ;
f[j+3 *pitch*YDIM*ZDIM]=f3 ;
f[j+4 *pitch*YDIM*ZDIM]=f4 ;
f[j+5 *pitch*YDIM*ZDIM]=f5 ;
f[j+6 *pitch*YDIM*ZDIM]=f6 ;
f[j+7 *pitch*YDIM*ZDIM]=f7 ;
f[j+8 *pitch*YDIM*ZDIM]=f8 ;
f[j+9 *pitch*YDIM*ZDIM]=f9 ;
f[j+10*pitch*YDIM*ZDIM]=f10;
f[j+11*pitch*YDIM*ZDIM]=f11;
f[j+12*pitch*YDIM*ZDIM]=f12;
f[j+13*pitch*YDIM*ZDIM]=f13;
f[j+14*pitch*YDIM*ZDIM]=f14;
f[j+15*pitch*YDIM*ZDIM]=f15;
f[j+16*pitch*YDIM*ZDIM]=f16;
f[j+17*pitch*YDIM*ZDIM]=f17;
f[j+18*pitch*YDIM*ZDIM]=f18;

    }


	if(x == XDIM-1){
	for(int i = XDIM; i<pitch; i++){
		j = i+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
		f[j+0 *pitch*YDIM*ZDIM]=0.f;
		f[j+1 *pitch*YDIM*ZDIM]=0.f;
		f[j+2 *pitch*YDIM*ZDIM]=0.f;
		f[j+3 *pitch*YDIM*ZDIM]=0.f;
		f[j+4 *pitch*YDIM*ZDIM]=0.f;
		f[j+5 *pitch*YDIM*ZDIM]=0.f;
		f[j+6 *pitch*YDIM*ZDIM]=0.f;
		f[j+7 *pitch*YDIM*ZDIM]=0.f;
		f[j+8 *pitch*YDIM*ZDIM]=0.f;
		f[j+9 *pitch*YDIM*ZDIM]=0.f;
		f[j+10*pitch*YDIM*ZDIM]=0.f;
		f[j+11*pitch*YDIM*ZDIM]=0.f;
		f[j+12*pitch*YDIM*ZDIM]=0.f;
		f[j+13*pitch*YDIM*ZDIM]=0.f;
		f[j+14*pitch*YDIM*ZDIM]=0.f;
		f[j+15*pitch*YDIM*ZDIM]=0.f;
		f[j+16*pitch*YDIM*ZDIM]=0.f;
		f[j+17*pitch*YDIM*ZDIM]=0.f;
		f[j+18*pitch*YDIM*ZDIM]=0.f;
	}
	}
}
__global__ void initialize_LR(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	float zcoord = LRZ0+z*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord,zcoord);
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = UMAX;//0.0f;
	w = 0.0f;
    if(im == 10 || im == 1){
    u = 0.0f;
    v = 0.0f;
    w = 0.0f;
    }
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

    if(MODEL == "BGK"){ 
	f[j+0 *pitch*YLRDIM*ZLRDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
	f[j+12*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
	f[j+14*pitch*YLRDIM*ZLRDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*YLRDIM*ZLRDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);   
    }
    else{
                                                                                                                
float f0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float f1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float f2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float f3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float f4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float f5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float f6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float f7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float f8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float f9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float f10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float f11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float f12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float f13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float f14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float f15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float f16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float f17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float f18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

f1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
f12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
f14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
f17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

f[j+0 *pitch*YLRDIM*ZLRDIM]=f0 ;
f[j+1 *pitch*YLRDIM*ZLRDIM]=f1 ;
f[j+2 *pitch*YLRDIM*ZLRDIM]=f2 ;
f[j+3 *pitch*YLRDIM*ZLRDIM]=f3 ;
f[j+4 *pitch*YLRDIM*ZLRDIM]=f4 ;
f[j+5 *pitch*YLRDIM*ZLRDIM]=f5 ;
f[j+6 *pitch*YLRDIM*ZLRDIM]=f6 ;
f[j+7 *pitch*YLRDIM*ZLRDIM]=f7 ;
f[j+8 *pitch*YLRDIM*ZLRDIM]=f8 ;
f[j+9 *pitch*YLRDIM*ZLRDIM]=f9 ;
f[j+10*pitch*YLRDIM*ZLRDIM]=f10;
f[j+11*pitch*YLRDIM*ZLRDIM]=f11;
f[j+12*pitch*YLRDIM*ZLRDIM]=f12;
f[j+13*pitch*YLRDIM*ZLRDIM]=f13;
f[j+14*pitch*YLRDIM*ZLRDIM]=f14;
f[j+15*pitch*YLRDIM*ZLRDIM]=f15;
f[j+16*pitch*YLRDIM*ZLRDIM]=f16;
f[j+17*pitch*YLRDIM*ZLRDIM]=f17;
f[j+18*pitch*YLRDIM*ZLRDIM]=f18;

    }



}
__global__ void initialize(float* f0, float* f1, float* f2,
							float* f3, float* f4, float* f5,
							float* f6, float* f7, float* f8, float* f9,
							float* f10, float* f11, float* f12,
							float* f13, float* f14, float* f15,
							float* f16, float* f17, float* f18,
							size_t pitch)//pitch in elements
//__global__ void initialize(void** f0in, void** f1in, 
//							int w, int h, int pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	int i = x+y*XDIM+z*XDIM*YDIM;//index on linear mem
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
//	f1out[j] = tex2D(texRef_f2A,x,y+h*z);
	
	float u,v,w,rho,feq,usqr;
	rho = 1.0f;
	u = 0.0f;
	v = 0.0f;
	w = 0.0f;
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

	feq = 1.0f/3.0f*(rho-1.5f*usqr);
	f0[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f1[j] = feq;
	feq = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f2[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f3[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f4[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f5[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f6[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f7[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f8[j] = feq;

	feq = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f9[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f10[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f11[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f12[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f13[j] = feq;
	feq = 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f14[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f15[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f16[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f17[j] = feq;
	feq = 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
	f18[j] = feq;
}



int main(int argc, char *argv[])
{

	//int *image_d, *image_h;

	ofstream output;
	ofstream output2;
	string FileName = CASENAME;
	//output.open ("LBM1_out.dat");
	output.open ((FileName+".dat").c_str());
	output2.open ((FileName+".force").c_str());

	size_t memsize, memsize2;
	size_t pitch = 0;
	size_t pitch2 = 0;
	int i, n, nBlocks, nBlocks2, n2;
	float omega, CharLength, omega2;

	if(abs(LRFACTOR-1.f/LRLEVEL)>0.001f){
		cout<<"LRLEVEL and LRFACTOR don't match! Exiting..."<<endl;
		return 0;
	}

	CharLength = OBSTR1*2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));
	if(LRFACTOR == 0.25f){
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega2-1.0f));
	}
	float SF_cf = omega*(1.0f-omega2)/((1.0f-omega)*omega2/LRFACTOR);
	//float SF_fc = 1.f/SF_cf;

	cout<<"omega : "<<omega<<endl;
	cout<<"omega2: "<<omega2<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<"x"<<BLOCKSIZEZ<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<"x"<<ZDIM<<endl;
	cout<<"LRblocksize: "<<BLOCKSIZELRX<<"x"<<BLOCKSIZELRY<<"x"<<BLOCKSIZELRZ<<endl;
	cout<<"LRgrid: "<<XLRDIM<<"x"<<YLRDIM<<"x"<<ZLRDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

	nBlocks = ((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX)*(YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY)
				*(ZDIM/BLOCKSIZEZ+ZDIM%BLOCKSIZEZ);
	nBlocks2 = (XLRDIM/BLOCKSIZELRX+XLRDIM%BLOCKSIZELRX)*(YLRDIM/BLOCKSIZELRY+YLRDIM%BLOCKSIZELRY)
				*(ZLRDIM/BLOCKSIZELRZ+ZLRDIM%BLOCKSIZELRZ);
	int B = BLOCKSIZEX*BLOCKSIZEY*BLOCKSIZEZ;
	int B2 = BLOCKSIZELRX*BLOCKSIZELRY*BLOCKSIZELRZ;
	n = nBlocks*B;
	n2 = nBlocks2*B2;
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
    dim3 grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),YDIM/BLOCKSIZEY,ZDIM/BLOCKSIZEZ);

    dim3 threads2(BLOCKSIZELRX, BLOCKSIZELRY, BLOCKSIZELRZ);
    dim3 grid2(XLRDIM/BLOCKSIZELRX,YLRDIM/BLOCKSIZELRY,ZLRDIM/BLOCKSIZELRZ);

	memsize = n*sizeof(float);
	//memsize_int = n*sizeof(int);
	memsize2 = n2*sizeof(float);

	//hipExtent extent = make_hipExtent(XDIM*sizeof(float),YDIM,ZDIM);

	//image_h = (int *)malloc(memsize_int);

	float *fA_h,*fA_d,*fB_d,*fC_h,*fC_d,*fD_d;
	float *FX_h,*FY_h,*FZ_h,*FX_d,*FY_d,*FZ_d;
	float *uAv_h,*vAv_h,*wAv_h,*uAv_d,*vAv_d,*wAv_d;
	float *uAvLR_h,*vAvLR_h,*wAvLR_h,*uAvLR_d,*vAvLR_d,*wAvLR_d;
	float *ufluc_h,*vfluc_h,*wfluc_h,*ufluc_d,*vfluc_d,*wfluc_d;
	float *uflucLR_h,*vflucLR_h,*wflucLR_h,*uflucLR_d,*vflucLR_d,*wflucLR_d;
	fA_h = (float *)malloc(memsize*19);
	fC_h = (float *)malloc(memsize2*19);
	FX_h = (float *)malloc(TMAX*sizeof(float));
	FY_h = (float *)malloc(TMAX*sizeof(float));
	FZ_h = (float *)malloc(TMAX*sizeof(float));
	uAv_h = (float *)malloc(XDIM*YDIM*ZDIM*sizeof(float));
	vAv_h = (float *)malloc(XDIM*YDIM*ZDIM*sizeof(float));
	wAv_h = (float *)malloc(XDIM*YDIM*ZDIM*sizeof(float));
	uAvLR_h = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM*sizeof(float));
	vAvLR_h = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM*sizeof(float));
	wAvLR_h = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM*sizeof(float));
	ufluc_h = (float *)malloc(XDIM*YDIM*ZDIM*sizeof(float));
	vfluc_h = (float *)malloc(XDIM*YDIM*ZDIM*sizeof(float));
	wfluc_h = (float *)malloc(XDIM*YDIM*ZDIM*sizeof(float));
	uflucLR_h = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM*sizeof(float));
	vflucLR_h = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM*sizeof(float));
	wflucLR_h = (float *)malloc(XLRDIM*YLRDIM*ZLRDIM*sizeof(float));
	hipMallocPitch((void **) &fA_d, &pitch, XDIM*sizeof(float), YDIM*ZDIM*19);
	hipMallocPitch((void **) &fB_d, &pitch, XDIM*sizeof(float), YDIM*ZDIM*19);
	hipMallocPitch((void **) &uAv_d,&pitch, XDIM*sizeof(float), YDIM*ZDIM);
	hipMallocPitch((void **) &vAv_d,&pitch, XDIM*sizeof(float), YDIM*ZDIM);
	hipMallocPitch((void **) &wAv_d,&pitch, XDIM*sizeof(float), YDIM*ZDIM);
	hipMallocPitch((void **) &ufluc_d,&pitch, XDIM*sizeof(float), YDIM*ZDIM);
	hipMallocPitch((void **) &vfluc_d,&pitch, XDIM*sizeof(float), YDIM*ZDIM);
	hipMallocPitch((void **) &wfluc_d,&pitch, XDIM*sizeof(float), YDIM*ZDIM);

	if(REFINEMENT == "YES"){
	hipMallocPitch((void **) &fC_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM*19);
	hipMallocPitch((void **) &fD_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM*19);
    if(VELAV == "YES"){
	hipMallocPitch((void **) &uAvLR_d,&pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM);
	hipMallocPitch((void **) &vAvLR_d,&pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM);
	hipMallocPitch((void **) &wAvLR_d,&pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM);
	hipMallocPitch((void **) &uflucLR_d,&pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM);
	hipMallocPitch((void **) &vflucLR_d,&pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM);
	hipMallocPitch((void **) &wflucLR_d,&pitch2, XLRDIM*sizeof(float), YLRDIM*ZLRDIM);
	}
	}

	hipMalloc((void **) &FX_d, TMAX*sizeof(float));
	hipMalloc((void **) &FY_d, TMAX*sizeof(float));
	hipMalloc((void **) &FZ_d, TMAX*sizeof(float));

	cout<<pitch<<", "<<pitch2<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);
	size_t pitch_elements2 = pitch2/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*19; i++)
	{
		fA_h[i] = i;
	}
	for (i = 0; i < n2*19; i++)
	{
		fC_h[i] = 0;
	}
	for (i = 0; i < TMAX; i++){
		FX_h[i] = 0.f;
		FY_h[i] = 0.f;
		FZ_h[i] = 0.f;
	}
	for (i = 0; i < n; i++)
	{
		uAv_h[i] = 0;
		vAv_h[i] = 0;
		wAv_h[i] = 0;
		ufluc_h[i] = 0;
		vfluc_h[i] = 0;
		wfluc_h[i] = 0;
	}
	for (i = 0; i < n2; i++)
	{
		uAvLR_h[i] = 0;
		vAvLR_h[i] = 0;
		wAvLR_h[i] = 0;
		uflucLR_h[i] = 0;
		vflucLR_h[i] = 0;
		wflucLR_h[i] = 0;
	}

	hipMemcpy(FX_d, FX_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FY_d, FY_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FZ_d, FZ_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
    if(VELAV == "YES"){
	hipMemcpy2D(uAv_d,   pitch, uAv_h, XDIM*sizeof(float), XDIM*sizeof(float), YDIM*ZDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(vAv_d,   pitch, vAv_h, XDIM*sizeof(float), XDIM*sizeof(float), YDIM*ZDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(wAv_d,   pitch, wAv_h, XDIM*sizeof(float), XDIM*sizeof(float), YDIM*ZDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(ufluc_d, pitch, uAv_h, XDIM*sizeof(float), XDIM*sizeof(float), YDIM*ZDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(vfluc_d, pitch, vAv_h, XDIM*sizeof(float), XDIM*sizeof(float), YDIM*ZDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(wfluc_d, pitch, wAv_h, XDIM*sizeof(float), XDIM*sizeof(float), YDIM*ZDIM, hipMemcpyHostToDevice);
    if(REFINEMENT == "YES"){
	hipMemcpy2D(uAvLR_d,  pitch2, uAvLR_h, XLRDIM*sizeof(float), XLRDIM*sizeof(float), YLRDIM*ZLRDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(vAvLR_d,  pitch2, vAvLR_h, XLRDIM*sizeof(float), XLRDIM*sizeof(float), YLRDIM*ZLRDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(wAvLR_d,  pitch2, wAvLR_h, XLRDIM*sizeof(float), XLRDIM*sizeof(float), YLRDIM*ZLRDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(uflucLR_d,pitch2, uAvLR_h, XLRDIM*sizeof(float), XLRDIM*sizeof(float), YLRDIM*ZLRDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(vflucLR_d,pitch2, vAvLR_h, XLRDIM*sizeof(float), XLRDIM*sizeof(float), YLRDIM*ZLRDIM, hipMemcpyHostToDevice);
	hipMemcpy2D(wflucLR_d,pitch2, wAvLR_h, XLRDIM*sizeof(float), XLRDIM*sizeof(float), YLRDIM*ZLRDIM, hipMemcpyHostToDevice);
    }
    }
//	for (i = 0; i < n; i++)
//	{
//		int x = i%XDIM;
//		int y = (i/XDIM)%YDIM;
//		int z = (i/XDIM)/YDIM;
////		image_h[i] = 0;
////		if(x < 1) image_h[i] = 1;//DirichletWest
////		if(x > XDIM-2) image_h[i] = 1;//BB
////		if(y < 1) image_h[i] = 1;//BB
////		if(y > YDIM-2) image_h[i] = 1;//BB
////		if(z < 1) image_h[i] = 1;//DirichletWest
////		if(z > ZDIM-2) image_h[i] = 1;//BB
//	}
	//hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;
	texRef_f9B.normalized = false;
	texRef_f10B.normalized = false;
	texRef_f11B.normalized = false;
	texRef_f12B.normalized = false;
	texRef_f13B.normalized = false;
	texRef_f14B.normalized = false;
	texRef_f15B.normalized = false;
	texRef_f16B.normalized = false;
	texRef_f17B.normalized = false;
	texRef_f18B.normalized = false;
	texRef_f0B.filterMode = hipFilterModeLinear;
	texRef_f1B.filterMode = hipFilterModeLinear;
	texRef_f2B.filterMode = hipFilterModeLinear;
	texRef_f3B.filterMode = hipFilterModeLinear;
	texRef_f4B.filterMode = hipFilterModeLinear;
	texRef_f5B.filterMode = hipFilterModeLinear;
	texRef_f6B.filterMode = hipFilterModeLinear;
	texRef_f7B.filterMode = hipFilterModeLinear;
	texRef_f8B.filterMode = hipFilterModeLinear;
	texRef_f9B.filterMode = hipFilterModeLinear;
	texRef_f10B.filterMode = hipFilterModeLinear;
	texRef_f11B.filterMode = hipFilterModeLinear;
	texRef_f12B.filterMode = hipFilterModeLinear;
	texRef_f13B.filterMode = hipFilterModeLinear;
	texRef_f14B.filterMode = hipFilterModeLinear;
	texRef_f15B.filterMode = hipFilterModeLinear;
	texRef_f16B.filterMode = hipFilterModeLinear;
	texRef_f17B.filterMode = hipFilterModeLinear;
	texRef_f18B.filterMode = hipFilterModeLinear;

	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f9A.normalized = false;
	texRef_f10A.normalized = false;
	texRef_f11A.normalized = false;
	texRef_f12A.normalized = false;
	texRef_f13A.normalized = false;
	texRef_f14A.normalized = false;
	texRef_f15A.normalized = false;
	texRef_f16A.normalized = false;
	texRef_f17A.normalized = false;
	texRef_f18A.normalized = false;
	texRef_f0A.filterMode = hipFilterModeLinear;
	texRef_f1A.filterMode = hipFilterModeLinear;
	texRef_f2A.filterMode = hipFilterModeLinear;
	texRef_f3A.filterMode = hipFilterModeLinear;
	texRef_f4A.filterMode = hipFilterModeLinear;
	texRef_f5A.filterMode = hipFilterModeLinear;
	texRef_f6A.filterMode = hipFilterModeLinear;
	texRef_f7A.filterMode = hipFilterModeLinear;
	texRef_f8A.filterMode = hipFilterModeLinear;
	texRef_f9A.filterMode = hipFilterModeLinear;
	texRef_f10A.filterMode = hipFilterModeLinear;
	texRef_f11A.filterMode = hipFilterModeLinear;
	texRef_f12A.filterMode = hipFilterModeLinear;
	texRef_f13A.filterMode = hipFilterModeLinear;
	texRef_f14A.filterMode = hipFilterModeLinear;
	texRef_f15A.filterMode = hipFilterModeLinear;
	texRef_f16A.filterMode = hipFilterModeLinear;
	texRef_f17A.filterMode = hipFilterModeLinear;
	texRef_f18A.filterMode = hipFilterModeLinear;

//	if(REFINEMENT == "YES"){
	texRef_f0C.normalized = false;
	texRef_f1C.normalized = false;
	texRef_f2C.normalized = false;
	texRef_f3C.normalized = false;
	texRef_f4C.normalized = false;
	texRef_f5C.normalized = false;
	texRef_f6C.normalized = false;
	texRef_f7C.normalized = false;
	texRef_f8C.normalized = false;
	texRef_f9C.normalized = false;
	texRef_f10C.normalized = false;
	texRef_f11C.normalized = false;
	texRef_f12C.normalized = false;
	texRef_f13C.normalized = false;
	texRef_f14C.normalized = false;
	texRef_f15C.normalized = false;
	texRef_f16C.normalized = false;
	texRef_f17C.normalized = false;
	texRef_f18C.normalized = false;
	texRef_f0C.filterMode = hipFilterModeLinear;
	texRef_f1C.filterMode = hipFilterModeLinear;
	texRef_f2C.filterMode = hipFilterModeLinear;
	texRef_f3C.filterMode = hipFilterModeLinear;
	texRef_f4C.filterMode = hipFilterModeLinear;
	texRef_f5C.filterMode = hipFilterModeLinear;
	texRef_f6C.filterMode = hipFilterModeLinear;
	texRef_f7C.filterMode = hipFilterModeLinear;
	texRef_f8C.filterMode = hipFilterModeLinear;
	texRef_f9C.filterMode = hipFilterModeLinear;
	texRef_f10C.filterMode = hipFilterModeLinear;
	texRef_f11C.filterMode = hipFilterModeLinear;
	texRef_f12C.filterMode = hipFilterModeLinear;
	texRef_f13C.filterMode = hipFilterModeLinear;
	texRef_f14C.filterMode = hipFilterModeLinear;
	texRef_f15C.filterMode = hipFilterModeLinear;
	texRef_f16C.filterMode = hipFilterModeLinear;
	texRef_f17C.filterMode = hipFilterModeLinear;
	texRef_f18C.filterMode = hipFilterModeLinear;
	texRef_f0D.normalized = false;
	texRef_f1D.normalized = false;
	texRef_f2D.normalized = false;
	texRef_f3D.normalized = false;
	texRef_f4D.normalized = false;
	texRef_f5D.normalized = false;
	texRef_f6D.normalized = false;
	texRef_f7D.normalized = false;
	texRef_f8D.normalized = false;
	texRef_f9D.normalized = false;
	texRef_f10D.normalized = false;
	texRef_f11D.normalized = false;
	texRef_f12D.normalized = false;
	texRef_f13D.normalized = false;
	texRef_f14D.normalized = false;
	texRef_f15D.normalized = false;
	texRef_f16D.normalized = false;
	texRef_f17D.normalized = false;
	texRef_f18D.normalized = false;
	texRef_f0D.filterMode = hipFilterModeLinear;
	texRef_f1D.filterMode = hipFilterModeLinear;
	texRef_f2D.filterMode = hipFilterModeLinear;
	texRef_f3D.filterMode = hipFilterModeLinear;
	texRef_f4D.filterMode = hipFilterModeLinear;
	texRef_f5D.filterMode = hipFilterModeLinear;
	texRef_f6D.filterMode = hipFilterModeLinear;
	texRef_f7D.filterMode = hipFilterModeLinear;
	texRef_f8D.filterMode = hipFilterModeLinear;
	texRef_f9D.filterMode = hipFilterModeLinear;
	texRef_f10D.filterMode = hipFilterModeLinear;
	texRef_f11D.filterMode = hipFilterModeLinear;
	texRef_f12D.filterMode = hipFilterModeLinear;
	texRef_f13D.filterMode = hipFilterModeLinear;
	texRef_f14D.filterMode = hipFilterModeLinear;
	texRef_f15D.filterMode = hipFilterModeLinear;
	texRef_f16D.filterMode = hipFilterModeLinear;
	texRef_f17D.filterMode = hipFilterModeLinear;
	texRef_f18D.filterMode = hipFilterModeLinear;
//	}

	for(int i = 0; i<2; i++){
	texRef_f0A.addressMode[i]  = hipAddressModeClamp;
	texRef_f1A.addressMode[i]  = hipAddressModeClamp;
	texRef_f2A.addressMode[i]  = hipAddressModeClamp;
	texRef_f3A.addressMode[i]  = hipAddressModeClamp;
	texRef_f4A.addressMode[i]  = hipAddressModeClamp;
	texRef_f5A.addressMode[i]  = hipAddressModeClamp;
	texRef_f6A.addressMode[i]  = hipAddressModeClamp;
	texRef_f7A.addressMode[i]  = hipAddressModeClamp;
	texRef_f8A.addressMode[i]  = hipAddressModeClamp;
	texRef_f9A.addressMode[i]  = hipAddressModeClamp;
	texRef_f10A.addressMode[i] = hipAddressModeClamp;
	texRef_f11A.addressMode[i] = hipAddressModeClamp;
	texRef_f12A.addressMode[i] = hipAddressModeClamp;
	texRef_f13A.addressMode[i] = hipAddressModeClamp;
	texRef_f14A.addressMode[i] = hipAddressModeClamp;
	texRef_f15A.addressMode[i] = hipAddressModeClamp;
	texRef_f16A.addressMode[i] = hipAddressModeClamp;
	texRef_f17A.addressMode[i] = hipAddressModeClamp;
	texRef_f18A.addressMode[i] = hipAddressModeClamp;
	texRef_f0B.addressMode[i]  = hipAddressModeClamp;
	texRef_f1B.addressMode[i]  = hipAddressModeClamp;
	texRef_f2B.addressMode[i]  = hipAddressModeClamp;
	texRef_f3B.addressMode[i]  = hipAddressModeClamp;
	texRef_f4B.addressMode[i]  = hipAddressModeClamp;
	texRef_f5B.addressMode[i]  = hipAddressModeClamp;
	texRef_f6B.addressMode[i]  = hipAddressModeClamp;
	texRef_f7B.addressMode[i]  = hipAddressModeClamp;
	texRef_f8B.addressMode[i]  = hipAddressModeClamp;
	texRef_f9B.addressMode[i]  = hipAddressModeClamp;
	texRef_f10B.addressMode[i] = hipAddressModeClamp;
	texRef_f11B.addressMode[i] = hipAddressModeClamp;
	texRef_f12B.addressMode[i] = hipAddressModeClamp;
	texRef_f13B.addressMode[i] = hipAddressModeClamp;
	texRef_f14B.addressMode[i] = hipAddressModeClamp;
	texRef_f15B.addressMode[i] = hipAddressModeClamp;
	texRef_f16B.addressMode[i] = hipAddressModeClamp;
	texRef_f17B.addressMode[i] = hipAddressModeClamp;
	texRef_f18B.addressMode[i] = hipAddressModeClamp;
	}

	}
	
	hipMemcpy2D(fA_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyHostToDevice);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*ZLRDIM*19,hipMemcpyHostToDevice);
	hipMemcpy2D(fD_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*ZLRDIM*19,hipMemcpyHostToDevice);
	}
//	for (i = 0; i < n*19; i++)
//	{
//		fA_h[i] = 0;
//		fC_h[i] = 1;
//	}


	if(true)//bind texture
	{
	hipBindTexture2D(0,&texRef_f0A, fA_d                            ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f1A, fA_d+pitch_elements*YDIM*ZDIM   ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f2A, fA_d+pitch_elements*YDIM*ZDIM*2 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f3A, fA_d+pitch_elements*YDIM*ZDIM*3 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f4A, fA_d+pitch_elements*YDIM*ZDIM*4 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f5A, fA_d+pitch_elements*YDIM*ZDIM*5 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f6A, fA_d+pitch_elements*YDIM*ZDIM*6 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f7A, fA_d+pitch_elements*YDIM*ZDIM*7 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f8A, fA_d+pitch_elements*YDIM*ZDIM*8 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f9A, fA_d+pitch_elements*YDIM*ZDIM*9 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f10A,fA_d+pitch_elements*YDIM*ZDIM*10,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f11A,fA_d+pitch_elements*YDIM*ZDIM*11,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f12A,fA_d+pitch_elements*YDIM*ZDIM*12,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f13A,fA_d+pitch_elements*YDIM*ZDIM*13,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f14A,fA_d+pitch_elements*YDIM*ZDIM*14,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f15A,fA_d+pitch_elements*YDIM*ZDIM*15,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f16A,fA_d+pitch_elements*YDIM*ZDIM*16,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f17A,fA_d+pitch_elements*YDIM*ZDIM*17,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f18A,fA_d+pitch_elements*YDIM*ZDIM*18,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f0B, fB_d                            ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f1B, fB_d+pitch_elements*YDIM*ZDIM   ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f2B, fB_d+pitch_elements*YDIM*ZDIM*2 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f3B, fB_d+pitch_elements*YDIM*ZDIM*3 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f4B, fB_d+pitch_elements*YDIM*ZDIM*4 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f5B, fB_d+pitch_elements*YDIM*ZDIM*5 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f6B, fB_d+pitch_elements*YDIM*ZDIM*6 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f7B, fB_d+pitch_elements*YDIM*ZDIM*7 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f8B, fB_d+pitch_elements*YDIM*ZDIM*8 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f9B, fB_d+pitch_elements*YDIM*ZDIM*9 ,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f10B,fB_d+pitch_elements*YDIM*ZDIM*10,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f11B,fB_d+pitch_elements*YDIM*ZDIM*11,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f12B,fB_d+pitch_elements*YDIM*ZDIM*12,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f13B,fB_d+pitch_elements*YDIM*ZDIM*13,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f14B,fB_d+pitch_elements*YDIM*ZDIM*14,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f15B,fB_d+pitch_elements*YDIM*ZDIM*15,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f16B,fB_d+pitch_elements*YDIM*ZDIM*16,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f17B,fB_d+pitch_elements*YDIM*ZDIM*17,&desc,XDIM,YDIM*ZDIM,pitch);
	hipBindTexture2D(0,&texRef_f18B,fB_d+pitch_elements*YDIM*ZDIM*18,&desc,XDIM,YDIM*ZDIM,pitch);

//	if(REFINEMENT == "YES"){
	hipBindTexture2D(0,&texRef_f0C, fC_d                                 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1C, fC_d+pitch_elements2*YLRDIM*ZLRDIM   ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*2 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*3 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*4 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*5 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*6 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*7 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*8 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f9C, fC_d+pitch_elements2*YLRDIM*ZLRDIM*9 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f10C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*10,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f11C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*11,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f12C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*12,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f13C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*13,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f14C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*14,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f15C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*15,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f16C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*16,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f17C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*17,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f18C,fC_d+pitch_elements2*YLRDIM*ZLRDIM*18,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f0D, fD_d                                 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1D, fD_d+pitch_elements2*YLRDIM*ZLRDIM   ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*2 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*3 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*4 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*5 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*6 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*7 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*8 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f9D, fD_d+pitch_elements2*YLRDIM*ZLRDIM*9 ,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f10D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*10,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f11D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*11,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f12D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*12,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f13D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*13,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f14D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*14,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f15D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*15,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f16D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*16,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f17D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*17,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f18D,fD_d+pitch_elements2*YLRDIM*ZLRDIM*18,&desc,XLRDIM,YLRDIM*ZLRDIM,pitch2);
//	}
	}


	initialize_single<<<grid, threads>>>(fA_d,pitch_elements);
	initialize_single<<<grid, threads>>>(fB_d,pitch_elements);
	if(REFINEMENT == "YES"){
	initialize_LR<<<grid2, threads2>>>(fC_d,pitch_elements2);
	initialize_LR<<<grid2, threads2>>>(fD_d,pitch_elements2);
	}

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(mrt_d_single),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_ABCD_force),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_ABCD),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_ABCD2),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_ABDC2),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_ABDC_Interp),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_BACD_force),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_BACD),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_BADC2),hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LR_d_BADC_Interp),hipFuncCachePreferL1);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ExtractFromC_d),hipFuncCachePreferL1);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(simple_copy),hipFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
	for(int t = 0; t<TMAX; t=t+2){
		if(METHOD == "SINGLE"){
		if(t >= STARTF)
		mrt_d_single_force<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,FX_d,FY_d,FZ_d,t,uAv_d,vAv_d,ufluc_d,vfluc_d);
		else
		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
		
		if(REFINEMENT == "YES"){
			if(LRFACTOR == 0.5f)
			{
			if(t >= STARTF)
			LR_d_ABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			else
			LR_d_ABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			//LR_d_ABDC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
			LR_d_ABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			}
			else if(LRFACTOR == 0.25f)
			{
			if(t >= STARTF)
			LR_d_ABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			else
			LR_d_ABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			LR_d_ABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			LR_d_ABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			LR_d_ABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,t,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			}
			
			//ExtractFromC_d<<<grid, threads>>>(fB_d,pitch_elements,SF_fc);
			ExtractFromC_d<<<grid, threads>>>(fB_d,pitch_elements,omega,omega2);
		}
		if(t >= STARTF)
		mrt_d_single_force<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,FX_d,FY_d,FZ_d,t+1,uAv_d,vAv_d,ufluc_d,vfluc_d);
		else
		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);

		if(REFINEMENT == "YES"){
			if(LRFACTOR == 0.5f)
			{
			if(t >= STARTF)
			LR_d_BACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			else
			LR_d_BACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			LR_d_BADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			}
			else if(LRFACTOR == 0.25f)
			{
			if(t >= STARTF)
			LR_d_BACD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			else
			LR_d_BACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			LR_d_BADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			LR_d_ABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			LR_d_BADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,t+1,uAvLR_d,vAvLR_d,uflucLR_d,vflucLR_d);
			}
	
			//ExtractFromC_d<<<grid, threads>>>(fA_d,pitch_elements,SF_fc);
			ExtractFromC_d<<<grid, threads>>>(fA_d,pitch_elements,omega,omega2);
		}


		}
//        else if(METHOD == "CACHE"){
//		    mrt_d_cache<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//		    mrt_d_cache<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//        }
//
//		else if(METHOD == "HYB"){
//			if(t >= STARTF && REFINEMENT == "NO")
//			mrt_d_hybAB_force<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,FX_d,FY_d,FZ_d,t);
//			else
//			mrt_d_hybAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//	
//			if(REFINEMENT == "YES"){
//			if(LRFACTOR == 0.5f)
//			{
//				if(t >= STARTF)
//				LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t);
//				else
//				LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
//	
//				LR_d_hybABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
//			}
//			else if(LRFACTOR == 0.25f)
//			{
//				if(t >= STARTF)
//				LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t);
//				else
//				LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
//	
//				LR_d_hybABDC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);
//
//				LR_d_hybABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3);
//				LR_d_hybABDC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
//			}
//			
//			ExtractFromC_d<<<grid, threads>>>(fB_d,pitch_elements,SF_fc);
//			}
//	
//			if(t >= STARTF && REFINEMENT == "NO")
//			mrt_d_hybBA_force<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,FX_d,FY_d,FZ_d,t+1);
//			else
//			mrt_d_hybBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//	
//			if(REFINEMENT == "YES"){
//			if(LRFACTOR == 0.5f)
//			{
//				if(t >= STARTF)
//				LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t+1);
//				else
//				LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
//	
//				LR_d_hybBADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
//			}
//			else if(LRFACTOR == 0.25f)
//			{
//				if(t >= STARTF)
//				LR_d_hybABCD_force<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,FX_d,FY_d,FZ_d,t+1);
//				else
//				LR_d_hybABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
//	
//				LR_d_hybBADC2<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf,2);
//
//				LR_d_hybABCD2<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2,3);
//				LR_d_hybBADC_Interp<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
//			}
//			
//			ExtractFromC_d<<<grid, threads>>>(fA_d,pitch_elements,SF_fc);
//			}
//		}

//		else if(METHOD == "TEXT"){
//		mrt_d_textAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//		mrt_d_textBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//		}
//
//		else if(METHOD == "SHARED"){
//		mrt_d_shared<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//		mrt_d_shared<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//		}

//		simple_copy<<<grid, threads>>>(fA_d,fB_d,pitch_elements);
//		simple_copy<<<grid, threads>>>(fB_d,fA_d,pitch_elements);
//
//		simple_text<<<grid, threads>>>(fA_d,fB_d,pitch_elements);
//		simple_text<<<grid, threads>>>(fB_d,fA_d,pitch_elements);



		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
	}
	hipDeviceSynchronize();

	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	int Nodes;
	if(REFINEMENT == "YES"){
		Nodes = (XDIM*YDIM*ZDIM+XLRDIM*YLRDIM*ZLRDIM*LRLEVEL);
	}
	else{
		Nodes = XDIM*YDIM*ZDIM;
	}
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)";
	if(REFINEMENT == "YES"){
	int effNodes = (XDIM*YDIM*ZDIM+XLRDIM*YLRDIM*ZLRDIM*LRLEVEL
					-(XLRDIM/LRLEVEL)*(YLRDIM/LRLEVEL)*(YLRDIM/LRLEVEL));
	cout<<" (eff: "<<double(effNodes*double(TMAX/1000000.f))/restime<<"MLUPS)";
	}
	cout<<endl;
	cout<<XDIM<<","<<YDIM<<","<<ZDIM<<","<<TMAX<<","<<restime<<endl;
	

	if(true){
	hipUnbindTexture(texRef_f0A);
	hipUnbindTexture(texRef_f1A);
	hipUnbindTexture(texRef_f2A);
	hipUnbindTexture(texRef_f3A);
	hipUnbindTexture(texRef_f4A);
	hipUnbindTexture(texRef_f5A);
	hipUnbindTexture(texRef_f6A);
	hipUnbindTexture(texRef_f7A);
	hipUnbindTexture(texRef_f8A);
	hipUnbindTexture(texRef_f9A);
	hipUnbindTexture(texRef_f10A);
	hipUnbindTexture(texRef_f11A);
	hipUnbindTexture(texRef_f12A);
	hipUnbindTexture(texRef_f13A);
	hipUnbindTexture(texRef_f14A);
	hipUnbindTexture(texRef_f15A);
	hipUnbindTexture(texRef_f16A);
	hipUnbindTexture(texRef_f17A);
	hipUnbindTexture(texRef_f18A);
	hipUnbindTexture(texRef_f0B);
	hipUnbindTexture(texRef_f1B);
	hipUnbindTexture(texRef_f2B);
	hipUnbindTexture(texRef_f3B);
	hipUnbindTexture(texRef_f4B);
	hipUnbindTexture(texRef_f5B);
	hipUnbindTexture(texRef_f6B);
	hipUnbindTexture(texRef_f7B);
	hipUnbindTexture(texRef_f8B);
	hipUnbindTexture(texRef_f9B);
	hipUnbindTexture(texRef_f10B);
	hipUnbindTexture(texRef_f11B);
	hipUnbindTexture(texRef_f12B);
	hipUnbindTexture(texRef_f13B);
	hipUnbindTexture(texRef_f14B);
	hipUnbindTexture(texRef_f15B);
	hipUnbindTexture(texRef_f16B);
	hipUnbindTexture(texRef_f17B);
	hipUnbindTexture(texRef_f18B);

	hipUnbindTexture(texRef_f0C);
	hipUnbindTexture(texRef_f1C);
	hipUnbindTexture(texRef_f2C);
	hipUnbindTexture(texRef_f3C);
	hipUnbindTexture(texRef_f4C);
	hipUnbindTexture(texRef_f5C);
	hipUnbindTexture(texRef_f6C);
	hipUnbindTexture(texRef_f7C);
	hipUnbindTexture(texRef_f8C);
	hipUnbindTexture(texRef_f9C);
	hipUnbindTexture(texRef_f10C);
	hipUnbindTexture(texRef_f11C);
	hipUnbindTexture(texRef_f12C);
	hipUnbindTexture(texRef_f13C);
	hipUnbindTexture(texRef_f14C);
	hipUnbindTexture(texRef_f15C);
	hipUnbindTexture(texRef_f16C);
	hipUnbindTexture(texRef_f17C);
	hipUnbindTexture(texRef_f18C);
	hipUnbindTexture(texRef_f0D);
	hipUnbindTexture(texRef_f1D);
	hipUnbindTexture(texRef_f2D);
	hipUnbindTexture(texRef_f3D);
	hipUnbindTexture(texRef_f4D);
	hipUnbindTexture(texRef_f5D);
	hipUnbindTexture(texRef_f6D);
	hipUnbindTexture(texRef_f7D);
	hipUnbindTexture(texRef_f8D);
	hipUnbindTexture(texRef_f9D);
	hipUnbindTexture(texRef_f10D);
	hipUnbindTexture(texRef_f11D);
	hipUnbindTexture(texRef_f12D);
	hipUnbindTexture(texRef_f13D);
	hipUnbindTexture(texRef_f14D);
	hipUnbindTexture(texRef_f15D);
	hipUnbindTexture(texRef_f16D);
	hipUnbindTexture(texRef_f17D);
	hipUnbindTexture(texRef_f18D);
	}

	hipMemcpy2D(fA_h,XDIM*sizeof(float),fA_d,pitch,XDIM*sizeof(float),YDIM*ZDIM*19,hipMemcpyDeviceToHost);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(fC_h,XLRDIM*sizeof(float),fC_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM*19,hipMemcpyDeviceToHost);
	}
    if(VELAV == "YES"){
	hipMemcpy2D(uAv_h,XDIM*sizeof(float),uAv_d,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(vAv_h,XDIM*sizeof(float),vAv_d,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(wAv_h,XDIM*sizeof(float),wAv_d,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(ufluc_h,XDIM*sizeof(float),ufluc_d,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(vfluc_h,XDIM*sizeof(float),vfluc_d,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(wfluc_h,XDIM*sizeof(float),wfluc_d,pitch,XDIM*sizeof(float),YDIM*ZDIM,hipMemcpyDeviceToHost);
	if(REFINEMENT == "YES"){
	hipMemcpy2D(uAvLR_h,XLRDIM*sizeof(float),uAvLR_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(vAvLR_h,XLRDIM*sizeof(float),vAvLR_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(wAvLR_h,XLRDIM*sizeof(float),wAvLR_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(uflucLR_h,XLRDIM*sizeof(float),uflucLR_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(vflucLR_h,XLRDIM*sizeof(float),vflucLR_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM,hipMemcpyDeviceToHost);
	hipMemcpy2D(wflucLR_h,XLRDIM*sizeof(float),wflucLR_d,pitch2,XLRDIM*sizeof(float),YLRDIM*ZLRDIM,hipMemcpyDeviceToHost);
    }
    }
	hipMemcpy(FX_h, FX_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FY_h, FY_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FZ_h, FZ_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);


	//hipMemcpy(image_h, image_d, memsize_int, hipMemcpyDeviceToHost);

	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\",\"uAv\",\"vAv\",\"ufluc\",\"vfluc\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<", K="<<ZDIM<<"\n";
	
	int row = 0;
	int col = 0;
	int dep = 0;
	i = 0;
	float rho, u, v, w;//, usqr;
	//int j;
    int check = 0;
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	for(dep = 0; dep<ZDIM; dep++){
	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = dep*XDIM*YDIM+row*XDIM+col;
            f0 = fA_h[i+XDIM*YDIM*ZDIM*0 ];
            f1 = fA_h[i+XDIM*YDIM*ZDIM*1 ];
            f2 = fA_h[i+XDIM*YDIM*ZDIM*2 ];
            f3 = fA_h[i+XDIM*YDIM*ZDIM*3 ];
            f4 = fA_h[i+XDIM*YDIM*ZDIM*4 ];
            f5 = fA_h[i+XDIM*YDIM*ZDIM*5 ];
            f6 = fA_h[i+XDIM*YDIM*ZDIM*6 ];
            f7 = fA_h[i+XDIM*YDIM*ZDIM*7 ];
            f8 = fA_h[i+XDIM*YDIM*ZDIM*8 ];
            f9 = fA_h[i+XDIM*YDIM*ZDIM*9 ];
            f10= fA_h[i+XDIM*YDIM*ZDIM*10];
            f11= fA_h[i+XDIM*YDIM*ZDIM*11];
            f12= fA_h[i+XDIM*YDIM*ZDIM*12];
            f13= fA_h[i+XDIM*YDIM*ZDIM*13];
            f14= fA_h[i+XDIM*YDIM*ZDIM*14];
            f15= fA_h[i+XDIM*YDIM*ZDIM*15];
            f16= fA_h[i+XDIM*YDIM*ZDIM*16];
            f17= fA_h[i+XDIM*YDIM*ZDIM*17];
            f18= fA_h[i+XDIM*YDIM*ZDIM*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));

			output<<col<<", "<<row<<", "<<dep<<", "<<u<<","<<v<<","<<w<<","<<Smag<<","
                  <<uAv_h[i]<<","<<vAv_h[i]<<", "<<ufluc_h[i]<<","<<vfluc_h[i]<<endl;
            if(rho>0.f && rho<2.f){
            }
            else{
                check = 1;
            }
		}
	}
	}
    if(check == 1) cout<<"error!"<<endl;

	if(REFINEMENT == "YES"){
	output<<endl;//<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\",\"uAv\"\n";
	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\",\"uAv\",\"vAv\",\"ufluc\",\"vfluc\"\n";
	output<<"ZONE F=POINT, I="<<XLRDIM-0<<", J="<<YLRDIM-0<<", K="<<ZLRDIM-0<<"\n";
    	
	for(dep = 0; dep<ZLRDIM-0; dep++){
	for(row = 0; row<YLRDIM-0; row++){
		for(col = 0; col<XLRDIM-0; col++){
			i = dep*XLRDIM*YLRDIM+row*XLRDIM+col;

            f0 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*0 ];
            f1 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*1 ];
            f2 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*2 ];
            f3 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*3 ];
            f4 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*4 ];
            f5 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*5 ];
            f6 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*6 ];
            f7 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*7 ];
            f8 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*8 ];
            f9 = fC_h[i+XLRDIM*YLRDIM*ZLRDIM*9 ];
            f10= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*10];
            f11= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*11];
            f12= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*12];
            f13= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*13];
            f14= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*14];
            f15= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*15];
            f16= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*16];
            f17= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*17];
            f18= fC_h[i+XLRDIM*YLRDIM*ZLRDIM*18];

        	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
            float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;

    		float PI11 = LRLEVEL*-0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    		float PI12 = LRLEVEL*-1.5f*omega*m13;
    		float PI23 = LRLEVEL*-1.5f*omega*m14;
    		float PI13 = LRLEVEL*-1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));

			output<<LRX0+col*LRFACTOR<<", "<<LRY0+row*LRFACTOR<<", "
                  <<LRZ0+dep*LRFACTOR<<", "<<u<<","<<v<<","<<w<<","<<Smag<<","
                  <<uAvLR_h[i]<<","<<vAvLR_h[i]<<", "<<uflucLR_h[i]<<","<<vflucLR_h[i]<<endl;
			//output<<LRX0+col*LRFACTOR<<", "<<LRY0+row*LRFACTOR<<", "<<LRZ0+dep*LRFACTOR<<", "<<u<<","<<v<<","<<w<<","<<rho<<endl;
        


		}
	}
	}
	}

	output.close();

	//for(int t = STARTF-1; t<TMAX; t++){
	for(int t = 0; t<TMAX; t++){
		output2<<t<<", "<<FX_h[t]/(0.5f*UMAX*UMAX*2.f*OBSTR1*ZDIM)<<", "
						<<FY_h[t]/(0.5f*UMAX*UMAX*2.f*OBSTR1*ZDIM)<<", "
						<<FZ_h[t]/(0.5f*UMAX*UMAX*2.f*OBSTR1*ZDIM)<<endl;
//		output2<<t<<", "<<FX_h[t]/(0.5f*UMAX*UMAX*OBSTR1*OBSTR1*3.14158f)<<", "
//						<<FY_h[t]/(0.5f*UMAX*UMAX*OBSTR1*OBSTR1*3.14158f)<<", "
//						<<FZ_h[t]/(0.5f*UMAX*UMAX*OBSTR1*OBSTR1*3.14158f)<<endl;
	}
	output2.close();



	//hipFree(image_d);


	hipFree(fA_d);
	hipFree(fB_d);
	hipFree(fC_d);
	hipFree(fD_d);
	return(0);

}
