#include "hip/hip_runtime.h"

    #include <stdlib.h>
    #include <stdio.h>
    #include <string.h>
    #include <math.h>
//    #include <conio.h>
    #include <cutil_inline.h>

    //-----------------------------------------------------------
    // Device code
    __global__ void SumRow(hipPitchedPtr devPitchedPtr, hipExtent extent, float* devPtr_out,int pitch_out)
    {
    char* devPtr   = (char*)devPitchedPtr.ptr;
    int pitch      = devPitchedPtr.pitch;
    int slicePitch = pitch*extent.height;

    //blockDim.x  : depth
    //threadIdx.x : height
    char* slice = devPtr + blockIdx.x * slicePitch;
    float* row  = (float*)(slice + threadIdx.x * pitch);
    float* sum  = (float*)((char*)devPtr_out + blockIdx.x*pitch_out);

    sum[threadIdx.x] = 0.f;
    for(int x = 0; x<extent.width;x++){
    sum[threadIdx.x] += row[x];
    }
    }

    //-----------------------------------------------------------
    // host code

    int main_impl();

    int main( int argc, char** argv)  {
     
    main_impl();
    }

    int main_impl(){

    //print_data();
//    getch();
    size_t s[3] = {256,256,209};
    size_t N    = s[2]*s[1]*s[0];
    size_t size = N * sizeof(float);
    size_t N_out    = s[2]*s[1];
    size_t size_out = N_out * sizeof(float);
    printf("size[%f]GB\n",size/1024.f/1204.f/1024.f);


    // memory allocation and init. in the host memory.
    float* h_A;
    h_A = (float*)malloc(size);
    float* h_B;
    h_B = (float*)malloc(size);
    float* h_C;
    h_C = (float*)malloc(size_out);
    float* h_D;
    h_D = (float*)malloc(size_out);

    for(int i=0; i<N;i++){
    h_A[i] = i;
    h_B[i] = N-i;
    }

    // memory allocation for device memory
    hipPitchedPtr devPitchedPtr;
    hipExtent     extent = make_hipExtent(s[0],s[1],s[2]);
    hipMalloc3D(&devPitchedPtr,extent);

    float* devPtr; //2D result. (y,z)
    size_t pitch_out ;    // This variable will be set by hipMallocPitch;
    hipMallocPitch( (void**)&devPtr,&pitch_out, (size_t)(s[1]*sizeof(float)), s[2]);


    unsigned int timer = 0;
        cutilCheckError( cutCreateTimer( &timer));
        cutilCheckError( cutStartTimer( timer));


    // copy from host memory data to device memory data
    hipMemcpy3DParms p = {0};
    p.srcPtr.ptr   = h_A;
    p.srcPtr.pitch = s[0] * sizeof(float);
    p.srcPtr.xsize = s[0];
    p.srcPtr.ysize = s[1];
    p.dstPtr.ptr   = devPitchedPtr.ptr;
    p.dstPtr.pitch = devPitchedPtr.pitch;
    p.dstPtr.xsize = s[0];
    p.dstPtr.ysize = s[1];
    p.extent.width  = s[0]*sizeof(float);
    p.extent.height = s[1];
    p.extent.depth  = s[2];
    p.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&p);

    int threadsPerBlock = s[1];
    int blocksPerGrid   = s[2];
    SumRow<<<blocksPerGrid,threadsPerBlock>>>(devPitchedPtr,extent,devPtr,pitch_out);
    hipGetErrorString(hipGetLastError());
    hipMemcpy2D(h_C,s[1]*sizeof(float),devPtr,pitch_out,s[1]*sizeof(float),s[2],hipMemcpyDeviceToHost);
    // free device memory.
    hipFree(devPitchedPtr.ptr);

    // finish Timer
    cutilCheckError( cutStopTimer( timer));
    printf( "Processing time1: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));


    // for reference.
    timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));
    int i=0;
    int j=0;
    for(int z=0; z< s[2];z++){
    for(int y=0; y< s[1];y++){
    h_D[j] =0.f;
    for(int x=0; x< s[0];x++){
    h_D[j] += h_A[i];
    i++;
    }
    j++;
    }
    }
    cutilCheckError( cutStopTimer( timer));
    printf( "Processing time2: %f (ms)\n", cutGetTimerValue( timer));
    cutilCheckError( cutDeleteTimer( timer));

    // comparison
    for(i=0; i< N_out;i++){

    // printf("test N[%d][%6.5f/%6.5f]\n",i,h_C[i],h_D[i]);
    if( 0.0f < (h_C[i] - h_D[i]) ){
    printf("error N[%d][%6.5f/%6.5f]\n",i,h_C[i],h_D[i]);
    }
    }
    return 0;
    };

