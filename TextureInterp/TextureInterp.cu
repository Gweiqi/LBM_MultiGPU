#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define BLOCKSIZEX 16
#define BLOCKSIZEY 1
#define BLOCKSIZELRX 16
#define BLOCKSIZELRY 1
#define XDIM 16
#define YDIM 16
#define TMAX 0
#define OBSTR 124.63f
#define OBSTX 498.53f//30.243f
#define OBSTY 499.03f//30.743f

#define STARTF 10000

#define LRFACTOR 0.5f
#define LRX0 0.75f	//minimum x coord of LR
#define XLRDIM 16	//number of nodes in x
#define LRY0 0.75f
#define YLRDIM 16

#define RE 100.f
#define UMAX 0.01f
#define METHOD "LR" //LR,SINGLE,HYB,TEXT,SHARED,SINGLEF,TEST
#define MODEL "MRT" //BGK,MRT,STREAM
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>

inline __device__ int ImageFcn(float x, float y){
	//if(y < 0.1f || (XDIM-x) < 1.1f || (YDIM-y) < 1.1f)
	if(y < 0.1f || (YDIM-y) < 1.1f)
		return 1;
	else if(x < 0.1f)
		return 3;//west
	else if((XDIM-x) < 1.1f)
		return 2;//east
	else if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
		return 1;
	else
		return 0;
}

inline __device__ int ImageFcn(int x, int y){
	//if(y == 0 || x == XDIM-1 || y == YDIM-1)
	if(y < 0.1f || (YDIM-y) < 1.1f)
		return 1;
	else if(x < 0.1f)
		return 3;//west
	else if((XDIM-x) < 1.1f)
		return 2;//east
	else if((x-OBSTX)*(x-OBSTX)+(y-OBSTY)*(y-OBSTY)<OBSTR*OBSTR)
		return 10;
	else
		return 0;
}

inline __device__ float PoisProf (float x){
	float radius = (YDIM-1-1)*0.5f;
//	double a = 2.0*radius;
	float result = -1.0f*(((1.0f-(x-0.5f)/radius))*((1.0f-(x-0.5f)/radius))-1.0f);
	return (result);
}



texture<float,2,hipReadModeElementType> texRef_f0A;
texture<float,2,hipReadModeElementType> texRef_f1A;
texture<float,2,hipReadModeElementType> texRef_f2A;
texture<float,2,hipReadModeElementType> texRef_f3A;
texture<float,2,hipReadModeElementType> texRef_f4A;
texture<float,2,hipReadModeElementType> texRef_f5A;
texture<float,2,hipReadModeElementType> texRef_f6A;
texture<float,2,hipReadModeElementType> texRef_f7A;
texture<float,2,hipReadModeElementType> texRef_f8A;

texture<float,2,hipReadModeElementType> texRef_f0B;
texture<float,2,hipReadModeElementType> texRef_f1B;
texture<float,2,hipReadModeElementType> texRef_f2B;
texture<float,2,hipReadModeElementType> texRef_f3B;
texture<float,2,hipReadModeElementType> texRef_f4B;
texture<float,2,hipReadModeElementType> texRef_f5B;
texture<float,2,hipReadModeElementType> texRef_f6B;
texture<float,2,hipReadModeElementType> texRef_f7B;
texture<float,2,hipReadModeElementType> texRef_f8B;

texture<float,2,hipReadModeElementType> texRef_f0C;
texture<float,2,hipReadModeElementType> texRef_f1C;
texture<float,2,hipReadModeElementType> texRef_f2C;
texture<float,2,hipReadModeElementType> texRef_f3C;
texture<float,2,hipReadModeElementType> texRef_f4C;
texture<float,2,hipReadModeElementType> texRef_f5C;
texture<float,2,hipReadModeElementType> texRef_f6C;
texture<float,2,hipReadModeElementType> texRef_f7C;
texture<float,2,hipReadModeElementType> texRef_f8C;

texture<float,2,hipReadModeElementType> texRef_f0D;
texture<float,2,hipReadModeElementType> texRef_f1D;
texture<float,2,hipReadModeElementType> texRef_f2D;
texture<float,2,hipReadModeElementType> texRef_f3D;
texture<float,2,hipReadModeElementType> texRef_f4D;
texture<float,2,hipReadModeElementType> texRef_f5D;
texture<float,2,hipReadModeElementType> texRef_f6D;
texture<float,2,hipReadModeElementType> texRef_f7D;
texture<float,2,hipReadModeElementType> texRef_f8D;



int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 ,  float omega)
{
	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	f0 = f0 -omega*(f0 -0.4444444444f*(rho-1.5f*usqr));
	f1 = f1 -omega*(f1 -0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 = f2 -omega*(f2 -0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 = f3 -omega*(f3 -0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 = f4 -omega*(f4 -0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 = f5 -omega*(f5 -0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	f6 = f6 -omega*(f6 -0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 = f7 -omega*(f7 -0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 = f8 -omega*(f8 -0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
}

__device__ void mrt_collide(float &f0, float &f1, float &f2,
						   float &f3, float &f4, float &f5,
						   float &f6, float &f7, float &f8, float omega)
{
	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float m1,m2,m4,m6,m7,m8;
	m1 =-4.f*f0 -    f1 -    f2 -    f3 -    f4+ 2.f*f5+ 2.f*f6+ 2.f*f7+ 2.f*f8-(-2.0f*rho+3.0f*(u*u+v*v));
	m2 = 4.f*f0 -2.f*f1 -2.f*f2 -2.f*f3 -2.f*f4+     f5+     f6+     f7+     f8-(rho-3.0f*(u*u+v*v)); //ep
	m4 =        -2.f*f1        + 2.f*f3        +     f5 -    f6 -    f7+     f8-(-u);//qx_eq
	m6 =                -2.f*f2        + 2.f*f4+     f5+     f6 -    f7 -    f8-(-v);//qy_eq
	m7 =             f1 -    f2+     f3 -    f4                                -(u*u-v*v);//pxx_eq
	m8 =                                             f5 -    f6+     f7 -    f8-(u*v);//pxy_eq

	f0=f0-(-m1+m2)*0.11111111f;//(-4.f*(m1)/36.0f+4.f *(m2)/36.0f);
	//f1=f1-(-m1-2.0f*(m2+m4)+m7*omega*9.0f)*0.027777777f;
	f1=f1-(-m1-2.f*m2-6.f*m4+m7*omega*9.0f)*0.027777777f;
	f2=f2-(-m1-2.f*m2-6.f*m6-m7*omega*9.0f)*0.027777777f;
	f3=f3-(-m1-2.f*m2+6.f*m4+m7*omega*9.0f)*0.027777777f;
	f4=f4-(-m1-2.f*m2+6.f*m6-m7*omega*9.0f)*0.027777777f;
	f5=f5-(2.f*m1+m2+3.f*m4+3.f*m6+m8*omega*9.0f)*0.027777777f;
	f6=f6-(2.f*m1+m2-3.f*m4+3.f*m6-m8*omega*9.0f)*0.027777777f;
	f7=f7-(2.f*m1+m2-3.f*m4-3.f*m6+m8*omega*9.0f)*0.027777777f;
	f8=f8-(2.f*m1+m2+3.f*m4-3.f*m6-m8*omega*9.0f)*0.027777777f;
}

inline __device__ int f_mem(int f_num, int x, int y, size_t pitch)
{

	return (x+y*pitch)+f_num*pitch*YDIM;
}

inline __device__ int f_memLR(int f_num, int x, int y, size_t pitch)
{

	return (x+y*pitch)+f_num*pitch*YLRDIM;
}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}

__global__ void simple_copy(float* fA, float* fB,
							int *image, float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)

	fB[j] = fA[j];//+0.01f;
}


__global__ void ExtractFromC_d(float* fout,
							size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
//	if(x < LRX0+1 || x > LRX0+XLRDIM*LRFACTOR-1 || y < LRY0+1 || y > LRY0+YLRDIM*LRFACTOR-1)
////	if(x < LRX0+2 || x > LRX0+XLRDIM-3 || y < LRY0+2 || y > LRY0+YLRDIM-3)
//	{
//	//do nothing
//	}
//	else
	//if(x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1)
	if( (x > LRX0+1 && x < LRX0+XLRDIM*LRFACTOR-1 && y > LRY0+1 && y < LRY0+YLRDIM*LRFACTOR-1) && 
	(x == int(LRX0+2) || x == int(LRX0+XLRDIM*LRFACTOR-1) || y == int(LRY0+2) || y == int(LRY0+YLRDIM*LRFACTOR-1)) )
	{
	float xcoord = 2.0f*(x-LRX0)+0.5f;
	float ycoord = 2.0f*(y-LRY0)+0.5f;
	f0 = tex2D(texRef_f0C ,xcoord,ycoord);
	f2 = tex2D(texRef_f2C ,xcoord,ycoord);
	f4 = tex2D(texRef_f4C ,xcoord,ycoord);
	f1 = tex2D(texRef_f1C ,xcoord,ycoord);
	f3 = tex2D(texRef_f3C ,xcoord,ycoord);
	f5 = tex2D(texRef_f5C ,xcoord,ycoord);
	f6 = tex2D(texRef_f6C ,xcoord,ycoord);
	f7 = tex2D(texRef_f7C ,xcoord,ycoord);
	f8 = tex2D(texRef_f8C ,xcoord,ycoord);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	f0 =SF*f0 +(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	f1 =SF*f1 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	f2 =SF*f2 +(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4 +(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	f6 =SF*f6 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7 +(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8 +(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
	
	fout[f_mem(0 ,x,y,pitch)] = f0 ;
	fout[f_mem(1 ,x,y,pitch)] = f1 ;
	fout[f_mem(2 ,x,y,pitch)] = f2 ;
	fout[f_mem(3 ,x,y,pitch)] = f3 ;
	fout[f_mem(4 ,x,y,pitch)] = f4 ;
	fout[f_mem(5 ,x,y,pitch)] = f5 ;
	fout[f_mem(6 ,x,y,pitch)] = f6 ;
	fout[f_mem(7 ,x,y,pitch)] = f7 ;
	fout[f_mem(8 ,x,y,pitch)] = f8 ;
	}
}

__global__ void LR_d_ABCD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);
	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_ABDC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int im = ImageFcn(xcoord,ycoord);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//interpolate for next time step. from B
	f0 = tex2D(texRef_f0B,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f2 = tex2D(texRef_f2B,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4B,xcoord+0.5f,ycoord+0.5f);
	f1 = tex2D(texRef_f1B,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3B,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5B,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6B,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7B,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8B,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
	fout[f_memLR(0,x,y,pitch)] = f0 ;
	fout[f_memLR(1,x,y,pitch)] = f1 ;
	fout[f_memLR(2,x,y,pitch)] = f2 ;
	fout[f_memLR(3,x,y,pitch)] = f3 ;
	fout[f_memLR(4,x,y,pitch)] = f4 ;
	fout[f_memLR(5,x,y,pitch)] = f5 ;
	fout[f_memLR(6,x,y,pitch)] = f6 ;
	fout[f_memLR(7,x,y,pitch)] = f7 ;
	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fin[f_memLR(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fin[f_memLR(2 ,x             ,y-1           ,pitch)];
	f5 = fin[f_memLR(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fin[f_memLR(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fin[f_memLR(4 ,x             ,y+1           ,pitch)];
	f7 = fin[f_memLR(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fin[f_memLR(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}
__global__ void LR_d_BACD(float* fin, float* fout,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);
	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2)
	{
	//dont do anything
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,x-1,y  ,pitch)];
	f3 = fin[f_memLR(3 ,x+1,y  ,pitch)];
	f2 = fin[f_memLR(2 ,x  ,y-1,pitch)];
	f5 = fin[f_memLR(5 ,x-1,y-1,pitch)];
	f6 = fin[f_memLR(6 ,x+1,y-1,pitch)];
	f4 = fin[f_memLR(4 ,x  ,y+1,pitch)];
	f7 = fin[f_memLR(7 ,x+1,y+1,pitch)];
	f8 = fin[f_memLR(8 ,x-1,y+1,pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

__global__ void LR_d_BADC(float* fin, float* fout,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);

	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3)
	{
	//interpolate for next time step. from A
	f0 = tex2D(texRef_f0A,xcoord+0.5f,ycoord+0.5f);//YDIM and not YLRDIM
	f2 = tex2D(texRef_f2A,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4A,xcoord+0.5f,ycoord+0.5f);
	f1 = tex2D(texRef_f1A,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3A,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5A,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6A,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7A,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8A,xcoord+0.5f,ycoord+0.5f);

	float rho,u,v;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8;
	u = f1-f3+f5-f6-f7+f8;
	v = f2-f4+f5+f6-f7-f8;
	float usqr = u*u+v*v;
	//scale
	f0 =SF*f0+(1.0f-SF)*(0.4444444444f*(rho-1.5f*usqr));
	f1 =SF*f1+(1.0f-SF)*(0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                 
	f2 =SF*f2+(1.0f-SF)*(0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 =SF*f3+(1.0f-SF)*(0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 =SF*f4+(1.0f-SF)*(0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 =SF*f5+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr));
	f6 =SF*f6+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 =SF*f7+(1.0f-SF)*(0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 =SF*f8+(1.0f-SF)*(0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr));
	fout[f_memLR(0,x,y,pitch)] = f0 ;
	fout[f_memLR(1,x,y,pitch)] = f1 ;
	fout[f_memLR(2,x,y,pitch)] = f2 ;
	fout[f_memLR(3,x,y,pitch)] = f3 ;
	fout[f_memLR(4,x,y,pitch)] = f4 ;
	fout[f_memLR(5,x,y,pitch)] = f5 ;
	fout[f_memLR(6,x,y,pitch)] = f6 ;
	fout[f_memLR(7,x,y,pitch)] = f7 ;
	fout[f_memLR(8,x,y,pitch)] = f8 ;
	}
	else{
	f0 = fin[j];
	f1 = fin[f_memLR(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fin[f_memLR(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fin[f_memLR(2 ,x             ,y-1           ,pitch)];
	f5 = fin[f_memLR(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fin[f_memLR(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fin[f_memLR(4 ,x             ,y+1           ,pitch)];
	f7 = fin[f_memLR(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fin[f_memLR(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fout[f_memLR(1 ,x,y,pitch)] = f3 ;
		fout[f_memLR(2 ,x,y,pitch)] = f4 ;
		fout[f_memLR(3 ,x,y,pitch)] = f1 ;
		fout[f_memLR(4 ,x,y,pitch)] = f2 ;
		fout[f_memLR(5 ,x,y,pitch)] = f7 ;
		fout[f_memLR(6 ,x,y,pitch)] = f8 ;
		fout[f_memLR(7 ,x,y,pitch)] = f5 ;
		fout[f_memLR(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fout[f_memLR(0 ,x,y,pitch)] = f0 ;
		fout[f_memLR(1 ,x,y,pitch)] = f1 ;
		fout[f_memLR(2 ,x,y,pitch)] = f2 ;
		fout[f_memLR(3 ,x,y,pitch)] = f3 ;
		fout[f_memLR(4 ,x,y,pitch)] = f4 ;
		fout[f_memLR(5 ,x,y,pitch)] = f5 ;
		fout[f_memLR(6 ,x,y,pitch)] = f6 ;
		fout[f_memLR(7 ,x,y,pitch)] = f7 ;
		fout[f_memLR(8 ,x,y,pitch)] = f8 ;
	}
	}//end else (not at edge of LR)
}

//__global__ void LR_d_BACD(float* fin, float* fout,
//							float omega, size_t pitch)//pitch in elements
//{
//	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
//	int y = threadIdx.y+blockIdx.y*blockDim.y;
//	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	float xcoord = LRX0+x*LRFACTOR;
//	float ycoord = LRY0+y*LRFACTOR;
//	float zcoord = LRZ0+z*LRFACTOR;
//	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
//	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//	int im = ImageFcn(xcoord,ycoord,zcoord);
////	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
////	im = -1;//not valid for extraction
////	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
////	{
////	im = -2;//not valid for second TS
////	}
//
//	if(x < 1 || x > XLRDIM-2 || y < 1 || y > YLRDIM-2 || z < 1 || z > ZLRDIM-2)
//	{
//	//dont do anything
//	}
//	else{
//	f0 = fin[j];
//	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
//	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
//	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
//	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
//	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
//	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
//	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
//	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
//	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
//	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
//	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
//	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
//	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
//	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
//	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
//	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
//	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
//	f18= fin[f_memLR(18,x  ,dmin(y+1,YLRDIM),dmin(z+1,ZLRDIM),pitch)];
//	if(im == 1){//BB
//		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
//		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
//		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
//		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
//		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
//		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
//		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
//		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
//		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
//		fout[f_memLR(10,x,y,z,pitch)] = f17;
//		fout[f_memLR(11,x,y,z,pitch)] = f18;
//		fout[f_memLR(12,x,y,z,pitch)] = f15;
//		fout[f_memLR(13,x,y,z,pitch)] = f16;
//		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
//		fout[f_memLR(15,x,y,z,pitch)] = f12;
//		fout[f_memLR(16,x,y,z,pitch)] = f13;
//		fout[f_memLR(17,x,y,z,pitch)] = f10;
//		fout[f_memLR(18,x,y,z,pitch)] = f11;
//	}
//	else{
//		if(MODEL == "MRT")
//		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
//		else if(MODEL == "BGK")
//		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
//
//		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
//		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
//		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
//		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
//		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
//		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
//		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
//		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
//		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
//		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
//		fout[f_memLR(10,x,y,z,pitch)] = f10;
//		fout[f_memLR(11,x,y,z,pitch)] = f11;
//		fout[f_memLR(12,x,y,z,pitch)] = f12;
//		fout[f_memLR(13,x,y,z,pitch)] = f13;
//		fout[f_memLR(14,x,y,z,pitch)] = f14;
//		fout[f_memLR(15,x,y,z,pitch)] = f15;
//		fout[f_memLR(16,x,y,z,pitch)] = f16;
//		fout[f_memLR(17,x,y,z,pitch)] = f17;
//		fout[f_memLR(18,x,y,z,pitch)] = f18;
//	}
//	}//end else (not at edge of LR)
//}
//
//__global__ void LR_d_BADC(float* fin, float* fout,
//							float omega, size_t pitch, float SF)//pitch in elements
//{
//	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
//	int y = threadIdx.y+blockIdx.y*blockDim.y;
//	int z = threadIdx.z+blockIdx.z*blockDim.z;
//	float xcoord = LRX0+x*LRFACTOR;
//	float ycoord = LRY0+y*LRFACTOR;
//	float zcoord = LRZ0+z*LRFACTOR;
//	int zminus = int(zcoord);
//	int zplus = zminus+1;
//	int j = x+y*pitch+z*YLRDIM*pitch;//index on padded mem (pitch in elements)
//	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;
//	int im = ImageFcn(xcoord,ycoord,zcoord);
////	if(x < 2 || x > LRX0+XLRDIM-3 || y < 2 || y > LRY0+YLRDIM-3 || z < 2 || z > LRZ0+ZLRDIM-3)
////	im = -1;//not valid for extraction
////	if(x < 1 || x > LRX0+XLRDIM-2 || y < 1 || y > LRY0+YLRDIM-2 || z < 1 || z > LRZ0+ZLRDIM-2)
////	{
////	im = -2;//not valid for second TS
////	}
//
//	if(x < 2 || x > XLRDIM-3 || y < 2 || y > YLRDIM-3 || z < 2 || z > ZLRDIM-3)
//	{
//	//interpolate for next time step. from A
//	f0 = (zplus-zcoord)*tex2D(texRef_f0A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f0A ,xcoord,ycoord+YDIM*(zplus));//YDIM and not YLRDIM
//	f2 = (zplus-zcoord)*tex2D(texRef_f2A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f2A ,xcoord,ycoord+YDIM*(zplus));
//	f4 = (zplus-zcoord)*tex2D(texRef_f4A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f4A ,xcoord,ycoord+YDIM*(zplus));
//	f9 = (zplus-zcoord)*tex2D(texRef_f9A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f9A ,xcoord,ycoord+YDIM*(zplus));
//	f11= (zplus-zcoord)*tex2D(texRef_f11A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f11A,xcoord,ycoord+YDIM*(zplus));
//	f13= (zplus-zcoord)*tex2D(texRef_f13A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f13A,xcoord,ycoord+YDIM*(zplus));
//	f14= (zplus-zcoord)*tex2D(texRef_f14A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f14A,xcoord,ycoord+YDIM*(zplus));
//	f16= (zplus-zcoord)*tex2D(texRef_f16A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f16A,xcoord,ycoord+YDIM*(zplus));
//	f18= (zplus-zcoord)*tex2D(texRef_f18A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f18A,xcoord,ycoord+YDIM*(zplus));
//	f1 = (zplus-zcoord)*tex2D(texRef_f1A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f1A ,xcoord,ycoord+YDIM*(zplus));
//	f3 = (zplus-zcoord)*tex2D(texRef_f3A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f3A ,xcoord,ycoord+YDIM*(zplus));
//	f5 = (zplus-zcoord)*tex2D(texRef_f5A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f5A ,xcoord,ycoord+YDIM*(zplus));
//	f6 = (zplus-zcoord)*tex2D(texRef_f6A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f6A ,xcoord,ycoord+YDIM*(zplus));
//	f7 = (zplus-zcoord)*tex2D(texRef_f7A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f7A ,xcoord,ycoord+YDIM*(zplus));
//	f8 = (zplus-zcoord)*tex2D(texRef_f8A ,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f8A ,xcoord,ycoord+YDIM*(zplus));
//	f15= (zplus-zcoord)*tex2D(texRef_f15A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f15A,xcoord,ycoord+YDIM*(zplus));
//	f17= (zplus-zcoord)*tex2D(texRef_f17A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f17A,xcoord,ycoord+YDIM*(zplus));
//	f10= (zplus-zcoord)*tex2D(texRef_f10A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f10A,xcoord,ycoord+YDIM*(zplus));
//	f12= (zplus-zcoord)*tex2D(texRef_f12A,xcoord,ycoord+YDIM*(zminus))+(zcoord-zminus)*tex2D(texRef_f12A,xcoord,ycoord+YDIM*(zplus));
//
//	float rho,u,v,w;	
//	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
//	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
//	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
//	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
//	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
//	float usqr = fma(u,u,fma(v,v,w*w));
//	//scale
//	f0 =SF*f0 +(1.0f-SF)*(0.3333333333f*(rho-1.5f*usqr));
//	f1 =SF*f1 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f2 =SF*f2 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =SF*f3 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =SF*f4 +(1.0f-SF)*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =SF*f5 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
//	f6 =SF*f6 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =SF*f7 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =SF*f8 +(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
//	f9 =SF*f9 +(1.0f-SF)*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
//	f10=SF*f10+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
//	f11=SF*f11+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
//	f12=SF*f12+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
//	f13=SF*f13+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
//	f14=SF*f14+(1.0f-SF)*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
//	f15=SF*f15+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
//	f16=SF*f16+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
//	f17=SF*f17+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
//	f18=SF*f18+(1.0f-SF)*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));
//
//	fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
//	fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
//	fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
//	fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
//	fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
//	fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
//	fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
//	fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
//	fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
//	fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
//	fout[f_memLR(10,x,y,z,pitch)] = f10;
//	fout[f_memLR(11,x,y,z,pitch)] = f11;
//	fout[f_memLR(12,x,y,z,pitch)] = f12;
//	fout[f_memLR(13,x,y,z,pitch)] = f13;
//	fout[f_memLR(14,x,y,z,pitch)] = f14;
//	fout[f_memLR(15,x,y,z,pitch)] = f15;
//	fout[f_memLR(16,x,y,z,pitch)] = f16;
//	fout[f_memLR(17,x,y,z,pitch)] = f17;
//	fout[f_memLR(18,x,y,z,pitch)] = f18;
//	}
//	else{
//	f0 = fin[j];
//	f1 = fin[f_memLR(1 ,x-1,y  ,z  ,pitch)];
//	f3 = fin[f_memLR(3 ,x+1,y  ,z  ,pitch)];
//	f2 = fin[f_memLR(2 ,x  ,y-1,z  ,pitch)];
//	f5 = fin[f_memLR(5 ,x-1,y-1,z  ,pitch)];
//	f6 = fin[f_memLR(6 ,x+1,y-1,z  ,pitch)];
//	f4 = fin[f_memLR(4 ,x  ,y+1,z  ,pitch)];
//	f7 = fin[f_memLR(7 ,x+1,y+1,z  ,pitch)];
//	f8 = fin[f_memLR(8 ,x-1,y+1,z  ,pitch)];
//	f9 = fin[f_memLR(9 ,x  ,y  ,z-1,pitch)];
//	f10= fin[f_memLR(10,x-1,y  ,z-1,pitch)];
//	f11= fin[f_memLR(11,x  ,y-1,z-1,pitch)];
//	f12= fin[f_memLR(12,x+1,y  ,z-1,pitch)];
//	f13= fin[f_memLR(13,x  ,y+1,z-1,pitch)];
//	f14= fin[f_memLR(14,x  ,y  ,z+1,pitch)];
//	f15= fin[f_memLR(15,x-1,y  ,z+1,pitch)];
//	f16= fin[f_memLR(16,x  ,y-1,z+1,pitch)];
//	f17= fin[f_memLR(17,x+1,y  ,z+1,pitch)];
//	//if(z != ZLRDIM-1){
//	f18= fin[f_memLR(18,x  ,dmin(y+1,YLRDIM),dmin(z+1,ZLRDIM),pitch)];
//	//}
//	//else f18 = 0.1f;
//	if(im == 1){//BB
//		fout[f_memLR(1 ,x,y,z,pitch)] = f3 ;
//		fout[f_memLR(2 ,x,y,z,pitch)] = f4 ;
//		fout[f_memLR(3 ,x,y,z,pitch)] = f1 ;
//		fout[f_memLR(4 ,x,y,z,pitch)] = f2 ;
//		fout[f_memLR(5 ,x,y,z,pitch)] = f7 ;
//		fout[f_memLR(6 ,x,y,z,pitch)] = f8 ;
//		fout[f_memLR(7 ,x,y,z,pitch)] = f5 ;
//		fout[f_memLR(8 ,x,y,z,pitch)] = f6 ;
//		fout[f_memLR(9 ,x,y,z,pitch)] = f14;
//		fout[f_memLR(10,x,y,z,pitch)] = f17;
//		fout[f_memLR(11,x,y,z,pitch)] = f18;
//		fout[f_memLR(12,x,y,z,pitch)] = f15;
//		fout[f_memLR(13,x,y,z,pitch)] = f16;
//		fout[f_memLR(14,x,y,z,pitch)] = f9 ;
//		fout[f_memLR(15,x,y,z,pitch)] = f12;
//		fout[f_memLR(16,x,y,z,pitch)] = f13;
//		fout[f_memLR(17,x,y,z,pitch)] = f10;
//		fout[f_memLR(18,x,y,z,pitch)] = f11;
//	}
//	else{
//		if(MODEL == "MRT")
//		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
//		else if(MODEL == "BGK")
//		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
//
//		fout[f_memLR(0 ,x,y,z,pitch)] = f0 ;
//		fout[f_memLR(1 ,x,y,z,pitch)] = f1 ;
//		fout[f_memLR(2 ,x,y,z,pitch)] = f2 ;
//		fout[f_memLR(3 ,x,y,z,pitch)] = f3 ;
//		fout[f_memLR(4 ,x,y,z,pitch)] = f4 ;
//		fout[f_memLR(5 ,x,y,z,pitch)] = f5 ;
//		fout[f_memLR(6 ,x,y,z,pitch)] = f6 ;
//		fout[f_memLR(7 ,x,y,z,pitch)] = f7 ;
//		fout[f_memLR(8 ,x,y,z,pitch)] = f8 ;
//		fout[f_memLR(9 ,x,y,z,pitch)] = f9 ;
//		fout[f_memLR(10,x,y,z,pitch)] = f10;
//		fout[f_memLR(11,x,y,z,pitch)] = f11;
//		fout[f_memLR(12,x,y,z,pitch)] = f12;
//		fout[f_memLR(13,x,y,z,pitch)] = f13;
//		fout[f_memLR(14,x,y,z,pitch)] = f14;
//		fout[f_memLR(15,x,y,z,pitch)] = f15;
//		fout[f_memLR(16,x,y,z,pitch)] = f16;
//		fout[f_memLR(17,x,y,z,pitch)] = f17;
//		fout[f_memLR(18,x,y,z,pitch)] = f18;
//	}
//	}//end else (not at edge of LR)
//}

__global__ void mrt_d_single(float* fA, float* fB,
							float omega, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;

	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		//atomicAdd();
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(im == 2)//NeumannEast
		{
			if(y == 0){
				f2 = f4;
				f5 = f8;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f8 = f5;
			}
			float u,v,rho;
	        v = 0.0;
	        rho = 1.0;
			u = -rho+((f0+f2+f4)+2.0f*f1+2.0f*f5+2.0f*f8);

	        f3 = f1-u*0.66666667f;
	        f7 = f5+0.5f*(f2-f4)-0.5f*v-u*0.16666667f;
	        f6 = f8-0.5f*(f2-f4)+0.5f*v-u*0.16666667f;
		}
		else if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
			}
			float u,v;//,rho;
			u = UMAX*1.5f*PoisProf(float(y));
	        v = 0.0f;//0.0;
			//rho = u+(f0+f2+f4+2.0f*f3+2.0f*f6+2.0f*f7);
			//f1 = f3+4.0f*u/6.0f;
			f1 = f3+u*0.66666667f;
	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;			
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}

}

__global__ void mrt_d_single_force(float* fA, float* fB,
							float omega, size_t pitch, float *FX, float *FY, int t)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	int check = 0; //check if block contains force calculation nodes
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	__shared__ float sumX[BLOCKSIZEX], sumY[BLOCKSIZEX];

	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		if(im == 10){
		check = 1;
		sumX[threadIdx.x]=2.f*f1-2.f*f3+2.f*f5+2.f*f8-2.f*f6-2.f*f7;
		sumY[threadIdx.x]=2.f*f2-2.f*f4+2.f*f5-2.f*f8+2.f*f6-2.f*f7;
		}
		else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		}
		//atomicAdd();
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		sumX[threadIdx.x]=0.f;
		sumY[threadIdx.x]=0.f;
		if(im == 2)//NeumannEast
		{
			if(y == 0){
				f2 = f4;
				f5 = f8;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f8 = f5;
			}
			float u,v,rho;
	        v = 0.0;
	        rho = 1.0;
			u = -rho+((f0+f2+f4)+2.0f*f1+2.0f*f5+2.0f*f8);

	        f3 = f1-u*0.66666667f;
	        f7 = f5+0.5f*(f2-f4)-0.5f*v-u*0.16666667f;
	        f6 = f8-0.5f*(f2-f4)+0.5f*v-u*0.16666667f;
		}
		else if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
			}
			float u,v;//,rho;
			u = UMAX*1.5f*PoisProf(float(y));
	        v = 0.0f;//0.0;
			//rho = u+(f0+f2+f4+2.0f*f3+2.0f*f6+2.0f*f7);
			//f1 = f3+4.0f*u/6.0f;
			f1 = f3+u*0.66666667f;
	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;			
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}


	if(check == 1 && t>STARTF){
	syncthreads();
	//reduction for force
	int nTotalThreads = blockDim.x;
	while(nTotalThreads > 1){
		int halfPoint = (nTotalThreads >> 1);
		if(threadIdx.x < halfPoint){
			sumX[threadIdx.x] += sumX[threadIdx.x+halfPoint];
			sumY[threadIdx.x] += sumY[threadIdx.x+halfPoint];
		}
		syncthreads();
		nTotalThreads = halfPoint;
	}
	if(threadIdx.x == 0){
		atomicAdd(&FX[t],sumX[0]);
		atomicAdd(&FY[t],sumY[0]);
	}
	}

}

__global__ void mrt_d_LR(float* fA, float* fB,
							float omega, size_t pitch, float SF)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;




	f0 = fA[j];
	f1 = fA[f_mem(1 ,dmax(x-1)     ,y             ,pitch)];
	f3 = fA[f_mem(3 ,dmin(x+1,XDIM),y             ,pitch)];
	f2 = fA[f_mem(2 ,x             ,y-1           ,pitch)];
	f5 = fA[f_mem(5 ,dmax(x-1)     ,y-1           ,pitch)];
	f6 = fA[f_mem(6 ,dmin(x+1,XDIM),y-1           ,pitch)];
	f4 = fA[f_mem(4 ,x             ,y+1           ,pitch)];
	f7 = fA[f_mem(7 ,dmin(x+1,XDIM),y+1           ,pitch)];
	f8 = fA[f_mem(8 ,dmax(x-1)     ,dmin(y+1,YDIM),pitch)];
	if(im == 1 || im == 10){//BB
		fB[f_mem(1 ,x,y,pitch)] = f3 ;
		fB[f_mem(2 ,x,y,pitch)] = f4 ;
		fB[f_mem(3 ,x,y,pitch)] = f1 ;
		fB[f_mem(4 ,x,y,pitch)] = f2 ;
		fB[f_mem(5 ,x,y,pitch)] = f7 ;
		fB[f_mem(6 ,x,y,pitch)] = f8 ;
		fB[f_mem(7 ,x,y,pitch)] = f5 ;
		fB[f_mem(8 ,x,y,pitch)] = f6 ;
	}
	else{
		if(im == 3)//DirichletWest
		{
			if(y == 0){
				f2 = f4;
				f6 = f7;
			}
			else if(y == YDIM-1){
				f4 = f2;
				f7 = f6;
			}
			float u,v;//,rho;
			u = 0.0f;
	        v = UMAX;//0.0;
			//rho = u+(f0+f2+f4+2.0f*f3+2.0f*f6+2.0f*f7);
			//f1 = f3+4.0f*u/6.0f;
			f1 = f3+u*0.66666667f;
	        f5 = f7-0.5f*(f2-f4)+v*0.5f+u*0.166666667f;
	        f8 = f6+0.5f*(f2-f4)-v*0.5f+u*0.166666667f;			
		}

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,omega);

		fB[f_mem(0 ,x,y,pitch)] = f0 ;
		fB[f_mem(1 ,x,y,pitch)] = f1 ;
		fB[f_mem(2 ,x,y,pitch)] = f2 ;
		fB[f_mem(3 ,x,y,pitch)] = f3 ;
		fB[f_mem(4 ,x,y,pitch)] = f4 ;
		fB[f_mem(5 ,x,y,pitch)] = f5 ;
		fB[f_mem(6 ,x,y,pitch)] = f6 ;
		fB[f_mem(7 ,x,y,pitch)] = f7 ;
		fB[f_mem(8 ,x,y,pitch)] = f8 ;
	}
}


__global__ void initialize_single(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	
	float u,v,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;

	f[j+0 *pitch*YDIM]= x*0.000000001f;//0.4444444444f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM]= x*0.000000001f;//0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);               
	f[j+2 *pitch*YDIM]= x*0.000000001f;//0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM]= x*0.000000001f;//0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM]= x*0.000000001f;//0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM]= x*0.000000001f;//0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM]= x*0.000000001f;//0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM]= x*0.000000001f;//0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM]= x*0.000000001f;//0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
}


__global__ void initialize_LR(float *f, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	
	float u,v,rho,usqr;
	rho = 1.f;
	u = 0.0f;
	v = 0.0f;
	usqr = u*u+v*v;

	f[j+0 *pitch*YLRDIM]= 1000;//0.4444444444f*(rho-1.5f*usqr);
	f[j+1 *pitch*YLRDIM]= 1000;//0.1111111111f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);               
	f[j+2 *pitch*YLRDIM]= 1000;//0.1111111111f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YLRDIM]= 1000;//0.1111111111f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YLRDIM]= 1000;//0.1111111111f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YLRDIM]= 1000;//0.02777777778*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
	f[j+6 *pitch*YLRDIM]= 1000;//0.02777777778*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YLRDIM]= 1000;//0.02777777778*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YLRDIM]= 1000;//0.02777777778*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
}

__global__ void text(float* fin, float* fout, size_t pitch)
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float xcoord = LRX0+x*LRFACTOR;//+0.5f is because textures are stored in a voxel centered fashion. we need to change this to vertex centered.
	float ycoord = LRY0+y*LRFACTOR;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	float f0,f1,f2,f3,f4,f5,f6,f7,f8;
	int im = ImageFcn(xcoord,ycoord);
	f0 = tex2D(texRef_f0A ,xcoord+0.5f,ycoord+0.5f);
	f2 = tex2D(texRef_f2A ,xcoord+0.5f,ycoord+0.5f);
	f4 = tex2D(texRef_f4A ,xcoord+0.5f,ycoord+0.5f);
	f1 = tex2D(texRef_f1A ,xcoord+0.5f,ycoord+0.5f);
	f3 = tex2D(texRef_f3A ,xcoord+0.5f,ycoord+0.5f);
	f5 = tex2D(texRef_f5A ,xcoord+0.5f,ycoord+0.5f);
	f6 = tex2D(texRef_f6A ,xcoord+0.5f,ycoord+0.5f);
	f7 = tex2D(texRef_f7A ,xcoord+0.5f,ycoord+0.5f);
	f8 = tex2D(texRef_f8A ,xcoord+0.5f,ycoord+0.5f);

	fout[f_memLR(0 ,x,y,pitch)] = f0 ;
	fout[f_memLR(1 ,x,y,pitch)] = f1 ;
	fout[f_memLR(2 ,x,y,pitch)] = f2 ;
	fout[f_memLR(3 ,x,y,pitch)] = f3 ;
	fout[f_memLR(4 ,x,y,pitch)] = f4 ;
	fout[f_memLR(5 ,x,y,pitch)] = f5 ;
	fout[f_memLR(6 ,x,y,pitch)] = f6 ;
	fout[f_memLR(7 ,x,y,pitch)] = f7 ;
	fout[f_memLR(8 ,x,y,pitch)] = f8 ;

}

int main(int argc, char *argv[])
{

	//int *image_d, *image_h;

	ofstream output;
	ofstream output2;
	output.open ("LBM1_out.dat");
	output2.open ("Forces.dat");

	size_t memsize, memsize2;
	size_t pitch;
	size_t pitch2;
	int i, n, nBlocks, nBlocks2, n2;
	float omega, CharLength, omega2;

	CharLength = OBSTR*2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);
	omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));
	float SF_cf = omega*(1.0f-omega2)/((1.0f-omega)*omega2/LRFACTOR);
	float SF_fc = 1.f/SF_cf;

	cout<<"omega: "<<omega<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<endl;
	cout<<"LRblocksize: "<<BLOCKSIZELRX<<"x"<<BLOCKSIZELRY<<endl;
	cout<<"LRgrid: "<<XLRDIM<<"x"<<YLRDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

	nBlocks = (XDIM/BLOCKSIZEX+XDIM%BLOCKSIZEX)*(YDIM/BLOCKSIZEY+YDIM%BLOCKSIZEY);
	nBlocks2 = (XLRDIM/BLOCKSIZELRX+XLRDIM%BLOCKSIZELRX)*(YLRDIM/BLOCKSIZELRY+YLRDIM%BLOCKSIZELRY);
	int B = BLOCKSIZEX*BLOCKSIZELRY;
	int B2 = BLOCKSIZELRX*BLOCKSIZELRY;
	n = nBlocks*B;
	n2 = nBlocks2*B2;
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
    dim3 grid(XDIM/BLOCKSIZEX,YDIM/BLOCKSIZEY);

    dim3 threads2(BLOCKSIZELRX, BLOCKSIZELRY);
    dim3 grid2(XLRDIM/BLOCKSIZELRX,YLRDIM/BLOCKSIZELRY);

	memsize = n*sizeof(float);
	//memsize_int = n*sizeof(int);
	memsize2 = n2*sizeof(float);

	//hipExtent extent = make_hipExtent(XDIM*sizeof(float),YDIM,ZDIM);

	//image_h = (int *)malloc(memsize_int);

	float *fA_h,*fA_d,*fB_d,*fC_h,*fC_d,*fD_d;
	float *FX_h,*FY_h,*FX_d,*FY_d;
	fA_h = (float *)malloc(memsize*9);
	fC_h = (float *)malloc(memsize2*9);
	FX_h = (float *)malloc(TMAX*sizeof(float));
	FY_h = (float *)malloc(TMAX*sizeof(float));
	hipMallocPitch((void **) &fA_d, &pitch, XDIM*sizeof(float), YDIM*9);
	hipMallocPitch((void **) &fB_d, &pitch, XDIM*sizeof(float), YDIM*9);

	if(METHOD == "LR"){
	hipMallocPitch((void **) &fC_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*9);
	hipMallocPitch((void **) &fD_d, &pitch2, XLRDIM*sizeof(float), YLRDIM*9);
	}

	hipMalloc((void **) &FX_d, TMAX*sizeof(float));
	hipMalloc((void **) &FY_d, TMAX*sizeof(float));

	//hipMalloc((void **) &image_d, memsize_int);

	cout<<pitch<<", "<<pitch2<<endl;
	
	size_t pitch_elements = pitch/sizeof(float);
	size_t pitch_elements2 = pitch2/sizeof(float);

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	for (i = 0; i < n*9; i++)
		fA_h[i] = i;
	for (i = 0; i < n2*9; i++)
		fC_h[i] = 0;
	for (i = 0; i < TMAX; i++){
		FX_h[i] = 0;
		FY_h[i] = 0;
	}
	hipMemcpy(FX_d, FX_h, TMAX*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(FY_d, FY_h, TMAX*sizeof(float), hipMemcpyHostToDevice);

//	for (i = 0; i < n; i++)
//	{
//		int x = i%XDIM;
//		int y = (i/XDIM)%YDIM;
//		int z = (i/XDIM)/YDIM;
////		image_h[i] = 0;
////		if(x < 1) image_h[i] = 1;//DirichletWest
////		if(x > XDIM-2) image_h[i] = 1;//BB
////		if(y < 1) image_h[i] = 1;//BB
////		if(y > YDIM-2) image_h[i] = 1;//BB
////		if(z < 1) image_h[i] = 1;//DirichletWest
////		if(z > ZDIM-2) image_h[i] = 1;//BB
//	}
	//hipMemcpy(image_d, image_h, memsize_int, hipMemcpyHostToDevice);
	if(true)//texture settings
	{
	texRef_f0B.normalized = false;
	texRef_f1B.normalized = false;
	texRef_f2B.normalized = false;
	texRef_f3B.normalized = false;
	texRef_f4B.normalized = false;
	texRef_f5B.normalized = false;
	texRef_f6B.normalized = false;
	texRef_f7B.normalized = false;
	texRef_f8B.normalized = false;

	texRef_f0B.filterMode = hipFilterModeLinear;
	texRef_f1B.filterMode = hipFilterModeLinear;
	texRef_f2B.filterMode = hipFilterModeLinear;
	texRef_f3B.filterMode = hipFilterModeLinear;
	texRef_f4B.filterMode = hipFilterModeLinear;
	texRef_f5B.filterMode = hipFilterModeLinear;
	texRef_f6B.filterMode = hipFilterModeLinear;
	texRef_f7B.filterMode = hipFilterModeLinear;
	texRef_f8B.filterMode = hipFilterModeLinear;
	texRef_f0A.normalized = false;
	texRef_f1A.normalized = false;
	texRef_f2A.normalized = false;
	texRef_f3A.normalized = false;
	texRef_f4A.normalized = false;
	texRef_f5A.normalized = false;
	texRef_f6A.normalized = false;
	texRef_f7A.normalized = false;
	texRef_f8A.normalized = false;
	texRef_f0A.filterMode = hipFilterModeLinear;
	texRef_f1A.filterMode = hipFilterModeLinear;
	texRef_f2A.filterMode = hipFilterModeLinear;
	texRef_f3A.filterMode = hipFilterModeLinear;
	texRef_f4A.filterMode = hipFilterModeLinear;
	texRef_f5A.filterMode = hipFilterModeLinear;
	texRef_f6A.filterMode = hipFilterModeLinear;
	texRef_f7A.filterMode = hipFilterModeLinear;
	texRef_f8A.filterMode = hipFilterModeLinear;
	
	texRef_f0C.normalized = false;
	texRef_f1C.normalized = false;
	texRef_f2C.normalized = false;
	texRef_f3C.normalized = false;
	texRef_f4C.normalized = false;
	texRef_f5C.normalized = false;
	texRef_f6C.normalized = false;
	texRef_f7C.normalized = false;
	texRef_f8C.normalized = false;

	texRef_f0C.filterMode = hipFilterModeLinear;
	texRef_f1C.filterMode = hipFilterModeLinear;
	texRef_f2C.filterMode = hipFilterModeLinear;
	texRef_f3C.filterMode = hipFilterModeLinear;
	texRef_f4C.filterMode = hipFilterModeLinear;
	texRef_f5C.filterMode = hipFilterModeLinear;
	texRef_f6C.filterMode = hipFilterModeLinear;
	texRef_f7C.filterMode = hipFilterModeLinear;
	texRef_f8C.filterMode = hipFilterModeLinear;

	texRef_f0D.normalized = false;
	texRef_f1D.normalized = false;
	texRef_f2D.normalized = false;
	texRef_f3D.normalized = false;
	texRef_f4D.normalized = false;
	texRef_f5D.normalized = false;
	texRef_f6D.normalized = false;
	texRef_f7D.normalized = false;
	texRef_f8D.normalized = false;

	texRef_f0D.filterMode = hipFilterModeLinear;
	texRef_f1D.filterMode = hipFilterModeLinear;
	texRef_f2D.filterMode = hipFilterModeLinear;
	texRef_f3D.filterMode = hipFilterModeLinear;
	texRef_f4D.filterMode = hipFilterModeLinear;
	texRef_f5D.filterMode = hipFilterModeLinear;
	texRef_f6D.filterMode = hipFilterModeLinear;
	texRef_f7D.filterMode = hipFilterModeLinear;
	texRef_f8D.filterMode = hipFilterModeLinear;

	}
	
	hipMemcpy2D(fA_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*9,hipMemcpyHostToDevice);
	hipMemcpy2D(fB_d,pitch ,fA_h,XDIM*sizeof(float),XDIM*sizeof(float),YDIM*9,hipMemcpyHostToDevice);
	if(METHOD == "LR"){
	hipMemcpy2D(fC_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyHostToDevice);
	hipMemcpy2D(fD_d,pitch2,fC_h,XLRDIM*sizeof(float),XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyHostToDevice);
	}
	for (i = 0; i < n*9; i++)
		fA_h[i] = 0;
	for (i = 0; i < n2*9; i++)
		fC_h[i] = 0;


	if(true)//bind texture
	{
	hipBindTexture2D(0,&texRef_f0A, fA_d                       ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f1A, fA_d+pitch_elements*YDIM   ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f2A, fA_d+pitch_elements*YDIM*2 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f3A, fA_d+pitch_elements*YDIM*3 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f4A, fA_d+pitch_elements*YDIM*4 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f5A, fA_d+pitch_elements*YDIM*5 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f6A, fA_d+pitch_elements*YDIM*6 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f7A, fA_d+pitch_elements*YDIM*7 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f8A, fA_d+pitch_elements*YDIM*8 ,&desc,XDIM,YDIM,pitch);

	hipBindTexture2D(0,&texRef_f0B, fB_d                       ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f1B, fB_d+pitch_elements*YDIM   ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f2B, fB_d+pitch_elements*YDIM*2 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f3B, fB_d+pitch_elements*YDIM*3 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f4B, fB_d+pitch_elements*YDIM*4 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f5B, fB_d+pitch_elements*YDIM*5 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f6B, fB_d+pitch_elements*YDIM*6 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f7B, fB_d+pitch_elements*YDIM*7 ,&desc,XDIM,YDIM,pitch);
	hipBindTexture2D(0,&texRef_f8B, fB_d+pitch_elements*YDIM*8 ,&desc,XDIM,YDIM,pitch);

	hipBindTexture2D(0,&texRef_f0C, fC_d                          ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1C, fC_d+pitch_elements2*YLRDIM   ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2C, fC_d+pitch_elements2*YLRDIM*2 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3C, fC_d+pitch_elements2*YLRDIM*3 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4C, fC_d+pitch_elements2*YLRDIM*4 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5C, fC_d+pitch_elements2*YLRDIM*5 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6C, fC_d+pitch_elements2*YLRDIM*6 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7C, fC_d+pitch_elements2*YLRDIM*7 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8C, fC_d+pitch_elements2*YLRDIM*8 ,&desc,XLRDIM,YLRDIM,pitch2);

	hipBindTexture2D(0,&texRef_f0D, fD_d                          ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f1D, fD_d+pitch_elements2*YLRDIM   ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f2D, fD_d+pitch_elements2*YLRDIM*2 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f3D, fD_d+pitch_elements2*YLRDIM*3 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f4D, fD_d+pitch_elements2*YLRDIM*4 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f5D, fD_d+pitch_elements2*YLRDIM*5 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f6D, fD_d+pitch_elements2*YLRDIM*6 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f7D, fD_d+pitch_elements2*YLRDIM*7 ,&desc,XLRDIM,YLRDIM,pitch2);
	hipBindTexture2D(0,&texRef_f8D, fD_d+pitch_elements2*YLRDIM*8 ,&desc,XLRDIM,YLRDIM,pitch2);
	}


	initialize_single<<<grid, threads>>>(fA_d,pitch_elements);
	initialize_single<<<grid, threads>>>(fB_d,pitch_elements);

	if(METHOD == "LR"){
	initialize_LR<<<grid2, threads2>>>(fC_d,pitch_elements2);
	}

//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(mrt_d_single),hipFuncCachePreferL1);

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);
//	for(int t = 0; t<TMAX; t=t+2){
//		if(METHOD == "LR"){
//		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//		
//		LR_d_ABCD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
//		LR_d_ABDC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
//		
//		ExtractFromC_d<<<grid, threads>>>(fB_d,pitch_elements,SF_fc);
//
//		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//
//		LR_d_BACD<<<grid2, threads2>>>(fC_d,fD_d,omega2,pitch_elements2);
//		LR_d_BADC<<<grid2, threads2>>>(fD_d,fC_d,omega2,pitch_elements2,SF_cf);
//		
//		ExtractFromC_d<<<grid, threads>>>(fA_d,pitch_elements,SF_fc);
//		}
//
//		else if(METHOD == "SINGLE"){
//		mrt_d_single<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
//		mrt_d_single<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
//		}
//
//		else if(METHOD == "SINGLEF"){
//		mrt_d_single_force<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements,FX_d,FY_d,t);
//		mrt_d_single_force<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements,FX_d,FY_d,t+1);
//		}
//
////		else if(METHOD == "HYB"){
////		mrt_d_hybAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
////		mrt_d_hybBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
////		}
////
////		else if(METHOD == "TEXT"){
////		mrt_d_textAB<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
////		mrt_d_textBA<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
////		}
////
////		else if(METHOD == "SHARED"){
////		mrt_d_shared<<<grid, threads>>>(fA_d,fB_d,omega,pitch_elements);
////		mrt_d_shared<<<grid, threads>>>(fB_d,fA_d,omega,pitch_elements);
////		}
//
////		simple_copy<<<grid, threads>>>(fA_d,fB_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);
////		simple_copy<<<grid, threads>>>(fB_d,fA_d,image_d,omega,UMAX,XDIM,YDIM,ZDIM,pitch_elements);
//
//		if(t%1000 == 0 && t>0) cout<<"finished "<<t<<" timesteps\n";
//	}

	text<<<grid2, threads2>>>(fA_d,fC_d,pitch_elements2);
	hipDeviceSynchronize();


	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	int Nodes;
	if(METHOD == "LR"){
		Nodes = (XDIM*YDIM+XLRDIM*YLRDIM*2);
	}
	else{
		Nodes = XDIM*YDIM;
	}
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
//			<<double((XDIM*YDIM*ZDIM)*double(TMAX/1000000.f))/restime<<"MLUPS)"<<endl;
	cout<<XDIM<<","<<YDIM<<","<<TMAX<<","<<restime<<endl;

	if(true){
	hipUnbindTexture(texRef_f0A);
	hipUnbindTexture(texRef_f1A);
	hipUnbindTexture(texRef_f2A);
	hipUnbindTexture(texRef_f3A);
	hipUnbindTexture(texRef_f4A);
	hipUnbindTexture(texRef_f5A);
	hipUnbindTexture(texRef_f6A);
	hipUnbindTexture(texRef_f7A);
	hipUnbindTexture(texRef_f8A);

	hipUnbindTexture(texRef_f0B);
	hipUnbindTexture(texRef_f1B);
	hipUnbindTexture(texRef_f2B);
	hipUnbindTexture(texRef_f3B);
	hipUnbindTexture(texRef_f4B);
	hipUnbindTexture(texRef_f5B);
	hipUnbindTexture(texRef_f6B);
	hipUnbindTexture(texRef_f7B);
	hipUnbindTexture(texRef_f8B);

	hipUnbindTexture(texRef_f0C);
	hipUnbindTexture(texRef_f1C);
	hipUnbindTexture(texRef_f2C);
	hipUnbindTexture(texRef_f3C);
	hipUnbindTexture(texRef_f4C);
	hipUnbindTexture(texRef_f5C);
	hipUnbindTexture(texRef_f6C);
	hipUnbindTexture(texRef_f7C);
	hipUnbindTexture(texRef_f8C);

	hipUnbindTexture(texRef_f0D);
	hipUnbindTexture(texRef_f1D);
	hipUnbindTexture(texRef_f2D);
	hipUnbindTexture(texRef_f3D);
	hipUnbindTexture(texRef_f4D);
	hipUnbindTexture(texRef_f5D);
	hipUnbindTexture(texRef_f6D);
	hipUnbindTexture(texRef_f7D);
	hipUnbindTexture(texRef_f8D);
	}

	hipMemcpy2D(fA_h,XDIM*sizeof(float),fA_d,pitch,XDIM*sizeof(float),YDIM*9,hipMemcpyDeviceToHost);
	if(METHOD == "LR"){
	hipMemcpy2D(fC_h,XLRDIM*sizeof(float),fC_d,pitch2,XLRDIM*sizeof(float),YLRDIM*9,hipMemcpyDeviceToHost);
	}	
	hipMemcpy(FX_h, FX_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(FY_h, FY_d, TMAX*sizeof(float), hipMemcpyDeviceToHost);

	output<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<"\n";
	
	int row = 0;
	int col = 0;
	i = 0;
	float rho, u, v;
	int j;

	for(row = 0; row<YDIM; row++){
		for(col = 0; col<XDIM; col++){
			i = row*XDIM+col;
			rho = fA_h[i];
			for(j = 1; j<9; j++)
				rho+=fA_h[i+XDIM*YDIM*j];
			u = fA_h[i+XDIM*YDIM*1]-fA_h[i+XDIM*YDIM*3]+fA_h[i+XDIM*YDIM*5]-fA_h[i+XDIM*YDIM*6]-
				fA_h[i+XDIM*YDIM*7]+fA_h[i+XDIM*YDIM*8];
			v = fA_h[i+XDIM*YDIM*2]-fA_h[i+XDIM*YDIM*4]+fA_h[i+XDIM*YDIM*5]+fA_h[i+XDIM*YDIM*6]-fA_h[i+XDIM*YDIM*7]-fA_h[i+XDIM*YDIM*8];
			output<<col<<", "<<row<<", "<<fA_h[i+XDIM*YDIM]<<","<<v<<","<<rho<<endl;
		}
	}

	if(METHOD == "LR"){
	output<<endl<<"VARIABLES = \"X\",\"Y\",\"u\",\"v\",\"rho\"\n";
	output<<"ZONE F=POINT, I="<<XLRDIM-0<<", J="<<YLRDIM-0<<"\n";
	
	for(row = 0; row<YLRDIM-0; row++){
		for(col = 0; col<XLRDIM-0; col++){
			i = row*XLRDIM+col;
			rho = fA_h[i];
			for(j = 1; j<9; j++)
				rho+=fC_h[i+XLRDIM*YLRDIM*j];
			u = fC_h[i+XLRDIM*YLRDIM*1]-fC_h[i+XLRDIM*YLRDIM*3]+fC_h[i+XLRDIM*YLRDIM*5]-fC_h[i+XLRDIM*YLRDIM*6]-
				fC_h[i+XLRDIM*YLRDIM*7]+fC_h[i+XLRDIM*YLRDIM*8];
			v = fC_h[i+XLRDIM*YLRDIM*2]-fC_h[i+XLRDIM*YLRDIM*4]+fC_h[i+XLRDIM*YLRDIM*5]+fC_h[i+XLRDIM*YLRDIM*6]-fC_h[i+XLRDIM*YLRDIM*7]-fC_h[i+XLRDIM*YLRDIM*8];
			output<<LRX0+LRFACTOR*col<<", "<<LRY0+LRFACTOR*row<<", "<<fC_h[i+XLRDIM*YLRDIM]<<","<<v<<","<<rho<<endl;
		}
	}
	}

	output.close();

	for(int t = STARTF-1; t<TMAX; t++){
		output2<<t<<", "<<FX_h[t]/(0.5f*UMAX*UMAX*OBSTR*2.0f)<<", "<<FY_h[t]/(0.5f*UMAX*UMAX*OBSTR*2.0f)<<endl;
	}
	output2.close();


	//hipFree(image_d);


	hipFree(fA_d);
	hipFree(fB_d);
	hipFree(fC_d);
	hipFree(fD_d);
	hipFree(FX_d);
	hipFree(FY_d);
	return(0);

}
